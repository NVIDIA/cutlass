/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example leveraging collective operation builders.

    This example showcases the use of CUTLASS's CollectiveBuilder to easily construct performant kernels
    targeting the NVIDIA Hopper architecture.

    Background and motivation
    -------------------------
    CUTLASS kernels are highly parameterizable via template parameters. To ease the selection of template
    parameters, CUTLASS 2 leveraged DefaultGemmConfigurations. Given a small set of parameters, such as
    the data types of operands and the compute capability of the GPU, DefaultGemmConfigurations defined sensible
    defaults for the many other parameters to the kernel (e.g., warp shape, stage count).

    However, DefaultGemmConfigurations leave multiple opportunities for improvement, which are addressed
    in CUTLASS 3:
      (1) DefaultGemmConfigurations do not allow one to use a more-performant set of parameters without
          specifying every parameter. For example, the DefaultGemmConfigurations for GEMMs targeting
          Ampere specify that three pipeline stages should be used regardless of the sizes of operands.
          If one wished to increase this value, one would also need to specify all other template parameters.
          This leaves a gap between a high-level ease-of-use interface and a lower-level detailed interface.
      (2) A new DefaultGemmConfiguration was required for each combination of operand types, GPU architecture,
          and operation type (e.g., Tensor Core or SIMT). This led to increased code size to cover each unique
          configuration and a lack of extensibility from one DefaultGemmConfiguration to another.

    Alongside these opportunities for improvement, the Hopper architecture offers new features that increase
    the number of valid configurations of a kernel. In addition to the many template parameters already available
    in CUTLASS 2 kernels, CUTLASS 3 kernels targeting Hopper also have various scheduling modes to select from that control:
      (1) how data is to be loaded (e.g., using the Hopper TMA feature or Ampere cp.async)
      (2) how work is to be divided among warps in a thread block (e.g., whether to use "warp specialization")
      (3) whether persistent thread blocks should be used
    This increased configuration space further motivates rethinking DefaultGemmConfigurations.

    Introduction to the CollectiveBuilder
    -------------------------------------
    CUTLASS 3 introduces the CollectiveBuilder to further ease the process of selecting template parameters
    for kernels targeting Hopper. Similar to the DefaultGemmConfigurations used in CUTLASS 2, the CollectiveBuilder
    takes in a small set of template parameters (e.g., the data types of operands A and B). It then automatically
    determines the data loading strategy to use depending on whether the Hopper TMA feature can be used with the provided
    parameters. If one does not indicate a particular scheduling policy or stage count to use (by using `Auto` template
    parameters), the CollectiveBuilder will also automatically select these.

    Unlike DefaultGemmConfigurations a partial specialization of the CollectiveBuilder is not needed for many
    configurations of operand types. Instead the CollectiveBuilder "builds" a configuration based on generic
    properties of the specified operands, layouts, and other parameters. For example, when the stage count
    is set to `Auto`, the CollectiveBuilder may automatically calculate the maximum number of stages that
    will fit in shared memory given the types of operands and the thread block shape, rather than simply using
    a single default value.

    CUTLASS 3.x provides builders for both collective mainloops and epilogues. The particular implementation of
    the collective is specified via the schedule tags that corresond to the underlying collective's
    dispatch policy. `gemm::collective::KernelScheduleAuto` and `epilogue::collective::EpilogueScheduleAuto`
    are special cases of these schedules that allow the builder to also decide the dispatch policy for you,
    therefore letting the builder pick the collective specialization.

    CUTLASS builders make an attempt to pick the best schedule when `Auto` is provided such that the
    assembled collectives have the best performance, but this is not a guarantee. A user relying on `Auto`
    may get a free performance upgrade with newer CUTLASS releases in case we can provide more optimized
    implementations that the builder can transparently assemble for `Auto`. But a user should not rely on 
    `Auto` if they require a specific scheduling policy and/or stage count to be used.

    If a user decides to let the builders pick the collective specialization via `Auto` schedules,
    they must be used for both mainloop and epilogue alike to ensure compatibility between the
    chosen collectives. Additionally, if a user chooses to opt in to a specific schedule, non-`Auto`
    schedules must be used for both mainloop and epilogue builder schedules, and these schedules
    must be compatible.

    One does not need to use the CollectiveBuilder to declare CUTLASS 3 kernels; one can still provide
    every template parameter to the `gemm::collective::CollectiveMma`. Specifying every template parameter
    in this manner remains the primary API for using CUTLASS 3 kernels. `CollectiveBuilder`s are
    simply meant to be a convenience interface.

    Details of this example
    -----------------------
    This example walks through the use of the CollectiveBuilder with various schedules and stage counts specified.
    This example also illustrates how CUTLASS 3 GEMMs targeting Hopper automatically support batched GEMMs by simply
    extending the problem size with an additional tensor rank.

    CUTLASS 3.2 provides initial support for epilogue visitor trees (EVT) for the TMA warp-specialized collective.
    EVTs allow users to define their own customized epilogue fusion patterns without having to write a new
    collective epilogue. This is done by representing the fusion as a compute graph, where each node is one of a
    fundamental set of load, store, or compute operations. These operations are either elementwise for tensor
    inputs/outputs, broadcasts for vector/scalar inputs, or reductions for vector/scalar outputs.
    This example shows how users can define their own custom EVT and use it with the CollectiveBuilder.

    Example usage:
      $ ./examples/49_hopper_with_collective_builder/49_collective_builder \
            --m=2048 --n=2048 --k=2048 --l=2
*/

#include <iostream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;

  int m, n, k, l;
  float alpha, beta;

  Options():
    help(false),
    error(false),
    m(2048), n(2048), k(2048), l(1),
    alpha(1.f), beta(0.f)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 2048);
    cmd.get_cmd_line_argument("n", n, 2048);
    cmd.get_cmd_line_argument("k", k, 2048);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "49_hopper_with_collective_builder\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  performant kernels targeting NVIDIA's Hopper architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

// Wrapper to construct, run, and verify a GEMM. This example showcases CUTLASS's collective
// operation builders by specializing the GEMM only on the kernel schedule it will use and the
// number of pipeline stages.
//
// One can use a special `Auto` type that tells the CollectiveBuilder
// to select an appropriate value on its own. The CollectiveBuilder will attempt to select
// configurations that will result in the most-performant kernel, but this is not a guarantee.
//
// If relying on 'Auto' schedules, all builders must use the 'Auto' schedule to ensure compatiblity.
// For example, if `KernelScheduleAuto` is used for the mainloop builder, `EpilogueScheduleAuto` must
// be used for the epilogue builder.
//
// Furthermore, if an override schedule is selected, both epilogue and mainloop schedules must
// be specifically opt into a compatible selection.
//
// Behavior of the CollectiveBuilder with `Auto` types is subject to change in future releases
// -- do not rely on `Auto` if you require a specific scheduling policy.
template <
  // Type of kernel schedule to generate
  class MainloopScheduleType = cutlass::gemm::collective::KernelScheduleAuto,
  // Type of epilogue schedule to generate
  class EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto,
  // Number of pipeline stages to use
  class StageCountType = cutlass::gemm::collective::StageCountAuto,
  // Type of tile scheduler to use
  class TileSchedulerType = cutlass::gemm::PersistentScheduler,
  // Do we use custom epilogue visitor tree (EVT) fusion
  bool UseCustomEVT = false
>
struct ExampleRunner {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::ColumnMajor;

  using ElementA = cutlass::half_t;
  using ElementB = cutlass::half_t;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::half_t;
  using ElementAccumulator = float;
  using ElementCompute = float;
  using ElementScalar = float;

  // 16B alignment lets us use TMA
  static constexpr int AlignmentA = 16 / sizeof(ElementA);
  static constexpr int AlignmentB = 16 / sizeof(ElementB);
  static constexpr int AlignmentC = 16 / sizeof(ElementC);
  static constexpr int AlignmentD = 16 / sizeof(ElementD);

  static_assert(not UseCustomEVT ||
    (cute::is_same_v<EpilogueScheduleType, cutlass::epilogue::TmaWarpSpecialized> ||
     cute::is_same_v<EpilogueScheduleType, cutlass::epilogue::TmaWarpSpecializedCooperative>),
    "Epilogue visitor trees are currently only supported by the TMA warp-specialized epilogue");
  static constexpr auto RoundStyle = cutlass::FloatRoundStyle::round_to_nearest;

  // EVTs can be constructed by composing the fundamental load/store/compute visitor operations defined in include/cutlass/epilogue/fusion
  // For more complex examples of EVT construction please refer to include/cutlass/epilogue/fusion/sm90_callbacks_tma_warpspecialized.hpp
  using CustomEVT =  // alpha * acc + beta * C
    cutlass::epilogue::fusion::Sm90EVT<cutlass::epilogue::fusion::Sm90Compute<cutlass::multiply_add, ElementD, ElementCompute, RoundStyle>, // beta * C + (alpha * acc)
      cutlass::epilogue::fusion::Sm90ScalarBroadcast<ElementScalar>, // beta
      cutlass::epilogue::fusion::Sm90SrcFetch, // C
      cutlass::epilogue::fusion::Sm90EVT<cutlass::epilogue::fusion::Sm90Compute<cutlass::multiplies, ElementCompute, ElementCompute, RoundStyle>, // alpha * acc
        cutlass::epilogue::fusion::Sm90ScalarBroadcast<ElementScalar>, // alpha
        cutlass::epilogue::fusion::Sm90AccFetch // acc
      >
    >;

  // A predefined set of fusion operations (implemented with EVT) are supported by the TMA warp-specialized epilogue.
  // Users can select one of these operations by passing one of the tags defined in include/cutlass/epilogue/fusion/operations.hpp
  // to the CollectiveBuilder. This frees the user from having to compute additional parameters such as stage counts and copy atoms/layouts.
  // These tags also provide additional metadata that can be queried at compile time.
  using DefaultOperation = cutlass::epilogue::fusion::LinearCombination<ElementD, ElementCompute, ElementScalar, RoundStyle>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      Shape<_128,_128,_64>, Shape<_1,_1,_1>,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      EpilogueScheduleType,
      cute::conditional_t<UseCustomEVT, CustomEVT, DefaultOperation>
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, AlignmentA,
      ElementB, LayoutB, AlignmentB,
      ElementAccumulator,
      Shape<_128,_128,_64>, Shape<_2,_1,_1>,
      cute::conditional_t<cute::is_same_v<StageCountType, cutlass::gemm::collective::StageCountAuto>,
          cutlass::gemm::collective::StageCountAutoCarveout<(int)sizeof(typename CollectiveEpilogue::SharedStorage)>,
          StageCountType>,
      MainloopScheduleType
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue,
      TileSchedulerType
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutTagA = cutlass::gemm::detail::StrideToLayoutTagA_t<StrideA>;
  using LayoutTagB = cutlass::gemm::detail::StrideToLayoutTagB_t<StrideB>;
  using LayoutTagC = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideC>;
  using LayoutTagD = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideD>;

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_D;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_ref_D;

  //
  // Methods
  //

  bool verify(const ProblemShapeType& problem_size, float alpha, float beta) {
    auto [M, N, K, L] = problem_size;

    cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({M, N}));

    cutlass::reference::device::GemmComplex(
          {M, N, K},
          ElementScalar(alpha),
          ref_A,
          cutlass::ComplexTransform::kNone,
          ref_B,
          cutlass::ComplexTransform::kNone,
          ElementScalar(beta),
          ref_C,
          ref_D,
          ElementAccumulator(0),
          L,     // batch_count
          M * K, // batch_stride_A
          K * N, // batch_stride_B
          M * N, // batch_stride_C
          M * N  // batch_stride_D
        );

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

    return passed;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(M * K * L);
    block_B.reset(K * N * L);
    block_C.reset(M * N * L);
    block_D.reset(M * N * L);
    block_ref_D.reset(M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  bool run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.m, options.n, options.k, options.l};

    initialize(problem_size);

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      problem_size,
      {block_A.get(), stride_A, block_B.get(), stride_B},
      {{}, // epilogue.thread
       block_C.get(), stride_C, block_D.get(), stride_D},
      hw_info
    };

    // Custom EVT fusions will have nested unnamed args, the structure of which
    // can be deduced from the type definition of the EVT.
    // Each node's arguments has the recursive structure of
    // {first_child_args, ..., last_child_args, op_args},
    // For more complex examples of EVT initialization please refer to
    // include/cutlass/epilogue/fusion/sm90_callbacks_tma_warpspecialized.hpp
    if constexpr (UseCustomEVT) {
      arguments.epilogue.thread =
        {    // ternary op : beta * C + (alpha * acc)
          {{options.beta}}, // leaf op+args : beta
          {},               // leaf op+args : C
          {                 // binary op : alpha * acc
            {{options.alpha}}, // leaf op+args : alpha
            {},                // leaf op+args : acc
            {}              // binary args : multiplies
          },                // end binary op
          {} // ternary args : multiply_add
        };   // end ternary op
    }
    // Pre-defined fusions will have flat, named args for user-friendlyness
    else {
      arguments.epilogue.thread.alpha = options.alpha;
      arguments.epilogue.thread.beta = options.beta;
    }

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    // Run the GEMM
    status = gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Verify that the result is correct
    bool passed = verify(problem_size, options.alpha, options.beta);
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
    }

    return passed;
  }

};

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, bool passed) {
  std::cout << description << ": " << (passed ? "Passed" : "Failed") << std::endl;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major < 9) {
    std::cout
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater) and CUDA 12.0 or greater.\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  bool passed;

  // This first example constructs a GEMM using the default schedule and stage count provided by
  // the CollectiveBuilder. The scheduling policy that is expected to be most performant will be
  // selected and the maximum number of stages that can fit in shared memory will be selected.
  //
  // This example is equivalent to declaring
  //    ExampleRunner<
  //        cutlass::gemm::collective::KernelScheduleAuto,
  //        cutlass::epilogue::collective::EpilogueScheduleAuto,
  //        cutlass::gemm::collective::StageCountAuto>
  // Each of the `Auto` types indicate that the CollectiveBuilder should determine the scheduling policy and
  // stage count. Note that the behavior of the CollectiveBuilder with `Auto` parameters is subject to change
  // -- do not rely on `Auto` if you require a specific scheduling policy.
  // If you opt in to a non-'Auto' schedule, make sure all collectives are built using specific, compatible schedules.
  ExampleRunner<> auto_schedule_auto_stage_runner;
  passed = auto_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Automatically-selected schedule and stage count", passed);

  // One can override the stage count used in the GEMM by replacing cutlass::gemm::collective::StageCountAuto
  // with the number of stages to use (5 in this case).
  ExampleRunner<
    cutlass::gemm::collective::KernelScheduleAuto,
    cutlass::epilogue::collective::EpilogueScheduleAuto,
    _5> auto_schedule_5_stage_runner;

  passed = auto_schedule_5_stage_runner.run(options, hw_info);
  print_result("Automatically-selected schedule with 5 stages", passed);

  // One can also override the scheduling policy to use. In this case, use the KernelTma scheduling
  // policy, which specifies that the Hopper TMA feature should be used, and we also use an epilogue
  // that does not use any shared memory.
  ExampleRunner<cutlass::gemm::KernelTma, cutlass::epilogue::NoSmemWarpSpecialized> tma_schedule_auto_stage_runner;
  passed = tma_schedule_auto_stage_runner.run(options, hw_info);
  print_result("TMA schedule with automatically-selected stage count", passed);

  // Here, we override the scheduling policy to use Hopper's TMA feature alongside the warp-specialized
  // scheduling policy, and an epilogue that does not use any shared memory.
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized, cutlass::epilogue::NoSmemWarpSpecialized> ws_schedule_auto_stage_runner;
  passed = ws_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Warp-specialized TMA schedule with automatically-selected stage count", passed);

  // Here, we override the scheduling policy to use Hopper's TMA feature, alongside the warp-specialized
  // scheduling policy, TMA-based epilogue, leveraging persistent thread blocks.
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecializedPingpong,
    cutlass::epilogue::TmaWarpSpecialized> ws_pingpong_schedule_auto_stage_runner;
  passed = ws_pingpong_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Ping-pong warp-specialized TMA schedule with automatically-selected stage count", passed);

  // Here, we override the scheduling policy to use stream-K problem decomposition atop the cooperative
  // warp-specialized scheduling policy. This kernel continues to leverage persistent thread blocks
  // as well aso TMA in both the mainloop and epilogue.
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecializedCooperative,
    cutlass::epilogue::TmaWarpSpecializedCooperative,
    cutlass::gemm::collective::StageCountAuto,
    cutlass::gemm::StreamKScheduler> ws_cooperative_stream_k_schedule_auto_stage_runner;
  passed = ws_cooperative_stream_k_schedule_auto_stage_runner.run(options, hw_info);
  print_result("Cooperative warp-specialized TMA schedule using stream-K with automatically-selected stage count", passed);

  // Here, we override the fusion operation to use a customized EVT fusion, in addition to the previous schedule overrides
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecializedCooperative,
    cutlass::epilogue::TmaWarpSpecializedCooperative,
    cutlass::gemm::collective::StageCountAuto,
    cutlass::gemm::PersistentScheduler,
    true> ws_cooperative_schedule_auto_stage_custom_evt_runner;
  passed = ws_cooperative_schedule_auto_stage_custom_evt_runner.run(options, hw_info);
  print_result("Cooperative warp-specialized TMA schedule using custom epilogue visitor tree with automatically-selected stage count", passed);

#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
