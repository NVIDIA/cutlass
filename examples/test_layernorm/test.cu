#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>
#include "utils.cuh"
#include <hip/hip_fp16.h>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/functional.h"
#include "customized_device_gemm.h"


#define DEBUG 1


hipError_t CutlassHgemmNN(
  int M,
  int N,
  int K,
  half alpha,
  half const *A,
  int lda,
  half const *B,
  int ldb,
  half beta,
  half *C,
  int ldc) {

  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  using MMAOp = cutlass::arch::OpClassTensorOp;
  using SmArch = cutlass::arch::Sm80;
  using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
  using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32
  using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 16>;  // <- MMA Op tile M = 16, N = 8, K = 8
  using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,                                     // <- data type of output matrix
      128 / cutlass::sizeof_bits<cutlass::half_t>::value,  // <- the number of elements per vectorized
                                                        // memory access. For a byte, it's 16
                                                        // elements. This becomes the vector width of
                                                        // math instructions in the epilogue too
      cutlass::half_t,                                            // <- data type of accumulator
      cutlass::half_t>;                                           // <- data type for alpha/beta in linear combination function
  constexpr int NumStages = 4;
  using CutlassGemm = cutlass::gemm::device::Gemm<cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  MMAOp,
                                                  SmArch,
                                                  ShapeMMAThreadBlock,
                                                  ShapeMMAWarp,
                                                  ShapeMMAOp,
                                                  EpilogueOp,
                                                  SwizzleThreadBlock,
                                                  NumStages>;

  CutlassGemm gemm_operator;
  cutlass::gemm::GemmCoord problem_size(M, N, K);
  typename CutlassGemm::Arguments args{problem_size,  // Gemm Problem dimensions
                              {(cutlass::half_t*)A, lda},    // Tensor-ref for source matrix A
                              {(cutlass::half_t*)B, ldb},    // Tensor-ref for source matrix B
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for source matrix C
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {(cutlass::half_t)alpha, (cutlass::half_t)beta}, // Scalars used in the Epilogue
                              1};// split_k_slices
  cutlass::Status status = gemm_operator(args);
  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }
  return hipSuccess;
}


hipError_t CutlassHgemmWithFusedPrologue(
  int M,
  int N,
  int K,
  half alpha,
  half const *A,
  int lda,
  half const *B,
  int ldb,
  half beta,
  half *C,
  int ldc,
  half* gamma_vec,
  half* beta_vec) {

  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  using MMAOp = cutlass::arch::OpClassTensorOp;
  using SmArch = cutlass::arch::Sm80;
  using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
  using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32
  using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 16>;  // <- MMA Op tile M = 16, N = 8, K = 8
  using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,                                     // <- data type of output matrix
      128 / cutlass::sizeof_bits<cutlass::half_t>::value,  // <- the number of elements per vectorized
                                                        // memory access. For a byte, it's 16
                                                        // elements. This becomes the vector width of
                                                        // math instructions in the epilogue too
      cutlass::half_t,                                            // <- data type of accumulator
      cutlass::half_t>;                                           // <- data type for alpha/beta in linear combination function
  constexpr int NumStages = 4;
  using CutlassGemm = cutlass::gemm::device::CustomizedGemm<cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  MMAOp,
                                                  SmArch,
                                                  ShapeMMAThreadBlock,
                                                  ShapeMMAWarp,
                                                  ShapeMMAOp,
                                                  EpilogueOp,
                                                  SwizzleThreadBlock,
                                                  NumStages>;

  CutlassGemm gemm_operator;
  cutlass::gemm::GemmCoord problem_size(M, N, K);
  typename CutlassGemm::Arguments args{problem_size,  // Gemm Problem dimensions
                              {(cutlass::half_t*)A, lda},    // Tensor-ref for source matrix A
                              {(cutlass::half_t*)B, ldb},    // Tensor-ref for source matrix B
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for source matrix C
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              (cutlass::half_t*)gamma_vec,
                              (cutlass::half_t*)beta_vec,
                              {(cutlass::half_t)alpha, (cutlass::half_t)beta}, // Scalars used in the Epilogue
                              1};// split_k_slices
  cutlass::Status status = gemm_operator(args);
  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }
  return hipSuccess;
}


/*

C(m*n) = A(m*k) * B(k*n);

m: batch number * seq len; {100:100:1000} x {32, 64}
n = k = feature size = 512 or 768

C layer norm on the feature dim.

E(m*4k) = C(m*k) * D(k*4k);

fused implementation - 
non fused implementation perf - 

*/

#define B(i, j) B[i + (j)*LDB]//colmajor

/*
 *
 * B = (B - mean) *rev_mean_sqr * gamma + beta
 *
 */

// void cpu_prologue_mat(int M, int N, int LDB, FLOAT *B, FLOAT *mean_vec, FLOAT *rev_mean_sqr_vec, FLOAT *beta, FLOAT *gamma) {
//   for (int i = 0; i < M; i++) {
//     for (int j = 0; j < N; j++) {
//       B(i, j) = (FLOAT)(((float)B(i, j) - (float)mean_vec[j])  * (float)rev_mean_sqr_vec[j] * (float)gamma[i] + (float)beta[i]  );
//     }
//   }
// }

void cpu_prologue_mat(int M, int N, int LDB, FLOAT *B, FLOAT *beta, FLOAT *gamma) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      B(i, j) = (FLOAT)( (float)B(i, j) * (float)gamma[i] + (float)beta[i] );
    }
  }
}

int main(int argc, char **argv){
    int m, n, k;
    
    if (argc == 4) {
      m = atoi(argv[1]);n = atoi(argv[2]);k = atoi(argv[3]);
    }else if (argc == 2) {
      n = atoi(argv[1]); k = 768; m = k * 4; // leading_dim = 4 * hidden_dim
    }else {
      printf("adopt the default input.\n");
      m = 3072, n = 6144, k = 768;
    }

    int n_block = (n + 128 - 1) / 128;

    // ensure m/n/k are all divisible by warp-mma dims - 16 should be safe here.
    m = m & -15; n = n & -15; k = k & -15;

    printf("m = %d, n = %d, k = %d, n_block = %d.\n", m, n, k, n_block);
    FLOAT *A=NULL;
    FLOAT *B=NULL;
    FLOAT *C=NULL;
    FLOAT *C_ref = NULL;
    
    FLOAT *dA=NULL;
    FLOAT *dB=NULL;
    FLOAT *dB_fused=NULL;
    FLOAT *dC=NULL;
    FLOAT *dC_ref=NULL;
    FLOAT alpha = 1, beta = 0;
    FLOAT *mean_vec, *mean_square_vec;
    FLOAT *vec_gamma, *vec_beta;


    FLOAT *dtensor_Reduction, *dtensor_Reduction_Square;
    FLOAT *d_vec_gamma;
    FLOAT *d_vec_beta;

    float elapsed_time1, elapsed_time2;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    A=(FLOAT *)malloc(sizeof(FLOAT)*m*k);
    B=(FLOAT *)malloc(sizeof(FLOAT)*k*n);
    C=(FLOAT *)malloc(sizeof(FLOAT)*m*n);
    C_ref=(FLOAT *)malloc(sizeof(FLOAT)*m*n);
    
    int mn = (m < n) ? n : m;
    mn = (mn < k) ? k : mn;

    mean_vec=(FLOAT *)malloc(sizeof(FLOAT)*mn*n_block);
    mean_square_vec=(FLOAT *)malloc(sizeof(FLOAT)*mn*n_block);
    vec_gamma=(FLOAT *)malloc(sizeof(FLOAT)*mn*1);
    vec_beta=(FLOAT *)malloc(sizeof(FLOAT)*mn*1);
    randomize_matrix(A,m*k);
    randomize_matrix_plus_one(B,k*n);
    randomize_matrix(C,m*n);
    
    randomize_matrix(mean_vec, mn*1);
    randomize_matrix(mean_square_vec, mn*1);
    randomize_matrix_plus_one(vec_beta, mn*1);
    randomize_matrix_plus_one(vec_gamma, mn*1);
    
    copy_matrix(C, C_ref, m*n);

    CUDA_CALLER(hipMalloc((void**) &dA, sizeof(FLOAT)*m*k));
    CUDA_CALLER(hipMalloc((void**) &dB, sizeof(FLOAT)*k*n));
    CUDA_CALLER(hipMalloc((void**) &dB_fused, sizeof(FLOAT)*k*n));
    CUDA_CALLER(hipMalloc((void**) &dC, sizeof(FLOAT)*m*n));
    CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(FLOAT)*m*n));

    CUDA_CALLER(hipMalloc((void**) &dtensor_Reduction, sizeof(FLOAT)*mn*n_block));
    CUDA_CALLER(hipMalloc((void**) &dtensor_Reduction_Square, sizeof(FLOAT)*mn*n_block));
    CUDA_CALLER(hipMalloc((void**) &d_vec_beta, sizeof(FLOAT)*mn*1));
    CUDA_CALLER(hipMalloc((void**) &d_vec_gamma, sizeof(FLOAT)*mn*1));

    CUDA_CALLER(hipMemcpy(dA, A, sizeof(FLOAT)*m*k, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB, B, sizeof(FLOAT)*k*n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC, C, sizeof(FLOAT)*m*n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC_ref, C_ref, sizeof(FLOAT)*m*n, hipMemcpyHostToDevice));

    CUDA_CALLER(hipMemcpy(d_vec_beta, vec_beta, sizeof(FLOAT)*mn*1, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(d_vec_gamma, vec_gamma, sizeof(FLOAT)*mn*1, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dtensor_Reduction, mean_vec, sizeof(FLOAT)*mn*n_block, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dtensor_Reduction_Square, mean_square_vec, sizeof(FLOAT)*mn*n_block, hipMemcpyHostToDevice));


    // cpu_prologue_mat(k, n, k, B, mean_vec, mean_square_vec, vec_beta, vec_gamma);
    cpu_prologue_mat(k, n, k, B, vec_beta, vec_gamma);

    CUDA_CALLER(hipMemcpy(dB_fused, B, sizeof(FLOAT)*k*n, hipMemcpyHostToDevice));

    CUDA_CALLER(hipEventRecord(beg));

    CUDA_CALLER(CutlassHgemmNN(
        m, n, k,
        alpha,
        dA, m,
        dB_fused, k,
        beta,
        dC, m
    ));

    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time1, beg, end);
    printf("Average elasped original time: %f ms, performance: %f TFLOPS.\n", elapsed_time1,2.*1e-9*m*n*k/elapsed_time1);
    
    CUDA_CALLER(hipEventRecord(beg));

    CUDA_CALLER(CutlassHgemmWithFusedPrologue(
      m, n, k,
      alpha,
      dA, m,
      dB, k,
      beta,
      dC_ref, m,
      d_vec_gamma,
      d_vec_beta
    ));

    CUDA_CALLER(hipEventRecord(end));
    CUDA_CALLER(hipEventSynchronize(beg));
    CUDA_CALLER(hipEventSynchronize(end));
    hipEventElapsedTime(&elapsed_time2, beg, end);

    printf("Average elasped fused time: %f ms, performance: %f TFLOPS.\n", elapsed_time2,2.*1e-9*m*n*k/elapsed_time2);

    printf("prologue fusion overhead = %5.2f %%.\n", (elapsed_time2 - elapsed_time1) / elapsed_time2 * 100);

    CUDA_CALLER(hipMemcpy(C, dC, sizeof(FLOAT)*m*n, hipMemcpyDeviceToHost));
    CUDA_CALLER(hipMemcpy(C_ref, dC_ref, sizeof(FLOAT)*m*n, hipMemcpyDeviceToHost));
    
    printf("******** validating C ********\n");
    if (verify_matrix(C, C_ref, m*n)) {
      printf("result is correct.\n");
    }

    hipDeviceSynchronize();
    free(A);free(B);free(C);free(C_ref);free(vec_beta);free(vec_gamma);
    free(mean_vec);free(mean_square_vec);
    hipFree(dA);hipFree(dB);hipFree(dC);hipFree(dC_ref);
    hipFree(dtensor_Reduction);hipFree(dtensor_Reduction_Square);
    hipFree(dB_fused);hipFree(d_vec_beta);hipFree(d_vec_gamma);

    hipDeviceSynchronize();
    return 0;
}
