#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>
#include "utils.cuh"
#include <hip/hip_fp16.h>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/functional.h"
#include "customized_device_gemm.h"


#define DEBUG 1

template <const int ite>
__inline__ __device__
void layernorm_v2(
    float2* local_out_fp2, const __half *gamma, const __half *beta, half2* out_ptr, const float *mean_vec, const float *mean_square_vec, int n)
{
    float2 gamma_val[ite], beta_val[ite];
    {
        #pragma unroll
        for(int i = 0; i < ite; i++)
        {
            int col_id = i * blockDim.x + threadIdx.x;
            gamma_val[i] = __half22float2(__ldg(&((const half2 *)gamma)[col_id]));
            beta_val[i]  = __half22float2(__ldg(&((const half2 *)beta)[col_id]));
        }
    }
    int bid = blockIdx.x;
    #pragma unroll
    for(int i = 0; i < ite; i++)
    {
        // local_out_fp2[i].x = local_out_fp2[i].x * s_[1] * gamma_val[i].x + beta_val[i].x;
        // local_out_fp2[i].y = local_out_fp2[i].y * s_[1] * gamma_val[i].y + beta_val[i].y;

        // local_out_fp2[i].x = local_out_fp2[i].x * s_[1];// * gamma_val[i].x + beta_val[i].x;
        // local_out_fp2[i].y = local_out_fp2[i].y * s_[1];// * gamma_val[i].y + beta_val[i].y;

        local_out_fp2[i].x = (local_out_fp2[i].x - mean_vec[bid]) * gamma_val[i].x * mean_square_vec[bid] + beta_val[i].x;
        local_out_fp2[i].y = (local_out_fp2[i].y  - mean_vec[bid]) * gamma_val[i].y * mean_square_vec[bid] + beta_val[i].y;

        //if(blockIdx.x == 0) {
            // printf("src_id = %d.\n", i * blockDim.x + threadIdx.x);
        //}

        out_ptr[i * blockDim.x + threadIdx.x] = __float22half2_rn(local_out_fp2[i]);
    }
}

template <const int ite>
__global__
void add_bias_input_layernorm_v2(
    __half *out, const __half *input, const __half *gamma, const __half *beta, const float *mean_vec, const float *mean_square_vec, int m, int n)
{
    half2 *out_ptr = (half2 *)out;
    const half2 *input_ptr = (const half2 *)input;
    int offset = blockIdx.x * n / 2;

    float2 local_out_fp2[ite];
#pragma unroll
    for(int i = 0; i < ite; i++)
    {
        int col_id = i * blockDim.x + threadIdx.x;
        int id = offset + col_id;
        local_out_fp2[i] = __half22float2(__ldg(&input_ptr[id]));
    }
    
    __shared__ float s_[2];
    layernorm_v2<ite>(local_out_fp2, gamma, beta, ((half2 *)out) + offset, mean_vec, mean_square_vec, n);
}

void my_add_bias_input_layernorm_kernel_launcher(
    __half *output, const __half *input, const __half *gamma, const __half *beta, const float *mean_vec, const float *mean_square_vec, int m, int n)
{
    dim3 grid(m), block(n / 2);
    printf("n = %d, k = %d, block.x = %d.\n", m, n, block.x);

    const int ite = 4;
    add_bias_input_layernorm_v2<ite><<<grid, block.x / ite, 0>>>(
            output, input, gamma, beta, mean_vec, mean_square_vec, m, n);

}



hipError_t CutlassHgemmNN(
  int M,
  int N,
  int K,
  half alpha,
  half const *A,
  int lda,
  half const *B,
  int ldb,
  half beta,
  half *C,
  int ldc) {

  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  using MMAOp = cutlass::arch::OpClassTensorOp;
  using SmArch = cutlass::arch::Sm80;
  using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
  using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32
  using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 16>;  // <- MMA Op tile M = 16, N = 8, K = 8
  using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,                                     // <- data type of output matrix
      128 / cutlass::sizeof_bits<cutlass::half_t>::value,  // <- the number of elements per vectorized
                                                        // memory access. For a byte, it's 16
                                                        // elements. This becomes the vector width of
                                                        // math instructions in the epilogue too
      cutlass::half_t,                                            // <- data type of accumulator
      cutlass::half_t>;                                           // <- data type for alpha/beta in linear combination function
  constexpr int NumStages = 4;
  using CutlassGemm = cutlass::gemm::device::Gemm<cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  MMAOp,
                                                  SmArch,
                                                  ShapeMMAThreadBlock,
                                                  ShapeMMAWarp,
                                                  ShapeMMAOp,
                                                  EpilogueOp,
                                                  SwizzleThreadBlock,
                                                  NumStages>;

  CutlassGemm gemm_operator;
  cutlass::gemm::GemmCoord problem_size(M, N, K);
  typename CutlassGemm::Arguments args{problem_size,  // Gemm Problem dimensions
                              {(cutlass::half_t*)A, lda},    // Tensor-ref for source matrix A
                              {(cutlass::half_t*)B, ldb},    // Tensor-ref for source matrix B
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for source matrix C
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {(cutlass::half_t)alpha, (cutlass::half_t)beta}, // Scalars used in the Epilogue
                              1};// split_k_slices
  cutlass::Status status = gemm_operator(args);
  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }
  return hipSuccess;
}


hipError_t CutlassHgemmWithFusedPrologue(
  int M,
  int N,
  int K,
  half alpha,
  half const *A,
  int lda,
  half const *B,
  int ldb,
  half beta,
  half *C,
  int ldc,
  half* gamma_vec,
  half* beta_vec,
  float *mean_vec,
  float *mean_square_vec) {

  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  using MMAOp = cutlass::arch::OpClassTensorOp;
  using SmArch = cutlass::arch::Sm80;
  using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 32>;  // <- threadblock tile M = 128, N = 128, K = 32
  using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;  // <- warp tile M = 64, N = 64, K = 32
  using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 16>;  // <- MMA Op tile M = 16, N = 8, K = 8
  using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;  // <- ??
  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
      cutlass::half_t,                                     // <- data type of output matrix
      128 / cutlass::sizeof_bits<cutlass::half_t>::value,  // <- the number of elements per vectorized
                                                        // memory access. For a byte, it's 16
                                                        // elements. This becomes the vector width of
                                                        // math instructions in the epilogue too
      cutlass::half_t,                                            // <- data type of accumulator
      cutlass::half_t>;                                           // <- data type for alpha/beta in linear combination function
  constexpr int NumStages = 4;
  using CutlassGemm = cutlass::gemm::device::CustomizedGemm<cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  ColumnMajor,
                                                  cutlass::half_t,
                                                  cutlass::half_t,
                                                  RowMajor,
                                                  MMAOp,
                                                  SmArch,
                                                  ShapeMMAThreadBlock,
                                                  ShapeMMAWarp,
                                                  ShapeMMAOp,
                                                  EpilogueOp,
                                                  SwizzleThreadBlock,
                                                  NumStages>;

  CutlassGemm gemm_operator;
  cutlass::gemm::GemmCoord problem_size(M, N, K);
  typename CutlassGemm::Arguments args{problem_size,  // Gemm Problem dimensions
                              {(cutlass::half_t*)A, lda},    // Tensor-ref for source matrix A
                              {(cutlass::half_t*)B, ldb},    // Tensor-ref for source matrix B
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for source matrix C
                              {(cutlass::half_t*)C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              (cutlass::half_t*)gamma_vec,
                              (cutlass::half_t*)beta_vec,
                              (float*)mean_vec,
                              (float*)mean_square_vec,
                              {(cutlass::half_t)alpha, (cutlass::half_t)beta}, // Scalars used in the Epilogue
                              1};// split_k_slices
  cutlass::Status status = gemm_operator(args);
  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }
  return hipSuccess;
}

__global__
void gpu_fuse(int m, int n, __half *C, __half *beta, __half *gamma, float *mean_vec, float *mean_square_vec) {
    half2 *input_ptr = (half2 *)C;
    half2 *out_ptr = (half2 *)C;
    int offset = blockIdx.x * n / 2;
    out_ptr = out_ptr + offset;
    float2 local_out_fp2[4];

#pragma unroll
    for(int i = 0; i < 4; i++)
    {
        int col_id = i * blockDim.x + threadIdx.x;
        int id = offset + col_id;
        local_out_fp2[i] = __half22float2(__ldg(&input_ptr[id]));
    }
    float2 gamma_val[4], beta_val[4];
    int bid = blockIdx.x;
    #pragma unroll
    for(int i = 0; i < 4; i++)
    {
        int col_id = i * blockDim.x + threadIdx.x;
        gamma_val[i] = __half22float2(__ldg(&((const half2 *)gamma)[col_id]));
        beta_val[i]  = __half22float2(__ldg(&((const half2 *)beta)[col_id]));
    }
    #pragma unroll
    for(int i = 0; i < 4; i++)
    {
        local_out_fp2[i].x = (local_out_fp2[i].x - mean_vec[bid]) * gamma_val[i].x * mean_square_vec[bid] + beta_val[i].x;
        local_out_fp2[i].y = (local_out_fp2[i].y  - mean_vec[bid]) * gamma_val[i].y * mean_square_vec[bid] + beta_val[i].y;

        out_ptr[i * blockDim.x + threadIdx.x] = __float22half2_rn(local_out_fp2[i]);
    }
}

void my_gpu_fuse(int m, int n, __half *C, __half *beta, __half *gamma, float *mean_vec, float *mean_square_vec) {
    dim3 grid(m);
    dim3 block(n / 2);
    gpu_fuse<<<grid, block.x / 4, 0>>>(m, n, C, beta, gamma, mean_vec, mean_square_vec);
}

/*

C(m*n) = A(m*k) * B(k*n);

m: batch number * seq len; {100:100:1000} x {32, 64}
n = k = feature size = 512 or 768

C layer norm on the feature dim.

E(m*4k) = C(m*k) * D(k*4k);

fused implementation - 
non fused implementation perf - 

*/

#define B(i, j) B[i + (j)*LDB]//colmajor

/*
 *
 * B = (B - mean) *rev_mean_sqr * gamma + beta
 *
 */

// void cpu_prologue_mat(int M, int N, int LDB, FLOAT *B, FLOAT *mean_vec, FLOAT *rev_mean_sqr_vec, FLOAT *beta, FLOAT *gamma) {
//   for (int i = 0; i < M; i++) {
//     for (int j = 0; j < N; j++) {
//       B(i, j) = (FLOAT)(((float)B(i, j) - (float)mean_vec[j])  * (float)rev_mean_sqr_vec[j] * (float)gamma[i] + (float)beta[i]  );
//     }
//   }
// }

void cpu_prologue_mat(int M, int N, int LDB, FLOAT *B, FLOAT *beta, FLOAT *gamma) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      B(i, j) = (FLOAT)( (float)B(i, j) * (float)gamma[i] + (float)beta[i] );
    }
  }
}

void cpu_prologue_mat(int M, int N, int LDB, FLOAT *B, float *beta, float *gamma) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      B(i, j) = (FLOAT)( (float)B(i, j) * (float)gamma[i] + (float)beta[i] );
    }
  }
}

int main(int argc, char **argv){
    int m, n, k;
    
    if (argc == 4) {
      m = atoi(argv[1]);n = atoi(argv[2]);k = atoi(argv[3]);
    }else if (argc == 2) {
      n = atoi(argv[1]); k = 768; m = k * 4; // leading_dim = 4 * hidden_dim
    }else {
      printf("adopt the default input.\n");
      m = 3072, n = 6144, k = 768;
    }

    int n_block = (n + 128 - 1) / 128;

    // ensure m/n/k are all divisible by warp-mma dims - 16 should be safe here.
    m = m & -15; n = n & -15; k = k & -15;

    printf("m = %d, n = %d, k = %d, n_block = %d.\n", m, n, k, n_block);
    FLOAT *A=NULL;
    FLOAT *B=NULL;
    FLOAT *B_fused=NULL;
    FLOAT *C=NULL;
    FLOAT *C_ref = NULL;
    
    FLOAT *dA=NULL;
    FLOAT *dB=NULL;
    FLOAT *dB_fused=NULL;
    FLOAT *dC=NULL;
    FLOAT *dC_ref=NULL;
    FLOAT alpha = 1, beta = 0;
    float *mean_vec, *mean_square_vec;
    FLOAT *vec_gamma, *vec_beta;


    float *dtensor_Reduction, *dtensor_Reduction_Square;
    FLOAT *d_vec_gamma;
    FLOAT *d_vec_beta;

    float elapsed_time1, elapsed_time2;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    A=(FLOAT *)malloc(sizeof(FLOAT)*m*k);
    B=(FLOAT *)malloc(sizeof(FLOAT)*k*n);
    B_fused=(FLOAT *)malloc(sizeof(FLOAT)*k*n);
    C=(FLOAT *)malloc(sizeof(FLOAT)*m*n);
    C_ref=(FLOAT *)malloc(sizeof(FLOAT)*m*n);
    
    int mn = (m < n) ? n : m;
    mn = (mn < k) ? k : mn;

    mean_vec=(float *)malloc(sizeof(float)*mn*1);
    mean_square_vec=(float *)malloc(sizeof(float)*mn*1);
    vec_gamma=(FLOAT *)malloc(sizeof(FLOAT)*mn*1);
    vec_beta=(FLOAT *)malloc(sizeof(FLOAT)*mn*1);
    randomize_matrix(A,m*k);
    randomize_matrix_plus_one(B,k*n);
    randomize_matrix(C,m*n);
    
    randomize_matrix(mean_vec, mn*1);
    randomize_matrix_plus_one(mean_square_vec, mn*1);
    randomize_matrix(vec_beta, mn*1);
    randomize_matrix_plus_one(vec_gamma, mn*1);
    
    copy_matrix(C, C_ref, m*n);

    CUDA_CALLER(hipMalloc((void**) &dA, sizeof(FLOAT)*m*k));
    CUDA_CALLER(hipMalloc((void**) &dB, sizeof(FLOAT)*k*n));
    CUDA_CALLER(hipMalloc((void**) &dB_fused, sizeof(FLOAT)*k*n));
    CUDA_CALLER(hipMalloc((void**) &dC, sizeof(FLOAT)*m*n));
    CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(FLOAT)*m*n));

    CUDA_CALLER(hipMalloc((void**) &dtensor_Reduction, sizeof(float)*mn*1));
    CUDA_CALLER(hipMalloc((void**) &dtensor_Reduction_Square, sizeof(float)*mn*1));
    CUDA_CALLER(hipMalloc((void**) &d_vec_beta, sizeof(FLOAT)*mn*1));
    CUDA_CALLER(hipMalloc((void**) &d_vec_gamma, sizeof(FLOAT)*mn*1));

    CUDA_CALLER(hipMemcpy(dA, A, sizeof(FLOAT)*m*k, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB, B, sizeof(FLOAT)*k*n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB_fused, B, sizeof(FLOAT)*k*n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC, C, sizeof(FLOAT)*m*n, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC_ref, C_ref, sizeof(FLOAT)*m*n, hipMemcpyHostToDevice));

    CUDA_CALLER(hipMemcpy(d_vec_beta, vec_beta, sizeof(FLOAT)*mn*1, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(d_vec_gamma, vec_gamma, sizeof(FLOAT)*mn*1, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dtensor_Reduction, mean_vec, sizeof(float)*mn*1, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dtensor_Reduction_Square, mean_square_vec, sizeof(float)*mn*1, hipMemcpyHostToDevice));

    // printf("*** matrix A ***\n");
    // print_matrix(A, m, k);
    // printf("*** matrix B ***\n");
    // print_matrix(B, n, k);
    // printf("*** matrix vec_beta ***\n");
    // print_matrix(vec_beta, 1, k);
    // printf("*** matrix vec_gamma ***\n");
    // print_matrix(vec_gamma, 1, k);

    // cpu_prologue_mat(k, n, k, B, mean_vec, mean_square_vec, vec_beta, vec_gamma);
    // cpu_prologue_mat(k, n, k, B, vec_beta, vec_gamma);

    // printf("*** matrix B after fused ***\n");
    // print_matrix(B, n, k);

    // my_add_bias_input_layernorm_kernel_launcher(dB_fused, dB, d_vec_gamma, d_vec_beta, n, k);
    my_gpu_fuse(n, k, dB_fused, d_vec_beta, d_vec_gamma, dtensor_Reduction, dtensor_Reduction_Square);
    // CUDA_CALLER(hipMemcpy(dB_fused, B, sizeof(FLOAT)*k*n, hipMemcpyHostToDevice));

    CUDA_CALLER(hipEventRecord(beg));

    CUDA_CALLER(CutlassHgemmNN(
        m, n, k,
        alpha,
        dA, m,
        dB_fused, k,
        beta,
        dC, m
    ));

    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time1, beg, end);
    printf("Average elasped original time: %f ms, performance: %f TFLOPS.\n", elapsed_time1,2.*1e-9*m*n*k/elapsed_time1);
    
    CUDA_CALLER(hipEventRecord(beg));

    CUDA_CALLER(CutlassHgemmWithFusedPrologue(
      m, n, k,
      alpha,
      dA, m,
      dB, k,
      beta,
      dC_ref, m,
      d_vec_gamma,
      d_vec_beta,
      dtensor_Reduction,
      dtensor_Reduction_Square
    ));

    CUDA_CALLER(hipEventRecord(end));
    CUDA_CALLER(hipEventSynchronize(beg));
    CUDA_CALLER(hipEventSynchronize(end));
    hipEventElapsedTime(&elapsed_time2, beg, end);

    printf("Average elasped fused time: %f ms, performance: %f TFLOPS.\n", elapsed_time2,2.*1e-9*m*n*k/elapsed_time2);

    printf("prologue fusion overhead = %5.2f %%.\n", (elapsed_time2 - elapsed_time1) / elapsed_time2 * 100);

    CUDA_CALLER(hipMemcpy(C, dC, sizeof(FLOAT)*m*n, hipMemcpyDeviceToHost));
    CUDA_CALLER(hipMemcpy(C_ref, dC_ref, sizeof(FLOAT)*m*n, hipMemcpyDeviceToHost));

    // CUDA_CALLER(hipMemcpy(B_fused, dB_fused, sizeof(FLOAT)*k*n, hipMemcpyDeviceToHost));
    // CUDA_CALLER(hipMemcpy(B, dB, sizeof(FLOAT)*k*n, hipMemcpyDeviceToHost));
    
    printf("******** validating C ********\n");
    if (verify_matrix(C, C_ref, m*n)) {
      printf("result is correct.\n");
    }

    // printf("******** validating B ********\n");
    // if (verify_matrix(B, B_fused, k*n)) {
    //   printf("result is correct.\n");
    // }

    // printf("*** matrix C ***\n");
    // print_matrix(C, m, n);
    // printf("*** matrix C_ref ***\n");
    // print_matrix(C_ref, m, n);

    // printf("*** matrix B ***\n");
    // print_matrix(B, n, k);
    // printf("*** matrix B_fused ***\n");
    // print_matrix(B_fused, n, k);

    hipDeviceSynchronize();
    free(A);free(B);free(B_fused);free(C);free(C_ref);free(vec_beta);free(vec_gamma);
    free(mean_vec);free(mean_square_vec);
    hipFree(dA);hipFree(dB);hipFree(dC);hipFree(dC_ref);
    hipFree(dtensor_Reduction);hipFree(dtensor_Reduction_Square);
    hipFree(dB_fused);hipFree(d_vec_beta);hipFree(d_vec_gamma);

    hipDeviceSynchronize();
    return 0;
}
