/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Simple Hopper FP8 GEMM example using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example demonstrate a simple way to instantiate and run a FP8 GEMM using the new CUTLASS 3.0
    APIs on NVIDIA Hopper architecture. New features that will be showcased in this example are as follows:

    1. NVIDIA Hopper architecture introduces a new series of tensor core instructions (GMMA) 
    which are more efficient than the Ampere tensor core instructions.

    2. NVIDIA Hopper architecture includes new Tensor Memory Accelerator (TMA) unit to transfer large 
    blocks of data efficiently between global memory and shared memory. TMA also supports asynchronous
    copies between thread blocks in a cluster.

    3. This example uses the Warp Specialized kernel design (see /media/docs/efficient_gemm.md for details).

    4. This example shows all important fusions used by FP8 gemm kernels, 
    i.e., scale factor for A, B, C, D tensor, the abs_max value of D tensor.

    Examples:

      $ ./examples/54_hopper_fp8_warp_specialized_gemm/54_hopper_fp8_warp_specialized_gemm --m=2048 --n=2048 --k=2048
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"


#include "helper.h"
#include "hopper_fp8_commandline.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::float_e4m3_t;                          // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::float_e5m2_t;                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C matrix configuration
using         ElementC    = cutlass::float_e4m3_t;                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = AlignmentC;

// Auxiliary matrix configuration
using         ElementAux   = ElementC;
using         LayoutAux    = LayoutC;

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_64,_128,_128>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_1,_2,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::KernelTmaWarpSpecialized;
using EpilogueSchedule    = cutlass::epilogue::TmaWarpSpecialized;
using EpilogueTileType    = cutlass::epilogue::collective::EpilogueTileAuto;
using FusionOperation     = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltActAmaxAux<
    LayoutAux, cutlass::epilogue::thread::ReLU, ElementD, ElementCompute, ElementAux>;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    TileShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementCompute,
    ElementC, LayoutC, AlignmentC,
    ElementD, LayoutD, AlignmentD,
    EpilogueSchedule,
    FusionOperation
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Extract information from Gemm kernel.
using EpilogueOutputOp  = typename Gemm::EpilogueOutputOp;
using ElementScalar     = typename EpilogueOutputOp::ElementScalar;
using ElementAmax       = typename EpilogueOutputOp::ElementAmax;
using ActivationFunctor = typename EpilogueOutputOp::ActivationFn<ElementCompute>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;
using StrideAux = StrideD;

constexpr bool IsDFp8 = 
    cute::is_same_v<ElementD, cutlass::float_e4m3_t> or
    cute::is_same_v<ElementD, cutlass::float_e5m2_t>;

constexpr bool IsAuxFp8 =
    cute::is_same_v<ElementAux, cutlass::float_e4m3_t> or
    cute::is_same_v<ElementAux, cutlass::float_e5m2_t>;

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
StrideAux stride_aux;
uint64_t seed;

cutlass::HostTensor<ElementA  , LayoutA  > tensor_A;
cutlass::HostTensor<ElementB  , LayoutB  > tensor_B;
cutlass::HostTensor<ElementC  , LayoutC  > tensor_C;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_D;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_ref_D;
cutlass::HostTensor<ElementAux, LayoutAux> tensor_aux;
cutlass::HostTensor<ElementAux, LayoutAux> tensor_ref_aux;

using LayoutScalar = cutlass::layout::PackedVectorLayout;
cutlass::HostTensor<ElementScalar, LayoutScalar> scalar_alpha;
cutlass::HostTensor<ElementScalar, LayoutScalar> scalar_beta;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_A;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_B;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_C;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_D;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_aux;
cutlass::HostTensor<ElementAmax  , LayoutScalar> abs_max_D;
cutlass::HostTensor<ElementAmax  , LayoutScalar> reference_abs_max_D;
cutlass::HostTensor<ElementAmax  , LayoutScalar> abs_max_aux;
cutlass::HostTensor<ElementAmax  , LayoutScalar> reference_abs_max_aux;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed) {

  double scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;
  int bits_output = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  }
  else if (bits_output == 16) {
    scope_max = 5;
    scope_min = -5;
  }
  else {
    scope_max = 8;
    scope_min = -8;
  }
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));
  stride_aux = stride_D;

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  initialize_tensor(tensor_A.host_view(), seed + 2022);
  initialize_tensor(tensor_B.host_view(), seed + 2023);
  initialize_tensor(tensor_C.host_view(), seed + 2024);

  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_C.sync_device();
  tensor_D.sync_device();

  if (options.save_aux) {
    tensor_aux.resize(c_coord);
    tensor_aux.sync_device();
    tensor_ref_aux.resize(c_coord);
  }

  if (options.device_scale) {
    scalar_alpha.resize(cutlass::make_Coord(1));
    scalar_beta.resize(cutlass::make_Coord(1));
    scale_A.resize(cutlass::make_Coord(1));
    scale_B.resize(cutlass::make_Coord(1));
    scale_C.resize(cutlass::make_Coord(1));
    scale_D.resize(cutlass::make_Coord(1));
    scale_aux.resize(cutlass::make_Coord(1));

    cutlass::reference::host::TensorFill(scalar_alpha.host_view(), options.alpha);
    cutlass::reference::host::TensorFill(scalar_beta.host_view(), options.beta);
    cutlass::reference::host::TensorFill(scale_A.host_view(), options.scale_a);
    cutlass::reference::host::TensorFill(scale_B.host_view(), options.scale_b);
    cutlass::reference::host::TensorFill(scale_C.host_view(), options.scale_c);
    cutlass::reference::host::TensorFill(scale_D.host_view(), options.scale_d);
    cutlass::reference::host::TensorFill(scale_aux.host_view(), options.scale_aux);

    scalar_alpha.sync_device();
    scalar_beta.sync_device();
    scale_A.sync_device();
    scale_B.sync_device();
    scale_C.sync_device();
    scale_D.sync_device();
    scale_aux.sync_device();
  }

  if (IsDFp8 && options.save_amax) {
    abs_max_D.resize(cutlass::make_Coord(1));
    abs_max_D.sync_device();
    reference_abs_max_D.resize(cutlass::make_Coord(1));
  }

  if (IsAuxFp8 && options.save_aux && options.save_amax) {
    abs_max_aux.resize(cutlass::make_Coord(1));
    abs_max_aux.sync_device();
    reference_abs_max_aux.resize(cutlass::make_Coord(1));
  }
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(), stride_A, tensor_B.device_data(), stride_B},
    {
      {}, // epilogue.thread
      tensor_C.device_data(), stride_C,
      tensor_D.device_data(), stride_D
    }
  };

  auto &fusion_args = arguments.epilogue.thread;
  fusion_args.alpha = options.alpha;
  fusion_args.beta = options.beta;
  fusion_args.alpha_ptr = scalar_alpha.device_data();
  fusion_args.beta_ptr = scalar_beta.device_data();
  fusion_args.scale_a = options.scale_a;
  fusion_args.scale_b = options.scale_b;
  fusion_args.scale_c = options.scale_c;
  fusion_args.scale_a_ptr = scale_A.device_data();
  fusion_args.scale_b_ptr = scale_B.device_data();
  fusion_args.scale_c_ptr = scale_C.device_data();

  // ignored if tensor types are not fp8
  fusion_args.scale_d = options.scale_d;
  fusion_args.scale_aux = options.scale_aux;
  fusion_args.scale_d_ptr = scale_D.device_data();
  fusion_args.scale_aux_ptr = scale_aux.device_data();

  // leaving/setting these as nullptr disables the fusion at runtime
  fusion_args.bias_ptr = nullptr;

  if (options.save_aux) {
    fusion_args.aux_ptr = tensor_aux.device_data();
    fusion_args.dAux = stride_aux;
    if (options.save_amax) {
      fusion_args.amax_aux_ptr = abs_max_aux.device_data();
    }
  }

  if (options.save_amax) {
    fusion_args.amax_D_ptr = abs_max_D.device_data();
  }

  return arguments;
}

bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.k, options.l), stride_A));
  auto B = cute::make_tensor(tensor_B.host_data(),
      cute::make_layout(cute::make_shape(options.n, options.k, options.l), stride_B));
  auto C = cute::make_tensor(tensor_C.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_C));
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_D));
  auto Aux = cute::make_tensor(tensor_ref_aux.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_aux));
  using unused_t = decltype(D);

  cutlass::reference::host::GettMainloopParams<ElementAccumulator, decltype(A), decltype(B)> mainloop_params{A, B};

  cutlass::reference::host::GettEpilogueParams<
      ElementScalar,
      ElementScalar,
      ElementAccumulator,
      ElementCompute,
      decltype(C),
      decltype(D),
      unused_t, // bias
      decltype(Aux),
      unused_t, // valpha
      unused_t, // vbeta
      ActivationFunctor
  > epilogue_params;

  epilogue_params.C = C;
  epilogue_params.D = D;
  epilogue_params.Aux = Aux;
  epilogue_params.alpha = options.alpha;
  epilogue_params.beta = options.beta;
  epilogue_params.scale_a = options.scale_a;
  epilogue_params.scale_b = options.scale_b;
  epilogue_params.scale_c = options.scale_c;
  epilogue_params.scale_d = options.scale_d;
  epilogue_params.scale_aux = options.scale_aux;
  epilogue_params.abs_max_D = reference_abs_max_D.host_data();
  epilogue_params.abs_max_Aux = reference_abs_max_aux.host_data();

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  // compare_reference
  tensor_D.sync_host();
  bool passed = cutlass::reference::host::TensorEquals(tensor_ref_D.host_view(), tensor_D.host_view());

  if (IsDFp8 && options.save_amax) {
    abs_max_D.sync_host();
    passed &= abs_max_D.at(cutlass::make_Coord(0)) == reference_abs_max_D.at(cutlass::make_Coord(0));
  }

  if (options.save_aux) {
    tensor_aux.sync_host();
    passed &= cutlass::reference::host::TensorEquals(tensor_ref_aux.host_view(), tensor_aux.host_view());
    if (IsAuxFp8 && options.save_amax) {
      abs_max_aux.sync_host();
      passed &= abs_max_aux.at(cutlass::make_Coord(0)) == reference_abs_max_aux.at(cutlass::make_Coord(0));
    }
  }

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
