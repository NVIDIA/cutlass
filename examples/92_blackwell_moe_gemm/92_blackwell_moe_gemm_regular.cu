/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
  \brief Example of Blackwell MoE-style GEMM implementation using TMA to load A and CPASYNC to load B.

  This example demonstrates an implementation of GEMM using mixed TMA+CPASYNC to load input matrices.
  In the decoding stage of Mixture of Experts (MoE) models, the number of tokens in different experts 
  can varies a lot, which requires frequently updates of TMA descriptors in TMA-based implementation.
  This examples uses CPASYNC to load activation (B) matrix to avoid the overhead of updating TMA descriptors.

  Usage:
  $ ./examples/92_blackwell_moe_gemm/92_blackwell_moe_gemm_regular 
  --m=28672 --n=4 --k=4096 --l=8

*/

#include <iostream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"


using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool verification;

  int m, n, k, l;

  int iterations;

  Options():
    help(false),
    error(false),
    verification(true),
    m(2048), n(2048), k(2048), l(1),
    iterations(10)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 2048);
    cmd.get_cmd_line_argument("n", n, 2048);
    cmd.get_cmd_line_argument("k", k, 2048);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("iterations", iterations, 10);

    if (cmd.check_cmd_line_flag("no_verif")) {
      verification = false;
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "92_blackwell_moe_gemm_regular\n\n"
      << "  Blackwell GEMM implementation using TMA to load A and CPASYNC to load B\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --iterations=<int>          Set the number of profiling iterations to perform\n"
      << "  --no_verif                  Do not run verification kernels\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = static_cast<Element>(2);
    scope_min = static_cast<Element>(0);
  }
  else if (bits_input <= 8) {
    scope_max = static_cast<Element>(2);
    scope_min = static_cast<Element>(-2);
  }
  else {
    scope_max = static_cast<Element>(8);
    scope_min = static_cast<Element>(-8);
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////



// Wrapper to construct, run, and verify a GEMM. This example showcases CUTLASS's collective
// operation builders by specializing the GEMM on the kernel+epilogue schedule it will use and the
// number of pipeline stages.
template <
  // Type of kernel schedule to generate
  class MainloopScheduleType = cutlass::gemm::KernelMixedTmaCpAsyncWarpSpecialized1SmSm100,
  // Type of epilogue schedule to generate
  class EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto
>
struct ExampleRunner {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::ColumnMajor;

  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::half_t;
  using ElementAccumulator = float;
  using ElementCompute = float;
  using ElementScalar = float;

  using ClusterShapeMNK = Shape<_1,_1,_1>;
  using MmaTileMNK    = Shape<_128,_16,Int<128 / sizeof(ElementA)>>;  // use tile size of N=16 to match real use cases (N is typically very small in decoding stage)

  // 16B alignment lets us use TMA
  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      MmaTileMNK, ClusterShapeMNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      EpilogueScheduleType,
      cutlass::epilogue::fusion::LinearCombination<ElementC, ElementAccumulator>
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, AlignmentA,
      ElementB, LayoutB, AlignmentB,
      ElementAccumulator,
      MmaTileMNK, ClusterShapeMNK,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopScheduleType
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutTagA = cutlass::gemm::detail::StrideToLayoutTagA_t<StrideA>;
  using LayoutTagB = cutlass::gemm::detail::StrideToLayoutTagB_t<StrideB>;
  using LayoutTagC = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideC>;
  using LayoutTagD = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideD>;

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_D;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_ref_D;

  //
  // Methods
  //

  bool verify(ProblemShapeType const& problem_size, float alpha, float beta) {
    auto [M, N, K, L] = problem_size;

    cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({M, N}));

    cutlass::reference::device::GemmComplex(
          {M, N, K},
          ElementScalar(alpha),
          ref_A,
          cutlass::ComplexTransform::kNone,
          ref_B,
          cutlass::ComplexTransform::kNone,
          ElementScalar(beta),
          ref_C,
          ref_D,
          ElementAccumulator(0),
          L,     // batch_count
          M * K, // batch_stride_A
          K * N, // batch_stride_B
          M * N, // batch_stride_C
          M * N  // batch_stride_D
        );

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

    return passed;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(ProblemShapeType const& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(size_t(1) * M * K * L);
    block_B.reset(size_t(1) * K * N * L);
    block_C.reset(size_t(1) * M * N * L);
    block_D.reset(size_t(1) * M * N * L);
    block_ref_D.reset(size_t(1) * M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  bool run(Options const& options, cutlass::KernelHardwareInfo const& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.m, options.n, options.k, options.l};

    initialize(problem_size);

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      problem_size,
      {block_A.get(), stride_A, block_B.get(), stride_B},
      {{}, // epilogue.thread
       block_C.get(), stride_C, block_D.get(), stride_D},
      hw_info
    };

    // arguments.scheduler.max_swizzle_size = options.swizzle;
    
    arguments.epilogue.thread.alpha = 1.0f;
    arguments.epilogue.thread.beta = 0.0f;

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    // Run the GEMM
    status = gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    if (options.verification) {
      // Verify that the result is correct
      bool passed = verify(problem_size, 1.0f, 0.0f);

      std::cout << "  Disposition: " << (passed ? "Passed" : "Failed") << std::endl;

      if (!passed) {
        exit(-1);
        return false;
      }
    }

    // Run profiling loop
    if (options.iterations > 0)
    {
      GpuTimer timer;
      timer.start();
      for (int iter = 0; iter < options.iterations; ++iter) {
        CUTLASS_CHECK(gemm_op.initialize(arguments, workspace.get()));
        CUTLASS_CHECK(gemm_op.run());
      }
      timer.stop();

      // Compute average setup and runtime and FLOPs.
      float elapsed_ms       = timer.elapsed_millis();
      double avg_runtime_ms  = double(elapsed_ms) / double(options.iterations);
      double flops           = double(int64_t(2) * options.m * options.n * options.k * options.l) / (avg_runtime_ms / 1000.0);

      std::cout << "  Avg runtime : " << avg_runtime_ms << " ms" << std::endl;
      std::cout << "  TFLOPS      : " << flops / 1e12 << std::endl;
    }

    return true;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)) {
    std::cerr << "This example requires CUDA 12.8 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
  
  if (!(props.major == 10 && props.minor == 0)) {
    std::cerr << "This example requires a GPU of NVIDIA's Blackwell architecture (compute capability 100)." << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  std::cout << "Running kernel with TMA load:" << std::endl;
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized1SmSm100> runner_tma;
  runner_tma.run(options, hw_info);

  std::cout << "Running kernel with CPASYNC load:" << std::endl;
  ExampleRunner<cutlass::gemm::KernelWarpSpecialized1SmSm100> runner_cpasync;
  runner_cpasync.run(options, hw_info);

  std::cout << "Running kernel with mixed TMA+CPASYNC load:" << std::endl;
  ExampleRunner<cutlass::gemm::KernelMixedTmaCpAsyncWarpSpecialized1SmSm100> runner_mixed_tma_cpasync;
  runner_mixed_tma_cpasync.run(options, hw_info);

#endif

  return 0;
}
