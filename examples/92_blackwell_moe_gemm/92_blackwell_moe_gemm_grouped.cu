#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
  \brief Example of Blackwell MoE-style grouped GEMM implementation using TMA to load A and CPASYNC to load B.

  This example demonstrates an implementation of GEMM using mixed TMA+CPASYNC to load input matrices.
  In the decoding stage of Mixture of Experts (MoE) models, the number of tokens in different experts 
  can varies a lot, which requires frequently updates of TMA descriptors in TMA-based implementation.
  This examples uses CPASYNC to load activation (B) matrix to avoid the overhead of updating TMA descriptors.

  Usage:
  $ ./examples/92_blackwell_moe_gemm/92_blackwell_moe_gemm_grouped
  --m=28672 --n=4 --k=4096 --l=8 --benchmark=benchmark.txt

*/

#include <iostream>
#include <fstream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"


using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool verification;

  int m, n, k, l;

  int iterations;

  std::string benchmark_path;

  Options():
    help(false),
    error(false),
    verification(true),
    m(2048), n(2048), k(2048), l(1),
    iterations(10)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 2048);
    cmd.get_cmd_line_argument("n", n, 2048);
    cmd.get_cmd_line_argument("k", k, 2048);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("iterations", iterations, 10);
    cmd.get_cmd_line_argument("benchmark", benchmark_path);


    if (cmd.check_cmd_line_flag("no_verif")) {
      verification = false;
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "92_blackwell_moe_gemm_grouped\n\n"
      << "  Blackwell MoE-style grouped GEMM implementation using TMA to load A and CPASYNC to load B\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --iterations=<int>          Set the number of profiling iterations to perform\n"
      << "  --benchmark=<file>          Executes a benchmark problem size\n"
      << "  --no_verif                  Do not run verification kernels\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = static_cast<Element>(2);
    scope_min = static_cast<Element>(0);
  }
  else if (bits_input <= 8) {
    scope_max = static_cast<Element>(2);
    scope_min = static_cast<Element>(-2);
  }
  else {
    scope_max = static_cast<Element>(8);
    scope_min = static_cast<Element>(-8);
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleRunner {

  // Type of kernel schedule to generate
  using MainloopScheduleType = cutlass::gemm::KernelMixedTmaCpAsyncWarpSpecialized1SmSm100;
  // Type of epilogue schedule to generate
  using EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto;

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::ColumnMajor;

  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::half_t;
  using ElementAccumulator = float;
  using ElementCompute = float;
  using ElementScalar = float;

  using ClusterShapeMNK = Shape<_1,_1,_1>;
  using MmaTileMNK    = Shape<_128,_16,Int<128 / sizeof(ElementA)>>;  // use tile size of N=16 to match real use cases (N is typically very small in decoding stage)

  // 16B alignment lets us use TMA
  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      MmaTileMNK, ClusterShapeMNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      EpilogueScheduleType,
      cutlass::epilogue::fusion::LinearCombination<ElementC, ElementAccumulator>
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, AlignmentA,
      ElementB, LayoutB, AlignmentB,
      ElementAccumulator,
      MmaTileMNK, ClusterShapeMNK,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopScheduleType
    >::CollectiveOp;

  using ProblemShapeGroup = cutlass::gemm::GroupProblemShape<Shape<int,int,int>>; // <M,N,K> per group
  using ProblemShapeMax = Shape<int,int,int,int>; // max <M,N,K,L>
  using ProblemShape = cutlass::gemm::MoEProblemShape<ProblemShapeGroup, ProblemShapeMax>;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      ProblemShape,
      CollectiveMainloop,
      CollectiveEpilogue
      //, cutlass::gemm::MoEScheduler
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  // using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutTagA = cutlass::gemm::detail::StrideToLayoutTagA_t<StrideA>;
  using LayoutTagB = cutlass::gemm::detail::StrideToLayoutTagB_t<StrideB>;
  using LayoutTagC = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideC>;
  using LayoutTagD = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideD>;

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_D;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_ref_D;

  cutlass::DeviceAllocation<typename ProblemShapeGroup::UnderlyingProblemShape> problem_sizes;


  //
  // Methods
  //

  bool verify(ProblemShape const& problem_size, float alpha, float beta) {
    auto [maxM, maxN, maxK, L] = problem_size.max_problem_shape;
    for (int i = 0; i < problem_size.problem_shape.num_groups; i++) {
      auto problem = problem_size.problem_shape.get_host_problem_shape(i);
      auto [M, N, K] = problem;

      cutlass::TensorRef ref_A(block_A.get() + size_t(1) * i * maxM * maxK, Gemm::LayoutA(maxK));
      cutlass::TensorRef ref_B(block_B.get() + size_t(1) * i * maxN * maxK, Gemm::LayoutB(maxK));
      cutlass::TensorRef ref_C(block_C.get() + size_t(1) * i * maxN * maxM, Gemm::LayoutC(maxM));
      cutlass::TensorRef ref_D(block_ref_D.get() + size_t(1) * i * maxN * maxM, Gemm::LayoutD(maxM));

      using DeviceGemmReference = cutlass::reference::device::Gemm<
        ElementA,
        LayoutA,
        ElementB,
        LayoutB,
        ElementC,
        LayoutC,
        ElementScalar,
        ElementAccumulator>;

      DeviceGemmReference gemm_reference;

      gemm_reference(
        {M, N, K},
        ElementScalar(alpha),
        ref_A,
        ref_B,
        ElementScalar(beta),
        ref_C,
        ref_D);

      hipError_t result = hipDeviceSynchronize();
      if (result != hipSuccess) {
        std::cerr << "Reference kernel failed. Last CUDA error: "
                  << hipGetErrorString(result) << std::endl;
        return false;
      }

      // Check if output from CUTLASS kernel and reference kernel are equal or not
      // assume all M == maxM
      bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get() + size_t(1) * i * maxN * maxM, block_D.get() + size_t(1) * i * maxN * maxM, M * N);
      if (!passed) {
        return false;
      }
    }

    return true;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(ProblemShape const& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size.max_problem_shape, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(size_t(1) * M * K * L);
    block_B.reset(size_t(1) * K * N * L);
    block_C.reset(size_t(1) * M * N * L);
    block_D.reset(size_t(1) * M * N * L);
    block_ref_D.reset(size_t(1) * M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  /// Load a benchmark
  std::vector<ProblemShapeGroup::UnderlyingProblemShape> benchmark_problems(std::string const& benchmark_path) {
    std::vector<ProblemShapeGroup::UnderlyingProblemShape> problem_sizes_host;

    std::ifstream file(benchmark_path);
    if (!file.good()) {
      return {};
    }

    while (file.good()) {

      int idx = -1;
      std::string extent_str;

      file >> idx >> extent_str;

      if (idx < 0 || extent_str.empty()) {
        break;
      }

      cutlass::gemm::GemmCoord extent;
      std::vector<std::string> tokens;

      cutlass::CommandLine::tokenize(tokens, extent_str, 'x');

      for (int i = 0; i < int(tokens.size()); ++i) {
        extent.at(i) = std::atoi(tokens.at(i).c_str());
      }
      problem_sizes_host.push_back({extent.m(), extent.n(), extent.k()});
    }

    return problem_sizes_host;
  }

  bool run(Options const& options, cutlass::KernelHardwareInfo const& hw_info) {
    auto problem_sizes_host = benchmark_problems(options.benchmark_path);
    if (problem_sizes_host.empty()) {
      return false;
    }

    problem_sizes.reset(problem_sizes_host.size());
    problem_sizes.copy_from_host(problem_sizes_host.data());

    ProblemShape problem_size;
    problem_size.max_problem_shape = ProblemShapeMax{options.m, options.n, options.k, options.l};
    problem_size.problem_shape.num_groups = problem_sizes_host.size();
    problem_size.problem_shape.problem_shapes = problem_sizes.get();
    problem_size.problem_shape.host_problem_shapes = problem_sizes_host.data();

    initialize(problem_size);

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGrouped,
      problem_size,
      {block_A.get(), stride_A, block_B.get(), stride_B},
      {{}, // epilogue.thread
       block_C.get(), stride_C, block_D.get(), stride_D},
      hw_info
    };

    // arguments.scheduler.max_swizzle_size = options.swizzle;
    
    arguments.epilogue.thread.alpha = 1.0f;
    arguments.epilogue.thread.beta = 0.0f;

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    // Run the GEMM
    status = gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    if (options.verification) {
      // Verify that the result is correct
      bool passed = verify(problem_size, 1.0f, 0.0f);

      std::cout << "  Disposition: " << (passed ? "Passed" : "Failed") << std::endl;

      if (!passed) {
        exit(-1);
        return false;
      }
    }

    // Run profiling loop
    if (options.iterations > 0)
    {
      GpuTimer timer;
      timer.start();
      for (int iter = 0; iter < options.iterations; ++iter) {
        CUTLASS_CHECK(gemm_op.initialize(arguments, workspace.get()));
        CUTLASS_CHECK(gemm_op.run());
      }
      timer.stop();

      // Compute average setup and runtime and FLOPs.
      float elapsed_ms       = timer.elapsed_millis();
      double avg_runtime_ms  = double(elapsed_ms) / double(options.iterations);
      double flops           = double(int64_t(2) * options.m * options.n * options.k * options.l) / (avg_runtime_ms / 1000.0);

      std::cout << "  Avg runtime : " << avg_runtime_ms << " ms" << std::endl;
      std::cout << "  TFLOPS      : " << flops / 1e12 << std::endl;
    }

    return true;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)) {
    std::cerr << "This example requires CUDA 12.8 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
  
  if (!(props.major == 10 && props.minor == 0)) {
    std::cerr << "This example requires a GPU of NVIDIA's Blackwell architecture (compute capability 100)." << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  std::cout << "Running kernel with mixed TMA+CPASYNC load:" << std::endl;
  ExampleRunner runner_mixed_tma_cpasync;
  runner_mixed_tma_cpasync.run(options, hw_info);

#endif

  return 0;
}
