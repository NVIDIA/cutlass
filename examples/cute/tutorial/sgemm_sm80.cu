#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

template <class ProblemShape, class CtaTiler,
          class TA, class AStride, class ASmemLayout, class TiledCopyA,
          class TB, class BStride, class BSmemLayout, class TiledCopyB,
          class TC, class CStride, class CSmemLayout, class TiledMma,
          class Alpha, class Beta>
__global__ static
__launch_bounds__(decltype(size(TiledMma{}))::value)
void
gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
            TA const* A, AStride dA, ASmemLayout sA_layout, TiledCopyA copy_a,
            TB const* B, BStride dB, BSmemLayout sB_layout, TiledCopyB copy_b,
            TC      * C, CStride dC, CSmemLayout          , TiledMma mma,
            Alpha alpha, Beta beta)
{
  using namespace cute;

  // Preconditions
  CUTE_STATIC_ASSERT_V(rank(shape_MNK) == Int<3>{});                   // (M, N, K)
  CUTE_STATIC_ASSERT_V(rank(cta_tiler) == Int<3>{});                   // (BLK_M, BLK_N, BLK_K)

  CUTE_STATIC_ASSERT_V(size(copy_a) == size(mma));                     // NumThreads
  CUTE_STATIC_ASSERT_V(size(copy_b) == size(mma));                     // NumThreads

  static_assert(is_static<ASmemLayout>::value);
  static_assert(is_static<BSmemLayout>::value);
  static_assert(is_static<CSmemLayout>::value);

  CUTE_STATIC_ASSERT_V(size<0>(ASmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(BSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(ASmemLayout{}) == size<2>(cta_tiler));  // BLK_K
  CUTE_STATIC_ASSERT_V(size<1>(BSmemLayout{}) == size<2>(cta_tiler));  // BLK_K

  CUTE_STATIC_ASSERT_V(congruent(select<0,2>(shape_MNK), dA));         // dA strides for shape MK
  CUTE_STATIC_ASSERT_V(congruent(select<1,2>(shape_MNK), dB));         // dB strides for shape NK
  CUTE_STATIC_ASSERT_V(congruent(select<0,1>(shape_MNK), dC));         // dC strides for shape MN

  //
  // Full and Tiled Tensors
  //

  // Represent the full tensors
  Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // (M,K)
  Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // (N,K)
  Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // (M,N)

  // Get the appropriate blocks for this thread block
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);              // (m,n,k)
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  // Shared memory buffers
  __shared__ TA smemA[cosize_v<ASmemLayout>];
  __shared__ TB smemB[cosize_v<BSmemLayout>];
  Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout);            // (BLK_M,BLK_K,PIPE)
  Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout);            // (BLK_N,BLK_K,PIPE)

  //
  // Partition the copying of A and B tiles across the threads
  //

  ThrCopy thr_copy_a = copy_a.get_slice(threadIdx.x);
  Tensor tAgA = thr_copy_a.partition_S(gA);                            // (CPY,CPY_M,CPY_K,k)
  Tensor tAsA = thr_copy_a.partition_D(sA);                            // (CPY,CPY_M,CPY_K,PIPE)

  ThrCopy thr_copy_b = copy_b.get_slice(threadIdx.x);
  Tensor tBgB = thr_copy_b.partition_S(gB);                            // (CPY,CPY_N,CPY_K,k)
  Tensor tBsB = thr_copy_b.partition_D(sB);                            // (CPY,CPY_N,CPY_K,PIPE)

  CUTE_STATIC_ASSERT_V(size<1>(tAgA) == size<1>(tAsA));                // CPY_M
  CUTE_STATIC_ASSERT_V(size<2>(tAgA) == size<2>(tAsA));                // CPY_K
  CUTE_STATIC_ASSERT_V(size<1>(tBgB) == size<1>(tBsB));                // CPY_N
  CUTE_STATIC_ASSERT_V(size<2>(tBgB) == size<2>(tBsB));                // CPY_K

  //
  // PREFETCH
  //

  auto K_PIPE_MAX = size<3>(tAsA);

  // Total count of tiles
  int k_tile_count = size<3>(tAgA);
  // Current tile index in gmem to read from
  int k_tile_next = 0;

  // Start async loads for all pipes but the last
  CUTE_UNROLL
  for (int k_pipe = 0; k_pipe < K_PIPE_MAX-1; ++k_pipe) {
    copy(copy_a, tAgA(_,_,_,k_tile_next), tAsA(_,_,_,k_pipe));
    copy(copy_b, tBgB(_,_,_,k_tile_next), tBsB(_,_,_,k_pipe));
    cp_async_fence();
    --k_tile_count;
    if (k_tile_count > 0) { ++k_tile_next; }
  }

  //
  // Define A/B partitioning and C accumulators
  //

  ThrMMA thr_mma = mma.get_slice(threadIdx.x);
  Tensor tCsA = thr_mma.partition_A(sA);                               // (MMA,MMA_M,MMA_K,PIPE)
  Tensor tCsB = thr_mma.partition_B(sB);                               // (MMA,MMA_N,MMA_K,PIPE)
  Tensor tCgC = thr_mma.partition_C(gC);                               // (MMA,MMA_M,MMA_N)

  // Allocate registers for pipelining
  Tensor tCrA = thr_mma.make_fragment_A(tCsA(_,_,_,0));                // (MMA,MMA_M,MMA_K)
  Tensor tCrB = thr_mma.make_fragment_B(tCsB(_,_,_,0));                // (MMA,MMA_N,MMA_K)
  // Allocate the accumulators -- same size as the projected data
  Tensor tCrC = thr_mma.make_fragment_C(tCgC);                         // (MMA,MMA_M,MMA_N)

  CUTE_STATIC_ASSERT_V(  shape(tCrA) ==   shape(tCsA));                // (MMA,MMA_M,MMA_K)
  CUTE_STATIC_ASSERT_V(  shape(tCrB) ==   shape(tCsB));                // (MMA,MMA_N,MMA_K)
  CUTE_STATIC_ASSERT_V(  shape(tCrC) ==   shape(tCgC));                // (MMA,MMA_M,MMA_N)
  CUTE_STATIC_ASSERT_V(size<1>(tCgC) == size<1>(tCsA));                // MMA_M
  CUTE_STATIC_ASSERT_V(size<2>(tCgC) == size<1>(tCsB));                // MMA_N
  CUTE_STATIC_ASSERT_V(size<2>(tCsA) == size<2>(tCsB));                // MMA_K

  // Clear the accumulators
  clear(tCrC);

#if 0
  if(thread0()) {
    print("  mA : "); print(  mA); print("\n");
    print("  gA : "); print(  gA); print("\n");
    print("  sA : "); print(  sA); print("\n");
    print("tAgA : "); print(tAgA); print("\n");
    print("tAsA : "); print(tAsA); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mB : "); print(  mB); print("\n");
    print("  gB : "); print(  gB); print("\n");
    print("  sB : "); print(  sB); print("\n");
    print("tBgB : "); print(tBgB); print("\n");
    print("tBsB : "); print(tBsB); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mC : "); print(  mC); print("\n");
    print("  gC : "); print(  gC); print("\n");
    print("tCsA : "); print(tCsA); print("\n");
    print("tCsB : "); print(tCsB); print("\n");
    print("tCgC : "); print(tCgC); print("\n");
    print("tCrA : "); print(tCrA); print("\n");
    print("tCrB : "); print(tCrB); print("\n");
    print("tCrC : "); print(tCrC); print("\n");
  }
#endif

#if 1

  // Current pipe index in smem to read from
  int smem_pipe_read  = 0;
  // Current pipe index in smem to write to
  int smem_pipe_write = K_PIPE_MAX-1;

  // Pipe slice
  Tensor tCsA_p = tCsA(_,_,_,smem_pipe_read);
  Tensor tCsB_p = tCsB(_,_,_,smem_pipe_read);

  // Size of the register pipeline
  auto K_BLOCK_MAX = size<2>(tCrA);

  // PREFETCH register pipeline
  if (K_BLOCK_MAX > 1) {
    // Wait until our first prefetched tile is loaded in
    cp_async_wait<K_PIPE_MAX-2>();
    __syncthreads();

    // Prefetch the first rmem from the first k-tile
    copy(tCsA_p(_,_,Int<0>{}), tCrA(_,_,Int<0>{}));
    copy(tCsB_p(_,_,Int<0>{}), tCrB(_,_,Int<0>{}));
  }

  //
  // PIPELINED MAIN LOOP
  // TUTORIAL: Example of a gemm loop that pipelines shared memory using SM80's cp.async instructions
  //           and explicit pipelines in shared memory.
  //   Data is read from global(k_tile_next) to shared(smem_pipe_write).
  //   Data is read from shared(smem_pipe_read) to registers(k_block_next).
  //   Data is computed on registers(b_block).
  //
  //   This allows all copies and compute to overlap:
  //     Copy from gmem->smem can overlap with copies from smem->rmem and compute on rmem.
  //     Copy from smem->rmem can overlap with compute on rmem.
  //

  CUTE_NO_UNROLL
  while (k_tile_count > -(K_PIPE_MAX-1))
  {
    CUTE_UNROLL
    for (int k_block = 0; k_block < K_BLOCK_MAX; ++k_block)
    {
      if (k_block == K_BLOCK_MAX - 1)
      {
        // Slice the smem_pipe_read smem
        tCsA_p = tCsA(_,_,_,smem_pipe_read);
        tCsB_p = tCsB(_,_,_,smem_pipe_read);

        // Commit the smem for smem_pipe_read
        cp_async_wait<K_PIPE_MAX-2>();
        __syncthreads();
      }

      // Load A, B shmem->regs for k_block+1
      auto k_block_next = (k_block + Int<1>{}) % K_BLOCK_MAX;      // static
      copy(tCsA_p(_,_,k_block_next), tCrA(_,_,k_block_next));
      copy(tCsB_p(_,_,k_block_next), tCrB(_,_,k_block_next));
      // Copy gmem to smem before computing gemm on each k-pipe
      if (k_block == 0)
      {
        copy(copy_a, tAgA(_,_,_,k_tile_next), tAsA(_,_,_,smem_pipe_write));
        copy(copy_b, tBgB(_,_,_,k_tile_next), tBsB(_,_,_,smem_pipe_write));
        cp_async_fence();

        // Advance the gmem tile
        --k_tile_count;
        if (k_tile_count > 0) { ++k_tile_next; }

        // Advance the smem pipe
        smem_pipe_write = smem_pipe_read;
        ++smem_pipe_read;
        smem_pipe_read = (smem_pipe_read == K_PIPE_MAX) ? 0 : smem_pipe_read;
      }
      // Thread-level register gemm for k_block
      gemm(mma, tCrA(_,_,k_block), tCrB(_,_,k_block), tCrC);
    }

  }

#endif

  //
  // Epilogue
  //

  axpby(alpha, tCrC, beta, tCgC);
}

// Setup params for a NT GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_nt(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define NT strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<3>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA = make_layout(make_shape(bM, bK, bP));             // (m,k,p) -> smem_idx; m-major
  auto sB = make_layout(make_shape(bN, bK, bP));             // (n,k,p) -> smem_idx; n-major
  auto sC = make_layout(make_shape(bM, bN));                 // (m,n) -> smem_idx; m-major

  // Define the thread layouts (static)

  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TA>{},
                                    Layout<Shape<_32,_8>>{}, // Thr layout 32x8 m-major
                                    Layout<Shape< _4,_1>>{});// Val layout  4x1 m-major
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TB>{},
                                    Layout<Shape<_32,_8>>{}, // Thr layout 32x8 n-major
                                    Layout<Shape< _4,_1>>{});// Val layout  4x1 n-major

  TiledMMA mmaC = make_tiled_mma(UniversalFMA<TC,TA,TB>{},
                                 Layout<Shape<_16,_16,_1>>{});  // 16x16x1 TiledMMA

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  dim3 dimBlock(size(mmaC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, copyA,
       B, dB, sB, copyB,
       C, dC, sC, mmaC,
       alpha, beta);
}

// Setup params for a NT GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_tn(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(ldA, Int<1>{});                      // (dM, dK)
  auto dB = make_stride(ldB, Int<1>{});                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<3>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA_atom = make_layout(make_shape (      bM,          bK),
                             make_stride(Int<1>{}, bM+Int<1>{}));   // (m,k) -> smem_idx; padded m-major
  auto sB_atom = make_layout(make_shape (      bN,          bK),
                             make_stride(Int<1>{}, bN+Int<1>{}));   // (n,k) -> smem_idx; padded n-major
  auto sA = tile_to_shape(sA_atom, make_shape(bM, bK, bP));
  auto sB = tile_to_shape(sA_atom, make_shape(bN, bK, bP));
  auto sC = make_layout(make_shape(bM, bN));                        // (m,n) -> smem_idx

  // Define the thread layouts (static)

  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<TA>, TA>{},
                                    Layout<Shape<_32,_8>,Stride<_8,_1>>{}, // Thr layout 32x8 k-major
                                    Layout<Shape< _1,_1>>{});              // Val layout  1x1
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<TB>, TB>{},
                                    Layout<Shape<_32,_8>,Stride<_8,_1>>{}, // Thr layout 32x8 k-major
                                    Layout<Shape< _1,_1>>{});              // Val layout  1x1

  TiledMMA mmaC = make_tiled_mma(UniversalFMA<TC,TA,TB>{},
                                 Layout<Shape<_16,_16,_1>>{});  // 16x16x1 TiledMMA

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  dim3 dimBlock(size(mmaC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, copyA,
       B, dB, sB, copyB,
       C, dC, sC, mmaC,
       alpha, beta);
}

template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm(char transA, char transB, int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  if (transA == 'N' && transB == 'T') {
    return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  } else
  if (transA == 'T' && transB == 'N') {
    return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  }
  assert(false && "Not implemented");
}


int main(int argc, char** argv)
{
  hipDeviceProp_t props;
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major < 8) {
    std::cout << "This example requires an Ampere GPU or newer (CC >= 80)" << std::endl;
    // Return 0 so tests pass if run on unsupported architectures or CUDA Toolkits.
    return 0;
  }

  int m = 5120;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 5120;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 4096;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'N';
  if (argc >= 5)
    sscanf(argv[4], "%c", &transA);

  char transB = 'T';
  if (argc >= 6)
    sscanf(argv[5], "%c", &transB);

  using TA = float;
  using TB = float;
  using TC = float;
  using TI = float;

  TI alpha = 1.0;
  TI beta  = 0.0;

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;
  std::cout << "C = A^" << transA << " B^" << transB << std::endl;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 100;
  GPU_Clock timer;

  int ldA = 0, ldB = 0, ldC = m;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }

  // Run once
  d_C = h_C;
  gemm(transA, transB, m, n, k,
       alpha,
       d_A.data().get(), ldA,
       d_B.data().get(), ldB,
       beta,
       d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(transA, transB, m, n, k,
         alpha,
         d_A.data().get(), ldA,
         d_B.data().get(), ldB,
         beta,
         d_C.data().get(), ldC);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

  return 0;
}
