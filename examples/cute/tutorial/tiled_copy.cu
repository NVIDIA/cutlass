#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

// This is a simple tutorial showing several ways to partition a tensor into tiles then
// perform efficient, coalesced copies. This example also shows how to vectorize accesses
// which may be a useful optimization or required for certain workloads.
//
// `copy_kernel()` and `copy_kernel_vectorized()` each assume a pair of tensors with
// dimensions (m, n) have been partitioned via `tiled_divide()`.
//
// The result are a part of compatible tensors with dimensions ((M, N), m', n'), where
// (M, N) denotes a statically sized tile, and m' and n' denote the number of such tiles
// within the tensor.
//
// Each statically sized tile is mapped to a CUDA threadblock which performs efficient
// loads and stores to Global Memory.
//
// `copy_kernel()` uses `cute::local_partition()` to partition the tensor and map
// the result to threads using a striped indexing scheme. Threads themselve are arranged
// in a (ThreadShape_M, ThreadShape_N) arrangement which is replicated over the tile.
//
// `copy_kernel_vectorized()` uses `cute::make_tiled_copy()` to perform a similar
// partitioning using `cute::Copy_Atom` to perform vectorization. The actual vector
// size is defined by `ThreadShape`.
//
// This example assumes the overall tensor shape is divisible by the tile size and
// does not perform predication.


/// Simple copy kernel.
//
// Uses local_partition() to partition a tile among threads arranged as (THR_M, THR_N).
template <class TensorS, class TensorD, class ThreadLayout>
__global__ void copy_kernel(TensorS S, TensorD D, ThreadLayout) 
{
  using namespace cute;

  // Slice the tiled tensors
  Tensor tile_S = S(make_coord(_,_), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)
  Tensor tile_D = D(make_coord(_,_), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)

  // Construct a partitioning of the tile among threads with the given thread arrangement.

  // Concept:                       Tensor    Layout          Index
  Tensor thr_tile_S = local_partition(tile_S, ThreadLayout{}, threadIdx.x);
  Tensor thr_tile_D = local_partition(tile_D, ThreadLayout{}, threadIdx.x);

  // Construct a register-backed Tensor with the same shape as each thread's partition
  auto fragment = make_fragment_like(thr_tile_S);

  // Copy from GMEM to RMEM and from RMEM to GMEM
  copy(thr_tile_S, fragment);
  copy(fragment, thr_tile_D);
}

/// Vectorized copy kernel.
///
/// Uses `make_tiled_copy()` to perform a copy using vector instructions. This operation
/// has the precondition that pointers are aligned to the vector size.
///
template <class TensorS, class TensorD, class ThreadLayout, class VecLayout>
__global__ void copy_kernel_vectorized(TensorS S, TensorD D, ThreadLayout, VecLayout) 
{
  using namespace cute;
  using Element = typename TensorS::value_type;

  // Slice the tensors to obtain a view into each tile.
  Tensor tile_S = S(make_coord(_, _), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)
  Tensor tile_D = D(make_coord(_, _), blockIdx.x, blockIdx.y);   // (BlockShape_M, BlockShape_N)

  // Define `AccessType` which controls the size of the actual memory access.
  using AccessType = cutlass::AlignedArray<Element, size(shape(VecLayout{}))>;

  // A copy atom corresponds to one hardware memory access.
  using Atom = Copy_Atom<UniversalCopy<AccessType>, Element>;

  // Construct tiled copy, a tiling of copy atoms.
  //
  // Note, this assumes the vector and thread layouts are aligned with contigous data
  // in GMEM. Alternative thread layouts are possible but may result in uncoalesced
  // reads. Alternative vector layouts are also possible, though incompatible layouts
  // will result in compile time errors.
  auto tiled_copy =
    make_tiled_copy(
      Atom{},                       // access size
      ThreadLayout{},               // thread layout
      VecLayout{});                 // vector layout (e.g. 4x1)

  // Construct a Tensor corresponding to each thread's slice.
  auto thr_copy = tiled_copy.get_thread_slice(threadIdx.x);

  Tensor thr_tile_S = thr_copy.partition_S(tile_S);
  Tensor thr_tile_D = thr_copy.partition_D(tile_D);

  // Construct a register-backed Tensor with the same shape as each thread's partition
  auto fragment = make_fragment_like(thr_tile_D);

  // Copy from GMEM to RMEM and from RMEM to GMEM
  copy(tiled_copy, thr_tile_S, fragment);
  copy(tiled_copy, fragment, thr_tile_D);
}

/// Helper to convert a shape to a dim3
template <class Shape>
dim3 shape_to_dim3(Shape shape)
{
  using namespace cute;

  CUTE_STATIC_ASSERT_V(rank(shape) <= Int<3>{});
  auto result = append<3>(product_each(shape), 1u);

  return dim3(get<0>(result), get<1>(result), get<2>(result));
}

/// Main function
int main(int argc, char** argv)
{
  //
  // Given a 2D shape, perform an efficient copy
  //

  using namespace cute;
  using Element = float;

  // Define a tensor shape with dynamic extents (m, n)
  auto tensor_shape = make_shape(256, 512);

  thrust::host_vector<Element> h_S(size(tensor_shape));
  thrust::host_vector<Element> h_D(size(tensor_shape));

  //
  // Initialize
  //

  for (size_t i = 0; i < h_S.size(); ++i) {
    h_S[i] = static_cast<Element>(i);
    h_D[i] = Element{};
  }

  thrust::device_vector<Element> d_S = h_S;
  thrust::device_vector<Element> d_D = h_D;

  //
  // Make tensors
  //

  Tensor tensor_S = make_tensor(make_gmem_ptr(d_S.data().get()), make_layout(tensor_shape));  
  Tensor tensor_D = make_tensor(make_gmem_ptr(d_D.data().get()), make_layout(tensor_shape));

  //
  // Partition
  //


  // Define a statically sized block (M, N).
  //
  // Note, by convention, capital letters are used to represent static modes.
  auto block_shape = make_shape(Int<128>{}, Int<64>{});

  if ((get<0>(tensor_shape) % get<0>(block_shape)) || (get<1>(tensor_shape) % get<1>(block_shape))) {
    std::cerr << "The tensor shape must be divisible by the block shape." << std::endl;
    return -1;
  }

  // Tile the tensor (m, m) ==> ((M, N), m', n') where (M, N) is the static tile
  // shape, and modes (m', n') correspond to the number of tiles.
  // 
  // These will be used to determine the CUDA kernel grid dimensinos.
  Tensor tiled_tensor_S = tiled_divide(tensor_S, block_shape);
  Tensor tiled_tensor_D = tiled_divide(tensor_D, block_shape);

  // Thread arrangement
  Layout thr_layout = make_layout(make_shape(Int<32>{}, Int< 8>{}));

  // Vector dimensions
  Layout vec_layout = make_layout(make_shape(Int<4>{}, Int<1>{}));

  //
  // Determine grid and block dimensions
  //

  dim3 gridDim = shape_to_dim3(select<1,2>(shape(tiled_tensor_D))); // Grid shape corresponds to  modes m' and n'
  dim3 blockDim(size(shape(thr_layout)));

  //
  // Launch the kernel
  //
  copy_kernel_vectorized<<< gridDim, blockDim >>>(
    tiled_tensor_S, 
    tiled_tensor_D, 
    thr_layout, 
    vec_layout);

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    std::cerr << "CUDA Runtime error: " << hipGetErrorString(result) << std::endl;
    return -1;
  }

  //
  // Verify
  //

  h_D = d_D;

  int32_t errors = 0;
  int32_t const kErrorLimit = 10;

  for (size_t i = 0; i < h_D.size(); ++i) {
    if (h_S[i] != h_D[i]) {
      std::cerr << "Error. S[" << i << "]: " << h_S[i] << ",   D[" << i << "]: " << h_D[i] << std::endl;

      if (++errors >= kErrorLimit) {
        std::cerr << "Aborting on " << kErrorLimit << "nth error." << std::endl;
        return -1;
      }
    }
  }

  std::cout << "Success." << std::endl;

  return 0;
}

