#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/cluster_launch.hpp"

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

using namespace cute;

template <class ElementA,
          class ElementB,
          class SmemLayoutA,  // (M,K,P)
          class SmemLayoutB>  // (N,K,P)
struct SharedStorage
{
  alignas(128) cute::ArrayEngine<ElementA, cosize_v<SmemLayoutA>> A;
  alignas(128) cute::ArrayEngine<ElementB, cosize_v<SmemLayoutB>> B;
};

template <class ProblemShape, class CtaTiler,
          class TA, class AStride, class ASmemLayout, class TiledCopyA,
          class TB, class BStride, class BSmemLayout, class TiledCopyB,
          class TC, class CStride, class TiledMma,
          class Alpha, class Beta>
__global__ static
__launch_bounds__(decltype(size(TiledMma{}))::value)
void
gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
            TA const* A, AStride dA, ASmemLayout sA_layout, TiledCopyA copy_a,
            TB const* B, BStride dB, BSmemLayout sB_layout, TiledCopyB copy_b,
            TC      * C, CStride dC, TiledMma mma,
            Alpha alpha, Beta beta)
{
  // Preconditions
  CUTE_STATIC_ASSERT_V(rank(shape_MNK) == Int<3>{});                   // (M, N, K)
  CUTE_STATIC_ASSERT_V(rank(cta_tiler) == Int<3>{});                   // (BLK_M, BLK_N, BLK_K)

  CUTE_STATIC_ASSERT_V(size(copy_a) == size(mma));                     // NumThreads
  CUTE_STATIC_ASSERT_V(size(copy_b) == size(mma));                     // NumThreads

  static_assert(is_static<ASmemLayout>::value);
  static_assert(is_static<BSmemLayout>::value);

  CUTE_STATIC_ASSERT_V(size<0>(ASmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(BSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(ASmemLayout{}) == size<2>(cta_tiler));  // BLK_K
  CUTE_STATIC_ASSERT_V(size<1>(BSmemLayout{}) == size<2>(cta_tiler));  // BLK_K

  CUTE_STATIC_ASSERT_V(congruent(select<0,2>(shape_MNK), dA));         // dA strides for shape MK
  CUTE_STATIC_ASSERT_V(congruent(select<1,2>(shape_MNK), dB));         // dB strides for shape NK
  CUTE_STATIC_ASSERT_V(congruent(select<0,1>(shape_MNK), dC));         // dC strides for shape MN

  //
  // Full and Tiled Tensors
  //

  // Represent the full tensors
  Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // (M,K)
  Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // (N,K)
  Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // (M,N)

  // Get the appropriate blocks for this thread block
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);              // (m,n,k)
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  // Shared memory tensors
  extern __shared__ char shared_memory[];
  using SharedStorage = SharedStorage<TA, TB, ASmemLayout, BSmemLayout>;
  SharedStorage& smem = *reinterpret_cast<SharedStorage*>(shared_memory);
  Tensor sA = make_tensor(make_smem_ptr(smem.A.begin()), ASmemLayout{}); // (BLK_M,BLK_K,PIPE)
  Tensor sB = make_tensor(make_smem_ptr(smem.B.begin()), BSmemLayout{}); // (BLK_N,BLK_K,PIPE)

  //
  // Partition the copying of A and B tiles across the threads
  //

  ThrCopy thr_copy_a = copy_a.get_slice(threadIdx.x);
  Tensor tAgA = thr_copy_a.partition_S(gA);                            // (CPY,CPY_M,CPY_K,k)
  Tensor sA_ = as_position_independent_swizzle_tensor(sA);
  Tensor tAsA = thr_copy_a.partition_D(sA_);                           // (CPY,CPY_M,CPY_K,PIPE)

  ThrCopy thr_copy_b = copy_b.get_slice(threadIdx.x);
  Tensor tBgB = thr_copy_b.partition_S(gB);                            // (CPY,CPY_N,CPY_K,k)
  Tensor sB_ = as_position_independent_swizzle_tensor(sB);
  Tensor tBsB = thr_copy_b.partition_D(sB_);                           // (CPY,CPY_N,CPY_K,PIPE)

  CUTE_STATIC_ASSERT_V(size<1>(tAgA) == size<1>(tAsA));                // CPY_M
  CUTE_STATIC_ASSERT_V(size<2>(tAgA) == size<2>(tAsA));                // CPY_K
  CUTE_STATIC_ASSERT_V(size<1>(tBgB) == size<1>(tBsB));                // CPY_N
  CUTE_STATIC_ASSERT_V(size<2>(tBgB) == size<2>(tBsB));                // CPY_K

  //
  // PREFETCH
  //

  // auto K_PIPE_MAX = size<3>(tAsA);

  // // Total count of tiles
  // int k_tile_count = size<3>(tAgA);
  // // Current tile index in gmem to read from
  // int k_tile_next = 0;

  // // Start async loads for all pipes but the last
  // CUTE_UNROLL
  // for (int k_pipe = 0; k_pipe < K_PIPE_MAX-1; ++k_pipe) {
  //   copy(copy_a, tAgA(_,_,_,k_tile_next), tAsA(_,_,_,k_pipe));
  //   copy(copy_b, tBgB(_,_,_,k_tile_next), tBsB(_,_,_,k_pipe));
  //   cp_async_fence();
  //   --k_tile_count;
  //   if (k_tile_count > 0) { ++k_tile_next; }
  // }

  //
  // Define A/B partitioning and C accumulators
  //

  ThrMMA thr_mma = mma.get_slice(threadIdx.x);
  Tensor tCsA = thr_mma.partition_A(sA);                               // (MMA,MMA_M,MMA_K,PIPE)
  Tensor tCsB = thr_mma.partition_B(sB);                               // (MMA,MMA_N,MMA_K,PIPE)
  Tensor tCgC = thr_mma.partition_C(gC);                               // (MMA,MMA_M,MMA_N)

  // Allocate registers for pipelining
  Tensor tCrA = thr_mma.make_fragment_A(tCsA);                         // (MMA,MMA_M,MMA_K,PIPE)
  Tensor tCrB = thr_mma.make_fragment_B(tCsB);                         // (MMA,MMA_N,MMA_K,PIPE)
  // Allocate the accumulators -- same size as the projected data
  Tensor tCrC = thr_mma.make_fragment_C(tCgC);                         // (MMA,MMA_M,MMA_N)

  CUTE_STATIC_ASSERT_V((size<1>(tCgC) == size<1>(tCsA)));              // MMA_M
  CUTE_STATIC_ASSERT_V((size<2>(tCgC) == size<1>(tCsB)));              // MMA_N
  CUTE_STATIC_ASSERT_V((size<2>(tCsA) == size<2>(tCsB)));              // MMA_K

  // Clear the accumulators
  clear(tCrC);

#if 0
  if(thread0()) {
    print("  mA : "); print(  mA); print("\n");
    print("  gA : "); print(  gA); print("\n");
    print("  sA : "); print(  sA); print("\n");
    print("tAgA : "); print(tAgA); print("\n");
    print("tAsA : "); print(tAsA); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mB : "); print(  mB); print("\n");
    print("  gB : "); print(  gB); print("\n");
    print("  sB : "); print(  sB); print("\n");
    print("tBgB : "); print(tBgB); print("\n");
    print("tBsB : "); print(tBsB); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mC : "); print(  mC); print("\n");
    print("  gC : "); print(  gC); print("\n");
    print("tCsA : "); print(tCsA); print("\n");
    print("tCsB : "); print(tCsB); print("\n");
    print("tCgC : "); print(tCgC); print("\n");
    print("tCrA : "); print(tCrA); print("\n");
    print("tCrB : "); print(tCrB); print("\n");
    print("tCrC : "); print(tCrC); print("\n");
  }
#endif

#if 1

  // Total number of k-tiles
  auto K_TILE_MAX  = size<3>(tAgA);
  // Number of pipelined k-tiles in smem
  auto K_PIPE_MAX  = size<3>(tAsA);

  //
  // PREFETCH
  //

  // Prefetch all but the last
  CUTE_UNROLL
  for (int k = 0; k < K_PIPE_MAX-1; ++k)
  {
    copy(copy_a, tAgA(_,_,_,k), tAsA(_,_,_,k));
    copy(copy_b, tBgB(_,_,_,k), tBsB(_,_,_,k));
    cp_async_fence();
  }

  // Clear the accumulators
  clear(tCrC);

  __syncthreads();

  //
  // PIPELINED MAIN LOOP
  //

  // Current pipe to read from
  int k_pipe_read  = 0;
  // Current pipe to write to
  int k_pipe_write = K_PIPE_MAX-1;

  CUTE_NO_UNROLL
  for (int k_tile = 0; k_tile < K_TILE_MAX; ++k_tile)
  {
    int k_tile_next = k_tile + (K_PIPE_MAX-1);
    k_tile_next = (k_tile_next >= K_TILE_MAX) ? K_TILE_MAX-1 : k_tile_next;

    //
    // Copy gmem to smem for k_tile_write
    //

    copy(copy_a, tAgA(_,_,_,k_tile_next), tAsA(_,_,_,k_pipe_write));
    copy(copy_b, tBgB(_,_,_,k_tile_next), tBsB(_,_,_,k_pipe_write));
    cp_async_fence();

    // Advance k_pipe_write
    ++k_pipe_write;
    k_pipe_write = (k_pipe_write == K_PIPE_MAX) ? 0 : k_pipe_write;

    //
    // Compute on k_tile
    //

    // Wait on all cp.async -- optimize by pipelining to overlap GMEM reads
    cp_async_wait<0>();

    warpgroup_fence_operand(tCrC);
    warpgroup_arrive();
    // (V,M,K) x (V,N,K) => (V,M,N)
    cute::gemm(mma, tCrA(_,_,_,k_pipe_read), tCrB(_,_,_,k_pipe_read), tCrC);
    warpgroup_commit_batch();
    /// Wait on the GMMA barrier for K_PIPE_MMAS (or fewer) outstanding to ensure smem_pipe_write is consumed
    warpgroup_wait<0>();
    warpgroup_fence_operand(tCrC);

    // Advance k_pipe_read
    ++k_pipe_read;
    k_pipe_read = (k_pipe_read == K_PIPE_MAX) ? 0 : k_pipe_read;
  }

#endif

  //
  // Epilogue
  //

  axpby(alpha, tCrC, beta, tCgC);
}

// Setup params for a NT GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_nt(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define NT strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 64>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<3>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA = tile_to_shape(GMMA::Layout_MN_SW128_Atom<TA>{}, make_shape(bM,bK,bP));
  auto sB = tile_to_shape(GMMA::Layout_MN_SW128_Atom<TB>{}, make_shape(bN,bK,bP));

  // Define the thread layouts (static)
  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TA>{},
                                    Layout<Shape<_16,_8>>{}, // Thr layout 32x4 m-major
                                    Layout<Shape< _8,_1>>{});// Val layout  8x1 m-major
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TB>{},
                                    Layout<Shape<_16,_8>>{}, // Thr layout 32x4 n-major
                                    Layout<Shape< _8,_1>>{});// Val layout  8x1 n-major

  TiledMMA tiled_mma = make_tiled_mma(SM90_64x64x16_F16F16F16_SS<GMMA::Major::MN,GMMA::Major::MN>{});

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  //
  // Setup and Launch
  //

  // Launch parameter setup
  dim3 dimBlock(size(tiled_mma));
  dim3 dimCluster(1, 1, 1);
  dim3 dimGrid(round_up(size(ceil_div(m, bM)), dimCluster.x),
               round_up(size(ceil_div(n, bN)), dimCluster.y));
  int  smemBytes = sizeof(SharedStorage<TA, TB, decltype(sA), decltype(sB)>);

  auto* kernel_ptr = &gemm_device<decltype(prob_shape), decltype(cta_tiler),
                                  TA, decltype(dA), decltype(sA), decltype(copyA),
                                  TB, decltype(dB), decltype(sB), decltype(copyB),
                                  TC, decltype(dC), decltype(tiled_mma),
                                  decltype(alpha), decltype(beta)>;

  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_ptr),
                                        hipFuncAttributeMaxDynamicSharedMemorySize,
                                        smemBytes));

  // Kernel Launch
  cutlass::ClusterLaunchParams params = {dimGrid, dimBlock, dimCluster, smemBytes};
  cutlass::Status status = cutlass::launch_kernel_on_cluster(params, (void const*) kernel_ptr,
                                                             prob_shape, cta_tiler,
                                                             A, dA, sA, copyA,
                                                             B, dB, sB, copyB,
                                                             C, dC, tiled_mma,
                                                             alpha, beta);
  CUTE_CHECK_LAST();

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }
}

// Setup params for a TN GEMM
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_tn(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(ldA, Int<1>{});                      // (dM, dK)
  auto dB = make_stride(ldB, Int<1>{});                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 64>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<3>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA = tile_to_shape(GMMA::Layout_K_SW128_Atom<TA>{}, make_shape(bM,bK,bP));
  auto sB = tile_to_shape(GMMA::Layout_K_SW128_Atom<TB>{}, make_shape(bN,bK,bP));

  // Define the thread layouts (static)
  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TA>{},
                                    Layout<Shape<_16,_8>,Stride<_8,_1>>{}, // Thr layout 16x8 k-major
                                    Layout<Shape< _1,_8>>{});              // Val layout  1x8
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TB>{},
                                    Layout<Shape<_16,_8>,Stride<_8,_1>>{}, // Thr layout 16x8 k-major
                                    Layout<Shape< _1,_8>>{});              // Val layout  1x8

  TiledMMA tiled_mma = make_tiled_mma(SM90_64x64x16_F16F16F16_SS<GMMA::Major::K,GMMA::Major::K>{});

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  //
  // Setup and Launch
  //

  // Launch parameter setup
  int smem_size = int(sizeof(SharedStorage<TA, TB, decltype(sA), decltype(sB)>));
  dim3 dimBlock(size(tiled_mma));
  dim3 dimCluster(1, 1, 1);
  dim3 dimGrid(round_up(size(ceil_div(m, bM)), dimCluster.x),
               round_up(size(ceil_div(n, bN)), dimCluster.y));
  cutlass::ClusterLaunchParams params = {dimGrid, dimBlock, dimCluster, smem_size};

  void const* kernel_ptr = reinterpret_cast<void const*>(
                              &gemm_device<decltype(prob_shape), decltype(cta_tiler),
                                           TA, decltype(dA), decltype(sA), decltype(copyA),
                                           TB, decltype(dB), decltype(sB), decltype(copyB),
                                           TC, decltype(dC), decltype(tiled_mma),
                                           decltype(alpha), decltype(beta)>);

  CUTE_CHECK_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
    kernel_ptr),
    hipFuncAttributeMaxDynamicSharedMemorySize,
    smem_size));

  // Kernel Launch
  cutlass::Status status = cutlass::launch_kernel_on_cluster(params, kernel_ptr,
                                                             prob_shape, cta_tiler,
                                                             A, dA, sA, copyA,
                                                             B, dB, sB, copyB,
                                                             C, dC, tiled_mma,
                                                             alpha, beta);
  CUTE_CHECK_LAST();

  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Error: Failed at kernel Launch" << std::endl;
  }
}

template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm(char transA, char transB, int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  if (transA == 'N' && transB == 'T') {
    return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  } else
  if (transA == 'T' && transB == 'N') {
    return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  }
  assert(false && "Not implemented");
}


int main(int argc, char** argv)
{
  hipDeviceProp_t props;
  int current_device_id;
  hipGetDevice(&current_device_id);
  hipGetDeviceProperties(&props, current_device_id);
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major != 9) {
    std::cout << "This example requires NVIDIA's Hopper Architecture GPU with compute capability 90a" << std::endl;
    // Return 0 so tests pass if run on unsupported architectures or CUDA Toolkits.
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  int m = 5120;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 5120;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 4096;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'N';
  if (argc >= 5)
    sscanf(argv[4], "%c", &transA);

  char transB = 'T';
  if (argc >= 6)
    sscanf(argv[5], "%c", &transB);

  using TA = cute::half_t;
  using TB = cute::half_t;
  using TC = cute::half_t;
  using TI = cute::half_t;

  TI alpha = TI(1.0f);
  TI beta  = TI(0.0f);

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  // Initialize the tensors
  for (int j = 0; j < m*k; ++j) h_A[j] = TA(int((rand() % 2) ? 1 : -1));
  for (int j = 0; j < n*k; ++j) h_B[j] = TB(int((rand() % 2) ? 1 : -1));
  for (int j = 0; j < m*n; ++j) h_C[j] = TC(0);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 100;
  GPU_Clock timer;

  int ldA = 0, ldB = 0, ldC = m;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }

  // Run once
  d_C = h_C;
  gemm(transA, transB, m, n, k,
       alpha,
       d_A.data().get(), ldA,
       d_B.data().get(), ldB,
       beta,
       d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(transA, transB, m, n, k,
         alpha,
         d_A.data().get(), ldA,
         d_B.data().get(), ldB,
         beta,
         d_C.data().get(), ldC);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

#else
  std::cout << "CUTLASS_ARCH_MMA_SM90_SUPPORTED must be enabled, but it is not. Test is waived \n" << std::endl;
#endif

  return 0;
}
