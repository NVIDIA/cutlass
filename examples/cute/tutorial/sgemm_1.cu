#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

template <class ProblemShape, class CtaTiler,
          class TA, class AStride, class ASmemLayout, class AThreadLayout,
          class TB, class BStride, class BSmemLayout, class BThreadLayout,
          class TC, class CStride, class CSmemLayout, class CThreadLayout,
          class Alpha, class Beta>
__global__ static
__launch_bounds__(decltype(size(CThreadLayout{}))::value)
void
gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
            TA const* A, AStride dA, ASmemLayout sA_layout, AThreadLayout tA,
            TB const* B, BStride dB, BSmemLayout sB_layout, BThreadLayout tB,
            TC      * C, CStride dC, CSmemLayout          , CThreadLayout tC,
            Alpha alpha, Beta beta)
{
  using namespace cute;

  // Preconditions
  CUTE_STATIC_ASSERT_V(rank(shape_MNK) == Int<3>{});                   // (M, N, K)
  CUTE_STATIC_ASSERT_V(rank(cta_tiler) == Int<3>{});                   // (BLK_M, BLK_N, BLK_K)

  static_assert(is_static<AThreadLayout>::value);
  static_assert(is_static<BThreadLayout>::value);
  static_assert(is_static<CThreadLayout>::value);

  CUTE_STATIC_ASSERT_V(size(tA) == size(tB));                          // NumThreads
  CUTE_STATIC_ASSERT_V(size(tC) == size(tA));                          // NumThreads

  CUTE_STATIC_ASSERT_V(size<0>(cta_tiler) % size<0>(tA) == Int<0>{});  // BLK_M / THR_M
  CUTE_STATIC_ASSERT_V(size<2>(cta_tiler) % size<1>(tA) == Int<0>{});  // BLK_K / THR_K
  CUTE_STATIC_ASSERT_V(size<1>(cta_tiler) % size<0>(tB) == Int<0>{});  // BLK_N / THR_N
  CUTE_STATIC_ASSERT_V(size<2>(cta_tiler) % size<1>(tB) == Int<0>{});  // BLK_K / THR_K
  CUTE_STATIC_ASSERT_V(size<0>(cta_tiler) % size<0>(tC) == Int<0>{});  // BLK_M / THR_M
  CUTE_STATIC_ASSERT_V(size<1>(cta_tiler) % size<1>(tC) == Int<0>{});  // BLK_N / THR_N

  static_assert(is_static<ASmemLayout>::value);
  static_assert(is_static<BSmemLayout>::value);
  static_assert(is_static<CSmemLayout>::value);

  CUTE_STATIC_ASSERT_V(size<0>(ASmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(BSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(ASmemLayout{}) == size<2>(cta_tiler));  // BLK_K
  CUTE_STATIC_ASSERT_V(size<1>(BSmemLayout{}) == size<2>(cta_tiler));  // BLK_K

  CUTE_STATIC_ASSERT_V(congruent(select<0,2>(shape_MNK), dA));         // dA strides for shape MK
  CUTE_STATIC_ASSERT_V(congruent(select<1,2>(shape_MNK), dB));         // dB strides for shape NK
  CUTE_STATIC_ASSERT_V(congruent(select<0,1>(shape_MNK), dC));         // dC strides for shape MN

  //
  // Full and Tiled Tensors
  //

  // Represent the full tensors
  Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // (M,K)
  Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // (N,K)
  Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // (M,N)

  // Get the appropriate blocks for this thread block
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);              // (m,n,k)
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  // Shared memory buffers
  __shared__ TA smemA[cosize_v<ASmemLayout>];
  __shared__ TB smemB[cosize_v<BSmemLayout>];
  Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout);            // (BLK_M,BLK_K)
  Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout);            // (BLK_N,BLK_K)

  //
  // Partition the copying of A and B tiles across the threads
  //

  // TUTORIAL: Example of simple raked partitioning of ThreadLayouts tA|tB over data A|B tiles

  Tensor tAgA = local_partition(gA, tA, threadIdx.x);                  // (THR_M,THR_K,k)
  Tensor tAsA = local_partition(sA, tA, threadIdx.x);                  // (THR_M,THR_K)

  Tensor tBgB = local_partition(gB, tB, threadIdx.x);                  // (THR_N,THR_K,k)
  Tensor tBsB = local_partition(sB, tB, threadIdx.x);                  // (THR_N,THR_K)

  CUTE_STATIC_ASSERT_V(size<0>(tAgA) == size<0>(tAsA));                // THR_M
  CUTE_STATIC_ASSERT_V(size<1>(tAgA) == size<1>(tAsA));                // THR_K
  CUTE_STATIC_ASSERT_V(size<0>(tBgB) == size<0>(tBsB));                // THR_N
  CUTE_STATIC_ASSERT_V(size<1>(tBgB) == size<1>(tBsB));                // THR_K

  //
  // Define A/B partitioning and C accumulators
  //

  // TUTORIAL: Example of partitioning via projections of a ThreadLayout tC

  // Partition sA (M,K) by the rows of tC
  Tensor tCsA = local_partition(sA, tC, threadIdx.x, Step<_1, X>{});   // (THR_M,BLK_K)
  // Partition sB (N,K) by the cols of tC
  Tensor tCsB = local_partition(sB, tC, threadIdx.x, Step< X,_1>{});   // (THR_N,BLK_K)
  // Partition gC (M,N) by the tile of tC
  Tensor tCgC = local_partition(gC, tC, threadIdx.x, Step<_1,_1>{});   // (THR_M,THR_N)

  // Allocate the accumulators -- same shape/layout as the partitioned data
  Tensor tCrC = make_tensor_like(tCgC);                                // (THR_M,THR_N)

  CUTE_STATIC_ASSERT_V(size<0>(tCrC) == size<0>(tCgC));                // THR_M
  CUTE_STATIC_ASSERT_V(size<0>(tCrC) == size<0>(tCsA));                // THR_M
  CUTE_STATIC_ASSERT_V(size<1>(tCrC) == size<1>(tCgC));                // THR_N
  CUTE_STATIC_ASSERT_V(size<1>(tCrC) == size<0>(tCsB));                // THR_N
  CUTE_STATIC_ASSERT_V(size<1>(tCsA) == size<1>(tCsB));                // BLK_K

  // Clear the accumulators
  clear(tCrC);

#if 0
  if(thread0()) {
    print("  mA : "); print(  mA); print("\n");
    print("  gA : "); print(  gA); print("\n");
    print("  sA : "); print(  sA); print("\n");
    print("tAgA : "); print(tAgA); print("\n");
    print("tAsA : "); print(tAsA); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mB : "); print(  mB); print("\n");
    print("  gB : "); print(  gB); print("\n");
    print("  sB : "); print(  sB); print("\n");
    print("tBgB : "); print(tBgB); print("\n");
    print("tBsB : "); print(tBsB); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mC : "); print(  mC); print("\n");
    print("  gC : "); print(  gC); print("\n");
    print("tCsA : "); print(tCsA); print("\n");
    print("tCsB : "); print(tCsB); print("\n");
    print("tCgC : "); print(tCgC); print("\n");
    print("tCrC : "); print(tCrC); print("\n");
  }
#endif

#if 1

  // TUTORIAL: Example of a simple mainloop that read tiles of data into shared memory,
  //           and then computes on those tiles.
  //   copy(.) operates on the global and shared memory via the tA|tB partitioning
  //   gemm(.) operates on the shared and register memory via the tC partitioning

  auto K_TILE_MAX = size<2>(tAgA);

  for (int k_tile = 0; k_tile < K_TILE_MAX; ++k_tile)
  {
    // Copy gmem to smem with tA|tB thread-partitioned tensors
    copy(tAgA(_,_,k_tile), tAsA);      // A   (THR_M,THR_K) -> (THR_M,THR_K)
    copy(tBgB(_,_,k_tile), tBsB);      // B   (THR_N,THR_K) -> (THR_N,THR_K)

    // TUTORIAL: The above call to copy(tAgA(_,_,k_tile), tAsA) is equivalent to
    //   Tensor tAgAk = tAgA(_,_,k_tile);
    //   CUTE_UNROLL
    //   for (int i = 0; i < size(tAsA); ++i) {
    //     tAsA(i) = tAgAk(i);
    //   }

    cp_async_fence();        // Label the end of (potential) cp.async instructions
    cp_async_wait<0>();      // Sync on all (potential) cp.async instructions
    __syncthreads();         // Wait for all threads to write to smem

    // Compute gemm on tC thread-partitioned smem
    gemm(tCsA, tCsB, tCrC);            // (THR_M,THR_N) += (THR_M,BLK_K) * (THR_N,BLK_K)

    // TUTORIAL: The above call to gemm(tCsA, tCsB, tCrC) is equivalent to
    //   CUTE_UNROLL
    //   for (int k = 0; k < size<1>(tCsA); ++k) {
    //     CUTE_UNROLL
    //     for (int m = 0; m < size<0>(tCrC); ++m) {
    //       CUTE_UNROLL
    //       for (int n = 0; n < size<1>(tCrC); ++n) {
    //         tCrC(m,n) += tCsA(m,k) * tCsB(n,k);
    //       }
    //     }
    //   }

    __syncthreads();         // Wait for all threads to read from smem
  }

#endif

  //
  // Epilogue
  //

  axpby(alpha, tCrC, beta, tCgC);

  // TUTORIAL: The above call to axpby(alpha, tCrC, beta, tCgC) is equivalent to
  //   CUTE_UNROLL
  //   for (int i = 0; i < size(tCsA); ++i) {
  //     tCgC(i) = alpha * tCrC(i) + beta * tCgC(i);
  //   }
}

// Setup params for an NT GEMM
// Use m-major smem sA, n-major smem sB, and mn-major threads tA|tB
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_nt(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define NT strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)

  // Define the smem layouts (static)
  auto sA = make_layout(make_shape(bM, bK));                 // (m,k) -> smem_idx; m-major
  auto sB = make_layout(make_shape(bN, bK));                 // (n,k) -> smem_idx; n-major
  auto sC = make_layout(make_shape(bM, bN));                 // (m,n) -> smem_idx; m-major

  // Define the thread layouts (static)
  auto tA = make_layout(make_shape(Int<32>{}, Int< 8>{}));   // (m,k) -> thr_idx
  auto tB = make_layout(make_shape(Int<32>{}, Int< 8>{}));   // (n,k) -> thr_idx
  auto tC = make_layout(make_shape(Int<16>{}, Int<16>{}));   // (m,n) -> thr_idx

  dim3 dimBlock(size(tC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, tA,
       B, dB, sB, tB,
       C, dC, sC, tC,
       alpha, beta);
}

// Setup params for a TN GEMM
// Use padded m-major smem sA, padded n-major smem sB, and k-major threads tA|tB
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_tn(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(ldA, Int<1>{});                      // (dM, dK)
  auto dB = make_stride(ldB, Int<1>{});                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)

  // Define the smem layouts (static)
  auto sA = make_layout(make_shape(bM,bK), LayoutRight{});   // (m,k) -> smem_idx; k-major
  auto sB = make_layout(make_shape(bN,bK), LayoutRight{});   // (n,k) -> smem_idx; k-major
  auto sC = make_layout(make_shape(bM, bN));                 // (m,n) -> smem_idx; m-major

  // Define the thread layouts (static)
  auto tA = make_layout(make_shape(Int<32>{}, Int< 8>{}), LayoutRight{});  // (m,k) -> thr_idx; k-major
  auto tB = make_layout(make_shape(Int<32>{}, Int< 8>{}), LayoutRight{});  // (n,k) -> thr_idx; k-major
  auto tC = make_layout(make_shape(Int<16>{}, Int<16>{}));                 // (m,n) -> thr_idx; m-major

  dim3 dimBlock(size(tC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, tA,
       B, dB, sB, tB,
       C, dC, sC, tC,
       alpha, beta);
}

template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm(char transA, char transB, int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  if (transA == 'N' && transB == 'T') {
    return gemm_nt(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  } else
  if (transA == 'T' && transB == 'N') {
    return gemm_tn(m, n, k, alpha, A, ldA, B, ldB, beta, C, ldC, stream);
  }
  assert(false && "Not implemented");
}


int main(int argc, char** argv)
{
  int m = 5120;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 5120;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 4096;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'N';
  if (argc >= 5)
    sscanf(argv[4], "%c", &transA);

  char transB = 'T';
  if (argc >= 6)
    sscanf(argv[5], "%c", &transB);

  using TA = float;
  using TB = float;
  using TC = float;
  using TI = float;

  TI alpha = 1.0;
  TI beta  = 0.0;

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;
  std::cout << "C = A^" << transA << " B^" << transB << std::endl;

  cute::device_init(0);

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 100;
  GPU_Clock timer;

  int ldA = 0, ldB = 0, ldC = m;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }
  // Run once
  d_C = h_C;
  gemm(transA, transB, m, n, k,
       alpha,
       d_A.data().get(), ldA,
       d_B.data().get(), ldB,
       beta,
       d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(transA, transB, m, n, k,
         alpha,
         d_A.data().get(), ldA,
         d_B.data().get(), ldB,
         beta,
         d_C.data().get(), ldC);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);
  return 0;
}
