/***************************************************************************************************
 * Copyright (c) 2024 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper Sparse GEMM example.

  This example demonstrates how to construct and run a structured sparse GEMM kernel
  on NVIDIA Hopper architecture.
    
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/transform/device/transform_universal_adapter.hpp"
#include "cutlass/transform/kernel/sparse_gemm_compressor.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::half_t;                                // Element type for A matrix operand
using         LayoutTagA  = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::half_t;                                // Element type for B matrix operand
using         LayoutTagB  = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;                                          // Element type for C and D matrix operands
using         LayoutTagC  = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using TileShape           = Shape<_128,_128,_128>;                          // Threadblock-level tile size for sparse kernel
using TileShapeRef        = Shape<_128,_128, _64>;                          // Threadblock-level tile size for reference (dense) kernel
using ClusterShape        = Shape<_1,_2,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::KernelTmaWarpSpecialized;        // Kernel schedule policy
using EpilogueSchedule    = cutlass::epilogue::TmaWarpSpecialized;          // Epilogue schedule policy

using ProblemShape = Shape<int,int,int,int>;

// Sparse kernel setup

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutTagC, AlignmentC,
    ElementC, LayoutTagC, AlignmentC,
    EpilogueSchedule
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
    ElementA, LayoutTagA, AlignmentA,
    ElementB, LayoutTagB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference (dense) kernel setup

using CollectiveEpilogueRef = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShapeRef, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutTagC, AlignmentC,
    ElementC, LayoutTagC, AlignmentC,
    EpilogueSchedule
  >::CollectiveOp;

using CollectiveMainloopRef = typename cutlass::gemm::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    ElementA, LayoutTagA, AlignmentA,
    ElementB, LayoutTagB, AlignmentB,
    ElementAccumulator,
    TileShapeRef, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelRef = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloopRef,
    CollectiveEpilogue
>;

using GemmRef = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelRef>;

// Layouts 
using LayoutA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutA;
using LayoutE = typename Gemm::GemmKernel::CollectiveMainloop::LayoutE;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

// Layouts for reference (non-sparse) tensors
using StrideA = cutlass::gemm::TagToStrideA_t<LayoutTagA>;
using StrideE = StrideA;

using ElementE = typename Gemm::GemmKernel::CollectiveMainloop::ElementE;
using SparseConfig = typename Gemm::GemmKernel::CollectiveMainloop::SparseConfig;

// Offline compressor kernel
using CompressorUtility = cutlass::transform::kernel::StructuredSparseCompressorUtility<
                            ProblemShape,
                            ElementA,
                            LayoutTagA,
                            SparseConfig>;

using CompressorKernel = cutlass::transform::kernel::StructuredSparseCompressor<
                            ProblemShape,
                            ElementA,
                            LayoutTagA,
                            SparseConfig,
                            cutlass::arch::Sm90>;

using Compressor = cutlass::transform::device::TransformUniversalAdapter<CompressorKernel>;

//
// Data members
//

ProblemShape problem_shape;

StrideA stride_A;
StrideA stride_A_compressed;
StrideE stride_E;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;

LayoutA layout_A;
LayoutE layout_E;

uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementA> block_A_compressed;
cutlass::DeviceAllocation<typename Gemm::CollectiveMainloop::ElementE> block_E;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D_ref;

#endif // defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k, l;

  Options():
    help(false),
    m(5120), n(4096), k(16384), l(1),
    alpha(1.f), beta(0.f),
    iterations(10)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "62_hopper_sparse_gemm\n\n"
      << "  Hopper Sparse GEMM example.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent of the GEMM (batch size)\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "62_hopper_sparse_gemm" << " --m=4096 --n=5120 --k=8192 --l=1 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

#if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = Element(2);
    scope_min = Element(0);
  } else if (bits_input <= 8) {
    scope_max = Element(2);
    scope_min = Element(-2);
  } else {
    scope_max = Element(8);
    scope_min = Element(-8);
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Make A structured sparse by replacing elements with 0 and compress it
bool sparsify_and_compress()
{
  auto [M, N, K, L] = problem_shape;
  CompressorUtility compressor_utility(problem_shape, stride_A);

  int ME = compressor_utility.get_metadata_m_physical();
  int KE = compressor_utility.get_metadata_k_physical();
  int KC = compressor_utility.get_tensorA_k_physical();

  block_A_compressed.reset(M * KC * L);
  block_E.reset(ME * KE * L);

  stride_A_compressed = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, KC, L));
  stride_E = cutlass::make_cute_packed_stride(StrideE{}, cute::make_shape(ME, KE, L));

  // Random sparsification is performed on host
  std::vector<ElementA> block_A_host(block_A.size());
  cutlass::device_memory::copy_to_host(block_A_host.data(), block_A.get(), block_A.size());
  compressor_utility.structure_sparse_zero_mask_fill(block_A_host.data(), static_cast<int>(seed + 2024));
  cutlass::device_memory::copy_to_device(block_A.get(), block_A_host.data(), block_A.size());

  cutlass::KernelHardwareInfo hw_info;
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  typename Compressor::Arguments arguments {
    problem_shape,
    { block_A.get(),
      stride_A,
      block_A_compressed.get(),
      block_E.get() },
    {hw_info} };

  Compressor compressor_op;
  size_t workspace_size = Compressor::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  CUTLASS_CHECK(compressor_op.can_implement(arguments));
  CUTLASS_CHECK(compressor_op.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(compressor_op.run());
  CUDA_CHECK(hipDeviceSynchronize());

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
bool initialize(Options const& options) {

  problem_shape = make_tuple(options.m, options.n, options.k, options.l);
  auto [M, N, K, L] = problem_shape;

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

  // Allocate memory for tensors
  block_A.reset(M * K * L);
  block_B.reset(N * K * L);
  block_C.reset(M * N * L);
  block_D.reset(M * N * L);
  block_D_ref.reset(M * N * L);

  // Fill input tensors with data
  initialize_block(block_A, seed + 2021);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2023);

  // Replace 0 in A with 1 to avoid metadata changes
  std::vector<ElementA> block_A_host(block_A.size());
  cutlass::device_memory::copy_to_host(block_A_host.data(), block_A.get(), block_A.size());
  for (size_t i = 0; i < block_A.size(); ++i) if (block_A_host[i] == ElementA(0)) block_A_host[i] = ElementA(1.0);
  cutlass::device_memory::copy_to_device(block_A.get(), block_A_host.data(), block_A.size());

  if (!sparsify_and_compress()) {
    return false;
  };

  // Build the compressed/metadata layouts
  layout_A = SparseConfig::fill_layoutA(problem_shape);
  layout_E = SparseConfig::fill_layoutE(problem_shape);

  return true;
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments make_args(Options const& options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    problem_shape,
    { block_A_compressed.get(), layout_A, block_B.get(), stride_B, block_E.get(), layout_E },
    { { ElementAccumulator(options.alpha), ElementAccumulator(options.beta) },
      block_C.get(), stride_C, block_D.get(), stride_D }
  };

  return arguments;
}

typename GemmRef::Arguments make_args_ref(Options const& options)
{
  typename GemmRef::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    problem_shape,
    { block_A.get(), stride_A, block_B.get(), stride_B },
    { { ElementAccumulator(options.alpha), ElementAccumulator(options.beta) },
      block_C.get(), stride_C, block_D_ref.get(), stride_D }
  };

  return arguments;
}

template<class Engine, class Layout>
void print_device_tensor(cute::Tensor<Engine, Layout> const& t)
{
  // Assumes size = cosize, i.e. compact tensor
  std::vector<typename Engine::value_type> data_host(t.size());
  cutlass::device_memory::copy_to_host(data_host.data(), t.data(), t.size());
  auto t_host = cute::make_tensor(data_host.data(), t.layout());
  cute::print_tensor(t_host);
}

bool verify(Options const& options) {
  CUDA_CHECK(hipDeviceSynchronize());

  bool passed = cutlass::reference::device::BlockCompareEqual(block_D_ref.get(), block_D.get(), block_D.size());

#if 0
  if (!passed) {
    auto [M, N, K, L] = problem_shape;
    CompressorUtility compressor_utility(problem_shape, stride_A);
    int ME = compressor_utility.get_metadata_m_physical();
    int KE = compressor_utility.get_metadata_k_physical();
    int KC = compressor_utility.get_tensorA_k_physical();

    cute::print("A (original): "); print_device_tensor(make_tensor(block_A.get(), make_shape(M, K, L), stride_A));
    cute::print("A (compressed): "); print_device_tensor(make_tensor(block_A_compressed.get(), make_shape(M, KC, L), stride_A_compressed));
    cute::print("E (physical): "); print_device_tensor(make_tensor(block_E.get(), make_shape(ME, KE, L), stride_E));
    cute::print("E (logical): "); print_device_tensor(make_tensor(block_E.get(), upcast<CollectiveMainloop::ElementEMmaSparsity>(layout_E)));
    cute::print("B: "); print_device_tensor(make_tensor(block_B.get(), make_shape(N, K, L), stride_B));
    cute::print("C: "); print_device_tensor(make_tensor(block_C.get(), make_shape(M, N, L), stride_C));
    cute::print("D reference: "); print_device_tensor(make_tensor(block_D_ref.get(), make_shape(M, N, L), stride_D));
    cute::print("D  computed: "); print_device_tensor(make_tensor(block_D.get(), make_shape(M, N, L), stride_D));
  }
#endif

  return passed;
}

template<typename Gemm>
struct Runner
{
  using Arguments = typename Gemm::Arguments;

  Runner(Arguments args): arguments(args) {
    // Using the arguments, query for extra workspace required for matrix multiplication computation
    size_t workspace_size = Gemm::get_workspace_size(arguments);

    // Allocate workspace memory
    workspace.reset(workspace_size);

    // Check if the problem size is supported or not
    CUTLASS_CHECK(gemm.can_implement(arguments));
  }

  void run() {
    CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
    CUTLASS_CHECK(gemm.run());
  }

  void benchmark(Options const& options) {
    if (options.iterations > 0)
    {
      GpuTimer timer;
      timer.start();
      for (int iter = 0; iter < options.iterations; ++iter) {
        run();
      }
      timer.stop();

      // Compute average runtime and GFLOPs.
      float elapsed_ms = timer.elapsed_millis();
      double avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
      double gflops = options.gflops(avg_runtime_ms / 1000.0);

      std::cout << "  Avg runtime: " << avg_runtime_ms << " ms" << std::endl;
      std::cout << "  GFLOPS: " << gflops << std::endl;
    }
  }

  Gemm gemm;
  Arguments arguments;
  cutlass::device_memory::allocation<uint8_t> workspace;
};

/// Execute the example (verification and timing)
void run(Options &options) {
  bool init = initialize(options);
  if (!init) {
    std::cout << "Initialization failure" << std::endl;
    exit(EXIT_FAILURE);
  }

  Runner<Gemm> gemm(make_args(options));
  Runner<GemmRef> gemm_ref(make_args_ref(options));

  gemm.run();
  gemm_ref.run();

  bool passed = verify(options);

  std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
  std::cout << "  Disposition: " << (passed ? "Passed" : "Failed") << std::endl;

  if (!passed) {
    exit(EXIT_FAILURE);
  }

  std::cout << "Sparse GEMM:" << std::endl;
  gemm.benchmark(options);

  std::cout << "Dense GEMM:" << std::endl;
  gemm_ref.benchmark(options);
}

#endif // defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.2 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 2)) {
    std::cerr << "This example requires CUDA 12.2 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }
  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)
  run(options);
#endif

  return EXIT_SUCCESS;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
