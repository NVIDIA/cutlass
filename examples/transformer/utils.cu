#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "utils.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <stdbool.h>

#define CEIL_DIV(m,n) ( (m) + (n) - 1 ) / (n)
void print_matrix(const FLOAT *A, int m, int n){
    int i;
    printf("[");
    for (i = 0; i < m * n; i++){
        if ((i + 1) % n == 0) printf("%5.2f ",__half2float(A[i]));
        else printf("%5.2f, ", __half2float(A[i]));
        if ((i + 1) % n == 0){
            if (i + 1 < m * n) printf(";\n");
        }
    }
    printf("]\n");
}

void print_matrix(const float *A, int m, int n){
    int i;
    printf("[");
    for (i = 0; i < m * n; i++){
        if ((i + 1) % n == 0) printf("%5.2f ",(A[i]));
        else printf("%5.2f, ", (A[i]));
        if ((i + 1) % n == 0){
            if (i + 1 < m * n) printf(";\n");
        }
    }
    printf("]\n");
}


void randomize_matrix(FLOAT* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        FLOAT tmp = (rand() % 2 == 0) ? (FLOAT)(rand() % 5) : (FLOAT)((rand() % 5) * -1);
        // FLOAT tmp = i ;
        mat[i] = tmp;
    }
}

void randomize_matrix_plus_one(FLOAT* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        FLOAT tmp = (rand() % 2 == 0) ? (FLOAT)((rand() % 5) + 1) : (FLOAT)(((rand() % 5) + 1) * -1);
        // FLOAT tmp = i + 1;
        mat[i] = tmp;
    }
}

void randomize_matrix_plus_one(float* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        float tmp = (rand() % 2 == 0) ? (float)((rand() % 5) + 1) : (float)(((rand() % 5) + 1) * -1);
        // FLOAT tmp = i + 1;
        mat[i] = tmp;
    }
}

void randomize_matrix_zero(float* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        // FLOAT tmp = (rand() % 2 == 0) ? (FLOAT)(rand() % 5) : (FLOAT)((rand() % 5) * -1);
        float tmp = 0;
        mat[i] = tmp;
    }
}

void randomize_matrix_zero(FLOAT* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        // FLOAT tmp = (rand() % 2 == 0) ? (FLOAT)(rand() % 5) : (FLOAT)((rand() % 5) * -1);
        FLOAT tmp = 0;
        mat[i] = tmp;
    }
}

void randomize_matrix_one(FLOAT* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        FLOAT tmp = 1;
        mat[i] = tmp;
    }
}

void randomize_matrix_one(float* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        float tmp = 1;
        mat[i] = tmp;
    }
}

void randomize_matrix(float* mat, int N){
    srand(time(NULL)); int i;
    for (i = 0; i < N; i++) {
        float tmp = (rand() % 2 == 0) ? (float)(rand() % 5) : (float)((rand() % 5) * -1);
        // float tmp = i;
        mat[i] = tmp;
    }
}

double get_sec(){
    struct timeval time;
    gettimeofday(&time, NULL); 
    return (time.tv_sec + 1e-6 * time.tv_usec);
}

bool verify_matrix(FLOAT *mat1, FLOAT *mat2, int n){
    double diff = 0.0;
    int i;
    int err_cnt = 0;
    for (i = 0; mat1 + i && mat2 + i && i < n; i++){
        diff = fabs( (double)mat1[i] - (double)mat2[i] ) / fabs((double)mat1[i]);
        if (diff > 1e-4) {
            if (err_cnt < 5) printf("error. %5.2f,%5.2f,%d\n", (double)mat1[i],(double)mat2[i],i);
            err_cnt++;
        }
    }
    return (err_cnt == 0);
}

bool verify_matrix(float *mat1, float *mat2, int n){
    double diff = 0.0;
    int i;
    int err_cnt = 0;
    for (i = 0; mat1 + i && mat2 + i && i < n; i++){
        diff = fabs( (double)mat1[i] - (double)mat2[i] ) / fabs((double)mat1[i]);
        if (diff > 5e-3) {
            if (err_cnt < 5) 
                printf("error. %8.5f,%8.5f,%d\n", (double)mat1[i],(double)mat2[i],i);
            err_cnt++;
        }
    }
    return (err_cnt == 0);
}

void copy_matrix(FLOAT *src, FLOAT *dest, int n){
    int i;
    for (i = 0; src + i && dest + i && i < n; i++) *(dest + i) = *(src + i);
    if (i != n) printf("copy failed at %d while there are %d elements in total.\n", i, n);
}
