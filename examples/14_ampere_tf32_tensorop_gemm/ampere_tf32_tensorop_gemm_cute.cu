/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * Copyright (c) 2024 - 2024 Codeplay Software Ltd. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Simple GEMM example using Cute and CUTLASS 3.x APIs for NVIDIA Ampere architecture

    This example demonstrate how to instantiate and run a TF32 GEMM using the Cute and
    CUTLASS 3.x APIs on NVIDIA Ampere architecture. Please check example 07 and 08 for
    the basics of tensor op gemm kernels. On NVIDIA Ampere architecture, most concept
    still holds. The two main differences are:

      (1) NVIDIA Ampere architecture introduces a new series of tensor core instructions
          (see include/cute/arch/mma_sm80.hpp) which are more efficient on Ampere.
      (2) NVIDIA Ampere architecture uses CP_ASYNC (see include/cute/arch/copy_sm80.hpp)
          to build a multistage software pipeline to better hide latency (see
          include/cutlass/gemm/collective/sm80_mma_multistage.hpp).

    Moreover, NVIDIA Ampere architecture starts supporting tfloat32 (see include/cutlass/tfloat32.h)
    data types in tensor cores.  One big advantage is that we can load in fp32 data and convert
    them implicitly to tf32 inside the GEMM kernel which means no change is needed to accelerate
    traditional fp32 data by using NVIDIA Ampere architecture.

    Examples:

      $ ./examples/14_ampere_tf32_tensorop_gemm/14_ampere_tf32_tensorop_gemm_cute

*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/collective/collective_mma.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

using namespace cute;

/// Result structure
struct Result {

  double avg_runtime_ms;
  double gflops;
  bool passed;

  //
  // Methods
  //

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), passed(false)
  {}
};

///////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  int m, n, k, l;
  float alpha, beta;
  int iterations;

  Options():
    help(false),
    m(5120), n(4096), k(4096), l(1),
    alpha(1), beta(0),
    iterations(100)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 5120);
    cmd.get_cmd_line_argument("n", n, 4096);
    cmd.get_cmd_line_argument("k", k, 4096);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);

  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "14_ampere_tf32_tensorop_gemm_cute example\n\n"
      << "  This example uses the CUTLASS Library to execute TF32 tensorop GEMM computations.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k * l;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
        cutlass::DeviceAllocation<Element>& block,
        uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
          block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Wrapper to run and verify a GEMM.
template <
        class Gemm
>
struct ExampleRunner {

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutA = typename Gemm::LayoutA;
  using LayoutB = typename Gemm::LayoutB;
  using LayoutC = typename Gemm::LayoutC;
  using LayoutD = typename Gemm::LayoutD;

  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementAcc = typename Gemm::ElementAccumulator;

  using CollectiveEpilogue = typename Gemm::CollectiveEpilogue;
  using ElementC = typename Gemm::ElementC;
  using ElementOutput = typename CollectiveEpilogue::ElementOutput;
  using ElementCompute = typename CollectiveEpilogue::ElementCompute;
  using ElementAccumulator = typename CollectiveEpilogue::ElementAccumulator;

  using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<ElementA> block_A;
  cutlass::DeviceAllocation<ElementB> block_B;
  cutlass::DeviceAllocation<ElementC> block_C;
  cutlass::DeviceAllocation<ElementOutput> block_D;
  cutlass::DeviceAllocation<ElementOutput> block_ref_D;

  //
  // Methods
  //

  bool verify(const ProblemShapeType& problem_size, ElementOutput alpha, ElementOutput beta) {
    auto [M, N, K, L] = problem_size;

    cutlass::TensorRef ref_A(block_A.get(), LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get(), LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get(), LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get(), LayoutD::packed({M, N}));

    cutlass::reference::device::GemmComplex(
            {M, N, K},
            ElementCompute(alpha),
            ref_A,
            cutlass::ComplexTransform::kNone,
            ref_B,
            cutlass::ComplexTransform::kNone,
            ElementCompute(beta),
            ref_C,
            ref_D,
            ElementAccumulator(0),
            L,     // batch_count
            M * K, // batch_stride_A
            K * N, // batch_stride_B
            M * N, // batch_stride_C
            M * N  // batch_stride_D
    );

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

    return passed;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(M * K * L);
    block_B.reset(K * N * L);
    block_C.reset(M * N * L);
    block_D.reset(M * N * L);
    block_ref_D.reset(M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  void run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.m, options.n, options.k, options.l};

    initialize(problem_size);

    typename Gemm::GemmKernel::Arguments arguments{
            cutlass::gemm::GemmUniversalMode::kGemm,
            problem_size,
            {block_A.get(), stride_A, block_B.get(), stride_B},
            {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D},
            hw_info
    };

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    CUTLASS_CHECK(gemm_op.can_implement(arguments));

    CUTLASS_CHECK(gemm_op.initialize(arguments, workspace.get()));

    // Run the GEMM
    CUTLASS_CHECK(gemm_op.run());

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    Result result;
    result.passed = verify(problem_size, options.alpha, options.beta);

    std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

    if (!result.passed) {
      exit(-1);
    }

    // Run profiling loop
    if (options.iterations > 0)
    {
      GpuTimer timer;
      timer.start();
      for (int iter = 0; iter < options.iterations; ++iter) {
        CUTLASS_CHECK(gemm_op.run());
      }
      timer.stop();

      // Compute average runtime and GFLOPs.
      float elapsed_ms = timer.elapsed_millis();
      result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
      result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

      std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x'
                << options.k << 'x' << options.l << std::endl;
      std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
      std::cout << "  GFLOPS: " << result.gflops << std::endl;
    }
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // Ampere Tensor Core operations exposed with mma.sync and ldmatrix are first available
  // in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    return 0;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!((props.major * 10 + props.minor) >= 80)) {
    std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. 
  // This information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  // Problem configuration
  using ElementA = float;
  using ElementB = float;
  using ElementAcc = float;
  using ElementOutput = float;

  using LayoutA = cutlass::layout::ColumnMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::ColumnMajor;

  // Tiling configuration selection
  using TileShape = Shape<_128,_128,_32>;

  //
  // Assembling the CollectiveMainloop type
  //

  // Number of pipelines you want to use
  constexpr int PipelineStages = 4;

  using DispatchPolicy = cutlass::gemm::MainloopSm80CpAsync<PipelineStages>;

  // This code section describes the MMA op and the tile size a warp will compute
  using TiledMma = TiledMMA<
    MMA_Atom<SM80_16x8x8_F32TF32TF32F32_TN>,
    Layout<Shape<_2,_2,_1>, Stride<_2,_1,_1>>, // 2x2x1 thread group
    Tile<_32,_32,_8>>;                         // 32x32x8 MMA for LDSM, 1x2x1 value group

  // Define the copy layout and atom for device memory copy.
  using GmemTiledCopyA = decltype(
          make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, float>{},
                          Layout<Shape<_16,_8>, Stride<_1,_16>>{},
                          Layout<Shape<_4,_1>>{}));

  using GmemTiledCopyB = decltype(
          make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, float>{},
                          Layout<Shape<_16,_8>, Stride<_8,_1>>{},
                          Layout<Shape<_1,_4>>{}));

  // Define the copy layout and atom for shared memory copy.
  using SmemLayoutAtomA = decltype(composition(Swizzle<2,3,2>{}, Layout<Shape <_32,_8>, Stride< _1,_32>>{}));
  using SmemCopyAtomA = Copy_Atom<UniversalCopy<float>, float>;

  using SmemLayoutAtomB = decltype(composition(Swizzle<3,2,3>{}, Layout<Shape <_8,_32>, Stride<_32, _1>>{}));
  using SmemCopyAtomB = Copy_Atom<SM75_U32x4_LDSM_N, float>;

  // Mainloop
  using CollectiveMainloop = cutlass::gemm::collective::CollectiveMma<
          DispatchPolicy,
          TileShape,
          ElementA,
          cutlass::detail::TagToStrideA_t<LayoutA>,
          ElementB,
          cutlass::detail::TagToStrideB_t<LayoutB>,
          TiledMma,
          GmemTiledCopyA, SmemLayoutAtomA, SmemCopyAtomA, cute::identity,  // A
          GmemTiledCopyB, SmemLayoutAtomB, SmemCopyAtomB, cute::identity   // B
  >;

  //
  // Assembling the Collective Epilogue Type
  //

  using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
          ElementOutput,                                     // <- data type of output matrix
          128 / cutlass::sizeof_bits<ElementOutput>::value,  // <- the number of elements per vectorized
                                                             // memory access. For a byte, it's 16
                                                             // elements. This becomes the vector width of
                                                             // math instructions in the epilogue too
          ElementAcc,                                        // <- data type of accumulator
          ElementOutput>;  // <- data type for alpha/beta in linear combination function

  using CollectiveEpilogue = cutlass::epilogue::collective::DefaultEpilogue<
          cutlass::detail::TagToStrideC_t<LayoutC>,
          cutlass::detail::TagToStrideC_t<LayoutD>,
          EpilogueOp,
          cutlass::gemm::EpilogueDefault>;

  //
  // Assembling the GemmKernel
  //

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
          Shape<int,int,int,int>,
          CollectiveMainloop,
          CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  ExampleRunner<Gemm> runner;

  runner.run(options, hw_info);

  return 0;
}
