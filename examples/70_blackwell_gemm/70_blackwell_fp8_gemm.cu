#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief A FP8 dense GEMM example for the NVIDIA Blackwell SM100 architecture using CUTLASS.

    This example demonstrates minimal set of changes needed to transition from a Hopper CUTLASS 3.x 
    FP8 GEMM kernel (see example 54_hopper_fp8_warp_specialized_gemm) to a Blackwell SM100 FP8 GEMM kernel.
    
    This example shows all important fusions used by FP8 gemm kernels, 
    i.e., scale factor for A, B, C, D tensor, the abs_max value of D tensor.
    
    The Blackwell SM100 CUTLASS kernel uses of the following Blackwell SM100 features:

     1. New series of Tensor Core MMA Instructions (tcgen05) introduced on the Blackwell architecture (sm100a) 
    which have 2x throughput compared to Hopper Tensor Core MMA instructions (WGMMA). 
    
    Note that Hopper WGMMA Tensor Core MMA instructions are not compatible on Blackwell (See https://docs.nvidia.com/cuda/parallel-thread-execution). 

    2. A new per-SM memory called Tensor Memory (TMEM) introduced on the Blackwell architecture (sm100a). 
    Blackwell SM100 Tensor Core MMA instructions store their accumulation results in TMEM instead of the 
    Register File. (Please refer to CUDA 12.8 docs on https://docs.nvidia.com/cuda/).

    3. An extended flavor of the warp-specialized kernel design introduced in Hopper enabled by use of TMEM 
    which allows us to decouple the execution of MMA and epilogue into separate warps. 

    4. A new SW controlled dynamic scheduler based on cluster launch control (See https://docs.nvidia.com/cuda/parallel-thread-execution).

    Usage:
      $ ./examples/70_blackwell_gemm/70_blackwell_fp8_gemm --m=8192 --n=8192 --k=8192
*/



#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"


#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
// A matrix configuration
using ElementA            = cutlass::float_e4m3_t;                          // Element type for A matrix operand
using LayoutA             = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using ElementB            = cutlass::float_e4m3_t;                          // Element type for B matrix operand
using LayoutB             = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using ElementC            = cutlass::float_e4m3_t;                          // Element type for C and D matrix operands
using LayoutC             = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

using ElementD           = ElementC;
using LayoutD            = LayoutC;
constexpr int AlignmentD = AlignmentC;

// MMA type
using ElementAccumulator = float;

// Epilogue types
using ElementBias        = cutlass::half_t;
using ElementCompute     = float;
using ElementAux         = ElementC;
using LayoutAux          = LayoutC;
using ElementAmax        = float;

// MMA and Cluster Tile Shapes
// Shape of the tile computed by tcgen05 MMA, could be across 2 SMs if Cluster Shape %2 == 0 
using MmaTileShape_MNK = Shape<_256,_128,_64>;                          
// Shape of the threadblocks in a cluster
using ClusterShape_MNK = Shape<_2,_2,_1>;
// Shape of the threadblocks participating in a tcgen05 MMA. <1, 1, 1> for cta_group = 1, <2, 1, 1> for cta_group = 2
using AtomThrShape_MNK = Shape<_2, _1, _1>;
// Shape of the tile computed by each SM
using PerSmTileShape_MNK = decltype(shape_div(MmaTileShape_MNK{}, AtomThrShape_MNK{}));

using FusionOp = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltActAmaxAux<
  LayoutC, cutlass::epilogue::thread::ReLU, ElementD, ElementCompute, ElementAux, ElementAmax, ElementBias>;
  
using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
    PerSmTileShape_MNK, ClusterShape_MNK,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementCompute,
    ElementC, LayoutC, AlignmentC,
    ElementD, LayoutC, AlignmentD,
    cutlass::epilogue::collective::EpilogueScheduleAuto,
    FusionOp
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    MmaTileShape_MNK, ClusterShape_MNK,
    cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    cutlass::gemm::collective::KernelScheduleAuto
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>,
    CollectiveMainloop,
    CollectiveEpilogue,
    void>;                // Default to ClusterLaunchControl (CLC) based tile scheduler

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Extract information from Gemm kernel.
using EpilogueOutputOp  = typename Gemm::EpilogueOutputOp;
using ElementScalar     = typename EpilogueOutputOp::ElementScalar;
using ElementAmax       = typename EpilogueOutputOp::ElementAmax;
using ActivationFunctor = typename EpilogueOutputOp::ActivationFn;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;
using StrideAux = StrideC;

constexpr bool IsDFp8 = 
    cute::is_same_v<ElementD, cutlass::float_e4m3_t> or
    cute::is_same_v<ElementD, cutlass::float_e5m2_t>;

constexpr bool IsAuxFp8 =
    cute::is_same_v<ElementAux, cutlass::float_e4m3_t> or
    cute::is_same_v<ElementAux, cutlass::float_e5m2_t>;

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
StrideAux stride_aux;
uint64_t seed;

cutlass::HostTensor<ElementA  , LayoutA  > tensor_A;
cutlass::HostTensor<ElementB  , LayoutB  > tensor_B;
cutlass::HostTensor<ElementC  , LayoutC  > tensor_C;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_D;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_ref_D;
cutlass::HostTensor<ElementAux, LayoutAux> tensor_aux;
cutlass::HostTensor<ElementAux, LayoutAux> tensor_ref_aux;

using LayoutScalar = cutlass::layout::PackedVectorLayout;
cutlass::HostTensor<ElementScalar, LayoutScalar> scalar_alpha;
cutlass::HostTensor<ElementScalar, LayoutScalar> scalar_beta;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_A;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_B;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_C;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_D;
cutlass::HostTensor<ElementScalar, LayoutScalar> scale_aux;
cutlass::HostTensor<ElementAmax  , LayoutScalar> abs_max_D;
cutlass::HostTensor<ElementAmax  , LayoutScalar> reference_abs_max_D;
cutlass::HostTensor<ElementAmax  , LayoutScalar> abs_max_aux;
cutlass::HostTensor<ElementAmax  , LayoutScalar> reference_abs_max_aux;

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  float alpha = 1.f, beta = 0.f;
  float scale_a = 1.f, scale_b = 1.f, scale_c = 1.f, scale_d = 1.f, scale_aux = 1.f;
  bool device_scale = false;
  bool save_aux = true;
  bool save_amax = true;
  int iterations = 1000;
  int m = 1024, n = 512, k = 1024, l = 1;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("scale_a", scale_a, 1.f);
    cmd.get_cmd_line_argument("scale_b", scale_b, 1.f);
    cmd.get_cmd_line_argument("scale_c", scale_c, 1.f);
    cmd.get_cmd_line_argument("scale_d", scale_d, 1.f);
    cmd.get_cmd_line_argument("scale_aux", scale_aux, 1.f);
    cmd.get_cmd_line_argument("device_scale", device_scale, false);
    cmd.get_cmd_line_argument("save_aux", save_aux, true);
    cmd.get_cmd_line_argument("save_amax", save_amax, true);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "70_blackwell_fp8_gemm\n\n"
      << "  Blackwell FP8 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the l extent (batch) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n"
      << "  --scale_a=<f32>             Scaling factor for A\n"
      << "  --scale_b=<f32>             Scaling factor for B\n"
      << "  --scale_c=<f32>             Scaling factor for C\n"
      << "  --scale_d=<f32>             Scaling factor for D (ignored for non-fp8 D)\n"
      << "  --scale_aux=<f32>           Scaling factor for the auxiliary tensor (ignored for non-fp8 aux)\n"
      << "  --device_scale=<bool>       Copy scalars to device memory before kernel launch (default: false)\n"
      << "  --save_aux=<bool>           Save the pre-activation as an auxiliary tensor (default: true)\n"
      << "  --save_amax=<bool>          Save the pre-scaled max absolute value of any fp8 outputs (aux and/or D) (default: true)\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "70_blackwell_fp8_gemm" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed) {

  double scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;
  int bits_output = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  }
  else if (bits_output == 16) {
    scope_max = 5;
    scope_min = -5;
  }
  else {
    scope_max = 8;
    scope_min = -8;
  }
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));
  stride_aux = stride_D;

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  initialize_tensor(tensor_A.host_view(), seed + 2022);
  initialize_tensor(tensor_B.host_view(), seed + 2023);
  initialize_tensor(tensor_C.host_view(), seed + 2024);

  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_C.sync_device();
  tensor_D.sync_device();

  if (options.save_aux) {
    tensor_aux.resize(c_coord);
    tensor_aux.sync_device();
    tensor_ref_aux.resize(c_coord);
  }

  if (options.device_scale) {
    scalar_alpha.resize(cutlass::make_Coord(1));
    scalar_beta.resize(cutlass::make_Coord(1));
    scale_A.resize(cutlass::make_Coord(1));
    scale_B.resize(cutlass::make_Coord(1));
    scale_C.resize(cutlass::make_Coord(1));
    scale_D.resize(cutlass::make_Coord(1));
    scale_aux.resize(cutlass::make_Coord(1));

    cutlass::reference::host::TensorFill(scalar_alpha.host_view(), options.alpha);
    cutlass::reference::host::TensorFill(scalar_beta.host_view(), options.beta);
    cutlass::reference::host::TensorFill(scale_A.host_view(), options.scale_a);
    cutlass::reference::host::TensorFill(scale_B.host_view(), options.scale_b);
    cutlass::reference::host::TensorFill(scale_C.host_view(), options.scale_c);
    cutlass::reference::host::TensorFill(scale_D.host_view(), options.scale_d);
    cutlass::reference::host::TensorFill(scale_aux.host_view(), options.scale_aux);

    scalar_alpha.sync_device();
    scalar_beta.sync_device();
    scale_A.sync_device();
    scale_B.sync_device();
    scale_C.sync_device();
    scale_D.sync_device();
    scale_aux.sync_device();
  }

  if (IsDFp8 && options.save_amax) {
    abs_max_D.resize(cutlass::make_Coord(1));
    abs_max_D.sync_device();
    reference_abs_max_D.resize(cutlass::make_Coord(1));
  }

  if (IsAuxFp8 && options.save_aux && options.save_amax) {
    abs_max_aux.resize(cutlass::make_Coord(1));
    abs_max_aux.sync_device();
    reference_abs_max_aux.resize(cutlass::make_Coord(1));
  }
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(), stride_A, tensor_B.device_data(), stride_B},
    {
      {}, // epilogue.thread
      tensor_C.device_data(), stride_C,
      tensor_D.device_data(), stride_D
    }
  };

  auto &fusion_args = arguments.epilogue.thread;
  fusion_args.alpha = options.alpha;
  fusion_args.beta = options.beta;
  fusion_args.alpha_ptr = scalar_alpha.device_data();
  fusion_args.beta_ptr = scalar_beta.device_data();
  fusion_args.scale_a = options.scale_a;
  fusion_args.scale_b = options.scale_b;
  fusion_args.scale_c = options.scale_c;
  fusion_args.scale_a_ptr = scale_A.device_data();
  fusion_args.scale_b_ptr = scale_B.device_data();
  fusion_args.scale_c_ptr = scale_C.device_data();

  // ignored if tensor types are not fp8
  fusion_args.scale_d = options.scale_d;
  fusion_args.scale_aux = options.scale_aux;
  fusion_args.scale_d_ptr = scale_D.device_data();
  fusion_args.scale_aux_ptr = scale_aux.device_data();

  // leaving/setting these as nullptr disables the fusion at runtime
  fusion_args.bias_ptr = nullptr;

  if (options.save_aux) {
    fusion_args.aux_ptr = tensor_aux.device_data();
    fusion_args.dAux = stride_aux;
    if (options.save_amax) {
      fusion_args.amax_aux_ptr = abs_max_aux.device_data();
    }
  }

  if (options.save_amax) {
    fusion_args.amax_D_ptr = abs_max_D.device_data();
  }

  return arguments;
}

bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.k, options.l), stride_A));
  auto B = cute::make_tensor(tensor_B.host_data(),
      cute::make_layout(cute::make_shape(options.n, options.k, options.l), stride_B));
  auto C = cute::make_tensor(tensor_C.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_C));
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_D));
  auto Aux = cute::make_tensor(tensor_ref_aux.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_aux));
  using unused_t = decltype(D);

  cutlass::reference::host::GettMainloopParams<ElementAccumulator, decltype(A), decltype(B)> mainloop_params{A, B};

  cutlass::reference::host::GettEpilogueParams<
      ElementScalar,
      ElementScalar,
      ElementAccumulator,
      ElementCompute,
      decltype(C),
      decltype(D),
      unused_t, // bias
      decltype(Aux),
      unused_t, // valpha
      unused_t, // vbeta
      ActivationFunctor
  > epilogue_params;

  epilogue_params.C = C;
  epilogue_params.D = D;
  epilogue_params.Aux = Aux;
  epilogue_params.alpha = options.alpha;
  epilogue_params.beta = options.beta;
  epilogue_params.scale_a = options.scale_a;
  epilogue_params.scale_b = options.scale_b;
  epilogue_params.scale_c = options.scale_c;
  epilogue_params.scale_d = options.scale_d;
  epilogue_params.scale_aux = options.scale_aux;
  epilogue_params.abs_max_D = reference_abs_max_D.host_data();
  epilogue_params.abs_max_Aux = reference_abs_max_aux.host_data();

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  // compare_reference
  tensor_D.sync_host();
  bool passed = cutlass::reference::host::TensorEquals(tensor_ref_D.host_view(), tensor_D.host_view());

  if (IsDFp8 && options.save_amax) {
    abs_max_D.sync_host();
    passed &= abs_max_D.at(cutlass::make_Coord(0)) == reference_abs_max_D.at(cutlass::make_Coord(0));
  }

  if (options.save_aux) {
    tensor_aux.sync_host();
    passed &= cutlass::reference::host::TensorEquals(tensor_ref_aux.host_view(), tensor_aux.host_view());
    if (IsAuxFp8 && options.save_amax) {
      abs_max_aux.sync_host();
      passed &= abs_max_aux.at(cutlass::make_Coord(0)) == reference_abs_max_aux.at(cutlass::make_Coord(0));
    }
  }

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  
  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  
  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

 
  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  
  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  
  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least sm100a.
  
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major != 10 || props.minor != 0) {
    std::cerr << "This example requires a GPU with compute capability 100a)." << std::endl;
    return 0;
  } 
  

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Run
  //
#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
  run<Gemm>(options);
#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
