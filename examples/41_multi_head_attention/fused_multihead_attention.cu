#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holdvr nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief CUTLASS Attention Example.

    This workload computes an attention example with non-fixed sequence length input. Pointers of arrays
    are fed into grouped-GEMM functions fused with softmax for computation.

    Examples:

      # Run an attention example with default setup (max sequence length = 1024, batch size = 16, head size = 64, head number = 12)
      $ ./examples/41_multi_head_attention/41_multi_head_attention

      # Run an attention example with batch size = 64 and head number = 16 without checking the correctness
      $ ./examples/41_multi_head_attention/41_multi_head_attention --head_number=16 --batch_size=64 --reference-check=false

      Acknowledgement: this example is inspired by the idea originally prototyped by ByteDance Inc.

*/

/////////////////////////////////////////////////////////////////////////////////////////////////

#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/gemm_transpose_operands.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/kernel/default_gemm_complex.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/epilogue/threadblock/epilogue_with_visitor.h"
#include "cutlass/fast_math.h"
#include "gemm_attention.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;
  bool use_mask;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes0;
  std::vector<cutlass::gemm::GemmCoord> problem_sizes1;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes0_real;
  std::vector<cutlass::gemm::GemmCoord> problem_sizes1_real;

  int alignment;
  int head_number;
  int batch_size;
  int head_size;
  int seq_length;
  int iterations;
  int cuda_streams;

  // alpha0, alpha1 and beta are fixed 
  // in this multi-head attention example
  float alpha0;
  float alpha1;
  float beta;

  //
  // Methods
  // 

  Options():
    help(false),
    error(false),
    alignment(16),
    reference_check(true),
    head_number(12),
    batch_size(16),
    head_size(64),
    seq_length(1024),
    use_mask(false),
    iterations(20),
    cuda_streams(0)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("alignment", alignment, 16);
    cmd.get_cmd_line_argument("head_number", head_number, 12);
    cmd.get_cmd_line_argument("batch_size", batch_size, 16);
    cmd.get_cmd_line_argument("head_size", head_size, 64);
    cmd.get_cmd_line_argument("seq_length", seq_length, 1024);
    cmd.get_cmd_line_argument("use_mask", use_mask, false);
    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("streams", cuda_streams, 0);
    cmd.get_cmd_line_argument("reference-check", reference_check, true);

    randomize_problems();

  }

  void randomize_problems() {

    int problem_count = head_number * batch_size;

    problem_sizes0.reserve(problem_count);
    problem_sizes1.reserve(problem_count);

    // When using mask, the original inputs are not padded
    // and we need to save these info.
    if (use_mask) {
      problem_sizes0_real.reserve(problem_count);
      problem_sizes1_real.reserve(problem_count);
    }

    for (int i = 0; i < batch_size; ++i) {
      // problems belonging to the same batch share the same seq len
      int m_real = (rand() % seq_length);
      int m = (m_real + 1 + alignment - 1) / alignment * alignment;
      int n = m;
      int k = head_size;

      for (int j = 0; j < head_number; ++j) {
        cutlass::gemm::GemmCoord problem0(m, n, k);
        cutlass::gemm::GemmCoord problem1(m, k, n);
        problem_sizes0.push_back(problem0);
        problem_sizes1.push_back(problem1);

        if (use_mask) {
          cutlass::gemm::GemmCoord problem0_real(m_real, m_real, k);
          cutlass::gemm::GemmCoord problem1_real(m_real, k, m_real);
          problem_sizes0_real.push_back(problem0_real);
          problem_sizes1_real.push_back(problem1_real);
        }

      }
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "41_multi_head_attention\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --head_number=<int>         Head number in multi-head attention (default: --head_number=12)\n"
      << "  --batch_size=<int>          Batch size in multi-head attention (default: --batch_size=16)\n"
      << "  --head_size=<int>           Head size in multi-head attention (default: --head_size=64)\n"
      << "  --seq_length=<int>          Max sequence length in multi-head attention (default: --seq_length=1024)\n"
      << "  --use_mask=<bool>           If true, performs padding-like masking in softmax.\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n"
      << "  --reference-check=<bool>    If true, performs reference check.\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = int64_t();

    for (auto const & problem : problem_sizes0) {
      // Two flops per multiply-add
      fmas += problem.product() * 2;
    }
    
    // Multiply another '2' because of the back-to-back GEMM problems in attention
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};



///////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Attention>
class TestbedAttention {
public:

  //
  // Type definitions
  //

  using ElementQ = typename Attention::ElementQ;
  using ElementK = typename Attention::ElementK;
  using ElementP = typename Attention::ElementP;
  using ElementAccumulator = typename Attention::GemmGrouped0::ElementAccumulator;
  using ElementV = typename Attention::ElementV;
  using ElementO = typename Attention::ElementOutput;

  using EpilogueOutputOp = typename Attention::GemmGrouped0::GemmKernel::EpilogueVisitor::ElementwiseFunctor;
  using ElementCompute = typename EpilogueOutputOp::ElementCompute;

  using ElementNorm = typename Attention::ElementNorm;
  using ElementSum = typename Attention::ElementSum;
  using ElementSoftmaxCompute = typename Attention::ElementSoftmaxCompute;

  using LayoutQ = typename Attention::LayoutQ;
  using LayoutK = typename Attention::LayoutK;
  using LayoutP = typename Attention::LayoutP;
  using LayoutV = typename Attention::LayoutV;
  using LayoutO = typename Attention::LayoutO;

  using MatrixCoord = typename LayoutP::TensorCoord;

  using ProblemVisitor0 = typename Attention::GemmKernel0::ProblemVisitor;
  using ProblemVisitor1 = typename Attention::GemmKernel1::ProblemVisitor;

private:

  //
  // Data members
  //

  Options & options;

  /// Initialization
  cutlass::Distribution::Kind init_Q;
  cutlass::Distribution::Kind init_K;
  cutlass::Distribution::Kind init_P;
  cutlass::Distribution::Kind init_V;
  cutlass::Distribution::Kind init_O;
  uint32_t seed;

  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device0;
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device1;
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device0_real;

  std::vector<int64_t> offset_Q;
  std::vector<int64_t> offset_K;
  std::vector<int64_t> offset_P;
  std::vector<int64_t> offset_V;
  std::vector<int64_t> offset_O;
  std::vector<int64_t> offset_Norm;
  std::vector<int64_t> offset_Sum;

  std::vector<int64_t> ldq_host;
  std::vector<int64_t> ldk_host;
  std::vector<int64_t> ldp_host;
  std::vector<int64_t> ldv_host;
  std::vector<int64_t> ldo_host;
  std::vector<int64_t> seqlen_host;

  cutlass::DeviceAllocation<int64_t> ldq;
  cutlass::DeviceAllocation<int64_t> ldk;
  cutlass::DeviceAllocation<int64_t> ldp;
  cutlass::DeviceAllocation<int64_t> ldv;
  cutlass::DeviceAllocation<int64_t> ldo;
  cutlass::DeviceAllocation<int64_t> seqlen;

  cutlass::DeviceAllocation<ElementQ> block_Q;
  cutlass::DeviceAllocation<ElementK> block_K;
  cutlass::DeviceAllocation<ElementP> block_P;
  cutlass::DeviceAllocation<ElementV> block_V;
  cutlass::DeviceAllocation<ElementO> block_O;
  cutlass::DeviceAllocation<ElementNorm> block_Norm;
  cutlass::DeviceAllocation<ElementSum> block_Sum;

  cutlass::DeviceAllocation<int64_t> offset_P_Device;
  cutlass::DeviceAllocation<int64_t> offset_Norm_Device;
  cutlass::DeviceAllocation<int64_t> offset_Sum_Device;

  cutlass::DeviceAllocation<ElementQ *> ptr_Q;
  cutlass::DeviceAllocation<ElementK *> ptr_K;
  cutlass::DeviceAllocation<ElementP *> ptr_P;
  cutlass::DeviceAllocation<ElementV *> ptr_V;
  cutlass::DeviceAllocation<ElementO *> ptr_O;
  cutlass::DeviceAllocation<ElementNorm *> ptr_Max;
  cutlass::DeviceAllocation<ElementSum *> ptr_Sum;

public:

  //
  // Methods
  //

  TestbedAttention(
    Options &options_,
    cutlass::Distribution::Kind init_Q_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_K_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_P_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_V_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_O_ = cutlass::Distribution::Uniform,
    uint32_t seed_ = 3080
  ):
    options(options_), init_Q(init_Q_), init_K(init_K_), init_P(init_P_), init_V(init_V_), init_O(init_O_), seed(seed_) { }

  int problem_count() const {
    return (options.head_number * options.batch_size);
  }

private:

  /// Helper to initialize a tensor view
  template <typename Element>
  void initialize_tensor_(
    Element *ptr,
    size_t capacity, 
    cutlass::Distribution::Kind dist_kind,
    uint32_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      Element scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<typename Attention::ElementP>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        scope_max = 8;
        scope_min = -8;
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::device::BlockFillRandomUniform(
        ptr, capacity, seed, scope_max, scope_min, 0);
    } 
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::device::BlockFillRandomGaussian(
        ptr, capacity, seed, Element(), Element(0.5f));
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      // Fill with increasing elements
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(1), Element());
    } 
    else {

      // Fill with all 1s
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(), Element(1));
    }
  }

  /// Initializes data structures
  void initialize_() {

    //
    // Set scalors for the mha example
    //

    options.alpha0 = 1.0f / sqrt(float(options.head_size));
    options.alpha1 = 1.0f;
    options.beta = 0;

    //
    // Choose random problem sizes
    //

    // construct a few problems of random sizes
    srand(seed);

    int64_t total_elements_Q = 0;
    int64_t total_elements_K = 0;
    int64_t total_elements_P = 0;
    int64_t total_elements_V = 0;
    int64_t total_elements_O = 0;

    int64_t total_elements_partial_norm = 0;

    ldq_host.resize(problem_count());
    ldk_host.resize(problem_count());
    ldp_host.resize(problem_count());
    ldv_host.resize(problem_count());
    ldo_host.resize(problem_count());
    seqlen_host.resize(problem_count());

    for (int32_t i = 0; i < problem_count(); ++i) {

      auto problem = options.problem_sizes0.at(i);

      ldq_host.at(i) = LayoutQ::packed({problem.m(), problem.k()}).stride(0);
      ldk_host.at(i) = LayoutK::packed({problem.k(), problem.n()}).stride(0);
      ldp_host.at(i) = LayoutP::packed({problem.m(), problem.n()}).stride(0);
      ldv_host.at(i) = LayoutV::packed({problem.n(), problem.k()}).stride(0);
      ldo_host.at(i) = LayoutO::packed({problem.m(), problem.k()}).stride(0);

      // m = n for attention problems.
      int64_t non_leading_dim = ldp_host.at(i);
      int64_t threadblock_n = Attention::GemmGrouped0::GemmKernel::EpilogueVisitor::ThreadblockShape::kN;
      int64_t threadblock_num = (ldp_host.at(i) + threadblock_n - 1) / threadblock_n;

      seqlen_host.at(i) = problem.m();

      offset_Q.push_back(total_elements_Q);
      offset_K.push_back(total_elements_K);
      offset_P.push_back(total_elements_P);
      offset_V.push_back(total_elements_V);
      offset_O.push_back(total_elements_O);
      offset_Norm.push_back(total_elements_partial_norm);
      offset_Sum.push_back(total_elements_partial_norm);

      int64_t elements_Q = problem.m() * problem.k();
      int64_t elements_K = problem.k() * problem.n();
      int64_t elements_P = problem.m() * problem.n();
      int64_t elements_V = problem.n() * problem.k();
      int64_t elements_O = problem.m() * problem.k();
      int64_t elements_norm = non_leading_dim * threadblock_num;

      total_elements_Q += elements_Q;
      total_elements_K += elements_K;
      total_elements_P += elements_P;
      total_elements_V += elements_V;
      total_elements_O += elements_O;
      total_elements_partial_norm += elements_norm;

    }

    problem_sizes_device0.reset(problem_count());
    problem_sizes_device1.reset(problem_count());
    problem_sizes_device0.copy_from_host(options.problem_sizes0.data());
    problem_sizes_device1.copy_from_host(options.problem_sizes1.data());

    if (options.use_mask) {
      problem_sizes_device0_real.reset(problem_count());
      problem_sizes_device0_real.copy_from_host(options.problem_sizes0_real.data());
    }

    ldq.reset(problem_count());
    ldk.reset(problem_count());
    ldp.reset(problem_count());
    ldv.reset(problem_count());
    ldo.reset(problem_count());
    seqlen.reset(problem_count());

    ldq.copy_from_host(ldq_host.data());
    ldk.copy_from_host(ldk_host.data());
    ldp.copy_from_host(ldp_host.data());
    ldv.copy_from_host(ldv_host.data());
    ldo.copy_from_host(ldo_host.data());
    seqlen.copy_from_host(seqlen_host.data());

    //
    // Assign pointers
    //

    block_Q.reset(total_elements_Q);
    block_K.reset(total_elements_K);
    block_P.reset(total_elements_P);
    block_V.reset(total_elements_V);
    block_O.reset(total_elements_O);
    block_Norm.reset(total_elements_partial_norm);
    block_Sum.reset(total_elements_partial_norm);

    offset_P_Device.reset(problem_count());
    offset_Norm_Device.reset(problem_count());
    offset_Sum_Device.reset(problem_count());

    // sync offset with device
    cutlass::device_memory::copy_to_device(offset_P_Device.get(), offset_P.data(), offset_P.size());
    cutlass::device_memory::copy_to_device(offset_Norm_Device.get(), offset_Norm.data(), offset_Norm.size());
    cutlass::device_memory::copy_to_device(offset_Sum_Device.get(), offset_Sum.data(), offset_Sum.size());

    std::vector<ElementQ *> ptr_Q_host(problem_count());
    std::vector<ElementK *> ptr_K_host(problem_count());
    std::vector<ElementP *> ptr_P_host(problem_count());
    std::vector<ElementV *> ptr_V_host(problem_count());
    std::vector<ElementO *> ptr_O_host(problem_count());
    std::vector<ElementNorm *> ptr_norm_host(problem_count());
    std::vector<ElementSum *> ptr_sum_host(problem_count());

    for (int32_t i = 0; i < problem_count(); ++i) {
      ptr_Q_host.at(i) = block_Q.get() + offset_Q.at(i);
      ptr_K_host.at(i) = block_K.get() + offset_K.at(i);
      ptr_P_host.at(i) = block_P.get() + offset_P.at(i);
      ptr_V_host.at(i) = block_V.get() + offset_V.at(i);
      ptr_O_host.at(i) = block_O.get() + offset_O.at(i);
      ptr_norm_host.at(i) = block_Norm.get() + offset_Norm.at(i);
      ptr_sum_host.at(i) = block_Sum.get() + offset_Sum.at(i);
    }

    ptr_Q.reset(problem_count());
    ptr_Q.copy_from_host(ptr_Q_host.data());
    
    ptr_K.reset(problem_count());
    ptr_K.copy_from_host(ptr_K_host.data());
    
    ptr_P.reset(problem_count());
    ptr_P.copy_from_host(ptr_P_host.data());

    ptr_V.reset(problem_count());
    ptr_V.copy_from_host(ptr_V_host.data());

    ptr_O.reset(problem_count());
    ptr_O.copy_from_host(ptr_O_host.data());

    ptr_Max.reset(problem_count());
    ptr_Max.copy_from_host(ptr_norm_host.data());

    ptr_Sum.reset(problem_count());
    ptr_Sum.copy_from_host(ptr_sum_host.data());

    //
    // Initialize the problems of the workspace
    //

    initialize_tensor_(block_Q.get(), total_elements_Q, init_Q, seed + 1);
    initialize_tensor_(block_K.get(), total_elements_K, init_K, seed + 2);
    initialize_tensor_(block_V.get(), total_elements_V, init_V, seed + 3);

  }

  template<typename Element>
  bool verify_tensor_(std::vector<Element> vector_Input, \
                       std::vector<Element> vector_Input_Ref,
                       int64_t verify_length = -1) {

    int64_t size = (vector_Input.size() < vector_Input_Ref.size()) ? vector_Input.size() : vector_Input_Ref.size();
    size = (verify_length == -1) ? size : verify_length;

    // 0.05 for absolute error
    float abs_tol = 5e-2f;
    // 10% for relative error
    float rel_tol = 1e-1f;
    for (int64_t i = 0; i < size; ++i) {
      float diff = (float)(vector_Input.at(i) - vector_Input_Ref.at(i));
      float abs_diff = fabs(diff);
      float abs_ref = fabs((float)vector_Input_Ref.at(i) + 1e-5f);
      float relative_diff = abs_diff / abs_ref;
      if ( (isnan(abs_diff) || isinf(abs_diff)) ||  (abs_diff > abs_tol && relative_diff > rel_tol)) {
        printf("diff = %f, rel_diff = %f, {%f, %f}.\n", abs_diff, relative_diff, (float)(vector_Input.at(i)), (float)(vector_Input_Ref.at(i)));
        return false;
      }

    }

    return true;
  }

  /// Verifies the result is a GEMM
  bool verify_() {

    bool passed = true;

    for (int32_t i = 0; i < problem_count(); ++i) {
      cutlass::gemm::GemmCoord problem = options.problem_sizes0.at(i);
      cutlass::gemm::GemmCoord problem1 = options.problem_sizes1.at(i);

      LayoutQ layout_Q(ldq_host.at(i));
      LayoutK layout_K(ldk_host.at(i));
      LayoutP layout_P(ldp_host.at(i));
      LayoutV layout_V(ldv_host.at(i));
      LayoutO layout_O(ldo_host.at(i));

      MatrixCoord extent_Q{problem.m(), problem.k()};
      MatrixCoord extent_K{problem.k(), problem.n()};
      MatrixCoord extent_P{problem.m(), problem.n()};
      MatrixCoord extent_V{problem.n(), problem.k()};
      MatrixCoord extent_O{problem.m(), problem.k()};

      cutlass::TensorView<ElementQ, LayoutQ> view_Q(block_Q.get() + offset_Q.at(i), layout_Q, extent_Q);
      cutlass::TensorView<ElementK, LayoutK> view_K(block_K.get() + offset_K.at(i), layout_K, extent_K);
      cutlass::TensorView<ElementP, LayoutP> view_P(block_P.get() + offset_P.at(i), layout_P, extent_P);
      cutlass::TensorView<ElementV, LayoutV> view_V(block_V.get() + offset_V.at(i), layout_V, extent_V);

      cutlass::DeviceAllocation<ElementP>    block_Ref(layout_P.capacity(extent_P));
      cutlass::TensorView<ElementP, LayoutP> view_Ref_device(block_Ref.get(), layout_P, extent_P);

      cutlass::DeviceAllocation<ElementO>    block_Ref_O(layout_O.capacity(extent_O));
      cutlass::TensorView<ElementO, LayoutO> view_Ref_O_device(block_Ref_O.get(), layout_O, extent_O);

      // Reference GEMM
      cutlass::reference::device::GemmComplex<
          ElementQ, LayoutQ,
          ElementK, LayoutK,
          ElementP, LayoutP, 
          ElementCompute, ElementAccumulator
      >(
        problem,
        ElementAccumulator(options.alpha0), 
        view_Q,
        Attention::GemmGrouped0::kTransformA,
        view_K,
        Attention::GemmGrouped0::kTransformB,
        ElementAccumulator(options.beta), 
        view_P, 
        view_Ref_device, 
        ElementAccumulator(0)
      );

      // Compute softmax for P. We need to explicitly compute softmax
      // over P because softmax is fused to the second GEMM in the
      // profiled implementation.
      std::vector<ElementP> matrix_Ref(layout_P.capacity(extent_P));
      cutlass::device_memory::copy_to_host(matrix_Ref.data(), block_Ref.get(), matrix_Ref.size());
      cutlass::TensorView<ElementP, LayoutP> view_Ref_host(matrix_Ref.data(), layout_P, extent_P);
      std::vector<ElementNorm> vector_Norm_Ref(problem.m());
      std::vector<ElementSum> vector_Sum_Ref(problem.m());

      int n_dim = options.use_mask ? options.problem_sizes0_real.at(i).n() : problem.n();

      // Compute softmax for referece matrix
      // Assumed a row-major storage
      for (int m = 0; m < problem.m(); m++) {
        ElementSoftmaxCompute max = ElementSoftmaxCompute(view_Ref_host.ref().at({m, 0}));
        for (int n = 1; n < n_dim; n++) {
           max = std::max(max, ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})));
        }

        vector_Norm_Ref.at(m) = ElementNorm(max);

        ElementSoftmaxCompute sum = ElementSoftmaxCompute();
        for (int n = 0; n < n_dim; n++) {
          sum += std::exp( ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})) - max );
        }
        ElementSoftmaxCompute inv_sum = ElementSoftmaxCompute(1.0f / sum);

        vector_Sum_Ref.at(m) = ElementSum(inv_sum);

        for (int n = 0; n < n_dim; n++) {
          view_Ref_host.ref().at({m, n}) = ElementP(
            std::exp( ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})) - max ) * inv_sum
          );
        }

      }

      // when not using mask, problem_real and problem share the same sizes
      if (options.use_mask) {
        for (int m = 0; m < problem.m(); m++) {
          for (int n = n_dim; n < problem.n(); n++) {
            view_Ref_host.ref().at({m, n}) = ElementP(0);
          }
        }
      }

      cutlass::device_memory::copy_to_device(block_P.get() + offset_P.at(i), matrix_Ref.data(), matrix_Ref.size());

      // Reference GEMM
      cutlass::reference::device::GemmComplex<
          ElementP, LayoutP,
          ElementV, LayoutV,
          ElementO, LayoutO, 
          ElementCompute, ElementAccumulator
      >(
        problem1,
        ElementAccumulator(options.alpha1), 
        view_P,
        Attention::GemmGrouped0::kTransformA,
        view_V,
        Attention::GemmGrouped0::kTransformB,
        ElementAccumulator(options.beta), 
        view_Ref_O_device, 
        view_Ref_O_device, 
        ElementAccumulator(0)
      );

      // Copy to host memory

      int64_t threadblock_n = Attention::GemmGrouped0::GemmKernel::EpilogueVisitor::ThreadblockShape::kN;
      int64_t threadblock_num = (problem.m() + threadblock_n - 1) / threadblock_n;

      std::vector<ElementNorm> vector_Norm(problem.m() * threadblock_num);
      std::vector<ElementSum> vector_Sum(problem.m() * threadblock_num);

      cutlass::device_memory::copy_to_host(vector_Norm.data(),   block_Norm.get() + offset_Norm.at(i), vector_Norm.size());
      cutlass::device_memory::copy_to_host(vector_Sum.data(),   block_Sum.get() + offset_Sum.at(i), vector_Sum.size());

      cutlass::TensorView<ElementP, LayoutP> view_Ref(matrix_Ref.data(), layout_P, extent_P);

      std::vector<ElementO> matrix_O(layout_O.capacity(extent_O));
      cutlass::device_memory::copy_to_host(matrix_O.data(),   block_O.get() + offset_O.at(i), matrix_O.size());
      std::vector<ElementP> matrix_Ref_O(layout_O.capacity(extent_O));
      cutlass::device_memory::copy_to_host(matrix_Ref_O.data(), block_Ref_O.get(), matrix_Ref_O.size());

      bool verified_N = false;
      bool verified_S = false;
      bool verified_O = false;

      if (!verified_N) {
        verified_N = verify_tensor_<ElementNorm>(vector_Norm, vector_Norm_Ref);
      }
      
      if (!verified_S) {
        verified_S = verify_tensor_<ElementSum>(vector_Sum, vector_Sum_Ref);
      }


      if (!verified_O) {
        verified_O = verify_tensor_<ElementO>(matrix_O, matrix_Ref_O);
      }

      passed = passed && verified_N && verified_S && verified_O;

      if (!passed) {
        std::cerr << "\n***\nError - problem " << i << " failed the QA check\n***\n" << std::endl;

        if (!verified_O) {
          std::cout << "Final matrix output is incorrect" << std::endl;
        }

        if (!verified_N) {
          std::cout << "Max is incorrect" << std::endl;
        }

        if (!verified_S) {
          std::cout << "Sum is incorrect" << std::endl;
        }

        return passed;
      }

    }

    return passed;
  }

public:

  /// Returns the number of threadblocks to launch if the kernel can run on the target
  /// device. Otherwise, returns zero.
  int sufficient() const {
    hipDeviceProp_t properties;
    int device_idx;
    hipError_t result = hipGetDevice(&device_idx);

    if (result != hipSuccess) {
      throw std::runtime_error("hipGetDevice() API call failed.");
    }

    result = hipGetDeviceProperties(&properties, device_idx);

    if (result != hipSuccess) {
      throw std::runtime_error("hipGetDeviceProperties() failed");
    }

    int occupancy = Attention::GemmGrouped0::maximum_active_blocks();

    return properties.multiProcessorCount * occupancy;

  }


  /// Executes a CUTLASS Attention kernel and measures runtime.
  Result profile_grouped() {

    Result result;

    int threadblock_count = sufficient();

    // Early exit
    if (!threadblock_count) {
      std::cout << "Active CUDA device lacks hardware resources to run CUTLASS Attention kernel." << std::endl;
      return result;
    }

    result.passed = false;

    // Initialize the problem
    initialize_();

    typename Attention::Arguments args(
      problem_sizes_device0.get(),
      problem_sizes_device1.get(),
      problem_count(),
      threadblock_count,
      ptr_Q.get(),
      ptr_K.get(),
      ptr_P.get(),
      ptr_V.get(),
      ptr_O.get(),
      ptr_Max.get(),
      ptr_Sum.get(),
      block_P.get(),
      block_Norm.get(),
      block_Sum.get(),
      offset_P_Device.get(),
      offset_Norm_Device.get(),
      offset_Sum_Device.get(),
      ldq.get(),
      ldk.get(),
      ldp.get(),
      ldv.get(),
      ldo.get(),
      ElementAccumulator(options.alpha0),
      ElementAccumulator(options.alpha1),
      ElementAccumulator(options.beta),
      options.head_number,
      options.batch_size,
      options.seq_length,
      options.problem_sizes0.data(),
      options.problem_sizes1.data(),
      problem_sizes_device0_real.get()
    );

    size_t workspace_size0 = ProblemVisitor0::kRequiresPrecomputation ?\
      ProblemVisitor0::get_workspace_size(options.problem_sizes0.data(),\
                                          problem_count(),\
                                          threadblock_count)\
      : 0;

    size_t workspace_size1 = ProblemVisitor1::kRequiresPrecomputation ?\
      ProblemVisitor1::get_workspace_size(options.problem_sizes1.data(),\
                                          problem_count(),\
                                          threadblock_count)\
      : 0;

    cutlass::DeviceAllocation<uint8_t> workspace0(workspace_size0);
    cutlass::DeviceAllocation<uint8_t> workspace1(workspace_size1);

    Attention attention;

    result.status = attention.initialize(args, workspace0.get(), workspace1.get());

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize CUTLASS Attention kernel." << std::endl;
      return result;
    }

    result.status = attention.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize CUTLASS Attention kernel." << std::endl;
      return result;
    }

    // Wait for completion
    result.error = hipDeviceSynchronize();

    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Verify correctness
    //
    result.passed = true;

    if (options.reference_check) {
      result.passed = verify_();
    }

    //
    // Warm-up run of the grouped GEMM object
    //

    result.status = attention.run();

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Attention kernel." << std::endl;
      return result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMM operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < options.iterations; ++iter) {
      attention();
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMM operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    //
    // Cleanup
    //

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    std::cout << std::endl;
    std::cout << "CUTLASS Attention:\n"
      << "====================================================" << std::endl;
    std::cout << "    " << " {max sequence length, head size, head number, batch size} = {" << options.seq_length \
      << ", " << options.head_size << ", " << options.head_number << ", " << options.batch_size << "}." << std::endl;
    std::cout << std::endl;
    std::cout << "    " << "Runtime: " << result.runtime_ms << " ms" << std::endl;
    std::cout << "    " << "GFLOPs: " << result.gflops << std::endl;

    return result;
  }


};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
  
    //
    // This example requires an NVIDIA Ampere-architecture GPU.
    //

    std::cout 
      << "CUTLASS's CUTLASS Attention example requires a GPU of NVIDIA's Ampere Architecture or "
      << "later (compute capability 80 or greater).\n";

    return 0;
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

  //
  // Define the CUTLASS Attention type
  //

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = cutlass::half_t;

  using ElementQ = cutlass::half_t;
  using ElementK = cutlass::half_t;
  using ElementP = ElementOutput;

  using LayoutQ = cutlass::layout::RowMajor;
  using LayoutK = cutlass::layout::ColumnMajor;
  using LayoutP = cutlass::layout::RowMajor;

  static bool const UseMask = false;

  if (UseMask != options.use_mask) {
    std::cerr << "UseMask and user-defined use_mask need to be consistant, "
    << " aborted execution.\n";
    return -2;
  }

  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using ArchTag = cutlass::arch::Sm80;

  using ThreadblockShape0 = cutlass::gemm::GemmShape<128, 128, 32>;
  using WarpShape0 = cutlass::gemm::GemmShape<64, 64, 32>;

  using ThreadblockShape1 = cutlass::gemm::GemmShape<64, 64, 32>;
  using WarpShape1 = cutlass::gemm::GemmShape<32, 32, 32>;
  
  static int const Stages0 = 3;
  static int const Stages1 = 4;

  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;

  using Attention = cutlass::FusedMultiHeadAttention<
    ElementQ,
    LayoutQ,
    ElementK,
    LayoutK,
    ElementP,
    LayoutP,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape0,
    ThreadblockShape1,
    WarpShape0,
    WarpShape1,
    InstructionShape,
    Stages0,
    Stages1,
    UseMask
  >;

  //
  // Test and profile
  //

  TestbedAttention<Attention> testbed(options);

  if (!testbed.sufficient()) {
    std::cout << "The active CUDA device lacks sufficient hardware resources to execute this kernel.\n";
    return 0;
  }

  Result result = testbed.profile_grouped();
  if (!result.passed) {
    std::cout << "Profiling CUTLASS attention has failed.\n";
    std::cout << "\nFailed\n";
    return -1;
  }

  std::cout << "\nPassed\n";

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
