/***************************************************************************************************
 * Copyright (c) 2023 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Grouped scale Hopper FP8 Grouped GEMM example using CUTLASS 3.0 APIs for NVIDIA Hopper architecture
    This example demonstrates a grouped scaled FP8 Grouped GEMM using the new CUTLASS 3.0.
    APIs on NVIDIA Hopper architecture. New features that will be showcased in this example are as follows:
    1. NVIDIA Hopper architecture introduces a new series of tensor core instructions (GMMA)
    which are more efficient than the Ampere tensor core instructions.
    2. NVIDIA Hopper architecture includes new Tensor Memory Accelerator (TMA) unit to transfer large
    blocks of data efficiently between global memory and shared memory. TMA also supports asynchronous
    copies between thread blocks in a cluster. This example also showcases on-the-fly modification of TMA
    descriptors to move between groups/problem_count (represented by groups).
    3. This example uses the Warp Specialized kernel design (see /media/docs/efficient_gemm.md for details).
    4. A simple way to tune the CTA rasterization direction and swizzle pattern of Hopper kernels. Both the
    CTA rasterization direction and swizzle pattern impact cross-CTA locality of accesses. By tuning we can
    improve performance.
    Examples:
      $ ./examples/68_hopper_fp8_warp_specialized_grouped_gemm_with_blockwise_scaling/68_hopper_fp8_warp_specialized_grouped_gemm_with_blockwise_scaling  \
        --m=2816 --n=3072 --k=16384 --save_aux=false --save_amax=false \
        --raster=h --swizzle=2 --benchmark=./test_benchmark.txt

      Where the test_benchmark.txt may look as such:
        0 256x512x128
        1 256x512x512
        2 512x256x128
        3 256x256x128
        4 256x512x1024
        5 1024x512x128 and so on
*/

#include <iostream>
#include <optional>
#include <fstream>
#include <sstream>
#include <vector>
#include <cfloat>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/device/tensor_fill.h"

// Includes from examples directory
#include "helper.h"
#include "hopper_fp8_commandline.hpp"
#include "reference/host/gemm_with_groupwise_scaling.h"

using namespace cute;

using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int,int,int>>; // <M,N,K> per group

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED) && defined(CUTLASS_ARCH_MMA_MODIFIABLE_TMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::float_e4m3_t;                          // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::float_e4m3_t;                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C matrix configuration
using         ElementC    = cutlass::float_e4m3_t;                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = AlignmentC;

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementBlockScale   = float;                                          // Element type for blockscaling during accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation

using TileShape_  = Shape<_128,_128,_128>;  // This one is just to make the compiler happy with verify()...

// ScaleGranularity{M,N}: number of {rows in A}/{columns in B} that share the same scaling factor
// Given TileShape = Shape<_128,_128,_128>:
//   ScaleGranularityM == 128 and ScaleGranularityN == 128 --> 2Dx2D (the shape of the scaling factor)
//   ScaleGranularityM == 1   and ScaleGranularityN == 128 --> 1Dx2D scaling
//   ScaleGranularityM == 128 and ScaleGranularityN == 1   --> 2Dx1D scaling
//   ScaleGranularityM == 1   and ScaleGranularityN == 1   --> 1Dx1D scaling
template <int ScaleGranularityM_, int ScaleGranularityN_>
struct GroupScaleConfig {
  using ArchTag       = cutlass::arch::Sm90;                          // Tag indicating the minimum SM that supports the intended feature
  using OperatorClass = cutlass::arch::OpClassTensorOp;               // Operator class tag
  using TileShape     = Shape<_128,_128,_128>;                        // Threadblock-level tile size
  using ClusterShape  = Shape<_1,_2,_1>;                              // Shape of the threadblocks in a cluster

  static constexpr int ScaleGranularityM = ScaleGranularityM_;
  static constexpr int ScaleGranularityN = ScaleGranularityN_;
  static constexpr int ScaleMsPerTile = size<0>(TileShape{}) / ScaleGranularityM;
  static constexpr int ScaleNsPerTile = size<1>(TileShape{}) / ScaleGranularityN;

  static_assert(size<0>(TileShape{}) == ScaleGranularityM * ScaleMsPerTile,
              "FP8 scaling granularity must evenly divide tile shape along M.");
  static_assert(size<1>(TileShape{}) == ScaleGranularityN * ScaleNsPerTile,
              "FP8 scaling granularity must evenly divide tile shape along N.");

  using KernelSchedule    = cutlass::gemm::KernelPtrArrayTmaWarpSpecializedCooperativeFP8BlockScaledAccum<ScaleGranularityM_, ScaleGranularityN_>;
  using EpilogueSchedule  = cutlass::epilogue::PtrArrayTmaWarpSpecializedCooperative;
  using EpilogueTileType  = cutlass::epilogue::collective::EpilogueTileAuto;
  using FusionOperation   = cutlass::epilogue::fusion::LinearCombination<ElementC, ElementAccumulator>;
};

using GroupScale1D1DConfig = GroupScaleConfig<                    1,                     1>;
using GroupScale1D2DConfig = GroupScaleConfig<                    1, size<1>(TileShape_{})>;
using GroupScale2D1DConfig = GroupScaleConfig<size<0>(TileShape_{}),                     1>;
using GroupScale2D2DConfig = GroupScaleConfig<size<0>(TileShape_{}), size<1>(TileShape_{})>;

template <typename ScheduleConfig>
struct GroupScaleGemm {
  using ArchTag           = typename ScheduleConfig::ArchTag;
  using OperatorClass     = typename ScheduleConfig::OperatorClass;
  using TileShape         = typename ScheduleConfig::TileShape;
  using ClusterShape      = typename ScheduleConfig::ClusterShape;
  using KernelSchedule    = typename ScheduleConfig::KernelSchedule;
  using EpilogueSchedule  = typename ScheduleConfig::EpilogueSchedule;
  using EpilogueTileType  = typename ScheduleConfig::EpilogueTileType;
  using FusionOperation   = typename ScheduleConfig::FusionOperation;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    TileShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementCompute,
    ElementC, LayoutC *, AlignmentC,
    ElementD, LayoutD *, AlignmentD,
    EpilogueSchedule,
    FusionOperation
  >::CollectiveOp;

  using CollectiveMainloopWithGroupWiseScaling = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA *, AlignmentA,
    ElementB, LayoutB *, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      ProblemShape,
      CollectiveMainloopWithGroupWiseScaling,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
};

using GroupScale1D1DGemm = GroupScaleGemm<GroupScale1D1DConfig>;
using GroupScale1D2DGemm = GroupScaleGemm<GroupScale1D2DConfig>;
using GroupScale2D1DGemm = GroupScaleGemm<GroupScale2D1DConfig>;
using GroupScale2D2DGemm = GroupScaleGemm<GroupScale2D2DConfig>;

// Extract information from Gemm kernel.
using EpilogueOutputOp  = typename GroupScale1D1DGemm::Gemm::EpilogueOutputOp;
using ElementScalar     = typename EpilogueOutputOp::ElementScalar;
using ActivationFunctor = typename EpilogueOutputOp::ActivationFn;

using StrideA = typename GroupScale1D1DGemm::Gemm::GemmKernel::InternalStrideA;
using StrideB = typename GroupScale1D1DGemm::Gemm::GemmKernel::InternalStrideB;
using StrideC = typename GroupScale1D1DGemm::Gemm::GemmKernel::InternalStrideC;
using StrideD = typename GroupScale1D1DGemm::Gemm::GemmKernel::InternalStrideD;

static_assert(cute::is_same_v<ElementAccumulator, ElementBlockScale>,
             "ElementAccumulator and ElementBlockScale should be same datatype");

/// Initialization

cutlass::DeviceAllocation<typename ProblemShape::UnderlyingProblemShape> problem_sizes;

std::vector<int64_t> offset_A;
std::vector<int64_t> offset_B;
std::vector<int64_t> offset_C;
std::vector<int64_t> offset_D;
std::vector<int64_t> offset_blockscale_A;
std::vector<int64_t> offset_blockscale_B;

std::vector<StrideA> stride_A_host;
std::vector<StrideB> stride_B_host;
std::vector<StrideC> stride_C_host;
std::vector<StrideD> stride_D_host;

std::vector<ElementAccumulator> alpha_host;
std::vector<ElementAccumulator> beta_host;

uint64_t seed;

cutlass::DeviceAllocation<ElementA> block_A;
cutlass::DeviceAllocation<ElementB> block_B;
cutlass::DeviceAllocation<ElementC> block_C;
cutlass::DeviceAllocation<ElementD> block_D;
cutlass::DeviceAllocation<ElementBlockScale> blockscale_block_A;
cutlass::DeviceAllocation<ElementBlockScale> blockscale_block_B;

cutlass::DeviceAllocation<const ElementA *> ptr_A;
cutlass::DeviceAllocation<const ElementB *> ptr_B;
cutlass::DeviceAllocation<const ElementC *> ptr_C;
cutlass::DeviceAllocation<ElementD *> ptr_D;
cutlass::DeviceAllocation<ElementD *> ptr_ref_D;
cutlass::DeviceAllocation<const ElementBlockScale *> ptr_blockscale_A;
cutlass::DeviceAllocation<const ElementBlockScale *> ptr_blockscale_B;

cutlass::DeviceAllocation<StrideA> stride_A;
cutlass::DeviceAllocation<StrideB> stride_B;
cutlass::DeviceAllocation<StrideC> stride_C;
cutlass::DeviceAllocation<StrideD> stride_D;

cutlass::DeviceAllocation<ElementAccumulator*> alpha_device;
cutlass::DeviceAllocation<ElementAccumulator*> beta_device;
cutlass::DeviceAllocation<ElementAccumulator> block_alpha;
cutlass::DeviceAllocation<ElementAccumulator> block_beta;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED) && defined(CUTLASS_ARCH_MMA_MODIFIABLE_TMA_SM90_SUPPORTED) 

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

using RasterOrderOptions = typename cutlass::gemm::kernel::detail::PersistentTileSchedulerSm90GroupParams<Shape<int,int,int>>::RasterOrderOptions;

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED) && defined(CUTLASS_ARCH_MMA_MODIFIABLE_TMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element, class ScopeMin = std::nullopt_t, class ScopeMax = std::nullopt_t>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023,
  ScopeMin scope_min = std::nullopt, ScopeMax scope_max = std::nullopt) {

  double _scope_max, _scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;
  if (bits_input == 1) {
    _scope_max = 2;
    _scope_min = 0;
  } else if (bits_input <= 8) {
    _scope_max = 2;
    _scope_min = -2;
  } else if (bits_input == 16) {
    _scope_max = 5;
    _scope_min = -5;
  } else {
    _scope_max = 8;
    _scope_min = -8;
  }
  if constexpr (!std::is_same_v<ScopeMax, std::nullopt_t>) {
    _scope_max = scope_max;
  }
  if constexpr (!std::is_same_v<ScopeMin, std::nullopt_t>) {
    _scope_min = scope_min;
  }
  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, (Element) _scope_max, (Element) _scope_min, 0);

  return true;
}

/// Allocates device-side data
template <typename OptionType>
void allocate(const OptionType &options) {

  using TileShape = typename OptionType::GroupScaleConfig::TileShape;
  const int ScaleMsPerTile = OptionType::GroupScaleConfig::ScaleMsPerTile;
  const int ScaleNsPerTile = OptionType::GroupScaleConfig::ScaleNsPerTile;

  int64_t total_elements_A = 0;
  int64_t total_elements_B = 0;
  int64_t total_elements_C = 0;
  int64_t total_elements_D = 0;
  int64_t total_elements_blockscale_A = 0;
  int64_t total_elements_blockscale_B = 0;

  offset_A.clear();
  offset_B.clear();
  offset_C.clear();
  offset_D.clear();
  offset_blockscale_A.clear();
  offset_blockscale_B.clear();
  stride_A_host.clear();
  stride_B_host.clear();
  stride_C_host.clear();
  stride_D_host.clear();
  
  for (int32_t i = 0; i < options.groups; ++i) {

    auto problem = options.problem_sizes_host.at(i);
    auto M = get<0>(problem);
    auto N = get<1>(problem);
    auto K = get<2>(problem);

    auto blockscale_shape = shape(get<1>(cute::zipped_divide(cute::make_layout(problem), TileShape{})));
    auto groupscale_m = cute::get<0>(blockscale_shape) * ScaleMsPerTile; // We need to pad along M in scale tensor of A to prevent illegal memory access.
    auto groupscale_n = cute::get<1>(blockscale_shape) * ScaleNsPerTile; // We need to pad along N in scale tensor of A to prevent illegal memory access.
    auto blockscale_k = cute::get<2>(blockscale_shape);

    offset_A.push_back(total_elements_A);
    offset_B.push_back(total_elements_B);
    offset_C.push_back(total_elements_C);
    offset_D.push_back(total_elements_D);
    offset_blockscale_A.push_back(total_elements_blockscale_A);
    offset_blockscale_B.push_back(total_elements_blockscale_B);

    int64_t elements_A = M * K;
    int64_t elements_B = K * N;
    int64_t elements_C = M * N;
    int64_t elements_D = M * N;
    int64_t elements_blockscale_A = groupscale_m * blockscale_k;
    int64_t elements_blockscale_B = groupscale_n * blockscale_k;

    total_elements_A += elements_A;
    total_elements_B += elements_B;
    total_elements_C += elements_C;
    total_elements_D += elements_D;
    total_elements_blockscale_A += elements_blockscale_A;
    total_elements_blockscale_B += elements_blockscale_B;

    stride_A_host.push_back(cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1}));
    stride_B_host.push_back(cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1}));
    stride_C_host.push_back(cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1}));
    stride_D_host.push_back(cutlass::make_cute_packed_stride(StrideD{}, {M, N, 1}));

  }

  block_A.reset(total_elements_A);
  block_B.reset(total_elements_B);
  block_C.reset(total_elements_C);
  block_D.reset(total_elements_D);
  block_alpha.reset(options.groups);
  block_beta.reset(options.groups);
  blockscale_block_A.reset(total_elements_blockscale_A);
  blockscale_block_B.reset(total_elements_blockscale_B);
}

/// Initialize operands to be used in the GEMM and reference GEMM
template <typename OptionType>
void initialize(const OptionType &options) {

  problem_sizes.reset(options.groups);
  problem_sizes.copy_from_host(options.problem_sizes_host.data());

  std::vector<ElementA *> ptr_A_host(options.groups);
  std::vector<ElementB *> ptr_B_host(options.groups);
  std::vector<ElementC *> ptr_C_host(options.groups);
  std::vector<ElementD *> ptr_D_host(options.groups);
  std::vector<ElementAccumulator *> ptr_alpha_host(options.groups);
  std::vector<ElementAccumulator *> ptr_beta_host(options.groups);
  std::vector<ElementBlockScale *> ptr_blockscale_A_host(options.groups);
  std::vector<ElementBlockScale *> ptr_blockscale_B_host(options.groups);

  alpha_host.clear();
  beta_host.clear();

  for (int i = 0; i < options.groups; i++) {
    ptr_A_host.at(i) = block_A.get() + offset_A.at(i);
    ptr_B_host.at(i) = block_B.get() + offset_B.at(i);
    ptr_C_host.at(i) = block_C.get() + offset_C.at(i);
    ptr_D_host.at(i) = block_D.get() + offset_D.at(i);
    ptr_blockscale_A_host.at(i) = blockscale_block_A.get() + offset_blockscale_A.at(i);
    ptr_blockscale_B_host.at(i) = blockscale_block_B.get() + offset_blockscale_B.at(i);
    alpha_host.push_back((options.alpha == FLT_MAX) ? static_cast<ElementAccumulator>((rand() % 5) + 1) : options.alpha);
    beta_host.push_back((options.beta == FLT_MAX) ? static_cast<ElementAccumulator>(rand() % 5) : options.beta);
    ptr_alpha_host.at(i) = block_alpha.get() + i;
    ptr_beta_host.at(i) = block_beta.get() + i;
  }

  ptr_A.reset(options.groups);
  ptr_A.copy_from_host(ptr_A_host.data());

  ptr_B.reset(options.groups);
  ptr_B.copy_from_host(ptr_B_host.data());

  ptr_C.reset(options.groups);
  ptr_C.copy_from_host(ptr_C_host.data());

  ptr_D.reset(options.groups);
  ptr_D.copy_from_host(ptr_D_host.data());

  ptr_blockscale_A.reset(options.groups);
  ptr_blockscale_A.copy_from_host(ptr_blockscale_A_host.data());

  ptr_blockscale_B.reset(options.groups);
  ptr_blockscale_B.copy_from_host(ptr_blockscale_B_host.data());

  stride_A.reset(options.groups);
  stride_A.copy_from_host(stride_A_host.data());

  stride_B.reset(options.groups);
  stride_B.copy_from_host(stride_B_host.data());

  stride_C.reset(options.groups);
  stride_C.copy_from_host(stride_C_host.data());

  stride_D.reset(options.groups);
  stride_D.copy_from_host(stride_D_host.data());

  alpha_device.reset(options.groups);
  alpha_device.copy_from_host(ptr_alpha_host.data());
  beta_device.reset(options.groups);
  beta_device.copy_from_host(ptr_beta_host.data());

  initialize_block(block_A, seed + 2022);
  initialize_block(block_B, seed + 2023);
  initialize_block(block_C, seed + 2024);
  initialize_block(blockscale_block_A, seed + 2025, -1, 1);
  initialize_block(blockscale_block_B, seed + 2026, -1, 1);

  block_alpha.copy_from_host(alpha_host.data());
  block_beta.copy_from_host(beta_host.data());

}

/// Populates a Gemm::Arguments structure from the given commandline options
template<typename GemmArguments, typename OptionType>
GemmArguments args_from_options(const OptionType &options, bool host_problem_shapes_available = true)
{
  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  int device_id = 0;
  cutlass::KernelHardwareInfo kernel_hw_info = cutlass::KernelHardwareInfo::make_kernel_hardware_info<typename GroupScale1D1DGemm::Gemm::GemmKernel>(device_id);

  GemmArguments arguments{
    cutlass::gemm::GemmUniversalMode::kGrouped,
    {options.groups, problem_sizes.get(), host_problem_shapes_available ? options.problem_sizes_host.data() : (decltype(options.problem_sizes_host.data())) nullptr},
    {ptr_A.get(), stride_A.get(), ptr_B.get(), stride_B.get(),
     ptr_blockscale_A.get(),
     ptr_blockscale_B.get()
    },
    {
      {}, // epilogue.thread
      ptr_C.get(), stride_C.get(),
      ptr_D.get(), stride_D.get()
    },
    kernel_hw_info
  };

  auto &fusion_args = arguments.epilogue.thread;
  if (options.alpha != FLT_MAX && options.beta != FLT_MAX) {
    // If both alpha/beta are provided (via cmd line args) and are scalar, i.e., same alpha/beta applies to all batches.
    fusion_args.alpha = options.alpha;
    fusion_args.beta = options.beta;
    fusion_args.alpha_ptr = nullptr;
    fusion_args.beta_ptr = nullptr;
    fusion_args.alpha_ptr_array = nullptr;
    fusion_args.beta_ptr_array = nullptr;
    // Single alpha and beta for all groups
    fusion_args.dAlpha = {cute::_0{}, cute::_0{}, 0};
    fusion_args.dBeta = {cute::_0{}, cute::_0{}, 0};
  }
  else {
    // If pointers to alpha/beta are provided, i.e., alpha/beta can differ between batches/groups.
    fusion_args.alpha = 0;
    fusion_args.beta = 0;
    fusion_args.alpha_ptr = nullptr;
    fusion_args.beta_ptr = nullptr;
    fusion_args.alpha_ptr_array = alpha_device.get();
    fusion_args.beta_ptr_array = beta_device.get();
    // One alpha and beta per each group
    fusion_args.dAlpha = {cute::_0{}, cute::_0{}, 1};
    fusion_args.dBeta = {cute::_0{}, cute::_0{}, 1};
  }

  arguments.scheduler.raster_order = options.raster;
  // The tile scheduler will swizzle up to 8 and with the nearest multiple of 2 (i.e., 1, 2, 4, and 8)
  arguments.scheduler.max_swizzle_size = options.swizzle;

  return arguments;
}

template <typename OptionType>
bool verify(const OptionType &options) {

  //
  // Compute reference output
  //

  std::vector<ElementA> block_A_host(block_A.size());
  std::vector<ElementB> block_B_host(block_B.size());
  std::vector<ElementC> block_C_host(block_C.size());
  std::vector<ElementD> block_D_host_kernel(block_D.size());
  std::vector<ElementD> block_D_host_ref(block_D.size());
  std::vector<ElementBlockScale> blockscale_block_A_host(blockscale_block_A.size());
  std::vector<ElementBlockScale> blockscale_block_B_host(blockscale_block_B.size());

  block_A.copy_to_host(block_A_host.data());
  block_B.copy_to_host(block_B_host.data());
  block_C.copy_to_host(block_C_host.data());
  block_D.copy_to_host(block_D_host_kernel.data());
  blockscale_block_A.copy_to_host(blockscale_block_A_host.data());
  blockscale_block_B.copy_to_host(blockscale_block_B_host.data());

  bool passed = true;
  for (int group_idx = 0; group_idx < options.groups; group_idx++) {
    // Group scaling tensors shapes based `ScaleGranularityM`, CTA Block (TileShape) and GEMM Problem shape
    auto [m, n, k] = options.problem_sizes_host.at(group_idx);
    auto gemm_problem_shape = cute::make_shape(m, n, k);
    auto blockscale_shape = shape(get<1>(cute::zipped_divide(cute::make_layout(gemm_problem_shape), TileShape_{})));
    auto blockscale_m = cute::get<0>(blockscale_shape);
    auto blockscale_n = cute::get<1>(blockscale_shape);
    auto blockscale_k = cute::get<2>(blockscale_shape);
    auto groupscale_m = blockscale_m * OptionType::GroupScaleConfig::ScaleMsPerTile;
    auto groupscale_n = blockscale_n * OptionType::GroupScaleConfig::ScaleNsPerTile;

    // Create instantiation for device reference gemm kernel
    auto A = cute::make_tensor(block_A_host.data() + offset_A.at(group_idx),
                              cute::make_layout(
                                  cute::make_shape(m, k, 1),
                                  stride_A_host.at(group_idx)
                                )
                              );
    auto B = cute::make_tensor(block_B_host.data() + offset_B.at(group_idx),
                              cute::make_layout(
                                cute::make_shape(n, k, 1),
                                stride_B_host.at(group_idx)
                                )
                              );
    auto C = cute::make_tensor(block_C_host.data() + offset_C.at(group_idx),
                              cute::make_layout(
                                  cute::make_shape(m, n, 1),
                                  stride_C_host.at(group_idx)
                                )
                              );
    auto D = cute::make_tensor(block_D_host_ref.data() + offset_D.at(group_idx),
                              cute::make_layout(
                                  cute::make_shape(m, n, 1),
                                  stride_D_host.at(group_idx)
                                )
                              );

    auto blockscale_A = cute::make_tensor(blockscale_block_A_host.data() + offset_blockscale_A.at(group_idx),
                                          cute::make_layout(
                                            cute::make_shape(groupscale_m, blockscale_k, 1),
                                            cute::make_stride(1, groupscale_m, groupscale_m * blockscale_k)
                                          )
                                        );
    auto blockscale_B = cute::make_tensor(blockscale_block_B_host.data() + offset_blockscale_B.at(group_idx),
                                          cute::make_layout(
                                            cute::make_shape(groupscale_n, blockscale_k, 1),
                                            cute::make_stride(1, groupscale_n, groupscale_n * blockscale_k)
                                          )
                                        );

    using unused_t = decltype(D);

    cutlass::reference::host::GettMainloopParams<
      ElementAccumulator,
      decltype(A),
      decltype(B),
      decltype(blockscale_A),
      decltype(blockscale_B),
      TileShape_
    > mainloop_params{
        A, B,                         // Operand Tensors
        blockscale_A, blockscale_B    // Groupwise scaling Tensors
    };

    cutlass::reference::host::GettEpilogueParams<
        ElementScalar,
        ElementScalar,
        ElementAccumulator,
        ElementCompute,
        decltype(C),
        decltype(D),
        unused_t, // bias
        unused_t, // Aux
        unused_t, // valpha
        unused_t, // vbeta
        ActivationFunctor
    > epilogue_params;

    epilogue_params.C = C;
    epilogue_params.D = D;
    epilogue_params.alpha = alpha_host.at(group_idx);
    epilogue_params.beta = beta_host.at(group_idx);

    // get reference result
    cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    auto this_group_passed = std::equal(
      // std::execution::par_unseq,
      block_D_host_ref.data() + offset_D.at(group_idx),
      block_D_host_ref.data() + offset_D.at(group_idx) + m * n,
      block_D_host_kernel.data() + offset_D.at(group_idx)
    );
    
    passed &= this_group_passed;

#if 0
    std::cout << "Group: " << group_idx << " M: " << m << " N: " << n << " K: " << k << " Status: " << this_group_passed << std::endl;
#endif

  }

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm, typename OptionType>
int run(OptionType &options, bool host_problem_shapes_available = true)
{
  using TileShape = typename OptionType::GroupScaleConfig::TileShape;
  const int ScaleGranularityM = OptionType::GroupScaleConfig::ScaleGranularityM;
  const int ScaleGranularityN = OptionType::GroupScaleConfig::ScaleGranularityN;
  const int ScaleMsPerTile    = OptionType::GroupScaleConfig::ScaleMsPerTile;
  const int ScaleNsPerTile    = OptionType::GroupScaleConfig::ScaleNsPerTile;

  allocate(options);
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<typename Gemm::Arguments>(options, host_problem_shapes_available);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
   exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::string raster = "Heuristic";

    if (options.raster == RasterOrderOptions::AlongN) {
      raster = "Along N";
    }
    else if (options.raster == RasterOrderOptions::AlongM) {
      raster = "Along M";
    }

    std::cout << "  Problem Sizes, Alpha, Beta " << std::endl;
    for (int32_t i = 0; i < options.groups; ++i) {
      std::cout << "    " << options.problem_sizes_host.at(i);
      std::cout << ", " << alpha_host.at(i) << ", " << beta_host.at(i) << std::endl;
    }
    std::cout << "  Groups      : " << options.groups  << std::endl;
    std::cout << "  Tile shape (M, N, K): " << size<0>(TileShape{}) << ", " << size<1>(TileShape{}) << ", " << size<2>(TileShape{}) << std::endl;
    std::cout << "  ScaleGranularityM: " << ScaleGranularityM << " (ScaleMsPerTile: " << ScaleMsPerTile << ")" << std::endl;
    std::cout << "  ScaleGranularityN: " << ScaleGranularityN << " (ScaleNsPerTile: " << ScaleNsPerTile << ")" << std::endl;
    std::cout << "  Rasterization: " << raster << " with a maximum CTA swizzle of " << options.swizzle << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
    fflush(stdout);
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED) && defined(CUTLASS_ARCH_MMA_MODIFIABLE_TMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 3)) {
    std::cerr << "This example requires CUDA 12.3 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major != 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED) && defined(CUTLASS_ARCH_MMA_MODIFIABLE_TMA_SM90_SUPPORTED)

  //
  // Parse options
  //

  Options<RasterOrderOptions, ProblemShape, GroupScale1D1DConfig> options_1d1d;
  Options<RasterOrderOptions, ProblemShape, GroupScale1D2DConfig> options_1d2d;
  Options<RasterOrderOptions, ProblemShape, GroupScale2D1DConfig> options_2d1d;
  Options<RasterOrderOptions, ProblemShape, GroupScale2D2DConfig> options_2d2d;

  options_1d1d.parse(argc, args);
  options_1d2d.parse(argc, args);
  options_2d1d.parse(argc, args);
  options_2d2d.parse(argc, args);

  if (options_1d1d.help) {
    options_1d1d.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

  auto run_tests = [&] (bool host_problem_shapes_available = true) {
    std::cout << "Grouped GEMM kernel with 1D1D group scale" << std::endl;
    run<GroupScale1D1DGemm::Gemm>(options_1d1d, host_problem_shapes_available);
    std::cout << "Grouped GEMM kernel with 1D2D group scale" << std::endl;
    run<GroupScale1D2DGemm::Gemm>(options_1d2d, host_problem_shapes_available);
    std::cout << "Grouped GEMM kernel with 2D1D group scale" << std::endl;
    run<GroupScale2D1DGemm::Gemm>(options_2d1d, host_problem_shapes_available);
    std::cout << "Grouped GEMM kernel with 2D2D group scale" << std::endl;
    run<GroupScale2D2DGemm::Gemm>(options_2d2d, host_problem_shapes_available);
    std::cout << std::endl;
  };

  std::cout << "Running tests with host problem shapes:" << std::endl;
  run_tests(true);
  std::cout << "Running tests without host problem shapes:" << std::endl;
  run_tests(false);

#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
