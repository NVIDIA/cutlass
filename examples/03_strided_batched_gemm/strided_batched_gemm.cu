/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include <iostream>
#include <vector>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/sgemm_traits.h"

#pragma warning( disable : 4503)

/*
This example demonstrates how to use cutlass to compute a batched strided gemm.
In this example, both A and B matrix are non-transpose and column major matrix
batched_C = batched_A x batched_B
As an example, matrix C can be seen as
-----------------------------------------------------------
(0,0,0) | (0,0,1) | (0,0,2) | (1,0,0) | (1,0,1) | (1,0,2) |
-----------------------------------------------------------
(0,1,0) | (0,1,1) | (0,1,2) | (1,1,0) | (1,1,1) | (1,1,2) |
-----------------------------------------------------------
(0,2,0) | (0,2,1) | (0,2,2) | (1,2,0) | (1,2,1) | (1,2,2) |
-----------------------------------------------------------
(0,3,0) | (0,3,1) | (0,3,2) | (1,3,0) | (1,3,1) | (1,3,2) |
-----------------------------------------------------------
(0,4,0) | (0,4,1) | (0,4,2) | (1,4,0) | (1,4,1) | (1,4,2) |
-----------------------------------------------------------
(0,5,0) | (0,5,1) | (0,5,2) | (1,5,0) | (1,5,1) | (1,5,2) |
-----------------------------------------------------------
           batch 0          |           batch 1
where we denote each element with (batch_idx, row_idx, column_idx)
In this example, batch size is 2, M is 6 and N is 3
The stride (batch_stride_C) between the first element of two batches is ldc * n

matrix A can be seen as
---------------------------------------
(0,0,0) | (0,0,1) | (1,0,0) | (1,0,1) |
---------------------------------------
(0,1,0) | (0,1,1) | (1,1,0) | (1,1,1) |
---------------------------------------
(0,2,0) | (0,2,1) | (1,2,0) | (1,2,1) |
---------------------------------------
(0,3,0) | (0,3,1) | (1,3,0) | (1,3,1) |
---------------------------------------
(0,4,0) | (0,4,1) | (1,4,0) | (1,4,1) |
---------------------------------------
(0,5,0) | (0,5,1) | (1,5,0) | (1,5,1) |
---------------------------------------
     batch 0      |      batch 1
, where batch size is 2, M is 6 and K is 2
The stride (batch_stride_B) between the first element of two batches is lda * k

matrix B can be seen as
-----------------------------
(0,0,0) | (0,0,1) | (0,0,2) |
----------------------------- batch 0
(0,1,0) | (0,1,1) | (0,1,2) |
-------------------------------------
(1,0,0) | (1,0,1) | (1,0,2) |
----------------------------- batch 1
(1,1,0) | (1,1,1) | (1,1,2) |
-----------------------------
, where the batch size is 2, N is 3 and K is 2
The stride (batch_stride_C) between the first element of two batches is k


*/

hipError_t cutlass_strided_batched_sgemm(float const *A,
  int lda,
  long long int batch_stride_A,
  float const *B,
  int ldb,
  long long int batch_stride_B,
  float *C,
  int ldc,
  long long int batch_stride_C,
  float alpha,
  float beta,
  int m, 
  int n,
  int k,
  int batch_count) {
  // create a cutlass traits
  typedef cutlass::gemm::SgemmTraits<cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor, cutlass::Shape<8, 128, 128> >
    SgemmTraits;
  
  // create a CUTLASS GEMM object.
  typedef cutlass::gemm::Gemm<SgemmTraits> Gemm;
  
  // Construct and initialize CUTLASS GEMM parameters object.
  typename Gemm::Params params;
  
  int result = params.initialize(
    m,                  // M dimension for each batch
    n,                  // N dimension for each batch
    k,                  // K dimension for each batch
    alpha,              // scalar alpha
    A,
    lda,
    batch_stride_A,     // distance in memory between the first element of neighboring batch
    B,
    ldb,
    batch_stride_B,     // distance in memory between the first element of neighboring batch
    beta,               // scalar beta
    C,                  // source matrix C
    ldc,
    batch_stride_C,     // distance in memory between the first element of neighboring batch
    C,                  // destination matrix C (may be different memory than source C matrix)
    ldc,
    batch_stride_C,    // distance in memory between the first element of neighboring batch
    batch_count
  );
  
  if (result != 0) {
    std::cerr << "Failed to initialize CUTLASS Gemm::Params object." << std::endl;
    return hipErrorInvalidValue;
  }
  
  // Launch the CUTLASS GEMM kernel.
  Gemm::launch(params);
  result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    std::cerr << "kernel launch result = " << result << std::endl;
  }
  return hipGetLastError();
}

template<typename T> 
hipError_t strided_batched_gemm_nn_reference(std::vector<T> const &A, 
  int lda,
  long long int batch_stride_A,
  std::vector<T> const &B, 
  int ldb,
  long long int batch_stride_B,
  std::vector<T> &C, 
  int ldc,
  long long int batch_stride_C,
  T alpha,
  T beta,
  int m,
  int n,
  int k,
  int batch_count) {
  /*
  strided batched gemm NN
  */
  
  hipError_t result = hipSuccess;

  if (A.size() < lda * k * batch_count) {
    std::cout << "the size of A is too small" << std::endl;
    return hipErrorInvalidValue;
  }
  if (B.size() < ldb * n) {
    std::cout << "the size of B is too small" << std::endl;
    return hipErrorInvalidValue;
  }
  if (C.size() < ldc * n * batch_count) {
    std::cout << "the size of C is too small" << std::endl;
    return hipErrorInvalidValue;
  }
  
  for (int batch_idx = 0; batch_idx < batch_count; batch_idx++) {
    for (int n_idx = 0; n_idx < n; n_idx++) {
      for (int m_idx = 0; m_idx < m; m_idx++) {
        T accum = beta * C[batch_idx * batch_stride_C + n_idx * ldc + m_idx];
        for (int k_idx = 0; k_idx < k; k_idx++) {
          accum += alpha 
            * A[batch_idx * batch_stride_A + k_idx * lda + m_idx]
            * B[batch_idx * batch_stride_B + n_idx * ldb + k_idx];
        }
        C[batch_idx * batch_stride_C + n_idx * ldc + m_idx] = accum;
      }
    }
  }

  return result;
}

int main() {
  int const m = 16;
  int const n = 24;
  int const k = 8;
  int const batch_count = 3;

  // A, B are non-transpose, column major
  int const lda = m;
  int const ldb = k * batch_count;
  int const ldc = m;

  int const count_A = batch_count * lda * k;
  int const count_B = ldb * n;
  int const count_C = batch_count * ldc * n;

  // the memory is batched along K dimension
  long long int batch_stride_A = static_cast<long long int>(lda) * static_cast<long long int>(k);
  long long int batch_stride_B = static_cast<long long int>(k);
  long long int batch_stride_C = static_cast<long long int>(ldc) * static_cast<long long int>(n);

  // alpha and beta
  float alpha = 1.0f;
  float beta = 2.0f;

  hipError_t result = hipSuccess;

  // allocate the host memory
  std::vector<float> host_A(count_A);
  std::vector<float> host_B(count_B);
  std::vector<float> host_C(count_C);
  std::vector<float> result_C(count_C);

  // allocate the device memory
  float *A;
  float *B;
  float *C;

  result = hipMalloc(&A, count_A * sizeof(float));
  if (result != hipSuccess) {
    std::cerr << "hipMalloc result = " << result << std::endl;
    return result;
  }
  result = hipMalloc(&B, count_B * sizeof(float));
  if (result != hipSuccess) {
    std::cerr << "hipMalloc result = " << result << std::endl;
    return result;
  }
  result = hipMalloc(&C, count_C * sizeof(float));
  if (result != hipSuccess) {
    std::cerr << "hipMalloc result = " << result << std::endl;
    return result;
  }

  // fill A
  for (int b_idx = 0; b_idx < batch_count; b_idx++) {
    for (int col_idx = 0; col_idx < k; col_idx++) {
      for (int row_idx = 0; row_idx < m; row_idx++) {
        host_A[row_idx + col_idx * lda + b_idx * lda * k] = static_cast<float>(row_idx + col_idx * lda + b_idx * lda * k);
      }
    }
  }
  // fill B
  for (int b_idx = 0; b_idx < batch_count; b_idx++) {
    for (int col_idx = 0; col_idx < n; col_idx++) {
      for (int row_idx = 0; row_idx < k; row_idx++) {
        host_B[row_idx + col_idx * ldb + b_idx * k] = static_cast<float>(n + k * ldb + batch_count * k) - static_cast<float>(row_idx + col_idx * ldb + b_idx * k);
      }
    }
  }
  // fill C
  for (int b_idx = 0; b_idx < batch_count; b_idx++) {
    for (int col_idx = 0; col_idx < n; col_idx++) {
      for (int row_idx = 0; row_idx < m; row_idx++) {
        host_C[row_idx + col_idx * ldc + b_idx * ldc * n] = 1.f;
      }
    }
  }

  // ref memory
  std::vector<float> ref_A(host_A);
  std::vector<float> ref_B(host_B);
  std::vector<float> ref_C(host_C);
  // copy host memory to device
  result = hipMemcpy(A, host_A.data(), count_A * sizeof(float), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }
  result = hipMemcpy(B, host_B.data(), count_B * sizeof(float), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }
  result = hipMemcpy(C, host_C.data(), count_C * sizeof(float), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }

  // run cutlass
  result = cutlass_strided_batched_sgemm(A, lda, batch_stride_A, B, ldb, batch_stride_B, C, ldc, batch_stride_C,
    alpha, beta, m, n, k, batch_count);
  if (result != hipSuccess)
    return result;

  // copy device memory to host
  result = hipMemcpy(result_C.data(), C, count_C * sizeof(float), hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    std::cerr << "hipMemcpy result = " << result << std::endl;
    return result;
  }

  //compare with reference code
  result = strided_batched_gemm_nn_reference(ref_A, lda, batch_stride_A, ref_B, ldb, batch_stride_B, ref_C, ldc, batch_stride_C,
    alpha, beta, m, n, k, batch_count);
  if (result != 0)
    return result;

  if (ref_C != result_C) {
    std::cout << "CUTLASS strided batched gemm does not run correctly" << std::endl;
    return hipErrorUnknown;
  }

  // free memory
  result = hipFree(A);
  if (result != hipSuccess) {
    std::cerr << "hipFree result = " << result << std::endl;
    return result;
  }
  result = hipFree(B);
  if (result != hipSuccess) {
    std::cerr << "hipFree result = " << result << std::endl;
    return result;
  }
  result = hipFree(C);
  if (result != hipSuccess) {
    std::cerr << "hipFree result = " << result << std::endl;
    return result;
  }


  if (result == hipSuccess) {
    std::cout << "Passed." << std::endl;
  }

  // Exit.
  return result == hipSuccess ? 0 : -1;
}
