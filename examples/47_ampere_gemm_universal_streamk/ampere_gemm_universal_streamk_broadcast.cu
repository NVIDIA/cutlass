#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/***************************************************************************************************
 Example contrasting the Stream-K parallel decomposition for GEMM threadblocks versus the
 "classic data-parallel" and "Split-K" decompositions + residual add.

 For more details regarding the Stream-K method, see "Stream-K: Work-centric Parallel Decomposition
 for Dense Matrix-Matrix Multiplication on the GPU" (https://arxiv.org/abs/2301.03598)

 Requires NVIDIA Ampere or newer device (SM80+).

 - To lock persistence mode, power (400W), clocks (1005MHz) for evaluation (assumes device 0 and A100)

     cutlass$ sudo nvidia-smi -pm 1 -i 0

     cutlass$ sudo nvidia-smi -i 0 -pl 400

     cutlass$ sudo nvidia-smi -i 0 -lgc 1005

 - Build and run:

     cutlass$ mkdir build

     cutlass$ cd build

     cutlass/build$ cmake .. -DCUTLASS_NVCC_ARCHS=80

     cutlass/build$ make 47_ampere_gemm_universal_streamk_broadcast

     cutlass/build$ ./examples/47_ampere_gemm_universal_streamk/47_ampere_gemm_universal_streamk_broadcast

 - Reset clocks when done:

     cutlass$ sudo nvidia-smi -rgc

 **************************************************************************************************/

#include <iostream>
#include <string>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm_universal_with_broadcast.h"
#include "cutlass/gemm/device/gemm_universal_streamk_with_broadcast.h"
#include "cutlass/epilogue/thread/linear_combination_residual_block.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/error_metrics.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_foreach.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"



/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations (cutlass_tensorop_h16816gemm_128x128_32x4_nn_align8)
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA         = cutlass::half_t;                                  // Element type for A matrix operand
using         LayoutA          = cutlass::layout::RowMajor;                        // Layout type for A matrix operand
constexpr int AlignmentA       = 128 / cutlass::sizeof_bits<ElementA>::value;      // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB         = cutlass::half_t;                                  // Element type for B matrix operand
using         LayoutB          = cutlass::layout::RowMajor;                        // Layout type for B matrix operand
constexpr int AlignmentB       = 128 / cutlass::sizeof_bits<ElementB>::value;      // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C1/C2/D matrix configuration
using         ElementC         = cutlass::half_t;                                  // Element type for C matrix operands
using         LayoutC          = cutlass::layout::RowMajor;                        // Layout type for C matrix operands
constexpr int AlignmentC       = 128 / cutlass::sizeof_bits<ElementC>::value;      // Memory access granularity/alignment of C matrices in units of elements (up to 16 bytes)

// Output matrix configuration
using         ElementOutput    = cutlass::half_t;                                  // Element type for output matrix operands
using         LayoutOutput     = cutlass::layout::RowMajor;                        // Layout type for output matrix operands
// constexpr int AlignmentOutput  = 128 / cutlass::sizeof_bits<ElementOutput>::value; // Memory access granularity/alignment of output matrices in units of elements (up to 16 bytes)

// Multiply-accumulate blocking/pipelining details
using ElementAccumulator  = cutlass::half_t;                          // Element type for internal accumulation
using ElementCompute      = cutlass::half_t;                          // Element type for compute
using ArchTag             = cutlass::arch::Sm80;                      // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;           // Operator class tag
using ThreadblockShape    = cutlass::gemm::GemmShape<128, 128, 32>;   // Threadblock-level tile size (concept: GemmShape)
using WarpShape           = cutlass::gemm::GemmShape<64, 64, 32>;     // Warp-level tile size (concept: GemmShape)
using InstructionShape    = cutlass::gemm::GemmShape<16, 8, 16>;      // Instruction-level tile size (concept: GemmShape)
constexpr int NumStages   = 4;                                        // Number of global->shared pipeline stages used in the GEMM mainloop

// Residual block configuration

// Epilogue output operator
/// Using LinearCombinationResidualBlock
/// Models a residual block of the form: UnaryOp(BinaryOp(BinaryOp(ActivationOp(TensorOp(X) + bias), residual1), residual2))
using EpilogueOp = cutlass::epilogue::thread::LinearCombinationResidualBlock<  
    ElementOutput,                        // Element type for output matrix
    ElementAccumulator,                   // Element type from internal accumulation
    ElementCompute,                       // Element type from internal accumulation
    ElementC,                             // Element type for C1/C2/D matrix operands
    AlignmentC,                           // Memory access granularity of C and D matrix in units of elements
    cutlass::epilogue::thread::Identity,  // Activation
    cutlass::plus,                        // Binary operation 1
    cutlass::epilogue::thread::Identity,  // Unary operation
    cutlass::plus                         // Binary operation 2
    >;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

// Classic data-parallel device GEMM implementation type
using DeviceGemmBasic = cutlass::gemm::device::GemmUniversalWithBroadcast<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    NumStages,
    AlignmentA,
    AlignmentB>;

// StreamK device GEMM implementation type
using DeviceGemmStreamK = cutlass::gemm::device::GemmUniversalStreamkWithBroadcast<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::ThreadblockSwizzleStreamK,
    NumStages,
    AlignmentA,
    AlignmentB>;


/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(true)
  {}

};


/// Command line options parsing
struct Options
{
  std::string               command_name;
  bool                      help;
  cutlass::gemm::GemmCoord  problem_size;
  float                     alpha;
  float                     beta;
  int                       split_k_factor;
  int                       avail_sms;
  int                       iterations;
  bool                      real;

  cutlass::HostTensor<ElementA, LayoutA> tensor_a;
  cutlass::HostTensor<ElementB, LayoutB> tensor_b;
  cutlass::HostTensor<ElementC, LayoutC> tensor_c1;
  cutlass::HostTensor<ElementC, LayoutC> tensor_c2;
  cutlass::HostTensor<ElementC, LayoutC> tensor_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_ref_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_Vector;
  // cutlass::HostTensor<ElementC, LayoutC> tensor_Tensor;

  Options(std::string command_name) :
    command_name(command_name),
    help(false),
    problem_size({2048, 2048, 2048}),
    alpha(1.0f),
    beta(1.0f),
    split_k_factor(1),
    avail_sms(-1),              // Number of device SMs to use is unlimited
    real(false),
    iterations(10000)
  {}

  bool valid() const
  {
    return true;
  }

  void parse(int argc, char const **args)
  {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("split", split_k_factor);
    cmd.get_cmd_line_argument("iterations", iterations);
    real = cmd.check_cmd_line_flag("real");
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const
  {
    out
      << "Performs a GEMM computation.\n"
      << "\n"
      << "Options:\n"
      << "\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --split=<int>               Split-K factor to emulate\n\n"
      << "  --real                      If specified, initializes with real values instead of whole numbers. Errors are to be expected.\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << command_name << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    return 2.0 * double(problem_size.product()) / double(1.0e9) / runtime_s;
  }
};


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Populates a DeviceGemmBasic::Arguments structure from the given commandline options
typename DeviceGemmBasic::Arguments args_from_options(
    const DeviceGemmBasic &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c1,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c2,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_Vector /*,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_Tensor */
    )
{
  return typename DeviceGemmBasic::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c1.device_data(),                  // ptr_C1
    tensor_c2.device_data(),                  // ptr_C2
    tensor_d.device_data(),                   // ptr_D
    tensor_Vector.device_data(),              // ptr_Vector
    /* tensor_Tensor.device_data(), */nullptr,// ptr_Tensor
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C1
    options.problem_size.mn().product(),      // batch_stride_C2
    options.problem_size.mn().product(),      // batch_stride_D
    options.problem_size.mn().product(),      // batch_stride_Vector
    options.problem_size.mn().product(),      // batch_stride_Tensor
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c1.layout().stride(0),             // stride_c1
    tensor_c2.layout().stride(0),             // stride_c2
    tensor_d.layout().stride(0),              // stride_d
    /*tensor_Vector.layout().stride(0)*/0,    // stride_Vector
    /*tensor_Tensor.layout().stride(0)*/0);   // stride_Tensor
}

/// Populates a DeviceGemmStreamK::Arguments structure from the given commandline options
typename DeviceGemmStreamK::Arguments args_from_options(
    const DeviceGemmStreamK &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c1,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c2,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_Vector/*,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_Tensor*/
    )
{
  return typename DeviceGemmStreamK::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c1.device_data(),                  // ptr_C1
    tensor_c2.device_data(),                  // ptr_C2
    tensor_d.device_data(),                   // ptr_D
    tensor_Vector.device_data(),              // ptr_Vector
    /* tensor_Tensor.device_data(), */nullptr,// ptr_Tensor    // We're not storing Tensor
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C1
    options.problem_size.mn().product(),      // batch_stride_C2
    options.problem_size.mn().product(),      // batch_stride_D
    options.problem_size.mn().product(),      // batch_stride_Vector
    options.problem_size.mn().product(),      // batch_stride_Tensor
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c1.layout().stride(0),             // stride_c1
    tensor_c2.layout().stride(0),             // stride_c2
    tensor_d.layout().stride(0),              // stride_d
    /*tensor_Vector.layout().stride(0)*/0,    // stride_Vector // Vector stride is always 0
    /*tensor_Tensor.layout().stride(0)*/0,    // stride_Tensor // We're not storing Tensor
    options.avail_sms);                       // avail_sms
}

/// Execute a given example GEMM computation
template <typename DeviceGemmT>
Result run(std::string description, Options &options)
{
  // Display test description
  std::cout << std::endl << description << std::endl;

  // Zero-initialize test output matrix D
  cutlass::reference::host::TensorFill(options.tensor_d.host_view());
  options.tensor_d.sync_device();

  // Instantiate CUTLASS kernel depending on templates
  DeviceGemmT device_gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of DeviceGemmT
  auto arguments = args_from_options(device_gemm, options, 
      options.tensor_a, options.tensor_b, options.tensor_c1, options.tensor_c2, options.tensor_d, 
      options.tensor_Vector/*, options.tensor_Tensor*/);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = DeviceGemmT::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  CUTLASS_CHECK(device_gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(device_gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(device_gemm());

  // Copy output data from CUTLASS and reference kernel to host for comparison
  options.tensor_d.sync_host();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = cutlass::reference::host::TensorEquals(
    options.tensor_d.host_view(),
    options.tensor_ref_d.host_view());

  double err = cutlass::reference::host::TensorRelativeErrorMetric(
    options.tensor_d.host_view(),
    options.tensor_ref_d.host_view());

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << " \t Relative error: " << err << std::endl;

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(device_gemm());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPs: " << result.gflops << std::endl;
  }

  // TODO: uncomment when results match
  //if (!result.passed) {
  //  exit(-1);
  //}

  return result;
}


/// Program entrypoint
int main(int argc, const char **argv)
{
  // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  // Current device must must have compute capability at least 80
  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  if (!((props.major * 10 + props.minor) >= 80))
  {
    std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  // Parse commandline options
  Options options("ampere_streamk_broadcast_gemm");
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  std::cout <<
    options.iterations << " timing iterations of " <<
    options.problem_size.m() << " x " <<
    options.problem_size.n() << " x " <<
    options.problem_size.k() << " matrix-matrix multiply" << std::endl;

  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }


  //
  // Initialize GEMM datasets
  //

  // Initialize tensors using CUTLASS helper functions
  options.tensor_a.resize(options.problem_size.mk());           // <- Create matrix A with dimensions M x K
  options.tensor_b.resize(options.problem_size.kn());           // <- Create matrix B with dimensions K x N
  options.tensor_c1.resize(options.problem_size.mn());          // <- Create matrix C1 with dimensions M x N
  options.tensor_c2.resize(options.problem_size.mn());          // <- Create matrix C2 with dimensions M x N
  options.tensor_d.resize(options.problem_size.mn());           // <- Create matrix D with dimensions M x N used to store output from CUTLASS kernel
  options.tensor_ref_d.resize(options.problem_size.mn());       // <- Create matrix D with dimensions M x N used to store output from reference kernel
  options.tensor_Vector.resize({1, options.problem_size.n()});  // <- Create broadcast vector with dimensions N x 1
  // options.tensor_Tensor.resize(options.problem_size.mn());   // <- Create T matrix with dimensions M x N

  int _init_bits = options.real ? -1 : 0;

  // Fill matrix A on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_a.host_view(),
      1,
      ElementA(2),
      ElementA(-2), _init_bits);

  // Fill matrix B on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_b.host_view(),
      1,
      ElementB(2),
      ElementB(-2), _init_bits);

  // Fill matrix C1 on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_c1.host_view(),
      1,
      ElementC(2),
      ElementC(-2), _init_bits);

  // Fill matrix C2 on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_c2.host_view(),
      1,
      ElementC(2),
      ElementC(-2), _init_bits);

  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_Vector.host_view(),
      1,
      ElementC(2),
      ElementC(-2), _init_bits);

  //
  // Compute reference output
  //

  // Copy data from host to GPU
  options.tensor_a.sync_device();
  options.tensor_b.sync_device();
  options.tensor_c1.sync_device();
  options.tensor_c2.sync_device();
  options.tensor_Vector.sync_device();
  // options.tensor_Tensor.sync_device();

  // Zero-initialize reference output matrix D
  cutlass::reference::host::TensorFill(options.tensor_ref_d.host_view());
  options.tensor_ref_d.sync_device();

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    options.problem_size,
    ElementAccumulator(options.alpha),
    options.tensor_a.device_ref(),
    options.tensor_b.device_ref(),
    ElementAccumulator(options.beta),
    options.tensor_c1.device_ref(),
    options.tensor_ref_d.device_ref());

  // Wait for kernels to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy output data from reference kernel to host for comparison
  options.tensor_ref_d.sync_host();

  // Add broadcast vector (without multiplier)
  // This is only possible because BinaryOp is addition, and UnaryOps are identity.
  // This makes the addition of broadcast vector commutable.
  /// identity(plus(identity(alpha * (a * b) + v), beta * c)) ==
  /// alpha * a * b + v + beta * c                            ==
  /// (alpha * a * b + beta * c) + v                          ==
  /// GEMM(a, b, c) + v
  // Vector broadcast on host
  for (int i=0; i < options.problem_size.m(); ++i) {
    for (int j=0; j < options.problem_size.n(); ++j) {
      options.tensor_ref_d.host_view().ref().at({i, j}) += options.tensor_Vector.host_view().ref().at({0, j});
      options.tensor_ref_d.host_view().ref().at({i, j}) += options.tensor_c2.host_view().ref().at({i, j});
    }
  }

  // Sync back with device just in case
  options.tensor_ref_d.sync_device();

  //
  // Evaluate CUTLASS kernels
  //

  // Test default operation
  if (options.split_k_factor == 1)
  {
    // Compare basic data-parallel version versus StreamK version using default load-balancing heuristics
    Result basic_dp         = run<DeviceGemmBasic>("Basic data-parallel GEMM", options);
    Result streamk_default  = run<DeviceGemmStreamK>("StreamK GEMM with default load-balancing", options);

    printf("  Speedup vs Basic-DP: %.3f\n", (basic_dp.avg_runtime_ms / streamk_default.avg_runtime_ms));

    // Show that StreamK can emulate basic data-parallel GEMM when we set the number of SMs to load-balance across = 1
    options.avail_sms       = 1;        // Set loadbalancing width to 1 SM (no load balancing)
    Result streamk_dp       = run<DeviceGemmStreamK>("StreamK emulating basic data-parallel GEMM", options);
    options.avail_sms       = -1;       // Reset loadbalancing width to unspecified SMs (i.e., the number of device SMs)

    printf("  Speedup vs Basic-DP: %.3f\n", (basic_dp.avg_runtime_ms / streamk_dp.avg_runtime_ms));

    options.split_k_factor++;     // Increment splitting factor for next evaluation

  }

  // Show that StreamK can emulate "Split-K" with a tile-splitting factor
  Result basic_splitk = run<DeviceGemmBasic>(
    std::string("Basic split-K GEMM with tile-splitting factor ") + std::to_string(options.split_k_factor),
    options);

  Result streamk_splitk = run<DeviceGemmStreamK>(
    std::string("StreamK emulating Split-K GEMM with tile-splitting factor ") + std::to_string(options.split_k_factor),
    options);

  printf("  Speedup vs Basic-SplitK: %.3f\n", (basic_splitk.avg_runtime_ms / streamk_splitk.avg_runtime_ms));

  return 0;
}
