#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief A Hopper CUTLASS example for Flash MLA.
*/

#include <cassert>
#include <iostream>

#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/numeric_conversion.h>
#include "cutlass/transform/device/transform_universal_adapter.hpp"

#include <thrust/universal_vector.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cutlass/util/command_line.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include <hip/hip_runtime.h>

#include "flash_fwd_mla_kernel.h"
#include "flash_mla.h"
#include "fill_nan.h"
#include "transform.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

#define CUDA_CHECK(status)                                                                 \
    {                                                                                      \
        hipError_t error = status;                                                       \
        if (error != hipSuccess) {                                                       \
            std::cerr << "CUDA error: " << hipGetErrorString(error) << " at " <<        \
                __FILE__ << ":" << __LINE__ << std::endl;                                 \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  int iterations;
  int b, s, h_q, s_q;
  int h_kv, d, dv;
  float softmax_scale;
  bool varlen;
  bool causal;

  static constexpr int block_size = 64;

  Options():
    help(false),
    b(128), s(4096), h_q(16), s_q(1),
    h_kv(1), d(576), dv(512),
    varlen(false),
    causal(true),
    iterations(10)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);
    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("b", b, defaults.b);
    cmd.get_cmd_line_argument("s", s, defaults.s);
    cmd.get_cmd_line_argument("h_q", h_q, defaults.h_q);
    cmd.get_cmd_line_argument("s_q", s_q, defaults.s_q);
    cmd.get_cmd_line_argument("h_kv", h_kv, defaults.h_kv);
    cmd.get_cmd_line_argument("d", d, defaults.d);
    cmd.get_cmd_line_argument("dv", dv, defaults.dv);

    if (cmd.check_cmd_line_flag("varlen")) {
      varlen = true;
    }

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);

    softmax_scale = 1 / std::sqrt(d);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "79_hopper_flash_mla\n\n"
      << "  Hopper Flash MLA kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the batch size\n"
      << "  --s=<int>                   Sets the sequence length\n"
      << "  --h_q=<int>                 Sets the number of heads\n"
      << "  --s_q=<int>                 Sets the sequence length of the query\n"
      << "  --varlen                    Sets the varlen as true or false\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    return out;
  }

  /// TOOD:Compute performance in GFLOP

};

/// Helper to initialize a block of device data
template <typename Element>
static void
initialize_values(
    thrust::universal_vector<Element>& dst_ptr,
    cutlass::Distribution::Kind dist_kind,
    uint64_t seed,
    Element var = Element(1.f)) {
  if (cutlass::Distribution::Uniform == dist_kind) {
    int scope = 2;
    cutlass::reference::host::BlockFillRandomUniform(
        dst_ptr.data().get(), dst_ptr.size(), seed, scope, -scope, 0);
  }
  else if (cutlass::Distribution::AllZeros == dist_kind) {
    cutlass::reference::host::BlockFillRandomUniform(
        dst_ptr.data().get(), dst_ptr.size(), seed, 0, 0, 0);
  }
  else if (cutlass::Distribution::AllOnes == dist_kind) {
    cutlass::reference::host::BlockFillRandomUniform(
        dst_ptr.data().get(), dst_ptr.size(), seed, 1, 1, 0);
  } 
  else if (cutlass::Distribution::Gaussian == dist_kind) {
    cutlass::reference::device::BlockFillRandomGaussian(
      dst_ptr.data().get(), dst_ptr.size(), seed, (Element) 0, var);
  }
  else if (cutlass::Distribution::Sequential == dist_kind) {
    cutlass::reference::host::BlockFillSequential(dst_ptr.data().get(), dst_ptr.size());
  }
  else {
    std::cerr << "Invalid distribution kind!\n.";
    exit(1);
  }
}

void initialize_varlen(thrust::universal_vector<int32_t>& block_C, const Options &options) {
  
  block_C.resize(options.b);

  std::vector<int32_t> cache_seqlens(options.b, options.s);

  std::random_device rd;
  std::mt19937 gen(rd());

  std::normal_distribution<float> distribution(options.s, options.s / 2.0f);

  for (int i = 0; i < options.b; ++i) {
    if (options.varlen) {
      float random_length = distribution(gen);
      cache_seqlens[i] = std::max(static_cast<int32_t>(random_length), options.s_q);
    } else {
      cache_seqlens[i] = options.s;
    }
  }

  cutlass::DeviceAllocation<int32_t> d_cache_seqlens(options.b);
  CUDA_CHECK(hipMemcpy(
      block_C.data().get(),
      cache_seqlens.data(),
      options.b * sizeof(int32_t),
      hipMemcpyHostToDevice
  ));
}

auto initialize_metadata(
  thrust::universal_vector<int32_t> &block_C,
  thrust::universal_vector<int32_t> &block_MD, thrust::universal_vector<int32_t> &block_S,
  int& num_sm_parts,
  const Options &options) {

  // This should match the logic in the MLA kernel.
  static constexpr int block_size_m = 64;
  static constexpr int block_size_n = 64;
  static constexpr int fixed_overhead_num_blocks = 5;
  
  hipDeviceProp_t props;
  int current_device;
  CUDA_CHECK(hipGetDevice(&current_device));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device));

  auto batch_size = options.b;
  int sm_count = props.multiProcessorCount;

  num_sm_parts = sm_count / options.h_kv / cutlass::ceil_div(options.h_kv, block_size_m);

  block_MD.resize(num_sm_parts * TileSchedulerMetaDataSize);
  block_S.resize(options.b + 1);

  Mla_metadata_params params{};
  params.seqlens_k_ptr = block_C.data().get();
  params.tile_scheduler_metadata_ptr = block_MD.data().get();
  params.num_splits_ptr = block_S.data().get();
  params.batch_size = batch_size;
  params.block_size_n = block_size_n;
  params.fixed_overhead_num_blocks = fixed_overhead_num_blocks;
  params.num_sm_parts = num_sm_parts;

  hipStream_t stream{nullptr};

  get_mla_metadata_func(params, stream);
}

// only transpose the dimensions 2 and 3
template <class Element>
void transpose(
  thrust::universal_vector<Element> &block_S,
  thrust::universal_vector<Element> &block_D,
  cute::tuple<int, int, int, int, int> problem_shape) {
  
  using Operator = cutlass::transform::device::TransformUniversalAdapter<TransposeKernel<Element>>;

  hipError_t result;
  result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    std::cerr << "Error running the Transpose kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
  }

  typename Operator::Arguments arguments{
    block_S.data().get(),
    block_D.data().get(),
    problem_shape,
  };

  Operator op;

  size_t workspace_size = Operator::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  cutlass::Status status = op.can_implement(arguments);
  if (status != cutlass::Status::kSuccess) {
    std::cerr << "This kernel is not supported. Last CUDA error is: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    return;
  }

  status = op.initialize(arguments, workspace.get());
  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    return;
  }

  status = op.run();
  if (status != cutlass::Status::kSuccess) {
    std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    return;
  }

  result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
              << hipGetErrorString(result) << std::endl;
    return;
  }
}

struct TestBed {
  using Element = cutlass::bfloat16_t;
  using ElementAcc = float;

  thrust::universal_vector<Element> block_Q;      // query
  thrust::universal_vector<Element> block_Q_T;    // query transpose
  thrust::universal_vector<Element> block_K;      // blocked key
  thrust::universal_vector<int32_t> block_T;      // block table
  thrust::universal_vector<int32_t> block_C;      // cache seqlens
  // TODO: block_V is not used in the example
  // thrust::universal_vector<Element> block_V;     // dv
  thrust::universal_vector<int32_t> block_MD;     // mla metadata
  thrust::universal_vector<int32_t> block_S;      // num splits
  thrust::universal_vector<Element> block_O;      // output
  thrust::universal_vector<Element> block_LSE;    // lse
  thrust::universal_vector<Element> block_O_T;    // output transpose
  thrust::universal_vector<Element> block_LSE_T;  // lse transpose
  thrust::universal_vector<ElementAcc> block_O_Accum;    // output
  thrust::universal_vector<ElementAcc> block_LSE_Accum;  // lse

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(
    const Options &options,
    int& total_blocks, int& blocks_per_seq, int& num_sm_parts,
    uint64_t seed = 2025) {

    initialize_varlen(block_C, options);

    thrust::device_ptr<int32_t> d_ptr(block_C.data().get());

    int64_t total_seqlens = thrust::reduce(d_ptr, d_ptr + options.b);
    float sum = static_cast<float>(total_seqlens);
    int32_t mean_seqlens = static_cast<int32_t>(sum / options.b);
    int32_t max_seqlen = thrust::reduce(d_ptr, d_ptr + options.b, 
                                        0, 
                                        thrust::maximum<int32_t>());
    int max_seqlen_pad = ((max_seqlen + 255) / 256) * 256;

    blocks_per_seq = max_seqlen_pad / options.block_size;
    total_blocks = options.b * blocks_per_seq;

    // Query: [b, s_q, h_q, d]
    block_Q.resize(options.b * options.s_q * options.h_q * options.d);
    block_Q_T.resize(options.b * options.s_q * options.h_q * options.d);

    // Block table: [b, max_num_blocks_per_seq]
    block_T.resize(total_blocks);

    // Key: [b, max_num_blocks_per_seq, block_size, h_kv, d]
    block_K.resize(total_blocks * options.block_size * options.h_kv * options.d);

    initialize_values(block_Q, cutlass::Distribution::Gaussian, seed + 1);
    initialize_values(block_T, cutlass::Distribution::Sequential, seed + 3);
    initialize_values(block_K, cutlass::Distribution::Gaussian, seed + 5);

    // Set the exceeding part to NaN
    fill_nan(block_K.data().get(), block_C.data().get(),
             options.b, max_seqlen_pad, options.h_kv, options.d);

    initialize_metadata(block_C, block_MD, block_S, num_sm_parts, options);

    int ngroups = options.h_q / options.h_kv;
    int num_heads = options.h_kv;
    int seqlen_q = options.s_q * ngroups;

    // LSE: [batch_size, num_heads, seqlen_q]
    block_LSE.resize(options.b * num_heads * seqlen_q);
    block_LSE_T.resize(options.b * seqlen_q * num_heads);
    // Output: [batch_size, seqlen_q, num_heads, head_size_v]
    block_O.resize(options.b * seqlen_q * num_heads * options.dv);
    block_O_T.resize(options.b * seqlen_q * num_heads * options.dv);

    auto softmax_lse_size = (options.b + num_sm_parts) * num_heads * seqlen_q;
    auto out_accum_size = (options.b + num_sm_parts) * num_heads * seqlen_q * options.dv;

    block_LSE_Accum.resize(softmax_lse_size);
    block_O_Accum.resize(out_accum_size);
  }

  /// Execute a given example Flash MLA computation
  void run(Options &options)
  {
    hipDeviceProp_t props;
    int current_device;
    CUDA_CHECK(hipGetDevice(&current_device));
    CUDA_CHECK(hipGetDeviceProperties(&props, current_device));

    // TODO: use vcache which is None in the example

    auto batch_size = options.b;
    auto seqlen_q_ori = options.s_q;
    auto num_heads_ori = options.h_q;
    auto head_size = options.d;
    auto head_size_v = options.dv;
    auto num_heads_k = options.h_kv;
    auto page_block_size = options.block_size;
    int total_blocks, max_num_blocks_per_seq;
    int num_sm_parts;

    assert(head_size % 8 == 0);
    assert(head_size_v % 32 == 0);

    initialize(options, total_blocks, max_num_blocks_per_seq, num_sm_parts);

    assert(batch_size > 0);
    assert(num_heads_ori % num_heads_k == 0);

    bool is_causal = seqlen_q_ori == 1 ? false : options.causal;

    int ngroups = num_heads_ori / num_heads_k;
    int seqlen_q = seqlen_q_ori * ngroups;
    int num_heads = num_heads_k;

    // preprocess the query
    transpose(
      block_Q, block_Q_T,
      cute::make_shape(options.b, seqlen_q_ori, num_heads_k, ngroups, options.d));

    hipStream_t stream{nullptr};

    // set the parameters
    Flash_fwd_mla_params kernel_params{};

    kernel_params.b = options.b;
    kernel_params.seqlen_q = options.s_q;
    kernel_params.d = options.d;
    kernel_params.d_v = options.dv;
    kernel_params.h = options.h_q;
    kernel_params.h_h_k_ratio = num_heads_ori / num_heads_k;
    kernel_params.ngroups = ngroups;

    kernel_params.q_ptr = block_Q_T.data().get();
    kernel_params.k_ptr = block_K.data().get();
    // TODO: block_V is not used in the example
    kernel_params.v_ptr = block_K.data().get();
    kernel_params.o_ptr = block_O.data().get();
    kernel_params.softmax_lse_ptr = block_LSE.data().get();

    kernel_params.q_batch_stride = seqlen_q * num_heads * options.d;
    kernel_params.k_batch_stride = page_block_size * options.h_kv * options.d;
    kernel_params.v_batch_stride = page_block_size * options.h_kv * options.dv;
    kernel_params.o_batch_stride = options.s_q * options.h_q * options.dv;

    kernel_params.q_row_stride = num_heads * options.d;
    kernel_params.k_row_stride = options.h_kv * options.d;
    kernel_params.v_row_stride = options.h_kv * options.dv;
    kernel_params.o_row_stride = options.h_q * options.dv;

    kernel_params.q_head_stride = options.d;
    kernel_params.k_head_stride = options.d;
    kernel_params.v_head_stride = options.dv;
    kernel_params.o_head_stride = options.dv; 

    kernel_params.block_table = block_T.data().get();
    kernel_params.block_table_batch_stride = max_num_blocks_per_seq;
    kernel_params.page_block_size = page_block_size;

    kernel_params.tile_scheduler_metadata_ptr = block_MD.data().get();
    kernel_params.num_splits_ptr = block_S.data().get(); 

    kernel_params.softmax_lseaccum_ptr = block_LSE_Accum.data().get();
    kernel_params.oaccum_ptr = block_O_Accum.data().get();

    kernel_params.is_causal = is_causal;
    kernel_params.scale_softmax = options.softmax_scale;
    kernel_params.scale_softmax_log2 = std::log2(options.softmax_scale);

    kernel_params.cu_seqlens_k = block_C.data().get();

    kernel_params.num_sm_parts = num_sm_parts;

    assert(head_size == 576);
    run_mha_fwd_splitkv_mla<cutlass::bfloat16_t, 576>(kernel_params, stream);

    CUDA_CHECK(hipDeviceSynchronize());

    transpose(
      block_O, block_O_T,
      cute::make_shape(options.b, seqlen_q_ori, ngroups, num_heads_k, options.dv));
    transpose(
      block_LSE, block_LSE_T,
      cute::make_shape(options.b, num_heads_k, seqlen_q_ori, ngroups, 1));

    // TODO: reference check

    printf("run done\n");
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 7)) {
    std::cerr << "This example requires CUDA 12.7 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major != 9 || props.minor != 0) {
    std::cerr << "This example requires a GPU with compute capability 90)." << std::endl;
    return 0;
  }
  
  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //
  TestBed testbed{};
  testbed.run(options);

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
