#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

// Referenced: 67_hopper_fp8_warp_specialized_gemm_with_blockwise_scaling

/*! \file
    \brief Blocked scale Ampere FP8 GEMM example using CUTLASS 3.0 APIs for NVIDIA Ampere Lovelace architecture.

    This example demonstrate a grouped scaled FP8 GEMM using the new CUTLASS 3.0.
    APIs on NVIDIA Ampere architecture. New features that will be showcased in this example are as follows:

    Examples:
      $ ./examples/85_ada_ampere_gemm_with_blockwise_scaling/85d_ampere_int8_gemm_with_blockwise_scaling_cute  \
        --m=2816 --n=3072 --k=16384
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/activation.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"

#include "helper.h"

using namespace cute;


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = int8_t;                                         // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = int8_t;                                         // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C matrix configuration
using         ElementC    = int;                                            // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = AlignmentC;

using ArchTag       = cutlass::arch::Sm89;                          // Tag indicating the minimum SM that supports the intended feature
using TileShape     = Shape<_128,_128,_128>;                        // Threadblock-level tile size
//using ClusterShape  = Shape<_1,_2,_1>;                              // Shape of the threadblocks in a cluster
using ScaleConfig = decltype(cutlass::detail::sm90_trivial_blockwise_scale_config(TileShape{}));

using LayoutSFA     = decltype(ScaleConfig::deduce_layoutSFA());    // Layout type for SFA matrix operand
using LayoutSFB     = decltype(ScaleConfig::deduce_layoutSFB());    // Layout type for SFB matrix operand

//
// Assembling the CollectiveMainloop type
//

// Number of pipelines you want to use
constexpr int PipelineStages = 3;
using DispatchPolicy = cutlass::gemm::MainloopSm80CpAsyncBlockScaling<PipelineStages>;

// This code section describes the MMA op and the tile size a warp will compute
using TiledMma = TiledMMA<
  MMA_Atom<SM80_16x8x32_S32S8S8S32_TN>,
  Layout<Shape<_2,_2,_1>>, // 2x2x1 thread group
  Tile<_32,_32,_32>>;      // 16x16x32 MMA for LDSM, 1x2x1 value group

// A (M,K)  K-major
using SmemLayoutAtomA = decltype(
  composition(
    Swizzle<2,4,3>{},
    Layout<Shape <_32,_64>,
            Stride<_64, _1>>{}));
using GmemTiledCopyA = decltype(
  make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, ElementA>{},
                  Layout<Shape <_32,_4>,
                          Stride< _4,_1>>{},
                  Layout<Shape<_1,Int<AlignmentA>>>{}));
// LDS.32- or LDSM-based copy atom
using SmemCopyAtomA = Copy_Atom<SM75_U32x4_LDSM_N, ElementA>;  // LDSM works

// B (N,K)  K-major
using SmemLayoutAtomB = decltype(
  composition(
    Swizzle<2,4,3>{},
    Layout<Shape <_16,_64>,
            Stride<_64, _1>>{}));
using GmemTiledCopyB = decltype(
  make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, ElementB>{},
                  Layout<Shape <_32,_4>,
                          Stride< _4,_1>>{},
                  Layout<Shape<_1,Int<AlignmentB>>>{}));

// LDS.32- or LDSM-based copy atom
using SmemCopyAtomB = Copy_Atom<SM75_U32x4_LDSM_N, ElementB>;  // LDSM works

// Mainloop
using CollectiveMainloop = cutlass::gemm::collective::CollectiveMma<
        DispatchPolicy, TileShape,
        ElementA,
        cute::tuple<cutlass::detail::TagToStrideA_t<LayoutA>, LayoutSFA>,
        ElementB,
        cute::tuple<cutlass::detail::TagToStrideB_t<LayoutB>, LayoutSFB>,
        TiledMma,
        GmemTiledCopyA, SmemLayoutAtomA, SmemCopyAtomA, cute::identity,  // A
        GmemTiledCopyB, SmemLayoutAtomB, SmemCopyAtomB, cute::identity   // B
>;

using ElementAccumulator  = typename CollectiveMainloop::ElementAccumulator;    // Element type for internal accumulation
using ElementBlockScale   = typename CollectiveMainloop::ElementBlockScale;     // Element type for blockscaling during accumulation
using ElementCompute      = ElementAccumulator;                                 // Element type for epilogue computation

//
// Assembling the Collective Epilogue Type
//

using CollectiveEpilogue = cutlass::epilogue::collective::DefaultEpilogue<
        ElementD,
        cutlass::detail::TagToStrideC_t<LayoutC>,
        cutlass::detail::TagToStrideC_t<LayoutD>,
        cutlass::epilogue::thread::LinearCombination<
          ElementD,
          AlignmentD,
          ElementAccumulator,
          ElementCompute>,
        cutlass::gemm::EpilogueDefault>;

//
// Assembling the GemmKernel
//

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>,
    CollectiveMainloop,
    CollectiveEpilogue>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Extract information from Gemm kernel.
using EpilogueOutputOp  = typename Gemm::EpilogueOutputOp;
using ElementScalar     = typename EpilogueOutputOp::ElementScalar;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

static_assert(cute::is_same_v<ElementAccumulator, ElementBlockScale>,
             "ElementAccumulator and ElementBlockScale should be same datatype");

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
LayoutSFA layout_SFA;
LayoutSFB layout_SFB;
uint64_t seed;

using LayoutScalar = cutlass::layout::PackedVectorLayout;

cutlass::HostTensor<ElementA  , LayoutA  > tensor_A;
cutlass::HostTensor<ElementB  , LayoutB  > tensor_B;
cutlass::HostTensor<ElementC  , LayoutC  > tensor_C;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_D;
cutlass::HostTensor<ElementBlockScale, LayoutScalar> blockscale_tensor_A;
cutlass::HostTensor<ElementBlockScale, LayoutScalar> blockscale_tensor_B;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_ref_D;

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;
  bool verify = true;

  ElementAccumulator alpha = 1, beta = 0;
#ifdef FP8_DEBUG
  int iterations = 0;
  int warmup = 0;
  int m = 1024, n = 128, k = 128, l = 1;
#else
  int iterations = 1000;
  int warmup = 1000;
  int m = 1024, n = 512, k = 1024, l = 1;
#endif
  float epsilon = 0.02f;
  float non_zero_floor = 1.f;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha, 1);
    cmd.get_cmd_line_argument("beta", beta, 0);
    cmd.get_cmd_line_argument("warmup", warmup);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("verify", verify);
    cmd.get_cmd_line_argument("epsilon", epsilon);
    cmd.get_cmd_line_argument("non-zero-floor", non_zero_floor);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "85d_ampere_int8_gemm_with_blockwise_scaling_cute\n\n"
      << "  Ampere INT8 GEMM using CUTLASS 3.0 API with Blockwise Scaling.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the l extent (batch) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n"
      << "  --verify=<bool>             Verify the results.\n\n"
      << "  --epsilon=<float>           The epsilon value for comparing the results.\n\n"
      << "  --non-zero-floor=<float>    The none zero floor for comparing the results.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "85d_ampere_int8_gemm_with_blockwise_scaling_cute" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;
    int bits_input = cutlass::sizeof_bits<Element>::value;
    int bits_output = cutlass::sizeof_bits<Element>::value;

    if (bits_input == 1) {
      scope_max = 2;
      scope_min = 0;
    } else if (bits_input <= 8) {
      scope_max = 2;
      scope_min = -2;
    } else if (bits_output == 16) {
      scope_max = 5;
      scope_min = -5;
    } else {
      scope_max = 8;
      scope_min = -8;
    }

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min, bits_input);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Helper to initialize a block of device data (scale_tensors)
template <typename Element, typename Layout>
bool initialize_scale_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;

    scope_min = -1;
    scope_max = 1;

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  assert(options.m % CollectiveMainloop::ScaleGranularityM == 0);
  assert(options.n % CollectiveMainloop::ScaleGranularityN == 0);

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));
  layout_SFA = ScaleConfig::tile_atom_to_shape_SFA(make_shape(options.m, options.n, options.k, options.l));
  layout_SFB = ScaleConfig::tile_atom_to_shape_SFB(make_shape(options.m, options.n, options.k, options.l));


  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto groupscale_a_coord = cutlass::make_Coord(size(filter_zeros(layout_SFA)));
  auto groupscale_b_coord = cutlass::make_Coord(size(filter_zeros(layout_SFB)));

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  blockscale_tensor_A.resize(groupscale_a_coord);
  blockscale_tensor_B.resize(groupscale_b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  cutlass::Distribution::Kind dist_A = cutlass::Distribution::Uniform;
  cutlass::Distribution::Kind dist_B = cutlass::Distribution::Uniform;
  cutlass::Distribution::Kind dist_C = cutlass::Distribution::Identity;
  cutlass::Distribution::Kind dist_scaleA = cutlass::Distribution::Uniform;
  cutlass::Distribution::Kind dist_scaleB = cutlass::Distribution::Uniform;

  initialize_tensor(tensor_A.host_view(), dist_A, seed + 2022);
  initialize_tensor(tensor_B.host_view(), dist_B, seed + 2023);
  initialize_tensor(tensor_C.host_view(), dist_C, seed + 2024);
  initialize_scale_tensor(blockscale_tensor_A.host_view(), dist_scaleA, seed + 2025);
  initialize_scale_tensor(blockscale_tensor_B.host_view(), dist_scaleB, seed + 2026);

#if 0 // Dump blockscaled tensors
  std::cout << "blockscale_tensor_A: " << groupscale_a_coord << std::endl;
  std::cout << blockscale_tensor_A.host_view() << "\n";
  std::cout << "blockscale_tensor_B: " << groupscale_b_coord << std::endl;
  std::cout << blockscale_tensor_B.host_view() << "\n";
#endif

  // Print group scaling tensors on the host side.
  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_C.sync_device();
  tensor_D.sync_device();
  blockscale_tensor_A.sync_device();
  blockscale_tensor_B.sync_device();
}

/// Populates a Gemm::Arguments structure from the given commandline options
template<typename GemmArguments>
GemmArguments args_from_options(const Options &options)
{
  GemmArguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(),
     stride_A,
     tensor_B.device_data(),
     stride_B,
     blockscale_tensor_A.device_data(),
     layout_SFA,
     blockscale_tensor_B.device_data(),
     layout_SFB
     },
    {
      {}, // epilogue.thread
      tensor_C.device_data(), stride_C,
      tensor_D.device_data(), stride_D
    }
  };

  auto &fusion_args = arguments.epilogue.thread;

  fusion_args.alpha = options.alpha;
  fusion_args.beta = options.beta;

  return arguments;
}

/// Don't know why the compiler does not like verify() being templated...
bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
                             cute::make_layout(
                                cute::make_shape(options.m, options.k, options.l),
                                stride_A
                              )
                            );
  auto B = cute::make_tensor(tensor_B.host_data(),
                             cute::make_layout(
                               cute::make_shape(options.n, options.k, options.l),
                               stride_B
                              )
                            );
  auto C = cute::make_tensor(tensor_C.host_data(),
                             cute::make_layout(
                                cute::make_shape(options.m, options.n, options.l),
                                stride_C
                              )
                            );
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
                             cute::make_layout(
                                cute::make_shape(options.m, options.n, options.l),
                                stride_D
                              )
                            );

  auto SFA = cute::make_tensor(blockscale_tensor_A.host_data(), layout_SFA);
  auto SFB = cute::make_tensor(blockscale_tensor_B.host_data(), layout_SFB);

  using unused_t = decltype(D);

  cutlass::reference::host::GettBlockScalingMainloopParams<
      ElementAccumulator,
      decltype(A),
      decltype(SFA),
      decltype(B),
      decltype(SFB)
    > mainloop_params{A, SFA, B, SFB};

  cutlass::reference::host::GettEpilogueParams<
      ElementScalar,
      ElementScalar,
      ElementAccumulator,
      ElementCompute,
      decltype(C),
      decltype(D),
      unused_t, // bias
      unused_t, // aux
      unused_t, // valpha
      unused_t  // vbeta
  > epilogue_params;

  epilogue_params.C = C;
  epilogue_params.D = D;
  epilogue_params.alpha = options.alpha;
  epilogue_params.beta = options.beta;

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  // compare_reference
  bool passed = true;
  tensor_D.sync_host();
  passed &= cutlass::reference::host::TensorRelativelyEquals(tensor_D.host_view(), tensor_ref_D.host_view(), ElementD(options.epsilon), ElementD(options.non_zero_floor));
  double mse = cutlass::reference::host::TensorMSE(tensor_D.host_view(), tensor_ref_D.host_view());
  double mre = cutlass::reference::host::TensorMRE(tensor_D.host_view(), tensor_ref_D.host_view());
  double max_error = cutlass::reference::host::TensorGreatestError(tensor_D.host_view(), tensor_ref_D.host_view());
  std::cout << "  Result MSE: " << mse << ", MRE: " << mre << ", greatest error: " << max_error << std::endl;

#if 0
  std::cout << "tensor_ref_D.host_view() {" << std::endl
            << tensor_ref_D.host_view() << std::endl
            << "}"  << std::endl;
  std::cout << "tensor_D.host_view() {" << std::endl
            << tensor_D.host_view() << std::endl
            << "}"  << std::endl;
#endif

  return passed;
}


/// Execute a given example GEMM computation
int run(Options &options) {

  bool skip = false;
  std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
  std::cout << "  Tile shape (M, N, K): " << size<0>(TileShape{}) << ", " << size<1>(TileShape{}) << ", " << size<2>(TileShape{}) << std::endl;
  std::cout << "  ScaleGranularityM: " << CollectiveMainloop::ScaleGranularityM << " (ScaleMsPerTile: " << CollectiveMainloop::ScaleMsPerTile << ")" << std::endl;
  std::cout << "  ScaleGranularityN: " << CollectiveMainloop::ScaleGranularityN << " (ScaleNsPerTile: " << CollectiveMainloop::ScaleNsPerTile << ")" << std::endl;


  if (options.m < CollectiveMainloop::ScaleGranularityM) {
    std::cout << "  Skippig (m size: " << options.m << " less than ScaleGranularityM: " << CollectiveMainloop::ScaleGranularityM << "):" << std::endl;
    skip = true;
  }

  if (options.n < CollectiveMainloop::ScaleGranularityN) {
    std::cout << "  Skippig (n size: " << options.n << " less than ScaleGranularityN: " << CollectiveMainloop::ScaleGranularityN << "):" << std::endl;
    skip = true;
  }

  if (options.k < size<2>(TileShape{})) {
    std::cout << "  Skippig (k size: " << options.k << " less than TileShape[2]: " << size<2>(TileShape{}) << "):" << std::endl;
    skip = true;
  }

  if (!skip) std::cout << "  Running... " << std::endl;
  else return -1;

  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<typename Gemm::Arguments>(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  if (options.verify) {
    result.passed = verify(options);

    std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;
  }
  else {
    result.passed = true;
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    for (int iter = 0; iter < options.warmup + options.iterations; ++iter) {
      if (iter == options.warmup)
        timer.start();
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
    fflush(stdout);
  }

  return result.passed;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 8) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Ampere Architecture or "
      << "later (compute capability 80 or greater).\n";
    return 0;
  }
  //
  // Parse options
  //

  Options options;
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

  bool passed = true;
  passed = run(options);
  if (!passed)
    return -1;

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
