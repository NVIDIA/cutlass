#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

// Referenced: 67_hopper_fp8_warp_specialized_gemm_with_blockwise_scaling

/*! \file
    \brief Blocked scale Ampere FP8 GEMM example using CUTLASS 3.0 APIs for NVIDIA Ampere Lovelace architecture.

    This example demonstrate a grouped scaled FP8 GEMM using the new CUTLASS 3.0.
    APIs on NVIDIA Ampere architecture. New features that will be showcased in this example are as follows:

    Examples:
      $ ./examples/85_ada_ampere_gemm_with_blockwise_scaling/85d_ampere_int8_gemm_with_blockwise_scaling_cute  \
        --m=2816 --n=3072 --k=16384
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/activation.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"

#include "helper.h"

using namespace cute;

template <typename ArchTag, typename Element, int BLK_M, int BLK_N, int BLK_K, int PipelineStages = 3, int WARP_M = 2, int WARP_N = 2>
struct SM8x_Byte_Gemm_Traits {
  static constexpr int MMA_WARP_M = WARP_M * 16;
  static constexpr int MMA_WARP_N = WARP_N * 16;
  static constexpr int MMA_WARP_K = 32;
  static constexpr int NUM_WARPS = WARP_M * WARP_N;
  static constexpr int NUM_THREADS = NUM_WARPS * 32;
  static constexpr int ELEMS_PER_COPY = sizeof(uint128_t) / sizeof(Element);
  static constexpr int THREADS_PER_ROW = BLK_K / ELEMS_PER_COPY;

  static_assert(
    !(std::is_same<Element, cutlass::float_e4m3_t>::value &&
      !std::is_same<ArchTag, cutlass::arch::Sm89>::value),
    "cutlass::float_e4m3_t is only supported on cutlass::arch::Sm89."
  );

  using MmaAtom = cute::conditional_t<cute::is_same_v<Element, cutlass::float_e4m3_t>,
    MMA_Atom<SM89_16x8x32_F32E4M3E4M3F32_TN>,
    MMA_Atom<SM80_16x8x32_S32S8S8S32_TN>>;

  using TileShape = Shape<Int<BLK_M>, Int<BLK_N>, Int<BLK_K>>;              // Threadblock-level tile size
  using ClusterShape  = Shape<_1,_1,_1>;                                    // Shape of the threadblocks in a cluster
  // This code section describes the MMA op and the tile size a warp will compute
  using TiledMma = TiledMMA<
    MmaAtom,
    Layout<Shape<Int<WARP_M>, Int<WARP_N>, _1>>, // WARP_M x WARP_N x 1 thread group
    Tile<Int<MMA_WARP_M>, Int<MMA_WARP_N>, Int<MMA_WARP_K>>>;

  using SmemLayoutAtom = decltype(composition(
      Swizzle<2,4,3>{},
      Layout<Shape <_8, Int<BLK_K>>,
              Stride<Int<BLK_K>, _1>>{}));
  using SmemLayoutAtomA = SmemLayoutAtom;
  using SmemLayoutAtomB = SmemLayoutAtom;

  using GmemTiledCopy = decltype(make_tiled_copy(
    Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, Element>{},
    Layout<Shape <Int<NUM_THREADS / THREADS_PER_ROW>, Int<THREADS_PER_ROW>>,
           Stride<Int<THREADS_PER_ROW>, _1>>{},
    Layout<Shape<_1,Int<ELEMS_PER_COPY>>>{}));
  using GmemTiledCopyA = GmemTiledCopy;
  using GmemTiledCopyB = GmemTiledCopy;

  using SmemCopyAtomA = Copy_Atom<SM75_U32x4_LDSM_N, Element>;
  using SmemCopyAtomB = Copy_Atom<SM75_U32x4_LDSM_N, Element>;
};

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = int8_t;                                         // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand

// B matrix configuration
using         ElementB    = int8_t;                                         // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand

// C matrix configuration
using         ElementC    = int;                                            // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands

// D matrix configuration
using         ElementD    = float;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;

using ArchTag       = cutlass::arch::Sm89;                          // Tag indicating the minimum SM that supports the intended feature

//
// Assembling the CollectiveMainloop type
//

// Number of pipelines you want to use
// 6: (64*128 + 128*128 + 64 + 1) * 6 = 147846 < 163840 (SM80 Shared Memory Size)
// 4: (128*128 + 128*128 + 128 + 1) * 4 = 131588 < 163840 (SM80 Shared Memory Size)
// 4: (64*128 + 128*128 + 64 + 1) * 4 = 98564 < 102400 (SM89 Shared Memory Size)
// 3: (128*128 + 128*128 + 128 + 1) * 3 = 98691 < 102400 (SM89 Shared Memory Size)
constexpr int PipelineStages = 4;
constexpr int BLK_M = 64;
constexpr int BLK_N = 128;
constexpr int BLK_K = 128;
using GemmTrait = SM8x_Byte_Gemm_Traits<ArchTag, ElementA, BLK_M, BLK_N, BLK_K, PipelineStages>;
using TileShape = GemmTrait::TileShape;
using DispatchPolicy = cutlass::gemm::MainloopSm80CpAsyncBlockScaling<PipelineStages, GemmTrait::ClusterShape>;

using ScaleConfig = decltype(cutlass::detail::sm90_trivial_blockwise_scale_config(TileShape{}));
using LayoutSFA     = decltype(ScaleConfig::deduce_layoutSFA());    // Layout type for SFA matrix operand
using LayoutSFB     = decltype(ScaleConfig::deduce_layoutSFB());    // Layout type for SFB matrix operand

// Mainloop
using CollectiveMainloop = cutlass::gemm::collective::CollectiveMma<
        DispatchPolicy, TileShape,
        ElementA,
        cute::tuple<cutlass::detail::TagToStrideA_t<LayoutA>, LayoutSFA>,
        ElementB,
        cute::tuple<cutlass::detail::TagToStrideB_t<LayoutB>, LayoutSFB>,
        GemmTrait::TiledMma,
        GemmTrait::GmemTiledCopyA, GemmTrait::SmemLayoutAtomA, GemmTrait::SmemCopyAtomA, cute::identity,  // A
        GemmTrait::GmemTiledCopyB, GemmTrait::SmemLayoutAtomB, GemmTrait::SmemCopyAtomB, cute::identity   // B
>;

using ElementAccumulator  = typename CollectiveMainloop::ElementAccumulator;    // Element type for internal accumulation
using ElementBlockScale   = typename CollectiveMainloop::ElementBlockScale;     // Element type for blockscaling during accumulation

//
// Assembling the Collective Epilogue Type
//

using CollectiveEpilogue = cutlass::epilogue::collective::DefaultEpilogue<
        ElementC,
        cutlass::detail::TagToStrideC_t<LayoutC>,
        cutlass::detail::TagToStrideC_t<LayoutD>,
        cutlass::epilogue::thread::LinearCombination<
          ElementD,
          AlignmentD,
          ElementAccumulator,
          ElementD,
          cutlass::epilogue::thread::ScaleType::Default,
          cutlass::FloatRoundStyle::round_to_nearest,
          ElementC>,                                                            // Provide if ElementD differs from ElementC
        cutlass::gemm::EpilogueDefault>;
//
// Assembling the GemmKernel
//

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>,
    CollectiveMainloop,
    CollectiveEpilogue>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Extract information from Gemm kernel.
using EpilogueOutputOp  = typename Gemm::EpilogueOutputOp;
using ElementScalar     = typename EpilogueOutputOp::ElementScalar;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

static_assert(cute::is_same_v<ElementAccumulator, ElementBlockScale>,
             "ElementAccumulator and ElementBlockScale should be same datatype");

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
LayoutSFA layout_SFA;
LayoutSFB layout_SFB;
uint64_t seed;

using LayoutScalar = cutlass::layout::PackedVectorLayout;

cutlass::HostTensor<ElementA  , LayoutA  > tensor_A;
cutlass::HostTensor<ElementB  , LayoutB  > tensor_B;
cutlass::HostTensor<ElementC  , LayoutC  > tensor_C;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_D;
cutlass::HostTensor<ElementBlockScale, LayoutScalar> blockscale_tensor_A;
cutlass::HostTensor<ElementBlockScale, LayoutScalar> blockscale_tensor_B;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_ref_D;

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;
  bool verify = true;

  float alpha = 1.f, beta = 0.f;
  int iterations = 1000;
  int warmup = 1000;
  int m = 1024, n = 512, k = 1024, l = 1;
  float epsilon = 0.02f;
  float non_zero_floor = 1.f;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("warmup", warmup);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("verify", verify);
    cmd.get_cmd_line_argument("epsilon", epsilon);
    cmd.get_cmd_line_argument("non-zero-floor", non_zero_floor);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "85d_ampere_int8_gemm_with_blockwise_scaling_cute\n\n"
      << "  Ampere INT8 GEMM using CUTLASS 3.0 API with Blockwise Scaling.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the l extent (batch) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n"
      << "  --verify=<bool>             Verify the results.\n\n"
      << "  --epsilon=<float>           The epsilon value for comparing the results.\n\n"
      << "  --non-zero-floor=<float>    The none zero floor for comparing the results.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "85d_ampere_int8_gemm_with_blockwise_scaling_cute" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;
    int bits_input = cutlass::sizeof_bits<Element>::value;
    int bits_output = cutlass::sizeof_bits<Element>::value;

    if (bits_input == 1) {
      scope_max = 2;
      scope_min = 0;
    } else if (bits_input <= 8) {
      scope_max = 2;
      scope_min = -2;
    } else if (bits_output == 16) {
      scope_max = 5;
      scope_min = -5;
    } else {
      scope_max = 8;
      scope_min = -8;
    }

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min, bits_input);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Helper to initialize a block of device data (scale_tensors)
template <typename Element, typename Layout>
bool initialize_scale_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;

    scope_min = -1;
    scope_max = 1;

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  assert(options.m % CollectiveMainloop::ScaleGranularityM == 0);
  assert(options.n % CollectiveMainloop::ScaleGranularityN == 0);

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));
  layout_SFA = ScaleConfig::tile_atom_to_shape_SFA(make_shape(options.m, options.n, options.k, options.l));
  layout_SFB = ScaleConfig::tile_atom_to_shape_SFB(make_shape(options.m, options.n, options.k, options.l));


  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto groupscale_a_coord = cutlass::make_Coord(size(filter_zeros(layout_SFA)));
  auto groupscale_b_coord = cutlass::make_Coord(size(filter_zeros(layout_SFB)));

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  blockscale_tensor_A.resize(groupscale_a_coord);
  blockscale_tensor_B.resize(groupscale_b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  cutlass::Distribution::Kind dist_A = cutlass::Distribution::Uniform;
  cutlass::Distribution::Kind dist_B = cutlass::Distribution::Uniform;
  cutlass::Distribution::Kind dist_C = cutlass::Distribution::Identity;
  cutlass::Distribution::Kind dist_scaleA = cutlass::Distribution::Uniform;
  cutlass::Distribution::Kind dist_scaleB = cutlass::Distribution::Uniform;

  initialize_tensor(tensor_A.host_view(), dist_A, seed + 2022);
  initialize_tensor(tensor_B.host_view(), dist_B, seed + 2023);
  initialize_tensor(tensor_C.host_view(), dist_C, seed + 2024);
  initialize_scale_tensor(blockscale_tensor_A.host_view(), dist_scaleA, seed + 2025);
  initialize_scale_tensor(blockscale_tensor_B.host_view(), dist_scaleB, seed + 2026);

#if 0 // Dump blockscaled tensors
  std::cout << "blockscale_tensor_A: " << groupscale_a_coord << std::endl;
  std::cout << blockscale_tensor_A.host_view() << "\n";
  std::cout << "blockscale_tensor_B: " << groupscale_b_coord << std::endl;
  std::cout << blockscale_tensor_B.host_view() << "\n";
#endif

  // Print group scaling tensors on the host side.
  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_C.sync_device();
  tensor_D.sync_device();
  blockscale_tensor_A.sync_device();
  blockscale_tensor_B.sync_device();
}

/// Populates a Gemm::Arguments structure from the given commandline options
template<typename GemmArguments>
GemmArguments args_from_options(const Options &options)
{
  GemmArguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(),
     stride_A,
     tensor_B.device_data(),
     stride_B,
     blockscale_tensor_A.device_data(),
     layout_SFA,
     blockscale_tensor_B.device_data(),
     layout_SFB
     },
    {
      {}, // epilogue.thread
      tensor_C.device_data(), stride_C,
      tensor_D.device_data(), stride_D
    }
  };

  auto &fusion_args = arguments.epilogue.thread;

  fusion_args.alpha = CollectiveEpilogue::ThreadEpilogueOp::ElementCompute(options.alpha);
  fusion_args.beta = CollectiveEpilogue::ThreadEpilogueOp::ElementCompute(options.beta);

  return arguments;
}

/// Don't know why the compiler does not like verify() being templated...
bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
                             cute::make_layout(
                                cute::make_shape(options.m, options.k, options.l),
                                stride_A
                              )
                            );
  auto B = cute::make_tensor(tensor_B.host_data(),
                             cute::make_layout(
                               cute::make_shape(options.n, options.k, options.l),
                               stride_B
                              )
                            );
  auto C = cute::make_tensor(tensor_C.host_data(),
                             cute::make_layout(
                                cute::make_shape(options.m, options.n, options.l),
                                stride_C
                              )
                            );
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
                             cute::make_layout(
                                cute::make_shape(options.m, options.n, options.l),
                                stride_D
                              )
                            );

  auto SFA = cute::make_tensor(blockscale_tensor_A.host_data(), layout_SFA);
  auto SFB = cute::make_tensor(blockscale_tensor_B.host_data(), layout_SFB);

  using unused_t = decltype(D);

  cutlass::reference::host::GettBlockScalingMainloopParams<
      CollectiveMainloop::ElementAccumulator,
      decltype(A),
      decltype(SFA),
      decltype(B),
      decltype(SFB)
    > mainloop_params{A, SFA, B, SFB};

  cutlass::reference::host::GettEpilogueParams<
      CollectiveEpilogue::ThreadEpilogueOp::ElementCompute,
      CollectiveEpilogue::ThreadEpilogueOp::ElementCompute,
      CollectiveEpilogue::ThreadEpilogueOp::ElementAccumulator,
      CollectiveEpilogue::ThreadEpilogueOp::ElementCompute,
      decltype(C),
      decltype(D),
      unused_t, // bias
      unused_t, // aux
      unused_t, // valpha
      unused_t  // vbeta
  > epilogue_params;

  epilogue_params.C = C;
  epilogue_params.D = D;
  epilogue_params.alpha = CollectiveEpilogue::ThreadEpilogueOp::ElementCompute(options.alpha);
  epilogue_params.beta = CollectiveEpilogue::ThreadEpilogueOp::ElementCompute(options.beta);

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  // compare_reference
  bool passed = true;
  tensor_D.sync_host();
  passed &= cutlass::reference::host::TensorRelativelyEquals(tensor_D.host_view(), tensor_ref_D.host_view(), ElementD(options.epsilon), ElementD(options.non_zero_floor));
  double mse = cutlass::reference::host::TensorMSE(tensor_D.host_view(), tensor_ref_D.host_view());
  double mre = cutlass::reference::host::TensorMRE(tensor_D.host_view(), tensor_ref_D.host_view());
  double max_error = cutlass::reference::host::TensorGreatestError(tensor_D.host_view(), tensor_ref_D.host_view());
  std::cout << "  Result MSE: " << mse << ", MRE: " << mre << ", greatest error: " << max_error << std::endl;

#if 0
  std::cout << "tensor_ref_D.host_view() {" << std::endl
            << tensor_ref_D.host_view() << std::endl
            << "}"  << std::endl;
  std::cout << "tensor_D.host_view() {" << std::endl
            << tensor_D.host_view() << std::endl
            << "}"  << std::endl;
#endif

  return passed;
}


/// Execute a given example GEMM computation
int run(Options &options) {

  bool skip = false;
  std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
  std::cout << "  Tile shape (M, N, K): " << size<0>(TileShape{}) << ", " << size<1>(TileShape{}) << ", " << size<2>(TileShape{}) << std::endl;
  std::cout << "  ScaleGranularityM: " << CollectiveMainloop::ScaleGranularityM << " (ScaleMsPerTile: " << CollectiveMainloop::ScaleMsPerTile << ")" << std::endl;
  std::cout << "  ScaleGranularityN: " << CollectiveMainloop::ScaleGranularityN << " (ScaleNsPerTile: " << CollectiveMainloop::ScaleNsPerTile << ")" << std::endl;


  if (options.m < CollectiveMainloop::ScaleGranularityM) {
    std::cout << "  Skippig (m size: " << options.m << " less than ScaleGranularityM: " << CollectiveMainloop::ScaleGranularityM << "):" << std::endl;
    skip = true;
  }

  if (options.n < CollectiveMainloop::ScaleGranularityN) {
    std::cout << "  Skippig (n size: " << options.n << " less than ScaleGranularityN: " << CollectiveMainloop::ScaleGranularityN << "):" << std::endl;
    skip = true;
  }

  if (options.k < size<2>(TileShape{})) {
    std::cout << "  Skippig (k size: " << options.k << " less than TileShape[2]: " << size<2>(TileShape{}) << "):" << std::endl;
    skip = true;
  }

  if (!skip) std::cout << "  Running... " << std::endl;
  else return -1;

  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<typename Gemm::Arguments>(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  if (options.verify) {
    result.passed = verify(options);

    std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;
  }
  else {
    result.passed = true;
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    for (int iter = 0; iter < options.warmup + options.iterations; ++iter) {
      if (iter == options.warmup)
        timer.start();
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
    fflush(stdout);
  }

  return result.passed;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 8) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Ampere Architecture or "
      << "later (compute capability 80 or greater).\n";
    return 0;
  }
  //
  // Parse options
  //

  Options options;
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

  bool passed = true;
  passed = run(options);
  if (!passed)
    return -1;

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
