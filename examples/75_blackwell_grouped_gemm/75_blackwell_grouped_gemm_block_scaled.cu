#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/



/*! \file
    \brief Grouped GEMM example using CUTLASS 3 APIs for the NVIDIA Blackwell SM100 architecture.

    This example demonstrates an implementation of Grouped GEMM using a TMA + Blackwell SM100 TensorOp-based warp-specialized kernel
    for narrow precisions (FP4) with Scale Factors (In and Out).
    For this example all scheduling work is performed on the device.
    The new feature showcased in this example is device-side modification of TMA descriptors
    to move between groups/problem_count (represented by groups).
    https://docs.nvidia.com/cuda/cuda-c-programming-guide/#encoding-a-tensor-map-on-device

    To run this example:

      $ ./examples/75_blackwell_grouped_gemm_block_scaled/75_blackwell_grouped_gemm_block_scaled --m=2048 --n=2048 --k=2048 --groups=10

      The above example command makes all 10 groups to be sized at the given m, n, k sizes.
      Skipping any of the problem dimensions randomizes it across the different groups.
      Same applies for alpha and beta values that are randomized across the different groups.

    To run this example for a set of problems using the benchmark option:

      $ ./examples/75_blackwell_grouped_gemm_block_scaled/75_blackwell_grouped_gemm_block_scaled --benchmark=./test_benchmark.txt

      Where the test_benchmark.txt may look as such:
        0 256x512x128
        1 256x512x512
        2 512x256x128
        3 256x256x128
        4 256x512x1024
        5 1024x512x128 and so on
*/

#include <iostream>
#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>
#include <float.h>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gett.hpp"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_compare.h"

#include "helper.h"
using namespace cute;

using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int,int,int>>; // <M,N,K> per group
using ElementInput = cutlass::float_e2m1_t;                                // Element type for Input matrix operands
using ElementSF    = cutlass::float_ue4m3_t;                               // Element type for SF matrix operands
using ElementC     = cutlass::half_t;                                      // Element type for C matrix operands

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
// A matrix configuration
using ElementA = cutlass::nv_float4_t<ElementInput>;                        // Element type for A matrix operand
using LayoutA  = cutlass::layout::RowMajor;                                 // Layout type for A matrix operand
constexpr int AlignmentA  = 32;                                             // Alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using ElementB = cutlass::nv_float4_t<ElementInput>;                        // Element type for B matrix operand
using LayoutB = cutlass::layout::ColumnMajor;                               // Layout type for B matrix operand
constexpr int AlignmentB  = 32;                                             // Alignment of A matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using ElementD = ElementC;                                                  // Element type for D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Alignment of C matrix in units of elements (up to 16 bytes)
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;    // Alignment of D matrix in units of elements (up to 16 bytes)
using ElementAccumulator  = float;                                          // Element type for internal accumulation

// using ElementD = cutlass::float_e2m1_t; // Enable for SF Output          // Element type for D matrix operands
constexpr int OutputSFVectorSize = 16;
using FusionOperation = cutlass::epilogue::fusion::LinCombEltActBlockScaleFactor<
    cutlass::epilogue::thread::SiLu,
    OutputSFVectorSize,
    ElementD, 
    ElementAccumulator, 
    ElementSF,
    LayoutC,
    ElementC>;

// Core kernel configurations
using ArchTag             = cutlass::arch::Sm100;                           // Tag indicating the minimum SM that supports the intended feature
using EpilogueOperatorClass = cutlass::arch::OpClassTensorOp;               // Epilogue Operator class tag
using MainloopOperatorClass = cutlass::arch::OpClassBlockScaledTensorOp;    // Mainloop Operator class tag
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size

// Runtime Cluster Shape
using ClusterShape = Shape<int32_t,int32_t,_1>;
/* // For Static Cluster Shape: 
use ClusterShape = Shape<_2,_1,_1> for example
using AtomThrShape   = decltype(shape_div(ClusterShape{}, Shape<_2,_1,_1>{}));    // for 2SM config
using OutputTileShape = decltype(shape_div(ClusterTileShape{}, ClusterShape{}));  // for epilogue builder
using MmaTileShape   = decltype(shape_div(ClusterTileShape{}, AtomThrShape{}));   // for mainloop builder
*/

// Different configs for 1SM and 2SM MMA kernel
struct MMA1SMConfig {
  using MmaTileShape     = Shape<_128,_256,_256>;
  using KernelSchedule   = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized1SmNvf4Sm100;   // Kernel to launch
  using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized1Sm;              // Epilogue to launch
  using OutputTileShape  = decltype(shape_div(MmaTileShape{}, Shape<_1,_1,_1>{}));
};

struct MMA2SMConfig {
  using MmaTileShape     = Shape<_256,_256,_256>;
  using KernelSchedule   = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized2SmNvf4Sm100;   // Kernel to launch
  using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized2Sm;              // Epilogue to launch
  using OutputTileShape  = decltype(shape_div(MmaTileShape{}, Shape<_2,_1,_1>{}));
};

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, EpilogueOperatorClass,
    typename MMA1SMConfig::OutputTileShape, ClusterShape,
    Shape<_128,_64>,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC *, AlignmentC,
    ElementD, LayoutC *, AlignmentD,
    typename MMA1SMConfig::EpilogueSchedule
    // , FusionOperation  // Enable for SF Output
>::CollectiveOp;
using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
  ArchTag, MainloopOperatorClass,
  ElementA, LayoutA *, AlignmentA,
  ElementB, LayoutB *, AlignmentB,
  ElementAccumulator,
    typename MMA1SMConfig::MmaTileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    typename MMA1SMConfig::KernelSchedule
>::CollectiveOp;
using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloop,
    CollectiveEpilogue
>;
using Gemm1SM = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
using Gemm = Gemm1SM;

using CollectiveEpilogue2SM = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, EpilogueOperatorClass,
    typename MMA2SMConfig::OutputTileShape, ClusterShape,
    Shape<_128,_64>,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC *, AlignmentC,
    ElementD, LayoutC *, AlignmentD,
    typename MMA2SMConfig::EpilogueSchedule
    // , FusionOperation  // Enable for SF Output
>::CollectiveOp;
using CollectiveMainloop2SM = typename cutlass::gemm::collective::CollectiveBuilder<
  ArchTag, MainloopOperatorClass,
  ElementA, LayoutA *, AlignmentA,
  ElementB, LayoutB *, AlignmentB,
  ElementAccumulator,
    typename MMA2SMConfig::MmaTileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    typename MMA2SMConfig::KernelSchedule
>::CollectiveOp;
using GemmKernel2SM = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloop2SM,
    CollectiveEpilogue2SM
>;
using Gemm2SM = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel2SM>;

using StrideA = typename Gemm::GemmKernel::InternalStrideA;
using StrideB = typename Gemm::GemmKernel::InternalStrideB;
using StrideC = typename Gemm::GemmKernel::InternalStrideC;
using StrideD = typename Gemm::GemmKernel::InternalStrideD;

using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::InternalLayoutSFA;
using LayoutSFB = typename Gemm::GemmKernel::CollectiveMainloop::InternalLayoutSFB;
using Sm100BlkScaledConfig =  typename Gemm::GemmKernel::CollectiveMainloop::Sm100BlkScaledConfig;
using Sm100BlockScaledOutputConfig = cutlass::detail::Sm100BlockScaledOutputConfig<
                                        OutputSFVectorSize, 
                                        cute::is_same_v<typename FusionOperation::GmemLayoutTagScalefactor,
                                            cutlass::layout::RowMajor> ? cute::UMMA::Major::K : cute::UMMA::Major::MN
                                     >;
using OutputSFAtom = typename Sm100BlockScaledOutputConfig::SfAtom;
using LayoutSFD = typename Sm100BlockScaledOutputConfig::LayoutSF;

// Host-side allocations
std::vector<StrideA> stride_A_host;
std::vector<StrideB> stride_B_host;
std::vector<LayoutSFA> layout_SFA_host;
std::vector<LayoutSFA> layout_SFB_host;
std::vector<StrideC> stride_C_host;
std::vector<StrideD> stride_D_host;

std::vector<ElementAccumulator> alpha_host;
std::vector<ElementAccumulator> beta_host;

using HostTensorA = cutlass::HostTensor<typename Gemm::ElementA, cutlass::layout::PackedVectorLayout>;
using HostTensorB = cutlass::HostTensor<typename Gemm::ElementB, cutlass::layout::PackedVectorLayout>;
using HostTensorSF = cutlass::HostTensor<typename Gemm::GemmKernel::ElementSF, cutlass::layout::PackedVectorLayout>;
using HostTensorC = cutlass::HostTensor<typename Gemm::ElementC, cutlass::layout::PackedVectorLayout>;
using HostTensorD = cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, cutlass::layout::PackedVectorLayout>;
std::vector<HostTensorA> block_A;
std::vector<HostTensorB> block_B;
std::vector<HostTensorSF> block_SFA;
std::vector<HostTensorSF> block_SFB;
std::vector<HostTensorC> block_C;
std::vector<HostTensorD> block_D;
std::vector<HostTensorSF> block_SFD;
std::vector<HostTensorD> block_ref_D;

// Device-side allocations
cutlass::DeviceAllocation<typename ProblemShape::UnderlyingProblemShape> problem_sizes;

cutlass::DeviceAllocation<const typename Gemm::ElementA *> ptr_A;
cutlass::DeviceAllocation<const typename Gemm::ElementB *> ptr_B;
cutlass::DeviceAllocation<const typename Gemm::GemmKernel::ElementSF *> ptr_SFA;
cutlass::DeviceAllocation<const typename Gemm::GemmKernel::ElementSF *> ptr_SFB;
cutlass::DeviceAllocation<const typename Gemm::ElementC *> ptr_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_D;
cutlass::DeviceAllocation<typename Gemm::GemmKernel::ElementSF *> ptr_SFD;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_ref_D;

cutlass::DeviceAllocation<StrideA> stride_A;
cutlass::DeviceAllocation<StrideB> stride_B;
cutlass::DeviceAllocation<LayoutSFA> layout_SFA;
cutlass::DeviceAllocation<LayoutSFB> layout_SFB;
cutlass::DeviceAllocation<StrideC> stride_C;
cutlass::DeviceAllocation<StrideD> stride_D;

// Note, this is an array of pointers to alpha and beta scaling values per group
cutlass::DeviceAllocation<ElementAccumulator*> alpha_device;
cutlass::DeviceAllocation<ElementAccumulator*> beta_device;
cutlass::DeviceAllocation<ElementAccumulator> block_alpha;
cutlass::DeviceAllocation<ElementAccumulator> block_beta;
// A matrix wide constant value to scale the output matrix
// Avoids generating small FP4 values.
// NormConst is a single device-side constant value, its not per-batch or per-group
cutlass::DeviceAllocation<ElementAccumulator> norm_constant_device;

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

template <typename T>
auto make_iterator(T* ptr) {
  using namespace cute;
  if constexpr (cute::is_subbyte_v<T>) {
    return subbyte_iterator<T>(ptr);
  }
  else {
    return ptr;
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

using RasterOrderOptions = typename cutlass::gemm::kernel::detail::PersistentTileSchedulerSm100GroupParams<typename ProblemShape::UnderlyingProblemShape>::RasterOrderOptions;
// Command line options parsing
struct Options {

  bool help = false;
  bool verification = true;

  float alpha = FLT_MAX;
  float beta  = FLT_MAX;
  float norm_constant = 1.0;
  int iterations = 10;
  int m = 1024, n = 2048, k = 512, groups = 10;
  dim3 cluster_shape = dim3(2,1,1);
  dim3 cluster_shape_fallback = dim3(2,1,1);
  RasterOrderOptions raster_order = RasterOrderOptions::AlongN;
  int max_sm_count = INT_MAX;
  std::string benchmark_path;
  std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes_host;
  int const tma_alignment_bits = 128;
  int const alignment = tma_alignment_bits / cutlass::sizeof_bits<ElementInput>::value;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }
    if (cmd.check_cmd_line_flag("no-verif")) {
      verification = false;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("groups", groups);
    cmd.get_cmd_line_argument("alpha", alpha, FLT_MAX);
    cmd.get_cmd_line_argument("beta",  beta,  FLT_MAX);
    cmd.get_cmd_line_argument("norm_constant",  norm_constant,  float(1.0));
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("benchmark", benchmark_path);
    cmd.get_cmd_line_argument("cluster_m", cluster_shape.x);
    cmd.get_cmd_line_argument("cluster_n", cluster_shape.y);
    cmd.get_cmd_line_argument("cluster_fallback_m", cluster_shape_fallback.x);
    cmd.get_cmd_line_argument("cluster_fallback_n", cluster_shape_fallback.y);
    cmd.get_cmd_line_argument("max_sm_count", max_sm_count, INT_MAX);

    // Decide how to initialize the problems
    if (!benchmark_path.empty()) {
      if (!benchmark_problems()) {
        problem_sizes_host.clear();
        return;
      }
    }
    else {
      randomize_problems(cmd);
    }

    char raster_char;
    cmd.get_cmd_line_argument("raster", raster_char);

    if (raster_char == 'N' || raster_char == 'n') {
      raster_order = RasterOrderOptions::AlongN;
    }
    else if (raster_char == 'M' || raster_char == 'm') {
      raster_order = RasterOrderOptions::AlongM;
    }
  }

  void randomize_problems(cutlass::CommandLine &cmd) {
    int cmd_line_m = -1, cmd_line_n = -1, cmd_line_k = -1;
    cmd.get_cmd_line_argument("m", cmd_line_m);
    cmd.get_cmd_line_argument("n", cmd_line_n);
    cmd.get_cmd_line_argument("k", cmd_line_k);

    problem_sizes_host.reserve(groups);

    for (int i = groups; i > 0; i--) {
      int m = cmd_line_m;
      int n = cmd_line_n;
      int k = cmd_line_k;
      if (m < 1) {
        m = alignment * ((rand() % 64) + 1);
      }
      if (n < 1) {
        n = alignment * ((rand() % 64) + 1);
      }
      if (k < 1) {
        k = alignment * ((rand() % 64) + 1);
      }
      problem_sizes_host.push_back({m, n, k});
    }
  }

  /// Load a benchmark
  bool benchmark_problems() {
    std::ifstream file(benchmark_path);
    if (!file.good()) {
      return false;
    }

    while (file.good()) {

      int idx = -1;
      std::string extent_str;

      file >> idx >> extent_str;

      if (idx < 0 || extent_str.empty()) {
        break;
      }

      cutlass::gemm::GemmCoord extent;
      std::vector<std::string> tokens;

      cutlass::CommandLine::tokenize(tokens, extent_str, 'x');

      for (int i = 0; i < int(tokens.size()); ++i) {
        int x = std::atoi(tokens.at(i).c_str());

        // round up
        if (x % alignment) {
          x += (alignment - (x % alignment));
        }

        extent.at(i) = x;
      }

      if (extent.product()) {
        problem_sizes_host.push_back({extent.m(), extent.n(), extent.k()});
      }
    }
    groups = static_cast<int>(problem_sizes_host.size());

    return true;
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "75_blackwell_grouped_gemm_block_scaled\n\n"
      << "  Blackwell Block Scaled Narrow Precision Grouped GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                                                       If specified, displays this usage statement\n\n"
      << "  --m=<int>                                                    Sets the M extent of the GEMM for all groups\n"
      << "  --n=<int>                                                    Sets the N extent of the GEMM for all groups\n"
      << "  --k=<int>                                                    Sets the K extent of the GEMM for all groups\n"
      << "  --groups=<int>                                               Sets the number of individual GEMM problems for Grouped GEMM\n"
      << "  --alpha=<f32>                                                Epilogue scalar alpha\n"
      << "  --beta=<f32>                                                 Epilogue scalar beta\n"
      << "  --norm_constant=<f32>                                        Epilogue scalar normalization constant for the output matrix\n\n"
      << "  --cluster_m=<int>          and --cluster_n=<int>             Sets the X,Y dims of the preferred cluster shape\n"
      << "  --cluster_fallback_m=<int> and --cluster_fallback_n=<int>    Sets the X,Y dims of the fallback cluster shape\n\n"
      << "  --raster=<char>                                              CTA Rasterization direction (N for along N, M for along M)\n\n"
      << "  --iterations=<int>                                           Number of profiling iterations to perform\n\n"
      << "  --benchmark=<str>                                            Executes a benchmark problem size\n"
      << "  --max_sm_count=<int>                                         Run kernels using only these number of SMs\n"
      << "  --no-verif                                                   Do not run (host-side) verification kernels\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "75_blackwell_grouped_gemm_block_scaled" << " --m=1024 --n=512 --k=1024 --groups=10 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s, std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes_host) const
  {
    // Number of real-valued multiply-adds
    uint64_t fmas = uint64_t();

    for (auto const & problem : problem_sizes_host) {
      fmas += static_cast<uint64_t>(get<0>(problem)) *
              static_cast<uint64_t>(get<1>(problem)) *
              static_cast<uint64_t>(get<2>(problem));
    }
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * uint64_t(fmas);
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms = 0.0;
  double gflops = 0.0;
  cutlass::Status status = cutlass::Status::kSuccess;
  hipError_t error = hipSuccess;
  bool passed = false;
};

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_block(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed) {

  double scope_max, scope_min;
  constexpr int bits_input = cutlass::sizeof_bits<Element>::value;

  if constexpr (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if constexpr (bits_input <= 6) {
    scope_max = 2;
    scope_min = -2;
  }
  else if constexpr (bits_input <= 8) {
    if constexpr (cute::is_same_v<Element, cutlass::float_ue8m0_t>) {
      scope_max = 4;
      scope_min = 1;
    }
    else {
      scope_max = 1;
      scope_min = -1;
    }
  }
  else{
    scope_max = 4;
    scope_min = -4;
  }
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min, 0);
  
  return true;
}

/// Allocates device-side data
void allocate(const Options &options) {
  for (int32_t i = 0; i < options.groups; ++i) {
    auto problem = options.problem_sizes_host.at(i);
    auto M = get<0>(problem);
    auto N = get<1>(problem);
    auto K = get<2>(problem);

    auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
    auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
    auto stride_C = cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1});
    auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {M, N, 1});

    auto layout_A = make_layout(make_shape(M, K, 1), stride_A);
    auto layout_B = make_layout(make_shape(N, K, 1), stride_B);
    auto layout_C = make_layout(make_shape(M, N, 1), stride_C);
    auto layout_D = make_layout(make_shape(M, N, 1), stride_D);
    auto layout_SFA = Sm100BlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(M, N, K, 1));
    auto layout_SFB = Sm100BlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(M, N, K, 1));
    auto layout_SFD = Sm100BlockScaledOutputConfig::tile_atom_to_shape_SFD(cute::make_shape(M, N, K, 1));

    stride_A_host.push_back(stride_A);
    stride_B_host.push_back(stride_B);
    layout_SFA_host.push_back(layout_SFA);
    layout_SFB_host.push_back(layout_SFB);
    stride_C_host.push_back(stride_C);
    stride_D_host.push_back(stride_D);

    block_A.push_back(HostTensorA(cutlass::make_Coord(size(layout_A))));
    block_B.push_back(HostTensorB(cutlass::make_Coord(size(layout_B))));
    block_SFA.push_back(HostTensorSF(cutlass::make_Coord(size(filter_zeros(layout_SFA)))));
    block_SFB.push_back(HostTensorSF(cutlass::make_Coord(size(filter_zeros(layout_SFB)))));
    block_C.push_back(HostTensorC(cutlass::make_Coord(size(layout_C))));
    block_D.push_back(HostTensorD(cutlass::make_Coord(size(layout_D))));
    block_SFD.push_back(HostTensorSF(cutlass::make_Coord(size(filter_zeros(layout_SFD)))));
    block_ref_D.push_back(HostTensorD(cutlass::make_Coord(size(layout_D))));
  }
  block_alpha.reset(options.groups);
  block_beta.reset(options.groups);
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {
  uint64_t seed = 2020;
  problem_sizes.reset(options.groups);
  problem_sizes.copy_from_host(options.problem_sizes_host.data());

  //
  // Assign pointers
  //

  std::vector<typename Gemm::ElementA *> ptr_A_host(options.groups);
  std::vector<typename Gemm::ElementB *> ptr_B_host(options.groups);
  std::vector<typename Gemm::GemmKernel::ElementSF *> ptr_SFA_host(options.groups);
  std::vector<typename Gemm::GemmKernel::ElementSF *> ptr_SFB_host(options.groups);
  std::vector<typename Gemm::ElementC *> ptr_C_host(options.groups);
  std::vector<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_D_host(options.groups);
  std::vector<typename Gemm::GemmKernel::ElementSF *> ptr_SFD_host(options.groups);
  std::vector<ElementAccumulator *> ptr_alpha_host(options.groups);
  std::vector<ElementAccumulator *> ptr_beta_host(options.groups);

  for (int32_t i = 0; i < options.groups; ++i) {

    initialize_block(block_A.at(i).host_view(), seed + 2021);
    initialize_block(block_B.at(i).host_view(), seed + 2022);
    initialize_block(block_C.at(i).host_view(), seed + 2023);
    initialize_block(block_SFA.at(i).host_view(), seed + 2024);
    initialize_block(block_SFB.at(i).host_view(), seed + 2025);

    block_A.at(i).sync_device();
    block_B.at(i).sync_device();
    block_C.at(i).sync_device();
    block_SFA.at(i).sync_device();
    block_SFB.at(i).sync_device();

    ptr_A_host.at(i) = block_A.at(i).device_data();
    ptr_B_host.at(i) = block_B.at(i).device_data();
    ptr_SFA_host.at(i) = block_SFA.at(i).device_data();
    ptr_SFB_host.at(i) = block_SFB.at(i).device_data();
    ptr_C_host.at(i) = block_C.at(i).device_data();
    ptr_D_host.at(i) = block_D.at(i).device_data();
    ptr_SFD_host.at(i) = block_SFD.at(i).device_data();

    alpha_host.push_back((options.alpha == FLT_MAX) ? static_cast<ElementAccumulator>((rand() % 5) + 1) : options.alpha);
    beta_host.push_back((options.beta == FLT_MAX) ? static_cast<ElementAccumulator>(rand() % 5) : options.beta);
    ptr_alpha_host.at(i) = block_alpha.get() + i;
    ptr_beta_host.at(i) = block_beta.get() + i;
  }

  ptr_A.reset(options.groups);
  ptr_A.copy_from_host(ptr_A_host.data());

  ptr_B.reset(options.groups);
  ptr_B.copy_from_host(ptr_B_host.data());

  ptr_SFA.reset(options.groups);
  ptr_SFA.copy_from_host(ptr_SFA_host.data());

  ptr_SFB.reset(options.groups);
  ptr_SFB.copy_from_host(ptr_SFB_host.data());

  ptr_C.reset(options.groups);
  ptr_C.copy_from_host(ptr_C_host.data());

  ptr_D.reset(options.groups);
  ptr_D.copy_from_host(ptr_D_host.data());

  ptr_SFD.reset(options.groups);
  ptr_SFD.copy_from_host(ptr_SFD_host.data());

  stride_A.reset(options.groups);
  stride_A.copy_from_host(stride_A_host.data());

  stride_B.reset(options.groups);
  stride_B.copy_from_host(stride_B_host.data());

  layout_SFA.reset(options.groups);
  layout_SFA.copy_from_host(layout_SFA_host.data());

  layout_SFB.reset(options.groups);
  layout_SFB.copy_from_host(layout_SFB_host.data());

  stride_C.reset(options.groups);
  stride_C.copy_from_host(stride_C_host.data());

  stride_D.reset(options.groups);
  stride_D.copy_from_host(stride_D_host.data());

  alpha_device.reset(options.groups);
  alpha_device.copy_from_host(ptr_alpha_host.data());
  beta_device.reset(options.groups);
  beta_device.copy_from_host(ptr_beta_host.data());

  block_alpha.copy_from_host(alpha_host.data());
  block_beta.copy_from_host(beta_host.data());

  norm_constant_device.reset(1);
  norm_constant_device.copy_from_host(&options.norm_constant);
}

/// Populates a Gemm::Arguments structure from the given commandline options
template <typename Gemm>
typename Gemm::Arguments args_from_options(Options &options, bool host_problem_shapes_available = true)
{
  cutlass::KernelHardwareInfo hw_info;
  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = min(cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id), options.max_sm_count);

  if (!is_static_v<ClusterShape>) {
    if (size<0>(typename Gemm::GemmKernel::CollectiveMainloop::AtomThrShapeMNK{}) == 2 &&
        (options.cluster_shape.x < 2 || options.cluster_shape_fallback.x < 2)) {
      std::cout << "Error: MMA2SMConfig kernel config needs cluster_dim.x >= 2" << std::endl;
    }
    hw_info.cluster_shape = options.cluster_shape;
    hw_info.cluster_shape_fallback = options.cluster_shape_fallback;
  }

  typename Gemm::Arguments arguments;
  decltype(arguments.epilogue.thread) fusion_args;
  fusion_args.alpha_ptr = nullptr;
  fusion_args.beta_ptr = nullptr;

  // If alpha/beta are provided (via cmd line args) and are scalar, i.e., same alpha/beta applies to all batches.
  // If pointers to alpha/beta are provided, i.e., alpha/beta can differ between batches/groups.
  if (options.alpha != FLT_MAX){
    // Single alpha for all groups
    fusion_args.alpha = options.alpha;
    fusion_args.alpha_ptr_array = nullptr;
    fusion_args.dAlpha = {_0{}, _0{}, 0};
  }
  else {
    fusion_args.alpha = 0;
    fusion_args.alpha_ptr_array = alpha_device.get();
    // Only one alpha per each group
    fusion_args.dAlpha = {_0{}, _0{}, 1};
  }
  if (options.beta != FLT_MAX) {
    // Single beta for all groups
    fusion_args.beta = options.beta;
    fusion_args.beta_ptr_array = nullptr;
    fusion_args.dBeta = {_0{}, _0{}, 0};
  }
  else {
    fusion_args.beta = 0;
    fusion_args.beta_ptr_array = beta_device.get();
    // Only one beta per each group
    fusion_args.dBeta = {_0{}, _0{}, 1};
  }
  // Output Block SF
  // fusion_args.block_scale_factor_ptr = ptr_SFD.get();          // Enable for SF Output
  // fusion_args.norm_constant_ptr = norm_constant_device.get();  // Enable for SF Output

  typename Gemm::GemmKernel::TileSchedulerArguments scheduler;
  scheduler.raster_order = options.raster_order;

  if (host_problem_shapes_available) {
    arguments = typename Gemm::Arguments {
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {options.groups, problem_sizes.get(), options.problem_sizes_host.data()},
      {ptr_A.get(), stride_A.get(), ptr_B.get(), stride_B.get(),
       ptr_SFA.get(), layout_SFA.get(), ptr_SFB.get(), layout_SFB.get()},
      {fusion_args, ptr_C.get(), stride_C.get(), ptr_D.get(), stride_D.get()},
      hw_info, scheduler
    };
  }
  else {
    arguments = typename Gemm::Arguments {
      cutlass::gemm::GemmUniversalMode::kGrouped,
      {options.groups, problem_sizes.get(), nullptr},
      {ptr_A.get(), stride_A.get(), ptr_B.get(), stride_B.get(),
       ptr_SFA.get(), layout_SFA.get(), ptr_SFB.get(), layout_SFB.get()},
      {fusion_args, ptr_C.get(), stride_C.get(), ptr_D.get(), stride_D.get()},
      hw_info, scheduler
    };
  }

  return arguments;
}

bool verify(const Options &options) {
  using namespace cute;
  bool passed = true;
  for (int32_t i = 0; i < options.groups; ++i) {
    auto problem = options.problem_sizes_host.at(i);
    auto M = get<0>(problem);
    auto N = get<1>(problem);
    auto K = get<2>(problem);

    auto stride_A = cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1});
    auto stride_B = cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1});
    auto stride_C = cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1});
    auto stride_D = cutlass::make_cute_packed_stride(StrideD{}, {M, N, 1});
    auto layout_A = make_layout(make_shape(M, K, 1), stride_A);
    auto layout_B = make_layout(make_shape(N, K, 1), stride_B);
    auto layout_C = make_layout(make_shape(M, N, 1), stride_C);
    auto layout_D = make_layout(make_shape(M, N, 1), stride_D);
    auto layout_SFA = Sm100BlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(M, N, K, 1));
    auto layout_SFB = Sm100BlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(M, N, K, 1));
    auto layout_SFD = Sm100BlockScaledOutputConfig::tile_atom_to_shape_SFD(cute::make_shape(M, N, K, 1));

    // Create the arguments for host reference implementation
    Tensor tensor_A = make_tensor(make_iterator(block_A.at(i).host_data()), layout_A);
    Tensor tensor_SFA = make_tensor(block_SFA.at(i).host_data(), layout_SFA);
    Tensor tensor_B = make_tensor(make_iterator(block_B.at(i).host_data()), layout_B);
    Tensor tensor_SFB = make_tensor(block_SFB.at(i).host_data(), layout_SFB);
    cutlass::reference::host::GettBlockScalingMainloopParams<ElementAccumulator,
        decltype(tensor_A),
        decltype(tensor_SFA),
        decltype(tensor_B),
        decltype(tensor_SFB)
      > 
    mainloop_params{tensor_A, tensor_SFA, tensor_B, tensor_SFB};
  
    auto tensor_C = cute::make_tensor(make_iterator(block_C.at(i).host_data()), layout_C);
    auto tensor_ref_D = cute::make_tensor(make_iterator(block_ref_D.at(i).host_data()), layout_D);

    cutlass::reference::host::GettEpilogueParams<
        float, float,
        ElementAccumulator, ElementAccumulator,
        decltype(tensor_C), decltype(tensor_ref_D)
      > epilogue_params{};

    epilogue_params.C = tensor_C;
    epilogue_params.D = tensor_ref_D;
    epilogue_params.alpha = alpha_host.at(i);
    epilogue_params.beta = beta_host.at(i);

    cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

    block_D.at(i).sync_host();
    // Check if output from CUTLASS kernel and reference kernel are equal or not
    passed &= cutlass::reference::host::TensorEquals(block_ref_D.at(i).host_view(), block_D.at(i).host_view());
  }
  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options, bool host_problem_shapes_available = true)
{
  std::cout << "  Problem Sizes, Alpha, Beta " << std::endl;
  for (int32_t i = 0; i < options.groups; ++i) {
    std::cout << "    " << options.problem_sizes_host.at(i);
    std::cout << ", " << alpha_host.at(i) << ", " << beta_host.at(i) << std::endl;
  }
  std::cout << "  Groups      : " << options.groups  << std::endl;

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<Gemm>(options, host_problem_shapes_available);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  hipDeviceSynchronize();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  if (options.verification) {
    std::cout << "  Host-side verification is now running - may be very slow for large cases." << std::endl;
    result.passed = verify(options);
    std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;
    if (!result.passed) {
      exit(-1);
    }
  }
  else {
    std::cout << "  Verfication is turned off for this run." << std::endl;
  } 

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average setup and runtime and GFLOPs.
    float elapsed_ms       = timer.elapsed_millis();
    result.avg_runtime_ms  = double(elapsed_ms) / double(options.iterations);
    result.gflops          = options.gflops(result.avg_runtime_ms / 1000.0, options.problem_sizes_host);

    std::cout << "  Avg runtime : " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS      : " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.8 Toolkit to run this example
  if (__CUDACC_VER_MAJOR__ < 12 ||
       ((__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)
       )
     ) {
    std::cerr << "This example requires CUDA 12.8 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (!(props.major == 10 && props.minor == 0)) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Blackwell Architecture (compute capability 100a).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
  allocate(options);
  initialize(options);

  //
  // Evaluate CUTLASS kernels
  //

  std::cout << "Running kernel with 1SM MMA config:" << std::endl;
  run<Gemm1SM>(options, false /*host_problem_shapes_available*/);
  std::cout << "Running kernel with 2SM MMA config:" << std::endl;
  run<Gemm2SM>(options, false /*host_problem_shapes_available*/); 
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
