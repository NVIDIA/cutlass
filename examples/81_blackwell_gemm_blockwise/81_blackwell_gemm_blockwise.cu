/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief A FP8 blockwise scaled GEMM example for the NVIDIA Blackwell SM100 architecture using CUTLASS.
*/



#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/util/reference/host/gett.hpp"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
// A matrix configuration
using ElementA            = cutlass::float_e4m3_t;                          // Element type for A matrix operand
using LayoutA             = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using ElementB            = cutlass::float_e4m3_t;                          // Element type for B matrix operand
using LayoutB             = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using ElementC            = cutlass::float_e4m3_t;                          // Element type for C and D matrix operands
using LayoutC             = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

using ElementD           = ElementC;
using LayoutD            = LayoutC;
constexpr int AlignmentD = AlignmentC;

// MMA type
using ElementAccumulator = float;                                           // Element Accumulator will also be our scale factor type
using ElementCompute = float;


// MMA and Cluster Tile Shapes
// Shape of the tile computed by tcgen05 MMA, could be across 2 SMs if Cluster Shape %2 == 0 
using MmaTileShape_MNK = Shape<_128,_128,_128>;                          
// Shape of the threadblocks in a cluster
using ClusterShape_MNK = Shape<_1,_1,_1>;
 
using ScaleConfig = decltype(cutlass::detail::sm100_trivial_blockwise_scale_config(MmaTileShape_MNK{}));

using LayoutSFA             = decltype(ScaleConfig::deduce_layoutSFA());                     // Layout type for SFA matrix operand
using LayoutSFB             = decltype(ScaleConfig::deduce_layoutSFB());                     // Layout type for SFB matrix operand


using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
    MmaTileShape_MNK, ClusterShape_MNK,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementCompute,
    ElementC, LayoutC, AlignmentC,
    ElementD, LayoutC, AlignmentD,
    cutlass::epilogue::TmaWarpSpecialized1Sm
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
    ElementA, cute::tuple<LayoutA, LayoutSFA>, AlignmentA,
    ElementB, cute::tuple<LayoutB, LayoutSFB>, AlignmentB,
    ElementAccumulator,
    MmaTileShape_MNK, ClusterShape_MNK,
    cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    cutlass::gemm::KernelTmaWarpSpecializedBlockwise1SmSm100 // Note: Groupwise and Blockwise only support 1 SM MMA at this moment
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>,
    CollectiveMainloop,
    CollectiveEpilogue,
    void>;                // Default to ClusterLaunchControl (CLC) based tile scheduler

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
// Strides just iterate over scalars and have no zeros
LayoutSFA layout_SFA;
LayoutSFB layout_SFB;
// Layouts are tiled to the problem size and the strides have zeros
uint64_t seed;

cutlass::HostTensor<ElementA          , LayoutA> tensor_A;
cutlass::HostTensor<ElementAccumulator, cutlass::layout::PackedVectorLayout> tensor_SFA;
cutlass::HostTensor<ElementB          , LayoutB> tensor_B;
cutlass::HostTensor<ElementAccumulator, cutlass::layout::PackedVectorLayout> tensor_SFB;
cutlass::HostTensor<ElementC          , LayoutC> tensor_C;
cutlass::HostTensor<ElementD          , LayoutD> tensor_D;
cutlass::HostTensor<ElementD          , LayoutD> tensor_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;
  bool skip_verification = false;

  float alpha = 1.f, beta = 0.f;
  int iterations = 1000;
  int m = 1024, n = 512, k = 1024, l = 1;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    if (cmd.check_cmd_line_flag("skip-verification")) {
      skip_verification = true;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "81_blackwell_gemm_blockwise\n\n"
      << "  Blackwell FP8 GEMM with Blockwise Scaling using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the l extent (batch) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n"
      << "  --skip-verification         Skip verification.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "81_blackwell_gemm_blockwise" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;
    int bits_input = cutlass::sizeof_bits<Element>::value;
    int bits_output = cutlass::sizeof_bits<Element>::value;

    if (bits_input == 1) {
      scope_max = 2;
      scope_min = 0;
    } else if (bits_input <= 8) {
      scope_max = 2;
      scope_min = -2;
    } else if (bits_output == 16) {
      scope_max = 5;
      scope_min = -5;
    } else {
      scope_max = 8;
      scope_min = -8;
    }

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min, 0);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Helper to initialize a block of device data (scale_tensors)
template <typename Element, typename Layout>
bool initialize_scale_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;

    scope_min = -8;
    scope_max = 8;

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min, 0);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {
  using namespace cute;

  auto gemm_problem_shape = cute::make_shape(options.m, options.n, options.k);

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));

  layout_SFA = ScaleConfig::tile_atom_to_shape_SFA(make_shape(options.m, options.n, options.k, options.l));
  layout_SFB = ScaleConfig::tile_atom_to_shape_SFB(make_shape(options.m, options.n, options.k, options.l));

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto blockscale_a_coord = cutlass::make_Coord(size(filter_zeros(layout_SFA)));
  auto blockscale_b_coord = cutlass::make_Coord(size(filter_zeros(layout_SFB)));

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);
  tensor_SFA.resize(blockscale_a_coord);
  tensor_SFB.resize(blockscale_b_coord);

  initialize_tensor(tensor_A.host_view(), cutlass::Distribution::Uniform, seed + 2022);
  initialize_tensor(tensor_B.host_view(), cutlass::Distribution::Uniform, seed + 2023);
  initialize_tensor(tensor_C.host_view(), cutlass::Distribution::Uniform, seed + 2024);

  initialize_scale_tensor(tensor_SFA.host_view(), cutlass::Distribution::Uniform, seed + 2025);
  initialize_scale_tensor(tensor_SFB.host_view(), cutlass::Distribution::Uniform, seed + 2026);

  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_C.sync_device();
  tensor_D.sync_device();

  tensor_SFA.sync_device();
  tensor_SFB.sync_device();

}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(), stride_A, 
     tensor_B.device_data(), stride_B,
     tensor_SFA.device_data(), layout_SFA,
     tensor_SFB.device_data(), layout_SFB},
    {
      {}, // epilogue.thread
      tensor_C.device_data(), stride_C,
      tensor_D.device_data(), stride_D
    }
  };

  auto &fusion_args = arguments.epilogue.thread;
  fusion_args.alpha = options.alpha;
  fusion_args.beta = options.beta;

  return arguments;
}

bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.k, options.l), stride_A));
  auto B = cute::make_tensor(tensor_B.host_data(),
      cute::make_layout(cute::make_shape(options.n, options.k, options.l), stride_B));
  auto C = cute::make_tensor(tensor_C.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_C));
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_D));
  auto SFA = cute::make_tensor(tensor_SFA.host_data(), layout_SFA);
  auto SFB = cute::make_tensor(tensor_SFB.host_data(), layout_SFB);

  using unused_t = decltype(D);

  cutlass::reference::host::GettBlockScalingMainloopParams<
      ElementAccumulator,
      decltype(A), 
      decltype(SFA), 
      decltype(B),
      decltype(SFB)
    > mainloop_params{A, SFA, B, SFB};

  cutlass::reference::host::GettEpilogueParams<
      ElementAccumulator,
      ElementAccumulator,
      ElementAccumulator,
      ElementCompute,
      decltype(C),
      decltype(D)
  > epilogue_params;

  epilogue_params.C = C;
  epilogue_params.D = D;
  epilogue_params.alpha = options.alpha;
  epilogue_params.beta = options.beta;

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  // compare_reference
  tensor_D.sync_host();
  bool passed = cutlass::reference::host::TensorEquals(tensor_ref_D.host_view(), tensor_D.host_view());

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  
  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  
  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

 
  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  
  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  Result result;
  if (!options.skip_verification) {
    // Check if output from CUTLASS kernel and reference kernel are equal or not
    result.passed = verify(options);

    std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

    if (!result.passed) {
      exit(-1);
    }
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least sm100a.
  
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major != 10 || props.minor != 0) {
    std::cerr << "This example requires a GPU with compute capability 100a)." << std::endl;
    return 0;
  } 
  

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Run
  //
#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
  run<Gemm>(options);
#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

