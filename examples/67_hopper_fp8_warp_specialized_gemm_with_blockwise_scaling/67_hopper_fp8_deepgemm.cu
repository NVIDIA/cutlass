#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Grouped scale Hopper FP8 GEMM example using CUTLASS 3.0 APIs for NVIDIA Hopper architecture
    This example demonstrate a grouped scaled FP8 GEMM using the new CUTLASS 3.0.
    APIs on NVIDIA Hopper architecture. New features that will be showcased in this example are as follows:
    1. NVIDIA Hopper architecture introduces a new series of tensor core instructions (GMMA)
    which are more efficient than the Ampere tensor core instructions.
    2. NVIDIA Hopper architecture includes new Tensor Memory Accelerator (TMA) unit to transfer large
    blocks of data efficiently between global memory and shared memory. TMA also supports asynchronous
    copies between thread blocks in a cluster.
    3. This example uses the Warp Specialized kernel design (see /media/docs/efficient_gemm.md for details).
    4. This example shows all important fusions used by FP8 gemm kernels, i.e., grouped scale factor along M for
    A, blocked scale factor along K for A tensor, blocked scale factor for B tensor, the abs_max value of D tensor.
    5. A simple way to tune the CTA rasterization direction and swizzle pattern of Hopper kernels. Both the
    CTA rasterization direction and swizzle pattern impact cross-CTA locality of accesses. By tuning we can
    improve performance.
    Examples:
      $ ./examples/67_hopper_fp8_warp_specialized_gemm_with_blockwise_scaling/67_hopper_fp8_deepgemm  \
        --m=4096 --iterations=1000
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
// #include "cutlass/util/reference/host/tensor_copy.h"
// #include "cutlass/util/reference/host/tensor_compare.h"
// #include "cutlass/util/reference/host/tensor_norm.h"

// Includes from examples directory
#include "helper.h"
// #include "reference/host/gemm_with_groupwise_scaling.h"

#include "deep_gemm/fp8_gemm.cuh"

// using namespace cute;
using namespace deep_gemm;

// #define CUTLASS_ARCH_MMA_SM90_SUPPORTED

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)


// Command line options parsing
struct Options {
  bool help;
  int iterations;
  int m, n, k;

  Options():
    help(false),
    m(4096),
    n(4096),
    k(4096),
    iterations(10)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);
    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, defaults.m);

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "67_hopper_fp8_deepgemm\n\n"
      << "  Hopper FP8 DeepGEMM kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the m size\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    return out;
  }

  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

constexpr int cdiv(int a, int b) {
  return (a + b - 1) / b;
}


// #if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;
    int bits_input = cutlass::sizeof_bits<Element>::value;
    int bits_output = cutlass::sizeof_bits<Element>::value;

    if (bits_input == 1) {
      scope_max = 2;
      scope_min = 0;
    } else if (bits_input <= 8) {
      scope_max = 2;
      scope_min = -2;
    } else if (bits_output == 16) {
      scope_max = 5;
      scope_min = -5;
    } else {
      scope_max = 8;
      scope_min = -8;
    }

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min, 0);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}

/// Helper to initialize a block of device data (scale_tensors)
template <typename Element, typename Layout>
bool initialize_scale_tensor(
  cutlass::TensorView<Element, Layout> view,
  cutlass::Distribution::Kind dist_kind,
  uint64_t seed) {

  if (dist_kind == cutlass::Distribution::Uniform) {

    double scope_max, scope_min;

    scope_min = -1;
    scope_max = 1;

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min, 0);
  }
  else if (dist_kind == cutlass::Distribution::AllZeros) {
    cutlass::reference::host::TensorFill(view);
  }
  else if (dist_kind == cutlass::Distribution::Identity) {

    cutlass::reference::host::TensorFillIdentity(view);
  }
  else if (dist_kind == cutlass::Distribution::Gaussian) {

    cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
  }
  else if (dist_kind == cutlass::Distribution::Sequential) {
    cutlass::reference::host::BlockFillSequential(view.data(), view.capacity());
  }
  else {
    throw std::runtime_error("Not implementated.");
  }

  return true;
}


/// Todo: add reference check
bool verify(const Options &options) {
  //
  // Compute reference output
  //
  return true;
}

struct TestBed {
  using Element = cutlass::float_e4m3_t;
  using ElementScale = float;
  using ElementAcc = float;
  using ElementOut = cutlass::bfloat16_t;

  cutlass::HostTensor<Element, cutlass::layout::RowMajor> Tensor_lhs;
  cutlass::HostTensor<Element, cutlass::layout::RowMajor> Tensor_rhs;
  cutlass::HostTensor<ElementScale, cutlass::layout::ColumnMajor> Tensor_lhs_scale;
  cutlass::HostTensor<ElementScale, cutlass::layout::RowMajor> Tensor_rhs_scale;
  cutlass::HostTensor<ElementOut, cutlass::layout::RowMajor> Tensor_out;


  /// Initialize operands to be used in the GEMM
  void initialize(
    const Options &options,
    uint64_t seed = 2025) {

    Tensor_lhs.resize({options.m, options.k}); //[m, k]
    Tensor_rhs.resize({options.n, options.k}); //[n, k]
    Tensor_lhs_scale.resize({options.m, cdiv(options.k, 128)}); // [m, cdiv(k, 128)] column major
    Tensor_rhs_scale.resize({cdiv(options.n, 128), cdiv(options.k, 128)}); // [cdiv(n, 128), cdiv(k, 128)]
    Tensor_out.resize({options.m, options.n}); // [m, n]

    initialize_tensor(Tensor_lhs.host_view(), cutlass::Distribution::Uniform, seed + 1);
    initialize_tensor(Tensor_rhs.host_view(), cutlass::Distribution::Uniform, seed + 2);
    initialize_scale_tensor(Tensor_lhs_scale.host_view(), cutlass::Distribution::Uniform, seed + 3);
    initialize_scale_tensor(Tensor_rhs_scale.host_view(), cutlass::Distribution::Uniform, seed + 4);

    Tensor_lhs.sync_device();
    Tensor_rhs.sync_device();
    Tensor_lhs_scale.sync_device();
    Tensor_rhs_scale.sync_device();
    Tensor_out.sync_device();

  }

  void run(Options &options)
  {
    hipDeviceProp_t props;
    int current_device;
    CUDA_CHECK(hipGetDevice(&current_device));
    CUDA_CHECK(hipGetDeviceProperties(&props, current_device));

    initialize(options);

    hipStream_t stream{nullptr};
    constexpr auto N = 4096;
    constexpr auto K = 4096;
    constexpr auto BLOCK_M = 128;
    constexpr auto BLOCK_N = 128;
    constexpr auto kNumStages = 5;
    constexpr auto kNumTMAMulticast = 2;
    const int num_sms = 132; // for H100 
    const int best_smem_size = 199376; 

    // Make a templated GEMM
    using GemmKernel = Gemm<N, K, BLOCK_M, BLOCK_N, 128, 1, kNumStages, kNumTMAMulticast, GemmType::Normal>;

    int m = options.m;
    // DeepGEMM requires __hip_fp8_e4m3_fnuz input and __hip_bfloat16 output
    __hip_fp8_e4m3_fnuz* lhs = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(Tensor_lhs.device_data());
    __hip_fp8_e4m3_fnuz* rhs = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(Tensor_rhs.device_data());
    float* lhs_scales = Tensor_lhs_scale.device_data();
    float* rhs_scales = Tensor_rhs_scale.device_data();
    __hip_bfloat16* out = reinterpret_cast<__hip_bfloat16*>(Tensor_out.device_data());

    // Launch kernel
    auto tma_a_desc = GemmKernel::make_2d_tma_a_desc(lhs, m);
    auto tma_b_desc = GemmKernel::make_2d_tma_b_desc(rhs);
    auto tma_scales_a_desc = GemmKernel::make_2d_tma_scales_a_desc(lhs_scales, m);
    auto tma_d_desc = GemmKernel::make_2d_tma_d_desc(out, m);
    GemmKernel::run(out, rhs_scales, nullptr,
                  m,
                  tma_a_desc, tma_b_desc, tma_scales_a_desc, tma_d_desc,
                  stream, num_sms, best_smem_size);

    CUDA_CHECK(hipStreamSynchronize(stream));

    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "run done\n";
    // TODO: reference check
    Result result;
    // result.passed = verify(options, ScaleMsPerTile, ScaleNsPerTile);

    // std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

    // if (!result.passed) {
    //  exit(-1);
    // }

    // Run profiling loop
    if (options.iterations > 0)
    {
      GpuTimer timer;
      timer.start();
      for (int iter = 0; iter < options.iterations; ++iter) {
        // initialize(options);
        GemmKernel::run(out, rhs_scales, nullptr,
                    m,
                    tma_a_desc, tma_b_desc, tma_scales_a_desc, tma_d_desc,
                    stream, num_sms, best_smem_size);
      }
      timer.stop();

      // Compute average runtime and GFLOPs.
      float elapsed_ms = timer.elapsed_millis();
      result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
      result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

      std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
      std::cout << "  Tile shape (M, N, K): (128, 128, 128)" << std::endl;
      std::cout << "  ScaleGranularityM: 1 (ScaleMsPerTile: 128)" << std::endl;
      std::cout << "  ScaleGranularityN: 128 (ScaleNsPerTile: 1)" << std::endl;
      std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
      std::cout << "  GFLOPS: " << result.gflops << std::endl;
      fflush(stdout);
    }
  }
};

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////


int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major != 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }
  //
  // Parse options
  //
  #if defined (CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  TestBed testbed{};
  testbed.run(options);

  #endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
