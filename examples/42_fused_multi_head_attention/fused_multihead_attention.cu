#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holdvr nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief CUTLASS Attention Example.

    This workload computes an attention example with non-fixed sequence length input. Pointers of arrays
    are fed into grouped-GEMM functions fused with softmax for computation.

    Examples:

      # Run an attention example with default setup (max sequence length = 1024, batch size = 16, head size = 64, head number = 12)
      $ ./examples/41_multi_head_attention/41_multi_head_attention

      # Run an attention example with batch size = 64 and head number = 16 without checking the correctness
      $ ./examples/41_multi_head_attention/41_multi_head_attention --head_number=16 --batch_size=64 --reference-check=false

      Acknowledgement: this example is inspired by the idea originally prototyped by ByteDance Inc.

*/

/////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/gemm_transpose_operands.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/kernel/default_gemm_complex.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/epilogue/threadblock/epilogue_with_visitor.h"
#include "cutlass/fast_math.h"
#include "kernel_forward.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result {

  double runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  //
  // Methods
  //

  Result(
    double runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess
  ):
    runtime_ms(runtime_ms), gflops(gflops), status(status), error(error), passed(true) { }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;
  bool use_mask;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes0;
  std::vector<cutlass::gemm::GemmCoord> problem_sizes1;

  std::vector<cutlass::gemm::GemmCoord> problem_sizes0_real;
  std::vector<cutlass::gemm::GemmCoord> problem_sizes1_real;

  int alignment;
  int head_number;
  int batch_size;
  int head_size;
  int seq_length;
  int iterations;
  int cuda_streams;

  // alpha0, alpha1 and beta are fixed 
  // in this multi-head attention example
  float alpha0;
  float alpha1;
  float beta;

  //
  // Methods
  // 

  Options():
    help(false),
    error(false),
    alignment(1),
    reference_check(true),
    head_number(12),
    batch_size(16),
    head_size(64),
    seq_length(1024),
    use_mask(false),
    iterations(20),
    cuda_streams(0)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("alignment", alignment, 1);
    cmd.get_cmd_line_argument("head_number", head_number, 12);
    cmd.get_cmd_line_argument("batch_size", batch_size, 16);
    cmd.get_cmd_line_argument("head_size", head_size, 64);
    cmd.get_cmd_line_argument("seq_length", seq_length, 1024);
    cmd.get_cmd_line_argument("use_mask", use_mask, false);
    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("streams", cuda_streams, 0);
    cmd.get_cmd_line_argument("reference-check", reference_check, true);

    randomize_problems();

  }

  void randomize_problems() {

    int problem_count = head_number * batch_size;

    problem_sizes0.reserve(problem_count);
    problem_sizes1.reserve(problem_count);

    // When using mask, the original inputs are not padded
    // and we need to save these info.
    if (use_mask) {
      problem_sizes0_real.reserve(problem_count);
      problem_sizes1_real.reserve(problem_count);
    }

    for (int i = 0; i < batch_size; ++i) {
      // problems belonging to the same batch share the same seq len
      int m_real = seq_length; // (rand() % seq_length);
      int m = (m_real + alignment - 1) / alignment * alignment;
      int n = m;
      int k = head_size;

      for (int j = 0; j < head_number; ++j) {
        cutlass::gemm::GemmCoord problem0(m, n, k);
        cutlass::gemm::GemmCoord problem1(m, k, n);
        problem_sizes0.push_back(problem0);
        problem_sizes1.push_back(problem1);

        if (use_mask) {
          cutlass::gemm::GemmCoord problem0_real(m_real, m_real, k);
          cutlass::gemm::GemmCoord problem1_real(m_real, k, m_real);
          problem_sizes0_real.push_back(problem0_real);
          problem_sizes1_real.push_back(problem1_real);
        }

      }
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "42_fused_multi_head_attention\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --head_number=<int>         Head number in multi-head attention (default: --head_number=12)\n"
      << "  --batch_size=<int>          Batch size in multi-head attention (default: --batch_size=16)\n"
      << "  --head_size=<int>           Head size in multi-head attention (default: --head_size=64)\n"
      << "  --seq_length=<int>          Max sequence length in multi-head attention (default: --seq_length=1024)\n"
      << "  --use_mask=<bool>           If true, performs padding-like masking in softmax.\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n"
      << "  --reference-check=<bool>    If true, performs reference check.\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = int64_t();

    for (auto const & problem : problem_sizes0) {
      // Two flops per multiply-add
      fmas += problem.product() * 2;
    }
    
    // Multiply another '2' because of the back-to-back GEMM problems in attention
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};



///////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Attention>
class TestbedAttention {
public:

  //
  // Type definitions
  //

  using ElementQ = typename Attention::scalar_t;
  using ElementK = typename Attention::scalar_t;
  using ElementP = typename Attention::accum_t;
  using ElementAccumulator = typename Attention::accum_t;
  using ElementV = typename Attention::scalar_t;
  using ElementO = typename Attention::output_t;

  using ElementCompute = typename Attention::accum_t;

  using ElementNorm = typename Attention::accum_t;
  using ElementSum = typename Attention::accum_t;
  using ElementSoftmaxCompute = typename Attention::accum_t;

  using LayoutQ = cutlass::layout::RowMajor;
  using LayoutK = cutlass::layout::RowMajor;
  using LayoutK_T = cutlass::layout::ColumnMajor; // transposed
  using LayoutP = cutlass::layout::RowMajor;
  using LayoutV = cutlass::layout::RowMajor;
  using LayoutO = cutlass::layout::RowMajor;

  using MatrixCoord = typename LayoutP::TensorCoord;

private:

  //
  // Data members
  //

  Options & options;

  /// Initialization
  cutlass::Distribution::Kind init_Q;
  cutlass::Distribution::Kind init_K;
  cutlass::Distribution::Kind init_P;
  cutlass::Distribution::Kind init_V;
  cutlass::Distribution::Kind init_O;
  uint32_t seed;

  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device0;
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device1;
  cutlass::DeviceAllocation<cutlass::gemm::GemmCoord> problem_sizes_device0_real;

  std::vector<int64_t> offset_Q;
  std::vector<int64_t> offset_K;
  std::vector<int64_t> offset_P;
  std::vector<int64_t> offset_V;
  std::vector<int64_t> offset_O;

  std::vector<int64_t> ldq_host;
  std::vector<int64_t> ldk_host;
  std::vector<int64_t> ldp_host;
  std::vector<int64_t> ldv_host;
  std::vector<int64_t> ldo_host;
  std::vector<int64_t> seqlen_host;

  cutlass::DeviceAllocation<int64_t> ldq;
  cutlass::DeviceAllocation<int64_t> ldk;
  cutlass::DeviceAllocation<int64_t> ldp;
  cutlass::DeviceAllocation<int64_t> ldv;
  cutlass::DeviceAllocation<int64_t> ldo;
  cutlass::DeviceAllocation<int64_t> seqlen;

  cutlass::DeviceAllocation<ElementQ> block_Q;
  cutlass::DeviceAllocation<ElementK> block_K;
  cutlass::DeviceAllocation<ElementP> block_P;
  cutlass::DeviceAllocation<ElementV> block_V;
  cutlass::DeviceAllocation<ElementO> block_O;
  cutlass::DeviceAllocation<ElementNorm> block_Norm;
  cutlass::DeviceAllocation<ElementSum> block_Sum;

  cutlass::DeviceAllocation<int64_t> offset_P_Device;

  cutlass::DeviceAllocation<ElementQ *> ptr_Q;
  cutlass::DeviceAllocation<ElementK *> ptr_K;
  cutlass::DeviceAllocation<ElementP *> ptr_P;
  cutlass::DeviceAllocation<ElementV *> ptr_V;
  cutlass::DeviceAllocation<ElementO *> ptr_O;

public:

  //
  // Methods
  //

  TestbedAttention(
    Options &options_,
    cutlass::Distribution::Kind init_Q_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_K_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_P_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_V_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_O_ = cutlass::Distribution::Uniform,
    uint32_t seed_ = 3080
  ):
    options(options_), init_Q(init_Q_), init_K(init_K_), init_P(init_P_), init_V(init_V_), init_O(init_O_), seed(seed_) { }

  int problem_count() const {
    return (options.head_number * options.batch_size);
  }

private:

  /// Helper to initialize a tensor view
  template <typename Element>
  void initialize_tensor_(
    Element *ptr,
    size_t capacity, 
    cutlass::Distribution::Kind dist_kind,
    uint32_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      Element scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<ElementP>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        scope_max = 8;
        scope_min = -8;
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::device::BlockFillRandomUniform(
        ptr, capacity, seed, scope_max, scope_min, 0);
    } 
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::device::BlockFillRandomGaussian(
        ptr, capacity, seed, Element(), Element(0.5f));
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      // Fill with increasing elements
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(1), Element());
    } 
    else {

      // Fill with all 1s
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(), Element(1));
    }
  }

  /// Initializes data structures
  void initialize_() {

    //
    // Set scalors for the mha example
    //

    options.alpha0 = 1.0f / sqrt(float(options.head_size));
    options.alpha1 = 1.0f;
    options.beta = 0;

    //
    // Choose random problem sizes
    //

    // construct a few problems of random sizes
    srand(seed);

    int64_t total_elements_Q = 0;
    int64_t total_elements_K = 0;
    int64_t total_elements_P = 0;
    int64_t total_elements_V = 0;
    int64_t total_elements_O = 0;

    ldq_host.resize(problem_count());
    ldk_host.resize(problem_count());
    ldp_host.resize(problem_count());
    ldv_host.resize(problem_count());
    ldo_host.resize(problem_count());
    seqlen_host.resize(problem_count());

    for (int32_t i = 0; i < problem_count(); ++i) {

      auto problem = options.problem_sizes0.at(i);

      ldq_host.at(i) = LayoutQ::packed({problem.m(), problem.k()}).stride(0);
      ldk_host.at(i) = LayoutK::packed({problem.n(), problem.k()}).stride(0);
      ldp_host.at(i) = LayoutP::packed({problem.m(), problem.n()}).stride(0);
      ldv_host.at(i) = LayoutV::packed({problem.n(), problem.k()}).stride(0);
      ldo_host.at(i) = LayoutO::packed({problem.m(), problem.k()}).stride(0);

      // m = n for attention problems.
      seqlen_host.at(i) = problem.m();

      offset_Q.push_back(total_elements_Q);
      offset_K.push_back(total_elements_K);
      offset_P.push_back(total_elements_P);
      offset_V.push_back(total_elements_V);
      offset_O.push_back(total_elements_O);

      int64_t elements_Q = problem.m() * problem.k();
      int64_t elements_K = problem.k() * problem.n();
      int64_t elements_P = problem.m() * problem.n();
      int64_t elements_V = problem.n() * problem.k();
      int64_t elements_O = problem.m() * problem.k();

      total_elements_Q += elements_Q;
      total_elements_K += elements_K;
      total_elements_P += elements_P;
      total_elements_V += elements_V;
      total_elements_O += elements_O;

    }

    problem_sizes_device0.reset(problem_count());
    problem_sizes_device1.reset(problem_count());
    problem_sizes_device0.copy_from_host(options.problem_sizes0.data());
    problem_sizes_device1.copy_from_host(options.problem_sizes1.data());

    if (options.use_mask) {
      problem_sizes_device0_real.reset(problem_count());
      problem_sizes_device0_real.copy_from_host(options.problem_sizes0_real.data());
    }

    ldq.reset(problem_count());
    ldk.reset(problem_count());
    ldp.reset(problem_count());
    ldv.reset(problem_count());
    ldo.reset(problem_count());
    seqlen.reset(problem_count());

    ldq.copy_from_host(ldq_host.data());
    ldk.copy_from_host(ldk_host.data());
    ldp.copy_from_host(ldp_host.data());
    ldv.copy_from_host(ldv_host.data());
    ldo.copy_from_host(ldo_host.data());
    seqlen.copy_from_host(seqlen_host.data());

    //
    // Assign pointers
    //

    block_Q.reset(total_elements_Q);
    block_K.reset(total_elements_K);
    block_P.reset(total_elements_P);
    block_V.reset(total_elements_V);
    block_O.reset(total_elements_O);

    offset_P_Device.reset(problem_count());

    // sync offset with device
    cutlass::device_memory::copy_to_device(offset_P_Device.get(), offset_P.data(), offset_P.size());

    std::vector<ElementQ *> ptr_Q_host(problem_count());
    std::vector<ElementK *> ptr_K_host(problem_count());
    std::vector<ElementP *> ptr_P_host(problem_count());
    std::vector<ElementV *> ptr_V_host(problem_count());
    std::vector<ElementO *> ptr_O_host(problem_count());
    std::vector<ElementNorm *> ptr_norm_host(problem_count());
    std::vector<ElementSum *> ptr_sum_host(problem_count());

    for (int32_t i = 0; i < problem_count(); ++i) {
      ptr_Q_host.at(i) = block_Q.get() + offset_Q.at(i);
      ptr_K_host.at(i) = block_K.get() + offset_K.at(i);
      ptr_P_host.at(i) = block_P.get() + offset_P.at(i);
      ptr_V_host.at(i) = block_V.get() + offset_V.at(i);
      ptr_O_host.at(i) = block_O.get() + offset_O.at(i);
    }

    ptr_Q.reset(problem_count());
    ptr_Q.copy_from_host(ptr_Q_host.data());
    
    ptr_K.reset(problem_count());
    ptr_K.copy_from_host(ptr_K_host.data());
    
    ptr_P.reset(problem_count());
    ptr_P.copy_from_host(ptr_P_host.data());

    ptr_V.reset(problem_count());
    ptr_V.copy_from_host(ptr_V_host.data());

    ptr_O.reset(problem_count());
    ptr_O.copy_from_host(ptr_O_host.data());

    //
    // Initialize the problems of the workspace
    //

    initialize_tensor_(block_Q.get(), total_elements_Q, init_Q, seed + 1);
    initialize_tensor_(block_K.get(), total_elements_K, init_K, seed + 2);
    initialize_tensor_(block_V.get(), total_elements_V, init_V, seed + 3);

  }

  template<typename Element>
  bool verify_tensor_(std::vector<Element> vector_Input, \
                       std::vector<Element> vector_Input_Ref,
                       int64_t verify_length = -1) {

    int64_t size = (vector_Input.size() < vector_Input_Ref.size()) ? vector_Input.size() : vector_Input_Ref.size();
    size = (verify_length == -1) ? size : verify_length;

    // 0.05 for absolute error
    float abs_tol = 5e-2f;
    // 10% for relative error
    float rel_tol = 1e-1f;
    for (int64_t i = 0; i < size; ++i) {
      float diff = (float)(vector_Input.at(i) - vector_Input_Ref.at(i));
      float abs_diff = fabs(diff);
      float abs_ref = fabs((float)vector_Input_Ref.at(i) + 1e-5f);
      float relative_diff = abs_diff / abs_ref;
      if ( (isnan(abs_diff) || isinf(abs_diff)) ||  (abs_diff > abs_tol && relative_diff > rel_tol)) {
        printf("[%d/%d] diff = %f, rel_diff = %f, {computed=%f, ref=%f}.\n", int(i), int(size), abs_diff, relative_diff, (float)(vector_Input.at(i)), (float)(vector_Input_Ref.at(i)));
        return false;
      }

    }

    return true;
  }

  /// Verifies the result is a GEMM
  bool verify_() {

    bool passed = true;

    for (int32_t i = 0; i < problem_count(); ++i) {
      cutlass::gemm::GemmCoord problem = options.problem_sizes0.at(i);
      cutlass::gemm::GemmCoord problem1 = options.problem_sizes1.at(i);

      LayoutQ layout_Q(ldq_host.at(i));
      LayoutK_T layout_K(ldk_host.at(i));
      LayoutP layout_P(ldp_host.at(i));
      LayoutV layout_V(ldv_host.at(i));
      LayoutO layout_O(ldo_host.at(i));

      MatrixCoord extent_Q{problem.m(), problem.k()};
      MatrixCoord extent_K{problem.n(), problem.k()};
      MatrixCoord extent_P{problem.m(), problem.n()};
      MatrixCoord extent_V{problem.n(), problem.k()};
      MatrixCoord extent_O{problem.m(), problem.k()};

      cutlass::TensorView<ElementQ, LayoutQ> view_Q(block_Q.get() + offset_Q.at(i), layout_Q, extent_Q);
      cutlass::TensorView<ElementK, LayoutK_T> view_K(block_K.get() + offset_K.at(i), layout_K, extent_K);
      cutlass::TensorView<ElementP, LayoutP> view_P(block_P.get() + offset_P.at(i), layout_P, extent_P);
      cutlass::TensorView<ElementV, LayoutV> view_V(block_V.get() + offset_V.at(i), layout_V, extent_V);

      cutlass::DeviceAllocation<ElementP>    block_Ref(layout_P.capacity(extent_P));
      cutlass::TensorView<ElementP, LayoutP> view_Ref_device(block_Ref.get(), layout_P, extent_P);

      cutlass::DeviceAllocation<ElementO>    block_Ref_O(layout_O.capacity(extent_O));
      cutlass::TensorView<ElementO, LayoutO> view_Ref_O_device(block_Ref_O.get(), layout_O, extent_O);

      // Reference GEMM
      cutlass::reference::device::GemmComplex<
          ElementQ, LayoutQ,
          ElementK, LayoutK_T,
          ElementP, LayoutP, 
          ElementCompute, ElementAccumulator
      >(
        problem,
        ElementAccumulator(options.alpha0), 
        view_Q,
        Attention::MM0::Mma::kTransformA,
        view_K,
        Attention::MM0::Mma::kTransformB,
        ElementAccumulator(options.beta), 
        view_P, 
        view_Ref_device, 
        ElementAccumulator(0)
      );

      // Compute softmax for P. We need to explicitly compute softmax
      // over P because softmax is fused to the second GEMM in the
      // profiled implementation.
      std::vector<ElementP> matrix_Ref(layout_P.capacity(extent_P));
      cutlass::device_memory::copy_to_host(matrix_Ref.data(), block_Ref.get(), matrix_Ref.size());
      cutlass::TensorView<ElementP, LayoutP> view_Ref_host(matrix_Ref.data(), layout_P, extent_P);
      std::vector<ElementNorm> vector_Norm_Ref(problem.m());
      std::vector<ElementSum> vector_Sum_Ref(problem.m());

      int n_dim = options.use_mask ? options.problem_sizes0_real.at(i).n() : problem.n();

      // Compute softmax for referece matrix
      // Assumed a row-major storage
      for (int m = 0; m < problem.m(); m++) {
        ElementSoftmaxCompute max = ElementSoftmaxCompute(view_Ref_host.ref().at({m, 0}));
        for (int n = 1; n < n_dim; n++) {
           max = std::max(max, ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})));
        }

        vector_Norm_Ref.at(m) = ElementNorm(max);

        ElementSoftmaxCompute sum = ElementSoftmaxCompute();
        for (int n = 0; n < n_dim; n++) {
          sum += std::exp( ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})) - max );
        }
        ElementSoftmaxCompute inv_sum = ElementSoftmaxCompute(1.0f / sum);

        vector_Sum_Ref.at(m) = ElementSum(inv_sum);

        for (int n = 0; n < n_dim; n++) {
          view_Ref_host.ref().at({m, n}) = ElementP(
            std::exp( ElementSoftmaxCompute(view_Ref_host.ref().at({m, n})) - max ) * inv_sum
          );
        }

      }

      // when not using mask, problem_real and problem share the same sizes
      if (options.use_mask) {
        for (int m = 0; m < problem.m(); m++) {
          for (int n = n_dim; n < problem.n(); n++) {
            view_Ref_host.ref().at({m, n}) = ElementP(0);
          }
        }
      }

      cutlass::device_memory::copy_to_device(block_P.get() + offset_P.at(i), matrix_Ref.data(), matrix_Ref.size());

      // Reference GEMM
      cutlass::reference::device::GemmComplex<
          ElementP, LayoutP,
          ElementV, LayoutV,
          ElementO, LayoutO, 
          ElementCompute, ElementAccumulator
      >(
        problem1,
        ElementAccumulator(options.alpha1), 
        view_P,
        Attention::MM0::Mma::kTransformA,
        view_V,
        Attention::MM0::Mma::kTransformB,
        ElementAccumulator(options.beta), 
        view_Ref_O_device, 
        view_Ref_O_device, 
        ElementAccumulator(0)
      );

      // Copy to host memory
      cutlass::TensorView<ElementP, LayoutP> view_Ref(matrix_Ref.data(), layout_P, extent_P);

      std::vector<ElementO> matrix_O(layout_O.capacity(extent_O));
      cutlass::device_memory::copy_to_host(matrix_O.data(),   block_O.get() + offset_O.at(i), matrix_O.size());
      std::vector<ElementO> matrix_Ref_O(layout_O.capacity(extent_O));
      cutlass::device_memory::copy_to_host(matrix_Ref_O.data(), block_Ref_O.get(), matrix_Ref_O.size());

      // printf("Pb %d: \n    Q=(offset=%d, ldq=%d)\n    K=(offset=%d, ldk=%d)\n    O=(offset=%d, ldo=%d)\n",
      //   int(i), int(offset_Q[i]), int(ldq_host[i]), int(offset_K[i]), int(ldk_host[i]), int(offset_O[i]), int(ldo_host[i]));
  
      bool verified_O = false;

      if (!verified_O) {
        verified_O = verify_tensor_<ElementO>(matrix_O, matrix_Ref_O);
      }

      passed = passed && verified_O;

      if (!passed) {
        std::cerr << "\n***\nError - problem " << i << " failed the QA check\n***\n" << std::endl;

        if (!verified_O) {
          std::cout << "Final matrix output is incorrect" << std::endl;
        }

        return passed;
      }

    }

    return passed;
  }

public:


  /// Executes a CUTLASS Attention kernel and measures runtime.
  Result profile_grouped() {

    Result result;
    result.passed = false;

    // Initialize the problem
    initialize_();

    typename Attention::Params p;
    { // set parameters
      p.query_ptr = block_Q.get();
      p.key_ptr = block_K.get();
      p.value_ptr = block_V.get();
      p.logsumexp_ptr = nullptr; // Only needed for bw
      p.output_accum_ptr = nullptr;
      if (Attention::kNeedsOutputAccumulatorBuffer) {
        hipMalloc(&p.output_accum_ptr, block_O.size() * sizeof(typename Attention::output_accum_t));
      }
      p.output_ptr = block_O.get();

      // TODO: support arbitrary seq lengths
      // if (cu_seqlens_q.has_value()) {
      //   p.cu_seqlens_q_ptr = (int32_t*)cu_seqlens_q->data_ptr();
      //   p.cu_seqlens_k_ptr = (int32_t*)cu_seqlens_k->data_ptr();
      // }

      p.num_heads = options.head_number;
      p.num_batches = options.batch_size;
      p.head_dim = options.head_size;
      p.head_dim_value = options.head_size;
      p.num_queries = options.seq_length;
      p.num_keys = options.seq_length;
      p.causal = false; // TODO: add an option to test causal as well

      p.q_strideM = ldq_host[0];
      p.k_strideM = ldk_host[0];
      p.v_strideM = ldv_host[0];
      p.q_strideH = p.q_strideM * options.seq_length;
      p.k_strideH = p.k_strideM * options.seq_length;
      p.v_strideH = p.v_strideM * options.seq_length;
      p.o_strideH = options.head_size * options.seq_length;
      p.q_strideB = p.q_strideH * options.head_number;
      p.k_strideB = p.k_strideH * options.head_number;
      p.v_strideB = p.v_strideH * options.head_number;
      p.o_strideB = options.head_size * options.seq_length * options.head_number;
    }

    // launch kernel :)
    constexpr auto kernel_fn = attention_kernel_batched_impl<Attention>;
    int smem_bytes = sizeof(typename Attention::SharedStorage);
    if (smem_bytes > 0xc000) {
      hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
    }
    if (!Attention::check_supported(p)) {
      std::cerr << "Kernel does not support these inputs" << std::endl;
      return result;
    }
    kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);

    // Wait for completion
    result.error = hipDeviceSynchronize();

    if (result.error != hipSuccess)  {
      std::cerr << "Kernel execution error: " << hipGetErrorString(result.error);
      return result;
    }

    //
    // Verify correctness
    //
    result.passed = true;

    if (options.reference_check) {
      result.passed = verify_();
    }

    //
    // Warm-up run of the grouped GEMM object
    //

    kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);

    if (result.status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to run CUTLASS Attention kernel." << std::endl;
      return result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return -1;
      }
    }

    // Record an event at the start of a series of GEMM operations
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    //
    // Run profiling loop
    //

    for (int iter = 0; iter < options.iterations; ++iter) {
      kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMM operations have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Compute average runtime and GFLOPs.
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    //
    // Cleanup
    //

    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    std::cout << std::endl;
    std::cout << "CUTLASS Attention:\n"
      << "====================================================" << std::endl;
    std::cout << "    " << " {max sequence length, head size, head number, batch size} = {" << options.seq_length \
      << ", " << options.head_size << ", " << options.head_number << ", " << options.batch_size << "}." << std::endl;
    std::cout << std::endl;
    std::cout << "    " << "Runtime: " << result.runtime_ms << " ms" << std::endl;
    std::cout << "    " << "GFLOPs: " << result.gflops << std::endl;

    return result;
  }


};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
  
    //
    // This example requires an NVIDIA Ampere-architecture GPU.
    //

    std::cout 
      << "CUTLASS's CUTLASS Attention example requires a GPU of NVIDIA's Ampere Architecture or "
      << "later (compute capability 80 or greater).\n";

    return 0;
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

  if (options.use_mask) {
    std::cerr << "--use_mask is not supported at the moment\n";
    return -2;
  }
  if (options.alignment != 1) {
    std::cerr << "--alignment=1 is the only supported value\n";
    return -2;
  }
  using ArchTag = cutlass::arch::Sm80;

  constexpr bool kIs64x64 = true;
  // Set grid size
  constexpr int64_t kQueriesPerBlock = kIs64x64 ? 64 : 32;
  constexpr int64_t kKeysPerBlock = kIs64x64 ? 64 : 128;
  if (kIs64x64 && options.head_size > kKeysPerBlock) {
    std::cerr << "WARNING: you will get better performance with `kIs64x64=false`\n";
  }

  constexpr bool kSingleValueIteration = true;
  if (kSingleValueIteration && options.head_size > kKeysPerBlock) {
    std::cerr << "ERROR  : Use kSingleValueIteration to keep output in RF. " \
    "This requires to have `head_size <= kKeysPerBlock` " \
    "but head_size=" << options.head_size << " and kKeysPerBlock=" << kKeysPerBlock << "\n";
    return -2;
  }
  if (!kSingleValueIteration && options.head_size <= kKeysPerBlock) {
    std::cerr << "WARNING: you will get better performance with `kSingleValueIteration=true` (keeps the output in RF rather than GMEM)\n";
  }

  using Attention = AttentionKernel<
    cutlass::half_t, // scalar_t
    ArchTag,
    true, // memory is aligned
    kQueriesPerBlock,
    kKeysPerBlock,
    kSingleValueIteration
  >;

  //
  // Test and profile
  //

  TestbedAttention<Attention> testbed(options);

  Result result = testbed.profile_grouped();
  if (!result.passed) {
    std::cout << "Profiling CUTLASS attention has failed.\n";
    std::cout << "\nFailed\n";
    return -1;
  }

  std::cout << "\nPassed\n";

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
