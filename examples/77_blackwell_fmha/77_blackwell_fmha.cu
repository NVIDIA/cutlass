#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Example implementation of fused multi-head attention for the NVIDIA Blackwell SM100
    architecture using CUTLASS 3.

    MQA/GQA
    -------

    The head dimension can be represented as a tuple, where the K/V strides in the
    first dimension is zero. This has the effect of MQA or GQA.
    * MHA is (head_size:head_stride).
    * MQA is (head_size:head_stride) in Q and (head_size:_0) in K and V.
    * GQA is (grouped_heads,heads_kv):(head_stride,grouped_heads*head_stride) in Q
      and (grouped_heads,heads_kv):(0,head_stride) in K and V

    Output Scale
    ------------

    The output scale gets passed to the collective mainloop, and is applied
    using FP32 compute pre-quantization

    Variable Sequence Length
    ------------------------

    For variable sequence length, pass in VariableLength objects
    (max_seqlen, cumulative_seqlen_ptr) in the problem shape for
    seqlen Q and KV.

    Support
    ---------

    Right now e4m3 with fp32 compute is using a 256x256 tiling and a head dimension
    of 128 is supported.


    Example usage:
      $ ./examples/77_blackell_fmha/77_blackell_fmha_fp8 \
            --b=2048 --h=2048 --d=2048 --q=2048 --k=2048
*/

#define DSHOW(x) print(#x ": "); print(x); print("\n");
#define DSHOWT(x) print(#x ": "); print_tensor(x); print("\n");

#include <iostream>
#include <random>
#include <regex>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/kernel_hardware_info.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "reference/fmha_fwd_reference.hpp"
#include "reference/reference_abs_error.hpp"

#include "device/fmha.hpp"
#include "collective/fmha_fusion.hpp"
#include "collective/sm100_fmha_fwd_mainloop_tma_warpspecialized.hpp"
#include "collective/sm100_fmha_fwd_epilogue_tma_warpspecialized.hpp"
#include "kernel/fmha_options.hpp"
#include "kernel/fmha_tile_scheduler.hpp"
#include "kernel/sm100_fmha_fwd_kernel_tma_warpspecialized.hpp"

///////////////////////////////////////////////////////////////////////////////////////////////////

using namespace cute;
using namespace cutlass::fmha::kernel;
using namespace cutlass::fmha::collective;
using namespace cutlass::fmha;

///////////////////////////////////////////////////////////////////////////////////////////////////

enum class InitStyle {
  kOne, kLinearStride128, kLinearStride1, kRandom, kNone
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help = false;
  bool error = false;

  int b = 1;
  int h = 1;
  int h_k = 1;
  int q = 256;
  int k = 256;
  int d = 128;
  int iterations = 3;
  bool verify = false;
  bool verbose = false;

  bool causal = false;
  bool residual = false;
  bool varlen = false;
  int sm_count = 0;

  std::string kernel_filter;

  InitStyle init_style_q = InitStyle::kRandom;
  InitStyle init_style_k = InitStyle::kRandom;
  InitStyle init_style_v = InitStyle::kRandom;

  static void get_init_style_argument(cutlass::CommandLine& cmd, const char* name, InitStyle& dst, InitStyle const& src) {
    std::string s;
    cmd.get_cmd_line_argument(name, s, s);
    if (s.empty()) {
      dst = src;
    }
    else {
      if (s == "r") {
        dst = InitStyle::kRandom;
      }
      else if (s == "1") {
        dst = InitStyle::kOne;
      }
      else if (s == "d") {
        dst = InitStyle::kLinearStride1;
      }
      else if (s == "s") {
        dst = InitStyle::kLinearStride128;
      }
      else if (s == "n") {
        dst = InitStyle::kNone;
      }
      else {
        std::cout << "Error: " << s << " is not a valid input type.\n";
        std::exit(-1);
      }
    }
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("d", d, defaults.d);
    cmd.get_cmd_line_argument("h", h, -1);
    if (h == -1) h = 2048 / d;

    cmd.get_cmd_line_argument("h_k", h_k, -1);
    if (h_k == -1) h_k = h;

    cmd.get_cmd_line_argument("q", q, -1);
    cmd.get_cmd_line_argument("k", k, -1);
    if (q == -1) q = k;
    if (k == -1) k = q;
    if (q == -1 && k == -1) q = k = defaults.q;

    cmd.get_cmd_line_argument("b", b, -1);
    if (b == -1) b = 16384 / k;
    if (b == 0) b = 1;

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
    verify = cmd.check_cmd_line_flag("verify");
    verbose = cmd.check_cmd_line_flag("verbose");
    varlen = cmd.check_cmd_line_flag("varlen");
    std::string mask;
    cmd.get_cmd_line_argument<std::string>("mask", mask, "");
    if (mask == "no" || mask == "") {
      causal = residual = false;
      if (varlen) {
        residual = true;
      }
    }
    else if (mask == "causal") {
      residual = false;
      causal = true;
    }
    else if (mask == "residual") {
      residual = true;
      causal = false;
    }
    cmd.get_cmd_line_argument("sm-count", sm_count, defaults.sm_count);
    
    get_init_style_argument(cmd, "init-style", init_style_q, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style", init_style_k, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style", init_style_v, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style-q", init_style_q, init_style_q);
    get_init_style_argument(cmd, "init-style-k", init_style_k, init_style_k);
    get_init_style_argument(cmd, "init-style-v", init_style_v, init_style_v);

    cmd.get_cmd_line_argument("kernel-filter", kernel_filter, defaults.kernel_filter);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "77_blackwell_fmha\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  fused multi-head attention forward-passkernels targeting NVIDIA's Blackwell architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the B extent\n"
      << "  --h=<int>                   Sets the H extent\n"
      << "  --h_k=<int>                 Sets the H_K/V extent (for GQA/MQA)\n"
      << "  --q=<int>                   Sets the Q extent\n"
      << "  --k=<int>                   Sets the K extent\n"
      << "  --d=<int>                   Sets the D extentn"
      << "  --iterations=<int>          Benchmarking iterations\n"
      << "  --verify                    Verify results\n"
      << "  --verbose                   Print smem and execution time per kernel\n"
      << "  --mask=<no|residual|causal> Enables masking\n"
      << "  --varlen                    Enables variable sequence length\n"
      << "                              B*Q and B*K become the total sequence length\n"
      << "                              and are split B-ways, alternatingly +10% and -10%\n"
      << "                              with the last batch sized to make it fit\n"
      << "                              implies at least residual masking for correctness\n"
      << " --sm-count                   Sets SM count rather than querying it\n"
      << " --kernel-filter=<filter>     Sets regexp to match kernel against\n"
      << "\n";

    return out;
  }
};


///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
void initialize_block(
    DeviceAllocation<Element>& block,
    uint64_t seed=2023, InitStyle init_style = InitStyle::kRandom) {

  switch (init_style) {
    case InitStyle::kOne: {
      cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, (Element) 1, (Element) 1);
      break;
    }
    case InitStyle::kRandom: {
      cutlass::reference::device::BlockFillRandomGaussian(
        block.get(), block.size(), seed, (Element) 0, (Element) 1);
      break;
    }
    case InitStyle::kLinearStride1: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (j % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kLinearStride128: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (i % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kNone: {
      break;
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleResult {
  bool passed = false;
  bool verified = false;
  float runtime_ms = 0;
  double tflops_tc_s = 0;
  double tops_exp2_s = 0;
  double tbytes_s = 0;
  size_t smem_size = 0;
};

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

template<
  bool kIsVarlen,
  class TileShape,
  class DispatchPolicy,
  class ActiveMask,
  class... KernelOptions
>
struct FwdRunner {

#ifdef FP8
  using Element = cutlass::float_e4m3_t;
#else
  using Element = cutlass::half_t;
#endif

  using ElementAccumulatorQK = float;
  using ElementAccumulatorPV = float;
  using ElementOut = cutlass::half_t;

  // Q K D (B H)
  using ProblemShapeRegular = cute::tuple<int, int, int, cute::tuple<cute::tuple<int, int>, int>>;
  using ProblemShapeVarlen = cute::tuple<VariableLength, VariableLength, int, cute::tuple<cute::tuple<int, int>, int>>;
  using ProblemShapeType = std::conditional_t<kIsVarlen, ProblemShapeVarlen, ProblemShapeRegular>;
  
  using StrideQ = cute::tuple<int, _1, cute::tuple<cute::tuple<int, int>, int>>;  // Q D (H_G H_R B)
  using StrideK = cute::tuple<int, _1, cute::tuple<cute::tuple<_0, int>, int>>;  // K D (H_G H_R B)
  using StrideV = StrideK;
  using StrideO = StrideQ;
  using StrideLSE = cute::tuple<_1, cute::tuple<cute::tuple<int, int>, int>>;     // Q   (H_G H_R B)

  static constexpr bool kIsPersistent = find_option_t<Tag::kIsPersistent, true_type, KernelOptions...>::value;
  using TileScheduler = std::conditional_t<kIsPersistent, cutlass::fmha::kernel::PersistentTileScheduler, cutlass::fmha::kernel::IndividualTileScheduler>;

  using Mainloop = 
    cutlass::fmha::collective::Sm100FmhaFwdMainloopTmaWarpspecialized<
      Element, ElementAccumulatorQK, ElementAccumulatorPV,
      TileShape, StrideQ, StrideK, StrideV,
      ActiveMask
    >;
  using Operation = cutlass::fmha::device::FMHA<
    cutlass::fmha::kernel::Sm100FmhaFwdKernelTmaWarpspecialized<
      ProblemShapeType,
      Mainloop,
      cutlass::fmha::collective::Sm100FmhaFwdEpilogueTmaWarpspecialized<
        ElementOut, ElementAccumulatorPV,
        typename Mainloop::TileShapePV,
        StrideO, StrideLSE
      >,
      TileScheduler
    >>;

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;
  uint64_t seed = 0;

  DeviceAllocation<Element> block_Q;
  DeviceAllocation<Element> block_K;
  DeviceAllocation<Element> block_V;
  DeviceAllocation<ElementOut> block_O;
  DeviceAllocation<ElementAccumulatorPV> block_LSE;
  DeviceAllocation<ElementOut> block_ref_O;
  DeviceAllocation<ElementAccumulatorPV> block_ref_LSE;

  std::vector<int> cumulative_seqlen_q;
  std::vector<int> cumulative_seqlen_kv;
  DeviceAllocation<int> device_cumulative_seqlen_q;
  DeviceAllocation<int> device_cumulative_seqlen_kv;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_shape) {
    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      select<0,2,3>(problem_shape),
      stride_Q);

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      select<1,2,3>(problem_shape),
      stride_K);

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      select<1,2,3>(problem_shape),
      stride_V);

    Tensor mO = make_tensor(make_gmem_ptr(block_ref_O.get()),
      select<0,2,3>(problem_shape),
      stride_O);

    Tensor mLSE = make_tensor(make_gmem_ptr(block_ref_LSE.get()),
      select<0,3>(problem_shape),
      stride_LSE);

    fmha_reference(problem_shape, mQ, mK, mV, mO, mLSE, ActiveMask{});

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    const double kMaxDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-2;
    const double kMeanDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-3;

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_O, block_ref_O, max_diff, mean_diff);

    bool passed_O = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_O) {
      std::cerr << "failed O: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    // reference_abs_diff(block_LSE, block_ref_LSE, max_diff, mean_diff);

    bool passed_LSE = true;  // future work
    // bool passed_LSE = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    // if ( ! passed_LSE) {
    //   std::cerr << "failed LSE: max diff " << max_diff 
    //             << " mean " << mean_diff << std::endl;
    // }

    return passed_O && passed_LSE;
  }

  template<class ProblemShape>
  auto initialize_varlen(const ProblemShape& problem_size, const bool kVarlenSame = true) {
    int num_batches = get<3,1>(problem_size);

    // generate Q as --b times
    //    gaussian (--Q, --Q / 2) sampled positive
    //    track cumulative 
    std::mt19937 rng(0x202305151552ull);
    std::normal_distribution<double> dist_q(get<0>(problem_size), get<0>(problem_size) / 2);
    std::normal_distribution<double> dist_kv(get<1>(problem_size), get<1>(problem_size) / 2);
    std::cout << "N: " << num_batches << ", Q: " << get<0>(problem_size) << ", KV: " << get<1>(problem_size) << std::endl;

    auto generate_positive_int = [](auto& dist, auto& gen) {
      int result = 0;
      do {
        result = static_cast<int>(dist(gen));
      } while (result <= 0);
      return result;
    };

    cumulative_seqlen_q = {0};
    cumulative_seqlen_kv = {0};

    int total_seqlen_q = 0;
    int total_seqlen_kv = 0;
    int max_seqlen_q = 0;
    int max_seqlen_kv = 0;

    for (int i = 0; i < num_batches; i++) {
      int seqlen_q = kVarlenSame ? get<0>(problem_size) : generate_positive_int(dist_q, rng);
      int seqlen_kv = kVarlenSame ? get<1>(problem_size) : generate_positive_int(dist_kv, rng);

      total_seqlen_q += seqlen_q;
      total_seqlen_kv += seqlen_kv;

      max_seqlen_q = std::max(max_seqlen_q, seqlen_q);
      max_seqlen_kv = std::max(max_seqlen_kv, seqlen_kv);

      cumulative_seqlen_q.push_back(cumulative_seqlen_q.back() + seqlen_q);
      cumulative_seqlen_kv.push_back(cumulative_seqlen_kv.back() + seqlen_kv);
    }
    std::cout << "Q max: " << max_seqlen_q << " total: " << total_seqlen_q << " vs even " << num_batches * get<0>(problem_size) << std::endl;
    std::cout << "KV max: " << max_seqlen_kv << " total: " << total_seqlen_kv << " vs even " << num_batches * get<1>(problem_size) << std::endl;

    ProblemShape problem_size_for_init = problem_size;
    get<3,1>(problem_size_for_init) = 1;
    get<0>(problem_size_for_init) = total_seqlen_q;
    get<1>(problem_size_for_init) = total_seqlen_kv;

    ProblemShapeType problem_size_for_launch;

    get<0>(problem_size_for_launch) = VariableLength{max_seqlen_q};
    get<1>(problem_size_for_launch) = VariableLength{max_seqlen_kv};
    get<2>(problem_size_for_launch) = get<2>(problem_size);
    get<3>(problem_size_for_launch) = get<3>(problem_size);

    return cute::make_tuple(problem_size_for_init, problem_size_for_launch);
  }


  /// Initialize operands to be used in the GEMM and reference GEMM

  ProblemShapeType initialize(const Options& options) {
    int h_r = options.h / options.h_k;
    assert(options.h % options.h_k == 0);
    auto problem_shape_in = cute::make_tuple(options.q, options.k, options.d, cute::make_tuple(cute::make_tuple(h_r, options.h_k), options.b));
    
    ProblemShapeType problem_shape;
    decltype(problem_shape_in) problem_size;

    if constexpr (kIsVarlen) {
      auto [problem_shape_init, problem_shape_launch] = initialize_varlen(problem_shape_in);
      problem_shape = problem_shape_launch;
      problem_size = problem_shape_init;
    }
    else {
      problem_size = problem_shape_in;
      problem_shape = problem_shape_in;
    }

    get<2>(problem_size) = cutlass::round_up(get<2>(problem_size), 8);  // alignment

    auto shape_QO = select<0,2,3>(problem_size);
    auto shape_KV = select<1,2,3>(problem_size);
    auto shape_LSE = select<0,3>(problem_size);

    int SQ = size<0>(problem_size);
    int SK = size<1>(problem_size);
    int D = size<2>(problem_size);
    int H  = size<3,0>(problem_size);
    int H_K = size<3,0,1>(problem_size);
    int H_Q = size<3,0,0>(problem_size);
    int B = size<3,1>(problem_size);

    stride_Q = make_stride(H*D , _1{}, make_stride(make_stride(D, H_Q*D), H*D*SQ));
    stride_O = stride_Q;
    stride_K = make_stride(H_K*D , _1{}, make_stride(make_stride(_0{}, D), H_K*D*SK));
    stride_V = stride_K;
    stride_LSE = make_stride(_1{}, make_stride(make_stride(SQ, SQ*H_Q), SQ*H));

    if (kIsVarlen) {
      get<2,1>(stride_Q) = 0;
      get<2,1>(stride_K) = 0;
      get<2,1>(stride_V) = 0;
      get<2,1>(stride_O) = 0;
      get<1,1>(stride_LSE) = 0;
    }

    block_Q.reset(size(shape_QO), kIsVarlen ? D*SQ*H : 0);
    block_K.reset(size(shape_KV), kIsVarlen ? D*SK*H_K : 0);
    block_V.reset(size(shape_KV), kIsVarlen ? D*SK*H_K : 0);
    block_O.reset(size(shape_QO), kIsVarlen ? D*SQ*H : 0);
    block_LSE.reset(size(shape_LSE));
    block_ref_O.reset(size(shape_QO));
    block_ref_LSE.reset(size(shape_LSE));

    initialize_block(block_Q, seed + 2023, options.init_style_q);
    initialize_block(block_K, seed + 2022, options.init_style_k);
    initialize_block(block_V, seed + 2021, options.init_style_v);

    if ( ! cumulative_seqlen_q.empty()) {
      device_cumulative_seqlen_q.reset(cumulative_seqlen_q.size());
      device_cumulative_seqlen_q.copy_from_host(
        cumulative_seqlen_q.data(), cumulative_seqlen_q.size());
    }
    if ( ! cumulative_seqlen_kv.empty()) {
      device_cumulative_seqlen_kv.reset(cumulative_seqlen_kv.size());
      device_cumulative_seqlen_kv.copy_from_host(
        cumulative_seqlen_kv.data(), cumulative_seqlen_kv.size());
    }

    if constexpr (kIsVarlen) {
      get<0>(problem_shape).cumulative_length = device_cumulative_seqlen_q.get();
      get<1>(problem_shape).cumulative_length = device_cumulative_seqlen_kv.get();
    }

    return problem_shape;
  }

  ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {

    ProblemShapeType problem_shape = initialize(options);

    typename Operation::Arguments arguments{
      problem_shape,
      { block_Q.get(), stride_Q,
        block_K.get(), stride_K,
        block_V.get(), stride_V },
      { block_O.get(), stride_O,
      block_LSE.get(), stride_LSE },
      hw_info
    };

    Operation op;

    ExampleResult example_result;

    example_result.smem_size = Operation::Kernel::SharedStorageSize;

    size_t workspace_size = 0;
    workspace_size = Operation::get_workspace_size(arguments);
    DeviceAllocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = cutlass::Status::kSuccess;
    status = op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    status = op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    // Run
    status = op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops;
    if (kIsVarlen) {
      flops = 0.0;
      for (int i = 0; i < size<3,1>(problem_shape); i++) {
        flops += (cumulative_seqlen_q[i+1] - cumulative_seqlen_q[i])
               * 1.0
               * (cumulative_seqlen_kv[i+1] - cumulative_seqlen_kv[i]);
      }
    }
    else {
      flops = 1.0;
      flops *= static_cast<double>(size<0>(problem_shape));
      flops *= static_cast<double>(size<1>(problem_shape));
      flops *= static_cast<double>(size<3,1>(problem_shape));
    }
    flops *= 4.0 * (std::is_same_v<ActiveMask, CausalMask> ? 0.5 : 1.0);
    flops *= static_cast<double>(size<2>(problem_shape));
    flops *= static_cast<double>(size<3,0>(problem_shape));
    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_tc_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_shape);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, ExampleResult result, bool verbose) {
  std::ios fmt(nullptr);
  fmt.copyfmt(std::cout);
  std::cout << (result.passed ? (result.verified ? " [OK]  " : " [--] ") : "[FAIL] ");
  std::cout << std::setw(32) << std::left << description;
  std::cout.copyfmt(fmt);
  std::cout << " : " << result.tflops_tc_s << " TFLOPS/s" << std::endl;
  if (verbose) {
    std::cout << "       t=" << result.runtime_ms << "ms, "
        "smem=" << result.smem_size << "b" << std::endl;
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Mask>
void run_fwd_128(Mask fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    if ((! options.kernel_filter.empty()) && (! std::regex_search(name, std::basic_regex(options.kernel_filter)))) {
        return;
    }
    if (options.varlen) {
      FwdRunner<true, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
      auto result = runner.run(options, hw_info);
      print_result(name, result, options.verbose);
    }
    else 
    {
      FwdRunner<false, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
      auto result = runner.run(options, hw_info);
      print_result(name, result, options.verbose);
    }
  };

  using HeadDim = _128;

  // Persistent Tile Scheduler
  run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 persistent", Option<Tag::kIsPersistent, true_type>{});
  // Individual Tile Scheduler
  run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 individual", Option<Tag::kIsPersistent, false_type>{});
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Mask>
void run_fwd_64(Mask fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    if ((! options.kernel_filter.empty()) && (! std::regex_search(name, std::basic_regex(options.kernel_filter)))) {
        return;
    }
    if (options.varlen) {
      FwdRunner<true, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
      auto result = runner.run(options, hw_info);
      print_result(name, result, options.verbose);
    }
    else 
    {
      FwdRunner<false, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
      auto result = runner.run(options, hw_info);
      print_result(name, result, options.verbose);
    }
  };

  using HeadDim = _64;

  // Persistent Tile Scheduler
  run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 persistent", Option<Tag::kIsPersistent, true_type>{});
  // Individual Tile Scheduler
  run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 individual", Option<Tag::kIsPersistent, false_type>{});
}


///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Mask>
void run_fwd_32(Mask fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    if (options.varlen) {
      FwdRunner<true, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
      auto result = runner.run(options, hw_info);
      print_result(name, result, options.verbose);
    }
    else {
      FwdRunner<false, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
      auto result = runner.run(options, hw_info);
      print_result(name, result, options.verbose);
    }
  };

  using HeadDim = _32;

#ifdef FP8
  // Persistent Tile Scheduler
  run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 persistent", Option<Tag::kIsPersistent, true_type>{});
  // Individual Tile Scheduler
  run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 individual", Option<Tag::kIsPersistent, false_type>{});
#endif
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_single(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major != 10) {
    std::cout
      << "This example requires a GPU of NVIDIA's Blackwell Architecture "
      << "(compute capability major 10) and CUDA 12.8 or greater.\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  if (options.sm_count == 0) {
    hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  }
  else {
    hw_info.sm_count = options.sm_count;
  }

  std::cout << "###### B " << options.b << " H " << options.h << " H_K " << options.h_k << " Q " << options.q << " K " << options.k << " D " << options.d << " ";
  std::cout << "Forward" << " " << (options.causal ? "Causal" : (options.residual ? "Residual" : "None")) << " ";
  std::cout << "#SM " << hw_info.sm_count << std::endl;

  auto with_mask = [&](auto fn) {
    if (options.causal) {
      fn(CausalMask{});
    }
    else if (options.residual) {
      fn(ResidualMask{});
    }
    else {
      fn(NoMask{});
    }
  };

  with_mask([&](auto fusion) {
    if (options.d <= 32) {
      run_fwd_32(fusion, options, hw_info);
    }
    else if (options.d <= 64) {
      run_fwd_64(fusion, options, hw_info);
    }
    else if (options.d <= 128) {
      run_fwd_128(fusion, options, hw_info);
    }
    else {
      std::cout << "No kernel instantiated for d=" << options.d << std::endl;
    }
  });
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  std::vector<std::string> full_arguments(args, args + argc);

  int result = 0;

  bool recursed = false;
  for (size_t i = 1; i < full_arguments.size(); i++) {
    if (full_arguments[i].find(',') != std::string::npos) {
      auto arg = full_arguments[i];
      size_t eq_pos = arg.find('=');
      std::string prefix = eq_pos == std::string::npos ? "" : arg.substr(0, eq_pos+1);
      std::string rest = eq_pos == std::string::npos ? arg : arg.substr(eq_pos+1);
      for (;;) {
        size_t comma_pos = rest.find(',');
        std::string current = rest.substr(0, comma_pos);
        full_arguments[i] = prefix + current;
        std::vector<const char*> next_args;
        for (auto& elem : full_arguments) { next_args.push_back(elem.data()); }
        main(argc, next_args.data());
        if (comma_pos == std::string::npos) break;
        rest = rest.substr(comma_pos+1);
      }
      recursed = true;
      break;
    }
  }

  if (! recursed) {
    main_single(argc, args);
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
