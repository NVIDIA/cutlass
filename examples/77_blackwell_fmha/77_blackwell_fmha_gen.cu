#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Example implementation of fused multi-head attention for the NVIDIA Blackwell SM100
    architecture using CUTLASS 3.

    MQA/GQA
    -------

    The head dimension can be represented as a tuple, where the K/V strides in the
    first dimension is zero. This has the effect of MQA or GQA.
    * MHA is (head_size:head_stride).
    * MQA is (head_size:head_stride) in Q and (head_size:_0) in K and V.
    * GQA is (grouped_heads,heads_kv):(head_stride,grouped_heads*head_stride) in Q
      and (grouped_heads,heads_kv):(0,head_stride) in K and V

    Example usage:
      $ ./examples/77_blackell_fmha/77_blackell_fmha_gen_fp8 \
            --b=2048 --h=2048 --d=2048 --k=2048
*/

#define DSHOW(x) print(#x ": "); print(x); print("\n");
#define DSHOWT(x) print(#x ": "); print_tensor(x); print("\n");

#include <iostream>
#include <random>
#include <regex>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/kernel_hardware_info.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "reference/fmha_fwd_gen_reference.hpp"
#include "reference/reference_abs_error.hpp"

#include "device/fmha.hpp"
#include "collective/fmha_fusion.hpp"
#include "collective/sm100_fmha_gen_mainloop_warpspecialized.hpp"
#include "collective/sm100_fmha_gen_epilogue_warpspecialized.hpp"
#include "kernel/sm100_fmha_gen_kernel_warpspecialized.hpp"
#include "kernel/fmha_tile_scheduler.hpp"

///////////////////////////////////////////////////////////////////////////////////////////////////

using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

enum class InitStyle {
  kZero, kOne, kLinearStride128, kLinearStride1, kRandom, kNone
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help = false;
  bool error = false;

  int b = 1;
  int h = 1;
  int h_k = 1;
  int k = 512;
  int d = 128;
  int iterations = 3;
  bool verify = false;
  bool verbose = false;
  bool remap = false;
  bool varlen = false;
  bool cache_only = false;

  int sm_count = 0;

  std::string kernel_filter;
  bool clear_cache = false;

  InitStyle init_style_q = InitStyle::kRandom;
  InitStyle init_style_cache_k = InitStyle::kRandom;
  InitStyle init_style_cache_v = InitStyle::kRandom;
  InitStyle init_style_new_k = InitStyle::kRandom;
  InitStyle init_style_new_v = InitStyle::kRandom;

  static void get_init_style_argument(cutlass::CommandLine& cmd, const char* name, InitStyle& dst, InitStyle const& src) {
    std::string s;
    cmd.get_cmd_line_argument(name, s, s);
    if (s.empty()) {
      dst = src;
    }
    else {
      if (s == "r") {
        dst = InitStyle::kRandom;
      }
      else if (s == "0") {
        dst = InitStyle::kZero;
      }
      else if (s == "1") {
        dst = InitStyle::kOne;
      }
      else if (s == "d") {
        dst = InitStyle::kLinearStride1;
      }
      else if (s == "s") {
        dst = InitStyle::kLinearStride128;
      }
      else if (s == "n") {
        dst = InitStyle::kNone;
      }
      else {
        std::cout << "Error: " << s << " is not a valid input type.\n";
        std::exit(-1);
      }
    }
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("d", d, defaults.d);
    cmd.get_cmd_line_argument("h", h, -1);
    if (h == -1) h = 2048 / d;

    cmd.get_cmd_line_argument("h_k", h_k, -1);
    if (h_k == -1) h_k = h;

    cmd.get_cmd_line_argument("k", k, defaults.k);

    cmd.get_cmd_line_argument("b", b, -1);
    if (b == -1) b = 16384 / k;
    if (b == 0) b = 1;

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
    verify = cmd.check_cmd_line_flag("verify");
    verbose = cmd.check_cmd_line_flag("verbose");
    varlen = cmd.check_cmd_line_flag("varlen");
    remap = cmd.check_cmd_line_flag("remap");
    cache_only = cmd.check_cmd_line_flag("cache-only");
    cmd.get_cmd_line_argument("sm-count", sm_count, defaults.sm_count);

    get_init_style_argument(cmd, "init-style", init_style_q, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style", init_style_cache_k, defaults.init_style_cache_k);
    get_init_style_argument(cmd, "init-style", init_style_cache_v, defaults.init_style_cache_v);
    get_init_style_argument(cmd, "init-style", init_style_new_k, defaults.init_style_new_k);
    get_init_style_argument(cmd, "init-style", init_style_new_v, defaults.init_style_new_v);
    get_init_style_argument(cmd, "init-style-q", init_style_q, init_style_q);
    get_init_style_argument(cmd, "init-style-cache-k", init_style_cache_k, init_style_cache_k);
    get_init_style_argument(cmd, "init-style-cache-v", init_style_cache_v, init_style_cache_v);
    get_init_style_argument(cmd, "init-style-new-k", init_style_new_k, init_style_new_k);
    get_init_style_argument(cmd, "init-style-new-v", init_style_new_v, init_style_new_v);

    clear_cache = cmd.check_cmd_line_flag("clear-cache");

    cmd.get_cmd_line_argument("kernel-filter", kernel_filter, defaults.kernel_filter);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "77_blackwell_fmha_gen\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  fused multi-head attention forward-pass gen-phase kernels targeting NVIDIA's Blackwell architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the B extent\n"
      << "  --h=<int>                   Sets the H extent\n"
      << "  --h_k=<int>                 Sets the H_K/V extent (for GQA/MQA)\n"
      << "  --k=<int>                   Sets the K extent (sampled around this length)\n"
      << "  --d=<int>                   Sets the D extentn"
      << "  --iterations=<int>          Benchmarking iterations\n"
      << "  --verify                    Verify results\n"
      << "  --verbose                   Print smem and execution time per kernel\n"
      << "  --remap                     Enables batch index remapping\n"
      << "  --cache-only                Only use data from KV cache, no reading or inserting new entry\n"
      << "  --varlen                    Varies sequence length between cache entries\n"
      << "  --sm-count                  Sets SM count rather than querying it\n"
      << "  --clear-cache               Clears the cache before benchmarking runs\n"
      << " --kernel-filter=<filter>     Sets regexp to match kernel against\n"
      << "\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
void initialize_block(
    DeviceAllocation<Element>& block,
    uint64_t seed=2023, InitStyle init_style = InitStyle::kRandom) {

  switch (init_style) {
    case InitStyle::kZero: {
      cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, (Element) 0, (Element) 0);
      break;
    }
    case InitStyle::kOne: {
      cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, (Element) 1, (Element) 1);
      break;
    }
    case InitStyle::kRandom: {
      cutlass::reference::device::BlockFillRandomGaussian(
        block.get(), block.size(), seed, (Element) 0, (Element) 1);
      break;
    }
    case InitStyle::kLinearStride1: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (j % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kLinearStride128: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (i % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kNone: {
      break;
    }

  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleResult {
  bool supported = false;
  bool passed = false;
  bool verified = false;
  float runtime_ms = 0;
  double tflops_tc_s = 0;
  double tops_exp2_s = 0;
  double tbytes_s = 0;
  size_t smem_size = 0;
};

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ClearCache {
  const int size = 1024 * 1024 * 1024 / 4;
  DeviceAllocation<float> data;
  bool active = false;

  ClearCache() = default;

  void set_active(bool the_active) {
    active = the_active;
    if (active) {
      data.reset(size);
    }
    else {
      data.reset(0);
    }
  }

  void operator ()() {
    if (active) {
      initialize_block(data, 0x49314, InitStyle::kRandom);
    }
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

enum class KernelType {
  UMMA_P, UMMA_I
};

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

template<KernelType kKernelType, class TileShape, class ThreadShape>
struct ExampleRunner {

  using Element = cutlass::float_e5m2_t;
  using ElementAcc = float;
  using ElementOut = cutlass::half_t;

  using ProblemShape = Shape<_1, int, int, Shape<Shape<int, int>, int>>;

  using StrideQ = Stride<_0, _1, Stride<Stride<int, int>, int>>;
  using StrideNewK = Stride<_0, _1, Stride<Stride<_0, int>, int>>;
  using StrideCacheK = Stride<int, _1, Stride<Stride<_0, int>, int>>;
  using StrideNewV = StrideNewK;
  using StrideCacheV = StrideCacheK;
  using StrideO = StrideQ;

  using Kernel = 
    cutlass::fmha::kernel::Sm100FmhaGenKernelWarpspecialized<
      ProblemShape,
      cutlass::fmha::collective::Sm100FmhaGenMainloopWarpspecialized<
        Element, ElementAcc, ElementAcc, ElementOut,
        TileShape,
        StrideQ, StrideNewK, StrideNewV,
        StrideCacheK, StrideCacheV, StrideO
      >,
      cutlass::fmha::collective::Sm100FmhaGenEpilogueWarpspecialized<ElementOut, StrideO>,
      std::conditional_t<kKernelType == KernelType::UMMA_P,
        cutlass::fmha::kernel::PersistentTileScheduler,
        cutlass::fmha::kernel::IndividualTileScheduler
      >
    >;
  
  using Operation = cutlass::fmha::device::FMHA<Kernel>;

  StrideQ stride_q;
  StrideNewK stride_new_k;
  StrideNewV stride_new_v;
  StrideCacheK stride_cache_k;
  StrideCacheV stride_cache_v;
  StrideO stride_o;
  uint64_t seed = 0;

  std::vector<int> seqlen_kv;

  DeviceAllocation<int> block_seqlen_kv;
  DeviceAllocation<int> block_cache_batch_idx;
  DeviceAllocation<Element> block_q;
  DeviceAllocation<Element> block_new_k;
  DeviceAllocation<Element> block_new_v;
  DeviceAllocation<Element> block_cache_k;
  DeviceAllocation<Element> block_cache_v;
  DeviceAllocation<ElementOut> block_o;

  DeviceAllocation<Element> block_ref_cache_k;
  DeviceAllocation<Element> block_ref_cache_v;
  DeviceAllocation<ElementOut> block_ref_o;

  ClearCache clear_cache;

  bool verify(const ProblemShape& problem_shape) {

    Tensor mQ = make_tensor(make_gmem_ptr(block_q.get()), select<0,2,3>(problem_shape), stride_q);
    Tensor mNewK = make_tensor(make_gmem_ptr(block_new_k.get()), select<0,2,3>(problem_shape), stride_new_k);
    Tensor mNewV = make_tensor(make_gmem_ptr(block_new_v.get()), select<0,2,3>(problem_shape), stride_new_v);
    Tensor mCacheK = make_tensor(make_gmem_ptr(block_ref_cache_k.get()), select<1,2,3>(problem_shape), stride_cache_k);
    Tensor mCacheV = make_tensor(make_gmem_ptr(block_ref_cache_v.get()), select<1,2,3>(problem_shape), stride_cache_v);
    Tensor mO = make_tensor(make_gmem_ptr(block_ref_o.get()), select<0,2,3>(problem_shape), stride_o);

    fmha_fwd_gen_reference<ElementAcc>(
        problem_shape, block_seqlen_kv.get(), block_cache_batch_idx.get(),
        mQ, mNewK, mNewV, mCacheK, mCacheV, mO);
    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    const double kMaxDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-2;
    const double kMeanDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-3;

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_o, block_ref_o, max_diff, mean_diff);
    bool passed_O = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_O) {
      std::cerr << "failed O: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_cache_k, block_ref_cache_k, max_diff, mean_diff);
    bool passed_K = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if ( ! passed_K) {
      std::cerr << "failed Cache K: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_cache_v, block_ref_cache_v, max_diff, mean_diff);
    bool passed_V = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if ( ! passed_V) {
      std::cerr << "failed Cache V: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    return passed_O && passed_K && passed_V;
  }

  ProblemShape initialize(const Options& options) {

    clear_cache.set_active(options.clear_cache);

    std::vector<int> cache_batch_idx;

    // set up stides and sizes
    if (options.remap) {
      for (int i = 0; i < options.b; i++) {
        cache_batch_idx.push_back(i);
      }
      std::mt19937 rng(0x202305291305ull);
      std::shuffle(cache_batch_idx.begin(), cache_batch_idx.end(), rng);
    }

    seqlen_kv = std::vector<int>(options.b, options.k);
    if (options.varlen) {
      std::mt19937 rng(0x202305151552ull);
      std::normal_distribution<double> dist_kv(options.k, options.k / 2);

      auto generate_positive_int = [](auto& dist, auto& gen) {
        int result = 0;
        do {
          result = static_cast<int>(dist(gen));
        } while (result <= 0);
        return result;
      };

      for (int i = 0; i < options.b; i++) {
        seqlen_kv[i] = generate_positive_int(dist_kv, rng);
      }
    }

    int max_seqlen_kv = 0;
    for (auto e : seqlen_kv) {
      max_seqlen_kv = std::max(e, max_seqlen_kv);
    }

    ProblemShape result = make_shape(_1{}, max_seqlen_kv + 1, options.d, make_shape(make_shape(options.h / options.h_k, options.h_k), options.b));

    stride_q = make_stride(_0{}, _1{}, make_stride(make_stride(options.d, options.d * size<3,0,0>(result)), options.d * size<3,0>(result)));
    stride_new_k = make_stride(_0{}, _1{}, make_stride(make_stride(_0{}, options.d), options.d * size<3,0,1>(result)));
    stride_cache_k = make_stride(options.d * size<3,0,1>(result), _1{}, make_stride(make_stride(_0{}, options.d), options.d * size<3,0,1>(result) * get<1>(result)));

    stride_new_v = stride_new_k;
    stride_cache_v = stride_cache_k;
    stride_o = stride_q;

    block_q.reset(options.b * get<2,1>(stride_q));
    if (! options.cache_only) {
      block_new_k.reset(options.b * get<2,1>(stride_new_k));
      block_new_v.reset(options.b * get<2,1>(stride_new_v));
    }
    block_cache_k.reset(options.b * get<2,1>(stride_cache_k));
    block_cache_v.reset(options.b * get<2,1>(stride_cache_v));
    block_o.reset(options.b * get<2,1>(stride_o));

    block_ref_cache_k.reset(options.b * get<2,1>(stride_cache_k));
    block_ref_cache_v.reset(options.b * get<2,1>(stride_cache_v));
    block_ref_o.reset(options.b * get<2,1>(stride_o));
    
    initialize_block(block_q, seed + 2023, options.init_style_q);
    if (! options.cache_only) {
      initialize_block(block_new_k, seed + 2022, options.init_style_new_k);
      initialize_block(block_new_v, seed + 2021, options.init_style_new_v);
    }

    initialize_block(block_cache_k, seed + 2024 - 2025, options.init_style_cache_k);
    initialize_block(block_cache_v, seed + 2025, options.init_style_cache_v);

    block_ref_cache_k.copy_from_device(block_cache_k.get(), block_cache_k.size());
    block_ref_cache_v.copy_from_device(block_cache_v.get(), block_cache_v.size());
    block_seqlen_kv.reset(seqlen_kv.size());
    block_seqlen_kv.copy_from_host(seqlen_kv.data(), seqlen_kv.size());

    if (! cache_batch_idx.empty()) {
      block_cache_batch_idx.reset(cache_batch_idx.size());
      block_cache_batch_idx.copy_from_host(cache_batch_idx.data(), cache_batch_idx.size());
    }

    return result;
  }

  ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    auto problem_shape = initialize(options);

    typename Operation::Arguments arguments{
      problem_shape,
      block_seqlen_kv.get(), block_cache_batch_idx.get(),
      block_q.get(), stride_q,
      block_new_k.get(), stride_new_k,
      block_new_v.get(), stride_new_v,
      block_cache_k.get(), stride_cache_k,
      block_cache_v.get(), stride_cache_v,
      block_o.get(), stride_o,
      hw_info
    };

    Operation op;

    ExampleResult example_result;

    example_result.smem_size = Operation::Kernel::SharedStorageSize;

    size_t workspace_size = 0;
    workspace_size = Operation::get_workspace_size(arguments);
    DeviceAllocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = cutlass::Status::kSuccess;
    status = op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      // std::cerr << "This kernel is not supported. Last CUDA error is: "
      //           << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }
    example_result.supported = true;

    status = op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    // Run
    status = op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    float total_runtime_ms = 0;

    for (int i = 0; i < options.iterations; i++) {

      clear_cache();

      // Record an event at the start of a series of GEMMs
      result = hipEventRecord(events[0]);
      if (result != hipSuccess) {
        std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }

      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }

      // Record an event when the GEMMs are complete
      result = hipEventRecord(events[1]);
      if (result != hipSuccess) {
        std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }

      //
      // Stop profiling loop
      //
  
      // Wait for work on the device to complete.
      result = hipEventSynchronize(events[1]);
      if (result != hipSuccess) {
        std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }

      // Measure elapsed runtime
      float runtime_ms = 0;
      result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
      if (result != hipSuccess) {
        std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }

      result = hipDeviceSynchronize();
      if (result != hipSuccess) {
        std::cerr << "hipDeviceSynchronize() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }

      total_runtime_ms += runtime_ms;

    }

    float runtime_ms = total_runtime_ms / static_cast<float>(options.iterations);

    double bytes;
    bytes = 0.0;
    bytes += double(sizeof(Element) * size<3>(problem_shape));  // Q
    bytes += double(sizeof(ElementOut) * size<3>(problem_shape));  // O
    bytes += 2.0 * double(sizeof(Element) * size<3>(problem_shape) / size<3,0,0>(problem_shape));  // NewK, NewV
    double total_seqlen_kv = 0;
    for (auto e : seqlen_kv) {
      total_seqlen_kv += double(e + 1);
    }
    bytes += 2.0 * double(sizeof(Element) * size<3,0,1>(problem_shape) * total_seqlen_kv);  // CacheK, CacheV
    bytes *= static_cast<double>(size<2>(problem_shape));
    double tbytes_s = bytes * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tbytes_s = tbytes_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_shape);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, ExampleResult result, bool verbose) {
  std::ios fmt(nullptr);
  fmt.copyfmt(std::cout);
  std::cout << (result.supported ? (result.passed ? (result.verified ? " [OK]  " : " [--] ") : "[FAIL] ") : "[NSUP] ");
  std::cout << std::setw(32) << std::left << description;
  std::cout.copyfmt(fmt);
  std::cout << " : " << result.tbytes_s << " TB/s" << std::endl;
  if (verbose) {
    std::cout << "       t=" << result.runtime_ms << "ms, "
        "smem=" << result.smem_size << "b" << std::endl;
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_single(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major < 10) {
    std::cout
      << "This example requires a GPU of NVIDIA's Blackwell Architecture or "
      << "later (compute capability 90 or greater) and CUDA 12.0 or greater.\n";
    return 0;
  }
  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  if (options.sm_count == 0) {
    hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  }
  else {
    hw_info.sm_count = options.sm_count;
  }

  std::cout << "###### B " << options.b << " H " << options.h << " H_K " << options.h_k << " K " << options.k << " D " << options.d << " ";
  std::cout << "Gen" << " " << (options.varlen ? "Variable" : "Uniform") << " " << (options.remap ? "Remap" : "Linear") << " ";
  std::cout << "#SM " << hw_info.sm_count << std::endl;

  using UMMA = true_type;
  using FFMA2 = false_type;
  auto run = [&](const char* name, auto kernel_type, auto tile, auto thr) {
    if ((! options.kernel_filter.empty()) && (! std::regex_search(name, std::basic_regex(options.kernel_filter)))) {
        return;
    }
    ExampleRunner<decltype(kernel_type)::value, decltype(tile), decltype(thr)> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };


  #define RUN(MODE, m, n, k, tm, tn, tk) \
    run( \
      #MODE " " #m "x" #n "x" #k " / " #tm "x" #tn "x" #tk, \
      std::integral_constant<KernelType, KernelType::MODE>{}, Shape<_##m, _##n, _##k>{}, Shape<_##tm, _##tn, _##tk>{} \
    )

  RUN(UMMA_I, 128, 64, 128, 1, 1, 1);
  RUN(UMMA_I, 128, 128, 128, 1, 1, 1);
  RUN(UMMA_I, 128, 256, 128, 1, 1, 1);
  RUN(UMMA_P, 128, 64, 128, 1, 1, 1);
  RUN(UMMA_P, 128, 128, 128, 1, 1, 1);
  RUN(UMMA_P, 128, 256, 128, 1, 1, 1);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  std::vector<std::string> full_arguments(args, args + argc);

  int result = 0;

  bool recursed = false;
  for (size_t i = 1; i < full_arguments.size(); i++) {
    if (full_arguments[i].find(',') != std::string::npos) {
      auto arg = full_arguments[i];
      size_t eq_pos = arg.find('=');
      std::string prefix = eq_pos == std::string::npos ? "" : arg.substr(0, eq_pos+1);
      std::string rest = eq_pos == std::string::npos ? arg : arg.substr(eq_pos+1);
      for (;;) {
        size_t comma_pos = rest.find(',');
        std::string current = rest.substr(0, comma_pos);
        full_arguments[i] = prefix + current;
        std::vector<const char*> next_args;
        for (auto& elem : full_arguments) { next_args.push_back(elem.data()); }
        main(argc, next_args.data());
        if (comma_pos == std::string::npos) break;
        rest = rest.substr(comma_pos+1);
      }
      recursed = true;
      break;
    }
  }

  if (! recursed) {
    main_single(argc, args);
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
