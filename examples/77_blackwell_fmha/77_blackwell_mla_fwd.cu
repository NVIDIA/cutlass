#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include <iostream>
#include <random>
#include <regex>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/kernel_hardware_info.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "reference/fmha_fwd_reference.hpp"
#include "reference/reference_abs_error.hpp"

#include "device/fmha.hpp"
#include "collective/fmha_fusion.hpp"
#include "collective/sm100_fmha_mla_fwd_mainloop_tma_warpspecialized.hpp"
#include "collective/sm100_fmha_fwd_epilogue_tma_warpspecialized.hpp"
#include "kernel/fmha_options.hpp"
#include "kernel/fmha_tile_scheduler.hpp"
#include "kernel/fmha_causal_tile_scheduler.hpp"
#include "kernel/sm100_fmha_fwd_kernel_tma_warpspecialized.hpp"

///////////////////////////////////////////////////////////////////////////////////////////////////

using namespace cute;
using namespace cutlass::fmha::kernel;
using namespace cutlass::fmha::collective;
using namespace cutlass::fmha;

///////////////////////////////////////////////////////////////////////////////////////////////////

enum class InitStyle {
  kOne, kLinearStride128, kLinearStride1, kRandom, kNone
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help = false;
  bool error = false;

  int b = 1;
  int h = 1;
  int h_k = 1;
  int q = 256;
  int k = 256;
  std::vector<int> varlen_q;
  std::vector<int> varlen_k;
  int dl = 128; // headdim latent
  int dr = 64;  // headdim rope
  int warmup_iterations = 1;
  int iterations = 3;
  int tensor_ring_buffers = 1;
  bool verify = false;
  bool verbose = false;

  bool causal = false;
  bool causal_q_begin = true;
  bool residual = false;
  bool varlen = false;
  bool persistent = false;
  int sm_count = 0;
  std::string kernel_filter;

  InitStyle init_style_q = InitStyle::kRandom;
  InitStyle init_style_k = InitStyle::kRandom;
  InitStyle init_style_v = InitStyle::kRandom;

  static void get_init_style_argument(cutlass::CommandLine& cmd, const char* name, InitStyle& dst, InitStyle const& src) {
    std::string s;
    cmd.get_cmd_line_argument(name, s, s);
    if (s.empty()) {
      dst = src;
    }
    else {
      if (s == "r") {
        dst = InitStyle::kRandom;
      }
      else if (s == "1") {
        dst = InitStyle::kOne;
      }
      else if (s == "d") {
        dst = InitStyle::kLinearStride1;
      }
      else if (s == "s") {
        dst = InitStyle::kLinearStride128;
      }
      else if (s == "n") {
        dst = InitStyle::kNone;
      }
      else {
        std::cout << "Error: " << s << " is not a valid input type.\n";
        std::exit(-1);
      }
    }
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("dl", dl, defaults.dl);
    cmd.get_cmd_line_argument("dr", dr, defaults.dr);
    cmd.get_cmd_line_argument("h", h, -1);
    if (h == -1) h = 2048 / dl;

    cmd.get_cmd_line_argument("h_k", h_k, -1);
    if (h_k == -1) h_k = h;

    varlen = cmd.check_cmd_line_flag("varlen");

    cmd.get_cmd_line_argument("q", q, -1);
    cmd.get_cmd_line_argument("k", k, -1);
    cmd.get_cmd_line_argument("b", b, -1);

    std::string varlen_q_str;
    cmd.get_cmd_line_argument("varlen-q", varlen_q_str);
    std::string varlen_k_str;
    cmd.get_cmd_line_argument("varlen-k", varlen_k_str);

    if (varlen && ! varlen_q_str.empty()) {
      varlen_q.clear();
      while (! varlen_q_str.empty()) {
        size_t pos = varlen_q_str.find(':');
        varlen_q.push_back(std::stoi(varlen_q_str.substr(0, pos)));
        if (pos == std::string::npos) {
          break;
        }
        varlen_q_str = varlen_q_str.substr(pos + 1);
      }
      if (b == -1) {
        b = static_cast<int>(varlen_q.size());
      }
      if (b != static_cast<int>(varlen_q.size())) {
        std::cout << "Error: Invalid --varlen-q length\n";
        std::exit(-1);
      }
      int new_q = 0;
      for (auto elem : varlen_q) {
        new_q += elem;
      }
      if (q != -1) {
        std::cout << "Error: Can't provide --q and --varlen-q\n";
        std::exit(-1);
      }
      q = new_q;
    }

    if (varlen && ! varlen_k_str.empty()) {
      varlen_k.clear();
      while (! varlen_k_str.empty()) {
        size_t pos = varlen_k_str.find(':');
        varlen_k.push_back(std::stoi(varlen_k_str.substr(0, pos)));
        if (pos == std::string::npos) {
          break;
        }
        varlen_k_str = varlen_k_str.substr(pos + 1);
      }
      if (b == -1) {
        b = static_cast<int>(varlen_k.size());
      }
      if (b != static_cast<int>(varlen_k.size())) {
        std::cout << " Error: Invalid --varlen-k length\n";
        std::exit(-1);
      }
      int new_k = 0;
      for (auto elem : varlen_k) {
        new_k += elem;
      }
      if (k != -1) {
        std::cout << "Error: Can't provide --k and --varlen-k\n";
        std::exit(-1);
      }
      k = new_k;
    }

    if (q == -1) q = k;
    if (k == -1) k = q;
    if (q == -1 && k == -1) q = k = defaults.q;
    if (b == -1) b = 16384 / k;
    if (b == 0) b = 1;

    cmd.get_cmd_line_argument("warmup_iterations", warmup_iterations, defaults.warmup_iterations);
    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
    cmd.get_cmd_line_argument("tensor_ring_buffers", tensor_ring_buffers, defaults.tensor_ring_buffers);

    verify = cmd.check_cmd_line_flag("verify");
    verbose = cmd.check_cmd_line_flag("verbose");
    persistent = cmd.check_cmd_line_flag("persistent");

    std::string mask;
    cmd.get_cmd_line_argument<std::string>("mask", mask, "");
    std::string causal_type;
    cmd.get_cmd_line_argument<std::string>("causal-type", causal_type, "");
    if (mask == "no" || mask == "") {
      causal = residual = false;
      if (varlen) {
        residual = true;
      }
    }
    else if (mask == "causal") {
      residual = false;
      causal = true;
      if(causal_type == "qend") {
        causal_q_begin = false;
      } else {
        causal_q_begin = true;
      }
    }
    else if (mask == "residual") {
      residual = true;
      causal = false;
    }
    cmd.get_cmd_line_argument("sm-count", sm_count, defaults.sm_count);
    get_init_style_argument(cmd, "init-style", init_style_q, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style", init_style_k, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style", init_style_v, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style-q", init_style_q, init_style_q);
    get_init_style_argument(cmd, "init-style-k", init_style_k, init_style_k);
    get_init_style_argument(cmd, "init-style-v", init_style_v, init_style_v);

    cmd.get_cmd_line_argument("kernel-filter", kernel_filter, defaults.kernel_filter);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "77_blackwell_mla_fwd\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  fused multi-head latent attention forward-passkernels targeting NVIDIA's Blackwell architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the B extent\n"
      << "  --h=<int>                   Sets the H extent\n"
      << "  --h_k=<int>                 Sets the H_K/V extent (for GQA/MQA)\n"
      << "  --q=<int>                   Sets the Q extent\n"
      << "  --k=<int>                   Sets the K extent\n"
      << "  --varlen-q=<int>:<int...>   Sets the variable Q extent per batch (colon separated)\n"
      << "  --varlen-k=<int>:<int...>   Sets the variable K extent per batch (colon separated)\n"
      << "  --dl=<int>                  Sets the D latent extent\n"
      << "  --dr=<int>                  Sets the D rope extent\n"
      << "  --tensor_ring_buffers=<int> Sets the number of tensor ring buffers\n"
      << "  --warmup_iterations=<int>   Sets the warmup iterations\n"
      << "  --iterations=<int>          Benchmarking iterations\n"
      << "  --verify                    Verify results\n"
      << "  --verbose                   Print smem and execution time per kernel\n"
      << "  --mask=<no|residual|causal> Enables masking\n"
      << "  --causal-type=<qbegin|qend> Causal mask type\n"
      << "  --persistent                Enables persistent scheduler\n"
      << "  --varlen                    Enables variable sequence length\n"
      << "                              B*Q and B*K become the total sequence length\n"
      << "                              and are split B-ways, alternatingly +10% and -10%\n"
      << "                              with the last batch sized to make it fit\n"
      << "                              implies at least residual masking for correctness\n"
      << "  --sm-count                  Sets SM count rather than querying it\n"
      << "  --kernel-filter=<filter>    Sets regexp to match kernel against\n"
      << "\n";

    return out;
  }
};


///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
void initialize_block(
    DeviceAllocation<Element>& block,
    uint64_t seed=2023, InitStyle init_style = InitStyle::kRandom) {

  switch (init_style) {
    case InitStyle::kOne: {
      cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, (Element) 1, (Element) 1);
      break;
    }
    case InitStyle::kRandom: {
      cutlass::reference::device::BlockFillRandomGaussian(
        block.get(), block.size(), seed, (Element) 0, (Element) 1);
      break;
    }
    case InitStyle::kLinearStride1: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (j % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kLinearStride128: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (i % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kNone: {
      break;
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleResult {
  bool passed = false;
  bool verified = false;
  float runtime_ms = 0;
  double tflops_tc_s = 0;
  double tops_exp2_s = 0;
  double tbytes_s = 0;
  size_t smem_size = 0;
};

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

template<
  bool kIsMaskTileSchedulerValid,
  bool kIsVarlen,
  class TileShape,
  class DispatchPolicy,
  class ActiveMask,
  class... KernelOptions
>
struct MlaFwdRunner {

#ifdef FP8
  using Element = cutlass::float_e4m3_t;
#else
  using Element = cutlass::half_t;
#endif

  using ElementAccumulatorQK = float;
  using ElementAccumulatorPV = float;
  using ElementOut = cutlass::half_t;

  // Q K (D_latent D_rope) (H B)
  using ProblemShapeRegular = cute::tuple<int, int, cute::tuple<int, int>, cute::tuple<cute::tuple<int, int>, int>>;
  using ProblemShapeVarlen = cute::tuple<VariableLength, VariableLength, cute::tuple<int, int>, cute::tuple<cute::tuple<int, int>, int>>;
  using ProblemShapeType = std::conditional_t<kIsVarlen, ProblemShapeVarlen, ProblemShapeRegular>;
  
  using StrideQ = cute::tuple<int, _1, cute::tuple<cute::tuple<int, int>, int>>;  // Q D (H_G H_R B)
  using StrideK = cute::tuple<int, _1, cute::tuple<cute::tuple<_0, int>, int>>;  // K D (H_G H_R B)
  using StrideV = StrideK;
  using StrideO = StrideQ;
  using StrideLSE = cute::tuple<_1, cute::tuple<cute::tuple<int, int>, int>>;     // Q   (H_G H_R B)

  static constexpr bool kIsPersistent = find_option_t<Tag::kIsPersistent, true_type, KernelOptions...>::value;
  using TileScheduler = std::conditional_t<kIsPersistent, 
                                          std::conditional_t<std::is_same_v<ActiveMask, CausalMask<false>> 
                                                                          || std::is_same_v<ActiveMask, CausalMask<true>>, 
                                                            cutlass::fmha::kernel::CausalPersistentTileScheduler,
                                                            cutlass::fmha::kernel::PersistentTileScheduler>,
                                          std::conditional_t<kIsMaskTileSchedulerValid, 
                                                            cutlass::fmha::kernel::CausalIndividualTileScheduler,
                                                            cutlass::fmha::kernel::IndividualTileScheduler>>;

  static constexpr bool IsOrderLoadEpilogue = kIsPersistent && (sizeof(Element) == sizeof(ElementOut));
  using OrderLoadEpilogue = std::conditional_t<IsOrderLoadEpilogue, true_type, false_type>;

  using Mainloop = 
    cutlass::fmha::collective::Sm100MlaFwdMainloopTmaWarpspecialized<
      Element, ElementAccumulatorQK, ElementAccumulatorPV,
      TileShape, StrideQ, StrideK, StrideV,
      ActiveMask, Shape<_2, _1, _1>, OrderLoadEpilogue
    >;
  using Operation = cutlass::fmha::device::FMHA<
    cutlass::fmha::kernel::Sm100FmhaFwdKernelTmaWarpspecialized<
      ProblemShapeType,
      Mainloop,
      cutlass::fmha::collective::Sm100FmhaFwdEpilogueTmaWarpspecialized<
        ElementOut, ElementAccumulatorPV,
        typename Mainloop::TileShapePV,
        StrideO, StrideLSE, OrderLoadEpilogue
      >,
      TileScheduler,
      cutlass::fmha::kernel::Sm100MlaFwdCtxKernelWarpspecializedSchedule
    >>;

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;
  uint64_t seed = 0;

  struct DeviceBuffer {
    DeviceAllocation<Element> block_Q;
    DeviceAllocation<Element> block_K;
    DeviceAllocation<Element> block_V;
    DeviceAllocation<ElementOut> block_O;
    DeviceAllocation<ElementAccumulatorPV> block_LSE;
    DeviceAllocation<ElementOut> block_ref_O;
    DeviceAllocation<ElementAccumulatorPV> block_ref_LSE;
    DeviceAllocation<int> device_cumulative_seqlen_q;
    DeviceAllocation<int> device_cumulative_seqlen_kv;

    DeviceBuffer() = default;
    DeviceBuffer(const DeviceBuffer&) = delete;
    DeviceBuffer& operator=(const DeviceBuffer&) = delete;

    size_t get_storage_size() const {
      return block_Q.get_storage_size() + block_K.get_storage_size() + block_V.get_storage_size()
          + block_O.get_storage_size() + block_LSE.get_storage_size() + block_ref_O.get_storage_size()
          + block_ref_LSE.get_storage_size() + device_cumulative_seqlen_q.get_storage_size()
          + device_cumulative_seqlen_kv.get_storage_size();
    }
  };

  std::vector<std::unique_ptr<DeviceBuffer>> buffers;

  std::vector<int> cumulative_seqlen_q;
  std::vector<int> cumulative_seqlen_kv;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_shape, DeviceBuffer& buffer) {
    int D_latent_rope = size<2, 0>(problem_shape) + size<2, 1>(problem_shape);
    Tensor mQ = make_tensor(make_gmem_ptr(buffer.block_Q.get()),
      replace<1>(select<0,2,3>(problem_shape), D_latent_rope),
      stride_Q);

    Tensor mK = make_tensor(make_gmem_ptr(buffer.block_K.get()),
      replace<1>(select<1,2,3>(problem_shape), D_latent_rope),
      stride_K);

    Tensor mV = make_tensor(make_gmem_ptr(buffer.block_V.get()),
      replace<1>(select<1,2,3>(problem_shape), get<2, 0>(problem_shape)),
      stride_V);

    Tensor mO = make_tensor(make_gmem_ptr(buffer.block_ref_O.get()),
      replace<1>(select<0,2,3>(problem_shape), get<2, 0>(problem_shape)),
      stride_O);

    Tensor mLSE = make_tensor(make_gmem_ptr(buffer.block_ref_LSE.get()),
      select<0,3>(problem_shape),
      stride_LSE);

    auto [Q, K, D, HB] = problem_shape;

    auto problem_shape_ref = cute::make_tuple(Q, K, D, D, HB);

    fmha_reference(problem_shape_ref, mQ, mK, mV, mO, mLSE, ActiveMask{});

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    const double kMaxDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-2;
    const double kMeanDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-3;

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(buffer.block_O, buffer.block_ref_O, max_diff, mean_diff);

    bool passed_O = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_O) {
      std::cerr << "failed O: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(buffer.block_LSE, buffer.block_ref_LSE, max_diff, mean_diff);

    bool passed_LSE = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if ( ! passed_LSE) {
      std::cerr << "failed LSE: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    return passed_O && passed_LSE;
  }

  template<class ProblemShape>
  auto initialize_varlen(
      const Options& options, const ProblemShape& problem_size,
      const bool kVarlenSame = true) {

    int num_batches = get<3,1>(problem_size);

    // generate Q as --b times
    //    gaussian (--Q, --Q / 2) sampled positive
    //    track cumulative 
    std::mt19937 rng(0x202305151552ull);
    std::normal_distribution<double> dist_q(get<0>(problem_size), get<0>(problem_size) / 2);
    std::normal_distribution<double> dist_kv(get<1>(problem_size), get<1>(problem_size) / 2);
    std::cout << "N: " << num_batches << ", Q: " << get<0>(problem_size) << ", KV: " << get<1>(problem_size) << std::endl;

    auto generate_positive_int = [](auto& dist, auto& gen) {
      int result = 0;
      do {
        result = static_cast<int>(dist(gen));
      } while (result <= 0);
      return result;
    };

    cumulative_seqlen_q = {0};
    cumulative_seqlen_kv = {0};

    int total_seqlen_q = 0;
    int total_seqlen_kv = 0;
    int max_seqlen_q = 0;
    int max_seqlen_kv = 0;

    for (int i = 0; i < num_batches; i++) {
      int seqlen_q = (! options.varlen_q.empty()) ? options.varlen_q.at(i) : 
              kVarlenSame ? get<0>(problem_size) :
              generate_positive_int(dist_q, rng);
      int seqlen_kv = (! options.varlen_k.empty()) ? options.varlen_k.at(i) :
              kVarlenSame ? get<1>(problem_size) :
              generate_positive_int(dist_kv, rng);

      total_seqlen_q += seqlen_q;
      total_seqlen_kv += seqlen_kv;

      max_seqlen_q = std::max(max_seqlen_q, seqlen_q);
      max_seqlen_kv = std::max(max_seqlen_kv, seqlen_kv);

      cumulative_seqlen_q.push_back(cumulative_seqlen_q.back() + seqlen_q);
      cumulative_seqlen_kv.push_back(cumulative_seqlen_kv.back() + seqlen_kv);
    }
    std::cout << "Q max: " << max_seqlen_q << " total: " << total_seqlen_q << " vs even " << num_batches * get<0>(problem_size) << std::endl;
    std::cout << "KV max: " << max_seqlen_kv << " total: " << total_seqlen_kv << " vs even " << num_batches * get<1>(problem_size) << std::endl;

    ProblemShape problem_size_for_init = problem_size;
    get<3,1>(problem_size_for_init) = 1;
    get<0>(problem_size_for_init) = total_seqlen_q;
    get<1>(problem_size_for_init) = total_seqlen_kv;

    ProblemShapeType problem_size_for_launch;

    get<0>(problem_size_for_launch) = VariableLength{max_seqlen_q};
    get<1>(problem_size_for_launch) = VariableLength{max_seqlen_kv};
    get<2>(problem_size_for_launch) = get<2>(problem_size);
    get<3>(problem_size_for_launch) = get<3>(problem_size);

    return cute::make_tuple(problem_size_for_init, problem_size_for_launch);
  }


  /// Initialize operands to be used in the GEMM and reference GEMM

  ProblemShapeType initialize(const Options& options) {
    int h_r = options.h / options.h_k;
    assert(options.h % options.h_k == 0);
    auto problem_shape_in = cute::make_tuple(options.q, options.k, cute::make_tuple(options.dl, options.dr), cute::make_tuple(cute::make_tuple(h_r, options.h_k), options.b));
    
    ProblemShapeType problem_shape;
    decltype(problem_shape_in) problem_size;

    if constexpr (kIsVarlen) {
      auto [problem_shape_init, problem_shape_launch] = initialize_varlen(options, problem_shape_in);
      problem_shape = problem_shape_launch;
      problem_size = problem_shape_init;
    }
    else {
      problem_size = problem_shape_in;
      problem_shape = problem_shape_in;
    }

    int D_latent_rope = size<2, 0>(problem_shape) + size<2, 1>(problem_shape);
    auto shape_Q = replace<1>(select<0,2,3>(problem_shape), D_latent_rope);
    auto shape_K = replace<1>(select<1,2,3>(problem_shape), D_latent_rope);

    auto shape_O = replace<1>(select<0,2,3>(problem_shape), get<2, 0>(problem_shape));
    auto shape_V = replace<1>(select<1,2,3>(problem_shape), get<2, 0>(problem_shape));

    auto shape_LSE = select<0,3>(problem_size);

    int SQ = size<0>(problem_size);
    int SK = size<1>(problem_size);
    int D = size<2, 0>(problem_size);
    int H  = size<3,0>(problem_size);
    int H_K = size<3,0,1>(problem_size);
    int H_Q = size<3,0,0>(problem_size);
    int B = size<3,1>(problem_size);

    stride_Q = make_stride(H*D_latent_rope , _1{}, make_stride(make_stride(D_latent_rope, H_Q*D_latent_rope), H*D_latent_rope*SQ));
    stride_O = make_stride(H*D , _1{}, make_stride(make_stride(D, H_Q*D), H*D*SQ));
    stride_K = make_stride(H_K*D_latent_rope , _1{}, make_stride(make_stride(_0{}, D_latent_rope), H_K*D_latent_rope*SK));
    stride_V = make_stride(H_K*D , _1{}, make_stride(make_stride(_0{}, D), H_K*D*SK));
    stride_LSE = make_stride(_1{}, make_stride(make_stride(SQ, SQ*H_Q), SQ*H));

    if (kIsVarlen) {
      get<2,1>(stride_Q) = 0;
      get<2,1>(stride_K) = 0;
      get<2,1>(stride_V) = 0;
      get<2,1>(stride_O) = 0;
      get<1,1>(stride_LSE) = 0;
    }

    auto buffer_init_fn = [&](auto& buffer) {
      buffer.block_Q.reset(size(shape_Q), kIsVarlen ? D_latent_rope*SQ*H : 0);
      buffer.block_K.reset(size(shape_K), kIsVarlen ? D_latent_rope*SK*H_K : 0);
      buffer.block_V.reset(size(shape_V), kIsVarlen ? D*SK*H_K : 0);
      buffer.block_O.reset(size(shape_O), kIsVarlen ? D*SQ*H : 0);
      buffer.block_LSE.reset(size(shape_LSE));
      buffer.block_ref_O.reset(size(shape_O), kIsVarlen ? D*SQ*H : 0);
      buffer.block_ref_LSE.reset(size(shape_LSE));

      initialize_block(buffer.block_Q, seed + 2023, options.init_style_q);
      initialize_block(buffer.block_K, seed + 2022, options.init_style_k);
      initialize_block(buffer.block_V, seed + 2021, options.init_style_v);

      if ( ! cumulative_seqlen_q.empty()) {
        buffer.device_cumulative_seqlen_q.reset(cumulative_seqlen_q.size());
        buffer.device_cumulative_seqlen_q.copy_from_host(
          cumulative_seqlen_q.data(), cumulative_seqlen_q.size());
      }
      if ( ! cumulative_seqlen_kv.empty()) {
        buffer.device_cumulative_seqlen_kv.reset(cumulative_seqlen_kv.size());
        buffer.device_cumulative_seqlen_kv.copy_from_host(
          cumulative_seqlen_kv.data(), cumulative_seqlen_kv.size());
      }   
    };

    buffers.push_back(std::make_unique<DeviceBuffer>());
    buffer_init_fn(*buffers.back());

    int tensor_ring_buffers = options.tensor_ring_buffers;
    for (int i = 1; i < tensor_ring_buffers; i++) {
      buffers.push_back(std::make_unique<DeviceBuffer>());
      buffer_init_fn(*buffers.back());
    }

    if constexpr (kIsVarlen) {
      get<0>(problem_shape).cumulative_length = buffers[0]->device_cumulative_seqlen_q.get();
      get<1>(problem_shape).cumulative_length = buffers[0]->device_cumulative_seqlen_kv.get();
    }

    return problem_shape;
  }

  auto get_arguments(const ProblemShapeType& problem_shape, const cutlass::KernelHardwareInfo& hw_info, int buffer_index) {
    auto problem_shape_ = problem_shape;
    if constexpr (kIsVarlen) {
      get<0>(problem_shape_).cumulative_length = buffers[buffer_index]->device_cumulative_seqlen_q.get();
      get<1>(problem_shape_).cumulative_length = buffers[buffer_index]->device_cumulative_seqlen_kv.get();
    }
    typename Operation::Arguments arguments{
      problem_shape_,
      { buffers[buffer_index]->block_Q.get(), stride_Q,
        buffers[buffer_index]->block_K.get(), stride_K,
        buffers[buffer_index]->block_V.get(), stride_V },
      { buffers[buffer_index]->block_O.get(), stride_O,
        buffers[buffer_index]->block_LSE.get(), stride_LSE },
      hw_info
    };
    return arguments;
  }

  ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {

    ProblemShapeType problem_shape = initialize(options);

    int buffer_index = 0;
    typename Operation::Arguments arguments = get_arguments(problem_shape, hw_info, buffer_index);

    Operation op;

    ExampleResult example_result;

    example_result.smem_size = Operation::Kernel::SharedStorageSize;

    size_t workspace_size = 0;
    workspace_size = Operation::get_workspace_size(arguments);
    DeviceAllocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = cutlass::Status::kSuccess;
    status = op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    status = op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    // Run
    for (int i = 0; i < options.warmup_iterations; i++) {
      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
      buffer_index = (buffer_index + 1) % buffers.size();
      arguments = get_arguments(problem_shape, hw_info, buffer_index);
      status = op.update(arguments, workspace.get());
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to update the CUTLASS kernel's parameters. Last CUDA error is: "
                  << std::endl;
        return example_result;
      }
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
      buffer_index = (buffer_index + 1) % buffers.size();
      arguments = get_arguments(problem_shape, hw_info, buffer_index);
      status = op.update(arguments, workspace.get());
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to update the CUTLASS kernel's parameters. Last CUDA error is: "
                  << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops;
    if (kIsVarlen) {
      flops = 0.0;
      for (int i = 0; i < size<3,1>(problem_shape); i++) {
        flops += (cumulative_seqlen_q[i+1] - cumulative_seqlen_q[i])
               * 1.0
               * (cumulative_seqlen_kv[i+1] - cumulative_seqlen_kv[i]);
      }
    }
    else {
      flops = 1.0;
      flops *= static_cast<double>(size<0>(problem_shape));
      flops *= static_cast<double>(size<1>(problem_shape));
      flops *= static_cast<double>(size<3,1>(problem_shape));
    }

    flops *= 2.0 * (std::is_same_v<ActiveMask, CausalMask<false>> ? 0.5 : 1.0);
    flops *= static_cast<double>(size<3,0>(problem_shape));

    double flops0 = flops * static_cast<double>(size<2, 0>(problem_shape) + size<2, 1>(problem_shape));
    double flops1 = flops * static_cast<double>(size<2, 0>(problem_shape));
    flops = flops0 + flops1;

    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_tc_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_shape, *buffers[0]);
      if (passed) example_result.verified = true;
    }

    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_result = 0;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, ExampleResult result, bool verbose) {
  std::ios fmt(nullptr);
  fmt.copyfmt(std::cout);
  std::cout << (result.passed ? (result.verified ? " [OK]  " : " [--] ") : "[FAIL] ");
  if (! result.passed) {
    main_result = -1;
  }
  std::cout << std::setw(32) << std::left << description;
  std::cout.copyfmt(fmt);
  std::cout << " : " << result.tflops_tc_s << " TFLOPS/s" << std::endl;
  if (verbose) {
    std::cout << "       t=" << result.runtime_ms << "ms, "
        "smem=" << result.smem_size << "b" << std::endl;
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Mask>
void run_prefill_mla_fwd(Mask fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    if ((! options.kernel_filter.empty()) && (! std::regex_search(name, std::basic_regex(options.kernel_filter)))) {
        return;
    }
    if (options.varlen) {
      if(options.h % cutlass::fmha::kernel::CausalIndividualTileScheduler::TileH == 0 && (!std::is_same_v<Mask, NoMask>)) {
        MlaFwdRunner<true, true, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
        auto result = runner.run(options, hw_info);
        print_result(name, result, options.verbose);
      } else {
        MlaFwdRunner<false, true, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
        auto result = runner.run(options, hw_info);
        print_result(name, result, options.verbose);
      }
    }
    else 
    {
      if(options.h % cutlass::fmha::kernel::CausalIndividualTileScheduler::TileH == 0 && (!std::is_same_v<Mask, NoMask>)) {
        MlaFwdRunner<true, false, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
        auto result = runner.run(options, hw_info);
        print_result(name, result, options.verbose);
      } else {
        MlaFwdRunner<false, false, decltype(shape), void, Mask, decltype(kernel_options)...> runner;
        auto result = runner.run(options, hw_info);
        print_result(name, result, options.verbose);
      }
    }
  };

  using HeadDimLatent = _128;
  using HeadDim = Shape<HeadDimLatent, _64>;

  if (options.persistent) {
    // Persistent Tile Scheduler
    run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 persistent", Option<Tag::kIsPersistent, true_type>{});
  }
  else {
    // Individual Tile Scheduler
    run(Shape<_256, _128, HeadDim>{}, "tma ws 256x128 acc fp32 individual", Option<Tag::kIsPersistent, false_type>{});
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_single(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major != 10) {
    std::cout
      << "This example requires a GPU of NVIDIA's Blackwell Architecture "
      << "(compute capability major 10) and CUDA 12.8 or greater.\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  if (options.sm_count == 0) {
    hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  }
  else {
    hw_info.sm_count = options.sm_count;
  }

  std::cout << "###### B " << options.b << " H " << options.h << " H_K " << options.h_k << " Q " << options.q << " K " << options.k << " D latent " << options.dl << " D rope " << options.dr << " ";
  std::cout << "MLA Forward" << " " << (options.causal ? "Causal" : (options.residual ? "Residual" : "None")) << " ";
  std::cout << "#SM " << hw_info.sm_count << std::endl;

  auto with_mask = [&](auto fn) {
    if (options.causal) {
      if(options.causal_q_begin) {
        fn(CausalMask{});
      } else {
        fn(CausalMask<false>{});
      }
    }
    else if (options.residual) {
      fn(ResidualMask{});
    }
    else {
      fn(NoMask{});
    }
  };

  with_mask([&](auto fusion) {
    if (options.dl == 128 && options.dr == 64) {
      run_prefill_mla_fwd(fusion, options, hw_info);
    }
    else {
      std::cout << "No kernel instantiated for dl=" << options.dl << " dr=" << options.dr << std::endl;
    }
  });
#endif

  return main_result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  std::vector<std::string> full_arguments(args, args + argc);

  bool recursed = false;
  for (size_t i = 1; i < full_arguments.size(); i++) {
    if (full_arguments[i].find(',') != std::string::npos) {
      auto arg = full_arguments[i];
      size_t eq_pos = arg.find('=');
      std::string prefix = eq_pos == std::string::npos ? "" : arg.substr(0, eq_pos+1);
      std::string rest = eq_pos == std::string::npos ? arg : arg.substr(eq_pos+1);
      for (;;) {
        size_t comma_pos = rest.find(',');
        std::string current = rest.substr(0, comma_pos);
        full_arguments[i] = prefix + current;
        std::vector<const char*> next_args;
        for (auto& elem : full_arguments) { next_args.push_back(elem.data()); }
        main(argc, next_args.data());
        if (comma_pos == std::string::npos) break;
        rest = rest.substr(comma_pos+1);
      }
      recursed = true;
      break;
    }
  }

  if (! recursed) {
    main_single(argc, args);
  }

  return main_result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
