#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Example implementation of fused multi-head attention for Blackwell using CUTLASS 3.

    This example showcases the use of CUTLASS to build backward fused
    multi-head attantion (FMHA) collectives from existing CUTLASS collectives targeting
    the NVIDIA Blackwell architecture.

    Background and motivation
    -------------------------
    CUTLASS is a highly flexible library that provides open-source building blocks
    for tensor core programming for GEMM or GEMM-like problems. Fused multi-head
    attention (FMHA) is a foundational kernel for large language models (LLMs) since it
    makes long sequence lengths feasible from a memory-usage perspective. It also
    improves computational efficiency since it transforms an outer-product-like and
    a matrix-vector-like GEMM into a fused operation with much higher arithmetic
    intensity. For more details, see Dao et al, 2022; Dao, 2023.
    Implementing this kernel in CUTLASS enabled easy customization and high
    performance.

    Introduction
    ------------
    The example targets the NVIDIA Blackwell architecture, and takes advantage of
    5th gen tensor cores and the Tensor Memory Accelerator (TMA), just like
    GEMMs do. It provides a backward pass (often abbreviated
    bwd in the code).
    The code is structured into three layers: The runner (and the reference kernels)
    takes care of initialization, measurement, and testing; the device layer
    orchestrates kernel calls and partitions workspace; and the kernel layer (just
    like the CUTLASS kernel layer.

    Support
    ---------

    We support fp16 and fp8 data types with a head dimension of 128.

    Example usage:
    $ ./examples/77_blackwell_fmha/77_blackwell_fmha_bwd_fp16 \
            --b=2048 --h=2048 --d=2048 --q=2048 --k=2048
*/

#include <iostream>
#include <random>
#include <regex>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/kernel_hardware_info.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "reference/fmha_fwd_reference.hpp"
#include "reference/fmha_bwd_reference.hpp"
#include "reference/reference_abs_error.hpp"

#include "collective/fmha_fusion.hpp"
#include "device/fmha_device_bwd.hpp"

///////////////////////////////////////////////////////////////////////////////////////////////////

using namespace cute;
using namespace cutlass::fmha::kernel;
using namespace cutlass::fmha::collective;
using namespace cutlass::fmha;

///////////////////////////////////////////////////////////////////////////////////////////////////

enum class InitStyle {
  kOne, kZero, kLinearStride128, kLinearStride1, kRandom, kNone
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help = false;
  bool error = false;

  int b = 16;
  int h = 16;
  int h_k = 1;
  int q = 1024;
  int k = 1024;
  int d = 128;
  int iterations = 3;
  bool verify = false;
  bool verbose = false;

  bool causal = false;
  int sm_count = 0;

  std::string kernel_filter;

  InitStyle init_style_q = InitStyle::kRandom;
  InitStyle init_style_k = InitStyle::kRandom;
  InitStyle init_style_v = InitStyle::kRandom;
  InitStyle init_style_do = InitStyle::kRandom;
  bool skip_reference = false;

  static void get_init_style_argument(cutlass::CommandLine& cmd, const char* name, InitStyle& dst, InitStyle const& src) {
    std::string s;
    cmd.get_cmd_line_argument(name, s, s);
    if (s.empty()) {
      dst = src;
    }
    else {
      if (s == "r") {
        dst = InitStyle::kRandom;
      }
      else if (s == "0") {
        dst = InitStyle::kZero;
      }
      else if (s == "1") {
        dst = InitStyle::kOne;
      }
      else if (s == "d") {
        dst = InitStyle::kLinearStride1;
      }
      else if (s == "s") {
        dst = InitStyle::kLinearStride128;
      }
      else if (s == "n") {
        dst = InitStyle::kNone;
      }
      else {
        std::cout << "Error: " << s << " is not a valid input type.\n";
        std::exit(-1);
      }
    }
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("d", d, defaults.d);
    cmd.get_cmd_line_argument("h", h, -1);
    if (h == -1) h = 2048 / d;

    cmd.get_cmd_line_argument("q", q, -1);
    cmd.get_cmd_line_argument("k", k, -1);
    if (q == -1) q = k;
    if (k == -1) k = q;
    if (q == -1 && k == -1) q = k = defaults.q;

    cmd.get_cmd_line_argument("b", b, -1);
    if (b == -1) b = 16384 / k;
    if (b == 0) b = 1;

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
    verify = cmd.check_cmd_line_flag("verify");
    verbose = cmd.check_cmd_line_flag("verbose");
    std::string mask;
    cmd.get_cmd_line_argument<std::string>("mask", mask, "");
    if (mask == "causal") {
      causal = true;
    }
    else {
      causal = defaults.causal;
    }

    skip_reference = cmd.check_cmd_line_flag("skip-reference");
    cmd.get_cmd_line_argument("sm-count", sm_count, defaults.sm_count);

    get_init_style_argument(cmd, "init-style", init_style_q, defaults.init_style_q);
    get_init_style_argument(cmd, "init-style", init_style_k, defaults.init_style_k);
    get_init_style_argument(cmd, "init-style", init_style_v, defaults.init_style_v);
    get_init_style_argument(cmd, "init-style", init_style_do, defaults.init_style_do);
    get_init_style_argument(cmd, "init-style-q", init_style_q, init_style_q);
    get_init_style_argument(cmd, "init-style-k", init_style_k, init_style_k);
    get_init_style_argument(cmd, "init-style-v", init_style_v, init_style_v);
    get_init_style_argument(cmd, "init-style-do", init_style_v, init_style_do);

    cmd.get_cmd_line_argument("kernel-filter", kernel_filter, defaults.kernel_filter);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "77_blackwell_fmha_bwd\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  fused multi-head attention kernels for the backward pass targeting NVIDIA's Blackwell architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the B extent\n"
      << "  --h=<int>                   Sets the H extent\n"
      << "  --q=<int>                   Sets the Q extent\n"
      << "  --k=<int>                   Sets the K extent\n"
      << "  --d=<int>                   Sets the D extentn"
      << "  --iterations=<int>          Benchmarking iterations\n"
      << "  --verify                    Verify results\n"
      << "  --verbose                   Print smem and execution time per kernel\n"
      << "  --mask=<no|causal>          Enables masking\n"
      << "  --sm-count                  Sets SM count rather than querying it\n"
      << "  --kernel-filter=<filter>    Sets regexp to match kernel against\n"
      << "\n";

    return out;
  }
};


///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
void initialize_block(
    DeviceAllocation<Element>& block,
    uint64_t seed=2023, InitStyle init_style = InitStyle::kRandom) {

  switch (init_style) {
    case InitStyle::kOne: {
      cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, (Element) 1, (Element) 1);
      break;
    }
    case InitStyle::kZero: {
      cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, (Element) 0, (Element) 0);
      break;
    }
    case InitStyle::kRandom: {
      cutlass::reference::device::BlockFillRandomGaussian(
        block.get(), block.size(), seed, (Element) 0, (Element) 1);
      break;
    }
    case InitStyle::kLinearStride1: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (j % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kLinearStride128: {
      std::vector<Element> data(block.size());
      for (size_t i = 0; i < block.size() / 128; i ++) {
        for (int j = 0; j < 128; j++) {
          data[j + 128*i] = static_cast<Element>((double) (i % 4));
        }
      }
      block.copy_from_host(data.data(), data.size());
      break;
    }
    case InitStyle::kNone: {
      break;
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleResult {
  bool passed = false;
  bool verified = false;
  float runtime_ms = 0;
  double tflops_tc_s = 0;
  size_t smem_size = 0;
};

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

template<
  class TileShape,
  class DispatchPolicy,
  class ActiveMask,
  class... KernelOptions
>
struct BwdRunner {

#ifdef FP8
  using Element = cutlass::float_e4m3_t;
#else
  using Element = cutlass::half_t;
#endif
  using ElementAccumulator = float;

  // Q K D (H B)
  using ProblemShapeType = cute::tuple<int, int, int, cute::tuple<int, int>>;

  using Operation = cutlass::fmha::device::Sm100FmhaBwd<Element, ElementAccumulator, TileShape, ActiveMask>;
  
  using TensorStride = Stride<int, _1, Stride<int, int>>; // Seq D (H B)
  using StrideQ = TensorStride;
  using StrideK = TensorStride;
  using StrideV = TensorStride;
  using StrideO = TensorStride;
  using StrideLSE = Stride<_1, Stride<int, int>>; // Seq (H B)

  // Backwards specific
  using StrideDQ = TensorStride;
  using StrideDK = TensorStride;
  using StrideDV = TensorStride;
  using StrideDO = TensorStride;

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;

  StrideDQ stride_dQ;
  StrideDK stride_dK;
  StrideDV stride_dV;
  StrideDO stride_dO;

  uint64_t seed = 0;

  DeviceAllocation<Element> block_Q;
  DeviceAllocation<Element> block_K;
  DeviceAllocation<Element> block_V;
  DeviceAllocation<Element> block_O;
  DeviceAllocation<ElementAccumulator> block_LSE;

  DeviceAllocation<Element> block_dQ;
  DeviceAllocation<Element> block_dK;
  DeviceAllocation<Element> block_dV;
  DeviceAllocation<Element> block_dO;

  DeviceAllocation<Element> block_ref_dQ;
  DeviceAllocation<Element> block_ref_dK;
  DeviceAllocation<Element> block_ref_dV;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_shape) {
    auto [Q, K, D, HB] = problem_shape;
    auto [H, B] = HB;

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      select<0,2,3>(problem_shape),
      stride_Q);

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      select<1,2,3>(problem_shape),
      stride_K);

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      select<1,2,3>(problem_shape),
      stride_V);

    Tensor mO = make_tensor(make_gmem_ptr(block_O.get()),
      select<0,2,3>(problem_shape),
      stride_O);

    // keep going here! (this might be better in cursor)

    Tensor mLSE = make_tensor(make_gmem_ptr(block_LSE.get()),
      select<0,3>(problem_shape),
      stride_LSE);

    Tensor mDQ = make_tensor(make_gmem_ptr(block_ref_dQ.get()),
      select<0,2,3>(problem_shape),
      stride_dQ);

    Tensor mDK = make_tensor(make_gmem_ptr(block_ref_dK.get()),
      select<1,2,3>(problem_shape),
      stride_dK);

    Tensor mDV = make_tensor(make_gmem_ptr(block_ref_dV.get()),
      select<1,2,3>(problem_shape),
      stride_dV);

    Tensor mDO = make_tensor(make_gmem_ptr(block_dO.get()),
      select<0,2,3>(problem_shape),
      stride_dO);

    fmha_bwd_reference(problem_shape, mQ, mK, mV, mO, mLSE, mDO, mDQ, mDK, mDV, ActiveMask{});

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    const double kMaxDiffThresh = sizeof(Element) == 1 ? 1e-0 : 1e-2;
    const double kMeanDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-3;

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_dQ, block_ref_dQ, max_diff, mean_diff);

    bool passed_dQ = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_dQ) {
      std::cerr << "failed dQ: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_dK, block_ref_dK, max_diff, mean_diff);

    bool passed_dK = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_dK) {
      std::cerr << "failed dK: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_dV, block_ref_dV, max_diff, mean_diff);

    bool passed_dV = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_dV) {
      std::cerr << "failed dV: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    return passed_dQ && passed_dK && passed_dV;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_shape, Options const& options) {
    auto [Q, K, D, HB] = problem_shape;
    auto [H, B] = HB;
    D = cutlass::round_up(D, 8);  // Alignment
    Q = cutlass::round_up(Q, 8);  // Alignment

    auto shape_QO = select<0,2,3>(problem_shape);
    auto shape_KV = select<1,2,3>(problem_shape);
    auto shape_LSE = select<0,3>(problem_shape);

    stride_Q = make_stride(D, _1{}, make_stride(D*Q, D*Q*H));
    stride_K = make_stride(D, _1{}, make_stride(D*K, D*K*H));
    stride_V = stride_K;
    stride_O = stride_Q;
    stride_LSE = make_stride(_1{}, make_stride(Q, Q*H));

    stride_dQ = stride_Q;
    stride_dK = stride_K;
    stride_dV = stride_V;
    stride_dO = stride_O;

    auto lsize = [](auto shape) {
      return size(make_shape(1ull, shape));
    };

    block_Q.reset(lsize(shape_QO));
    block_K.reset(lsize(shape_KV));
    block_V.reset(lsize(shape_KV));
    block_O.reset(lsize(shape_QO));
    block_LSE.reset(lsize(shape_LSE));

    block_dQ.reset(lsize(shape_QO));
    block_dK.reset(lsize(shape_KV));
    block_dV.reset(lsize(shape_KV));
    block_dO.reset(lsize(shape_QO));

    block_ref_dQ.reset(lsize(shape_QO));
    block_ref_dK.reset(lsize(shape_KV));
    block_ref_dV.reset(lsize(shape_KV));

    initialize_block(block_Q, seed + 2023, options.init_style_q);
    initialize_block(block_K, seed + 2022, options.init_style_k);
    initialize_block(block_V, seed + 2021, options.init_style_v);
    initialize_block(block_dO, seed + 2020, options.init_style_do);

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      select<0,2,3>(problem_shape),
      stride_Q);

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      select<1,2,3>(problem_shape),
      stride_K);

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      select<1,2,3>(problem_shape),
      stride_V);

    Tensor mO = make_tensor(make_gmem_ptr(block_O.get()),
      select<0,2,3>(problem_shape),
      stride_O);

    Tensor mLSE = make_tensor(make_gmem_ptr(block_LSE.get()),
      select<0,3>(problem_shape),
      stride_LSE);

    if (! options.skip_reference) {
      fmha_reference(problem_shape, mQ, mK, mV, mO, mLSE, ActiveMask{});
    }
  }

  ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    auto problem_shape = make_shape(options.q, options.k, options.d, make_shape(options.h, options.b));

    initialize(problem_shape, options);

    ElementAccumulator softmax_scale = 1.0f / sqrtf(options.d);

    typename Operation::Arguments arguments{
      problem_shape,
      block_Q.get(), stride_Q,
      block_K.get(), stride_K,
      block_V.get(), stride_V,
      block_O.get(), stride_O,
      block_LSE.get(), stride_LSE,
      block_dO.get(), stride_dO,
      block_dQ.get(), stride_dQ,
      block_dK.get(), stride_dK,
      block_dV.get(), stride_dV,
      softmax_scale,
      hw_info
    };

    Operation op;

    ExampleResult example_result;

    example_result.smem_size = Operation::Kernel::SharedStorageSize;

    size_t workspace_size = 0;
    workspace_size = Operation::get_workspace_size(arguments);
    DeviceAllocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = cutlass::Status::kSuccess;
    status = op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    status = op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    // Run
    status = op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops = 10.0 * (std::is_same_v<ActiveMask, CausalMask> ? 0.5 : 1.0);
    flops *= static_cast<double>(get<0>(problem_shape));
    flops *= static_cast<double>(get<1>(problem_shape));
    flops *= static_cast<double>(get<2>(problem_shape));
    flops *= static_cast<double>(get<3,0>(problem_shape));
    flops *= static_cast<double>(get<3,1>(problem_shape));
    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_tc_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_shape);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, ExampleResult result, bool verbose) {
  std::ios fmt(nullptr);
  fmt.copyfmt(std::cout);
  std::cout << (result.passed ? (result.verified ? " [OK]  " : " [--] ") : "[FAIL] ");
  std::cout << std::setw(32) << std::left << description;
  std::cout.copyfmt(fmt);
  std::cout << " : " << result.tflops_tc_s << " TFLOPS/s" << std::endl;
  if (verbose) {
    std::cout << "       t=" << result.runtime_ms << "ms, "
        "smem=" << result.smem_size << "b" << std::endl;
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct KernelCoop {};

//////////////////////////////////////////////////////////////////////////////////////////////////

template<class Mask>
void run_bwd_64(Mask fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel), Mask, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _64;

  run(Shape<_128, _128, HeadDim>{}, KernelCoop{}, "tma");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Mask>
void run_bwd_128(Mask fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel), Mask, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _128;

  run(Shape<_128, _128, HeadDim>{}, KernelCoop{}, "tma");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_single(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major != 10) {
    std::cout
      << "This example requires a GPU of NVIDIA's Blackwell Architecture "
      << "(compute capability 100a) and CUDA 12.8 or greater.\n";
    return 0;
  }
  
  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  if (options.sm_count == 0) {
    hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);
  }
  else {
    hw_info.sm_count = options.sm_count;
  }

  std::cout << "###### B " << options.b << " H " << options.h << " Q " << options.q << " K " << options.k << " D " << options.d << " ";
  std::cout << "Backward" << " " << (options.causal ? "Causal" : "Full") << " ";
  std::cout << "#SM " << hw_info.sm_count << std::endl;

  auto with_causal = [&](auto fn) {
    if (options.causal) {
      fn(CausalMask{});
    }
    else {
      fn(NoMask{});
    }
  };

  with_causal([&](auto fusion) {
    if (options.d <= 64) {
      run_bwd_64(fusion, options, hw_info);
    }
    else if (options.d <= 128) {
      run_bwd_128(fusion, options, hw_info);
    }
    else {
      std::cout << "No kernel instantiated for d=" << options.d << std::endl;
    }
  });
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  std::vector<std::string> full_arguments(args, args + argc);

  int result = 0;

  bool recursed = false;
  for (size_t i = 1; i < full_arguments.size(); i++) {
    if (full_arguments[i].find(',') != std::string::npos) {
      auto arg = full_arguments[i];
      size_t eq_pos = arg.find('=');
      std::string prefix = eq_pos == std::string::npos ? "" : arg.substr(0, eq_pos+1);
      std::string rest = eq_pos == std::string::npos ? arg : arg.substr(eq_pos+1);
      for (;;) {
        size_t comma_pos = rest.find(',');
        std::string current = rest.substr(0, comma_pos);
        full_arguments[i] = prefix + current;
        std::vector<const char*> next_args;
        for (auto& elem : full_arguments) { next_args.push_back(elem.data()); }
        main(argc, next_args.data());
        if (comma_pos == std::string::npos) break;
        rest = rest.substr(comma_pos+1);
      }
      recursed = true;
      break;
    }
  }

  if (! recursed) {
    main_single(argc, args);
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
