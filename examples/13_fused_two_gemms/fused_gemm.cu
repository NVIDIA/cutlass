/***************************************************************************************************
 * Copyright (c) 2017-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*

This example shows fusing two GEMM mainloops into one kernel. The first GEMM computes relu(alpha*A*B) and 
the second GEMM computes relu(alpha*A*B+beta*C). The performance measuring environment compares against
two unfused GEMM operations, demonstrating a speedup of the fused kernel on the 
NVIDIA Turing GPU architecture.

Problem size:
  GEMM1 (M,N,K): 128*1600, 64, 576
  GEMM2 (M,N,K): 128*1600, 128, 64

Note that GEMM1_N = GEMM2_K

The example requires the number of threadblocks be the same across 2 GEMMs and 
thread_block_tile_N = problem_N so the data required by each layer is threadblock-resident. It 
also requires warp_tile_N = thread_block_tile_N so the data required by each warp is 
register-file-resident.

Performance:
  - fp16 on Tesla T4 @ 1590MHz (non-fused vs. fused): 1.39011 ms vs. 1.26035 ms
  - int8 on Tesla T4 @ 1590MHz (non-fused vs. fused): 0.751759 ms vs. 0.62971 ms
  - fp16 on Quadro RTX 8000 @ 1890MHz (non-fused vs. fused): 0.721144 ms vs. 0.629864 ms
  - int8 on Quadro RTX 8000 @ 1890MHz (non-fused vs. fused): 0.379049 ms vs. 0.324764 ms
  - int8 on GA100 @ 1200MHz (non-fused vs. fused): 0.153795 ms vs. 0.129874 ms

*/

#include "b2b_gemm_f16t_f16n_f16t_tensor_op_f16_sm75.h"
#include "b2b_gemm_s8n_s8t_s8n_tensor_op_s32_sm75.h"
#include "b2b_gemm_s8n_s8t_s8n_tensor_op_s32_sm80.h"

int run() {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!(props.major * 10 + props.minor >= 75)) {
    std::cerr << "Turing Tensor Ops must be run on a machine with compute capability at least 75."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
  run_nonfused_gemm_s8_sm80();
  run_fused_gemm_s8_sm80();
#elif defined(CUTLASS_ARCH_MMA_SM75_SUPPORTED)
  run_nonfused_gemm_f16();
  run_fused_gemm_f16();
  run_nonfused_gemm_s8();
  run_fused_gemm_s8();
#endif

  return 0;
}

int main() {
  // Turing Tensor Core operations exposed with mma.sync are first available in CUDA 10.2.
  //
  // CUTLASS must be compiled with CUDA 10.1 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
    std::cerr << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
  else {
    return run();
  }
}

