/***************************************************************************************************
 * Copyright (c) 2017-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
*/

#include "b2b_gemm_f16t_f16n_f16t_tensor_op_f16_sm75.h"
#include "b2b_gemm_s8n_s8t_s8n_tensor_op_s32_sm75.h"

int run() {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!(props.major * 10 + props.minor >= 75)) {
    std::cerr << "Turing Tensor Ops must be run on a machine with compute capability at least 75."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

#if defined(CUTLASS_ARCH_MMA_SM75_SUPPORTED)
  run_nonfused_gemm_f16();
  run_fused_gemm_f16();
  run_nonfused_gemm_s8();
  run_fused_gemm_s8();
#endif

  return 0;
}

int main() {
  // Turing Tensor Core operations exposed with mma.sync are first available in CUDA 10.2.
  //
  // CUTLASS must be compiled with CUDA 10.1 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
    std::cerr << "Turing Tensor Core operations must be compiled with CUDA 10.2 Toolkit or later." << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
  else {
    return run();
  }
}

