/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief A GEMM example using CUTLASS for the NVIDIA Blackwell SM100 architecture with preferred cluster.
    
    With the introduction of NVIDIA Compute Capability 9.0, the CUDA programming model introduced 
    an optional hierarchy level known as Thread Block Clusters, which consist of multiple Thread Blocks. 
    While the CUDA programming model has supported the specification of cluster shapes at runtime 
    (Dynamic Clusters) since the Hopper architecture, CUTLASS has only provided support for Static 
    Clusters, meaning that cluster shapes must be defined at compile time.  

    Larger cluster shapes can achieve higher TMA multicast but may result in poor SM occupancy due 
    to quantization. For instance, a 2x2 cluster on an 18 SM GPU would only utilize 16 SMs, leaving 
    2 SMs idle.

    Starting with Compute Capability 10.0, the CUDA programming model adds the ability to specify 
    two clusters: preferred cluster and fallback cluster. For brevity, we refer to this as 
    Preferred Clusters. In the previous example, users can now launch an additional 2x1 cluster to 
    utilize the 2 idle SMs.
      
    With CUTLASS 3.8, in addition to Dynamic Clusters, CUTLASS adds support for Preferred Dynamic Cluster,
    the ability for users to specify two clusters shapes at runtime.

    Terminology
      * Static cluster: cluster shape is specified at compile time.
      * Dynamic cluster: cluster shape is specified at runtime and set by the host.
      * Preferred cluster: Kernel can be launched with two cluster shapes (preferred and fallback).

    Preferred and fallback cluster shapes are subject to several constraints.
      * Preferred cluster depth (Z dimension) must be the same as that of fallback cluster.
      * Fallback cluster shape must evenly divide the preferred cluster shape.
      * Preferred cluster shape must evenly divide the kernel launch grid shape.

    This example demonstrates how to use the Dynamic Clusters and Preferred Clusters features in
    CUTLASS 3.x Blackwell SM100 kernels. Users can specify preferred and fallback cluster shapes via GEMM arguments.

    # Example:
    ./73_blackwell_gemm_preferred_cluster" --m=4096 --n=4096 --k=4096 --preferred_cluster_m=4 --preferred_cluster_n=4 --fallback_cluster_m=2 --fallback_cluster_m=1
*/



#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = half_t;                                         // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = half_t;                                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;                                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Kernel functional config
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm100;                           // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag

// MMA and Cluster Tile Shapes
// Shape of the tile computed by tcgen05 MMA, could be across 2 SMs if Cluster Shape % 2 == 0 
using MmaTileShape_MNK = Shape<_256,_128,_64>;                          
// Shape of the threadblocks participating in a tcgen05 MMA. <1, 1, 1> for cta_group = 1, <2, 1, 1> for cta_group = 2
using AtomThrShape_MNK = Shape<_2, _1, _1>;
// Shape of the tile computed by each SM
using PerSmTileShape_MNK = decltype(shape_div(MmaTileShape_MNK{}, AtomThrShape_MNK{}));
// Shape of the cluster set to <int,int,_1> to indicate dynamic cluster shape
using ClusterShape_MNK = Shape<int,int,_1>;
// When dynamic cluster is used, KernelScheduleAuto always selects mainloop dispatch policy that 
// lowers to tcgen05 MMA cta_group = 1 as we don't know if the dynamic cluster M dimension will be a multiple of 2
// To use KernelScheduleAuto, users need to set AtomThrShape_MNK to Shape<1, 1, 1>
using KernelSchedule = cute::conditional_t<cute::size(AtomThrShape_MNK{}) == 2, 
  cutlass::gemm::KernelTmaWarpSpecialized2SmSm100, 
  cutlass::gemm::collective::KernelScheduleAuto>;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    PerSmTileShape_MNK, ClusterShape_MNK,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC, AlignmentC,
    ElementC, LayoutC, AlignmentC,
    cutlass::epilogue::collective::EpilogueScheduleAuto
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    MmaTileShape_MNK, ClusterShape_MNK,
    cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int, int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue,
    void // <--- Default to cluster launch control (CLC) scheduler
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;
  int preferred_cluster_m, preferred_cluster_n, fallback_cluster_m, fallback_cluster_n;
  
  Options():
    help(false),
    m(4096), n(4096), k(4096),
    alpha(1.f), beta(0.f),
    iterations(10),
    preferred_cluster_m(4),
    preferred_cluster_n(4),
    fallback_cluster_m(2),
    fallback_cluster_n(1)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("preferred_cluster_m", preferred_cluster_m, 4);
    cmd.get_cmd_line_argument("preferred_cluster_n", preferred_cluster_n, 4);
    cmd.get_cmd_line_argument("fallback_cluster_m", fallback_cluster_m, 2);
    cmd.get_cmd_line_argument("fallback_cluster_n", fallback_cluster_n, 1);

    if (!validate_cluster_shape()){
      std::cout << "--Invalid cluster shapes" << std::endl;
      help = true;
      return;
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "73_blackwell_gemm_preferred_cluster\n\n"
      << "  Blackwell FP16 GEMM using preferred cluster.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n"
      << "  --preferred_cluster_m=<str> Sets the M extent of preferred cluster shape\n"
      << "  --preferred_cluster_n=<str> Sets the N extent of preferred cluster shape\n"
      << "  --fallback_cluster_m=<str>  Sets the M extent of fallback cluster shape\n"
      << "  --fallback_cluster_n=<str>  Sets the N extent of fallback cluster shape\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out << "Preferred cluster shape cannot be smaller than fallback cluster shape.\n"
      << "Preferred cluster shape must be a multiple of fallback cluster shape.\n\n";
  
    out << "\n\nExamples:\n\n"
      << "$ " << "73_blackwell_gemm_preferred_cluster" << " --m=4096 --n=4096 --k=4096 --preferred_cluster_m=4 --preferred_cluster_n=4 --fallback_cluster_m=2 --fallback_cluster_m=1\n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }

 private:
  /// Validate preferred and fallback cluster shapes
  bool validate_cluster_shape() {
    if (preferred_cluster_m < fallback_cluster_m || preferred_cluster_n < fallback_cluster_n) {
      std::cout << "--Preferred cluster cannot be smaller than fallback cluster" << std::endl;
      return false;
    }

    if (preferred_cluster_m % fallback_cluster_m != 0 || preferred_cluster_n % fallback_cluster_n != 0) {
      std::cout << "--Preferred cluster must be a multiple of fallback cluster" << std::endl;
      return false;
    }
    return true;
  }

};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(cutlass::DeviceAllocation<Element>& block, uint64_t seed=2023) {
  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = Element(2);
    scope_min = Element(0);
  } else if (bits_input <= 8) {
    scope_max = Element(2);
    scope_min = Element(-2);
  } else {
    scope_max = Element(8);
    scope_min = Element(-8);
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, {options.m, options.k, 1});
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, {options.n, options.k, 1});
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, {options.m, options.n, 1});
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, {options.m, options.n, 1});

  block_A.reset(options.m * options.k);
  block_B.reset(options.k * options.n);
  block_C.reset(options.m * options.n);
  block_D.reset(options.m * options.n);
  block_ref_D.reset(options.m * options.n);

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options) {
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, 1},
    {block_A.get(), stride_A, block_B.get(), stride_B},
    {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
  };

  arguments.hw_info.cluster_shape = dim3(options.preferred_cluster_m, options.preferred_cluster_n,1);
  arguments.hw_info.cluster_shape_fallback = dim3(options.fallback_cluster_m, options.fallback_cluster_n,1);
  
  return arguments;
}

bool verify(const Options &options) {
  cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({options.m, options.k}));
  cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({options.k, options.n}));
  cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({options.m, options.n}));
  cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({options.m, options.n}));

  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    {options.m, options.n, options.k},
    ElementAccumulator(options.alpha),
    ref_A,
    ref_B,
    ElementAccumulator(options.beta),
    ref_C,
    ref_D);

  // Wait for kernel to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

  return passed;
}

/// Execute a given example GEMM computation
int run(Options &options) {

  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "GEMM with"
            << " Problem Size: " << options.m << 'x' << options.n << 'x' << options.k 
            << " Preferred Cluster = (" << options.preferred_cluster_m << ", " << options.preferred_cluster_n << ", 1)"
            << " Fallback Cluster = (" << options.fallback_cluster_m << ", " << options.fallback_cluster_n << ", 1)"
            << std::endl;

  std::cout << "--------------------------------------------------------------------------------" << std::endl;

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.8 Toolkit to run this example
  // and must have compute capability at least 100.
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)) {
    std::cerr << "This example requires CUDA 12.8 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));

  if (props.major != 10 || props.minor != 0) {
    std::cerr << "This example requires a GPU of NVIDIA's Blackwell architecture (compute capability 100)." << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //
#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
  run(options);
#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  return 0;
}
