/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**

This example shows how to run CUTLASS's convolution kernels
based on the Implicit GEMM algorithm, that use the Tensor Cores
on an NVIDIA Ampere GPU.

Writing a single high-performance convolution kernel is hard enough,
let alone writing kernels that perform well for multiple problem sizes
and use good software abstractions.
CUTLASS provides simplified abstractions
to compose multiple sections of a convolution kernel.
When used properly, the kernels can reach peak GPU performance.

CUTLASS divides a kernel into hierarchical composable sections
for each level of the GPU hardware hierarchy:
thread, warp, and threadblock.
Each section computes on its own tile shape,
with each higher level's tile shape
being composed from lower-level tile shapes.
Multiple thread tiles (the tile shape each thread computes)
can be used to form warp tiles (the tile shape each warp computes),
and multiple warp tiles can be used to compute threadblock tiles
(the tile shape computed by a threadblock).

In this example, we split variable initialization into two parts.

1. Setting up data properties: describes how tensors are laid out in the memory
   and how the kernel can view them (logical to physical mapping)

2. Setting up computation properties: describes how the above tensors
   will be used to compute the output of convolution

We begin by setting up the data types
of all the input and output elements of a convolution.
A convolution computes
C = alpha * Conv2dFprop(A, B) + beta * C,
so we set up data types for the input tensor A,
weights tensor B, output tensor C,
and the scaling factors alpha and beta.
CUTLASS divides the convolution into two parts:
the "mainloop" that computes X = Conv2dFprop(A, B),
and the "epilogue" that computes C = alpha * X + beta * C.
The epilogue is an element-wise operation on X and C.
In this case, it is a linear combination,
but other epilogues are possible.

In this example, we want

* the scaling factors alpha and beta to be float,

* the elements of A and B to be cutlass::half_t
  (a 16-bit floating-point type),

* the elements of C to be float, and

* intermediate sums to be accumulated in float.

We convey this to the CUTLASS kernel
by setting the following template parameters.

* alpha and beta: ElementComputeEpilogue = float

* Elements of input tensor A: ElementInputA = cutlass::half_t

* Elements of input tensor B: ElementInputB = cutlass::half_t

* Elements of output tensor C: ElementOutput = float

* Accumulation type: ElementAccumulator = float

Next, we describe the layout of the input and output tensors.
We convey this to the CUTLASS kernel
by setting the following template parameters.

* Layout of input tensor A: LayoutInputA = TensorNHWC

* Layout of input tensor B: LayoutInputB = TensorNHWC

* Layout of output tensor C: LayoutOutput = TensorNHWC

After that, we set up rules to compute the epilogue.
The epilogue in this case is a simple linear combination
C = alpha * X + beta * C.
Thus, we set the kernel's template parameter EpilogueOp
to LinearCombination.  LinearCombination itself
has template parameters:

* the element type of the output tensor (ElementOutput),

* the number of elements per vector memory access (8),

* the data type of the accumulator (ElementAccumulator),

* and the data type used to compute the linear combination
  (ElementComputeEpilogue).

We then define the tile shapes
that each level of the computation uses.
We define these as types that encode the tile shapes
as compile-time integer values.
Each shape expresses the dimensions M x N x K.
Here, the letters refer to the dimensions
of a matrix-matrix multiply.

* ThreadblockShape defines the threadblock tile shape
  as 128 x 128 x 64.

* WarpShape defines the warp tile shape as 64 x 64 x 64.

* InstructionShape defines the MMA
  (matrix multiply-accumulate) operation shape
  as 16 x 8 x 16.

These types become template arguments
of the kernel properties type
cutlass::conv::kernel::DefaultConv2dFprop.
The kernel uses these shapes to deduce
the number of threads needed per threadblock,
the required amount of shared memory,
the internal layouts needed to access
shared memory without bank conflicts,
and many other properties that the kernel needs
for good performance.
CUTLASS deduces all these properties automatically,
so that users don't have to.
DefaultConv2dFprop accepts other template parameters
that describe things like the target CUDA SM architecture.

CUTLASS also supports multiple MMA pipelines in a threadblock.
An MMA pipeline constitutes the whole process
of loading input data from global memory to shared memory,
loading data from shared memory to registers,
doing matrix multiplication,
and storing the result to global memory.
The below flow sequence shows a typical MMA multistage pipeline
(see include/cutlass/conv/threadblock/implicit_gemm_multistage.h).

tensor in global memory
--cp_async-->
tile in shared memory
--smem loads-->
registers
--mma-->
registers
--global stores-->
output to global memory

On NVIDIA Ampere, the kernel uses `cp_async`
to build a multistage software pipeline.
This helps it better hide latency.

At this point, we can define the actual CUTLASS kernel type
as the alias ImplicitGemm, a specialization of
cutlass::conv::device::ImplicitGemmConvolution.
The latter accepts the kernel properties type alias
Conv2dFpropKernel as its one template argument.

This example then sets up a test problem
and arguments to the kernel.
We use CUTLASS utilities to allocate
the input and output tensors
and fill them with sample input data.
We then create the kernel arguments
as an instance of ImplicitGemm::Arguments.
The arguments include
the problem size (N = 1, H = 64, W = 64, C = 128),
filter size (K = 64, R = 3, S = 3, C = 128),
padding, strides, dilation, tensors, alpha, beta,
and the split k-dimension factor.
We also query CUTLASS if the kernel we instantiated
requires any memory for scratch space.
If yes, we reserve scratch space and pass it along
with other arguments to initialize the CUTLASS kernel.

After lauching the CUTLASS kernel, this example runs
a reference convolution kernel (from CUTLASS utilities)
to check correctness.
*/

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/convolution.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"

// Data types for input and output tensors
// and computation between elements
using ElementAccumulator = float;                  // Data type of accumulator
using ElementComputeEpilogue = float;              // Data type of epilogue computation (alpha, beta)
using ElementInputA = cutlass::half_t;             // Data type of elements in input tensor
using ElementInputB = cutlass::half_t;             // Data type of elements in input tensor
using ElementOutput = float;                       // Data type of elements in output tensor

using LayoutInputA = cutlass::layout::TensorNHWC;
using LayoutInputB = cutlass::layout::TensorNHWC;
using LayoutOutput = cutlass::layout::TensorNHWC;

// Whether to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// SM architecture number
using SmArch = cutlass::arch::Sm80;

// Threadblock tile shape
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 64>;

// Warp tile shape
using WarpShape = cutlass::gemm::GemmShape<64, 64, 64>;

// MMA (Tensor Core instruction, in this case) tile shape
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;

// How the kernel schedules threadblocks
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipeline stages to use
constexpr int NumStages = 3;

// Which iterator algorithm to use: Analytic or Optimized
static cutlass::conv::IteratorAlgorithm const IteratorAlgorithm = cutlass::conv::IteratorAlgorithm::kOptimized;

// The epilogue part of the kernel
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // The number of elements per vectorized
                                                       // memory access. This becomes the vector width of
                                                       // math instructions in the epilogue too.
    ElementAccumulator,                                // Data type of accumulator
    ElementComputeEpilogue>;                           // Data type for alpha/beta in linear combination

// Kernel properties type
using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  cutlass::arch::OpMultiplyAdd,
  IteratorAlgorithm
>::Kernel;

// Type of the actual kernel
using ImplicitGemm = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  cutlass::Tensor4DCoord input_size;
  cutlass::Tensor4DCoord filter_size;
  cutlass::Tensor4DCoord padding;
  cutlass::MatrixCoord conv_stride;
  cutlass::MatrixCoord dilation;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    input_size(1, 32, 32, 32),
    filter_size(32, 3, 3, 32),
    padding(1, 1, 1, 1),
    conv_stride(1, 1),
    dilation(1, 1),
    reference_check(false),
    measure_performance(true),
    iterations(20),
    save_workspace(false),
    alpha(1),
    beta(0),
    benchmark(false) { }

  // Verify that the problem size is compatible with CUTLASS's convolution implementation
  bool valid() {

    //
    // CUTLASS attempts to load 128b vectors of cutlass::half_t (F16) elements. Consequently,
    // all pointers, strides, and tensor extents must be divisible by 8 elements.
    //
    int const kAlignment = 8;

    if ((input_size.c() % kAlignment) ||
      (filter_size.n() % kAlignment)) {

      // misaligned tensors
      return false;
    }

    // Invalid padding
    if ((padding.h() != filter_size.h() / 2) ||
      (padding.w() != filter_size.w() / 2)) {

      return false;
    }

    return true;
  }

  /// Update input and filter sizes
  void update(
    cutlass::Tensor4DCoord input_size,
    cutlass::Tensor4DCoord filter_size) {

    this->input_size = input_size;
    this->filter_size = filter_size;

    padding.n() = filter_size.h() / 2;
    padding.h() = filter_size.h() / 2;
    padding.w() = filter_size.w() / 2;
    padding.c() = filter_size.w() / 2;
  }

  // Parse command-line arguments
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("n", input_size.n());
    cmd.get_cmd_line_argument("h", input_size.h());
    cmd.get_cmd_line_argument("w", input_size.w());
    cmd.get_cmd_line_argument("c", input_size.c());

    cmd.get_cmd_line_argument("k", filter_size.n());
    cmd.get_cmd_line_argument("r", filter_size.h());
    cmd.get_cmd_line_argument("s", filter_size.w());
    filter_size.c() = input_size.c();

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);

    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);

    if (filter_size.h() == 3 && filter_size.w() == 3) {
      padding = {1, 1, 1, 1};
    }
    else {
      filter_size.h() = 1;
      filter_size.w() = 1;
      padding = {0, 0, 0, 0};
    }
  }

  /// Print an explanation of the command-line arguments
  std::ostream & print_usage(std::ostream &out) const {

    out << "16_ampere_tensorop_conv2dfprop example\n\n"
      << "  This example uses Ampere's Tensor Core operators on F16 data types\n"
      << "  to compute forward convolution on tensors of layout NHWC.\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --n=<int>            Input tensor extent N\n"
      << "  --h=<int>            Input tensor extent H\n"
      << "  --w=<int>            Input tensor extent W\n"
      << "  --c=<int>            Input tensor extent C\n"
      << "  --k=<int>            Filter extent K\n"
      << "  --r=<int>            Filter extent R\n"
      << "  --s=<int>            Filter extent S\n\n"
      << "  --alpha=<float>      Epilogue scalar alpha\n"
      << "  --beta=<float>       Epilogue scalar beta\n\n"
      << "  --ref-check          If set (true), reference check on the host is computed\n"
      << "  --perf-check         If set (true), performance is measured.\n"
      << "  --benchmark          If set (true), performance benchmarking on several layers and batch-size.\n"
      << "  --iterations=<int>   Number of profiling iterations to perform.\n"
      << "  --save-workspace     If set, workspace is written to a text file.\n"
      << "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/16_ampere_tensorop_conv2dfprop/16_ampere_tensorop_conv2dfprop  --n=32 --h=224 --w=224 --c=128 --k=256 --r=1 --s=1\n\n"
      << "$ ./examples/16_ampere_tensorop_conv2dfprop/16_ampere_tensorop_conv2dfprop  --n=1 --h=224 --w=224 --c=32 --k=32 --r=3 --s=3 --ref-check\n\n";

    return out;
  }

  /// Computes the output tensor size (NPQK)
  cutlass::Tensor4DCoord output_size() const {
    return cutlass::Tensor4DCoord(
      input_size.n(),
      (input_size.h() + padding.n() + padding.h() - filter_size.h()) / conv_stride.row() + 1,
      (input_size.w() + padding.w() + padding.c() - filter_size.w()) / conv_stride.column() + 1,
      filter_size.n());
  }

  /// Compute performance in Gflop/s
  ///
  /// Gflop/s stands for billions (10^9) of
  /// floating-point operations per second (Gflop/s).
  double gflops(double runtime_s) const {

    // Number of multiply-adds = NPQK * CRS
    int64_t fmas = output_size().product() * int64_t(filter_size.h() * filter_size.w() * filter_size.c());

    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

struct Result {
  double runtime_ms;
  double gflops;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result():
    runtime_ms(0),
    gflops(0),
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream& print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "Layer,N,H,W,C,K,R,S,Runtime,GFLOPs";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out
      << "conv_" << idx << ","
      << options.input_size.n() << ","
      << options.input_size.h() << ","
      << options.input_size.w() << ","
      << options.input_size.c() << ","
      << options.filter_size.n() << ","
      << options.filter_size.h() << ","
      << options.filter_size.w() << ","
      << runtime_ms << ","
      << gflops;

    return out;
  }
};

/// Runs one benchmark
Result profile_convolution(Options const &options) {

  Result result;

  //
  // Allocate host-device tensors using the CUTLASS Utilities.
  //

  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.input_size);
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.filter_size);
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.output_size());
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.output_size());

  //
  // Initialize tensors
  //

  // Fill tensor A on host with uniformly distributed random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1,
      ElementInputA(7),
      ElementInputA(-8),
      0);

  // Fill tensor B on host with uniformly distributed random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      1,
      ElementInputB(7),
      ElementInputB(-8),
      0);

  // Fill tensor C on host with uniformly distributed random data
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      1,
      ElementOutput(7),
      ElementOutput(-8),
      0);

  // Fill tensor D on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());

  // Fill tensor D for reference on host with zeros
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());

  // Copy data from host to GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();

  //
  // Define arguments for CUTLASS Convolution
  //

  cutlass::conv::Mode mode = cutlass::conv::Mode::kCrossCorrelation;

  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Construct Conv2dProblemSize with user defined output size
  cutlass::conv::Conv2dProblemSize problem_size(
      options.input_size,
      options.filter_size,
      options.padding,
      options.conv_stride,
      options.dilation,
      options.output_size(),
      mode,
      split_k_slices
  );

  // Construct ImplicitGemm::Argument structure with conv2d
  // problem size, data pointers, and epilogue values
  typename ImplicitGemm::Arguments arguments{
    problem_size,
    tensor_a.device_ref(),
    tensor_b.device_ref(),
    tensor_c.device_ref(),
    tensor_d.device_ref(),
    {options.alpha, options.beta},
  };

  //
  // Initialize CUTLASS Convolution
  //

  ImplicitGemm implicit_gemm_op;

  size_t workspace_size = implicit_gemm_op.get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  result.status = implicit_gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  result.status = implicit_gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  //
  // Launch initialized CUTLASS kernel
  //
  result.status = implicit_gemm_op();

  CUTLASS_CHECK(result.status);

  //
  // Optional reference check
  //

  if (options.reference_check) {
    std::cout << "Verification on host...\n";

    // Compute with reference implementation
    cutlass::reference::host::Conv2dFprop<
      ElementInputA,
      LayoutInputA,
      ElementInputB,
      LayoutInputB,
      ElementOutput,
      LayoutOutput,
      ElementComputeEpilogue,
      ElementAccumulator
    >(
      problem_size,
      tensor_a.host_ref(),
      tensor_b.host_ref(),
      tensor_c.host_ref(),
      tensor_ref_d.host_ref(),
      options.alpha,
      options.beta
    );

    // Check if CUTLASS kernel and reference kernel produced the same output
    tensor_d.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(
      tensor_d.host_view(),
      tensor_ref_d.host_view());

    if (!passed) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    }
    else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  }
  else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "16_ampere_workspace_conv2dfprop_"
      << options.input_size.n() << "x" << options.input_size.h() << "x" << options.input_size.w() << "x" << options.input_size.c()
      << "_"
      << options.filter_size.n() << "x" << options.filter_size.h() << "x" << options.filter_size.w() << "x" << options.filter_size.c()
      << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace
      << "Input = \n" << tensor_a.host_view() << "\n\n"
      << "Filters = \n" << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference = \n" << tensor_ref_d.host_view() << "\n\n";
    }

    output_workspace << "Computed = \n" << tensor_d.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }

  //
  // Performance measurement
  //

  if (options.measure_performance) {

    hipEvent_t events[2];

    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // Record an event at the start of a series of convolution operations.
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Launch a sequence of implicit GEMM operations on the device.
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = implicit_gemm_op();
      CUTLASS_CHECK(result.status);
    }

    // Record an event when the convolutions have been launched.
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Wait for work on the device to complete.
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Measure elapsed runtime.
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // Print average run time and floating-point throughput (Gflop/s).
    result.runtime_ms = double(runtime_ms) / double(options.iterations);
    result.gflops = options.gflops(result.runtime_ms / 1000.0);

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

int main(int argc, char const **args) {

  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major >= 8)) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // Benchmark several layers

    int batch_sizes[] = {1, 32, 64, 128, 256, 512};

    struct Benchmark {
      int h, w, c, k, r, s;
    } layers[] = {
      {56,  56,   64,   256, 1, 1},
      {56,  56,   64,    64, 1, 1},
      {56,  56,   64,    64, 3, 3},
      {56,  56,  256,    64, 1, 1},
      {56,  56,  256,   512, 1, 1},
      {56,  56,  256,   128, 1, 1},
      {28,  28,  128,   128, 3, 3},
      {28,  28,  128,   512, 1, 1},
      {28,  28,  512,   128, 1, 1},
      {28,  28,  512,  1024, 1, 1},
      {28,  28,  512,   256, 1, 1},
      {14,  14,  256,   256, 3, 3},
      {14,  14,  256,  1024, 1, 1},
      {14,  14,  1024,  256, 1, 1},
      {14,  14,  1024, 2048, 1, 1},
      {14,  14,  1024,  512, 1, 1},
      {7,    7,   512,  512, 3, 3},
    };

    Result::print_header(std::cout, options) << std::endl;

    int idx = 1;

    for (auto const &layer : layers) {
      for (auto N : batch_sizes) {

        options.update({N, layer.h, layer.w, layer.c}, {layer.k, layer.r, layer.s, layer.c});

        Result result = profile_convolution(options);
        result.print(std::cout, idx, options) << std::endl;
      }

      ++idx;
    }
  }
  else {

    // Execute one problem size
    if (!options.valid()) {
      std::cerr << "Invalid problem." << std::endl;
      return -1;
    }

    Result result = profile_convolution(options);

    Result::print_header(std::cout, options) << std::endl;
    result.print(std::cout, 1, options) << std::endl;
  }

  return 0;
}
