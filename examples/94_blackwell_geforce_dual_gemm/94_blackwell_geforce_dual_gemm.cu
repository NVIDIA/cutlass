/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Blackwell GeForce Dual-GEMM example using CUTLASS 3.0 APIs for the NVIDIA Blackwell SM120 architecture.

    This example is based on example 79a_blackwell_geforce_nvfp4_bf16_gemm.cu but for Dual-GEMM

```
D0 = epilogue0(X @ B0, C0)
D1 = epilogue1(X @ B1, C1)
D2 = element_wise(D0, D1)
```    
    Usage:

      $ ./examples/94_blackwell_geforce_dual_gemm/94_blackwell_geforce_dual_gemm --m=2048 --n=2048 --k=2048
*/

#include <iostream>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/detail/sm100_blockscaled_layout.hpp"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gett.hpp"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_compare.h"

#include "helper.h"

#include "collective/dispatch_policy_extra.hpp"
#include "collective/builder.hpp"
#include "kernel/sm90_gemm_tma_warpspecialized_cooperative_dual.hpp"
#include "device/gemm_universal_adapter.h"
#include "collective/builder_epilogue.hpp"
#include "thread/left_silu_and_mul.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::nv_float4_t<cutlass::float_e2m1_t>;    // Element type for A matrix operand
using         LayoutATag  = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 32;                                             // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::nv_float4_t<cutlass::float_e2m1_t>;    // Element type for B matrix operand
using         LayoutB0Tag = cutlass::layout::ColumnMajor;                   // Layout type for B0 matrix operand
using         LayoutB1Tag = cutlass::layout::ColumnMajor;                   // Layout type for B1 matrix operand
constexpr int AlignmentB  = 32;                                             // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementD    = cutlass::bfloat16_t;                            // Element type for D matrix operand
using         ElementC    = cutlass::bfloat16_t;                            // Element type for C matrix operand
using         LayoutCTag  = cutlass::layout::RowMajor;                      // Layout type for C matrix operand
using         LayoutDTag  = cutlass::layout::RowMajor;                      // Layout type for D matrix operand
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;    // Memory access granularity/alignment of D matrix in units of elements (up to 16 bytes)
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)
// Kernel functional config
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm120;                           // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassBlockScaledTensorOp;      // Operator class tag

// Kernel Perf config
using ThreadBlockShape    = Shape<_128,_128,_128>;                          // Threadblock's tile size
using ClusterShape        = Shape<_1,_1,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::DualKernelTmaWarpSpecializedCooperativeBlockScaledSm120<3>;
using EpilogueSchedule    = cutlass::epilogue::DualTmaWarpSpecialized;
using EpilogueTileType    = cutlass::epilogue::collective::EpilogueTileAuto;
using ElementCompute      = float;

using OpLeft  = cutlass::epilogue::fusion::DualLinearCombination<ElementD, float, ElementC, float>;
using OpRight = cutlass::epilogue::fusion::DualLinearCombination<ElementD, float, ElementC, float>;
using DualPairOp = cutlass::epilogue::fusion::DualOpPair<OpLeft, OpRight>;

using CollectiveEpilogue = typename cutlass::epilogue::collective::DualCollectiveBuilder<
    ArchTag, OperatorClass,
    ThreadBlockShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutCTag, AlignmentC,
    ElementD, LayoutDTag, AlignmentD,
    EpilogueSchedule,
    DualPairOp
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::DualCollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutATag, AlignmentA,
    ElementB, LayoutB0Tag, LayoutB1Tag, AlignmentB,
    ElementAccumulator,
    ThreadBlockShape, ClusterShape,
    cutlass::gemm::collective::StageCount<3>,
    KernelSchedule                             
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>,                                                   // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue,
    void>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using StrideA   = typename Gemm::GemmKernel::StrideA;
using LayoutA   = decltype(cute::make_layout(make_shape(0,0,0), StrideA{}));
using LayoutSFA = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFA;      // Scale Factor tensors have an interleaved layout. Bring Layout instead of stride.
using StrideB0   = typename Gemm::GemmKernel::StrideB0;
using LayoutB0   = decltype(cute::make_layout(make_shape(0,0,0), StrideB0{}));
using LayoutSFB0 = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB0;    // Scale Factor tensors have an interleaved layout. Bring Layout instead of stride.
using StrideB1   = typename Gemm::GemmKernel::StrideB1;
using LayoutB1   = decltype(cute::make_layout(make_shape(0,0,0), StrideB1{}));
using LayoutSFB1 = typename Gemm::GemmKernel::CollectiveMainloop::LayoutSFB1;    // Scale Factor tensors have an interleaved layout. Bring Layout instead of stride.
using StrideC   = typename Gemm::GemmKernel::StrideC;
using LayoutC   = decltype(cute::make_layout(make_shape(0,0,0), StrideC{}));
using StrideD   = typename Gemm::GemmKernel::StrideD;
using LayoutD   = decltype(cute::make_layout(make_shape(0,0,0), StrideD{}));

//
// Data members
//

/// Initialization
StrideA stride_A;
LayoutA layout_A;
LayoutSFA layout_SFA;
StrideB0 stride_B0;
LayoutB0 layout_B0;
LayoutSFB0 layout_SFB0;
StrideB1 stride_B1;
LayoutB1 layout_B1;
LayoutSFB1 layout_SFB1;
StrideC stride_C;
LayoutC layout_C;
StrideD stride_D;
LayoutD layout_D;
uint64_t seed;

// The HostTensors are only used for allocating memory on host and device, and transferring data between host and device
// Use cute::Tensor and cute::Layout for iterating thru the matrix elements
cutlass::HostTensor<ElementA::DataType, cutlass::layout::PackedVectorLayout> block_A;
cutlass::HostTensor<ElementA::ScaleFactorType, cutlass::layout::PackedVectorLayout> block_SFA;
cutlass::HostTensor<ElementB::DataType, cutlass::layout::PackedVectorLayout> block_B0;
cutlass::HostTensor<ElementB::DataType, cutlass::layout::PackedVectorLayout> block_B1;
cutlass::HostTensor<ElementB::ScaleFactorType, cutlass::layout::PackedVectorLayout> block_SFB0;
cutlass::HostTensor<ElementB::ScaleFactorType, cutlass::layout::PackedVectorLayout> block_SFB1;
cutlass::HostTensor<ElementC, cutlass::layout::PackedVectorLayout> block_C;
// Output Tensor
cutlass::HostTensor<ElementD, cutlass::layout::PackedVectorLayout> block_D;
// Reference Output Tensor
cutlass::HostTensor<ElementD, cutlass::layout::PackedVectorLayout> block_reference_D;
#endif // defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)

template <typename T>
auto make_iterator(T* ptr) {
  return cute::recast_ptr<T>(ptr);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;

  Options():
    help(false),
    m(1024), n(1024), k(1024),
    alpha(1.f), beta(0.f),
    iterations(10)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "94_blackwell_geforce_dual_gemm\n\n"
      << "  Blackwell NVFP4 Dual-GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out << "\n\nExamples:\n\n"
      << "$ " << "./examples/94_blackwell_geforce_dual_gemm/94_blackwell_geforce_dual_gemm.cu" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add x2 for dual gemm
    uint64_t flop = uint64_t(4) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_block(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed) {

  double scope_max, scope_min;
  constexpr int bits_input = cutlass::sizeof_bits<Element>::value;

  if constexpr (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if constexpr (bits_input <= 6) {
    scope_max = 2;
    scope_min = -2;
  }
  else if constexpr (bits_input <= 8) {
    if constexpr (cute::is_same_v<Element, cutlass::float_ue8m0_t>) {
      scope_max = 4;
      scope_min = 1;
    }
    else {
      scope_max = 1;
      scope_min = -1;
    }
  }
  else{
    scope_max = 4;
    scope_min = -4;
  }
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {
  using namespace cute;
  // For SFA and SFB tensors layouts
  using Sm1xxBlkScaledConfig =  typename Gemm::GemmKernel::CollectiveMainloop::Sm1xxBlkScaledConfig;

  stride_A  = cutlass::make_cute_packed_stride(StrideA{}, {options.m, options.k, 1});
  stride_B0 = cutlass::make_cute_packed_stride(StrideB0{}, {options.n, options.k, 1});
  stride_B1 = cutlass::make_cute_packed_stride(StrideB1{}, {options.n, options.k, 1});
  stride_C  = cutlass::make_cute_packed_stride(StrideC{}, {options.m, options.n, 1});
  stride_D  = cutlass::make_cute_packed_stride(StrideD{}, {options.m, options.n, 1});

  layout_A  = make_layout(make_shape(options.m, options.k, 1), stride_A);
  layout_B0 = make_layout(make_shape(options.n, options.k, 1), stride_B0);
  layout_B1 = make_layout(make_shape(options.n, options.k, 1), stride_B1);
  layout_C  = make_layout(make_shape(options.m, options.n, 1), stride_C);
  layout_D  = make_layout(make_shape(options.m, options.n, 1), stride_D);
  layout_SFA  = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFA(cute::make_shape(options.m, options.n, options.k, 1));
  layout_SFB0 = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(options.m, options.n, options.k, 1));
  layout_SFB1 = Sm1xxBlkScaledConfig::tile_atom_to_shape_SFB(cute::make_shape(options.m, options.n, options.k, 1));

  block_A.reset(cutlass::make_Coord(size(layout_A)));
  block_B0.reset(cutlass::make_Coord(size(layout_B0)));
  block_B1.reset(cutlass::make_Coord(size(layout_B1)));
  block_C.reset(cutlass::make_Coord(size(layout_C)));
  block_D.reset(cutlass::make_Coord(size(layout_D)));
  block_reference_D.reset(cutlass::make_Coord(size(layout_D)));
  block_SFA.reset(cutlass::make_Coord(size(filter_zeros(layout_SFA))));
  block_SFB0.reset(cutlass::make_Coord(size(filter_zeros(layout_SFB0))));
  block_SFB1.reset(cutlass::make_Coord(size(filter_zeros(layout_SFB1))));

  initialize_block(block_A.host_view(), seed + 2021);
  initialize_block(block_B0.host_view(), seed + 2022);
  initialize_block(block_B1.host_view(), seed + 3022);
  initialize_block(block_C.host_view(), seed + 2023);
  initialize_block(block_SFA.host_view(), seed + 2024);
  initialize_block(block_SFB0.host_view(), seed + 2025);
  initialize_block(block_SFB1.host_view(), seed + 3025);

  block_A.sync_device();
  block_B0.sync_device();
  block_B1.sync_device();
  block_C.sync_device();
  block_SFA.sync_device();
  block_SFB0.sync_device();
  block_SFB1.sync_device();
}

// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments {
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, 1},
    { // Mainloop arguments
      block_A.device_data(), stride_A,
      block_B0.device_data(), stride_B0,
      block_B1.device_data(), stride_B1,
      block_SFA.device_data(), layout_SFA,
      block_SFB0.device_data(), layout_SFB0,
      block_SFB1.device_data(), layout_SFB1
    },
    { // Epilogue arguments
      {
        { options.alpha, options.beta },   // op0
        { options.alpha, options.beta }    // op1
      },
      block_C.device_data(), stride_C,
      block_D.device_data(), stride_D
    }
  };

  return arguments;
}

bool verify(const Options &options) {
  using namespace cute;
  // Create the arguments for host reference implementation
  Tensor tensor_A = make_tensor(make_iterator(block_A.host_data()), layout_A);
  Tensor tensor_SFA = make_tensor(block_SFA.host_data(), layout_SFA);
  Tensor tensor_B0 = make_tensor(make_iterator(block_B0.host_data()), layout_B0);
  Tensor tensor_SFB0 = make_tensor(block_SFB0.host_data(), layout_SFB0);
  Tensor tensor_B1 = make_tensor(make_iterator(block_B1.host_data()), layout_B1);
  Tensor tensor_SFB1 = make_tensor(block_SFB1.host_data(), layout_SFB1);

  // Prepare host buffers for two separate GEMM outputs:
  cutlass::HostTensor<ElementD, cutlass::layout::PackedVectorLayout> block_reference_D0;
  cutlass::HostTensor<ElementD, cutlass::layout::PackedVectorLayout> block_reference_D1;
  block_reference_D0.reset(cutlass::make_Coord(size(layout_D)));
  block_reference_D1.reset(cutlass::make_Coord(size(layout_D)));

  auto tensor_C  = cute::make_tensor(make_iterator(block_C.host_data()), layout_C);
  auto tensor_D0 = cute::make_tensor(make_iterator(block_reference_D0.host_data()), layout_D);
  auto tensor_D1 = cute::make_tensor(make_iterator(block_reference_D1.host_data()), layout_D);

  // First GEMM: D0 = alpha * (A @ B0) + beta * C  (linear combination epilogue)
  cutlass::reference::host::GettBlockScalingMainloopParams<
      ElementAccumulator,                 // ElementAccumulator
      decltype(tensor_A),                 // TensorA
      decltype(tensor_SFA),               // TensorSfA
      decltype(tensor_B0),                // TensorB0
      decltype(tensor_SFB0)               // TensorSfB0
    > mainloop_params0{tensor_A, tensor_SFA, tensor_B0, tensor_SFB0};

  cutlass::reference::host::GettBlockScalingEpilogueParams<
      ElementAccumulator,                 // ElementScalar
      ElementAccumulator,                 // ElementAccumulator
      ElementAccumulator,                 // ElementCompute
      decltype(tensor_C),                 // TensorC
      decltype(tensor_D0)                 // TensorD0
    > epilogue_params0{options.alpha, options.beta, tensor_C, tensor_D0};

  cutlass::reference::host::Gemm3x(mainloop_params0, epilogue_params0);

  // Second GEMM: D1 = alpha * (A @ B1) + beta * C  (linear combination epilogue)
  cutlass::reference::host::GettBlockScalingMainloopParams<
      ElementAccumulator,
      decltype(tensor_A),
      decltype(tensor_SFA),
      decltype(tensor_B1),
      decltype(tensor_SFB1)
    > mainloop_params1{tensor_A, tensor_SFA, tensor_B1, tensor_SFB1};

  cutlass::reference::host::GettBlockScalingEpilogueParams<
      ElementAccumulator,
      ElementAccumulator,
      ElementAccumulator,
      decltype(tensor_C),
      decltype(tensor_D1)
    > epilogue_params1{options.alpha, options.beta, tensor_C, tensor_D1};

  cutlass::reference::host::Gemm3x(mainloop_params1, epilogue_params1);

  // Compute final D2_ref = LeftSiLUAndMul(D0_ref, D1_ref) 
  int64_t total_elems = static_cast<int64_t>(options.m) * static_cast<int64_t>(options.n);

  using LeftOp = cutlass::epilogue::thread::LeftSiLUAndMul<
      ElementD,                     // ElementOutput (store as ElementD)
      1,                            // Count: scalar path
      ElementAccumulator,           // ElementAccumulator (accum type)
      ElementAccumulator,           // ElementCompute (compute type)
      cutlass::FloatRoundStyle::round_to_nearest
    >;

  LeftOp leftop(typename LeftOp::Params{}); // Params is empty struct

  // elementwise loop (host)
  ElementD *d0_ptr = block_reference_D0.host_data();
  ElementD *d1_ptr = block_reference_D1.host_data();
  ElementD *d2_ptr = block_reference_D.host_data(); 

  for (int64_t i = 0; i < total_elems; ++i) {
    ElementAccumulator lhs_acc = ElementAccumulator(d0_ptr[i]);
    ElementAccumulator rhs_acc = ElementAccumulator(d1_ptr[i]);

    d2_ptr[i] = leftop(lhs_acc, rhs_acc);
  }

  // Comparison
  block_D.sync_host();
  bool passed = cutlass::reference::host::TensorEquals(block_reference_D.host_view(), block_D.host_view());
  passed &= (cutlass::reference::host::TensorNorm(block_reference_D.host_view()) > 0);
  passed &= (cutlass::reference::host::TensorNorm(block_D.host_view()) > 0);

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  hipDeviceSynchronize();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);
    
  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);


    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.8 or higher Toolkit to run this example
  // and must have compute capability at least 100.
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)) {
    std::cerr << "This example requires CUDA 12.8 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));

  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));

  if (!(props.major == 12 && props.minor == 0)) {
    std::cerr << "This example requires a GPU of NVIDIA's Blackwell architecture (compute capability 120)." << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //
#if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)
  run<Gemm>(options);
#endif // defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
