#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief A GEMM example using CUTLASS for the NVIDIA Blackwell SM100 architecture with the Stream-K scheduler.

    Stream-K is a GEMM parallelization technique that attempts to reduce load imbalance across SMs
    by parallelizing certain output tiles across the K mode of the GEMM, without using a static splitting factor.
    For complete details on Stream-K, please see https://arxiv.org/abs/2301.03598.

    CUTLASS's Stream-K scheduler using the CUTLASS 3.x API is capable of supporting various modes of
    decomposing a GEMM (referred to as "decomposition modes" in this example):
      * DataParallel: basic GEMM parallelized spatially via tiling, but without splitting the K mode
      * SplitK: `split_factor` CTAs compute portions of the K mode for a given output tile and reduce their results
      * StreamK: parallelizes work according to the stream-K load balancing method described in https://arxiv.org/abs/2301.03598
      * Heuristic: applies an internal heuristic in attempt to choose the most performant among the three preceding decomposition modes

    Additionally, the Stream-K scheduler supports two different means of performing reductions for
    decomposition modes that require reduction (SplitK, StreamK, and Heuristic):
      * Deterministic: Participating CTAs perform reduction in a turnstile fashion in order of the K mode
                       covered by each CTA. This requires a lock to be held exclusively by the CTA that is
                       currently accumulating.
      * Nondeterministic: Participating CTAs perform reduction atomically to the same workspace (mostly) without locking.
                          Locks are used only to wait for the first CTA to write its partial values (to initialize the
                          workspace), and for all but the final CTA to have accumulated (so that the final CTA can load
                          the accumulated value and accumulate it into registers on top of which the epilogue will
                          be performed). Due to the nondeterminsitic ordering of accumulation, deterministic numeric
                          behavior cannot be guaranteed with this mode (e.g., floating-point rounding error will depend
                          on the order of accumulation)

    This example allows one to try out different decomposition modes, reduction modes, and (when using Split-K) splitting factors.
    Here are a few examples of usage:
       # Heuristic mode with deterministic reduction
      ./74_blackwell_gemm_streamk" --m=256 --n=256 --k=16384 --decomposition=Heuristic --reduction=Deterministic

      # Stream-K mode with determinsitic reduction
      ./74_blackwell_gemm_streamk" --m=256 --n=256 --k=16384 --decomposition=StreamK --reduction=Deterministic

      # Split-K mode with a splitting factor of 2 and deterministic reduction
      ./74_blackwell_gemm_streamk" --m=256 --n=256 --k=16384 --decomposition=SplitK --reduction=Deterministic --splits=2

      # Stream-K mode with nondeterministic reduction
      ./74_blackwell_gemm_streamk" --m=256 --n=256 --k=16384 --decomposition=StreamK --reduction=Nondeterministic
*/



#include <iostream>
#include <string>
#include <unordered_map>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler_params.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = half_t;                                         // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = half_t;                                          // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;                                          // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Kernel functional config
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm100;                           // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag

// MMA and Cluster Tile Shapes
// Shape of the tile computed by tcgen05 MMA, could be across 2 SMs if Cluster Shape % 2 == 0
using MmaTileShape_MNK = Shape<_256,_128,_64>;
// Shape of the cluster set to <int,int,_1> to indicate dynamic cluster shape
using ClusterShape_MNK = Shape<int,int,_1>;
// When dynamic cluster is used, KernelScheduleAuto always selects mainloop dispatch policy that 
// lowers to tcgen05 MMA cta_group = 1 as we don't know if the dynamic cluster M dimension will be a multiple of 2
// To use tcgen05 MMA cta_group = 2, users must explicitly use 2sm builder schedules
using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecialized2SmSm100;
using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecialized2Sm;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    MmaTileShape_MNK, ClusterShape_MNK,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC, AlignmentC,
    ElementC, LayoutC, AlignmentC,
    cutlass::epilogue::collective::EpilogueScheduleAuto
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    MmaTileShape_MNK, ClusterShape_MNK,
    cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;
using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int, int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue,
    cutlass::gemm::StreamKScheduler // <--- Change needed to enable the stream-K scheduler
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;
  int preferred_cluster_m, preferred_cluster_n, fallback_cluster_m, fallback_cluster_n;
  using DecompositionMode = cutlass::gemm::kernel::detail::PersistentTileSchedulerSm90StreamKParams::DecompositionMode;
  using ReductionMode = cutlass::gemm::kernel::detail::PersistentTileSchedulerSm90StreamKParams::ReductionMode;
  DecompositionMode decomposition_mode;
  ReductionMode reduction_mode;
  int splits;

  std::unordered_map<DecompositionMode, std::vector<std::string>> dec_mappings = {
    {DecompositionMode::Heuristic,    {"Heuristic", "heuristic", "h", "H", ""}},
    {DecompositionMode::SplitK,       {"SplitK", "split-k", "split-K", "Split-K", "Split-k", "splitk", "Splitk", "splitK", "spk", "SpK", "spK"}},
    {DecompositionMode::StreamK,      {"StreamK", "stream-k", "stream-K", "Stream-K", "Stream-k", "streamk", "Streamk", "streamK", "stk", "StK", "stK"}},
    {DecompositionMode::DataParallel, {"DataParallel", "data-parallel", "dataparallel", "dp", "DP"}}
  };

  std::unordered_map<ReductionMode, std::vector<std::string>> red_mappings = {
    {ReductionMode::Deterministic,    {"Deterministic", "deterministic", "d", "D", ""}},
    {ReductionMode::Nondeterministic, {"Nondeterministic", "nondeterministic", "n", "N"}}
  };

  Options():
    help(false),
    m(256), n(256), k(16384),
    alpha(1.f), beta(0.f),
    iterations(10),
    preferred_cluster_m(4),
    preferred_cluster_n(4),
    fallback_cluster_m(2),
    fallback_cluster_n(1),
    decomposition_mode(DecompositionMode::Heuristic),
    reduction_mode(ReductionMode::Deterministic),
    splits(1)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("splits", splits, 1);
    cmd.get_cmd_line_argument("preferred_cluster_m", preferred_cluster_m, 4);
    cmd.get_cmd_line_argument("preferred_cluster_n", preferred_cluster_n, 4);
    cmd.get_cmd_line_argument("fallback_cluster_m", fallback_cluster_m, 2);
    cmd.get_cmd_line_argument("fallback_cluster_n", fallback_cluster_n, 1);

    // Parse decompsition mode
    std::string decomp_mode;
    cmd.get_cmd_line_argument("decomposition", decomp_mode);
    bool found = parse_from_options_map(decomp_mode, dec_mappings, decomposition_mode);
    if (!found) {
      std::cout << "--decomposition must be one of Heuristic, SplitK, StreamK, or DataParallel" << std::endl;
      help = true;
      return;
    }

    // Parse reduction mode
    std::string red_mode;
    cmd.get_cmd_line_argument("reduction", red_mode);
    found = parse_from_options_map(red_mode, red_mappings, reduction_mode);
    if (!found) {
      std::cout << "--reduction must be one of Deterministic and Nondeterministic" << std::endl;
      help = true;
      return;
    }
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "74_blackwell_gemm_streamk\n\n"
      << "  Blackwell FP16 GEMM using a stream-K kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n"
      << "  --preferred_cluster_m=<str> Sets the M extent of preferred cluster shape\n"
      << "  --preferred_cluster_n=<str> Sets the N extent of preferred cluster shape\n"
      << "  --fallback_cluster_m=<str>  Sets the M extent of fallback cluster shape\n"
      << "  --fallback_cluster_n=<str>  Sets the N extent of fallback cluster shape\n"
      << "  --decomposition=<str>       Mode in which the stream-K kernel should decompose the problem. Options: Heuristic (default), SplitK, StreamK, DataParallel\n"
      << "  --reduction=<str>           Mode in which the stream-K kernel's reduction should be performed. Options: Deterministic (default), Nondeterministic\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "74_blackwell_gemm_streamk" << " --m=256 --n=256 --k=16384 --decomposition=Heuristic --reduction=Deterministic \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }

  std::string decomposition_mode_str() const {
    return dec_mappings.at(decomposition_mode).at(0);
  }

  std::string reduction_mode_str() const {
    return red_mappings.at(reduction_mode).at(0);
  }

 private:
  template <class T>
  bool parse_from_options_map(std::string val, std::unordered_map<T, std::vector<std::string>> options, T& result) const {
    for (const auto & [key, values] : options) {
      if (std::find(values.begin(), values.end(), val) != values.end()) {
        result = key;
        return true;
      }
    }
    return false;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(cutlass::DeviceAllocation<Element>& block, uint64_t seed=2023) {
  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = Element(2);
    scope_min = Element(0);
  } else if (bits_input <= 8) {
    scope_max = Element(2);
    scope_min = Element(-2);
  } else {
    scope_max = Element(8);
    scope_min = Element(-8);
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, {options.m, options.k, 1});
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, {options.n, options.k, 1});
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, {options.m, options.n, 1});
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, {options.m, options.n, 1});

  block_A.reset(options.m * options.k);
  block_B.reset(options.k * options.n);
  block_C.reset(options.m * options.n);
  block_D.reset(options.m * options.n);
  block_ref_D.reset(options.m * options.n);

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options) {
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, 1},
    {block_A.get(), stride_A, block_B.get(), stride_B},
    {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
  };

  arguments.hw_info.cluster_shape = dim3(options.preferred_cluster_m, options.preferred_cluster_n,1);
  arguments.hw_info.cluster_shape_fallback = dim3(options.fallback_cluster_m, options.fallback_cluster_n,1);

  arguments.scheduler.splits = options.splits;
  arguments.scheduler.decomposition_mode = options.decomposition_mode;
  arguments.scheduler.reduction_mode = options.reduction_mode;

  return arguments;
}

bool verify(const Options &options) {
  cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({options.m, options.k}));
  cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({options.k, options.n}));
  cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({options.m, options.n}));
  cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({options.m, options.n}));

  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    {options.m, options.n, options.k},
    ElementAccumulator(options.alpha),
    ref_A,
    ref_B,
    ElementAccumulator(options.beta),
    ref_C,
    ref_D);

  // Wait for kernel to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

  return passed;
}

/// Execute a given example GEMM computation
int run(Options &options) {

  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "Stream-K GEMM with"
            << " Problem Size: " << options.m << 'x' << options.n << 'x' << options.k
            << " Preferred Cluster = (" << options.preferred_cluster_m << ", " << options.preferred_cluster_n << ", 1)"
            << " Fallback Cluster = (" << options.fallback_cluster_m << ", " << options.fallback_cluster_n << ", 1)\n"
            << " Decomposition_mode=" << options.decomposition_mode_str()
            << " Split_count=" << options.splits
            << " Reduction_mode=" << options.reduction_mode_str()
            << std::endl;

  std::cout << "--------------------------------------------------------------------------------" << std::endl;

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.8 Toolkit to run this example
  // and must have compute capability at least 100.
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)) {
    std::cerr << "This example requires CUDA 12.8 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));

  if (props.major != 10 && (props.minor != 0 || props.minor != 1)) {
    std::cerr << "This example requires a GPU of NVIDIA's Blackwell architecture (compute capability 100 or 101)." << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //
#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
  run(options);
#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
