#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper Grouped GEMM example using CUTLASS 3 APIs for NVIDIA Hopper architecture.

    This example demonstrates an implementation of Grouped GEMM using a TMA + GMMA 
    warp-specialized cooperative kernel.
    For this example all scheduling work is performed on the device.
    The new feature showcased in this example is on-the-fly modification of TMA descriptors
    to move between groups/problem_count (represented by groups).

    To run this example:

      $ ./examples/57_hopper_grouped_gemm/57_hopper_grouped_gemm --m=2048 --n=2048 --k=2048 --groups=10

      The above example command makes all 10 groups to be sized at the given m, n, k sizes. 
      Skipping any of the problem dimensions randomizes it across the different groups.

    To run this example for a set of problems using the benchmark option:

      $ ./examples/57_hopper_grouped_gemm/57_hopper_grouped_gemm --benchmark=./test_benchmark.txt

      Where the test_benchmark.txt may look as such:
        0 256x512x128
        1 256x512x512
        2 512x256x128
        3 256x256x128
        4 256x512x1024
        5 1024x512x128 and so on
*/

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;
using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int,int,int>>; // <M,N,K> per group
using ElementA = cutlass::float_e4m3_t;                          // Element type for A matrix operand
using ElementB = cutlass::float_e5m2_t;                          // Element type for B matrix operand
using ElementC = float;                                          // Element type for C and D matrix operands

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_256,_128,_64>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_1,_2,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::KernelGroupTmaWarpSpecializedCooperativeFP8FastAccum; // Kernel to launch
using EpilogueSchedule = cutlass::epilogue::NoSmemWarpSpecializedGroup;                     // Epilogue to launch

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC, AlignmentC,
    ElementC, LayoutC, AlignmentC,
    EpilogueSchedule
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    ProblemShape,
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

// Host-side allocations
std::vector<int64_t> offset_A;
std::vector<int64_t> offset_B;
std::vector<int64_t> offset_C;
std::vector<int64_t> offset_D;

std::vector<StrideA> stride_A_host;
std::vector<StrideB> stride_B_host;
std::vector<StrideC> stride_C_host;
std::vector<StrideD> stride_D_host;

// Device-side allocations
cutlass::DeviceAllocation<typename ProblemShape::UnderlyingProblemShape> problem_sizes;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

cutlass::DeviceAllocation<const typename Gemm::ElementA *> ptr_A;
cutlass::DeviceAllocation<const typename Gemm::ElementB *> ptr_B;
cutlass::DeviceAllocation<const typename Gemm::ElementC *> ptr_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_ref_D;

cutlass::DeviceAllocation<StrideA> stride_A;
cutlass::DeviceAllocation<StrideB> stride_B;
cutlass::DeviceAllocation<StrideC> stride_C;
cutlass::DeviceAllocation<StrideD> stride_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  float alpha = 1.0f;
  float beta = 0.0f;
  int iterations = 10;
  int m = 1024, n = 2048, k = 512, groups = 10;
  std::string benchmark_path;
  std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes_host;
  int const tma_alignment_bits = 128;
  int const alignment = tma_alignment_bits / cutlass::sizeof_bits<ElementA>::value; 

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("groups", groups);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("benchmark", benchmark_path);

    // Decide how to initialize the problems
    if (!benchmark_path.empty()) {
      if (!benchmark_problems()) {
        problem_sizes_host.clear();
        return;
      }
    }
    else {
      randomize_problems(cmd);
    }
  }

  void randomize_problems(cutlass::CommandLine &cmd) {
    int cmd_line_m = -1;
    int cmd_line_n = -1;
    int cmd_line_k = -1;

    cmd.get_cmd_line_argument("m", cmd_line_m);
    cmd.get_cmd_line_argument("n", cmd_line_n);
    cmd.get_cmd_line_argument("k", cmd_line_k);

    problem_sizes_host.reserve(groups);

    for (int i = groups; i > 0; i--) {

      int m = cmd_line_m;
      int n = cmd_line_n;
      int k = cmd_line_k;

      if (m < 1) {
        m = ((rand() % 512) + 1);
      }

      if (n < 1) {
        n = ((rand() % 512) + 1);
      }

      if (k < 1) {
        k = alignment * ((rand() % 64) + 1);
      }
      problem_sizes_host.push_back({m, n, k});
    }
  }

  /// Load a benchmark
  bool benchmark_problems() {
    std::ifstream file(benchmark_path);
    if (!file.good()) {
      return false;
    }

    while (file.good()) {

      int idx = -1;
      std::string extent_str;

      file >> idx >> extent_str;

      if (idx < 0 || extent_str.empty()) {
        break;
      }

      cutlass::gemm::GemmCoord extent;
      std::vector<std::string> tokens;

      cutlass::CommandLine::tokenize(tokens, extent_str, 'x');

      for (int i = 0; i < int(tokens.size()); ++i) {
        int x = std::atoi(tokens.at(i).c_str());

        // round up
        if (x % alignment) {
          x += (alignment - (x % alignment));
        }

        extent.at(i) = x;
      }

      if (extent.product()) {
        problem_sizes_host.push_back({extent.m(), extent.n(), extent.k()});
      }
    }

    return true;
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "57_hopper_grouped_gemm\n\n"
      << "  Hopper FP8 Grouped GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM for all groups\n"
      << "  --n=<int>                   Sets the N extent of the GEMM for all groups\n"
      << "  --k=<int>                   Sets the K extent of the GEMM for all groups\n"
      << "  --groups=<int>              Sets the number of individual GEMM problems for Grouped GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform\n\n"
      << "  --benchmark=<str>           Executes a benchmark problem size.\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "57_hopper_grouped_gemm" << " --m=1024 --n=512 --k=1024 --groups=10 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s, std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes_host) const
  {
    // Number of real-valued multiply-adds
    uint64_t fmas = uint64_t();

    for (auto const & problem : problem_sizes_host) {
      fmas += cute::size(problem);
    }
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * uint64_t(fmas);
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms = 0.0;
  double gflops = 0.0;
  cutlass::Status status = cutlass::Status::kSuccess;
  hipError_t error = hipSuccess;
  bool passed = false;
};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = static_cast<Element>(2);
    scope_min = static_cast<Element>(0);
  } else if (bits_input <= 8) {
    scope_max = static_cast<Element>(2);
    scope_min = static_cast<Element>(-2);
  } else {
    scope_max = static_cast<Element>(8);
    scope_min = static_cast<Element>(-8);
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Allocates device-side data
void allocate(const Options &options) {
  int64_t total_elements_A = 0;
  int64_t total_elements_B = 0;
  int64_t total_elements_C = 0;
  int64_t total_elements_D = 0;

  for (int32_t i = 0; i < options.groups; ++i) {

    auto problem = options.problem_sizes_host.at(i);
    auto M = get<0>(problem);
    auto N = get<1>(problem);
    auto K = get<2>(problem);

    offset_A.push_back(total_elements_A);
    offset_B.push_back(total_elements_B);
    offset_C.push_back(total_elements_C);
    offset_D.push_back(total_elements_D);

    int64_t elements_A = M * K;
    int64_t elements_B = K * N;
    int64_t elements_C = M * N;
    int64_t elements_D = M * N;

    total_elements_A += elements_A;
    total_elements_B += elements_B;
    total_elements_C += elements_C;
    total_elements_D += elements_D;

    stride_A_host.push_back(cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, Int<1>{})));
    stride_B_host.push_back(cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, Int<1>{})));
    stride_C_host.push_back(cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, Int<1>{})));
    stride_D_host.push_back(cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, Int<1>{})));
  }

  block_A.reset(total_elements_A);
  block_B.reset(total_elements_B);
  block_C.reset(total_elements_C);
  block_D.reset(total_elements_D);
  block_ref_D.reset(total_elements_D);
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {
  
  uint64_t seed = 2020;

  problem_sizes.reset(options.groups);
  problem_sizes.copy_from_host(options.problem_sizes_host.data());

  //
  // Assign pointers
  //

  std::vector<ElementA *> ptr_A_host(options.groups);
  std::vector<ElementB *> ptr_B_host(options.groups);
  std::vector<ElementC *> ptr_C_host(options.groups);
  std::vector<ElementC *> ptr_D_host(options.groups);

  for (int32_t i = 0; i < options.groups; ++i) {
    ptr_A_host.at(i) = block_A.get() + offset_A.at(i);
    ptr_B_host.at(i) = block_B.get() + offset_B.at(i);
    ptr_C_host.at(i) = block_C.get() + offset_C.at(i);
    ptr_D_host.at(i) = block_D.get() + offset_D.at(i);
  }

  ptr_A.reset(options.groups);
  ptr_A.copy_from_host(ptr_A_host.data());

  ptr_B.reset(options.groups);
  ptr_B.copy_from_host(ptr_B_host.data());

  ptr_C.reset(options.groups);
  ptr_C.copy_from_host(ptr_C_host.data());

  ptr_D.reset(options.groups);
  ptr_D.copy_from_host(ptr_D_host.data());

  stride_A.reset(options.groups);
  stride_A.copy_from_host(stride_A_host.data());

  stride_B.reset(options.groups);
  stride_B.copy_from_host(stride_B_host.data());

  stride_C.reset(options.groups);
  stride_C.copy_from_host(stride_C_host.data());

  stride_D.reset(options.groups);
  stride_D.copy_from_host(stride_D_host.data());

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  cutlass::KernelHardwareInfo hw_info;
  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGrouped,
    {options.groups, problem_sizes.get(), options.problem_sizes_host.data()},
    {ptr_A.get(), stride_A.get(), ptr_B.get(), stride_B.get()},
    {{options.alpha, options.beta}, ptr_C.get(), stride_C.get(), ptr_D.get(), stride_D.get()},
    hw_info
  };

  return arguments;
}

bool verify(const Options &options) {
  bool passed = true;
  for (int32_t i = 0; i < options.groups; ++i) {
    auto problem = options.problem_sizes_host.at(i);
    auto M = get<0>(problem);
    auto N = get<1>(problem);
    auto K = get<2>(problem);
    cutlass::TensorRef ref_A(block_A.get() + offset_A.at(i), Gemm::LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get() + offset_B.at(i), Gemm::LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get() + offset_C.at(i), Gemm::LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get() + offset_D.at(i), Gemm::LayoutD::packed({M, N}));

    //
    // Compute reference output
    //

    // Create instantiation for device reference gemm kernel
    DeviceGemmReference gemm_reference;

    // Launch device reference gemm kernel
    gemm_reference(
      {M, N, K},
      ElementAccumulator(options.alpha),
      ref_A,
      ref_B,
      ElementAccumulator(options.beta),
      ref_C,
      ref_D);

    // Wait for kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    passed &= cutlass::reference::device::BlockCompareEqual(block_ref_D.get() + offset_D.at(i), block_D.get() + offset_D.at(i), M * N);
    #if 0
    std::cout << "Group: " << i << " Status: " << passed << std::endl;
    #endif
  }
  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  allocate(options);
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average setup and runtime and GFLOPs.
    float elapsed_ms       = timer.elapsed_millis();
    result.avg_runtime_ms  = double(elapsed_ms) / double(options.iterations);
    result.gflops          = options.gflops(result.avg_runtime_ms / 1000.0, options.problem_sizes_host);

    std::cout << "  Problem Sizes: " << std::endl;
    for (auto const & problem : options.problem_sizes_host) {
      std::cout << "    " << problem << std::endl;
    }
    std::cout << "  Groups      : " << options.groups  << std::endl;
    std::cout << "  Alpha, Beta : " << options.alpha << ',' << options.beta << std::endl;
    std::cout << "  Avg runtime : " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS      : " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.3 Toolkit to run this example
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 3)) {
    std::cerr << "This example requires CUDA 12.3 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
