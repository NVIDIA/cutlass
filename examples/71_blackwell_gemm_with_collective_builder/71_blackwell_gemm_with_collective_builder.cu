/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Blackwell SM100 GEMM example demonstrating compatible mainloop+epilogue builder schedules
    and epilogue visitor tree (EVT) construction

    Example usage:
      $ ./examples/71_blackwell_gemm_with_collective_builder/71_blackwell_gemm_with_collective_builder \
            --m=2048 --n=2048 --k=2048 --l=2
*/

#include <iostream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/kernel/tile_scheduler.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

using namespace cute;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;

  int m, n, k, l;
  float alpha, beta;

  Options():
    help(false),
    error(false),
    m(2048), n(2048), k(2048), l(1),
    alpha(1.f), beta(0.f)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m, 2048);
    cmd.get_cmd_line_argument("n", n, 2048);
    cmd.get_cmd_line_argument("k", k, 2048);
    cmd.get_cmd_line_argument("l", l, 1);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "71_blackwell_gemm_with_collective_builder\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  performant kernels targeting NVIDIA's Blackwell architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the L extent (batch count) of the GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

// Wrapper to construct, run, and verify a GEMM. This example showcases CUTLASS's collective
// operation builders by specializing the GEMM on the kernel+epilogue schedule it will use and the
// number of pipeline stages.
template <
  // Type of kernel schedule to generate
  class MainloopScheduleType = cutlass::gemm::collective::KernelScheduleAuto,
  // Type of epilogue schedule to generate
  class EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto,
  // Number of pipeline stages to use
  class StageCountType = cutlass::gemm::collective::StageCountAuto,
  // Do we use custom epilogue visitor tree (EVT) fusion
  bool UseCustomEVT = false
>
struct ExampleRunner {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using LayoutD = cutlass::layout::ColumnMajor;

  using ElementA = cutlass::half_t;
  using ElementB = cutlass::half_t;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::half_t;
  using ElementAccumulator = float;
  using ElementCompute = float;
  using ElementScalar = float;

  using ClusterShapeMNK = Shape<_2,_2,_1>;
  static constexpr bool Use2SmMma =
      // Manually specified 2sm cluster MMA schedule, will error if cluster M is not a multiple of 2
      std::is_same_v<MainloopScheduleType, cutlass::gemm::KernelTmaWarpSpecialized2SmSm100> ||
      // Auto schedule will try to select 2sm cluster MMA based on cluster M
      std::is_same_v<MainloopScheduleType, cutlass::gemm::collective::KernelScheduleAuto> && size<0>(ClusterShapeMNK{}) % 2 == 0;
  // The MNK layout of CTAs within a cluster MMA
  using AtomThrMNK    = std::conditional_t<Use2SmMma, Shape<_2,_1,_1>, Shape<_1,_1,_1>>;
  // The MMA tile used by the mainloop collective. Blackwell 1sm MMA supports up to MMA tile M = 128, 2sm MMA supports up to MMA tile M = 256
  using MmaTileMNK    = std::conditional_t<Use2SmMma, Shape<_256,_128,_64>, Shape<_128,_128,_64>>;
  // The Output tile used by the epilogue collective
  using OutputTileMNK = decltype(shape_div(MmaTileMNK{}, AtomThrMNK{}));

  // 16B alignment lets us use TMA
  static constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
  static constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;
  static constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
  static constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

  static constexpr auto RoundStyle = cutlass::FloatRoundStyle::round_to_nearest;

  // Blackwell fusions for the most part use the same EVT nodes used in Hopper. Most Blackwell EVTs will alias to their Hopper counterparts.
  // EVT nodes new to Blackwell mainly relate to narrow precision scale factor generation and are contained in include/cutlass/epilogue/fusion/sm100_visitor_*.hpp
  // See include/cutlass/epilogue/fusion/sm100_callbacks_tma_warpspecialized.hpp for EVT construction using these new nodes
  // Fusions relating to narrow-precision scale factor generation are demonstrated in example 72b and can only be used in blackwell kernels
  using CustomEVT =  // alpha * acc + beta * C
    cutlass::epilogue::fusion::Sm90EVT<cutlass::epilogue::fusion::Sm90Compute<cutlass::homogeneous_multiply_add, ElementD, ElementCompute, RoundStyle>, // beta * C + (alpha * acc)
      cutlass::epilogue::fusion::Sm90ScalarBroadcast<ElementScalar>, // beta
      cutlass::epilogue::fusion::Sm90SrcFetch<ElementC>, // C
      cutlass::epilogue::fusion::Sm90EVT<cutlass::epilogue::fusion::Sm90Compute<cutlass::multiplies, ElementCompute, ElementCompute, RoundStyle>, // alpha * acc
        cutlass::epilogue::fusion::Sm90ScalarBroadcast<ElementScalar>, // alpha
        cutlass::epilogue::fusion::Sm90AccFetch // acc
      >
    >;

  // As in Hopper, a predefined set of fusion operations are provided in include/cutlass/epilogue/fusion/operations.hpp and can be passed to the epilogue builder
  // Fusions operations supported by the Hopper TMA epilogue will also be supported by the Blackwell TMA epilogue
  // Fusions relating to narrow-precision scale factor generation are demonstrated in example 72b and can only be used in blackwell kernels
  using DefaultOperation = cutlass::epilogue::fusion::LinearCombination<ElementD, ElementCompute, ElementC, ElementScalar, RoundStyle>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      OutputTileMNK, ClusterShapeMNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      EpilogueScheduleType,
      cute::conditional_t<UseCustomEVT, CustomEVT, DefaultOperation>
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, AlignmentA,
      ElementB, LayoutB, AlignmentB,
      ElementAccumulator,
      MmaTileMNK, ClusterShapeMNK,
      cute::conditional_t<cute::is_same_v<StageCountType, cutlass::gemm::collective::StageCountAuto>,
          cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
          StageCountType>,
      MainloopScheduleType
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  using ProblemShapeType = typename Gemm::GemmKernel::ProblemShape;

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  using LayoutTagA = cutlass::gemm::detail::StrideToLayoutTagA_t<StrideA>;
  using LayoutTagB = cutlass::gemm::detail::StrideToLayoutTagB_t<StrideB>;
  using LayoutTagC = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideC>;
  using LayoutTagD = cutlass::gemm::detail::StrideToLayoutTagC_t<StrideD>;

  //
  // Data members
  //

  /// Initialization
  StrideA stride_A;
  StrideB stride_B;
  StrideC stride_C;
  StrideD stride_D;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
  cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
  cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_D;
  cutlass::DeviceAllocation<typename Gemm::ElementD> block_ref_D;

  //
  // Methods
  //

  bool verify(const ProblemShapeType& problem_size, float alpha, float beta) {
    auto [M, N, K, L] = problem_size;

    cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({M, K}));
    cutlass::TensorRef ref_B(block_B.get(), Gemm::LayoutB::packed({K, N}));
    cutlass::TensorRef ref_C(block_C.get(), Gemm::LayoutC::packed({M, N}));
    cutlass::TensorRef ref_D(block_ref_D.get(), Gemm::LayoutD::packed({M, N}));

    cutlass::reference::device::GemmComplex(
          {M, N, K},
          ElementScalar(alpha),
          ref_A,
          cutlass::ComplexTransform::kNone,
          ref_B,
          cutlass::ComplexTransform::kNone,
          ElementScalar(beta),
          ref_C,
          ref_D,
          ElementAccumulator(0),
          L,     // batch_count
          M * K, // batch_stride_A
          K * N, // batch_stride_B
          M * N, // batch_stride_C
          M * N  // batch_stride_D
        );

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

    return passed;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto problem_shape_MNKL = cute::append<4>(problem_size, 1);
    auto [M, N, K, L] = problem_shape_MNKL;

    stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(M, K, L));
    stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(N, K, L));
    stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(M, N, L));
    stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(M, N, L));

    block_A.reset(M * K * L);
    block_B.reset(K * N * L);
    block_C.reset(M * N * L);
    block_D.reset(M * N * L);
    block_ref_D.reset(M * N * L);

    initialize_block(block_A, seed + 2023);
    initialize_block(block_B, seed + 2022);
    initialize_block(block_C, seed + 2021);
  }

  bool run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.m, options.n, options.k, options.l};

    initialize(problem_size);

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      problem_size,
      {block_A.get(), stride_A, block_B.get(), stride_B},
      {{}, // epilogue.thread
       block_C.get(), stride_C, block_D.get(), stride_D},
      hw_info
    };

    // See example 48 for details on custom EVT construction
    if constexpr (UseCustomEVT) {
      arguments.epilogue.thread =
        {    // ternary op : beta * C + (alpha * acc)
          {{options.beta}}, // leaf op+args : beta
          {},               // leaf op+args : C
          {                 // binary op : alpha * acc
            {{options.alpha}}, // leaf op+args : alpha
            {},                // leaf op+args : acc
            {}              // binary args : multiplies
          },                // end binary op
          {} // ternary args : multiply_add
        };   // end ternary op
    }
    // Pre-defined fusions will have flat, named args for user-friendlyness
    else {
      arguments.epilogue.thread.alpha = options.alpha;
      arguments.epilogue.thread.beta = options.beta;
    }

    Gemm gemm_op;

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    // Run the GEMM
    status = gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return false;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Verify that the result is correct
    bool passed = verify(problem_size, options.alpha, options.beta);
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
    }

    return passed;
  }

};

#endif // defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, bool passed) {
  std::cout << description << ": " << (passed ? "Passed" : "Failed") << std::endl;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 8)) {
    std::cerr << "This example requires CUDA 12.8 or newer." << std::endl;
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
  
  if (!(props.major == 10 && props.minor == 0)) {
    std::cerr << "This example requires a GPU of NVIDIA's Blackwell architecture (compute capability 100)." << std::endl;
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  bool passed;

  // Auto mainloop and epilogue schedules must be used together to guarantee functionality
  ExampleRunner<> runner_0;
  passed = runner_0.run(options, hw_info);
  print_result("KernelScheduleAuto mainloop schedule with EpilogueScheduleAuto epilogue schedule", passed);

  // Mainloop stage counts can be specified manually
  // It is the user's responsibility to ensure there is enough device smem to allocate manual stage counts
  ExampleRunner<
    cutlass::gemm::collective::KernelScheduleAuto,
    cutlass::epilogue::collective::EpilogueScheduleAuto,
    _3> runner_1;
  passed = runner_1.run(options, hw_info);
  print_result("KernelScheduleAuto mainloop schedule with EpilogueScheduleAuto epilogue schedule and 3 mainloop stages", passed);

  // 1SM cluster MMA mainloop schedules can be used with direct store ("no-smem") epilogue schedules
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized1SmSm100, cutlass::epilogue::NoSmemWarpSpecialized> runner_2;
  passed = runner_2.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized1SmSm100 mainloop schedule with NoSmemWarpSpecialized epilogue schedule", passed);

  // 1SM cluster MMA mainloop schedules can also be used with 1SM TMA epilogue schedules
  // 1SM cluster MMA mainloop schedules will not work with 2SM TMA epilogue schedules
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized1SmSm100, cutlass::epilogue::TmaWarpSpecialized1Sm> runner_3;
  passed = runner_3.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized1SmSm100 mainloop schedule with NoSmemWarpSpecialized epilogue schedule", passed);

  // 2SM cluster MMA mainloop schedules can be used with direct store ("no-smem") epilogue schedules
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized2SmSm100, cutlass::epilogue::NoSmemWarpSpecialized> runner_4;
  passed = runner_4.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized2SmSm100 mainloop schedule with NoSmemWarpSpecialized epilogue schedule", passed);

  // 2SM cluster MMA mainloop schedules can also be used with 2SM TMA epilogue schedules
  // 2SM cluster MMA mainloop schedules will not work with SM TMA epilogue schedules
  ExampleRunner<cutlass::gemm::KernelTmaWarpSpecialized2SmSm100, cutlass::epilogue::TmaWarpSpecialized2Sm> runner_5;
  passed = runner_5.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized2SmSm100 mainloop schedule with TmaWarpSpecialized2Sm epilogue schedule", passed);

  // Blackwell Auto schedule supports custom EVT fusions
  constexpr bool UseCustomEVT = true;
  ExampleRunner<
    cutlass::gemm::collective::KernelScheduleAuto,
    cutlass::epilogue::collective::EpilogueScheduleAuto,
    cutlass::gemm::collective::StageCountAuto,
    UseCustomEVT> runner_6;
  passed = runner_6.run(options, hw_info);
  print_result("KernelScheduleAuto mainloop schedule with EpilogueScheduleAuto epilogue schedule and custom EVT", passed);

  // 1SM TMA epilogue schedules support custom EVT fusions
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecialized1SmSm100,
    cutlass::epilogue::TmaWarpSpecialized1Sm,
    cutlass::gemm::collective::StageCountAuto,
    UseCustomEVT> runner_7;
  passed = runner_7.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized1SmSm100 mainloop schedule with TmaWarpSpecialized1Sm epilogue and custom EVT", passed);

  // 2SM TMA epilogue schedules support custom EVT fusions
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecialized2SmSm100,
    cutlass::epilogue::TmaWarpSpecialized2Sm,
    cutlass::gemm::collective::StageCountAuto,
    UseCustomEVT> runner_8;
  passed = runner_8.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized2SmSm100 mainloop schedule with TmaWarpSpecialized2Sm epilogue and custom EVT", passed);


  // Blackwell direct store epilogue schedule supports custom EVTs and named fusion operations as well (not supported for pre-Blackwell kernels)
  ExampleRunner<
    cutlass::gemm::KernelTmaWarpSpecialized1SmSm100,
    cutlass::epilogue::NoSmemWarpSpecialized,
    cutlass::gemm::collective::StageCountAuto,
    UseCustomEVT> runner_9;
  passed = runner_9.run(options, hw_info);
  print_result("KernelTmaWarpSpecialized1SmSm100 mainloop schedule with NoSmemWarpSpecialized epilogue and custom EVT", passed);

#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
