/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper Ptr-Array Batched GEMM example using CUTLASS 3 APIs for NVIDIA Hopper architecture.

    This example demonstrates an implementation of Ptr-Array Batched GEMM using a TMA + GMMA 
    warp-specialized cooperative kernel.
    The new feature showcased in this example is on-the-fly modification of TMA descriptors
    to move between batches (represented by l).

    To run this example:

      $ ./examples/56_hopper_ptr_array_batched_gemm/56_hopper_ptr_array_batched_gemm --m=2048 --n=2048 --k=2048 --l=10
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::half_t;                                // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::half_t;                                // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::ColumnMajor;                   // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_256,_128,_64>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_1,_2,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::KernelArrayTmaWarpSpecializedCooperative; // Kernel to launch
using EpilogueSchedule = cutlass::epilogue::NoSmemWarpSpecializedArray;         // Epilogue to launch

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    ElementC, LayoutC, AlignmentC,
    ElementC, LayoutC, AlignmentC,
    EpilogueSchedule
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    cutlass::gemm::ArrayProblemShape<Shape<int,int,int,int>>,
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

std::vector<int64_t> offset_A;
std::vector<int64_t> offset_B;
std::vector<int64_t> offset_C;
std::vector<int64_t> offset_D;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

cutlass::DeviceAllocation<const typename Gemm::ElementA *> ptr_A;
cutlass::DeviceAllocation<const typename Gemm::ElementB *> ptr_B;
cutlass::DeviceAllocation<const typename Gemm::ElementC *> ptr_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  float alpha = 1.0f;
  float beta = 0.0f;
  int iterations = 10;
  int m = 1024, n = 512, k = 1024, l = 10;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "56_hopper_ptr_array_batched_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the batch count for Ptr-Array GEMM\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "56_hopper_ptr_array_batched_gemm" << " --m=1024 --n=512 --k=1024 --l=10 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k * l;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms = 0.0;
  double gflops = 0.0;
  cutlass::Status status = cutlass::Status::kSuccess;
  hipError_t error = hipSuccess;
  bool passed = false;
};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

/// Allocates device-side data
void allocate(const Options &options) {
  int64_t total_elements_A = 0;
  int64_t total_elements_B = 0;
  int64_t total_elements_C = 0;
  int64_t total_elements_D = 0;

  for (int32_t i = 0; i < options.l; ++i) {

    offset_A.push_back(total_elements_A);
    offset_B.push_back(total_elements_B);
    offset_C.push_back(total_elements_C);
    offset_D.push_back(total_elements_D);

    int64_t elements_A = options.m * options.k;
    int64_t elements_B = options.k * options.n;
    int64_t elements_C = options.m * options.n;
    int64_t elements_D = options.m * options.n;

    total_elements_A += elements_A;
    total_elements_B += elements_B;
    total_elements_C += elements_C;
    total_elements_D += elements_D;
  }

  block_A.reset(total_elements_A);
  block_B.reset(total_elements_B);
  block_C.reset(total_elements_C);
  block_D.reset(total_elements_D);
  block_ref_D.reset(total_elements_D);
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.m, options.n, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));

  //
  // Assign pointers
  //

  std::vector<ElementA *> ptr_A_host(options.l);
  std::vector<ElementB *> ptr_B_host(options.l);
  std::vector<ElementC *> ptr_C_host(options.l);
  std::vector<ElementC *> ptr_D_host(options.l);

  for (int32_t i = 0; i < options.l; ++i) {
    ptr_A_host.at(i) = block_A.get() + offset_A.at(i);
    ptr_B_host.at(i) = block_B.get() + offset_B.at(i);
    ptr_C_host.at(i) = block_C.get() + offset_C.at(i);
    ptr_D_host.at(i) = block_D.get() + offset_D.at(i);
  }

  ptr_A.reset(options.l);
  ptr_A.copy_from_host(ptr_A_host.data());

  ptr_B.reset(options.l);
  ptr_B.copy_from_host(ptr_B_host.data());

  ptr_C.reset(options.l);
  ptr_C.copy_from_host(ptr_C_host.data());

  ptr_D.reset(options.l);
  ptr_D.copy_from_host(ptr_D_host.data());

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_block(block_C, seed + 2021);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  cutlass::KernelHardwareInfo hw_info;
  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kArray,
    {{options.m, options.n, options.k, options.l}},
    {ptr_A.get(), stride_A, ptr_B.get(), stride_B},
    {{options.alpha, options.beta}, ptr_C.get(), stride_C, ptr_D.get(), stride_D},
    hw_info
  };

  return arguments;
}

bool verify(const Options &options) {
  bool passed = true;
  for (int32_t i = 0; i < options.l; ++i) {
    cutlass::TensorRef ref_A(block_A.get() + offset_A.at(i), Gemm::LayoutA::packed({options.m, options.k}));
    cutlass::TensorRef ref_B(block_B.get() + offset_B.at(i), Gemm::LayoutB::packed({options.k, options.n}));
    cutlass::TensorRef ref_C(block_C.get() + offset_C.at(i), Gemm::LayoutC::packed({options.m, options.n}));
    cutlass::TensorRef ref_D(block_ref_D.get() + offset_D.at(i), Gemm::LayoutD::packed({options.m, options.n}));

    //
    // Compute reference output
    //

    // Create instantiation for device reference gemm kernel
    DeviceGemmReference gemm_reference;

    // Launch device reference gemm kernel
    gemm_reference(
      {options.m, options.n, options.k},
      ElementAccumulator(options.alpha),
      ref_A,
      ref_B,
      ElementAccumulator(options.beta),
      ref_C,
      ref_D);

    // Wait for kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    passed &= cutlass::reference::device::BlockCompareEqual(block_ref_D.get() + offset_D.at(i), block_D.get() + offset_D.at(i), options.m * options.n);
  }
  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  allocate(options);
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average setup and runtime and GFLOPs.
    float elapsed_ms       = timer.elapsed_millis();
    result.avg_runtime_ms  = double(elapsed_ms) / double(options.iterations);
    result.gflops          = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Batches     : " << options.l  << std::endl;
    std::cout << "  Alpha, Beta : " << options.alpha << ',' << options.beta << std::endl;
    std::cout << "  Avg runtime : " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS      : " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.3 Toolkit to run this example
  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 3)) {
    std::cerr << "This example requires CUDA 12.3 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
