#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example with different data types using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example shows how to perform INT4 x FP8 GEMM and scale up the INT4 weight during dequantization. It uses a look-up table to avoid the multiplications
    between INT4 and FP8. To trigger this method, use cutlass::Array<ElementScale, 8> as the scale type in the collective's arguments.
    
    However, this algorithm requires changes to the encoding of INT4 weights and scale factors. These changes must happen before launching the GEMM. See the helper functions
    `unify_quant_encoding`, `initialize_packed_scale`, and header `fp8_packed_scale.hpp` for details.

    In a nutshell, the positive values of INT4 weights need to be encoded in the same way as negative values except for the sign bit. For each scale factor,
    8 negative results (-8 x scale, -7 x scale, ... -1 x scale) are packed together, forming a cutlass::Array<ElementScale, 8> value.

    The narrower type always passes through the register file. Therefore, in cases where the narrower type is operand B, the collective will implicitly swap 
    A and B in the main loop. However, as a result of this collective performing implicit swaps, it does not support TMA epilogues. Consequently, it is essential to consider this when constructing the epilogue, 
    as illustrated in this example.

    Note that in this example, we explicitly swap A and B in order to use TMA epilogues. We do this since TMA epilogues are more performant on problem sizes of interest.

    As an additional optimization, we can reorder the narrow data type tensor such that elements read into register file by the same thread are contiguous in global and shared memory.
    This promotes vectorization of shared memory loads and removes additional instructions on the critical path. For example, when MMA is performed in FP8 data type, each thread reads
    4 groups of 4 elements that are logically contiguous in the same row (refer to https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#wgmma-64n32-a for thread-value layout).
    If the narrow type is INT4 and tensor is major in K dim, only 16 bits can be read at a time, leading to extra load instructions and suboptimal utilization of shared memory throughput.
    If we reorder the data offline to place all 16 elements read by a thread contiguously in memory, a single 64-bit load is sufficient. This reordering is often feasible when the quantized
    tensor is static (e.g. weight tensor of a NN layer at inference time). This example demonstrates how such a reordering can be performed and communicated to the kernel when the macro
    OPTIMIZE_WEIGHT_LAYOUT is set to 1.

    It is expected that the scale's K dimension be scale_k = ceil_div(problem_k, group_size). 
    
    Scales are always expected to be MN major. This means the fastest changing dimension must be M if A is scaled or N if B is scaled.
    
    If A is being scaled, the scales must have shape [M, scale_k],  while if B is scaled, it must have shape [N, scale_k].

    The implementation only supports "group-wise" scales. However, we can make it work for per-column scales by setting the group's size
    equal to the gemm problem K.

    Limitations:
      1) Only supports INT4 x { FP8, INT8, UINT8 }. The scales must be the same as mma Type. Scale with zero-point mode is not supported.
      2) The INT4 weights and scale factors have additional encoding requirements.
      3) The scales must be MN major. That means if A is scaled, it must be column major, but if B is scaled it must be row major.
      4) The scales must have the same layout and groupsize.
      5) The groupsize must be greater or equal to the tile shape k.
      6) Currently, TMA epilogues cannot be used when the narrow type is the B operand. This limitation arises because the implementation always swaps the 
         operands to ensure that the narrow type passes through the register file, and TMA epilogues do not currently support implicit swap + transpose operations. 
         We plan to address this limitation in the future. However, we address this in the example by explicitly swapping and transposing the operands.
    
    Optimizing suggestions:
      1) Use a small tile size, since the register pressure for this GEMM (and RS GEMM in general) is high (it uses a lot of register space).

    Examples:
      
      Runs the mixed input batched gemm (with batch size 2), converting B to the type of A (mode 0)
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_int4_fp8_gemm --m=2048 --n=2048 --k=2048 --l=2 --mode=0

      Runs the mixed input gemm, and applies a scaling factor to B before mma (mode 1). Applies a vector of scales to the entire
      matrix (group size is the same as the gemm k dimension).
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_int4_fp8_gemm --m=4096 --n=5120 --k=8192 --g=8192 --mode=1
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_compare.h"

#include "helper.h"
#include "unfused_weight_dequantize.hpp"
#include "packed_scale.hpp"
#include "reorder_utils.hpp"

using namespace cute;

#define OPTIMIZE_WEIGHT_LAYOUT 1

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
using MmaType = cutlass::float_e4m3_t;
using QuantType = cutlass::int4b_t;
constexpr int TileShapeK = 128 * 8 / sizeof_bits<MmaType>::value;

// A matrix configuration
using         ElementA    = MmaType;                                        // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = QuantType;                                      // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// This example manually swaps and transposes, so keep transpose of input layouts
using LayoutA_Transpose = typename cutlass::layout::LayoutTranspose<LayoutA>::type;
using LayoutB_Transpose = typename cutlass::layout::LayoutTranspose<LayoutB>::type;

using StrideA = cutlass::detail::TagToStrideA_t<LayoutA>;
using StrideB = cutlass::detail::TagToStrideB_t<LayoutB>;

#if OPTIMIZE_WEIGHT_LAYOUT
// Define the CuTe layout for reoredered quantized tensor B
// LayoutAtomQuant places values that will be read by the same thread in contiguous locations in global memory.
// It specifies the reordering within a single warp's fragment
using LayoutAtomQuant = decltype(compute_memory_reordering_atom<MmaType>());
using LayoutB_Reordered = decltype(tile_to_shape(LayoutAtomQuant{}, Layout<Shape<int,int,int>, StrideB>{}));
#endif

using ElementScale = MmaType;
using ElementZero = ElementScale; // only for verify
using LayoutScale = cutlass::layout::RowMajor;

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_128,cute::Int<TileShapeK>>;         // Threadblock-level tile size
using ClusterShape        = Shape<_1,_1,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::KernelTmaWarpSpecializedCooperativeMixedInput;  // Kernel to launch based on the default setting in the Collective Builder 
using EpilogueSchedule    = cutlass::epilogue::TmaWarpSpecializedCooperative;
using EpilogueTileType    = cutlass::epilogue::collective::EpilogueTileAuto;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementAccumulator,
    // Transpose layout of D here since we use explicit swap + transpose
    // the void type for C tells the builder to allocate 0 smem for the C matrix.
    // We can enable this if beta == 0 by changing ElementC to void below.
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    ElementD, typename cutlass::layout::LayoutTranspose<LayoutD>::type, AlignmentD,
    EpilogueSchedule // This is the only epi supporting the required swap + transpose.
  >::CollectiveOp;

// =========================================================== MIXED INPUT WITH SCALES ===========================================================================
// The Scale information must get paired with the operand that will be scaled. In this example, B is scaled so we make a tuple of B's information and the scale information.
using CollectiveMainloopScaleOnly = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
#if OPTIMIZE_WEIGHT_LAYOUT
    cute::tuple<ElementB, cutlass::Array<ElementScale, 8>>, LayoutB_Reordered, AlignmentB,
#else
    cute::tuple<ElementB, cutlass::Array<ElementScale, 8>>, LayoutB_Transpose, AlignmentB,
#endif
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleOnly = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleOnly,
    CollectiveEpilogue
>;

using GemmScaleOnly = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleOnly>;

using StrideC = typename GemmKernelScaleOnly::StrideC;
using StrideD = typename GemmKernelScaleOnly::StrideD;

using StrideC_ref = cutlass::detail::TagToStrideC_t<LayoutC>;
using StrideD_ref = cutlass::detail::TagToStrideC_t<LayoutD>;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideC_ref stride_C_ref;
StrideD stride_D;
StrideD_ref stride_D_ref;
uint64_t seed;

#if OPTIMIZE_WEIGHT_LAYOUT
LayoutB_Reordered layout_B_reordered;
#endif

using StrideS = typename CollectiveMainloopScaleOnly::StrideScale;
using StrideS_ref = cutlass::detail::TagToStrideB_t<LayoutScale>;
StrideS stride_S;
StrideS_ref stride_S_ref;

cutlass::DeviceAllocation<ElementA> block_A;
cutlass::DeviceAllocation<ElementB> block_B;
cutlass::DeviceAllocation<ElementB> block_B_modified;
cutlass::DeviceAllocation<ElementA> block_B_dq;
cutlass::DeviceAllocation<ElementScale> block_scale;
cutlass::DeviceAllocation<cutlass::Array<ElementScale, 8>> block_scale_packed;
cutlass::DeviceAllocation<ElementZero> block_zero;
cutlass::DeviceAllocation<ElementC> block_C;
cutlass::DeviceAllocation<typename GemmScaleOnly::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename GemmScaleOnly::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  float alpha = 1.0f;
  float beta = 0.0f;
  int iterations = 10;
  int m = 5120, n = 4096, k = 4096;
  int g = 128;
  int l = 1;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("g", g);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "55_hopper_warp_specialized_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   The number of independent gemm problems with mnk shape\n"
      << "  --g=<int>                   The size of each group for the scales. To broadcast a vector of scales or zeros, set the group size to K.\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "55_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 -g 0 --l=10 --alpha=2 --mode=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k * l;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms = 0.0;
  double gflops = 0.0;
  cutlass::Status status = cutlass::Status::kSuccess;
  hipError_t error = hipSuccess;
  bool passed = false;

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_tensor(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  double scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;
  int bits_output = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  }
  else if (bits_output == 16) {
    scope_max = 5;
    scope_min = -5;
  }
  else {
    scope_max = 8;
    scope_min = -8;
  }
  cutlass::reference::device::BlockFillRandomUniform(
      block.get(), block.size(), seed, Element(scope_max), Element(scope_min));

  return true;
}

template <typename Element>
bool initialize_quant_tensor(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {
  
  float scope_min = float(cutlass::platform::numeric_limits<Element>::lowest());
  float scope_max = float(cutlass::platform::numeric_limits<Element>::max());

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, Element(scope_max), Element(scope_min));

  return true;
}

// In the mainloop, PRMT selects 1 byte from only 8 bytes so the sign bit is handled in an extra PRMT.
// Here the encodings of positive values and negative values are unified (except for the sign bit). 
// For instance, 1 becomes 0b0111, which is the same encoding as -1 (0b1111).
bool unify_quant_encoding(
  cutlass::DeviceAllocation<cutlass::int4b_t> const& block_in,
  cutlass::DeviceAllocation<cutlass::int4b_t>& block_out) {

  using StorageType = cutlass::int4b_t::Storage;

  if (block_in.size() != block_out.size()) {
    std::cerr << "block_in and block_out must have same size.\n";
    return false;
  }
  constexpr int pack = sizeof_bits_v<StorageType> / 4;
  std::vector<StorageType> data(block_in.size() / pack);
  cutlass::device_memory::copy_to_host(data.data(), (StorageType*)block_in.get(), block_in.size() / pack);

  for (auto&& d : data) {
    StorageType out = 0;
    StorageType mask = 0x0f;
    for (int i = 0; i < pack; ++i) {
      cutlass::int4b_t curr;
      curr.storage = (d >> (i * 4)) & 0x0f;
      switch (curr) {
        case 1: curr.storage = StorageType(0b0111); break; // 2's complement
        case 2: curr.storage = StorageType(0b0110); break; // 2's complement
        case 3: curr.storage = StorageType(0b0101); break; // 2's complement
        case 4: curr.storage = StorageType(0b0100); break; // 2's complement
        case 5: curr.storage = StorageType(0b0011); break; // 2's complement
        case 6: curr.storage = StorageType(0b0010); break; // 2's complement
        case 7: curr.storage = StorageType(0b0001); break; // 2's complement
        default: break;
      }
      out |= (curr.storage << (4 * i)) & mask;
      mask <<= 4;
    }
    d = out;
  }

  cutlass::device_memory::copy_to_device((StorageType*)block_out.get(), data.data(), block_out.size() / pack);
  return true;
}

template <class Element>
bool initialize_scale(
  cutlass::DeviceAllocation<Element>& block, 
  Options const& options) {
  
  float elt_max_f = float(cutlass::platform::numeric_limits<QuantType>::max());
  float const max_dequant_val = 4.f;
  float const min_dequant_val = 0.5f;

  float scope_max(max_dequant_val / elt_max_f);
  float scope_min(min_dequant_val / elt_max_f);

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, Element(scope_max), Element(scope_min));
  return true;
}

bool initialize_packed_scale(
  cutlass::DeviceAllocation<ElementScale> const& block_in, 
  cutlass::DeviceAllocation<cutlass::Array<ElementScale, 8> > & block_out) {
  
  std::vector<ElementScale> data_in(block_in.size());
  std::vector<cutlass::Array<ElementScale, 8> > data_out(block_in.size());
  try {
    block_in.copy_to_host(data_in.data());
  } catch (cutlass::cuda_exception const& e)
  {
    std::cerr << "CUDA Error: " << hipGetErrorString(e.hipError_t()) << std::endl;
    return false;
  }
  for (size_t i = 0; i < block_in.size(); ++i)
  {
    cutlass::packed_scale_t<ElementScale> tmp(data_in[i]);
    data_out[i] = reinterpret_cast<cutlass::Array<ElementScale, 8> const&>(tmp);
    // std::cout << data_in[i] << ":" << std::hex << static_cast<uint16_t>(data_in[i].storage) << ",\t" << -data_in[i] << ":" << std::hex << static_cast<uint16_t>((-data_in[i]).storage) << std::endl;
  }
  try {
    block_out.copy_from_host(data_out.data());
  } catch (cutlass::cuda_exception const& e)
  {
    std::cerr << "CUDA Error: " << hipGetErrorString(e.hipError_t()) << std::endl;
    return false;
  }
  return true;
}

template <class Element>
bool initialize_zero(
  cutlass::DeviceAllocation<Element>& block,
  Options const& options) {
  std::vector<Element> stage(block.size(), Element(0.0f));
  block.copy_from_host(stage.data());
  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(Options const& options) {

  auto shape_B = cute::make_shape(options.n, options.k, options.l);
  int const scale_k = (options.k + options.g - 1) / options.g;
  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, shape_B);
  // Reverse stride here due to swap and transpose
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.n, options.m, options.l));
  stride_C_ref = cutlass::make_cute_packed_stride(StrideC_ref{}, cute::make_shape(options.m, options.n, options.l));
  // Reverse stride here due to swap and transpose
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.n, options.m, options.l));
  stride_D_ref = cutlass::make_cute_packed_stride(StrideD_ref{}, cute::make_shape(options.m, options.n, options.l));

  auto layout_B = make_layout(shape_B, stride_B);

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);

  block_A.reset(a_coord.product());
  block_B.reset(b_coord.product());
  block_B_modified.reset(b_coord.product());
  block_B_dq.reset(b_coord.product());
  block_C.reset(c_coord.product());
  block_D.reset(c_coord.product());
  block_ref_D.reset(c_coord.product());

  block_scale.reset(scale_k * options.l * options.n);
  block_scale_packed.reset(scale_k * options.l * options.n);
  block_zero.reset(scale_k * options.l * options.n);

  initialize_tensor(block_A, seed + 2022);
  initialize_quant_tensor(block_B, seed + 2021);
  unify_quant_encoding(block_B, block_B_modified);
  initialize_tensor(block_C, seed + 2020);
  initialize_scale(block_scale, options);
  initialize_packed_scale(block_scale, block_scale_packed);
  initialize_zero(block_zero, options);

  auto shape_scale_zero = cute::make_shape(options.n, scale_k, options.l);
  stride_S = cutlass::make_cute_packed_stride(StrideS{}, cute::make_shape(options.n, scale_k, options.l));
  stride_S_ref = cutlass::make_cute_packed_stride(StrideS_ref{}, cute::make_shape(options.n, scale_k, options.l));
  auto layout_scale_zero = make_layout(shape_scale_zero, stride_S_ref);

  dequantize_weight(block_B_dq.get(), block_B.get(), layout_B, block_scale.get(), block_zero.get(), layout_scale_zero, options.g);

  #if OPTIMIZE_WEIGHT_LAYOUT
  // Repeat the reorder layout atom to tile the whole tensor shape 
  layout_B_reordered = tile_to_shape(LayoutAtomQuant{}, shape_B);
  reorder_tensor(block_B_modified.get(), layout_B, layout_B_reordered);

  print("Quantized tensor layout: ");
  print(layout_B_reordered);
  print("\n");
#endif
}

/// Populates a Gemm::Arguments structure from the given commandline options
template <typename Args>
Args args_from_options(Options const& options)
{
// Swap the A and B tensors, as well as problem shapes here.
  
  return Args {
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.n, options.m, options.k, options.l},
#if OPTIMIZE_WEIGHT_LAYOUT
    {block_B_modified.get(), layout_B_reordered, block_A.get(), stride_A, block_scale_packed.get(), stride_S, options.g},
#else
    {block_B_modified.get(), stride_B,           block_A.get(), stride_A, block_scale_packed.get(), stride_S, options.g},
#endif
    {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
  };
}

bool verify(Options const& options) {
  //
  // Compute reference output
  //

  // In this example, we use the GPU default kernels as a reference (unfused scale).
  // This avoids numerical differences due to different accumulation order.

  // Again, due to numerical differences, we must use fast acc here when the mma type is
  // FP8 as the fused implementation only supports fast acc at the moment.
  constexpr bool IsFP8Input = cute::is_same_v<MmaType, cutlass::float_e4m3_t> || cute::is_same_v<MmaType, cutlass::float_e5m2_t>;
  using FP8Sched = cute::conditional_t<size<0>(TileShape{}) == 64, cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum, cutlass::gemm::KernelTmaWarpSpecializedCooperativeFP8FastAccum>;
  using ScheduleRef = cute::conditional_t<IsFP8Input, FP8Sched, cutlass::gemm::collective::KernelScheduleAuto>;

  using CollectiveMainloopRef = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      MmaType, LayoutA, AlignmentA,
      MmaType, LayoutB, AlignmentB,
      ElementAccumulator,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAuto,
      ScheduleRef
    >::CollectiveOp;

  using CollectiveEpilogueRef = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementAccumulator,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      cutlass::epilogue::NoSmemWarpSpecialized
    >::CollectiveOp;

  using GemmKernelRef = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>, // Indicates ProblemShape
      CollectiveMainloopRef,
      CollectiveEpilogueRef
  >;

  using GemmRef = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelRef>;

  typename GemmRef::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {block_A.get(), stride_A, block_B_dq.get(), stride_B},
    {{options.alpha, options.beta}, block_C.get(), stride_C_ref, block_ref_D.get(), stride_D_ref}
  };

  // Run the gemm where the scaling is performed outside of the kernel.
  GemmRef gemm_ref;
  size_t workspace_size = GemmRef::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
  CUTLASS_CHECK(gemm_ref.can_implement(arguments));
  CUTLASS_CHECK(gemm_ref.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(gemm_ref.run());

  // compare_reference
  ElementD const epsilon(1e-2f);
  ElementD const non_zero_floor(1e-4f);
  bool passed = cutlass::reference::device::BlockCompareRelativelyEqual(block_ref_D.get(), block_D.get(), block_D.size(), epsilon, non_zero_floor);

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<typename Gemm::Arguments>(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }
  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  if (options.g == options.k) {
    std::cout << "Running in per-column scale mode." << std::endl;
  } else {
    std::cout << "Running in group scale mode." << std::endl;
  }
  run<GemmScaleOnly>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
