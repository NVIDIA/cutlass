#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example with different data types using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example shows how to perform GEMM where the input tensors A and B have different element types. CUTLASS currently supports upcasting
    from a narrower (fewer bits) to a wider (more bits) type and utilizing the tensor core instruction for the wider type. For instance, when doing
    INT8 x FP16, CUTLASS will convert INT8 -> FP16 and do math using FP16 tensor cores. Similarly, for INT4 x INT8, it will upcast to INT8 and issue math 
    using INT8 tensor cores.

    The narrower type always passes through the register file. Therefore, in cases where the narrower type is operand B, the collective will implicitly swap 
    A and B in the main loop. Consequently, it is essential to consider this when constructing the epilogue, as illustrated in this example.

    Limitations:
      1) Only supported combinations are 16-bit x {8-bit, 4-bit, 2-bit} and {8-bit} x {4-bit, 2-bit}.
      2) The narrow type must always be in K-major format.
      3) When dealing with 8-bit x {4-bit, 2-bit}, both inputs must be in K-major format.
      4) Currently, TMA epilogues cannot be used when the narrow type is the B operand. This limitation arises because the implementation always swaps the 
         operands to ensure the narrow type passes through the register file, and TMA epilogues do not currently support swap + transpose operations. 
         We plan to address this limitation in the future.

    Examples:

      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_mixed_dtype_gemm --m=2048 --n=2048 --k=2048 --l=2
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"

#include "helper.h"
#include "unfused_weight_dequantize.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
using MmaType = cutlass::half_t;
using QuantType = int8_t;

// A matrix configuration
using         ElementA    = MmaType;                                // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = QuantType;                                      // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_256,_64>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_2,_1,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::collective::KernelScheduleAuto;       // Kernel to launch based on the default setting in the Collective Builder 


using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAuto,
    cutlass::gemm::collective::KernelScheduleAuto
  >::CollectiveOp;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    // Lie here about layout of C and D since we do swap and transpose trick
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    cutlass::epilogue::NoSmemWarpSpecialized // This is the only epi supporting the required swap + transpose.
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

// Initialization functions don't handle sub-byte types so we use uint8 to initialize and a separate
// kernel to pack the data if it is necessary.
using InitializationType = cute::conditional_t<cute::sizeof_bits_v<QuantType> < 8, uint8_t, QuantType>;

cutlass::HostTensor<typename Gemm::ElementA, LayoutA> tensor_A;
cutlass::HostTensor<InitializationType, LayoutB> tensor_B_init;
cutlass::HostTensor<typename Gemm::ElementB, LayoutB> tensor_B;
cutlass::HostTensor<typename Gemm::ElementA, LayoutB> tensor_B_dq;
cutlass::HostTensor<typename Gemm::ElementC, LayoutC> tensor_C;
cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, LayoutD> tensor_D;
cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, LayoutD> tensor_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  float alpha = 1.0f;
  float beta = 0.0f;
  int iterations = 1000;
  int m = 5120, n = 4096, k = 4096;
  int l = 1;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "55_hopper_warp_specialized_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   The number of independent gemm problems with mnk shape\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "55_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 --l=10 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k * l;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms = 0.0;
  double gflops = 0.0;
  cutlass::Status status = cutlass::Status::kSuccess;
  hipError_t error = hipSuccess;
  bool passed = false;

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element, class Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed=2023) {

  double scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;
  int bits_output = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  }
  else if (bits_output == 16) {
    scope_max = 5;
    scope_min = -5;
  }
  else {
    scope_max = 8;
    scope_min = -8;
  }
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min, 0);

  return true;
}

template <class QuantElement, typename Element, typename Layout>
bool initialize_quant_tensor(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed=2023) {
  
  Element scope_max, scope_min;
  constexpr int bits_input = cute::sizeof_bits_v<QuantElement>;
  static_assert(bits_input <= 8, "Quantization type can be at most 8 bits");

  if constexpr (bits_input == 8) {
    // Directly init 1-byte types
    static_assert(cute::is_same_v<QuantElement, Element>, "Init type should equal quant type for 1 byte types");
    scope_max = std::numeric_limits<QuantElement>::max();
    scope_min = std::numeric_limits<QuantElement>::min();
  } else {
    static_assert(cute::is_same_v<uint8_t, Element>, "Init type should be uint8_t for sub-byte types");
    scope_max = (1 << bits_input);
    scope_min = 0;
  }

  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min, 0);

  return true;
}

template <class Element, class Layout>
bool initialize_with_one(
  cutlass::TensorView<Element, Layout> view) {
  cutlass::reference::host::TensorFill(view, Element(1.0f));
  return true;
}

template <class ElementDst, class ElementSrc, class Layout, class L>
void prepare_packed_data(cutlass::HostTensor<ElementDst, Layout> view_dst_data,
                         cutlass::HostTensor<ElementSrc, Layout> view_src_data,
                         const L& cute_layout) {
  if constexpr (cute::is_same_v<ElementSrc, ElementDst>) {
    view_dst_data.copy_in_device_to_device(view_src_data.device_data());
  } 
  else {
    pack_data(view_dst_data.device_data(), view_src_data.device_data(), cute_layout);
  }
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  auto shape_b = cute::make_shape(options.n, options.k, options.l);
  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, shape_b);
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.n, options.m, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.n, options.m, options.l));

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);

  tensor_A.resize(a_coord);
  tensor_B_init.resize(b_coord);
  tensor_B.resize(b_coord);
  tensor_B_dq.resize(b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  // We need scales since the "dequantize" kernels expects them. We just set them to 1 so the values get converted
  // to the mma type.
  cutlass::HostTensor<MmaType, cutlass::layout::RowMajor> tensor_scale;
  tensor_scale.resize({1 * options.l, options.n});

  initialize_tensor(tensor_A.host_view(), seed + 2022);
  initialize_quant_tensor<QuantType>(tensor_B_init.host_view(), seed + 2021);
  initialize_tensor(tensor_C.host_view(), seed + 2020);
  initialize_with_one(tensor_scale.host_view());

  tensor_A.sync_device();
  tensor_B_init.sync_device();
  tensor_C.sync_device();
  tensor_scale.sync_device();

  auto layout_B = make_layout(shape_b, stride_B);
  prepare_packed_data(tensor_B, tensor_B_init, layout_B);

  auto shape_scale = cute::make_shape(options.n, 1, options.l);
  auto layout_scale = make_layout(shape_scale);
  dequantize_weight(tensor_B_dq.device_data(), tensor_B.device_data(), layout_B, tensor_scale.device_data(), layout_scale);

  tensor_B.sync_host();
  tensor_B_dq.sync_host();
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(), stride_A, tensor_B.device_data(), stride_B},
    {{options.alpha, options.beta}, tensor_C.device_data(), stride_C, tensor_D.device_data(), stride_D}
  };

  return arguments;
}

bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.k, options.l), stride_A));
  auto B = cute::make_tensor(tensor_B_dq.host_data(),
      cute::make_layout(cute::make_shape(options.n, options.k, options.l), stride_B));
  auto C = cute::make_tensor(tensor_C.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_C));
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_D));

  using unused_t = decltype(D);

  cutlass::reference::host::GettMainloopParams<ElementAccumulator, decltype(A), decltype(B)> mainloop_params{A, B};

  cutlass::reference::host::GettEpilogueParams<
      typename Gemm::EpilogueOutputOp::ElementScalar,
      typename Gemm::EpilogueOutputOp::ElementScalar,
      ElementAccumulator,
      ElementCompute,
      decltype(C),
      decltype(D),
      unused_t, // bias
      unused_t, // aux
      unused_t, // valpha
      unused_t  // vbeta
  > epilogue_params;

  epilogue_params.C = C;
  epilogue_params.D = D;
  epilogue_params.alpha = options.alpha;
  epilogue_params.beta = options.beta;

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  // compare_reference
  tensor_D.sync_host();
  bool passed = cutlass::reference::host::TensorEquals(tensor_ref_D.host_view(), tensor_D.host_view());
  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
