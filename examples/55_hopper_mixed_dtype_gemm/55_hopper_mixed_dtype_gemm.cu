#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example with different data types using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example shows how to perform GEMM where the input tensors A and B have different element types. CUTLASS currently supports upcasting
    from a narrower (fewer bits) to a wider (more bits) type and utilizing the tensor core instruction for the wider type. For instance, when doing
    INT8 x FP16, CUTLASS will convert INT8 -> FP16 and do math using FP16 tensor cores. Similarly, for INT4 x INT8, it will upcast to INT8 and issue math 
    using INT8 tensor cores.

    The narrower type always passes through the register file. Therefore, in cases where the narrower type is operand B, the collective will implicitly swap 
    A and B in the main loop. However, implicit swaps do not support TMA epilogues. Consequently, it is essential to consider this when constructing the epilogue, 
    as illustrated in this example.

    Note that in this example, we explicitly swap A and B in order to use TMA epilogues. We do this since TMA epilogues are more performant on problem sizes of interest.

    It is expected that the scale's K dimension be scale_k = ceil_div(problem_k, group_size). 
    
    Scales are always expected to be MN major. This means the fastest changing dimension must be M if A is scaled or N if B is scaled.
    
    If A is being scaled, the scales should have shape [M, scale_k],  while if B is scaled, it must have shape [N, scale_k].

    The implementation only supports "group-wise" scales. However, we can make it work for per-column scales by setting the groups size
    equal to the gemm problem K.

    Limitations:
      1) Only supported combinations are 16-bit x {8-bit, 4-bit, 2-bit} and {8-bit} x {4-bit, 2-bit}.
      2) The narrow type must always be in K-major format.
      3) The scales and zeros must be MN major. That means if A is scaled, it must be column major, but if B is scaled it must be row major.
      4) The scales and the zeros must have the same layout and groupsize.
      5) When dealing with 8-bit x {4-bit, 2-bit}, both inputs must be in K-major format.
      6) Currently, TMA epilogues cannot be used when the narrow type is the B operand. This limitation arises because the implementation always swaps the 
         operands to ensure that the narrow type passes through the register file, and TMA epilogues do not currently support implicit swap + transpose operations. 
         We plan to address this limitation in the future. However, we address this in the example by explicitly swapping and transposing the operands.

    Examples:
      
      Runs the mixed input batched gemm (with batch size 2), converting B to the type of A (mode 0)
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_mixed_dtype_gemm --m=2048 --n=2048 --k=2048 --l=2 --mode=0

      Runs the mixed input gemm, and applies a scaling factor to B before mma (mode 1). Applies a vector of scales to the entire
      matrix (group size is the same as the gemm k dimension).
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_mixed_dtype_gemm --m=4096 --n=5120 --k=8192 --g=8192 --mode=1

      Runs the mixed input gemm, and applies a scaling factor and adds a zero-point to B before mma (mode 2). Uses a group size of 128.
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_mixed_dtype_gemm --m=2048 --n=5120 --k=8192 --g=128 --mode=2
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"

#include "helper.h"
#include "unfused_weight_dequantize.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

// This is just an example, so we use a regular enum so we can compare directly to the command-line int.
enum GemmMode {
  ConvertOnly,
  ScaleOnly,
  ScaleWithZeroPoint
};

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
using MmaType = cutlass::float_e4m3_t;
using QuantType = cutlass::int4b_t;
constexpr int TileShapeK = 128 * 8 / sizeof_bits<MmaType>::value;

// A matrix configuration
using         ElementA    = MmaType;                                        // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = QuantType;                                      // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// This example manually swaps and transposes, so keep transpose of input layouts
using LayoutA_Transpose = typename cutlass::layout::LayoutTranspose<LayoutA>::type;
using LayoutB_Transpose = typename cutlass::layout::LayoutTranspose<LayoutB>::type;

using ElementZero = cutlass::half_t;
using ElementScale = cutlass::half_t;
using LayoutScale = cutlass::layout::RowMajor;

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_256,cute::Int<TileShapeK>>;         // Threadblock-level tile size
using ClusterShape        = Shape<_2,_1,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::KernelTmaWarpSpecializedCooperativeMixedInput;  // Kernel to launch based on the default setting in the Collective Builder 
using EpilogueSchedule    = cutlass::epilogue::TmaWarpSpecializedCooperative;
using EpilogueTileType    = cutlass::epilogue::collective::EpilogueTileAuto;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementAccumulator,
    // Transpose layout of D here since we use explicit swap + transpose
    // the void type for C tells the builder to allocate 0 smem for the C matrix.
    // We can enable this if beta == 0 by changing ElementC to void below.
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    ElementD, typename cutlass::layout::LayoutTranspose<LayoutD>::type, AlignmentD,
    EpilogueSchedule // This is the only epi supporting the required swap + transpose.
  >::CollectiveOp;

// ============================================================ MIXED INPUT NO SCALES ============================================================================
// The collective will infer that the narrow type should be upcasted to the wide type.
// We swap A and B operands to the builder here
using CollectiveMainloopConvertOnly = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementB, LayoutB_Transpose, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelConvertOnly = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopConvertOnly,
    CollectiveEpilogue
>;

using GemmConvertOnly = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelConvertOnly>;

// =========================================================== MIXED INPUT WITH SCALES ===========================================================================
// The Scale information must get paired with the operand that will be scaled. In this example, B is scaled so we make a tuple of B's information and the scale information.
using CollectiveMainloopScaleOnly = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<ElementB, ElementScale>, LayoutB_Transpose, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleOnly = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleOnly,
    CollectiveEpilogue
>;

using GemmScaleOnly = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleOnly>;

// =========================================================== MIXED INPUT WITH SCALES AND ZEROS ==================================================================
// We specify scale + zero elements to indicate that we require both. Scales and biases have the same format.
using CollectiveMainloopScaleWithZeroPoint = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<ElementB, ElementScale, ElementZero>, LayoutB_Transpose, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleWithZeroPoint = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleWithZeroPoint,
    CollectiveEpilogue
>;

using GemmScaleWithZeroPoint = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleWithZeroPoint>;
// =================================================================================================================================================================

using StrideA = cutlass::detail::TagToStrideA_t<LayoutA>;
using StrideB = cutlass::detail::TagToStrideB_t<LayoutB>;
using StrideC = typename GemmKernelScaleWithZeroPoint::StrideC;
using StrideD = typename GemmKernelScaleWithZeroPoint::StrideD;

using StrideC_ref = cutlass::detail::TagToStrideC_t<LayoutC>;
using StrideD_ref = cutlass::detail::TagToStrideC_t<LayoutD>;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideC_ref stride_C_ref;
StrideD stride_D;
StrideD_ref stride_D_ref;
uint64_t seed;

// Scale and Zero share a stride since the layout and shapes must be the same.
using StrideS = typename CollectiveMainloopScaleWithZeroPoint::StrideScale;
using StrideS_ref = cutlass::detail::TagToStrideB_t<LayoutScale>;
StrideS stride_S;
StrideS_ref stride_S_ref;

cutlass::HostTensor<MmaType, LayoutA> tensor_A;
cutlass::HostTensor<QuantType, LayoutB> tensor_B;
cutlass::HostTensor<MmaType, LayoutB> tensor_B_dq;
cutlass::HostTensor<ElementScale, LayoutScale> tensor_scale;
cutlass::HostTensor<ElementZero, LayoutScale> tensor_zero;
cutlass::HostTensor<ElementC, LayoutC> tensor_C;
cutlass::HostTensor<typename GemmScaleWithZeroPoint::EpilogueOutputOp::ElementOutput, LayoutD> tensor_D;
cutlass::HostTensor<typename GemmScaleWithZeroPoint::EpilogueOutputOp::ElementOutput, LayoutD> tensor_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  float alpha = 1.0f;
  float beta = 0.0f;
  int iterations = 1000;
  int mode = 2;
  int m = 5120, n = 4096, k = 4096;
  int g = 128;
  int l = 1;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("g", g);
    cmd.get_cmd_line_argument("mode", mode);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "55_hopper_warp_specialized_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   The number of independent gemm problems with mnk shape\n"
      << "  --g=<int>                   The size of each group for the scales and zeros. To broadcast a vector of scales or zeros, set the group size to K.\n"
      << "  --mode=<int>                The mode to run the gemm. 0 does (A @ B), 1 means A @ (scale * B), 2 means A @ (scale * B + zero-point).\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "55_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 -g 0 --l=10 --alpha=2 --mode=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k * l;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms = 0.0;
  double gflops = 0.0;
  cutlass::Status status = cutlass::Status::kSuccess;
  hipError_t error = hipSuccess;
  bool passed = false;

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element, class Layout>
bool initialize_tensor(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed=2023) {

  double scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;
  int bits_output = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  }
  else if (bits_input <= 8) {
    scope_max = 2;
    scope_min = -2;
  }
  else if (bits_output == 16) {
    scope_max = 5;
    scope_min = -5;
  }
  else {
    scope_max = 8;
    scope_min = -8;
  }
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min);

  return true;
}

template <typename Element, typename Layout>
bool initialize_quant_tensor(
  cutlass::TensorView<Element, Layout> view,
  uint64_t seed=2023) {
  
  float scope_min = float(cutlass::platform::numeric_limits<Element>::lowest());
  float scope_max = float(cutlass::platform::numeric_limits<Element>::max());

  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, scope_max, scope_min);

  return true;
}

template <class Element, class Layout>
bool initialize_scale(
  cutlass::TensorView<Element, Layout> view, 
  const Options &options) {
  
  if (options.mode == GemmMode::ConvertOnly) {
    // No scales, so just initialize with 1 so we can use the same kernel to dequantize the data.
    cutlass::reference::host::TensorFill(view, Element(1.0f));
  } 
  else {
    float elt_max_f = float(cutlass::platform::numeric_limits<QuantType>::max());
    const float max_dequant_val = 4.f;
    const float min_dequant_val = 0.5f;

    float scope_max(max_dequant_val / elt_max_f);
    float scope_min(min_dequant_val / elt_max_f);

    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, scope_max, scope_min);
  }
  return true;
}

template <class Element, class Layout>
bool initialize_zero(
  cutlass::TensorView<Element, Layout> view,
  const Options &options) {
  
  if (options.mode == GemmMode::ScaleWithZeroPoint) {
    cutlass::reference::host::TensorFillRandomUniform(
      view, seed, 2.0f, -2.0f);
  } else {
    // No bias, so just initialize with 1 so we can use the same kernel to dequantize the data.
    cutlass::reference::host::TensorFill(view, Element(0.0f));
  }
  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  auto shape_b = cute::make_shape(options.n, options.k, options.l);
  const int scale_k = (options.k + options.g - 1) / options.g;
  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, shape_b);
  // Reverse stride here due to swap and transpose
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.n, options.m, options.l));
  stride_C_ref = cutlass::make_cute_packed_stride(StrideC_ref{}, cute::make_shape(options.m, options.n, options.l));
  // Reverse stride here due to swap and transpose
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.n, options.m, options.l));
  stride_D_ref = cutlass::make_cute_packed_stride(StrideD_ref{}, cute::make_shape(options.m, options.n, options.l));

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  tensor_B_dq.resize(b_coord);
  tensor_C.resize(c_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  tensor_scale.resize({scale_k * options.l, options.n});
  tensor_zero.resize({scale_k * options.l, options.n});

  initialize_tensor(tensor_A.host_view(), seed + 2022);
  initialize_quant_tensor(tensor_B.host_view(), seed + 2021);
  initialize_tensor(tensor_C.host_view(), seed + 2020);
  initialize_scale(tensor_scale.host_view(), options);
  initialize_zero(tensor_zero.host_view(), options);

  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_C.sync_device();
  tensor_scale.sync_device();
  tensor_zero.sync_device();

  auto layout_B = make_layout(shape_b, stride_B);

  auto shape_scale_zero = cute::make_shape(options.n, scale_k, options.l);
  stride_S = cutlass::make_cute_packed_stride(StrideS{}, cute::make_shape(options.n, scale_k, options.l));
  stride_S_ref = cutlass::make_cute_packed_stride(StrideS_ref{}, cute::make_shape(options.n, scale_k, options.l));
  auto layout_scale_zero = make_layout(shape_scale_zero, stride_S_ref);

  dequantize_weight(tensor_B_dq.device_data(), tensor_B.device_data(), layout_B, tensor_scale.device_data(), tensor_zero.device_data(), layout_scale_zero, options.g);
  tensor_B_dq.sync_host();
}

/// Populates a Gemm::Arguments structure from the given commandline options
template <typename Args>
Args args_from_options(const Options &options)
{
// Swap the A and B tensors, as well as problem shapes here.
  if (options.mode == GemmMode::ConvertOnly) {
    return Args {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {options.n, options.m, options.k, options.l},
      {tensor_B.device_data(), stride_B, tensor_A.device_data(), stride_A},
      {{options.alpha, options.beta}, tensor_C.device_data(), stride_C, tensor_D.device_data(), stride_D}
    };
  } 
  else if (options.mode == GemmMode::ScaleOnly) {
    return Args {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {options.n, options.m, options.k, options.l},
      {tensor_B.device_data(), stride_B, tensor_A.device_data(), stride_A, tensor_scale.device_data(), stride_S, options.g},
      {{options.alpha, options.beta}, tensor_C.device_data(), stride_C, tensor_D.device_data(), stride_D}
    };
  } 
  else if (options.mode == GemmMode::ScaleWithZeroPoint) {
    return Args {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {options.n, options.m, options.k, options.l},
      {tensor_B.device_data(), stride_B, tensor_A.device_data(), stride_A, tensor_scale.device_data(), stride_S, options.g, tensor_zero.device_data()},
      {{options.alpha, options.beta}, tensor_C.device_data(), stride_C, tensor_D.device_data(), stride_D}
    };
  } else {
    std::cerr << "Invalid mode " << options.mode << ". Must be 0, 1 or 2." << std::endl;
    exit(-1);
  }
}

bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // In this example, we use the GPU default kernels as a reference (unfused scale)
  // This is to avoid numerical differences from different accumulation order.

  // Again, due to numerical differences, we must use fast acc here when the mma type is
  // FP8 as the fused implementation only supports fast acc at the moment.
  constexpr bool IsFP8Input = cute::is_same_v<MmaType, cutlass::float_e4m3_t> || cute::is_same_v<MmaType, cutlass::float_e5m2_t>;
  using FP8Sched = cute::conditional_t<size<0>(TileShape{}) == 64, cutlass::gemm::KernelTmaWarpSpecializedPingpongFP8FastAccum, cutlass::gemm::KernelTmaWarpSpecializedCooperativeFP8FastAccum>;
  using ScheduleRef = cute::conditional_t<IsFP8Input, FP8Sched, cutlass::gemm::collective::KernelScheduleAuto>;

  using CollectiveMainloopRef = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      MmaType, LayoutA, AlignmentA,
      MmaType, LayoutB, AlignmentB,
      ElementAccumulator,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAuto,
      ScheduleRef
    >::CollectiveOp;

  using CollectiveEpilogueRef = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementAccumulator,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      cutlass::epilogue::NoSmemWarpSpecialized
    >::CollectiveOp;

  using GemmKernelRef = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>, // Indicates ProblemShape
      CollectiveMainloopRef,
      CollectiveEpilogueRef
  >;

  using GemmRef = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelRef>;

  typename GemmRef::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(), stride_A, tensor_B_dq.device_data(), stride_B},
    {{options.alpha, options.beta}, tensor_C.device_data(), stride_C_ref, tensor_ref_D.device_data(), stride_D_ref}
  };

  // Run the gemm where the scaling is performed outside of the kernel.
  GemmRef gemm_ref;
  size_t workspace_size = GemmRef::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
  CUTLASS_CHECK(gemm_ref.can_implement(arguments));
  CUTLASS_CHECK(gemm_ref.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(gemm_ref.run());

  // compare_reference
  tensor_D.sync_host();
  tensor_ref_D.sync_host();
  const ElementD epsilon(1e-2f);
  const ElementD non_zero_floor(1e-4f);
  bool passed = cutlass::reference::host::TensorRelativelyEquals(tensor_ref_D.host_view(), tensor_D.host_view(), epsilon, non_zero_floor);
  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<typename Gemm::Arguments>(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  if (options.mode == GemmMode::ConvertOnly) {
    std::cout << "Running in no scale mode." << std::endl;
    run<GemmConvertOnly>(options);
  }
  else if (options.mode == GemmMode::ScaleOnly) {
    if (options.g == options.k) {
      std::cout << "Running in per-column scale mode." << std::endl;
    } else {
      std::cout << "Running in group scale mode." << std::endl;
    }
    run<GemmScaleOnly>(options);
  }
  else if (options.mode == GemmMode::ScaleWithZeroPoint) {
    if (options.g == options.k) {
      std::cout << "Running in per-column scale and zero mode." << std::endl;
    } else {
      std::cout << "Running in group scale and zero mode." << std::endl;
    }
    run<GemmScaleWithZeroPoint>(options);
  }
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
