#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example for f16 x s8 using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    Examples:

      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_mixed_dtype_gemm --m=2048 --n=2048 --k=2048
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/mixed_input_collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "helper.h"
#include "unfused_weight_dequantize.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
using MmaType = cutlass::half_t;
using QuantType = int8_t;

// A matrix configuration
using         ElementA    = MmaType;                                // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = QuantType;                                      // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// Scale matrix configuration. This is only applied to the matrix with
// the narrower type
using         ElementScale   = MmaType;
using         LayoutScale    = cutlass::layout::RowMajor;
constexpr int AlignmentScale = 128 / cutlass::sizeof_bits<ElementScale>::value; 

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_256,_64>;                           // Threadblock-level tile size
using ClusterShape        = Shape<_2,_1,_1>;                                // Shape of the threadblocks in a cluster
using StageCountType = cutlass::gemm::collective::StageCountAuto;           // Stage count maximized based on the tile size
using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecializedCooperative;  // Kernel to launch based on the default setting in the Collective Builder 
using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;   // Epilogue to use. This MUST match main loop schedule.


// We must be aware of when swap + transpose happen. The narrow type always gets swapped to A internally. Therefore,
// we must tell the epiloogue to output column major (and change the strides) if we want to output row-major tensors.
using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator, ElementAccumulator,
    // Lie here about layout of C and D since we do swap and transpose trick
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    EpilogueSchedule
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::MixedInputCollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementScale, LayoutScale, AlignmentScale,
    cutlass::gemm::collective::MixedInputCastDirection::NARROW_TO_WIDE,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  MmaType,
  LayoutA,
  MmaType,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideScale = typename CollectiveMainloop::StrideScale;
using StrideC = typename Gemm::GemmKernel::StrideC;
using StrideD = typename Gemm::GemmKernel::StrideD;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideScale stride_scale;
StrideC stride_C;
StrideD stride_D;
uint64_t seed;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementA> block_B_dq;
cutlass::DeviceAllocation<ElementScale> block_scale;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;

  float alpha, beta;
  int iterations;
  int m, n, k;
  int num_groups;

  Options():
    help(false),
    m(5120), n(4096), k(4096),
    num_groups(1),
    alpha(1.f), beta(0.f),
    iterations(1000)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("num_groups", num_groups);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "55_hopper_warp_specialized_gemm\n\n"
      << "  Hopper FP32 GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --num_groups=<int>          The number of groups in the scales\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "55_hopper_warp_specialized_gemm" << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }
};

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  Element scope_max, scope_min;
  int bits_input = cutlass::sizeof_bits<Element>::value;

  if (bits_input == 1) {
    scope_max = 2;
    scope_min = 0;
  } else if (bits_input <= 8) {
    // This is the quantized value
    scope_max = std::numeric_limits<Element>::max();
    scope_min = std::numeric_limits<Element>::min();
  } else {
    scope_max = 8;
    scope_min = -8;
  }

  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min, 0);

  return true;
}

template <class Element>
bool initialize_scale_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023) {

  float elt_max_f(std::numeric_limits<QuantType>::max());
  const float max_dequant_val = 2.f;
  const float min_dequant_val = 0.15f;

  float scope_max_f = max_dequant_val / elt_max_f;
  float scope_min_f = min_dequant_val / elt_max_f;

  // CUTLASS kernel does not take scales at the moment, so we use 1.f to dequantize.
  Element scope_min(1.f); 
  Element scope_max(1.f);
  cutlass::reference::device::BlockFillRandomUniform(
    block.get(), block.size(), seed, scope_max, scope_min);

  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  auto shape_b = cute::make_shape(options.n, options.k, Int<1>{});
  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, Int<1>{}));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, shape_b);
  stride_scale = cutlass::make_cute_packed_stride(StrideScale{}, cute::make_shape(options.n, Int<1>{}, Int<1>{}));
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.n, options.m, Int<1>{}));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.n, options.m, Int<1>{}));

  block_A.reset(options.m * options.k);
  block_B.reset(options.k * options.n);
  block_B_dq.reset(options.k * options.n);
  block_scale.reset(options.n * options.num_groups);
  block_C.reset(options.m * options.n);
  block_D.reset(options.m * options.n);
  block_ref_D.reset(options.m * options.n);

  initialize_block(block_A, seed + 2023);
  initialize_block(block_B, seed + 2022);
  initialize_scale_block(block_scale, seed + 2021);

  auto layout_B = make_layout(shape_b, stride_B);
  auto shape_scale = cute::make_shape(options.n, options.num_groups, Int<1>{});
  auto layout_scale = make_layout(shape_scale, stride_scale);
  dequantize_weight(block_B_dq.get(), block_B.get(), layout_B, block_scale.get(), layout_scale);
  initialize_block(block_C, seed + 2020);
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options)
{
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k},
    {block_A.get(), stride_A, block_B.get(), stride_B},
    {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
  };

  return arguments;
}

bool verify(const Options &options) {
  cutlass::TensorRef ref_A(block_A.get(), Gemm::LayoutA::packed({options.m, options.k}));
  cutlass::TensorRef ref_B(block_B_dq.get(), Gemm::LayoutB::packed({options.k, options.n}));
  cutlass::TensorRef ref_C(block_C.get(), LayoutC::packed({options.m, options.n}));
  cutlass::TensorRef ref_D(block_ref_D.get(), LayoutC::packed({options.m, options.n}));

  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    {options.m, options.n, options.k},
    ElementAccumulator(options.alpha),
    ref_A,
    ref_B,
    ElementAccumulator(options.beta),
    ref_C,
    ref_D);

  // Wait for kernel to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  bool passed = cutlass::reference::device::BlockCompareEqual(block_ref_D.get(), block_D.get(), block_D.size());

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
