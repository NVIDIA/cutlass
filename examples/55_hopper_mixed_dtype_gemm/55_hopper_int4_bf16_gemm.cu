/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Hopper GEMM example with different data types using CUTLASS 3.0 APIs for NVIDIA Hopper architecture

    This example shows how to perform INT4 x BF16 GEMM and scale up the INT4 weight during dequantization.

    The narrower type always passes through the register file. Therefore, in cases where the narrower type is operand B, the collective will implicitly swap 
    A and B in the main loop. However, as a result of this collective performing implicit swaps, it does not support TMA epilogues. Consequently, it is essential to consider this when constructing the epilogue, 
    as illustrated in this example.

    Note that in this example, we explicitly swap A and B in order to use TMA epilogues. We do this since TMA epilogues are more performant on problem sizes of interest.

    As an additional optimization, we can reorder the narrow data type tensor such that elements read into register file by the same thread are contiguous in global and shared memory.
    This promotes vectorization of shared memory loads and removes additional instructions on the critical path. For example, when MMA is performed in 16-bit data type, each thread reads
    4 groups of 2 elements that are logically contiguous in the same row (refer to https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#wgmma-64n16-a for thread-value layout).
    If the narrow type is INT4 and tensor is major in K dim, only 8 bits can be read at a time, leading to extra load instructions and suboptimal utilization of shared memory throughput.
    If we reorder the data offline to place all 16 elements read by a thread contiguously in memory, a single 64-bit load is sufficient. This reordering is often feasible when the quantized
    tensor is static (e.g. weight tensor of a NN layer at inference time). This example demonstrates how such a reordering can be performed and communicated to the kernel when the options.shuffle is set to true.

    Furthermore, the conversion from {INT4, UINT4} to {FP16, BF16} can benefit from pre-shuffling the weights in the order [0,2,4,6,1,3,5,7]. This allows multiple nibbles to be efficiently extracted and up-converted
    in parallel. The reordering is enabled by defining the layout type `ValueShuffle`. Refer to the partial specializations of `NumericArrayShuffleConverter` in "include/cutlass/detail/collective/mixed_input_utils.hpp"
    for more details.

    It is expected that the scale's K dimension be scale_k = ceil_div(problem_k, group_size). 
    
    Scales are always expected to be MN major. This means the fastest changing dimension must be M if A is scaled or N if B is scaled.
    
    If A is being scaled, the scales must have shape [M, scale_k],  while if B is scaled, it must have shape [N, scale_k].

    The implementation only supports "group-wise" scales. However, we can make it work for per-column scales by setting the group's size
    equal to the gemm problem K.

    Limitations:
      1) The INT4 weights have additional encoding requirements.
      2) The scales must be MN major. That means if A is scaled, it must be column major, but if B is scaled it must be row major.
      3) The scales must have the same layout and groupsize.
      4) The groupsize must be greater or equal to the tile shape k.
      5) Currently, TMA epilogues cannot be used when the narrow type is the B operand. This limitation arises because the implementation always swaps the 
         operands to ensure that the narrow type passes through the register file, and TMA epilogues do not currently support implicit swap + transpose operations. 
         We plan to address this limitation in the future. However, we address this in the example by explicitly swapping and transposing the operands.
    
    Optimizing suggestions:
      1) Use a small tile size, since the register pressure for this GEMM (and RS GEMM in general) is high (it uses a lot of register space).

    Examples:
      
      Runs the mixed input batched gemm (with batch size 2), converting B to the type of A (mode 0)
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_int4_bf16_gemm --m=2048 --n=2048 --k=2048 --l=2 --mode=0

      Runs the mixed input gemm, and applies a scaling factor to B before mma (mode 1). Applies a vector of scales to the entire
      matrix (group size is the same as the gemm k dimension).
      $ ./examples/55_hopper_mixed_dtype_gemm/55_hopper_int4_bf16_gemm --m=4096 --n=5120 --k=8192 --g=8192 --mode=1
*/

#include <iostream>

#include "cutlass/cutlass.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/device/tensor_compare.h"

#include "helper.h"
#include "mixed_dtype_utils.hpp"
#include "packed_scale.hpp"
#include "reorder_utils.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////
using MmaType = cutlass::bfloat16_t;
using QuantType = cutlass::int4b_t;
constexpr int TileShapeK = 128 * 8 / sizeof_bits<MmaType>::value;

// A matrix configuration
using         ElementA    = MmaType;                                        // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = QuantType;                                      // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// This example manually swaps and transposes, so keep transpose of input layouts
using LayoutA_Transpose = typename cutlass::layout::LayoutTranspose<LayoutA>::type;
using LayoutB_Transpose = typename cutlass::layout::LayoutTranspose<LayoutB>::type;

using StrideA = cutlass::detail::TagToStrideA_t<LayoutA>;
using StrideB = cutlass::detail::TagToStrideB_t<LayoutB>;

// Define the CuTe layout for reoredered quantized tensor B
// LayoutAtomQuant places values that will be read by the same thread in contiguous locations in global memory.
// It specifies the reordering within a single warp's fragment
//using ValueShuffle = Layout<_1>;                          // no value reordering
using ValueShuffle = Layout<Shape<_2,_4>, Stride<_4,_1>>; // order [0,2,4,6,1,3,5,7]
int constexpr NumShuffleAtoms = 1;
using MmaAtomShape = Layout<Shape<_1,Int<NumShuffleAtoms>>>;
using LayoutAtomQuant = decltype(compute_memory_reordering_atom<MmaType, MmaAtomShape, ValueShuffle>());
using LayoutB_Reordered = decltype(tile_to_shape(LayoutAtomQuant{}, Layout<Shape<int,int,int>, StrideB>{}));

using ElementScale = MmaType;
using ElementZero = ElementScale;
using LayoutScale = cutlass::layout::RowMajor;

// C/D matrix configuration
using         ElementC    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C matrix in units of elements (up to 16 bytes)

// D matrix configuration
using         ElementD    = ElementC;
using         LayoutD     = LayoutC;
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_128,_128,cute::Int<TileShapeK>>;         // Threadblock-level tile size
using ClusterShape        = Shape<_1,_1,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::KernelTmaWarpSpecializedCooperative;  // Kernel to launch based on the default setting in the Collective Builder 
using EpilogueSchedule    = cutlass::epilogue::TmaWarpSpecializedCooperative;
using EpilogueTileType    = cutlass::epilogue::collective::EpilogueTileAuto;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
    TileShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementAccumulator,
    // Transpose layout of D here since we use explicit swap + transpose
    // the void type for C tells the builder to allocate 0 smem for the C matrix.
    // We can enable this if beta == 0 by changing ElementC to void below.
    ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
    ElementD, typename cutlass::layout::LayoutTranspose<LayoutD>::type, AlignmentD,
    EpilogueSchedule // This is the only epi supporting the required swap + transpose.
  >::CollectiveOp;

// ============================================================ MIXED INPUT NO SCALES ============================================================================
// The collective will infer that the narrow type should be upcasted to the wide type.
// We swap A and B operands to the builder here
using CollectiveMainloopConvertOnly = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementB, LayoutB_Transpose, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelConvertOnly = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopConvertOnly,
    CollectiveEpilogue
>;

using GemmConvertOnly = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelConvertOnly>;

using CollectiveMainloopConvertOnlyShuffled = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementB, LayoutB_Reordered, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelConvertOnlyShuffled = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopConvertOnlyShuffled,
    CollectiveEpilogue
>;

using GemmConvertOnlyShuffled = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelConvertOnlyShuffled>;

// =========================================================== MIXED INPUT WITH SCALES ===========================================================================
// The Scale information must get paired with the operand that will be scaled. In this example, B is scaled so we make a tuple of B's information and the scale information.
using CollectiveMainloopScaleOnly = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<ElementB, ElementScale>, LayoutB_Transpose, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleOnly = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleOnly,
    CollectiveEpilogue
>;

using GemmScaleOnly = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleOnly>;

using CollectiveMainloopScaleOnlyShuffled = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<ElementB, ElementScale>, LayoutB_Reordered, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleOnlyShuffled = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleOnlyShuffled,
    CollectiveEpilogue
>;

using GemmScaleOnlyShuffled = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleOnlyShuffled>;

// =========================================================== MIXED INPUT WITH SCALES AND ZEROS ==================================================================
// We specify scale + zero elements to indicate that we require both. Scales and biases have the same format.
using CollectiveMainloopScaleWithZeroPoint = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<ElementB, ElementScale, ElementZero>, LayoutB_Transpose, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleWithZeroPoint = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleWithZeroPoint,
    CollectiveEpilogue
>;

using GemmScaleWithZeroPoint = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleWithZeroPoint>;

using CollectiveMainloopScaleWithZeroPointShuffled = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    cute::tuple<ElementB, ElementScale, ElementZero>, LayoutB_Reordered, AlignmentB,
    ElementA, LayoutA_Transpose, AlignmentA,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernelScaleWithZeroPointShuffled = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloopScaleWithZeroPointShuffled,
    CollectiveEpilogue
>;

using GemmScaleWithZeroPointShuffled = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelScaleWithZeroPointShuffled>;
// =================================================================================================================================================================

using StrideC = typename GemmKernelScaleOnly::StrideC;
using StrideD = typename GemmKernelScaleOnly::StrideD;

using StrideC_ref = cutlass::detail::TagToStrideC_t<LayoutC>;
using StrideD_ref = cutlass::detail::TagToStrideC_t<LayoutD>;

//
// Data members
//

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideC stride_C;
StrideC_ref stride_C_ref;
StrideD stride_D;
StrideD_ref stride_D_ref;
uint64_t seed;

LayoutB_Reordered layout_B_reordered;

using StrideS = typename CollectiveMainloopScaleOnly::StrideScale;
using StrideS_ref = cutlass::detail::TagToStrideB_t<LayoutScale>;
StrideS stride_S;
StrideS_ref stride_S_ref;

cutlass::DeviceAllocation<ElementA> block_A;
cutlass::DeviceAllocation<ElementB> block_B;
cutlass::DeviceAllocation<ElementA> block_B_dq;
cutlass::DeviceAllocation<ElementScale> block_scale;
cutlass::DeviceAllocation<ElementZero> block_zero;
cutlass::DeviceAllocation<ElementC> block_C;
cutlass::DeviceAllocation<typename GemmScaleOnly::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename GemmScaleOnly::EpilogueOutputOp::ElementOutput> block_ref_D;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options : MixedDtypeOptions{
  bool shuffle = true;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);
    cmd.get_cmd_line_argument("shuffle", shuffle);

    this->MixedDtypeOptions::parse(argc, args);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "55_hopper_int4_bf16_gemm\n\n"
      << "  Hopper Mixed Data Type GEMM using a Warp Specialized kernel.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   The number of independent gemm problems with mnk shape\n"
      << "  --g=<int>                   The size of each group for the scales. To broadcast a vector of scales or zeros, set the group size to K.\n"
      << "  --mode=<int>                The mode to run the gemm. 0 does (A @ B), 1 means A @ (scale * B), 2 means A @ (scale * B + zero-point).\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n"
      << "  --warmup=<int>              Number of warmup iterations to perform.\n\n"
      << "  --shuffle=<boolean>         Enable the offline layout swizzling.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "55_hopper_int4_bf16_gemm" << " --m=1024 --n=512 --k=1024 -g=1024 --l=10 --alpha=2 --mode=2 --beta=0.707 \n\n";

    return out;
  }
};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(Options const& options) {

  auto shape_B = cute::make_shape(options.n, options.k, options.l);
  int const scale_k = (options.k + options.g - 1) / options.g;
  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, shape_B);
  // Reverse stride here due to swap and transpose
  stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(options.n, options.m, options.l));
  stride_C_ref = cutlass::make_cute_packed_stride(StrideC_ref{}, cute::make_shape(options.m, options.n, options.l));
  // Reverse stride here due to swap and transpose
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.n, options.m, options.l));
  stride_D_ref = cutlass::make_cute_packed_stride(StrideD_ref{}, cute::make_shape(options.m, options.n, options.l));

  auto layout_B = make_layout(shape_B, stride_B);

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);

  block_A.reset(a_coord.product());
  block_B.reset(b_coord.product());
  block_B_dq.reset(b_coord.product());
  block_C.reset(c_coord.product());
  block_D.reset(c_coord.product());
  block_ref_D.reset(c_coord.product());

  block_scale.reset(scale_k * options.l * options.n);
  block_zero.reset(scale_k * options.l * options.n);

  initialize_tensor(block_A, seed + 2022);
  initialize_quant_tensor(block_B, seed + 2021);
  initialize_tensor(block_C, seed + 2020);
  initialize_scale(block_scale, options);
  initialize_zero(block_zero, options);

  auto shape_scale_zero = cute::make_shape(options.n, scale_k, options.l);
  stride_S = cutlass::make_cute_packed_stride(StrideS{}, cute::make_shape(options.n, scale_k, options.l));
  stride_S_ref = cutlass::make_cute_packed_stride(StrideS_ref{}, cute::make_shape(options.n, scale_k, options.l));
  auto layout_scale_zero = make_layout(shape_scale_zero, stride_S_ref);

  dequantize_weight(block_B_dq.get(), block_B.get(), layout_B, block_scale.get(), block_zero.get(), layout_scale_zero, options.g);

  if (options.shuffle) {
    // Repeat the reorder layout atom to tile the whole tensor shape 
    layout_B_reordered = tile_to_shape(LayoutAtomQuant{}, shape_B);
    reorder_tensor(block_B.get(), layout_B, layout_B_reordered);

    print("Quantized tensor layout: ");
    print(layout_B_reordered);
    print("\n");
  }
}

/// Populates a Gemm::Arguments structure from the given commandline options
/// Swap the A and B tensors, as well as problem shapes here.
template <typename Gemm>
typename Gemm::Arguments args_from_options(Options const& options)
{
  using Args = typename Gemm::Arguments;
  auto&& dB = [&]() {
    if constexpr (cute::is_same_v<Gemm, GemmConvertOnlyShuffled> ||
                  cute::is_same_v<Gemm, GemmScaleOnlyShuffled> ||
                  cute::is_same_v<Gemm, GemmScaleWithZeroPointShuffled>) {
      // offline swizzling is enabled.
      return layout_B_reordered;
    }
    else {
      return stride_B;
    }
  }();
  if (options.mode == MixedDtypeGemmMode::ConvertOnly) {
    return Args {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {options.n, options.m, options.k, options.l},
      {block_B.get(), dB, block_A.get(), stride_A},
      {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
    };
  } 
  else if (options.mode == MixedDtypeGemmMode::ScaleOnly) {
    return Args {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {options.n, options.m, options.k, options.l},
      {block_B.get(), dB, block_A.get(), stride_A, block_scale.get(), stride_S, options.g},
      {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
    };
  } 
  else if (options.mode == MixedDtypeGemmMode::ScaleWithZeroPoint) {
    return Args {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {options.n, options.m, options.k, options.l},
      {block_B.get(), dB, block_A.get(), stride_A, block_scale.get(), stride_S, options.g, block_zero.get()},
      {{options.alpha, options.beta}, block_C.get(), stride_C, block_D.get(), stride_D}
    };
  } else {
    std::cerr << "Invalid mode " << options.mode << ". Must be 0, 1 or 2." << std::endl;
    exit(-1);
  }
}

bool verify(Options const& options) {
  //
  // Compute reference output
  //

  using CollectiveMainloopRef = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag, OperatorClass,
      MmaType, LayoutA, AlignmentA,
      MmaType, LayoutB, AlignmentB,
      ElementAccumulator,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using CollectiveEpilogueRef = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementAccumulator,
      ElementC, LayoutC, AlignmentC,
      ElementD, LayoutD, AlignmentD,
      cutlass::epilogue::NoSmemWarpSpecialized
    >::CollectiveOp;

  using GemmKernelRef = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>, // Indicates ProblemShape
      CollectiveMainloopRef,
      CollectiveEpilogueRef
  >;

  using GemmRef = cutlass::gemm::device::GemmUniversalAdapter<GemmKernelRef>;

  typename GemmRef::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {block_A.get(), stride_A, block_B_dq.get(), stride_B},
    {{options.alpha, options.beta}, block_C.get(), stride_C_ref, block_ref_D.get(), stride_D_ref}
  };

  // Run the gemm where the scaling is performed outside of the kernel.
  GemmRef gemm_ref;
  size_t workspace_size = GemmRef::get_workspace_size(arguments);
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
  CUTLASS_CHECK(gemm_ref.can_implement(arguments));
  CUTLASS_CHECK(gemm_ref.initialize(arguments, workspace.get()));
  CUTLASS_CHECK(gemm_ref.run());

  // compare_reference
  ElementD const epsilon(1e-2f);
  ElementD const non_zero_floor(1e-4f);
  bool passed = cutlass::reference::device::BlockCompareRelativelyEqual(block_ref_D.get(), block_D.get(), block_D.size(), epsilon, non_zero_floor);

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options)
{
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options<Gemm>(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  MixedDtypeResult result;
  result.passed = verify(options);
  mixed_dtype_profiling(gemm, options, result);
  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;
  if (!result.passed) {
    exit(-1);
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  if (options.mode == MixedDtypeGemmMode::ConvertOnly) {
    std::cout << "Running in no scale mode." << std::endl;
    if (options.shuffle) {
      std::cout << "Offline shuffle enabled." << std::endl;
      run<GemmConvertOnlyShuffled>(options);
    } else {
      std::cout << "Offline shuffle disabled." << std::endl;
      run<GemmConvertOnly>(options);
    }
  }
  else if (options.mode == MixedDtypeGemmMode::ScaleOnly) {
    if (options.g == options.k) {
      std::cout << "Running in per-column scale mode." << std::endl;
    } else {
      std::cout << "Running in group scale mode." << std::endl;
    }
    if (options.shuffle) {
      std::cout << "Offline shuffle enabled." << std::endl;
      run<GemmScaleOnlyShuffled>(options);
    } else {
      std::cout << "Offline shuffle disabled." << std::endl;
      run<GemmScaleOnly>(options);
    }
  }
  else if (options.mode == MixedDtypeGemmMode::ScaleWithZeroPoint) {
    if (options.g == options.k) {
      std::cout << "Running in per-column scale and zero mode." << std::endl;
    } else {
      std::cout << "Running in group scale and zero mode." << std::endl;
    }
    if (options.shuffle) {
      std::cout << "Offline shuffle enabled." << std::endl;
      run<GemmScaleWithZeroPointShuffled>(options);
    } else {
      std::cout << "Offline shuffle disabled." << std::endl;
      run<GemmScaleWithZeroPoint>(options);
    }
  }
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////