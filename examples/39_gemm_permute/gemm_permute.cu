/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief GEMM Permute Example.

    This example computes batched GEMM operations with output results permuted as reshaped tensors.

    We provide layout plugin as a flexible tool for users to add any customized input/output tensor permute operation, 
    or any other generalized global memory writeout address computation. To add a customized layout, add new class
    in include/cutlass/layout/permute.h

    In this example we use several permute operations (permute([0, 2, 1, 3]))

    In this example, we used Tensor4DPermuteBMM0213 layout to perform Batched GEMM with permute([0, 2, 1, 3]) on BMM
    whole output tensor, and used Tensor5DPermute20314 layout to perform Normal GEMM with permute([2, 0, 3, 1, 4]) on
    output matrix. The address computations are performed in compute(col_init, row_init, stride_init, 
    BMM_batch_idx) with {col_permute, row_permute and stride_permute} as new addresses after permute op.
    (check include/cutlass/layout/permute.h)

    Tips:
    
      1) Make sure to set batch_stride to zero for BMM permute; also the BMM GEMM should be in mode
      cutlass::gemm::GemmUniversalMode::kBatched instead of kArray.

      2) When the contiguous dimension is touched in permute op (for example [0, 2, 3, 1] for row-major matrix 
      or [1, 0, 2, 3] for column-major), Alignment should be set to 1 for the corresponding matrix. 
      If the last dimension is untouched,  one can set Alignment to be larger like 8 in our example.
      As a result, permute op without touching the unit stride dimension is recommended to obtain the best performance.

    Examples:

      # Runs a batched GEMM with 96 batches
      $ ./examples/39_gemm_permute/39_gemm_permute --problem-count=96

      # Runs a batched GEMM with 96 batches (with GEMM-K dimension equal to 1024)
      $ ./examples/39_gemm_permute/39_gemm_permute --problem-count=96 --k=1024 --verbose=true

      # Execute batched GEMM and profile with NSight
      $ nv-nsight-cu-cli ./examples/39_gemm_permute/39_gemm_permute --m=256 --n=192 --k=256 --verbose=true --iterations=1 --reference-check=false

*/

/////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/permute.h"

#include "layouts.h"
#include "permute_info.h"

/// Tensor4DPermuteBMM0213 --->
/// Permute layout function for 4-D permuted tensors for BMM with BMM tensor (dimension as [B, M, N]) reshaped
/// as [B/D1, D1, M, N]. Then perform permute([0, 2, 1, 3]) on the corresponding whole BMM tensor.
int constexpr D1 = 12;

/// Tensor5DPermute20314 --->
/// Permute layout function for 5-D permuted tensors with matrix (dimension as [M, N]) reshaped
/// as [M/T1, T1, T2, T3, N/T2/T3]. Then perform permute([2, 0, 3, 1, 4]) on the corresponding tensor.
int constexpr T1 = 16; 
int constexpr T2 = 3;
int constexpr T3 = 8;

/// Tensor4DPermute0213 --->
/// Permute layout function for 4-D permuted tensors with matrix (dimension as [M, N]) reshaped
/// as [M/S1, S1, S2, N/S2]. Then perform permute([0, 2, 1, 3]) on the corresponding tensor.
int constexpr S1 = 8; 
int constexpr S2 = 4;

// // // Alignments
int constexpr AlignmentA = 8;
int constexpr AlignmentB = 8;
int constexpr AlignmentC = 8;

/// GEMM element types
using ElementInput = cutlass::half_t;
using ElementOutput = cutlass::half_t;
using ElementAccumulator = float;

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Useful macros

#define CHECK_CUDA_CALL(call, handler) \
do { \
  hipError_t __err = (call); \
  if (__err != hipSuccess) { \
    std::cerr << #call " failed: " << hipGetErrorString(__err) << std::endl; \
    handler; \
  } \
} while(0)

#define CHECK_CUTLASS_CALL(call, handler) \
do { \
  cutlass::Status __status = (call); \
  if (__status != cutlass::Status::kSuccess) { \
    std::cerr << #call " failed: " << cutlass::cutlassGetStatusString(__status) << std::endl; \
    handler; \
  } \
} while(0)

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;

  cutlass::gemm::GemmCoord problem_each;

  int batch_count;
  int iterations;
  int cuda_streams;
  bool verbose;
  float alpha;
  float beta;

  //
  // Methods
  // 

  Options():
    help(false),
    error(false),
    reference_check(true),
    batch_count(-1),
    iterations(20),
    cuda_streams(0),
    verbose(false),
    alpha(1),
    beta()
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("alpha", alpha, 1.0f);
    cmd.get_cmd_line_argument("beta", beta, 0.0f);    
    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("streams", cuda_streams, 0);
    cmd.get_cmd_line_argument("verbose", verbose, false);
    cmd.get_cmd_line_argument("reference-check", reference_check, true);

    int m, n, k;

    cmd.get_cmd_line_argument("m", m, 384);
    cmd.get_cmd_line_argument("n", n, 192);
    cmd.get_cmd_line_argument("k", k, 384);
    cmd.get_cmd_line_argument("batch-count", batch_count, 96);

    problem_each = cutlass::gemm::GemmCoord(m, n, k);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << 
      "39_gemm_permute\n"
      "\n"
      " This example tests and profiles the performance of normal GEMM and batched GEMM with different"
      " combinations of fused permutations of input and output tensors."
      "\n"
      " Permutations considered in this example:\n"
      "\n"
      " Normal GEMM:\n"
      " 1) Tensor4DPermute0213: matrix of shape [X, Y] is reshaped as [X/S1, S1, S2, Y/S2] and has its dimensions"
      " permuted as [0, 2, 1, 3], resulting in shape [X/S1, S2, S1, Y/S2] viewed as matrix of shape [X*S2/S1, Y*S1/S2].\n"
      " 2) Tensor5DPermute20314: matrix of shape [X, Y] is reshaped as [X/T1, T1, T2, T3, Y/T2/T3] and has its dimensions"
      " permuted as [2, 0, 3, 1, 4], resulting in shape [T2, X/T1, T3, T1, Y/T2/T3] viewed as matrix of shape [X*T2/T1, Y*T1/T2].\n"
       "\n"
      " Batched GEMM:\n"
      " 3) Tensor4DPermuteBMM0213: batched tensor of 3D shape [B, X, Y] is reshaped as 4D shape [B/D1, D1, X, Y]"
      " and has its dimensions permuted as [0, 2, 1, 3], resulting in shape [B/D1, X, D1, Y] viewed as"
      " a matrix of shape [B/D1, X, Y*D1] for batched GEMM purposes.\n"
      "\n"
      " Note: S1, S2, D1, D2, T1, T2, T3 are compile-time constants defined in gemm_permute.cu."
      " Runtime specification of these values is not supported."
      " These values along with alignment requirements place constraints on supported matrix sizes.\n"
      "\n"
      " Note: X, Y above may refer to M, N or K dimensions of GEMM problem, depending on the tensor considered (A, B or D)."
      " For the output tensor D the values correspond directly to dimensions of D, whereas for A and B the original dimensions"
      " X', Y' are inferred from the ones supplied to the GEMM, taking into account the permute operation.\n"
      "\n"
      "Options:\n"
      "\n"
      "  --help                      If specified, displays this usage statement.\n\n"
      "  --batch-count=<int>         Sets the number of batches in batched GEMM (batch number for BMM). (default: --batch-count=768)\n"
      "  --m=<int>                   Sets the M dimension for both batched GEMM and normal GEMM problems. (default: --m=128)\n"
      "  --n=<int>                   Sets the N dimension for both batched GEMM and normal GEMM problems. (default: --n=192)\n"
      "  --k=<int>                   Sets the K dimension for both batched GEMM and normal GEMM problems. (default: --k=384)\n"
      "  --alpha=<f32>               Epilogue scalar alpha (real part)\n"
      "  --beta=<f32>                Epilogue scalar beta (real part)\n\n"
      "  --iterations=<int>          Number of profiling iterations to perform.\n"
      "  --reference-check=<bool>    If true, performs reference check.\n"
      "  --verbose=<bool>            If true, prints problem sizes and batching structure.\n"
      "\n"
      "Examples:\n"
      "\n"
      "# Runs a batched GEMM with 96 batches\n"
      "$ ./examples/39_gemm_permute/39_gemm_permute --batch-count=96\n"
      "\n"
      "# Runs a batched GEMM with 96 batches (with GEMM-K dimension equal to 1024)\n"
      "$ ./examples/39_gemm_permute/39_gemm_permute --batch-count=96 --k=1024 --verbose=true\n"
      "\n"
      "# Execute batched GEMM and profile with NSight\n"
      "$ nv-nsight-cu-cli ./examples/39_gemm_permute/39_gemm_permute --m=256 --n=192 --k=256 --verbose=true --iterations=1 --reference-check=false\n"
      "\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s, bool batched) const {

    // Number of real-valued multiply-adds 
    int64_t fmas = int64_t();

    fmas += problem_each.product() * (batched ? batch_count : 1);
    
    // Two flops per multiply-add
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

namespace { // (anonymous)

/// Dimension-generic permutation loop
template<int I, typename Element, typename Layout, typename PermuteOp, typename Coord>
void permute_host_impl(
    cutlass::TensorView<Element const, Layout> const & input,
    cutlass::TensorView<Element, Layout> const & output,
    PermuteOp && permute,
    Coord & coord
) {
  static_assert(Layout::kRank == Coord::kRank, "Incompatible Layout and Coord types");
  if constexpr (I == Coord::kRank) {
    output.at(permute(coord)) = input.at(coord);
  }
  else {
    for (coord[I] = 0; coord[I] < input.extent(I); ++coord[I]) {
      permute_host_impl<I+1>(input, output, std::forward<PermuteOp>(permute), coord);
    }
  }
}

} // namespace (anonymous)

/// Perform a reference (host-based) permutation of an input tensor
template<typename PermuteLayout, typename Element, typename Layout>
void permute_host(
    cutlass::TensorView<Element const, Layout> const &input,
    cutlass::TensorView<Element, Layout> const &output,
    int batch_count) {
  Layout layout = input.layout();
  cutlass::MatrixCoord extent = input.extent();

  std::size_t num_elems = layout.capacity(extent) * batch_count;
  std::vector<Element> h_input(num_elems);
  cutlass::device_memory::copy_to_host(h_input.data(), input.data(), num_elems);

  std::vector<Element> h_output(num_elems);

  using Info = PermuteInfo<PermuteLayout>;
  using TensorLayout = typename Info::Layout;

  auto shape_orig = Info::original_shape(extent, batch_count);
  auto shape_perm = Info::permute(shape_orig);

  cutlass::TensorView<Element const, TensorLayout> view_input(h_input.data(), TensorLayout::packed(shape_orig), shape_orig); 
  cutlass::TensorView<Element, TensorLayout> view_output(h_output.data(), TensorLayout::packed(shape_perm), shape_perm);

  decltype(shape_orig) coord;
  permute_host_impl<0>(view_input, view_output, Info::permute, coord);

  cutlass::device_memory::copy_to_device(output.data(), h_output.data(), num_elems);
}

template<typename Layout>
struct LayoutInfo;

template<>
struct LayoutInfo<cutlass::layout::RowMajor> {
  static std::string name() { return "RowMajor"; }
};

template<>
struct LayoutInfo<cutlass::layout::ColumnMajor> {
  static std::string name() { return "ColumnMajor"; }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

template <typename ElementA, typename ElementB, typename ElementC>
class Testbed {
private:

  //
  // Data members
  //

  Options & options;

  /// Initialization
  cutlass::Distribution::Kind init_A;
  cutlass::Distribution::Kind init_B;
  cutlass::Distribution::Kind init_C;
  uint32_t seed;

  cutlass::DeviceAllocation<ElementA> block_A;
  cutlass::DeviceAllocation<ElementB> block_B;
  cutlass::DeviceAllocation<ElementC> block_C;
  cutlass::DeviceAllocation<ElementC> block_D;

public:

  //
  // Methods
  //

  Testbed(
    Options &options_,
    cutlass::Distribution::Kind init_A_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_B_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_C_ = cutlass::Distribution::Uniform,
    uint32_t seed_ = 3090
  ):
    options(options_), init_A(init_A_), init_B(init_B_), init_C(init_C_), seed(seed_) { }

private:

  /// Print permutation info for one tensor
  template<typename PermuteLayout>
  void print_tensor_info(
      std::ostream & os,
      std::string const &tensor_name,
      int row_dim,
      int col_dim) {

    cutlass::MatrixCoord extent(options.problem_each.at(row_dim), options.problem_each.at(col_dim));
    using Info = PermuteInfo<PermuteLayout>;

    os << "tensor " << tensor_name << ": " << Info::desc() << "\n";
    os << "    extent: [" << extent.row() << ", " << extent.column() << "]";
    if (Info::kBatched) {
      os << ", batch count: " << options.batch_count;
    }
    os << "\n";
    if (!cutlass::layout::is_trivial_permute<PermuteLayout>) {
      auto shape_orig = Info::original_shape(extent, options.batch_count);
      auto shape_perm = Info::permute(shape_orig);
      os << "  original: [" << shape_orig << "]\n";
      os << "  permuted: [" << shape_perm << "]\n";
    }
  }

  /// Check shape compatibility for one tensor
  template<typename Layout, typename PermuteLayout, int Alignment>
  bool check_tensor_shape(
      std::string const &tensor_name,
      int row_dim,
      int col_dim) {

    cutlass::MatrixCoord extent(options.problem_each.at(row_dim), options.problem_each.at(col_dim));

    using Info = PermuteInfo<PermuteLayout>;

    auto rowAlign = cutlass::platform::is_same<Layout, cutlass::layout::ColumnMajor>::value ? Alignment : 1;
    auto colAlign = cutlass::platform::is_same<Layout, cutlass::layout::RowMajor>::value ? Alignment : 1;

    auto rowFactor = Info::kRowFactor * rowAlign;
    auto colFactor = Info::kColumnFactor * colAlign;

    // Assumes row-major layout
    bool const valid_row = extent.row() % rowFactor == 0;
    if (!valid_row) {
      std::cerr << "Invalid tensor " << tensor_name << " row size = " << extent.row() << ", "
                   "must be divisible by " << rowFactor << ", "
                   "required by " << Info::name() << 
                   (rowAlign > 1 ? (" and alignment of " + std::to_string(rowAlign)) : "") << std::endl;
    }

    bool const valid_col = extent.column() % colFactor == 0;
    if (!valid_col) {
      std::cerr << "Invalid tensor " << tensor_name << " column size = " << extent.column() << ", "
                   "must be divisible by " << colFactor << ", "
                   "required by " << Info::name() << 
                   (colAlign > 1 ? (" and alignment of " + std::to_string(colAlign)) : "") << std::endl;
    }

    bool const valid_bsz = options.batch_count % Info::kBatchFactor == 0;
    if (!valid_bsz) {
      std::cerr << "Invalid batch count = " << options.batch_count << ", "
                   "must be divisible by " << Info::kBatchFactor << ", "
                   "required by " << Info::name() << std::endl;
    }

    return valid_row && valid_col && valid_bsz;
  }

  /// Helper to initialize a tensor view
  template <typename Element>
  void initialize_tensor_(
      Element *ptr,
      size_t capacity, 
      cutlass::Distribution::Kind dist_kind,
      uint32_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      Element scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<ElementC>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        if (cutlass::sizeof_bits<ElementAccumulator>::value <= 16) {
          scope_max = 5;
          scope_min = -5;
        }
        else {
          scope_max = 8;
          scope_min = -8;
        }
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::device::BlockFillRandomUniform(
        ptr, capacity, seed, scope_max, scope_min, 0);
    } 
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::device::BlockFillRandomGaussian(
        ptr, capacity, seed, Element(), Element(0.5f));
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      // Fill with increasing elements
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(1), Element());
    } 
    else {

      // Fill with all 1s
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(), Element(1));
    }
  }

  /// Initializes data structures
  void initialize(int batch_count) {

    srand(seed);

    int64_t total_elements_A = options.problem_each.m() * options.problem_each.k() * batch_count;
    int64_t total_elements_B = options.problem_each.n() * options.problem_each.k() * batch_count;
    int64_t total_elements_C = options.problem_each.m() * options.problem_each.n() * batch_count;
    int64_t total_elements_D = options.problem_each.m() * options.problem_each.n() * batch_count;

    // Allocate space
    block_A.reset(total_elements_A);
    block_B.reset(total_elements_B);
    block_C.reset(total_elements_C);
    block_D.reset(total_elements_D);

    // Initialize input tensors
    initialize_tensor_(block_A.get(), total_elements_A, init_A, seed * 2021);
    initialize_tensor_(block_B.get(), total_elements_B, init_B, seed * 2022);
    initialize_tensor_(block_C.get(), total_elements_C, init_C, seed * 2023);

    cutlass::reference::device::BlockFillSequential(
      block_D.get(), total_elements_D, ElementC(), ElementC());
  }


  /// Check device GEMM results against a reference implementation with separate host-based permutation
  template<typename Gemm>
  bool validate(Gemm const &gemm) {

    bool constexpr kBatched = PermuteInfo<typename Gemm::PermuteALayout>::kBatched 
                           || PermuteInfo<typename Gemm::PermuteBLayout>::kBatched 
                           || PermuteInfo<typename Gemm::PermuteDLayout>::kBatched;
                      
    int const batch_count = kBatched ? options.batch_count : 1;

    cutlass::gemm::GemmCoord problem = options.problem_each;

    cutlass::MatrixCoord extent_A{problem.m(), problem.k()};
    cutlass::MatrixCoord extent_B{problem.k(), problem.n()};
    cutlass::MatrixCoord extent_C{problem.m(), problem.n()};

    using LayoutA = typename Gemm::LayoutA;
    using LayoutB = typename Gemm::LayoutB;
    using LayoutC = typename Gemm::LayoutC;

    LayoutA layout_A(LayoutA::packed(extent_A));
    LayoutB layout_B(LayoutB::packed(extent_B));
    LayoutC layout_C(LayoutC::packed(extent_C));

    auto size_A = layout_A.capacity(extent_A) * batch_count;
    auto size_B = layout_B.capacity(extent_B) * batch_count;
    auto size_C = layout_C.capacity(extent_C) * batch_count;
    
    cutlass::TensorView<ElementA, LayoutA> view_A(block_A.get(), layout_A, extent_A);
    cutlass::TensorView<ElementB, LayoutB> view_B(block_B.get(), layout_B, extent_B);
    cutlass::TensorView<ElementC, LayoutC> view_C(block_C.get(), layout_C, extent_C);
    cutlass::TensorView<ElementC, LayoutC> view_D(block_D.get(), layout_C, extent_C);

    cutlass::DeviceAllocation<ElementA> block_A_perm(size_A);
    cutlass::DeviceAllocation<ElementA> block_B_perm(size_B);

    cutlass::TensorView<ElementA, LayoutA> view_A_perm(block_A_perm.get(), layout_A, extent_A);
    cutlass::TensorView<ElementB, LayoutB> view_B_perm(block_B_perm.get(), layout_B, extent_B);

    permute_host<typename Gemm::PermuteALayout>(view_A.const_view(), view_A_perm, batch_count);
    permute_host<typename Gemm::PermuteBLayout>(view_B.const_view(), view_B_perm, batch_count);

    cutlass::DeviceAllocation<ElementC>    block_D_ref(size_C);
    cutlass::TensorView<ElementC, LayoutC> view_D_ref(block_D_ref.get(), layout_C, extent_C);

    using EpilogueOutputOp = typename Gemm::GemmKernel::Epilogue::OutputOp;

    // Reference GEMM
    cutlass::reference::device::GemmComplex<
        ElementA, LayoutA,
        ElementB, LayoutB,
        ElementC, LayoutC, 
        typename EpilogueOutputOp::ElementCompute,
        typename Gemm::ElementAccumulator
    >(
      problem,
      options.alpha, 
      view_A_perm,
      Gemm::kTransformA,
      view_B_perm,
      Gemm::kTransformB,
      options.beta, 
      view_C, 
      view_D_ref, 
      ElementAccumulator(0),
      batch_count,
      options.problem_each.m() * options.problem_each.k(),
      options.problem_each.n() * options.problem_each.k(),
      options.problem_each.m() * options.problem_each.n(),
      options.problem_each.m() * options.problem_each.n()
    );

    cutlass::DeviceAllocation<ElementC>    block_D_perm(size_C);
    cutlass::TensorView<ElementC, LayoutC> view_D_perm(block_D_perm.get(), layout_C, extent_C);
    permute_host<typename Gemm::PermuteDLayout>(view_D_ref.const_view(), view_D_perm, batch_count);

    // Reference check
    return cutlass::reference::device::BlockCompareEqual(view_D_perm.data(), view_D.data(), size_C);
}

public:

  template<typename Gemm>
  bool profile_GEMM_permute() {

    using LayoutA = typename Gemm::LayoutA;
    using LayoutB = typename Gemm::LayoutB;
    using LayoutC = typename Gemm::LayoutC;

    using PermuteALayout = typename Gemm::PermuteALayout;
    using PermuteBLayout = typename Gemm::PermuteBLayout;
    using PermuteDLayout = typename Gemm::PermuteDLayout;

    bool constexpr kBatched = PermuteInfo<PermuteALayout>::kBatched 
                           || PermuteInfo<PermuteBLayout>::kBatched 
                           || PermuteInfo<PermuteDLayout>::kBatched;

    std::cout << "\n"
                 "====================================================\n"
                 << (kBatched ? "Batched" : "Normal") << " GEMM:"
                 << "\n  A=" << LayoutInfo<LayoutA>::name() << "," << PermuteInfo<PermuteALayout>::name()
                 << "\n  B=" << LayoutInfo<LayoutB>::name() << "," << PermuteInfo<PermuteBLayout>::name()
                 << "\n  D=" << LayoutInfo<LayoutC>::name() << "," << PermuteInfo<PermuteDLayout>::name()
                 << "\n"
                 "====================================================\n";

    if (options.verbose) {
      print_tensor_info<PermuteALayout>(std::cout, "A", 0, 2);
      print_tensor_info<PermuteBLayout>(std::cout, "B", 2, 1);
      print_tensor_info<PermuteDLayout>(std::cout, "D", 0, 1);
    }
    std::cout << std::endl;

    bool valid = true;
    valid &= check_tensor_shape<LayoutA, PermuteALayout, Gemm::kAlignmentA>("A", 0, 2);
    valid &= check_tensor_shape<LayoutB, PermuteBLayout, Gemm::kAlignmentB>("B", 2, 1);
    valid &= check_tensor_shape<LayoutC, PermuteDLayout, Gemm::kAlignmentC>("D", 0, 1);
    if (!valid)
    {
      std::cout << "Skipped test" << std::endl;
      return true;
    }

    int const batch_count = kBatched ? options.batch_count : 1;

    // Initialize the problem
    initialize(batch_count);

    // Configure the GEMM arguments
    using EpilogueOutputOp = typename Gemm::GemmKernel::Epilogue::OutputOp;
    typename EpilogueOutputOp::Params epilogue_op(options.alpha, options.beta);

    // Please make sure all problem_sizes are the same for kBatched mode
    auto problem = options.problem_each;

    cutlass::MatrixCoord extent_A{problem.m(), problem.k()};
    cutlass::MatrixCoord extent_B{problem.k(), problem.n()};
    cutlass::MatrixCoord extent_C{problem.m(), problem.n()};

    LayoutA layout_A(LayoutA::packed(extent_A));
    LayoutB layout_B(LayoutB::packed(extent_B));
    LayoutC layout_C(LayoutC::packed(extent_C));

    // Configure GEMM arguments
    typename Gemm::Arguments arguments{
      kBatched ? cutlass::gemm::GemmUniversalMode::kBatched : cutlass::gemm::GemmUniversalMode::kGemm,
      problem,
      batch_count,
      epilogue_op,
      (void*)block_A.get(),
      (void*)block_B.get(),
      (void*)block_C.get(),
      (void*)block_D.get(),
      // For any non-trivial permute the batch stride must be set to 0
      cutlass::layout::is_trivial_permute<PermuteALayout> ? layout_A.capacity(extent_A) : 0,
      cutlass::layout::is_trivial_permute<PermuteBLayout> ? layout_B.capacity(extent_B) : 0,
      layout_C.capacity(extent_C),
      cutlass::layout::is_trivial_permute<PermuteDLayout> ? layout_C.capacity(extent_C) : 0,
      layout_A.stride(0),
      layout_B.stride(0),
      layout_C.stride(0),
      layout_C.stride(0),
    };

    // Initialize the GEMM object
    Gemm gemm_normal;

    CHECK_CUTLASS_CALL(gemm_normal.initialize(arguments, nullptr), return false);

    // Run the normal GEMM object
    CHECK_CUTLASS_CALL(gemm_normal.run(), return false);

    // Wait for completion
    CHECK_CUDA_CALL(hipDeviceSynchronize(), return false);

    //
    // Verify correctness
    //
    if (options.reference_check) {
      if (validate(gemm_normal)) {
        std::cout << "\nPassed verification\n" << std::endl;
      }
      else {
        std::cerr << "\n*** Error - problem failed the QA check ***\n" << std::endl;
        return false;
      }
    }

    // Warm-up run of the normal GEMM object
    CHECK_CUTLASS_CALL(gemm_normal.run(), return false);

    // Construct events
    hipEvent_t events[2];
    for (auto & event : events) {
      CHECK_CUDA_CALL(hipEventCreate(&event), return false);
    }

    // Record an event at the start of a series of GEMM operations
    CHECK_CUDA_CALL(hipEventRecord(events[0]), return false);

    // Run profiling loop
    for (int iter = 0; iter < options.iterations; ++iter) {
      gemm_normal();
    }

    // Record an event when the GEMM operations have been launched.
    CHECK_CUDA_CALL(hipEventRecord(events[1]), return false);

    // Wait for work on the device to complete.
    CHECK_CUDA_CALL(hipEventSynchronize(events[1]), return false);

    // Measure elapsed runtime
    float runtime_total_ms = 0;
    CHECK_CUDA_CALL(hipEventElapsedTime(&runtime_total_ms, events[0], events[1]), return false);

    // Compute average runtime and GFLOPs.
    double runtime_avg_ms = double(runtime_total_ms) / double(options.iterations);
    double gflops = options.gflops(runtime_avg_ms / 1000.0, kBatched);

    // Cleanup
    for (auto event : events) {
      CHECK_CUDA_CALL(hipEventDestroy(event), return false);
    }

    std::cout << "    Runtime: " << runtime_avg_ms << " ms\n"
                 "     GFLOPs: " << gflops << std::endl;

    return true;
  }
};

/// Shorthand alist for GEMM instantiations
template<typename LayoutA, typename PermuteALayout,
         typename LayoutB, typename PermuteBLayout,
         typename LayoutC, typename PermuteDLayout>
using GemmPermute = cutlass::gemm::device::GemmUniversal<
  ElementInput, LayoutA,
  ElementInput, LayoutB,
  ElementOutput, LayoutC,
  ElementAccumulator,
  cutlass::arch::OpClassTensorOp,
  cutlass::arch::Sm80,
  cutlass::gemm::GemmShape<128, 128, 32>,
  cutlass::gemm::GemmShape<64, 64, 32>,
  cutlass::gemm::GemmShape<16, 8, 16>,
  cutlass::epilogue::thread::LinearCombination<
    ElementOutput, 
    AlignmentC, //128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator, 
    ElementAccumulator
  >,
  cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
  4,              /*kStages*/
  AlignmentA,     /*AlignmentA*/
  AlignmentB,     /*AlignmentB*/
  cutlass::arch::OpMultiplyAdd,
  cutlass::ComplexTransform::kNone,
  cutlass::ComplexTransform::kNone,
  false,  /*GatherA*/
  false,  /*GatherB*/
  false,  /*ScatterD*/
  PermuteDLayout,  /*PermuteDLayout*/
  typename cutlass::layout::InversePermute<PermuteALayout>::type,  /*PermuteALayout*/
  typename cutlass::layout::InversePermute<PermuteBLayout>::type   /*PermuteBLayout*/
>;

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  //
  // This example uses mma.sync to directly access Tensor Cores to achieve peak performance.
  //

  hipDeviceProp_t props;

  CHECK_CUDA_CALL(hipGetDeviceProperties(&props, 0), return EXIT_FAILURE);

  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
  
    //
    // This example requires an NVIDIA Ampere-architecture GPU.
    //

    std::cout << "CUTLASS's GEMM+Permute example requires a GPU of NVIDIA's Ampere Architecture "
                 "or later (compute capability 80 or greater).\n";

    return EXIT_SUCCESS;
  }

  //
  // Parse options
  //

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return EXIT_SUCCESS;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return EXIT_FAILURE;
  }

  //
  // Define GEMM types to test
  //

  //
  // TTT (Row-major) GEMMs
  //

  using TTTGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteA = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using TTTGemmNormalPermuteB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using TTTGemmNormalPermuteD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using TTTGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  //
  // NNN (Col-major) GEMMs
  //

  using NNNGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteA = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using NNNGemmNormalPermuteB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using NNNGemmNormalPermuteD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using NNNGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  //
  // NNT (Col-major inputs, row-major output) GEMMs
  //

  using NNTGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteA = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using NNTGemmNormalPermuteB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using NNTGemmNormalPermuteD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using NNTGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  //
  // TTN (Row-major inputs, col-major output) GEMMs
  //

  using TTNGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteA = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using TTNGemmNormalPermuteB = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using TTNGemmNormalPermuteD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using TTNGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  //
  // TTT (Row-major) BMMs
  //

  using TTTGemmBatchedPermuteA = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmBatchedPermuteAD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmBatchedPermuteBD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteAB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteABD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  //
  // NNN (Col-major) BMMs
  //

  using NNNGemmBatchedPermuteA = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmBatchedPermuteAD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  using NNNGemmBatchedPermuteB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmBatchedPermuteBD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  using NNNGemmBatchedPermuteD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  using NNNGemmBatchedPermuteAB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmBatchedPermuteABD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  //
  // Profile it
  //

  Testbed<ElementInput, ElementInput, ElementOutput> testbed(options);

  bool result = true;

  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteA>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteAD>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteB>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteBD>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteD>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteAB>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteABD>();

  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteA>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteAD>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteB>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteBD>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteD>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteAB>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteABD>();

  std::cout << "\n"
               "====================================================\n"
               "Finished (" << (result ? "PASS" : "FAIL") << ")\n"
               "====================================================" << std::endl;

  return result ? EXIT_SUCCESS : EXIT_FAILURE;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
