#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief  Hopper GEMM + Top-K + Softmax fusion

    This example illustrates how to use the LinCombTopKSoftmaxCol EVT node to fuse
    Top-K and Softmax into the GEMM epilogue, with certain assumptions made.

    Those assumptions are as:
      1. Fusion is over the N dimension.
      2. Top-K is either 2 or 4 elements, and the value is static (meaning two kernels have to be
         compiled to support both.)
      3. The GEMM tile shape along N is greater than or equal to problem size
         along N.


    The example runs the fused GEMM kernel, along with a standard unfused host reference, and
    manually performs Top-K and softmax, and compares the error between tensors.

    Note that some numerical error (smaller than 1e-5) is to be expected, but this is true
    in most efficient reduction kernels, because floating point addition is not necessarily
    associative.
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_types.h"

#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/error_metrics.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gett.hpp"


#include "helper.h"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

static constexpr int TopK = 2;
static constexpr bool EnableTopKSoftmax = TopK > 1;

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = cutlass::half_t;                                // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = cutlass::half_t;                                // Element type for B matrix operand
using         LayoutB     = cutlass::layout::ColumnMajor;                   // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C matrix configuration
using         ElementC    = void;
using         LayoutC     = cutlass::layout::RowMajor;
constexpr int AlignmentC  = 1;

// D matrix configuration
using         ElementD    = cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutD     = cutlass::layout::RowMajor;                      // Layout type for output
constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;    // Memory access granularity/alignment of output in units of elements (up to 16 bytes)

// Core kernel configurations
using ElementAccumulator  = float;                                          // Element type for internal accumulation
using ElementCompute      = float;                                          // Element type for epilogue computation
using ArchTag             = cutlass::arch::Sm90;                            // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;                 // Operator class tag
using TileShape           = Shape<_64,_64,_128>;                            // Threadblock-level tile size
using ClusterShape        = Shape<_1,_1,_1>;                                // Shape of the threadblocks in a cluster
using KernelSchedule      = cutlass::gemm::KernelTmaWarpSpecialized;
using EpilogueSchedule    = cutlass::epilogue::TmaWarpSpecialized;

// Top-K + Softmax fusion operation
using FusionOperation     = std::conditional_t<EnableTopKSoftmax,
  typename cutlass::epilogue::fusion::LinCombTopKSoftmaxCol<TopK, ElementD, ElementCompute>,
  typename cutlass::epilogue::fusion::LinearCombination<ElementD, ElementCompute, ElementC, ElementCompute>
>;

// The fusion op only allows for epilogue tiles matching the mainloop tile.
using EpilogueTileType    = decltype(cute::take<0,2>(TileShape{}));

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    TileShape, ClusterShape,
    EpilogueTileType,
    ElementAccumulator, ElementCompute,
    ElementC, LayoutC, AlignmentC,
    ElementD, LayoutD, AlignmentD,
    EpilogueSchedule,
    FusionOperation
  >::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag, OperatorClass,
    ElementA, LayoutA, AlignmentA,
    ElementB, LayoutB, AlignmentB,
    ElementAccumulator,
    TileShape, ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<
      static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))
    >,
    KernelSchedule
  >::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
    Shape<int,int,int,int>, // Indicates ProblemShape
    CollectiveMainloop,
    CollectiveEpilogue
>;

using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Extract information from Gemm kernel.
using EpilogueOutputOp  = typename Gemm::EpilogueOutputOp;
using ElementScalar     = typename EpilogueOutputOp::ElementScalar;

using StrideA = typename Gemm::GemmKernel::StrideA;
using StrideB = typename Gemm::GemmKernel::StrideB;
using StrideD = typename Gemm::GemmKernel::StrideD;

/// Initialization
StrideA stride_A;
StrideB stride_B;
StrideD stride_D;
uint64_t seed;

cutlass::HostTensor<ElementA  , LayoutA  > tensor_A;
cutlass::HostTensor<ElementB  , LayoutB  > tensor_B;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_D;
cutlass::HostTensor<ElementD  , LayoutD  > tensor_ref_D;

using LayoutScalar = cutlass::layout::PackedVectorLayout;

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
struct Options {

  bool help = false;

  int iterations = 1000;
  int m = 16, n = 8, k = 64, l = 1;
  double eps = 1e-5;

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("m", m);
    cmd.get_cmd_line_argument("n", n);
    cmd.get_cmd_line_argument("k", k);
    cmd.get_cmd_line_argument("l", l);
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("eps", eps);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "61_hopper_gemm_with_topk_and_softmax\n\n"
      << "  Hopper FP8 GEMM with Top-K and softmax fusion.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --m=<int>                   Sets the M extent of the GEMM\n"
      << "  --n=<int>                   Sets the N extent of the GEMM\n"
      << "  --k=<int>                   Sets the K extent of the GEMM\n"
      << "  --l=<int>                   Sets the l extent (batch) of the GEMM\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n"
      << "  --eps=<float>               Threshold of numerical verification. Default: 1e-5.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << "61_hopper_gemm_with_topk_and_softmax" << " --m=16 --n=8 --k=1024 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * m * n * k;
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
  }

  float alpha() const {
    return 1.f / static_cast<float>(k);
  }
};

/// Result structure
struct Result {
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(false)
  {}

};

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM setup and evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <typename Element, typename Layout>
bool initialize_tensor(
    cutlass::TensorView<Element, Layout> view,
    uint64_t seed) {
  cutlass::reference::host::TensorFillRandomUniform(
    view, seed, /* max = */ 1, /* min = */ -1, /* bits = */ 2);
  return true;
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const Options &options) {

  stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(options.m, options.k, options.l));
  stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(options.n, options.k, options.l));
  stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(options.m, options.n, options.l));

  auto a_coord = cutlass::make_Coord(options.m * options.l, options.k);
  auto c_coord = cutlass::make_Coord(options.m * options.l, options.n);
  auto b_coord = cutlass::make_Coord(options.k, options.n * options.l);

  tensor_A.resize(a_coord);
  tensor_B.resize(b_coord);
  tensor_D.resize(c_coord);
  tensor_ref_D.resize(c_coord);

  initialize_tensor(tensor_A.host_view(), seed + 2022);
  initialize_tensor(tensor_B.host_view(), seed + 2023);

  tensor_A.sync_device();
  tensor_B.sync_device();
  tensor_D.sync_device();
}

/// Populates a Gemm::Arguments structure from the given commandline options
typename Gemm::Arguments args_from_options(const Options &options) {
  typename Gemm::Arguments arguments{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {options.m, options.n, options.k, options.l},
    {tensor_A.device_data(), stride_A, tensor_B.device_data(), stride_B},
    {
      {options.alpha(), 0.f}, // alpha, beta
      nullptr, stride_D,
      tensor_D.device_data(), stride_D
    }
  };

  return arguments;
}

bool verify(const Options &options) {
  //
  // Compute reference output
  //

  // Create instantiation for device reference gemm kernel
  auto A = cute::make_tensor(tensor_A.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.k, options.l), stride_A));
  auto B = cute::make_tensor(tensor_B.host_data(),
      cute::make_layout(cute::make_shape(options.n, options.k, options.l), stride_B));
  auto D = cute::make_tensor(tensor_ref_D.host_data(),
      cute::make_layout(cute::make_shape(options.m, options.n, options.l), stride_D));
  using unused_t = decltype(D);

  cutlass::reference::host::GettMainloopParams<ElementAccumulator, decltype(A), decltype(B)> mainloop_params{A, B};

  cutlass::reference::host::GettEpilogueParams<
      ElementScalar,
      ElementScalar,
      ElementAccumulator,
      ElementCompute,
      unused_t,
      decltype(D),
      unused_t, // bias
      unused_t, // aux
      unused_t, // valpha
      unused_t  // vbeta
  > epilogue_params;

  epilogue_params.D = D;
  epilogue_params.alpha = options.alpha();
  epilogue_params.beta = 0.f;

  // get reference result
  cutlass::reference::host::Gemm3x(mainloop_params, epilogue_params);

  if constexpr (EnableTopKSoftmax) {
    // top-K + softmax
    for (int i = 0; i < options.m; ++i) {

      // Find Top-K
      cutlass::Array<ElementAccumulator, TopK> top_k;
      top_k.fill(-cutlass::platform::numeric_limits<ElementCompute>::infinity());
      for (int j = 0; j < options.n; ++j) {
        auto val = static_cast<ElementAccumulator>(tensor_ref_D.host_view().ref().at({i, j}));
        for (int top_k_idx = 0; top_k_idx < TopK; ++top_k_idx) {
          if (val > top_k[top_k_idx]) {
            // Shift down
            for (int l = TopK - 1; l > top_k_idx; --l) {
              top_k[l] = top_k[l - 1];
            }
            top_k[top_k_idx] = val;
            break;
          }
        }
      }

      // This formulation of top-K + softmax only works when it is
      // guaranteed that none of the top-K elements are repeated!
      // If this is the case, the device kernel can also make mistakes, because
      //   A. Once the top-K values are reduced, and the operation is being applied,
      //      there is no way to tell repeated elements apart, so none are masked.
      //   B. The softmax sum of exps will be incorrect (because the repeated elements
      //      are not repeated in it.)

      ElementAccumulator max = top_k[0];
      ElementAccumulator sum = ElementAccumulator(0.f);
      for (int top_k_idx = 0; top_k_idx < TopK; ++top_k_idx) {
        sum = sum + cutlass::fast_exp(top_k[top_k_idx] - max);
      }

      for (int j=0; j < options.n; ++j) {
        auto val = tensor_ref_D.host_view().ref().at({i, j});
        if (val < top_k[TopK - 1]) {
          tensor_ref_D.host_view().ref().at({i, j}) = static_cast<ElementD>(0.f);
        } else {
          // Softmax
          auto softmax_val = cutlass::fast_exp(val - max) / sum;
          tensor_ref_D.host_view().ref().at({i, j}) = static_cast<ElementD>(softmax_val);
        }
      }
    }
  }

  // compare_reference
  tensor_D.sync_host();

  double err = cutlass::reference::host::TensorRelativeErrorMetric(
    tensor_D.host_view(),
    tensor_ref_D.host_view());
  bool passed = err < options.eps;

  if (options.m <= 32 && options.n <= 32) {
    std::cout << "GEMM output:\n" << tensor_D.host_view() << "\n\n";
    std::cout << "Reference output:\n" << tensor_ref_D.host_view() << "\n\n";
  }

  std::cout << "  Disposition: " << (passed ? "Passed" : "Failed") << " \t Relative error: " << err << std::endl;

  return passed;
}

/// Execute a given example GEMM computation
template <typename Gemm>
int run(Options &options) {
  initialize(options);

  // Instantiate CUTLASS kernel depending on templates
  Gemm gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
  auto arguments = args_from_options(options);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check if the problem size is supported or not
  CUTLASS_CHECK(gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(gemm.run());

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = verify(options);

  if (!result.passed) {
    exit(-1);
  }

  // Run profiling loop
  if (options.iterations > 0) {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(gemm.run());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Problem Size: " << options.m << 'x' << options.n << 'x' << options.k << 'x' << options.l << std::endl;
    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPS: " << result.gflops << std::endl;
  }

  return 0;
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {

  // CUTLASS must be compiled with CUDA 12.0 Toolkit to run this example
  // and must have compute capability at least 90.
  if (__CUDACC_VER_MAJOR__ < 12) {
    std::cerr << "This example requires CUDA 12 or newer.\n";
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (props.major < 9) {
    std::cerr
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater).\n";
    return 0;
  }
  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  //
  // Evaluate CUTLASS kernels
  //

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  run<Gemm>(options);
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
