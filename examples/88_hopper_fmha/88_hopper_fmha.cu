#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Example implementation of fused multi-head attention for Hopper using CUTLASS 3.

    This example showcases the use of CUTLASS to build forward and backward fused
    multi-head attention (FMHA) collectives from existing CUTLASS collectives targeting
    the NVIDIA Hopper architecture.

    Background and motivation
    -------------------------
    CUTLASS is a highly flexible library that provides open-source building blocks
    for tensor core programming for GEMM or GEMM-like problems. Fused multi-head
    attention (FMHA) is a foundational kernel for large language models (LLMs) since it
    makes long sequence lengths feasible from a memory-usage perspective. It also
    improves computational efficiency since it transforms an outer-product-like and
    a matrix-vector-like GEMM into a fused operation with much higher arithmetic
    intensity. For more details, see Dao et al, 2022; Dao, 2023.
    Implementing this kernel in CUTLASS enabled easy customization and high
    performance.

    Introduction
    ------------
    The example targets the NVIDIA Hopper architecture, and takes advantage of
    warpgroup-wide tensor cores, the Tensor Memory Accelerator (TMA), just like
    GEMMs do. It provides both a forward and a backward pass (often abbreviated
    fwd and bwd in the code), and an optional FP8 mode for the forward pass.
    The code is structured into four layers: The runner (and the reference kernels)
    takes care of initialization, measurement, and testing; the device layer
    orchestrates kernel calls and partitions workspace; the kernel layer (just
    like the CUTLASS kernel layer); and the collective layer (most of the logic
    of FMHA is implemented here).

    Details
    -------
    This example contains a considerable amount of code. For a more detailed
    look at it, please refer to the README.md.

    Example usage:
      $ ./examples/88_hopper_fmha/88_hopper_fmha \
            --b=2048 --h=2048 --d=2048 --q=2048 --k=2048
*/

#include <iostream>

#include "cute/tensor.hpp"

#include "cutlass/cutlass.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "collective/fmha_fusion.hpp"
#include "device/fmha_device_bwd.hpp"
#include "device/device_universal.hpp"
#include "kernel/fmha_kernel_builder.hpp"
#include "reference/fmha_reference.hpp"
#include "reference/fmha_bwd_reference.hpp"
#include "reference/reference_abs_error.hpp"

using namespace cute;
using namespace cutlass::fmha::kernel;
using namespace cutlass::fmha::collective;
using namespace cutlass::fmha;

// Uncomment for FP8
// #define FP8

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Command line options parsing
struct Options {

  bool help;
  bool error;

  int b, h, q, k, d;
  int iterations;
  bool verify;
  bool verbose;
  bool causal;
  bool residual;
  bool bwd;

  Options():
    help(false),
    error(false),
    b(16), h(16), q(1024), k(1024), d(128),
    iterations(3), verify(false),
    causal(false), residual(false), bwd(false), verbose(false)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("d", d, defaults.d);
    cmd.get_cmd_line_argument("h", h, -1);
    if (h == -1) h = 2048 / d;

    cmd.get_cmd_line_argument("q", q, -1);
    cmd.get_cmd_line_argument("k", k, -1);
    if (q == -1) q = k;
    if (k == -1) k = q;
    if (q == -1 && k == -1) q = k = defaults.q;

    cmd.get_cmd_line_argument("b", b, -1);
    if (b == -1) b = 16384 / k;
    if (b == 0) b = 1;

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
    verify = cmd.check_cmd_line_flag("verify");
    verbose = cmd.check_cmd_line_flag("verbose");

    std::string mask;
    cmd.get_cmd_line_argument<std::string>("mask", mask, "");
    if (mask == "no" || mask == "") {
      causal = residual = false;
    }
    else if (mask == "causal") {
      residual = false;
      causal = true;
    }
    else if (mask == "residual") {
      residual = true;
      causal = false;
    }

    bwd = cmd.check_cmd_line_flag("bwd");
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "88_hopper_fmha\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  fused multi-head attention forward-pass kernels targeting NVIDIA's Hopper architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the B extent\n"
      << "  --h=<int>                   Sets the H extent\n"
      << "  --q=<int>                   Sets the Q extent\n"
      << "  --k=<int>                   Sets the K extent\n"
      << "  --d=<int>                   Sets the D extent\n"
      << "  --iterations=<int>          Benchmarking iterations\n"
      << "  --verify                    Verify results\n"
      << "  --verbose                   Print smem and execution time per kernel\n"
      << "  --mask=<no|residual|causal> Enables masking\n"
      << "  --bwd                       Runs the backwards pass\n"
      << "\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023, bool init_one=false) {

  if (init_one) {
    cutlass::reference::device::BlockFillRandomUniform(
      block.get(), block.size(), seed, (Element) 1, (Element) 1);
  } else {
    cutlass::reference::device::BlockFillRandomGaussian(
      block.get(), block.size(), seed, (Element) 0, (Element) 1);
  }

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleResult {
  bool passed = false;
  bool verified = false;
  float runtime_ms = 0;
  double tflops_s = 0;
  size_t smem_size = 0;
};

///////////////////////////////////////////////////////////////////////////////////////////////////

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

template<
  class TileShape,
  class DispatchPolicy,
  class ActiveFusion,
  class... KernelOptions
>
struct FwdRunner {

#ifdef FP8
  using Element = cutlass::float_e4m3_t;
  using ElementAccumulatorQK = find_option_t<Tag::kAccQK, float, KernelOptions...>;
#else
  using Element = cutlass::half_t;
  using ElementAccumulatorQK = float;
#endif

  using ElementAccumulatorPV = float;

  // B H Q K D
  using ProblemShapeType = cute::tuple<int, int, int, int, int>;

  
  using StrideQ = cute::tuple<int, _1, cute::tuple<int, int>>;  // Q D (B H)
  using StrideK = cute::tuple<int, _1, cute::tuple<int, int>>;  // K D (B H)
  using StrideV = std::conditional_t<sizeof(Element) == 1,
    cute::tuple<_1, int, cute::tuple<int, int>>,
    cute::tuple<int, _1, cute::tuple<int, int>>>;  // K D (B H)
  using StrideO = cute::tuple<int, _1, cute::tuple<int, int>>; // Q D (B H)
  using StrideLSE = cute::tuple<_1, cute::tuple<int, int>>; // Q (B H)

  using Operation = cutlass::device::Universal<
    typename cutlass::fmha::kernel::FmhaBuilder<
      Element, ElementAccumulatorQK, ElementAccumulatorPV,
      TileShape, StrideQ, StrideK, StrideV,
      ActiveFusion, DispatchPolicy, KernelOptions...
    >::Kernel>;

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<Element> block_Q;
  cutlass::DeviceAllocation<Element> block_K;
  cutlass::DeviceAllocation<Element> block_V;
  cutlass::DeviceAllocation<Element> block_O;
  cutlass::DeviceAllocation<ElementAccumulatorPV> block_LSE;
  cutlass::DeviceAllocation<Element> block_ref_O;
  cutlass::DeviceAllocation<ElementAccumulatorPV> block_ref_LSE;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      make_shape(Q, D, make_shape(B, H)),
      stride_Q);

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      make_shape(K, D, make_shape(B, H)),
      stride_K);

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      make_shape(K, D, make_shape(B, H)),
      stride_V);

    Tensor mO = make_tensor(make_gmem_ptr(block_ref_O.get()),
      make_shape(Q, D, make_shape(B, H)),
      stride_O);

    Tensor mLSE = make_tensor(make_gmem_ptr(block_ref_LSE.get()),
      make_shape(Q, make_shape(B, H)),
      stride_LSE);

    fmha_reference(problem_size, mQ, mK, mV, mO, mLSE, ActiveFusion{});
    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    const double kMaxDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-2;
    const double kMeanDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-3;

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_O, block_ref_O, max_diff, mean_diff);
    bool passed_O = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_O) {
      std::cerr << "failed O: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_LSE, block_ref_LSE, max_diff, mean_diff);
    bool passed_LSE = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if ( ! passed_LSE) {
      std::cerr << "failed LSE: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    return passed_O && passed_LSE;
  }

  void initialize_stride(cute::tuple<int, int, int> const& shape, cute::tuple<_1, cute::tuple<int, int>>& stride) {
    auto [B, H, Q] = shape;
    stride = make_stride(_1{}, make_stride(H*Q, Q));
  }

  void initialize_stride(cute::tuple<int, int, int, int> const& shape, cute::tuple<int, _1, cute::tuple<int, int>>& stride) {
    auto [B, H, Q, D] = shape;
    stride = make_stride(D, _1{}, make_stride(H*Q*D, Q*D));
  }

  void initialize_stride(cute::tuple<int, int, int, int> const& shape, cute::tuple<_1, int, cute::tuple<int, int>>& stride) {
    auto [B, H, Q, D] = shape;
    stride = make_stride(_1{}, Q, make_stride(H*Q*D, Q*D));
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;
    D = cutlass::round_up(D, 8); // Alignment

    auto shape_QO = cute::make_shape(B, H, Q, D);
    auto shape_KV = cute::make_shape(B, H, K, D);
    auto shape_LSE = cute::make_shape(B, H, Q);

    initialize_stride(shape_QO, stride_Q);
    initialize_stride(shape_KV, stride_K);
    initialize_stride(shape_KV, stride_V);
    initialize_stride(shape_QO, stride_O);
    initialize_stride(shape_LSE, stride_LSE);

    block_Q.reset(size(shape_QO));
    block_K.reset(size(shape_KV));
    block_V.reset(size(shape_KV));
    block_O.reset(size(shape_QO));
    block_LSE.reset(size(shape_LSE));
    block_ref_O.reset(size(shape_QO));
    block_ref_LSE.reset(size(shape_LSE));

    initialize_block(block_Q, seed + 2023, false);
    initialize_block(block_K, seed + 2022, false);
    initialize_block(block_V, seed + 2021, false);
  }

  ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.b, options.h, options.q, options.k, options.d};

    initialize(problem_size);

    typename Operation::Arguments arguments{
      problem_size,
      { block_Q.get(), stride_Q,
        block_K.get(), stride_K,
        block_V.get(), stride_V },
      { block_O.get(), stride_O,
      block_LSE.get(), stride_LSE },
      hw_info
    };

    Operation op;

    ExampleResult example_result;

    example_result.smem_size = Operation::Kernel::SharedStorageSize;

    size_t workspace_size = Operation::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    status = op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    // Run
    status = op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops = 4.0 * (std::is_same_v<ActiveFusion, CausalFusion> ? 0.5 : 1.0);
    flops *= static_cast<double>(get<0>(problem_size));
    flops *= static_cast<double>(get<1>(problem_size));
    flops *= static_cast<double>(get<2>(problem_size));
    flops *= static_cast<double>(get<3>(problem_size));
    flops *= static_cast<double>(get<4>(problem_size));
    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_size);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

template<
  class TileShape,
  class DispatchPolicy,
  class ActiveFusion,
  class... KernelOptions
>
struct BwdRunner {

  using Element = cutlass::half_t;
  using ElementAccumulator = float;

  // B H Q K D
  using ProblemShapeType = cute::tuple<int, int, int, int, int>;

  using Operation = cutlass::fmha::device::FmhaBwd<Element, ElementAccumulator, TileShape, ActiveFusion, KernelOptions...>;
  
  // Just like forward
  using StrideQ = cute::tuple<int, int, int, _1>; // B H Q D
  using StrideK = cute::tuple<int, int, int, _1>; // B H K D
  using StrideV = cute::tuple<int, int, int, _1>; // B H K D
  using StrideO = cute::tuple<int, int, int, _1>; // B H Q D
  using StrideLSE = cute::tuple<int, int, _1>; // B H Q

  // Backwards specific
  using StrideDQ = cute::tuple<int, int, int, _1>; // B H Q D
  using StrideDK = cute::tuple<int, int, int, _1>; // B H K D
  using StrideDV = cute::tuple<int, int, int, _1>; // B H K D
  using StrideDO = cute::tuple<int, int, int, _1>; // B H Q D

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;

  StrideDQ stride_dQ;
  StrideDK stride_dK;
  StrideDV stride_dV;
  StrideDO stride_dO;

  uint64_t seed = 0;

  cutlass::DeviceAllocation<Element> block_Q;
  cutlass::DeviceAllocation<Element> block_K;
  cutlass::DeviceAllocation<Element> block_V;
  cutlass::DeviceAllocation<Element> block_O;
  cutlass::DeviceAllocation<ElementAccumulator> block_LSE;

  cutlass::DeviceAllocation<Element> block_dQ;
  cutlass::DeviceAllocation<Element> block_dK;
  cutlass::DeviceAllocation<Element> block_dV;
  cutlass::DeviceAllocation<Element> block_dO;

  cutlass::DeviceAllocation<Element> block_ref_dQ;
  cutlass::DeviceAllocation<Element> block_ref_dK;
  cutlass::DeviceAllocation<Element> block_ref_dV;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_Q), get<3>(stride_Q), make_stride(get<0>(stride_Q), get<1>(stride_Q))));

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_K), get<3>(stride_K), make_stride(get<0>(stride_K), get<1>(stride_K))));

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_V), get<3>(stride_V), make_stride(get<0>(stride_V), get<1>(stride_V))));

    Tensor mO = make_tensor(make_gmem_ptr(block_O.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_O), get<3>(stride_O), make_stride(get<0>(stride_O), get<1>(stride_O))));

    Tensor mLSE = make_tensor(make_gmem_ptr(block_LSE.get()),
      make_shape(Q, make_shape(B, H)),
      make_stride(get<2>(stride_LSE), make_stride(get<0>(stride_LSE), get<1>(stride_LSE))));

    Tensor mDQ = make_tensor(make_gmem_ptr(block_ref_dQ.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_dQ), get<3>(stride_dQ), make_stride(get<0>(stride_dQ), get<1>(stride_dQ))));

    Tensor mDK = make_tensor(make_gmem_ptr(block_ref_dK.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_dK), get<3>(stride_dK), make_stride(get<0>(stride_dK), get<1>(stride_dK))));

    Tensor mDV = make_tensor(make_gmem_ptr(block_ref_dV.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_dV), get<3>(stride_dV), make_stride(get<0>(stride_dV), get<1>(stride_dV))));

    Tensor mDO = make_tensor(make_gmem_ptr(block_dO.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_dO), get<3>(stride_dO), make_stride(get<0>(stride_dO), get<1>(stride_dO))));


    fmha_bwd_reference(problem_size, mQ, mK, mV, mO, mLSE, mDO, mDQ, mDK, mDV, ActiveFusion{});
    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_dQ, block_ref_dQ, max_diff, mean_diff);
    bool passed_dQ = (max_diff < 1e-2) && (mean_diff < 1e-3);
    if (! passed_dQ) {
      std::cerr << "failed dQ: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_dK, block_ref_dK, max_diff, mean_diff);
    bool passed_dK = (max_diff < 1e-2) && (mean_diff < 1e-3);
    if (! passed_dK) {
      std::cerr << "failed dK: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_dV, block_ref_dV, max_diff, mean_diff);
    bool passed_dV = (max_diff < 1e-2) && (mean_diff < 1e-3);
    if (! passed_dV) {
      std::cerr << "failed dV: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    return passed_dQ && passed_dK && passed_dV;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;
    D = cutlass::round_up(D, 8);  // Alignment
    Q = cutlass::round_up(Q, 8);  // Alignment

    auto shape_QO = cute::make_shape(B, H, Q, D);
    auto shape_KV = cute::make_shape(B, H, K, D);
    auto shape_LSE = cute::make_shape(B, H, Q);

    stride_Q = cute::compact_row_major(shape_QO);
    stride_K = cute::compact_row_major(shape_KV);
    stride_V = cute::compact_row_major(shape_KV);
    stride_O = cute::compact_row_major(shape_QO);
    stride_LSE = cute::compact_row_major(shape_LSE);

    stride_dQ = stride_Q;
    stride_dK = stride_K;
    stride_dV = stride_V;
    stride_dO = stride_O;

    block_Q.reset(size(shape_QO));
    block_K.reset(size(shape_KV));
    block_V.reset(size(shape_KV));
    block_O.reset(size(shape_QO));
    block_LSE.reset(size(shape_LSE));

    block_dQ.reset(size(shape_QO));
    block_dK.reset(size(shape_KV));
    block_dV.reset(size(shape_KV));
    block_dO.reset(size(shape_QO));

    block_ref_dQ.reset(size(shape_QO));
    block_ref_dK.reset(size(shape_KV));
    block_ref_dV.reset(size(shape_KV));

    initialize_block(block_Q, seed + 2023, false);
    initialize_block(block_K, seed + 2022, false);
    initialize_block(block_V, seed + 2021, false);
    initialize_block(block_dO, seed + 2020, false);
    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_Q), get<3>(stride_Q), make_stride(get<0>(stride_Q), get<1>(stride_Q))));

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_K), get<3>(stride_K), make_stride(get<0>(stride_K), get<1>(stride_K))));

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_V), get<3>(stride_V), make_stride(get<0>(stride_V), get<1>(stride_V))));

    Tensor mO = make_tensor(make_gmem_ptr(block_O.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_O), get<3>(stride_O), make_stride(get<0>(stride_O), get<1>(stride_O))));

    Tensor mLSE = make_tensor(make_gmem_ptr(block_LSE.get()),
      make_shape(Q, make_shape(B, H)),
      make_stride(get<2>(stride_LSE), make_stride(get<0>(stride_LSE), get<1>(stride_LSE))));

    fmha_reference(problem_size, mQ, mK, mV, mO, mLSE, ActiveFusion{});
  }

  ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
    ProblemShapeType problem_size = ProblemShapeType{options.b, options.h, options.q, options.k, options.d};

    initialize(problem_size);

    typename Operation::Arguments arguments{
      problem_size,
      block_Q.get(), stride_Q,
      block_K.get(), stride_K,
      block_V.get(), stride_V,
      block_O.get(), stride_O,
      block_LSE.get(), stride_LSE,
      block_dO.get(), stride_dO,
      block_dQ.get(), stride_dQ,
      block_dK.get(), stride_dK,
      block_dV.get(), stride_dV,
      hw_info
    };

    Operation op;

    ExampleResult example_result;

    example_result.smem_size = Operation::Operation::Kernel::SharedStorageSize;

    size_t workspace_size = Operation::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    cutlass::Status status = op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "This kernel is not supported. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    status = op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to initialize the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    // Run
    hipMemset(block_dQ.get(), 0, block_dQ.size() * sizeof(Element));
    hipMemset(block_dK.get(), 0, block_dK.size() * sizeof(Element));
    hipMemset(block_dV.get(), 0, block_dV.size() * sizeof(Element));
    
    status = op.run();
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(hipGetLastError()) << std::endl;
      return example_result;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    //
    // Construct events
    //

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      status = op.run();
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops = 10.0 * (std::is_same_v<ActiveFusion, CausalFusion> ? 0.5 : 1.0);
    flops *= static_cast<double>(get<0>(problem_size));
    flops *= static_cast<double>(get<1>(problem_size));
    flops *= static_cast<double>(get<2>(problem_size));
    flops *= static_cast<double>(get<3>(problem_size));
    flops *= static_cast<double>(get<4>(problem_size));
    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_size);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, ExampleResult result, bool verbose) {
  std::ios fmt(nullptr);
  fmt.copyfmt(std::cout);
  std::cout << (result.passed ? (result.verified ? " [OK]  " : " [--] ") : "[FAIL] ");
  std::cout << std::setw(32) << std::left << description;
  std::cout.copyfmt(fmt);
  std::cout << " : " << result.tflops_s << " TFLOPS/s" << std::endl;
  if (verbose) {
    std::cout << "       t=" << result.runtime_ms << "ms, "
        "smem=" << result.smem_size << "b" << std::endl;
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

using KernelTma = cutlass::gemm::KernelTma;
using KernelCooperative = cutlass::gemm::KernelTmaWarpSpecializedCooperative;
using KernelPingpong = cutlass::gemm::KernelTmaWarpSpecializedPingpong;

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_fwd_32(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _32;

  run(Shape< _64, _128, HeadDim>{}, KernelTma{}, "tma 64x128x32");
  run(Shape< _128, _64, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x64x32");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_fwd_64(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _64;

  run(Shape< _64, _128, HeadDim>{}, KernelTma{}, "tma 64x128x64");
  run(Shape< _128, _64, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x64x64");
  run(Shape< _128, _64, HeadDim>{}, KernelPingpong{}, "tma ws ping-pong 128x64x64");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_fwd_128(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _128;

  run(Shape<_128, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x128x128");
#ifdef FP8
  run(Shape<_128, _256, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x256x128 acc fp16", Option<Tag::kAccQK, cutlass::half_t>{});
  run(Shape<_128, _256, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x256x128 acc fp32");
#endif
  run(Shape<_128, _128, HeadDim>{}, KernelPingpong{}, "tma ws ping-pong 128x128x128");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_fwd_256(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _256;

#ifdef FP8
  run(Shape<_128, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x128x256");
  run(Shape<_128, _128, HeadDim>{}, KernelPingpong{}, "tma ws ping-pong 128x128x256");
#else
  run(Shape<_128, _64, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x64x256");
#endif
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_bwd_32(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _32;

  run(Shape< _64, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 64x128x32");
  run(Shape<_128, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x128x32");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_bwd_64(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _64;

  run(Shape< _64, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 64x128x64");
  run(Shape<_128, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 128x128x64");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

template<class Fusion>
void run_bwd_128(Fusion fusion, Options const & options, cutlass::KernelHardwareInfo const& hw_info) {
  auto run = [&](auto shape, auto kernel, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel), Fusion, decltype(kernel_options)...> runner;
    auto result = runner.run(options, hw_info);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _128;

  run(Shape<_64, _128, HeadDim>{}, KernelCooperative{}, "tma ws cooperative 64x128x128");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_single(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || props.major < 9) {
    std::cout
      << "This example requires a GPU of NVIDIA's Hopper Architecture or "
      << "later (compute capability 90 or greater) and CUDA 12.0 or greater.\n";
    return 0;
  }
  
  else if (__CUDACC_VER_MAJOR__ < 12 || (props.major != 9 || props.minor != 0)) {
    std::cout
      << "This example requires a GPU of NVIDIA's Hopper Architecture "
      << "(compute capability 90) and CUDA 12.0 or greater.\n";
    return 0;
  }
  

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  //
  // Run examples
  //

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  std::cout << "###### B " << options.b << " H " << options.h << " Q " << options.q << " K " << options.k << " D " << options.d << " ";
  std::cout << (options.bwd ? "Backward" : "Forward") << " " << (options.causal ? "Causal" : "Full") << " ";
  std::cout << "#SM " << hw_info.sm_count << std::endl;

  auto with_fusion = [&](auto fn) {
    if (options.causal) {
      fn(CausalFusion{});
    } else if (options.residual){
      fn(ResidualFusion{});
    } else {
      fn(DefaultFusion{});
    }
  };

  with_fusion([&](auto fusion) {
    if (options.bwd) {
#ifndef FP8
      if (options.d <= 32) {
        run_bwd_32(fusion, options, hw_info);
      } else if (options.d <= 64) {
        run_bwd_64(fusion, options, hw_info);
      } else if (options.d <= 128) {
        run_bwd_128(fusion, options, hw_info);
      } else
#endif
      {
#ifdef FP8
        std::cout << "Backward is not implemented for FP8." << std::endl;
#else
        std::cout << "No backward kernel instantiated for d=" << options.d << std::endl;
#endif
      }
    } else {
#ifndef FP8
      if (options.d <= 32) {
        run_fwd_32(fusion, options, hw_info);
      } else
      if (options.d <= 64) {
        run_fwd_64(fusion, options, hw_info);
      } else
#endif
      if (options.d <= 128) {
        run_fwd_128(fusion, options, hw_info);
      } else
      if (options.d <= 256) {
        run_fwd_256(fusion, options, hw_info);
      }
      else {
        std::cout << "No forward kernel instantiated for d=" << options.d << std::endl;
      }
    }
  });
#endif

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  std::vector<std::string> full_arguments(args, args + argc);

  int result = 0;

  bool recursed = false;
  for (size_t i = 1; i < full_arguments.size(); i++) {
    if (full_arguments[i].find(',') != std::string::npos) {
      auto arg = full_arguments[i];
      size_t eq_pos = arg.find('=');
      std::string prefix = eq_pos == std::string::npos ? "" : arg.substr(0, eq_pos+1);
      std::string rest = eq_pos == std::string::npos ? arg : arg.substr(eq_pos+1);
      for (;;) {
        size_t comma_pos = rest.find(',');
        std::string current = rest.substr(0, comma_pos);
        full_arguments[i] = prefix + current;
        std::vector<const char*> next_args;
        for (auto& elem : full_arguments) { next_args.push_back(elem.data()); }
        main(argc, next_args.data());
        if (comma_pos == std::string::npos) break;
        rest = rest.substr(comma_pos+1);
      }
      recursed = true;
      break;
    }
  }

  if (! recursed) {
    main_single(argc, args);
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
