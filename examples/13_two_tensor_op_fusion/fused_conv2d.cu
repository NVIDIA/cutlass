/***************************************************************************************************
 * Copyright (c) 2017-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "b2b_conv2d_fprop_implicit_gemm_s8ncxhwx_s8cxrskx_s8ncxhwx_tensor_op_s32_sm75.h"
#include "b2b_conv2d_fprop_implicit_gemm_s8ncxhwx_s8cxrskx_s8ncxhwx_tensor_op_s32_sm80.h"
#include "b2b_conv2d_fprop_implicit_gemm_f16nhwc_f16nhwc_f16nhwc_tensor_op_f16_sm75.h"
#include "b2b_conv2d_fprop_implicit_gemm_f16nhwc_f16nhwc_f16nhwc_tensor_op_f16_sm80.h"

int run_sm75() {
  bool notSupported = false;

  // Turing Tensor Core operations exposed with mma.sync are first available in CUDA 10.2.
  //
  // CUTLASS must be compiled with CUDA 10.2 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!(props.major == 7 && props.minor >= 5)) {
    notSupported = true;
  }

  if (notSupported) {
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  bool pass = 1;
 
  std::cout << "Running on SM75" << std::endl;
  pass &= run_nonfused_conv2d_fprop_optimized_f16_sm75();
  pass &= run_fused_conv2d_fprop_optimized_f16_sm75();
  pass &= run_fused_conv2d_fprop_optimized_f16_sm75_rf_res();
  pass &= run_nonfused_conv2d_fprop_optimized_s8_sm75();
  pass &= run_fused_conv2d_fprop_optimized_s8_sm75();
  pass &= run_fused_conv2d_fprop_optimized_s8_sm75_rf_res();

  if(pass)
    return 1;
  else
    return -1;

}

int run_sm80() {
  bool notSupported = false;

  // Ampere Tensor Core operations exposed with mma.sync are first available in CUDA 11.0.
  //
  // CUTLASS must be compiled with CUDA 11 Toolkit to run Conv2dFprop examples.
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!(props.major == 8 && props.minor >= 0)) {
    notSupported = true;
  }

  if (notSupported) {
    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  bool pass = 1;
 
  std::cout << "Running on SM80" << std::endl;
  pass &= run_nonfused_conv2d_fprop_optimized_f16_sm80();
  pass &= run_fused_conv2d_fprop_optimized_f16_sm80();
  pass &= run_nonfused_conv2d_fprop_optimized_s8_sm80();
  pass &= run_fused_conv2d_fprop_optimized_s8_sm80();

  if(pass)
    return 1;
  else
    return -1;

}


int main() {

  int result = 0;

  result = run_sm80();

  if(!result) { // not supported
    result = run_sm75();

    if(!result) {
      std::cout << "This example isn't supported on current architecture" << std::endl;
    }

  }

  if(result >= 0)
    return 0;
  else
    return -1;
}

