#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Example of running an Ada FP8 GEMM.

    In addition to using FP8 Tensor Core instructions, the Ada FP8 GEMM uses a distinct epilogue
    that enables additional scaling of operands/outputs, storing a pre-activation-function output
    tensor (called the "auxiliary" output), and computing the absolute maximum value of the
    outputs.

    Pseudocode for this epilogue is as follows:

    Aux = ((alpha * scale_a * scale_b) * accumulator) + ((beta * scale_c) * source) + bias
    D = activation(Aux)

    if Aux is fp8 type:
        abs_max_output = max( abs(aux) | (for every aux in Aux))
        Aux = scale_aux * Aux
    endif

    if D is fp8 type:
        abs_max_output = max( abs(d) | (for every d in D))
        D = scale_d * D
    endif

    Parameter Aux is optionally stored to global memory
*/

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/gemm.h"

#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/epilogue/thread/linear_combination_generic_with_scaling.h"
#include "cutlass/gemm/device/gemm_universal_with_absmax.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/matrix_coord.h"
#include "cutlass/gemm/device/gemm_universal_adapter.h"


using ElementA = cutlass::float_e4m3_t;
using ElementB = cutlass::float_e4m3_t;
using ElementOutput = cutlass::float_e4m3_t;
using ElementAuxOutput = ElementOutput;
using ElementAccumulator = float;
using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::ColumnMajor;
using LayoutC = cutlass::layout::RowMajor;
static int const kStages = 3;
static int const kAlignmentA = 16;
static int const kAlignmentB = 16;

using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::ReLu,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
    >;

template <typename MathOperator>
using Gemm_ = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages,
    kAlignmentA, kAlignmentB, MathOperator
  >;

using ElementAbsmax = typename EpilogueOutputOp::ElementAbsmax;


// Command line options parsing
struct Options {

  bool help;
  bool error;
  bool reference_check;
  cutlass::gemm::GemmCoord problem_size;

  int iterations;
  int warmup_iterations;

  bool scale_A;
  bool scale_B;
  bool scale_C;

  float alpha;
  float beta;

  Options():
    help(false),
    error(false),
    reference_check(false),
    iterations(20),
    warmup_iterations(5),
    scale_A(true),
    scale_B(true),
    scale_C(true),
    alpha(1.f),
    beta(0.f)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("warmup_iterations", warmup_iterations, 5);
    cmd.get_cmd_line_argument("reference-check", reference_check, false);
    cmd.get_cmd_line_argument("scale-A", scale_A, true);
    cmd.get_cmd_line_argument("scale-B", scale_B, true);
    cmd.get_cmd_line_argument("scale-C", scale_C, true);
    cmd.get_cmd_line_argument("alpha", alpha, 1.f);
    cmd.get_cmd_line_argument("beta", beta, 0.f);

    int m, n, k;
    cmd.get_cmd_line_argument("m", m, 1024);
    cmd.get_cmd_line_argument("n", n, 1024);
    cmd.get_cmd_line_argument("k", k, 1024);

    problem_size = cutlass::gemm::GemmCoord{m, n, k};
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "58_ada_fp8_gemm\n\n"
      << "  This example executes a GEMM using Ada FP8 Tensor Core operations. In addition to performing\n"
      << "  a normal GEMM, the kernel performs the following operations:\n"
      << "      Aux = ((alpha * scale_a * scale_b) * accumulator) + ((beta * scale_c) * source) + bias\n"
      << "        D = activation(Aux)\n\n"
      << "      if Aux is fp8:\n"
      << "         abs_max_output = max( abs(aux) | (for every aux in Aux) )\n"
      << "         Aux = scale_aux * Aux\n\n"
      << "      if D is fp8 type:\n"
      << "         abs_max_output = max( abs(d) | (for every d in D) )\n"
      << "         D = scale_d * D\n\n"
      << "Options:\n\n"
      << "  --help                           If specified, displays this usage statement\n\n"
      << "  --m=<int>                        Sets the M dimension of the GEMM\n"
      << "  --n=<int>                        Sets the N dimension of the GEMM\n"
      << "  --k=<int>                        Sets the K dimension of the GEMM\n"
      << "  --scale-A=<bool>                 Whether to apply a scaling factor to operand A (default: true)\n"
      << "  --scale-B=<bool>                 Whether to apply a scaling factor to operand B (default: true)\n"
      << "  --scale-C=<bool>                 Whether to apply a scaling factor to operand C (default: true)\n"
      << "  --iterations=<int>               Number of profiling iterations to perform\n"
      << "  --warmup-iterations=<int>        Number of warmup iterations to perform\n"
      << "  --reference-check=<bool>         If true, performs reference check\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  float gflops(float runtime_s) const {
    // Two flops per multiply-add
    return 2.0f * float(problem_size.product()) / float(1.0e9) / runtime_s;
  }
};

/// Helper class to run the kernel
template <typename Gemm>
struct TestbedRunner {

  using ElementAccumulator = typename Gemm::ElementAccumulator;
  using ElementCompute = typename Gemm::GemmKernel::Epilogue::OutputOp::ElementCompute;
  using ElementScalingFactor = typename Gemm::EpilogueOutputOp::ElementScalingFactor;

  static bool const kScaleAux = Gemm::EpilogueOutputOp::kIsScalingAndAmaxAuxOutputNeeded;
  static bool const kScaleOutput = Gemm::EpilogueOutputOp::kIsScalingAndAmaxOutputNeeded;

  /// Initialization
  cutlass::Distribution::Kind init_A;
  cutlass::Distribution::Kind init_B;
  cutlass::Distribution::Kind init_C;
  uint64_t seed;

  cutlass::HostTensor<typename Gemm::ElementA, typename Gemm::LayoutA> tensor_A;
  cutlass::HostTensor<typename Gemm::ElementB, typename Gemm::LayoutB> tensor_B;
  cutlass::HostTensor<typename Gemm::ElementC, typename Gemm::LayoutC> tensor_C;
  cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementAuxOutput, typename Gemm::LayoutC> tensor_Aux;
  cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, typename Gemm::LayoutC> tensor_D;
  cutlass::HostTensor<typename Gemm::ElementC, typename Gemm::LayoutC> tensor_Vector;
  cutlass::HostTensor<ElementAccumulator, typename Gemm::LayoutC> tmp_D;
  cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementOutput, typename Gemm::LayoutC> reference_D;
  cutlass::HostTensor<typename Gemm::EpilogueOutputOp::ElementAuxOutput, typename Gemm::LayoutC> reference_Aux;
  cutlass::HostTensor<ElementScalingFactor, typename Gemm::LayoutC> scale_A;
  cutlass::HostTensor<ElementScalingFactor, typename Gemm::LayoutC> scale_B;
  cutlass::HostTensor<ElementScalingFactor, typename Gemm::LayoutC> scale_C;
  cutlass::HostTensor<ElementScalingFactor, typename Gemm::LayoutC> scale_D;
  cutlass::HostTensor<ElementScalingFactor, typename Gemm::LayoutC> scale_Aux;
  cutlass::HostTensor<ElementAbsmax, typename Gemm::LayoutC> abs_max_Aux;
  cutlass::HostTensor<ElementAbsmax, typename Gemm::LayoutC> abs_max_D;
  cutlass::HostTensor<ElementAbsmax, typename Gemm::LayoutC> reference_abs_max_Aux;
  cutlass::HostTensor<ElementAbsmax, typename Gemm::LayoutC> reference_abs_max_D;

  //
  // Methods
  //

  TestbedRunner(
    bool scaleA = true,
    bool scaleB = true,
    bool scaleC = true,
    cutlass::Distribution::Kind init_A_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_B_ = cutlass::Distribution::Uniform,
    cutlass::Distribution::Kind init_C_ = cutlass::Distribution::Uniform,
    uint64_t seed_ = 2080
  ):
    init_A(init_A_), init_B(init_B_), init_C(init_C_), seed(seed_) { }

  /// Helper to initialize scaling factors
  template <typename Element, typename Layout>
  bool initialize_scale_factor(cutlass::TensorView<Element, Layout> view, uint64_t seed, int bits=0) {
    cutlass::reference::host::TensorFillRandomUniform(view, seed, double(1.), double(0.), bits);
    return true;
  }

  /// Helper to initialize a tensor view
  template <typename Element, typename Layout>
  bool initialize_tensor(
    cutlass::TensorView<Element, Layout> view,
    cutlass::Distribution::Kind dist_kind,
    uint64_t seed) {

    if (dist_kind == cutlass::Distribution::Uniform) {

      double scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<typename Gemm::ElementC>::value;

      if (bits_input == 1) {
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        scope_max = 5;
        scope_min = -5;
      } else {
        scope_max = 8;
        scope_min = -8;
      }

      cutlass::reference::host::TensorFillRandomUniform(
        view, seed, scope_max, scope_min, 0);
    }
    else if (dist_kind == cutlass::Distribution::Identity) {

      cutlass::reference::host::TensorFillIdentity(view);
    }
    else if (dist_kind == cutlass::Distribution::Gaussian) {

      cutlass::reference::host::TensorFillRandomGaussian(view, seed, 0, 0.5);
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {

      cutlass::reference::host::BlockFillSequential(
        view.data(), view.capacity());
    }
    else {
      std::cerr << "Not implemented";
      return false;
    }

    return true;
  }

  /// Initializes data structures
  void initialize(const Options& options) {
    //
    // Allocate the GEMM workspace
    //

    tensor_A.resize(options.problem_size.mk());
    tensor_B.resize(options.problem_size.kn());
    tensor_C.resize(options.problem_size.mn());
    tensor_D.resize(options.problem_size.mn());
    tensor_Vector.resize({1, options.problem_size.n()});
    reference_D.resize(options.problem_size.mn(), false);
    tmp_D.resize(options.problem_size.mn(), false);

    initialize_tensor(tensor_A.host_view(), init_A, seed + 2019);
    initialize_tensor(tensor_B.host_view(), init_B, seed + 2018);
    initialize_tensor(tensor_C.host_view(), init_C, seed + 2017);
    initialize_tensor(tensor_Vector.host_view(), init_C, seed + 2020);

    // It is possible to randomly initialize to all zeros, so override this with non-zeros
    // in the upper left corner of each operand.
    cutlass::Coord<2> origin(0);
    tensor_A.host_view().at(origin) = typename Gemm::ElementA(1);
    tensor_B.host_view().at(origin) = typename Gemm::ElementB(1);
    tensor_C.host_view().at(origin) = typename Gemm::ElementC(1);
    tensor_Vector.host_view().at(origin) = typename Gemm::ElementC(1);

    cutlass::reference::host::TensorFill(tensor_D.host_view());
    cutlass::reference::host::TensorCopy(reference_D.host_view(), tensor_C.host_view());

    tensor_A.sync_device();
    tensor_B.sync_device();
    tensor_C.sync_device();
    tensor_D.sync_device();
    tensor_Vector.sync_device();

    int scale_bits = 2;
    if (options.scale_A) {
      scale_A.resize({1, 1});
      initialize_scale_factor(scale_A.host_view(), seed + 2021, scale_bits);
      scale_A.sync_device();
    }

    if (options.scale_B) {
      scale_B.resize({1, 1});
      initialize_scale_factor(scale_B.host_view(), seed + 2022, scale_bits);
      scale_B.sync_device();
    }

    if (options.scale_C) {
      scale_C.resize({1, 1});
      initialize_scale_factor(scale_C.host_view(), seed + 2023, scale_bits);
      scale_C.sync_device();
    }

    if (kScaleOutput) {
      scale_D.resize({1, 1});
      initialize_scale_factor(scale_D.host_view(), seed + 2024, scale_bits);
      scale_D.sync_device();

      abs_max_D.resize({1, 1});
      cutlass::reference::host::TensorFill(abs_max_D.host_view());
      abs_max_D.sync_device();

      reference_abs_max_D.resize({1, 1});
    }

    if (kScaleAux) {
      tensor_Aux.resize(options.problem_size.mn());
      cutlass::reference::host::TensorFill(tensor_Aux.host_view());
      tensor_Aux.sync_device();

      scale_Aux.resize({1, 1});
      initialize_scale_factor(scale_Aux.host_view(), seed + 2025, scale_bits);
      scale_Aux.sync_device();

      abs_max_Aux.resize({1, 1});
      cutlass::reference::host::TensorFill(abs_max_Aux.host_view());
      abs_max_Aux.sync_device();

      reference_Aux.resize(options.problem_size.mn(), false);
      reference_abs_max_Aux.resize({1, 1});
    }
  }

  /// Compares computed reference with device reference and outputs to a file if incorrect
  bool compare_reference(const Options& options) {

    tensor_D.sync_host();

    bool passed = cutlass::reference::host::TensorEquals(reference_D.host_view(), tensor_D.host_view());

    if (kScaleAux) {
      tensor_Aux.sync_host();
      abs_max_Aux.sync_host();
      passed &= cutlass::reference::host::TensorEquals(reference_Aux.host_view(), tensor_Aux.host_view());
      passed &= cutlass::reference::host::TensorEquals(abs_max_Aux.host_view(), reference_abs_max_Aux.host_view());
    }

    if (kScaleOutput) {
      abs_max_D.sync_host();
      passed &= cutlass::reference::host::TensorEquals(abs_max_D.host_view(), reference_abs_max_D.host_view());
    }

    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;

      std::string output_file = "testbed_with_amax_errors.txt";
      std::ofstream file(output_file);

      file
        << "problem: " << options.problem_size
        << ", alpha: " << options.alpha << ", beta: " << options.beta << "\n\n";

      file
        << "A =\n" << tensor_A.host_view()
        << "\nB =\n" << tensor_B.host_view()
        << "\nC =\n" << tensor_C.host_view()
        << "\nVector =\n" << tensor_Vector.host_view()
        << "\nScaleA = " << scale_A.host_view()
        << "\nScaleB = " << scale_B.host_view()
        << "\nScaleC = " << scale_C.host_view()
        << "\nScaleD = " << scale_D.host_view()
        << "\nScaleAux = " << scale_Aux.host_view()
        << "\n\nReference D =\n" << reference_D.host_view()
        << "\nComputed D =\n" << tensor_D.host_view();
      if (kScaleAux) {
        file
          << "\n\nReference Aux =\n" << reference_Aux.host_view()
          << "\nComputed Aux =\n" << tensor_Aux.host_view()
          << "\n\nReference Absmax Aux = " << reference_abs_max_Aux.host_view()
          << "\nComputed Absmax Aux = " << abs_max_Aux.host_view();
      }
      if (kScaleOutput) {
        file
          << "\n\nReference Absmax D = " << reference_abs_max_D.host_view()
          << "\nComputed Absmax D = " << abs_max_D.host_view();
      }

      std::cerr << "Dumped results to " << output_file << std::endl;

    }

    return passed;
  }

  /// Verifies the result is a GEMM
  bool verify(const Options& options) {

    cutlass::Coord<2> origin(0);
    ElementCompute scaled_alpha = options.alpha;
    if (options.scale_A) {
      scaled_alpha *= scale_A.host_view().at(origin);
    }
    if (options.scale_B) {
      scaled_alpha *= scale_B.host_view().at(origin);
    }

    ElementCompute scaled_beta = options.beta;
    if (options.scale_C) {
      scaled_beta *= scale_C.host_view().at(origin);
    }

    //
    // Verify
    //

    cutlass::reference::host::GemmComplex<
        typename Gemm::ElementA, typename Gemm::LayoutA,
        typename Gemm::ElementB, typename Gemm::LayoutB,
        typename Gemm::ElementC, typename Gemm::LayoutC,
        ElementCompute, ElementAccumulator, ElementAccumulator
    >(
      options.problem_size,
      scaled_alpha,
      tensor_A.host_ref(),
      Gemm::kTransformA,
      tensor_B.host_ref(),
      Gemm::kTransformB,
      scaled_beta,
      tensor_C.host_ref(),
      tmp_D.host_ref(),
      ElementAccumulator(0)
    );

    ElementCompute tmp_abs_max_Aux(0.);
    ElementCompute tmp_abs_max_D(0.);

    cutlass::NumericConverter<ElementCompute, typename Gemm::ElementC> cvt_c_to_compute;
    cutlass::NumericConverter<ElementCompute, ElementAccumulator> cvt_accum_to_compute;
    cutlass::NumericConverter<ElementAccumulator, ElementCompute> cvt_compute_to_accum;
    cutlass::NumericConverter<typename Gemm::EpilogueOutputOp::ElementOutput, ElementCompute> cvt_compute_to_d;
    cutlass::NumericConverter<typename Gemm::EpilogueOutputOp::ElementAuxOutput, ElementCompute> cvt_compute_to_aux;

    cutlass::absolute_value_op<ElementCompute> abs;
    cutlass::maximum_with_nan_propogation<ElementCompute> max;
    cutlass::epilogue::thread::ReLu<ElementCompute> act;

    ElementScalingFactor d_scale = kScaleOutput ? scale_D.host_view().at(origin) : ElementScalingFactor(1.);

    for (int m = 0; m < options.problem_size.m(); ++m) {
      for (int n = 0; n < options.problem_size.n(); ++n) {
        ElementCompute intermediate = cvt_accum_to_compute(tmp_D.host_view().at({m, n}));
        ElementCompute bias = cvt_c_to_compute(tensor_Vector.host_view().at({0, n}));
        ElementCompute aux = intermediate + bias;
        ElementCompute d = act(aux);
        tmp_abs_max_Aux = max(abs(aux), tmp_abs_max_Aux);
        tmp_abs_max_D = max(abs(d), tmp_abs_max_D);
        reference_D.host_view().at({m, n}) = cvt_compute_to_d(d * d_scale);

        if (kScaleAux) {
          reference_Aux.host_view().at({m, n}) = cvt_compute_to_aux(aux * scale_Aux.host_view().at(origin));
        }
      }
    }

    if (kScaleAux) {
      reference_abs_max_Aux.host_view().at(origin) = cvt_compute_to_accum(tmp_abs_max_Aux);
    }

    if (kScaleOutput) {
      reference_abs_max_D.host_view().at(origin) = cvt_compute_to_accum(tmp_abs_max_D);
    }

    return compare_reference(options);
  }

  /// Returns true if the CUDA device is sufficient to execute the kernel.
  bool sufficient() const {

    if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 4)) {
      std::cerr << "This example requires CUDA 12.4 or greater." << std::endl;
      return false;
    }

    size_t smem_size = sizeof(typename Gemm::GemmKernel::SharedStorage);

    hipDeviceProp_t properties;
    int device_idx;
    hipError_t result = hipGetDevice(&device_idx);

    if (result != hipSuccess) {
      std::cerr << "hipGetDevice() failed with error: " << hipGetErrorString(result) << std::endl;
      return false;
    }

    result = hipGetDeviceProperties(&properties, device_idx);

    if (result != hipSuccess) {
      std::cerr << "hipGetDeviceProperties() failed with error: " << hipGetErrorString(result) << std::endl;
      return false;
    }

    if (properties.major < 8 || (properties.major == 8 && properties.minor < 9)) {
      std::cerr << "CUTLASS's Ada FP8 GEMM example requires a device of compute capability 89 or higher.\n" << std::endl;
      return false;
    }

    if (properties.sharedMemPerBlockOptin < smem_size) {
      std::cerr << "Insufficient shared memory. Need " << smem_size
                << ", but device only has " << properties.sharedMemPerBlockOptin << std::endl;
      return false;
    }

    return true;
  }

  /// Executes one test
  bool run(Options& options)
  {

    // Waive test if insufficient CUDA device
    if (!sufficient()) {
      std::cerr << "Insufficient resources to run the kernel." << std::endl;
      return false;
    }

    this->initialize(options);

    //
    // Initialize the GEMM operator
    //

    typename Gemm::EpilogueOutputOp::Params::ActivationParams activation_params{
      ElementCompute(options.alpha),
      ElementCompute(options.beta)
    };
    typename Gemm::EpilogueOutputOp::Params epilogue_params{
      activation_params,
      scale_A.device_data(),
      scale_B.device_data(),
      scale_C.device_data(),
      scale_D.device_data(),
      scale_Aux.device_data(),
      abs_max_Aux.device_data(),
      abs_max_D.device_data()
    };

    typename Gemm::Arguments arguments{
      cutlass::gemm::GemmUniversalMode::kGemm,
      options.problem_size,
      /* batch_count = */ 1,
      epilogue_params,
      tensor_A.device_data(),
      tensor_B.device_data(),
      tensor_C.device_data(),
      tensor_D.device_data(),
      tensor_Aux.device_data(),
      tensor_Vector.device_data(),
      options.problem_size.m() * options.problem_size.k(),
      options.problem_size.n() * options.problem_size.k(),
      options.problem_size.m() * options.problem_size.n(),
      options.problem_size.m() * options.problem_size.n(),
      (int)options.problem_size.m(), // Batch stride vector
      tensor_A.layout().stride(0),
      tensor_B.layout().stride(0),
      tensor_C.layout().stride(0),
      tensor_D.layout().stride(0),
      (int64_t)0 // Leading dimension of vector. This must be 0
    };

    Gemm gemm_op;

    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Gemm::can_implement() failed" << std::endl;
      return false;
    }

    size_t workspace_size = Gemm::get_workspace_size(arguments);
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    status = gemm_op.initialize(arguments, workspace.get());
    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Gemm::initialize() failed" << std::endl;
      return false;
    }

    //
    // Run the GEMM
    //

    status = gemm_op();

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Gemm::run() failed" << std::endl;
      return false;
    }

    hipError_t cuda_error = hipDeviceSynchronize();
    if (cuda_error != hipSuccess) {
      std::cerr << "CUDA error: " << hipGetErrorString(cuda_error) << std::endl;
      return false;
    }

    //
    // Verify
    //

    bool passed = true;
    if (options.reference_check) {
      passed &= this->verify(options);
    } else {
      std::cout << "Skipped reference check" << std::endl;
    }

    //
    // Warm up
    //

    for (int i = 0; i < options.warmup_iterations; ++i) {
      gemm_op();
    }

    //
    // Profile
    //

    hipEvent_t events[2];
    hipError_t error;
    for (auto & event : events) {
      error = hipEventCreate(&event);
      if (error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(error) << std::endl;
        return false;
      }
    }

    // Record an event at the start of a series of GEMM operations
    error = hipEventRecord(events[0]);
    if (error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Run profiling loop
    for (int iter = 0; iter < options.iterations; ++iter) {
      gemm_op();
    }

    // Record an event when the GEMM operations have been launched.
    error = hipEventRecord(events[1]);
    if (error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Wait for work on the device to complete.
    error = hipEventSynchronize(events[1]);
    if (error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(error) << std::endl;
      return false;
    }

    // Compute average runtime and GFLOPs.
    runtime_ms = runtime_ms / float(options.iterations);
    float gflops = options.gflops(runtime_ms / 1000.0f);

    std::cout << "Problem size: " << options.problem_size.m() << 'x' << options.problem_size.n() << 'x' << options.problem_size.k() << std::endl;
    std::cout << "Runtime (ms): " << runtime_ms << std::endl;
    std::cout << "GFLOPs/sec:   " << gflops << std::endl;

    // Cleanup
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }

    return passed;
  }

};

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const** argv) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (__CUDACC_VER_MAJOR__ < 12 || (__CUDACC_VER_MAJOR__ == 12 && __CUDACC_VER_MINOR__ < 4) ||
      (props.major != 8 && props.minor != 9)) {

    //
    // This example requires an NVIDIA Ada-architecture GPU.
    //

    std::cout
      << "CUTLASS's FP8 SM89 example requires a GPU of NVIDIA's Ada architecture "
      << "and CUDA toolkit version 12.4 or later.\n";

    return 0;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

  std::cout << "Running GEMM with staged accumulation (OpMultiplyAdd)" << std::endl;
  std::cout << "=====================================================" << std::endl;
  TestbedRunner<Gemm_<cutlass::arch::OpMultiplyAdd>> testbed_staged_accum;
  bool passed = testbed_staged_accum.run(options);

  if (passed) {
    std::cout << "Passed" << std::endl;
  } else {
    std::cout << "Failed" << std::endl;
  }

  std::cout << "\nRunning GEMM with fast accumulation (OpMultiplyAddFastAccum)" << std::endl;
  std::cout << "============================================================" << std::endl;
  TestbedRunner<Gemm_<cutlass::arch::OpMultiplyAddFastAccum>> testbed_fast_accum;
  passed = testbed_fast_accum.run(options);

  if (passed) {
    std::cout << "Passed" << std::endl;
  } else {
    std::cout << "Failed" << std::endl;
  }

  return 0;
}
