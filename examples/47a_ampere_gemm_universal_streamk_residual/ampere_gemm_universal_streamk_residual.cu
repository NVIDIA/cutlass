#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/***************************************************************************************************
 Example contrasting the Stream-K parallel decomposition for GEMM threadblocks versus the
 "classic data-parallel" and "Split-K" decompositions + residual add.

 For more details regarding the Stream-K method, see "Stream-K: Work-centric Parallel Decomposition
 for Dense Matrix-Matrix Multiplication on the GPU" (https://arxiv.org/abs/2301.03598)

 Requires NVIDIA Ampere or newer device (SM80+).

 - To lock persistence mode, power (400W), clocks (1005MHz) for evaluation (assumes device 0 and A100)

     cutlass$ sudo nvidia-smi -pm 1 -i 0

     cutlass$ sudo nvidia-smi -i 0 -pl 400

     cutlass$ sudo nvidia-smi -i 0 -lgc 1005

 - Build and run:

     cutlass$ mkdir build

     cutlass$ cd build

     cutlass/build$ cmake .. -DCUTLASS_NVCC_ARCHS=80

     cutlass/build$ make 47a_ampere_gemm_universal_streamk_residual

     cutlass/build$ ./examples/47a_ampere_gemm_universal_streamk_residual/47_ampere_gemm_universal_streamk_residual

 **************************************************************************************************/

#include <iostream>
#include <string>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/gemm/device/gemm_universal_with_broadcast.h"
#include "cutlass/gemm/device/gemm_universal_streamk_with_broadcast.h"
#include "cutlass/epilogue/thread/linear_combination_residual_block.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_foreach.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"



/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations (cutlass_tensorop_h16816gemm_128x128_32x4_nn_align8)
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA         = cutlass::half_t;                                  // Element type for A matrix operand
using         LayoutA          = cutlass::layout::RowMajor;                        // Layout type for A matrix operand
constexpr int AlignmentA       = 128 / cutlass::sizeof_bits<ElementA>::value;      // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB         = cutlass::half_t;                                  // Element type for B matrix operand
using         LayoutB          = cutlass::layout::RowMajor;                        // Layout type for B matrix operand
constexpr int AlignmentB       = 128 / cutlass::sizeof_bits<ElementB>::value;      // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D/T/Z matrix configuration
using         ElementC         = cutlass::half_t;                                  // Element type for C matrix operands
using         LayoutC          = cutlass::layout::RowMajor;                        // Layout type for C matrix operands
constexpr int AlignmentC       = 128 / cutlass::sizeof_bits<ElementC>::value;      // Memory access granularity/alignment of C matrices in units of elements (up to 16 bytes)

// Output matrix configuration
using         ElementOutput    = cutlass::half_t;                                  // Element type for output matrix operands
using         LayoutOutput     = cutlass::layout::RowMajor;                        // Layout type for output matrix operands
// constexpr int AlignmentOutput  = 128 / cutlass::sizeof_bits<ElementOutput>::value; // Memory access granularity/alignment of output matrices in units of elements (up to 16 bytes)

// Multiply-accumulate blocking/pipelining details
using ElementAccumulator  = cutlass::half_t;                          // Element type for internal accumulation
using ElementCompute      = cutlass::half_t;                          // Element type for compute
using ArchTag             = cutlass::arch::Sm80;                      // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;           // Operator class tag
using ThreadblockShape    = cutlass::gemm::GemmShape<128, 128, 32>;   // Threadblock-level tile size (concept: GemmShape)
using WarpShape           = cutlass::gemm::GemmShape<64, 64, 32>;     // Warp-level tile size (concept: GemmShape)
using InstructionShape    = cutlass::gemm::GemmShape<16, 8, 16>;      // Instruction-level tile size (concept: GemmShape)
constexpr int NumStages   = 4;                                        // Number of global->shared pipeline stages used in the GEMM mainloop

// Residual block configuration

// Epilogue output operator
/// Using LinearCombinationResidualBlock
/// Models a residual block of the form: UnaryOp(BinaryOp(BinaryOp(ActivationOp(TensorOp(X) + bias), residual1), residual2))
using EpilogueOp = cutlass::epilogue::thread::LinearCombinationResidualBlock<  
    ElementOutput,                        // Element type for output matrix
    ElementAccumulator,                   // Element type from internal accumulation
    ElementCompute,                       // Element type from internal accumulation
    ElementC,                             // Element type for C/D/T/Z matrix operands
    AlignmentC,                           // Memory access granularity of C and D matrix in units of elements
    cutlass::epilogue::thread::Identity,  // Activation
    cutlass::plus,                        // Binary operation 1
    cutlass::epilogue::thread::Identity   // Unary operation
    >;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

// Classic data-parallel device GEMM implementation type
using DeviceGemmBasic = cutlass::gemm::device::GemmUniversalWithBroadcast<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    NumStages,
    AlignmentA,
    AlignmentB>;

// StreamK device GEMM implementation type
using DeviceGemmStreamK = cutlass::gemm::device::GemmUniversalStreamkWithBroadcast<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::ThreadblockSwizzleStreamK,
    NumStages,
    AlignmentA,
    AlignmentB>;


/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(true)
  {}

};


/// Command line options parsing
struct Options
{
  std::string               command_name;
  bool                      help;
  cutlass::gemm::GemmCoord  problem_size;
  float                     alpha;
  float                     beta;
  int                       split_k_factor;
  int                       avail_sms;
  bool                      reference_check;
  int                       iterations;

  cutlass::HostTensor<ElementA, LayoutA> tensor_a;
  cutlass::HostTensor<ElementB, LayoutB> tensor_b;
  cutlass::HostTensor<ElementC, LayoutC> tensor_c;
  cutlass::HostTensor<ElementC, LayoutC> tensor_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_ref_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_z;
  cutlass::HostTensor<ElementC, LayoutC> tensor_t;

  Options(std::string command_name) :
    command_name(command_name),
    help(false),
    problem_size({2048, 2048, 2048}),
    alpha(1.0f),
    beta(0.0f),
    split_k_factor(1),
    avail_sms(-1),              // Number of device SMs to use is unlimited
    reference_check(true),
    iterations(10000)
  {}

  bool valid() const
  {
    return true;
  }

  void parse(int argc, char const **args)
  {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("split", split_k_factor);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const
  {
    out
      << "Performs a GEMM computation.\n"
      << "\n"
      << "Options:\n"
      << "\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --split=<int>               Split-K factor to emulate\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << command_name << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    return 2.0 * double(problem_size.product()) / double(1.0e9) / runtime_s;
  }
};


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Populates a DeviceGemmBasic::Arguments structure from the given commandline options
typename DeviceGemmBasic::Arguments args_from_options(
    const DeviceGemmBasic &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_z,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_t
    )
{
  return typename DeviceGemmBasic::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    tensor_z.device_data(),                   // ptr_Vector
    tensor_t.device_data(),                   // ptr_Tensor
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    options.problem_size.mn().product(),      // batch_stride_Vector
    options.problem_size.mn().product(),      // batch_stride_Tensor
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0),              // stride_d
    tensor_z.layout().stride(0),              // stride_Vector
    tensor_t.layout().stride(0));             // stride_Tensor
}

/// Populates a DeviceGemmStreamK::Arguments structure from the given commandline options
typename DeviceGemmStreamK::Arguments args_from_options(
    const DeviceGemmStreamK &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_z,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_t
    )
{
  return typename DeviceGemmStreamK::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    tensor_z.device_data(),                   // ptr_Vector
    tensor_t.device_data(),                   // ptr_Tensor
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    options.problem_size.mn().product(),      // batch_stride_Vector
    options.problem_size.mn().product(),      // batch_stride_Tensor
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0),              // stride_d
    tensor_z.layout().stride(0),              // stride_Vector
    tensor_t.layout().stride(0),              // stride_Tensor
    options.avail_sms);                       // avail_sms
}


/// Tensor MSE function
//// Useful when we expect outputs to approximately match, but 
//// not exactly.
template <
  typename Element,               ///< Element type
  typename Layout>                ///< Layout function
struct TensorMSEFunc {

  //
  // Data members
  //

  cutlass::TensorView<Element, Layout> lhs;
  cutlass::TensorView<Element, Layout> rhs;
  double err;
  double cnt;

  /// Ctor
  TensorMSEFunc(): err(0), cnt(0) { }

  /// Ctor
  TensorMSEFunc(
    cutlass::TensorView<Element, Layout> const &lhs_,
    cutlass::TensorView<Element, Layout> const &rhs_
  ) :
    lhs(lhs_), rhs(rhs_), err(0), cnt(0) { }

  /// Visits a coordinate
  void operator()(cutlass::Coord<Layout::kRank> const &coord) {

    Element lhs_ = lhs.at(coord);
    Element rhs_ = rhs.at(coord);
    
    cnt += double(1.0);
    double diff = double(lhs_) - double(rhs_);
    err += double(diff * diff);
  }

  double get_err() const {
    if (cnt == double(0.0))
      return double(0.0);
    return err / cnt;
  }
};

/// Mean squared error
template <
  typename Element,               ///< Element type
  typename Layout>                ///< Layout function
double TensorMSE(
  cutlass::TensorView<Element, Layout> const &lhs, 
  cutlass::TensorView<Element, Layout> const &rhs) {

  // Extents must be identical
  if (lhs.extent() != rhs.extent()) {
    return double(-1);
  }

  TensorMSEFunc<Element, Layout> func(lhs, rhs);
  cutlass::reference::host::TensorForEach(
    lhs.extent(),
    func
  );

  return func.get_err();
}

/// Execute a given example GEMM computation
template <typename DeviceGemmT>
Result run(std::string description, Options &options)
{
  // Display test description
  std::cout << std::endl << description << std::endl;

  // Zero-initialize test output matrix D
  cutlass::reference::host::TensorFill(options.tensor_d.host_view());
  options.tensor_d.sync_device();

  // Instantiate CUTLASS kernel depending on templates
  DeviceGemmT device_gemm;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of DeviceGemmT
  auto arguments = args_from_options(device_gemm, options, 
      options.tensor_a, options.tensor_b, options.tensor_c, options.tensor_d, 
      options.tensor_z, options.tensor_t);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = DeviceGemmT::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  CUTLASS_CHECK(device_gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(device_gemm.initialize(arguments, workspace.get()));

  // Correctness / Warmup iteration
  CUTLASS_CHECK(device_gemm());

  // Copy output data from CUTLASS and reference kernel to host for comparison
  options.tensor_d.sync_host();

  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = cutlass::reference::host::TensorEquals(
    options.tensor_d.host_view(),
    options.tensor_ref_d.host_view());

  double err = TensorMSE(
    options.tensor_d.host_view(),
    options.tensor_ref_d.host_view());

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << " - MSE: " << err << std::endl;

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(device_gemm());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPs: " << result.gflops << std::endl;
  }

  // TODO: uncomment when results match
  //if (!result.passed) {
  //  exit(-1);
  //}

  return result;
}


/// Program entrypoint
int main(int argc, const char **argv)
{
  // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  // Current device must must have compute capability at least 80
  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  if (!((props.major * 10 + props.minor) >= 80))
  {
    std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }

  // Parse commandline options
  Options options("ampere_streamk_residual_gemm");
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  std::cout <<
    options.iterations << " timing iterations of " <<
    options.problem_size.m() << " x " <<
    options.problem_size.n() << " x " <<
    options.problem_size.k() << " matrix-matrix multiply" << std::endl;

  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }


  //
  // Initialize GEMM datasets
  //

  // Initialize tensors using CUTLASS helper functions
  options.tensor_a.resize(options.problem_size.mk());       // <- Create matrix A with dimensions M x K
  options.tensor_b.resize(options.problem_size.kn());       // <- Create matrix B with dimensions K x N
  options.tensor_c.resize(options.problem_size.mn());       // <- Create matrix C with dimensions M x N
  options.tensor_d.resize(options.problem_size.mn());       // <- Create matrix D with dimensions M x N used to store output from CUTLASS kernel
  options.tensor_ref_d.resize(options.problem_size.mn());   // <- Create matrix D with dimensions M x N used to store output from reference kernel
  options.tensor_z.resize(options.problem_size.mn());       // <- Create residual matrix Z with dimensions M x N
  options.tensor_t.resize(options.problem_size.mn());       // <- Create residual matrix T with dimensions M x N

  // Fill matrix A on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_a.host_view(),
      1,
      ElementA(2),
      ElementA(-2),
      8);

  // Fill matrix B on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_b.host_view(),
      1,
      ElementB(2),
      ElementB(-2),
      8);

  // Fill matrix C on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_c.host_view(),
      1,
      ElementC(2),
      ElementC(-2),
      8);

  // Fill matrix Z on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_z.host_view(),
      1,
      ElementC(2),
      ElementC(-2),
      8);

  // Fill matrix T on host with uniform-random data [4, -4]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_t.host_view(),
      1,
      ElementC(2),
      ElementC(-2),
      8);


  //
  // Compute reference output
  //

  // Copy data from host to GPU
  options.tensor_a.sync_device();
  options.tensor_b.sync_device();
  options.tensor_c.sync_device();
  options.tensor_z.sync_device();
  options.tensor_t.sync_device();

  // Zero-initialize reference output matrix D
  cutlass::reference::host::TensorFill(options.tensor_ref_d.host_view());
  options.tensor_ref_d.sync_device();

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;

  // Launch device reference gemm kernel
  gemm_reference(
    options.problem_size,
    ElementAccumulator(options.alpha),
    options.tensor_a.device_ref(),
    options.tensor_b.device_ref(),
    ElementAccumulator(options.beta),
    options.tensor_c.device_ref(),
    options.tensor_ref_d.device_ref());

  // Wait for kernels to finish
  CUDA_CHECK(hipDeviceSynchronize());

  // Copy output data from reference kernel to host for comparison
  options.tensor_ref_d.sync_host();

  // TODO: move over to reference? 
  // TODO: move to device?
  for (int idx=0; idx < options.problem_size.mn().product(); ++idx) {
      ElementC* ptr_D = (ElementC*)(options.tensor_ref_d.host_view().ref().data());
      ElementC* ptr_Z = (ElementC*)(options.tensor_z.host_view().ref().data());
      ElementC* ptr_T = (ElementC*)(options.tensor_t.host_view().ref().data());
      ptr_D[idx] = (ptr_D[idx] + ptr_Z[idx]) + ptr_T[idx];
  }

  //
  // Evaluate CUTLASS kernels
  //

  // Test default operation
  if (options.split_k_factor == 1)
  {
    // Compare basic data-parallel version versus StreamK version using default load-balancing heuristics
    Result basic_dp         = run<DeviceGemmBasic>("Basic data-parallel GEMM", options);
    Result streamk_default  = run<DeviceGemmStreamK>("StreamK GEMM with default load-balancing", options);

    printf("  Speedup vs Basic-DP: %.3f\n", (basic_dp.avg_runtime_ms / streamk_default.avg_runtime_ms));

    // Show that StreamK can emulate basic data-parallel GEMM when we set the number of SMs to load-balance across = 1
    options.avail_sms       = 1;        // Set loadbalancing width to 1 SM (no load balancing)
    Result streamk_dp       = run<DeviceGemmStreamK>("StreamK emulating basic data-parallel GEMM", options);
    options.avail_sms       = -1;       // Reset loadbalancing width to unspecified SMs (i.e., the number of device SMs)

    printf("  Speedup vs Basic-DP: %.3f\n", (basic_dp.avg_runtime_ms / streamk_dp.avg_runtime_ms));

    options.split_k_factor++;     // Increment splitting factor for next evaluation

  }

  // Show that StreamK can emulate "Split-K" with a tile-splitting factor
  Result basic_splitk = run<DeviceGemmBasic>(
    std::string("Basic split-K GEMM with tile-splitting factor ") + std::to_string(options.split_k_factor),
    options);

  Result streamk_splitk = run<DeviceGemmStreamK>(
    std::string("StreamK emulating Split-K GEMM with tile-splitting factor ") + std::to_string(options.split_k_factor),
    options);

  printf("  Speedup vs Basic-SplitK: %.3f\n", (basic_splitk.avg_runtime_ms / streamk_splitk.avg_runtime_ms));

  return 0;
}
