
#include <hip/hip_runtime.h>
template <unsigned N> struct foo_t {
    static __device__ constexpr unsigned size = N;
};

template <class TA, class TB>
constexpr __device__ unsigned smem(TA t0, TB t1) {
    return TA::size + TB::size;
}