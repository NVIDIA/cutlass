/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file 
    \brief Unit tests for thread-level GEMM
*/

#include "../../common/cutlass_unit_test.h"

#include "cutlass/aligned_buffer.h"
#include "cutlass/half.h"

#include "cutlass/gemm/warp/default_mma_tensor_op.h"

#include "cutlass/core_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"

#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/gemm.h"

#include "testbed.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)


////////////////////////////////////////////////////////////////////////////////
/// F32 <= F16 * I8 + F32 (Upcast on Operand B)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_f16_i8, 128x128x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = cutlass::half_t;
  using ElementB = int8_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<128, 128, 64> >()
      .run();
}

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_f16_i8, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = cutlass::half_t;
  using ElementB = int8_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= I8 * F16 + F32 (Upcast on Operand A)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_i8_f16, 128x128x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = int8_t;
  using ElementB = cutlass::half_t;;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<128, 128, 64> >()
      .run();
}

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_i8_f16, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = int8_t;
  using ElementB = cutlass::half_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= F16 * U8 + F32 (Upcast on Operand B)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_f16_u8, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = cutlass::half_t;
  using ElementB = uint8_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_f16_u8, 128x128x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = cutlass::half_t;
  using ElementB = uint8_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<128, 128, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= U8 * F16 + F32 (Upcast on Operand A)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_u8_f16, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = uint8_t;
  using ElementB = cutlass::half_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_u8_f16, 128x128x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = uint8_t;
  using ElementB = cutlass::half_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<128, 128, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= B16 * U8 + F32 (Upcast on Operand B)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_bf16_u8, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = cutlass::bfloat16_t;
  using ElementB = uint8_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= U8 * BF16 + F32 (Upcast on Operand A)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_u8_bf16, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = uint8_t;
  using ElementB = cutlass::bfloat16_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= I8 * BF16 + F32 (Upcast on Operand A)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_bf16_i8, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = cutlass::bfloat16_t;
  using ElementB = int8_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

////////////////////////////////////////////////////////////////////////////////
/// F32 <= B16 * I8 + F32 (Upcast on Operand B)
////////////////////////////////////////////////////////////////////////////////

TEST(SM80_warp_gemm_mixed_input_tensor_op_crosswise_i8_bf16, 64x64x64_64x64x64_16x8x16) {
  using Shape = cutlass::gemm::GemmShape<64, 64, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
  using ElementA = int8_t;
  using ElementB = cutlass::bfloat16_t;
  using ElementC = float;
  using LayoutA = cutlass::layout::RowMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementA>::value, 64>;
  using LayoutB = cutlass::layout::ColumnMajorTensorOpMultiplicandCrosswise<
      cutlass::sizeof_bits<ElementB>::value, 64>;

  using MmaTensorOp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
      Shape, InstructionShape, ElementA, LayoutA, ElementB, LayoutB, ElementC,
      cutlass::layout::RowMajor, cutlass::arch::OpMultiplyAddMixedInputUpcast>::Type;

  test::gemm::warp::TransformTestbed<MmaTensorOp,
                            cutlass::gemm::GemmShape<64, 64, 64> >()
      .run();
}

#endif // if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)
