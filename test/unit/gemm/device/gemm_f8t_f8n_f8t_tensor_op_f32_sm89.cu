/***************************************************************************************************
 * Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Tests for device-wide GEMM interface with:
        A: row major, of type FE4M4 or FE5M2
        B: column major, of type FE4M3 or FE5M2
        C: row major, of FE4M3 or FE5M2
        Accum: F32
*/

#include <iostream>

#include "../../common/cutlass_unit_test.h"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/thread/activation.h"
#include "cutlass/epilogue/thread/linear_combination_generic_with_scaling.h"
#include "cutlass/gemm/device/gemm_universal_with_absmax.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "testbed.h"
#include "testbed_with_absmax.h"

#if defined(CUTLASS_ARCH_MMA_SM89_SUPPORTED)

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe4m3t_tensor_op_f32, identity_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe4m3t_tensor_op_f32, identity_fastacc_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;
  static int const kAlignment = 16;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages,
    kAlignment, kAlignment, cutlass::arch::OpMultiplyAddFastAccum
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe4m3t_tensor_op_f32, relu_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::ReLu,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::ReLu>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe5m2n_fe4m3t_tensor_op_f32, identity_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e5m2_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe5m2t_fe4m3n_fe4m3t_tensor_op_f32, identity_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e5m2_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe5m2t_fe5m2n_fe4m3t_tensor_op_f32, identity_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e5m2_t;
  using ElementB = cutlass::float_e5m2_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe5m2t_tensor_op_f32, identity_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e5m2_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe5m2t_fe5m2n_fe5m2t_tensor_op_f32, identity_diff_aux_output_types_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e5m2_t;
  using ElementB = cutlass::float_e5m2_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = cutlass::float_e5m2_t;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe4m3t_tensor_op_f32, identity_128x128x64_32x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 128, 64>, cutlass::gemm::GemmShape<32, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe4m3t_tensor_op_f32, identity_noScale_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = ElementOutput;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>(
    /* scaleA = */false,
    /* scaleB = */false,
    /* scaleC = */false
  );
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

TEST(SM89_Device_Gemm_fe4m3t_fe4m3n_fe4m3t_tensor_op_f32, identity_noAux_128x256x64_64x64x64) {
  using ElementA = cutlass::float_e4m3_t;
  using ElementB = cutlass::float_e4m3_t;
  using ElementOutput = cutlass::float_e4m3_t;
  using ElementAuxOutput = float;
  using ElementAccumulator = float;
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  static int const kStages = 3;

  using EpilogueOutputOp = cutlass::epilogue::thread::LinearCombinationGenericWithScalingAndAbsMax<
    cutlass::epilogue::thread::Identity,
    ElementOutput,
    ElementAuxOutput,
    128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator,
    ElementAccumulator
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalWithAbsMax<
    ElementA, LayoutA, ElementB, LayoutB, ElementOutput, LayoutC,
    ElementAccumulator, cutlass::arch::OpClassTensorOp, cutlass::arch::Sm89,
    cutlass::gemm::GemmShape<128, 256, 64>, cutlass::gemm::GemmShape<64, 64, 64>, cutlass::gemm::GemmShape<16, 8, 32>,
    EpilogueOutputOp, cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, kStages
  >;

  bool passed = test::gemm::device::TestAllGemmWithAbsmax<Gemm, test::gemm::device::Testbed<Gemm>, cutlass::epilogue::thread::Identity>();
  EXPECT_TRUE(passed);
}

////////////////////////////////////////////////////////////////////////////////

#endif  // CUTLASS_ARCH_MMA_SM89_SUPPORTED
