/***************************************************************************************************
 * Copyright (c) 2024 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"
#include "cutlass/arch/mma_sm90.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)

TEST(SM90_Device_Sparse_Gemm_s8t_s8n_s32t_tensorop_s32, 128x128x128_1x1x1) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_128>;
  using ClusterShape = Shape<_1,_1,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      int32_t, int32_t,
      int32_t, LayoutC, 4,
      int32_t, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      int8_t, LayoutA, 32,
      int8_t, LayoutB, 16,
      int32_t,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

TEST(SM90_Device_Sparse_Gemm_s8t_s8n_s32t_tensorop_s32, 128x128x128_1x1x1_warpspecialized) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_128>;
  using ClusterShape = Shape<_1,_1,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      int32_t, int32_t,
      int32_t, LayoutC, 4,
      int32_t, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      int8_t, LayoutA, 32,
      int8_t, LayoutB, 16,
      int32_t,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

TEST(SM90_Device_Sparse_Gemm_s8t_s8n_s32t_tensorop_s32, 128x128x256_1x2x1_cooperative) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_256>;
  using ClusterShape = Shape<_1,_2,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      int32_t, int32_t,
      int32_t, LayoutC, 4,
      int32_t, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      int8_t, LayoutA, 32,
      int8_t, LayoutB, 16,
      int32_t,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

TEST(SM90_Device_Sparse_Gemm_s8t_s8n_s32t_tensorop_s32, 128x128x64_2x1x1_pingpong) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_64>;
  using ClusterShape = Shape<_2,_1,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      int32_t, int32_t,
      int32_t, LayoutC, 4,
      int32_t, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      int8_t, LayoutA, 32,
      int8_t, LayoutB, 16,
      int32_t,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecializedPingpong
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

#endif // #if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)
