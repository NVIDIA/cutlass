/***************************************************************************************************
 * Copyright (c) 2024 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"
#include "cutlass/arch/mma_sm90.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)

TEST(SM90_Device_Sparse_Gemm_tf16t_tf16n_f32t_tensorop_f32, 128x128x64_1x1x1) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_64>;
  using ClusterShape = Shape<_1,_1,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      float, LayoutA, 8,
      float, LayoutB, 4,
      float,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

TEST(SM90_Device_Sparse_Gemm_tf16t_tf16n_f32t_tensorop_f32, 128x128x64_1x1x1_warpspecialized) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_64>;
  using ClusterShape = Shape<_1,_1,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      float, LayoutA, 8,
      float, LayoutB, 4,
      float,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

TEST(SM90_Device_Sparse_Gemm_tf32t_tf32n_f32t_tensorop_f32, 128x128x32_1x2x1_cooperative) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_32>;
  using ClusterShape = Shape<_1,_2,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::TmaWarpSpecializedCooperative
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      float, LayoutA, 8,
      float, LayoutB, 4,
      float,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

TEST(SM90_Device_Sparse_Gemm_tf32t_tf32n_f32t_tensorop_f32, 128x128x16_2x1x1_pingpong) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape = Shape<_128,_128,_16>;
  using ClusterShape = Shape<_2,_1,_1>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::TmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassSparseTensorOp,
      float, LayoutA, 8,
      float, LayoutB, 4,
      float,
      TileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecializedPingpong
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  bool result = TestAll<Gemm>(1.0, 1.0, CheckEquality::EXACT);
  EXPECT_TRUE(result);
}

#endif // #if defined(CUTLASS_ARCH_MMA_SPARSE_SM90_SUPPORTED)
