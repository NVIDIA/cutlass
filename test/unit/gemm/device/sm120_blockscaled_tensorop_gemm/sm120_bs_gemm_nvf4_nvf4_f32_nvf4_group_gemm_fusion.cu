/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/


/*! \file
    \brief Tests for device-wide grouped GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/epilogue/thread/activation.h"

#include "../../../common/cutlass_unit_test.h"
#include "../gemm_testbed_3x_ptr_array.hpp"


using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)

// Pingpong kernel schedule
TEST(SM120_Device_Gemm_e2m1t_e2m1n_e2m1t_tensorop_f32_epilogue_VS16_group_pingpong, row_sf) {
  using ElementInput = float_e2m1_t;
  using ElementA = cutlass::nv_float4_t<ElementInput>;
  using ElementB = cutlass::nv_float4_t<ElementInput>;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::float_e2m1_t;
  using ElementCompute = float;
  using ElementAccumulator = float;
  using ElementSF = cutlass::float_ue4m3_t;
  using ElementSFD  = ElementSF;
  using ElementAccumulator = float;
  using GmemLayoutA = cutlass::layout::RowMajor;
  using GmemLayoutB = cutlass::layout::ColumnMajor;
  using GmemLayoutC = cutlass::layout::RowMajor;
  constexpr int SFVectorSize = 16;
  using TileShape_MNK = Shape<_128,_128,_128>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementInput>::value;
  constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementInput>::value;  
  constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;
  constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;  

  //
  // Construct CollectiveEpilogue
  //

  constexpr int OutputSFVectorSize = SFVectorSize;
  // D = alpha * acc + beta * C
  // With Row-major BlockScaleFactor generation.
  using FusionOperation = cutlass::epilogue::fusion::LinCombBlockScaleFactor<
      OutputSFVectorSize,
      ElementD, 
      ElementCompute, 
      ElementSFD, GmemLayoutC,
      ElementC>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, GmemLayoutC *, AlignmentC,
      ElementD, GmemLayoutC *, AlignmentD,
      cutlass::epilogue::collective::EpilogueScheduleAuto,
      FusionOperation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      ElementA, GmemLayoutA *, AlignmentA,
      ElementB, GmemLayoutB *, AlignmentB,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpong
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::GroupProblemShape<Shape<int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = test::gemm::device::TestSmallFusion<Gemm>(1.0, 0.5);
  EXPECT_TRUE(pass);
}



TEST(SM120_Device_Gemm_e2m1t_e2m1n_e2m1t_tensorop_f32_epilogue_VS16_group_pingpong, silu_row_sf) {
  using ElementInput = float_e2m1_t;
  using ElementA = cutlass::nv_float4_t<ElementInput>;
  using ElementB = cutlass::nv_float4_t<ElementInput>;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::float_e2m1_t;
  using ElementCompute = float;
  using ElementAccumulator = float;
  using ElementSF = cutlass::float_ue4m3_t;
  using ElementSFD  = ElementSF;
  using ElementAccumulator = float;
  using GmemLayoutA = cutlass::layout::RowMajor;
  using GmemLayoutB = cutlass::layout::ColumnMajor;
  using GmemLayoutC = cutlass::layout::RowMajor;
  constexpr int SFVectorSize = 16;
  using TileShape_MNK = Shape<_128,_128,_256>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementInput>::value;
  constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementInput>::value;  
  constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;
  constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;  

  //
  // Construct CollectiveEpilogue
  //

  constexpr int OutputSFVectorSize = SFVectorSize;
  // D = SiLu(alpha * acc + beta * C)
  // With Row-major BlockScaleFactor generation.
  using FusionOperation = cutlass::epilogue::fusion::LinCombEltActBlockScaleFactor<
      cutlass::epilogue::thread::SiLu,
      OutputSFVectorSize,
      ElementD, 
      ElementCompute, 
      ElementSFD, GmemLayoutC,
      ElementC>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, GmemLayoutC *, AlignmentC,
      ElementD, GmemLayoutC *, AlignmentD,
      cutlass::epilogue::collective::EpilogueScheduleAuto,
      FusionOperation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      ElementA, GmemLayoutA *, AlignmentA,
      ElementB, GmemLayoutB *, AlignmentB,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelPtrArrayTmaWarpSpecializedPingpong
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::GroupProblemShape<Shape<int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = test::gemm::device::TestSmallFusion<Gemm>(1.0, 0.5);
  EXPECT_TRUE(pass);
}


// Cooperative kenel schedule
TEST(SM120_Device_Gemm_e2m1t_e2m1n_e2m1t_tensorop_f32_epilogue_VS16_group_cooperative, row_sf) {
  using ElementInput = float_e2m1_t;
  using ElementA = cutlass::nv_float4_t<ElementInput>;
  using ElementB = cutlass::nv_float4_t<ElementInput>;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::float_e2m1_t;
  using ElementCompute = float;
  using ElementAccumulator = float;
  using ElementSF = cutlass::float_ue4m3_t;
  using ElementSFD  = ElementSF;
  using ElementAccumulator = float;
  using GmemLayoutA = cutlass::layout::RowMajor;
  using GmemLayoutB = cutlass::layout::ColumnMajor;
  using GmemLayoutC = cutlass::layout::RowMajor;
  constexpr int SFVectorSize = 16;
  using TileShape_MNK = Shape<_128,_128,_128>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementInput>::value;
  constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementInput>::value;  
  constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;
  constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;  

  //
  // Construct CollectiveEpilogue
  //

  constexpr int OutputSFVectorSize = SFVectorSize;
  // D = alpha * acc + beta * C
  // With Row-major BlockScaleFactor generation.
  using FusionOperation = cutlass::epilogue::fusion::LinCombBlockScaleFactor<
      OutputSFVectorSize,
      ElementD, 
      ElementCompute, 
      ElementSFD, GmemLayoutC,
      ElementC>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, GmemLayoutC *, AlignmentC,
      ElementD, GmemLayoutC *, AlignmentD,
      cutlass::epilogue::collective::EpilogueScheduleAuto,
      FusionOperation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      ElementA, GmemLayoutA *, AlignmentA,
      ElementB, GmemLayoutB *, AlignmentB,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::GroupProblemShape<Shape<int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = test::gemm::device::TestSmallFusion<Gemm>(1.0, 0.5);
  EXPECT_TRUE(pass);
}



TEST(SM120_Device_Gemm_e2m1t_e2m1n_e2m1t_tensorop_f32_epilogue_VS16_group_cooperative, silu_row_sf) {
  using ElementInput = float_e2m1_t;
  using ElementA = cutlass::nv_float4_t<ElementInput>;
  using ElementB = cutlass::nv_float4_t<ElementInput>;
  using ElementC = cutlass::half_t;
  using ElementD = cutlass::float_e2m1_t;
  using ElementCompute = float;
  using ElementAccumulator = float;
  using ElementSF = cutlass::float_ue4m3_t;
  using ElementSFD  = ElementSF;
  using ElementAccumulator = float;
  using GmemLayoutA = cutlass::layout::RowMajor;
  using GmemLayoutB = cutlass::layout::ColumnMajor;
  using GmemLayoutC = cutlass::layout::RowMajor;
  constexpr int SFVectorSize = 16;
  using TileShape_MNK = Shape<_128,_128,_256>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementInput>::value;
  constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementInput>::value;  
  constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;
  constexpr int AlignmentD  = 128 / cutlass::sizeof_bits<ElementD>::value;  

  //
  // Construct CollectiveEpilogue
  //

  constexpr int OutputSFVectorSize = SFVectorSize;
  // D = SiLu(alpha * acc + beta * C)
  // With Row-major BlockScaleFactor generation.
  using FusionOperation = cutlass::epilogue::fusion::LinCombEltActBlockScaleFactor<
      cutlass::epilogue::thread::SiLu,
      OutputSFVectorSize,
      ElementD, 
      ElementCompute, 
      ElementSFD, GmemLayoutC,
      ElementC>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, GmemLayoutC *, AlignmentC,
      ElementD, GmemLayoutC *, AlignmentD,
      cutlass::epilogue::collective::EpilogueScheduleAuto,
      FusionOperation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm120, cutlass::arch::OpClassBlockScaledTensorOp,
      ElementA, GmemLayoutA *, AlignmentA,
      ElementB, GmemLayoutB *, AlignmentB,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::GroupProblemShape<Shape<int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = test::gemm::device::TestSmallFusion<Gemm>(1.0, 0.5);
  EXPECT_TRUE(pass);
}
#endif // #if defined(CUTLASS_ARCH_MMA_SM120_SUPPORTED)
