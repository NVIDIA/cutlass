#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Tests for Sm90 f8_f8_f32 with EVT epilogue
    ScaledLinCombPerRowBiasEltAct and ScaledLinCombPerRowBiasEltActAmaxAux
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/sm70_epilogue_vectorized.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/epilogue/thread/linear_combination_bias_elementwise.h"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x_evt.hpp"
#include "sm90_evt_operations.hpp"


#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

using namespace cute;

// Z = scale_a * scale_b * alpha * acc + beta * scale_c * C + per-row bias
// if D is fp8 
//   D = scale_d * activation(Z)
// else
//   D = activation(Z)
TEST(SM90_Device_Gemm_e4m3t_e4m3n_f32n_tensor_op_gmma_f32_cooperative_epilogue, 128x128x128_1x4x1_ScaledLinCombPerRowBiasEltAct) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_128,_128,_128>;
  using ClusterShape_MNK = Shape<_1,_4,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionCallbacks = cutlass::epilogue::fusion::Sm90ScaledLinCombPerRowBiasEltAct<
    TileShape_MNK,                      // CtaTileShapeMNK
    cutlass::epilogue::thread::ReLu,    // ActivationFn
    float,                              // ElementOutput
    float,                              // ElementCompute
    float                               // ElementBias
  >;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      EpilogueSchedule,
      FusionCallbacks
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::float_e4m3_t, LayoutA, 16,
      cutlass::float_e4m3_t, LayoutB, 16,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  // Host reference
  using HostReference = test::gemm::device::HostScaledLinCombPerRowBiasEltAct<
    Gemm, cutlass::epilogue::thread::ReLu, float
  >;
  bool passed = test::gemm::device::TestAllEVT<Gemm, HostReference>(true);
  EXPECT_TRUE(passed);
}

// Z = scale_a * scale_b * alpha * acc + scale_c * beta * C + per-row bias
// if D is fp8 
//   amax_d = max(abs(elements in activation(Z)))
//   D = scale_d * activation(Z)
// else
//   D = activation(Z)
// if Aux is fp8 
//   amax_aux = max(abs(elements in Z))
//   Aux = scale_aux * Z
// else
//   Aux = Z
TEST(SM90_Device_Gemm_e4m3t_e4m3n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 128x128x128_1x2x1_ScaledLinCombPerRowBiasEltActAmaxAux) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_128,_128,_128>;
  using ClusterShape_MNK = Shape<_1,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using EpilogueTileType = cutlass::epilogue::collective::EpilogueTileAuto;
  using EpilogueDescriptor = cutlass::epilogue::collective::detail::EpilogueDescriptor<
    TileShape_MNK, EpilogueTileType, float, float, EpilogueSchedule>;
  using AuxStoreDescriptor = cutlass::epilogue::collective::detail::AuxStoreDescriptor<
    EpilogueDescriptor, cutlass::layout::RowMajor, float>;
    
  using FusionCallbacks = cutlass::epilogue::fusion::Sm90ScaledLinCombPerRowBiasEltActAmaxAux<
    TileShape_MNK,                               // CtaTileShapeMNK
    typename EpilogueDescriptor::EpilogueTile,   // EpilogueTile
    EpilogueDescriptor::StagesD,                 // StagesD
    typename AuxStoreDescriptor::Stride,         // StrideAux
    typename AuxStoreDescriptor::SmemLayoutAtom, // SmemLayoutAtom
    typename AuxStoreDescriptor::CopyOpR2S,      // CopyOpR2S
    cutlass::epilogue::thread::ReLu,             // ActivationFn
    float,                                       // ElementOutput
    float,                                       // ElementCompute
    float,                                       // ElementBias
    float                                        // ElementScalar
  >;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      EpilogueTileType,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      EpilogueSchedule,
      FusionCallbacks
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::float_e4m3_t, LayoutA, 16,
      cutlass::float_e4m3_t, LayoutB, 16,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  // Host reference
  using HostReference = test::gemm::device::HostScaledLinCombPerRowBiasEltActAmaxAux<
    Gemm, cutlass::epilogue::thread::ReLu, float
  >;
  bool passed = test::gemm::device::TestAllEVT<Gemm, HostReference>(true);
  EXPECT_TRUE(passed);
}
#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
