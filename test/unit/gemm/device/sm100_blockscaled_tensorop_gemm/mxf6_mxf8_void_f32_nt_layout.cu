/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/



/*! \file
    \brief Unit tests for mxfp6xmxfp8 Block Scaled Gemm

    * A tensor: 
      * Types: {e2m3,e3m2}xue8m0
      * Layout: Column Major (N)
      * Alignment: 128 elements
    * B tensor: 
      * Types: {e5m2,e4m3}xue8m0
      * Layout: Row Major (T)
      * Alignment: 16 elements
    * Mma Tile Shapes supported:
      For the A tensor (mxfp6 type) the tile dimension with stride-1 should be divisible by 128, i.e., 128 element aligned.
      Support Matrix (Y: Yes, N: No)
      | 1/2 SM | Mma Tile Size | TN | TT | NT (*) | NN |
      |--------|---------------|----|----|--------|----|
      | 1SM    | 128x128x128   | Y  | Y  | Y      | Y  |
      | 1SM    | 128x192x128   | Y  | Y  | Y      | Y  |
      | 1SM    | 128x256x128   | Y  | Y  | Y      | Y  |
      | 2SM    | 256x128x128   | Y  | Y  | Y      | Y  |
      | 2SM    | 256x192x128   | Y  | Y  | Y      | Y  |
      | 2SM    | 256x256x128   | Y  | Y  | Y      | Y  |

      (*) Unit tests in this file
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"

#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"

#include "cutlass/epilogue/thread/activation.h"
#include "../../../common/cutlass_unit_test.h"

#include "../gemm_testbed_3x.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

TEST(SM100Only_Device_Gemm_ue8m0xe2m3n_ue8m0xe5m2t_void_f32t_bstensorop_f32, 128x128x128_4x4x1_1sm_auto) {
  // Describe A and B tensors
  using ElementA = cutlass::mx_float6_t<cutlass::float_e2m3_t>;
  constexpr int AlignA = 128;
  using GmemLayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e5m2_t>;
  constexpr int AlignB = 16;
  using GmemLayoutB = cutlass::layout::RowMajor;

  // Describe C and D tensors
  using ElementC = void;
  constexpr int AlignC = 4;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_128,_128>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_4,_4,_1>;
  // Collective Epilogue takes the output tile shape for 1 CTA
  using PerSmTileShape_MNK = Shape<_128,_128,_128>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      PerSmTileShape_MNK, ClusterShape_MNK,                                     // Epilogue tile shape, and cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto                   // Epilogue schedule policy
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecialized1SmBlockScaledSm100
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}


TEST(SM100Only_Device_Gemm_ue8m0xe2m3n_ue8m0xe4m3t_void_f32t_bstensorop_f32, 128x192x128_2x2x1_1sm_auto) {
  // Describe A and B tensors
  using ElementA = cutlass::mx_float6_t<cutlass::float_e2m3_t>;
  constexpr int AlignA = 128;
  using GmemLayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
  constexpr int AlignB = 16;
  using GmemLayoutB = cutlass::layout::RowMajor;

  // Describe C and D tensors
  using ElementC = void;
  constexpr int AlignC = 4;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_192,_128>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_2,_2,_1>;
  // Collective Epilogue takes the output tile shape for 1 CTA
  using PerSmTileShape_MNK = Shape<_128,_192,_128>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      PerSmTileShape_MNK, ClusterShape_MNK,                                     // Epilogue tile shape, and cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto                   // Epilogue schedule policy
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto    // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

TEST(SM100Only_Device_Gemm_ue8m0xe3m2n_ue8m0xe5m2t_void_f32t_bstensorop_f32, 128x256x128_4x2x1_1sm_auto) {
  // Describe A and B tensors
  using ElementA = cutlass::mx_float6_t<cutlass::float_e3m2_t>;
  constexpr int AlignA = 128;
  using GmemLayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e5m2_t>;
  constexpr int AlignB = 16;
  using GmemLayoutB = cutlass::layout::RowMajor;

  // Describe C and D tensors
  using ElementC = void;
  constexpr int AlignC = 4;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_256,_128>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_4,_2,_1>;
  // Collective Epilogue takes the output tile shape for 1 CTA
  using PerSmTileShape_MNK = Shape<_128,_256,_128>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      PerSmTileShape_MNK, ClusterShape_MNK,                                     // Epilogue tile shape, and cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto                   // Epilogue schedule policy
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecialized1SmMxf8f6f4Sm100
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

TEST(SM100Only_Device_Gemm_ue8m0xe3m2n_ue8m0xe4m3t_void_f32t_bstensorop_f32, 256x128x128_4x1x1_2sm_auto) {
  // Describe A and B tensors
  using ElementA = cutlass::mx_float6_t<cutlass::float_e3m2_t>;
  constexpr int AlignA = 128;
  using GmemLayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
  constexpr int AlignB = 16;
  using GmemLayoutB = cutlass::layout::RowMajor;

  // Describe C and D tensors
  using ElementC = void;
  constexpr int AlignC = 4;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_256,_128,_128>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_4,_1,_1>;
  // Collective Epilogue takes the output tile shape for 1 CTA
  using PerSmTileShape_MNK = Shape<_128,_128,_128>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      PerSmTileShape_MNK, ClusterShape_MNK,                                     // Epilogue tile shape, and cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto                   // Epilogue schedule policy
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto    // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

TEST(SM100Only_Device_Gemm_ue8m0xe2m3n_ue8m0xe4m3t_void_f32t_bstensorop_f32, 256x192x128_2x2x1_2sm_auto) {
  // Describe A and B tensors
  using ElementA = cutlass::mx_float6_t<cutlass::float_e2m3_t>;
  constexpr int AlignA = 128;
  using GmemLayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e4m3_t>;
  constexpr int AlignB = 16;
  using GmemLayoutB = cutlass::layout::RowMajor;

  // Describe C and D tensors
  using ElementC = void;
  constexpr int AlignC = 4;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_256,_192,_128>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_2,_2,_1>;
  // Collective Epilogue takes the output tile shape for 1 CTA
  using PerSmTileShape_MNK = Shape<_128,_192,_128>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      PerSmTileShape_MNK, ClusterShape_MNK,                                     // Epilogue tile shape, and cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto                   // Epilogue schedule policy
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecialized2SmBlockScaledSm100
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

TEST(SM100Only_Device_Gemm_ue8m0xe3m2n_ue8m0xe5m2t_void_f32t_bstensorop_f32, 256x256x128_4x2x1_2sm_auto) {
  // Describe A and B tensors
  using ElementA = cutlass::mx_float6_t<cutlass::float_e3m2_t>;
  constexpr int AlignA = 128;
  using GmemLayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::mx_float8_t<cutlass::float_e5m2_t>;
  constexpr int AlignB = 16;
  using GmemLayoutB = cutlass::layout::RowMajor;

  // Describe C and D tensors
  using ElementC = void;
  constexpr int AlignC = 4;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_256,_256,_128>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_4,_2,_1>;
  // Collective Epilogue takes the output tile shape for 1 CTA
  using PerSmTileShape_MNK = Shape<_128,_256,_128>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      PerSmTileShape_MNK, ClusterShape_MNK,                                     // Epilogue tile shape, and cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto                   // Epilogue schedule policy
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassBlockScaledTensorOp,      // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::KernelTmaWarpSpecialized2SmMxf8f6f4Sm100
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

#endif // #if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)
