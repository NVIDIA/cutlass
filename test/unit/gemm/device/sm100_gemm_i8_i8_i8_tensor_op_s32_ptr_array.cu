/***************************************************************************************************
 * Copyright (c) 2023 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/



/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x_ptr_array.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////// 128x64x128 Cluster1x1x1 TMEM 4x1 ////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM100_Device_Gemm_s8t_s8n_s8n_tensorop_1cta_s32_ptr_array, 128x64x128_1x1x1) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementA = int8_t;
  using ElementB = int8_t;
  using ElementC = int8_t;
  using ElementD = int8_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = float;
  using ElementBias = int8_t;
  using ClusterTileShape = cute::Shape<_128,_64,Int<128 / sizeof(ElementA)>>;
  using ClusterShape = Shape<_1,_1,_1>;
  using AtomThrShape = decltype(shape_div(ClusterShape{}, Shape<_1,_1,_1>{}));

  using OutputCtaShape = decltype(shape_div(ClusterTileShape{}, ClusterShape{}));
  using MmaTileShape = decltype(shape_div(ClusterTileShape{}, AtomThrShape{}));

  using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized1Sm;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      OutputCtaShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, 16 / sizeof(ElementC),
      ElementD, LayoutC, 16 / sizeof(ElementD),
      EpilogueSchedule
    >::CollectiveOp;

  using MainloopSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized1SmSm100;
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 16 / sizeof(ElementA),
      ElementB, LayoutB, 16 / sizeof(ElementB),
      ElementAccumulator,
      MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopSchedule
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::ArrayProblemShape<Shape<int,int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = TestSmall<Gemm>(2, 0.5, CheckEquality::EXACT);
  EXPECT_TRUE(pass);
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////// 128x64x128 Cluster4x2x1 TMEM 4x1 ////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM100_Device_Gemm_s8t_s8n_s8n_tensorop_1cta_s32_ptr_array, 512x128x128_4x2x1) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementA = int8_t;
  using ElementB = int8_t;
  using ElementC = int8_t;
  using ElementD = int8_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = float;
  using ElementBias = int8_t;
  using ClusterTileShape = Shape<_512,_128,Int<128 / sizeof(ElementA)>>;
  using ClusterShape = Shape<_4,_2,_1>;
  using OutputCtaShape = decltype(shape_div(ClusterTileShape{}, ClusterShape{})); 
  using AtomThrShape = decltype(shape_div(ClusterShape{}, Shape<_1,_1,_1>{}));

  using OutputCtaShape = decltype(shape_div(ClusterTileShape{}, ClusterShape{}));
  using MmaTileShape = decltype(shape_div(ClusterTileShape{}, AtomThrShape{}));

  using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized1Sm;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      OutputCtaShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, 16 / sizeof(ElementC),
      ElementD, LayoutC, 16 / sizeof(ElementD),
      EpilogueSchedule
    >::CollectiveOp;

  using MainloopSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized1SmSm100;
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 16 / sizeof(ElementA),
      ElementB, LayoutB, 16 / sizeof(ElementB),
      ElementAccumulator,
      MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopSchedule
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::ArrayProblemShape<Shape<int,int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = TestSmall<Gemm>(2, 0.5, CheckEquality::EXACT);
  EXPECT_TRUE(pass);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////// 64x256x128 Cluster1x1x1 TMEM 4x1 ////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM100_Device_Gemm_s8t_s8n_s32n_tensorop_1cta_s32_ptr_array, 64x256x128_1x1x1) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementA = int8_t;
  using ElementB = int8_t;
  using ElementC = int32_t;
  using ElementD = int32_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = int32_t;
  using ElementBias = int32_t;
  using ClusterTileShape = cute::Shape<_64,_256,Int<128 / sizeof(ElementA)>>;
  using ClusterShape = Shape<_1,_1,_1>;
  using AtomThrShape = decltype(shape_div(ClusterShape{}, Shape<_1,_1,_1>{}));

  using OutputCtaShape = decltype(shape_div(ClusterTileShape{}, ClusterShape{}));
  using MmaTileShape = decltype(shape_div(ClusterTileShape{}, AtomThrShape{}));

  using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized1Sm;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      OutputCtaShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, 16 / sizeof(ElementC),
      ElementD, LayoutC, 16 / sizeof(ElementD),
      EpilogueSchedule
    >::CollectiveOp;

  using MainloopSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized1SmSm100;
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 16 / sizeof(ElementA),
      ElementB, LayoutB, 16 / sizeof(ElementB),
      ElementAccumulator,
      MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopSchedule
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::ArrayProblemShape<Shape<int,int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = TestSmall<Gemm>(2, 0.5, CheckEquality::EXACT);
  EXPECT_TRUE(pass);
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////// 64x256x128 Cluster2x4x1 TMEM 2x2 ////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM100_Device_Gemm_s8t_s8n_s8n_tensorop_2cta_s32_ptr_array, 128x1024x128_2x4x1) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using ElementA = int8_t;
  using ElementB = int8_t;
  using ElementC = int8_t;
  using ElementD = int8_t;
  using ElementAccumulator = int32_t;
  using ElementCompute = float;
  using ElementBias = int8_t;
  using ClusterTileShape = Shape<_128,_1024,Int<128 / sizeof(ElementA)>>;
  using ClusterShape = Shape<_2,_4,_1>;
  using AtomThrShape = decltype(shape_div(ClusterShape{}, Shape<_2,_1,_1>{}));

  using OutputCtaShape = decltype(shape_div(ClusterTileShape{}, ClusterShape{}));
  using MmaTileShape = decltype(shape_div(ClusterTileShape{}, AtomThrShape{}));

  using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized2Sm;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      OutputCtaShape, ClusterShape,
      cutlass::epilogue::collective::EpilogueTileAuto,
      ElementAccumulator, ElementCompute,
      ElementC, LayoutC, 16 / sizeof(ElementC),
      ElementD, LayoutC, 16 / sizeof(ElementD),
      EpilogueSchedule
    >::CollectiveOp;

  using MainloopSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized2SmSm100;
  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 16 / sizeof(ElementA),
      ElementB, LayoutB, 16 / sizeof(ElementB),
      ElementAccumulator,
      MmaTileShape, ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      MainloopSchedule
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      cutlass::gemm::ArrayProblemShape<Shape<int,int,int,int>>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using namespace test::gemm::device;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  auto pass = TestSmall<Gemm>(2, 0.5, CheckEquality::EXACT);
  EXPECT_TRUE(pass);
}

#endif // #if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

