/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"

#include "../../common/cutlass_unit_test.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/gemm.h"

#include "testbed.h"

#if defined(CUTLASS_ARCH_MMA_SM80_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 16x64x64_16x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<16, 64, 64>,
    cutlass::gemm::GemmShape<16, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 16x128x64_16x32x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<16, 128, 64>,
    cutlass::gemm::GemmShape<16, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 16x256x64_16x64x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<16, 256, 64>,
    cutlass::gemm::GemmShape<16, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 64x16x64_16x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 16, 64>,
    cutlass::gemm::GemmShape<16, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 128x16x64_32x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 16, 64>,
    cutlass::gemm::GemmShape<32, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 256x16x64_64x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 16, 64>,
    cutlass::gemm::GemmShape<64, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x32x32_16x16x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 32, 32>,
    cutlass::gemm::GemmShape<16, 16, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x64x32_16x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 64, 32>,
    cutlass::gemm::GemmShape<16, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x128x32_16x64x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 128, 32>,
    cutlass::gemm::GemmShape<16, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x64x32_32x16x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 64, 32>,
    cutlass::gemm::GemmShape<32, 16, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x128x32_32x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 128, 32>,
    cutlass::gemm::GemmShape<32, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x256x32_32x64x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 256, 32>,
    cutlass::gemm::GemmShape<32, 64, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 64x32x32_32x16x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 32, 32>,
    cutlass::gemm::GemmShape<32, 16, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 128x32x32_64x16x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 32, 32>,
    cutlass::gemm::GemmShape<64, 16, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 64x32x32_16x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 32, 32>,
    cutlass::gemm::GemmShape<16, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 128x32x32_32x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 32, 32>,
    cutlass::gemm::GemmShape<32, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 256x32x32_64x32x32) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 32, 32>,
    cutlass::gemm::GemmShape<64, 32, 32>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x32x64_16x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x64x64_16x32x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 64, 64>,
    cutlass::gemm::GemmShape<16, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x128x64_16x64x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 128, 64>,
    cutlass::gemm::GemmShape<16, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x64x64_32x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 64, 64>,
    cutlass::gemm::GemmShape<32, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x128x64_32x32x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 128, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 32x256x64_32x64x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<32, 256, 64>,
    cutlass::gemm::GemmShape<32, 64, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 64x32x64_32x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 32, 64>,
    cutlass::gemm::GemmShape<32, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 128x32x64_64x16x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 32, 64>,
    cutlass::gemm::GemmShape<64, 16, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      64 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 64x32x64_16x32x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 32, 64>,
    cutlass::gemm::GemmShape<16, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 128x32x64_32x32x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 32, 64>,
    cutlass::gemm::GemmShape<32, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}

TEST(SM80_Device_Gemm_f16t_f16n_f16t_tensor_op_f32, 256x32x64_64x32x64) {

  using ElementOutput = cutlass::half_t;
  using ElementAccumulator = float;

  using Gemm = cutlass::gemm::device::Gemm<
    cutlass::half_t,
    cutlass::layout::RowMajor,
    cutlass::half_t,
    cutlass::layout::ColumnMajor,
    ElementOutput,
    cutlass::layout::RowMajor,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<256, 32, 64>,
    cutlass::gemm::GemmShape<64, 32, 64>,
    cutlass::gemm::GemmShape<16, 8, 16>,
    cutlass::epilogue::thread::LinearCombination<
      ElementOutput,
      128 / cutlass::sizeof_bits<ElementOutput>::value,
      ElementAccumulator,
      ElementAccumulator
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    3
  >;

  EXPECT_TRUE(test::gemm::device::TestAllGemm<Gemm>());
}
/////////////////////////////////////////////////////////////////////////////////////////////////

#endif
