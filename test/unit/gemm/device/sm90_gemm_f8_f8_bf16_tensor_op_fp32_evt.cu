#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief Tests for Sm90 f8_f8_bf16 with EVT epilogue 
    ScaledLinCombPerRowBiasEltAct and ScaledLinCombPerRowBiasEltActAmaxAux
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/sm70_epilogue_vectorized.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/epilogue/thread/linear_combination_bias_elementwise.h"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x_evt.hpp"
#include "sm90_evt_operations.hpp"


#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

using namespace cute;

// Z = scale_a * scale_b * alpha * acc + beta * scale_c * C + per-row bias
// if D is fp8 
//   D = scale_d * activation(Z)
// else
//   D = activation(Z)
TEST(SM90_Device_Gemm_e4m3t_e4m3n_bf16t_tensor_op_gmma_f32_epilogue, 64x128x128_ScaledLinCombPerRowBiasEltAct) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_64,_128,_128>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecialized;
  using FusionCallbacks = cutlass::epilogue::fusion::Sm90ScaledLinCombPerRowBiasEltAct<
    TileShape_MNK,                      // CtaTileShapeMNK
    cutlass::epilogue::thread::ReLu,    // ActivationFn
    cutlass::bfloat16_t,                // ElementOutput
    float,                              // ElementCompute
    cutlass::bfloat16_t                 // ElementBias
  >;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::bfloat16_t, LayoutC, 8,
      cutlass::bfloat16_t, LayoutC, 8,
      EpilogueSchedule,
      FusionCallbacks
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::float_e4m3_t, LayoutA, 16,
      cutlass::float_e4m3_t, LayoutB, 16,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  // Host reference
  using HostReference = test::gemm::device::HostScaledLinCombPerRowBiasEltAct<
    Gemm, cutlass::epilogue::thread::ReLu, cutlass::bfloat16_t
  >;
  bool passed = test::gemm::device::TestAllEVT<Gemm, HostReference>(true);
  EXPECT_TRUE(passed);
}

// Z = scale_a * scale_b * alpha * acc + scale_c * beta * C + per-row bias
// if D is fp8 
//   amax_d = max(abs(elements in activation(Z)))
//   D = scale_d * activation(Z)
// else
//   D = activation(Z)
// if Aux is fp8 
//   amax_aux = max(abs(elements in Z))
//   Aux = scale_aux * Z
// else
//   Aux = Z
TEST(SM90_Device_Gemm_e4m3t_e4m3n_bf16n_tensor_op_gmma_f32_epilogue, 64x128x128_4x1x1_ScaledLinCombPerRowBiasEltActAmaxAux) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_128>;
  using ClusterShape_MNK = Shape<_2,_4,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecialized;
  using EpilogueTileType = cutlass::epilogue::collective::EpilogueTileAuto;
  using EpilogueDescriptor = cutlass::epilogue::collective::detail::EpilogueDescriptor<
    TileShape_MNK, EpilogueTileType, cutlass::bfloat16_t, cutlass::bfloat16_t, EpilogueSchedule>;
  using AuxStoreDescriptor = cutlass::epilogue::collective::detail::AuxStoreDescriptor<
    EpilogueDescriptor, cutlass::layout::RowMajor, cutlass::bfloat16_t>;
    
  using FusionCallbacks = cutlass::epilogue::fusion::Sm90ScaledLinCombPerRowBiasEltActAmaxAux<
    TileShape_MNK,                               // CtaTileShapeMNK
    typename EpilogueDescriptor::EpilogueTile,   // EpilogueTile
    EpilogueDescriptor::StagesD,                 // StagesD
    typename AuxStoreDescriptor::Stride,         // StrideAux
    typename AuxStoreDescriptor::SmemLayoutAtom, // SmemLayoutAtom
    typename AuxStoreDescriptor::CopyOpR2S,      // CopyOpR2S
    cutlass::epilogue::thread::ReLu,             // ActivationFn
    cutlass::bfloat16_t,                         // ElementOutput
    float,                                       // ElementCompute
    cutlass::bfloat16_t,                         // ElementBias
    float                                        // ElementScalar
  >;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      EpilogueTileType,
      float, float,
      cutlass::bfloat16_t, LayoutC, 16,
      cutlass::bfloat16_t, LayoutC, 16,
      EpilogueSchedule,
      FusionCallbacks
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::float_e4m3_t, LayoutA, 16,
      cutlass::float_e4m3_t, LayoutB, 16,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  // Host reference
  using HostReference = test::gemm::device::HostScaledLinCombPerRowBiasEltActAmaxAux<
    Gemm, cutlass::epilogue::thread::ReLu, cutlass::bfloat16_t
  >;
  bool passed = test::gemm::device::TestAllEVT<Gemm, HostReference>(true);
  EXPECT_TRUE(passed);
}
#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
