/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x.hpp"

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

using namespace cute;

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32t_tf32n_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::RowMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32n_tf32n_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32t_tf32t_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::RowMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::RowMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::gemm::EpilogueTransposed
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32n_tf32t_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::RowMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_1,_1,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32t_tf32n_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::RowMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32n_tf32n_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32t_tf32t_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::RowMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::RowMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::gemm::EpilogueTransposed
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32n_tf32t_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::RowMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::KernelTmaWarpSpecialized
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////
////////////  CollectiveBuilder with KernelScheduleAuto  //////////////////////
///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32t_tf32n_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1_auto_schedule) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::RowMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32n_tf32n_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1_auto_schedule) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32t_tf32t_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1_auto_schedule) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::RowMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::RowMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::gemm::EpilogueTransposed
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

TEST(SM90_Device_Gemm_tf32n_tf32t_f32n_tensor_op_gmma_rs_ws_f32, 64x128x32_4x2x1_auto_schedule) {
  using ElementA = cutlass::tfloat32_t;
  using LayoutA = cutlass::layout::ColumnMajor;
  using ElementB = cutlass::tfloat32_t;
  using LayoutB = cutlass::layout::RowMajor;
  using ElementAccumulator = float;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_64,_128,_32>;
  using ClusterShape_MNK = Shape<_4,_2,_1>;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      ElementA, LayoutA, 4,
      ElementB, LayoutB, 4,
      ElementAccumulator,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAuto,
      cutlass::gemm::collective::KernelScheduleAuto
    >::CollectiveOp;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      float, LayoutC, 4,
      float, LayoutC, 4,
      cutlass::epilogue::collective::EpilogueScheduleAuto
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  EXPECT_TRUE(test::gemm::device::TestAll<Gemm>());
}

///////////////////////////////////////////////////////////////////////////////

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
