/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for device-wide GEMM interface with bias and elementwise epilogues.
*/

#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/sm70_epilogue_vectorized.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/epilogue/thread/linear_combination_bias_elementwise.h"

#include "../../common/cutlass_unit_test.h"

#include "gemm_testbed_3x.hpp"


#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

using namespace cute;

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_ReLU) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombEltAct<
      cutlass::epilogue::thread::ReLu, cutlass::half_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  test::gemm::device::Testbed3x<Gemm, cutlass::epilogue::thread::ReLu> testbed;
  bool passed = test::gemm::device::TestAll<Gemm>(1, 1, testbed);
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_ReLU_Legacy) {
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wdeprecated-declarations" // Suppress deprecation warnings
#ifdef _MSC_VER
#pragma warning( push )
#pragma warning( disable : 4996 )
#endif // _MSC_VER
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  static constexpr bool StoreT = true;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperativeBiasElementwise<
        cutlass::epilogue::thread::ReLu, cutlass::half_t, cutlass::plus, StoreT, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1, 1);
  EXPECT_TRUE(passed);
#ifdef _MSC_VER
#pragma warning( pop )
#endif // _MSC_VER
#pragma GCC diagnostic pop // Re-enable deprecation warnings
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_ReLU) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::epilogue::thread::ReLu, cutlass::half_t, float, cutlass::half_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1, 1);
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_GELU) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::epilogue::thread::GELU, cutlass::half_t, float, cutlass::half_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool check_relative_equality = true;
  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1, 1, check_relative_equality);
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_ReLU_NoStoreT) {
  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltAct<
      cutlass::epilogue::thread::ReLu, cutlass::half_t, float, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1, 1);
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_Negate) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::negate, cutlass::half_t, float, cutlass::half_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1, 1);
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32n_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_ReLU) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::ColumnMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::epilogue::thread::ReLu, cutlass::half_t, float, cutlass::half_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      cutlass::half_t, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1, 1);
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF32_ReLU_VoidC) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::epilogue::thread::ReLu, cutlass::half_t, float, cutlass::half_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      void, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>();
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasF16_ReLU_VoidC) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::epilogue::thread::ReLu, cutlass::half_t, float, cutlass::half_t, cutlass::half_t>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      void, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>();
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_BiasS8_ReLU_VoidC_U1Aux) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  // ReLU with uint1b_t aux will compute dReLU/dZ as the aux output, i.e. Aux(i) = (Z(i) >= 0) ? 1 : 0
  using FusionOperation = cutlass::epilogue::fusion::LinCombPerRowBiasEltActAux<
      LayoutC, cutlass::epilogue::thread::ReLU, cutlass::half_t, float, cutlass::uint1b_t, int8_t>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      void, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>();
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_dReLU_dBias_VoidC) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombDeEltActDePerRowBias<
      LayoutC, cutlass::epilogue::thread::dReLU, cutlass::half_t, float, cutlass::uint1b_t, float>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      void, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>();
  EXPECT_TRUE(passed);
}

TEST(SM90_Device_Gemm_f16t_f16n_f32t_tensor_op_gmma_f32_cooperative_epilogue, 256x128x64_2x2x1_dGELU_VoidC) {

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;
  using LayoutC = cutlass::layout::RowMajor;
  using TileShape_MNK = Shape<_256,_128,_64>;
  using ClusterShape_MNK = Shape<_2,_2,_1>;

  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using FusionOperation = cutlass::epilogue::fusion::LinCombDeEltAct<
      LayoutC, cutlass::epilogue::thread::dGELU, cutlass::half_t, float, cutlass::half_t>;

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::epilogue::collective::EpilogueTileAuto,
      float, float,
      void, LayoutC, 8,
      cutlass::half_t, LayoutC, 8,
      EpilogueSchedule,
      FusionOperation
    >::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm90, cutlass::arch::OpClassTensorOp,
      cutlass::half_t, LayoutA, 8,
      cutlass::half_t, LayoutB, 8,
      float,
      TileShape_MNK, ClusterShape_MNK,
      cutlass::gemm::collective::StageCountAutoCarveout<sizeof(typename CollectiveEpilogue::SharedStorage)>,
      cutlass::gemm::KernelTmaWarpSpecializedCooperative
    >::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
  >;

  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  bool passed = test::gemm::device::TestAllBiasElementwise<Gemm>(1.0, 0.0, /*check_relative_equality=*/true);
  EXPECT_TRUE(passed);
}

#endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
