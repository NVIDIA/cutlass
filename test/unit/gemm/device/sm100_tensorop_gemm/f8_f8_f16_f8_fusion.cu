#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/



#include <iostream>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cute/atom/mma_atom.hpp"

#include "cutlass/numeric_types.h"

#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"

#include "cutlass/epilogue/dispatch_policy.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"

#include "cutlass/epilogue/thread/activation.h"
#include "../../../common/cutlass_unit_test.h"

#include "../gemm_testbed_3x.hpp"

using namespace cute;

#if defined(CUTLASS_ARCH_MMA_SM100_SUPPORTED)

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Inference fprop fusions
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM100Only_Device_Gemm_e4m3t_e4m3n_f16t_e4m3t_tensor_op_f32, 128x128x128_1x2x1_1sm_bias_relu) {
  // Describe A and B tensors
  using ElementA = cutlass::float_e4m3_t;
  constexpr int AlignA = 16;
  using GmemLayoutA = cutlass::layout::RowMajor;
  constexpr int AlignB = 16;
  using ElementB = cutlass::float_e4m3_t;
  using GmemLayoutB = cutlass::layout::ColumnMajor;

  // Describe C and D tensors
  using ElementC = cutlass::half_t;
  constexpr int AlignC = 8;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = cutlass::float_e4m3_t;
  constexpr int AlignD = 16;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_128,_64>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_1,_2,_1>;

  // Epilogue fusion operation
  // Z = alpha * scale_a * scale_b * acc + beta * scale_c * C + per-row bias
  // D = scale_d * ReLU(Z)
  using ElementBias = cutlass::half_t;
  using FusionOperation = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltAct<
      cutlass::epilogue::thread::ReLU,
      ElementD,
      ElementCompute,
      ElementBias,
      ElementC>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto,                  // Epilogue schedule policy
      FusionOperation                                                       // Epilogue fusion operation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto    // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

TEST(SM100Only_Device_Gemm_e4m3t_e4m3n_f16t_f32t_tensor_op_f32, 128x128x128_1x2x1_1sm_bias_relu) {
  // Describe A and B tensors
  using ElementA = cutlass::float_e4m3_t;
  constexpr int AlignA = 16;
  using GmemLayoutA = cutlass::layout::RowMajor;
  constexpr int AlignB = 16;
  using ElementB = cutlass::float_e4m3_t;
  using GmemLayoutB = cutlass::layout::ColumnMajor;

  // Describe C and D tensors
  using ElementC = cutlass::half_t;
  constexpr int AlignC = 8;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_128,_64>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_1,_2,_1>;

  // Epilogue fusion operation
  // Z = alpha * scale_a * scale_b * acc + beta * scale_c * C + per-row bias
  // D = ReLU(Z)
  // scale_d is only applied if D is an fp8 type
  using ElementBias = float;
  using FusionOperation = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltAct<
      cutlass::epilogue::thread::ReLU,
      ElementD,
      ElementCompute,
      ElementBias,
      ElementC>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto,                  // Epilogue schedule policy
      FusionOperation                                                       // Epilogue fusion operation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto    // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Training fprop fusions
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(SM100Only_Device_Gemm_e4m3t_e4m3n_f16t_e4m3t_tensor_op_f32, 128x128x128_1x2x1_1sm_bias_relu_amax_aux) {
  // Describe A and B tensors
  using ElementA = cutlass::float_e4m3_t;
  constexpr int AlignA = 16;
  using GmemLayoutA = cutlass::layout::RowMajor;
  constexpr int AlignB = 16;
  using ElementB = cutlass::float_e4m3_t;
  using GmemLayoutB = cutlass::layout::ColumnMajor;

  // Describe C and D tensors
  using ElementC = cutlass::half_t;
  constexpr int AlignC = 8;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = cutlass::float_e4m3_t;
  constexpr int AlignD = 16;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_128,_64>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_1,_2,_1>;

  // Epilogue fusion operation
  // Z = alpha * scale_a * scale_b * acc + beta * scale_c * C + per-row bias
  // D = scale_d * ReLU(Z)
  // Amax_D = max absolute value of ReLU(Z)
  // Aux = Z
  // scale_d and Amax_D are only computed if D is fp8
  using ElementBias = cutlass::half_t;
  using ElementAmax = float;
  using ElementAux = float;
  using GmemLayoutAux = GmemLayoutC;
  using FusionOperation = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltActAmaxAux<
      GmemLayoutAux,
      cutlass::epilogue::thread::ReLU,
      ElementD,
      ElementCompute,
      ElementAux,
      ElementAmax,
      ElementBias,
      ElementC>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto,                  // Epilogue schedule policy
      FusionOperation                                                       // Epilogue fusion operation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto    // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}

TEST(SM100Only_Device_Gemm_e4m3t_e4m3n_f16t_f32t_tensor_op_f32, 128x128x128_1x2x1_1sm_bias_relu_amax_aux) {
  // Describe A and B tensors
  using ElementA = cutlass::float_e4m3_t;
  constexpr int AlignA = 16;
  using GmemLayoutA = cutlass::layout::RowMajor;
  constexpr int AlignB = 16;
  using ElementB = cutlass::float_e4m3_t;
  using GmemLayoutB = cutlass::layout::ColumnMajor;

  // Describe C and D tensors
  using ElementC = cutlass::half_t;
  constexpr int AlignC = 8;
  using GmemLayoutC = cutlass::layout::RowMajor;
  using ElementD = float;
  constexpr int AlignD = 4;
  using GmemLayoutD = cutlass::layout::RowMajor;

  // Mma's accumulator type
  using ElementAccumulator = float;
  // Epilogue computation's precision type
  using ElementCompute = float;
  
  // Tile and cluster shapes
  // Collective MMA takes tile shape of the MMA operation as input
  using MmaTileShape_MNK = Shape<_128,_128,_64>;
  // Cluster size for multicast
  using ClusterShape_MNK = Shape<_1,_2,_1>;

  // Epilogue fusion operation
  // Z = alpha * scale_a * scale_b * acc + beta * scale_c * C + per-row bias
  // D = ReLU(Z)
  // Aux = scale_aux * Z
  // Amax_Aux = max absolute value of Z
  // scale_aux and Amax_Aux are only computed if Aux is fp8
  using ElementBias = float;
  using ElementAmax = float;
  using ElementAux = cutlass::float_e4m3_t;
  using GmemLayoutAux = GmemLayoutC;
  using FusionOperation = cutlass::epilogue::fusion::ScaledLinCombPerRowBiasEltActAmaxAux<
      GmemLayoutAux,
      cutlass::epilogue::thread::ReLU,
      ElementD,
      ElementCompute,
      ElementAux,
      ElementAmax,
      ElementBias,
      ElementC>;

  //
  // Construct CollectiveEpilogue
  //

  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      cutlass::epilogue::collective::EpilogueTileAuto,                      // Epilogue subtile shape. Auto will find a suitable tile shape
      ElementAccumulator, ElementCompute,                                   // Mma instr's accumulator type and compute precision for epilogue
      ElementC, GmemLayoutC, AlignC,                                        // C tensor description
      ElementD, GmemLayoutD, AlignD,                                        // D tensor description
      cutlass::epilogue::collective::EpilogueScheduleAuto,                  // Epilogue schedule policy
      FusionOperation                                                       // Epilogue fusion operation
    >::CollectiveOp;

  //
  // Construct CollectiveMainloop
  //

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      cutlass::arch::Sm100, cutlass::arch::OpClassTensorOp,                 // Arch and Tensorop spec
      ElementA, GmemLayoutA, AlignA,                                        // A tensor elem type, layout and alignment requirement
      ElementB, GmemLayoutB, AlignB,                                        // B tensor elem type, layout and alignment requirement
      ElementAccumulator,                                                   // Mma instruction accumulator type
      MmaTileShape_MNK, ClusterShape_MNK,                                   // Mma instruction tile shape, cluster shape
      // Epilogue's SMEM usage that needs to be subtracted from overall SMEM capacity 
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
      cutlass::gemm::collective::KernelScheduleAuto    // Kernel schedule policy. Auto or using targeted scheduling policy
    >::CollectiveOp;

  // Create Gemm Kernel using CollectiveEpilogue and CollectiveMainloop created by the builders
  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int,int,int,int>,
      CollectiveMainloop,
      CollectiveEpilogue
    >;
  
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;
  // Run tests
  auto pass = test::gemm::device::TestAll<Gemm>();
  // Check results
  EXPECT_TRUE(pass);
}


#endif
