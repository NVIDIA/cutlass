#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for conversion operators.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/numeric_conversion.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/util/host_tensor.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace core {
namespace kernel {

/// Simple conversion function
template <typename Destination, typename Source, int Count>
__global__ void convert(
  cutlass::Array<Destination, Count> *destination,
  cutlass::Array<Source, Count> const *source) {

  cutlass::FastNumericArrayConverter<Destination, Source, Count> convert;

  *destination = convert(*source);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Destination, typename Source, int Count>
void run_test_integer_range_limited() {
  const int kN = Count;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<Destination, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<Source, cutlass::layout::RowMajor> source({1, kN});

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = Source(i % 4);
  }

  source.sync_device();

  convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination.host_data()[i]) == float(source.host_data()[i]));
  }
}


template <typename Destination, typename Source, int Count>
void run_test_integer_range_all() {
  const int kN = Count;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<Destination, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<Source, cutlass::layout::RowMajor> source({1, kN});

  int const kIntSourceMin = std::numeric_limits<Source>::min();
  int const kIntSourceMax = std::numeric_limits<Source>::max();
  int const kIntRange = kIntSourceMax - kIntSourceMin + 1;

  for (int i = 0; i < kN; ++i) {
    source.host_data()[i] = Source(kIntSourceMin + (i % kIntRange));

  }

  source.sync_device();

  convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  // Verify conversion
  bool passed = true;
  for (int i = 0; i < kN; ++i) {
    if(!(float(destination.host_data()[i]) == float(source.host_data()[i]))) {
      passed = false;
      break;
    }
  }
  EXPECT_TRUE(passed) << " FastNumericArrayConverter failed";

   // Print out results for the failed conversion.
   if (!passed) {
    for (int i = 0; i < kN; ++i) {
        std::cout << "source(" << float(source.host_data()[i]) << ") -> "
                  << "destination ("<< float(destination.host_data()[i]) << ")" << std::endl;
    }
   }
   std::flush(std::cout);
}

} // namespace kernel
} // namespace core
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////
TEST(FastNumericConversion, s32_to_f32) {
  int const kN = 4;
  using Source = int;
  using Destination = float;
  test::core::kernel::run_test_integer_range_limited<Destination, Source, kN>();
}

TEST(FastNumericConversion, s8_to_f16_array) {
  int const kN = 256;
  using Source = int8_t;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test_integer_range_all<Destination, Source, kN>();
}

TEST(FastNumericConversion, u8_to_f16_array) {
  int const kN = 256;
  using Source = uint8_t;
  using Destination = cutlass::half_t;
  test::core::kernel::run_test_integer_range_all<Destination, Source, kN>();
}

TEST(FastNumericConversion, u8_to_bf16_array) {
  int const kN = 256;
  using Source = uint8_t;
  using Destination = cutlass::bfloat16_t;
  test::core::kernel::run_test_integer_range_all<Destination, Source, kN>();
}

TEST(FastNumericConversion, s8_to_bf16_array) {
  int const kN = 256;
  using Source = int8_t;
  using Destination = cutlass::bfloat16_t;
  test::core::kernel::run_test_integer_range_all<Destination, Source, kN>();
}
