#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Unit tests for conversion operators.
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/numeric_conversion.h"

#include "cutlass/layout/matrix.h"
#include "cutlass/util/host_tensor.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {
namespace core {
namespace kernel {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Simple conversion function
template <typename Destination, typename Source, int Count>
__global__ void convert(
  cutlass::Array<Destination, Count> *destination,
  cutlass::Array<Source, Count> const *source) {

  cutlass::NumericArrayConverter<Destination, Source, Count> convert;

  *destination = convert(*source);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Destination, typename Source, int Count>
void run_test(const char dest_name[], const char source_name[], const int range = 4, const int offset = 0) {
  const int kN = Count;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<Destination, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<Source, cutlass::layout::RowMajor> source({1, kN});
  auto source_ref = source.host_ref();
  auto destination_ref = destination.host_ref();

  for (int i = 0; i < kN; ++i) {
    source_ref.at({0, i}) = Source(i % range + offset);
  }

  source.sync_device();

  convert<Destination, Source, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    EXPECT_TRUE(float(destination_ref.at({0, i})) == float(source_ref.at({0, i})))
      << "Destination type: " << dest_name << " "<< float(destination_ref.at({0, i}))
      << ", Source type: " << source_name << " " << float(source_ref.at({0, i}))
      << ", Count: " << Count;
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Destination, typename Source, typename ScaleFactor, int Count>
__global__ void convert_with_scale_factor(
  cutlass::Array<Destination, Count> *destination,
  cutlass::Array<Source, Count> const *source,
  cutlass::Array<ScaleFactor, Count> const *scale_factor) {

  cutlass::NumericArrayConverter<Destination, Source, Count> convert;

  *destination = convert(*source, *scale_factor);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Destination, typename Source, typename ScaleFactor,  int Count, int Range = 4>
void run_test_with_scalefactor(const char dest_name[], const char source_name[], const char scale_factor_name[]) {
  const int kN = Count;

  dim3 grid(1, 1);
  dim3 block(1, 1);

  cutlass::HostTensor<Destination, cutlass::layout::RowMajor> destination({1, kN});
  cutlass::HostTensor<Source, cutlass::layout::RowMajor> source({1, kN});
  cutlass::HostTensor<ScaleFactor, cutlass::layout::RowMajor> scale_factor({1, kN});
  auto source_ref = source.host_ref();
  auto destination_ref = destination.host_ref();
  auto scale_factor_ref = scale_factor.host_ref();


  for (int i = 0; i < kN; ++i) {
    source_ref.at({0, i}) = Source(i % Range);
  }

  for (int i = 0; i < kN; ++i) {
    scale_factor_ref.at({0, i}) = ScaleFactor(1 + i % 8);
  }

  source.sync_device();
  scale_factor.sync_device();

  convert_with_scale_factor<Destination, Source, ScaleFactor, kN><<< grid, block >>>(
    reinterpret_cast<cutlass::Array<Destination, kN> *>(destination.device_data()),
    reinterpret_cast<cutlass::Array<Source, kN> const *>(source.device_data()),
    reinterpret_cast<cutlass::Array<ScaleFactor, kN> const *>(scale_factor.device_data())
  );

  destination.sync_host();

  for (int i = 0; i < kN; ++i) {
    float ref = float(source_ref.at({0, i})) / float(scale_factor_ref.at({0, i}));
    EXPECT_TRUE(float(destination_ref.at({0, i})) == ref)
      << "Destination type: " << dest_name << " "<< float(destination_ref.at({0, i}))
      << ", Source type: " << source_name << " " << float(source_ref.at({0, i}))
      << ", Count: " << Count;
  }
}

} // namespace kernel
} // namespace core
} // namespace test

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32_to_f16_rn) {
  constexpr int kN = 1;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f32x2_to_f16x2_rn) {
  constexpr int kN = 2;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f32x8_to_f16x8_rn) {
  constexpr int kN = 8;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f16_to_f32_rn) {  
  int const kN = 1;
  using Source = cutlass::half_t;
  const char source_name[] = "half_t";
  using Destination = float;
  const char dest_name[] = "float";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f16x8_to_f32x8_rn) {
  int const kN = 8;
  using Source = cutlass::half_t;
  const char source_name[] = "half_t";
  using Destination = float;
  const char dest_name[] = "float";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32_to_fe4m3_rn) {
  int const kN = 1;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f32_to_fe4m3_rn_array) {
  int const kN = 27;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f32_to_fe5m2_rn) {
  int const kN = 1;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f32_to_fe5m2_rn_array) {
  int const kN = 27;
  using Source = float;
  const char source_name[] = "float";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f16_to_fe4m3_rn) {
  int const kN = 1;
  using Source = cutlass::half_t;
  const char source_name[] = "half_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f16_to_fe4m3_rn_array) {
  int const kN = 27;
  using Source = cutlass::half_t;
  const char source_name[] = "half_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f16_to_fe5m2_rn) {
  int const kN = 1;
  using Source = cutlass::half_t;
  const char source_name[] = "half_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, f16_to_fe5m2_rn_array) {
  int const kN = 27;
  using Source = cutlass::half_t;
  const char source_name[] = "half_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, bf16_to_fe4m3_rn) {
  int const kN = 1;
  using Source = cutlass::bfloat16_t;
  const char source_name[] = "bfloat16_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, bf16_to_fe4m3_rn_array) {
  int const kN = 27;
  using Source = cutlass::bfloat16_t;
  const char source_name[] = "bfloat16_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, bf16_to_fe5m2_rn) {
  int const kN = 1;
  using Source = cutlass::bfloat16_t;
  const char source_name[] = "bfloat16_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, bf16_to_fe5m2_rn_array) {
  int const kN = 27;
  using Source = cutlass::bfloat16_t;
  const char source_name[] = "bfloat16_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, fe4m3_to_fe5m2_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_fe5m2_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_fe4m3_rn) {
  int const kN = 1;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_fe4m3_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_f32_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = float;
  const char dest_name[] = "float";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(NumericConversion, f32x8_to_s8x8_rn) {

  int const kN = 8;
  using Source = float;
  const char source_name[] = "float";
  using Destination = int8_t;
  const char dest_name[] = "int8_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_f32_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = float;
  const char dest_name[] = "float";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_f32_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = float;
  const char dest_name[] = "float";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_f16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_f16_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_f16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_f16_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = cutlass::half_t;
  const char dest_name[] = "half_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_bf16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = cutlass::bfloat16_t;
  const char dest_name[] = "bfloat16_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe4m3_to_bf16_array) {
  int const kN = 27;
  using Source = cutlass::float_e4m3_t;
  const char source_name[] = "float_e4m3_t";
  using Destination = cutlass::bfloat16_t;
  const char dest_name[] = "bfloat16_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_bf16_rn) {
  int const kN = 1;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = cutlass::bfloat16_t;
  const char dest_name[] = "bfloat16_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_to_bf16_array) {
  int const kN = 27;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = cutlass::bfloat16_t;
  const char dest_name[] = "bfloat16_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

// These are included as regression tests for a special case when N = 4.
TEST(NumericConversion, int4b_t_to_fe5m2_t_array_4) {
  int const kN = 4;
  using Source = cutlass::int4b_t;
  const char source_name[] = "int4b_t";
  using Destination = cutlass::float_e5m2_t;
  const char dest_name[] = "float_e5m2_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, int_to_fe4m3_t_array_4) {
  int const kN = 4;
  using Source = int;
  const char source_name[] = "int";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, int2b_t_to_fe4m3_t_array_4) {
  int const kN = 4;
  using Source = cutlass::int2b_t;
  const char source_name[] = "int2b_t";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, fe5m2_t_to_double_array_4) {
  int const kN = 4;
  using Source = cutlass::float_e5m2_t;
  const char source_name[] = "float_e5m2_t";
  using Destination = double;
  const char dest_name[] = "double";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

TEST(NumericConversion, int_to_fe4m3_t_array_32) {
  int const kN = 32;
  using Source = int;
  const char source_name[] = "int";
  using Destination = cutlass::float_e4m3_t;
  const char dest_name[] = "float_e4m3_t";
  test::core::kernel::run_test<Destination, Source, kN>(dest_name, source_name);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
struct GetName {
  static constexpr char name[] = "UNSUPPORTED";
};

template <>
struct GetName<cutlass::int4b_t> {
  static constexpr char name[] = "int4b_t";
};

template <>
struct GetName<uint8_t> {
  static constexpr char name[] = "uint8_t";
};

template <>
struct GetName<int8_t> {
  static constexpr char name[] = "int8_t";
};

template <>
struct GetName<cutlass::float_e4m3_t> {
  static constexpr char name[] = "float_e4m3_t";
};

template <>
struct GetName<cutlass::half_t> {
  static constexpr char name[] = "half_t";
};

template <>
struct GetName<cutlass::bfloat16_t> {
  static constexpr char name[] = "bfloat16_t";
};

template <>
struct GetName<float> {
  static constexpr char name[] = "float";
};

template <typename Result_, typename Source_>
struct ResultSourcePair {
  using Result = Result_;
  using Source = Source_;
};

template <typename ResultSourcePair>
class VectorArrayConverterTest : public testing::Test {
 public:
  using Result = typename ResultSourcePair::Result;
  using Source = typename ResultSourcePair::Source;
  
  template <int N>
  static void emit_test() { 
    const int range = 1 << cutlass::sizeof_bits<Source>::value;
    const int offset = cutlass::platform::numeric_limits<Source>::lowest();
    test::core::kernel::run_test<Result, Source, N>(GetName<Result>::name, GetName<Source>::name, range, offset);
  }
};

using VectorConvertTypes = ::testing::Types<
  ResultSourcePair<float, int8_t>,
  ResultSourcePair<float, uint8_t>,

  ResultSourcePair<cutlass::half_t, int8_t>,
  ResultSourcePair<cutlass::half_t, uint8_t>,

  ResultSourcePair<cutlass::bfloat16_t, uint8_t>,
  ResultSourcePair<cutlass::bfloat16_t, int8_t>,

  ResultSourcePair<cutlass::float_e4m3_t, cutlass::int4b_t>,
  ResultSourcePair<cutlass::half_t, cutlass::int4b_t>,
  ResultSourcePair<cutlass::bfloat16_t, cutlass::int4b_t>,
  ResultSourcePair<float, cutlass::int4b_t>
>;

TYPED_TEST_SUITE(VectorArrayConverterTest, VectorConvertTypes);

TYPED_TEST(VectorArrayConverterTest, array_1) {
  TestFixture::template emit_test<1>();
}

TYPED_TEST(VectorArrayConverterTest, array_2) {
  TestFixture::template emit_test<2>();
}

TYPED_TEST(VectorArrayConverterTest, array_3) {
  TestFixture::template emit_test<3>();
}

TYPED_TEST(VectorArrayConverterTest, array_4) {
  TestFixture::template emit_test<4>();
}

TYPED_TEST(VectorArrayConverterTest, array_5) {
  TestFixture::template emit_test<5>();
}

TYPED_TEST(VectorArrayConverterTest, array_8) {
  TestFixture::template emit_test<8>();
}

TYPED_TEST(VectorArrayConverterTest, array_10) {
  // N > 8 and N is not a multiple of 4
  TestFixture::template emit_test<10>();
}

TYPED_TEST(VectorArrayConverterTest, array_12) {
  // N > 8 and N is a multiple of 4
  TestFixture::template emit_test<12>();
}

TYPED_TEST(VectorArrayConverterTest, array_16) {
  // N > 8 and N is a multiple of 8
  TestFixture::template emit_test<16>();
}

TYPED_TEST(VectorArrayConverterTest, array_17) {
  // N > 8 and N is not a multiple of 8
  TestFixture::template emit_test<17>();
}

TYPED_TEST(VectorArrayConverterTest, array_27) {
  // Test entire conversion range with residue (for int4)
  TestFixture::template emit_test<27>();
}

TYPED_TEST(VectorArrayConverterTest, array_31) {
  // Force use of converters for 16, 8, 4, 2 and scalar 
  // if max width is 16
  TestFixture::template emit_test<31>();
}

TYPED_TEST(VectorArrayConverterTest, array_63) {
  // Force use of converters for 32, 16, 8, 4, 2 and scalar 
  // if max width is 32
  TestFixture::template emit_test<63>();
}

TYPED_TEST(VectorArrayConverterTest, array_256) {
  // Test entire conversion range (for int8)
  TestFixture::template emit_test<256>();
}

TYPED_TEST(VectorArrayConverterTest, array_259) {
  // Force use of 4, 2 and scalar converter (if max width is 4)
  TestFixture::template emit_test<259>();
}

TYPED_TEST(VectorArrayConverterTest, array_263) {
  // Force use of 8, 4, 2 and scalar converter (if max width is 8)
  TestFixture::template emit_test<263>();
}

/////////////////////////////////////////////////////////////////////////////////////////////////
