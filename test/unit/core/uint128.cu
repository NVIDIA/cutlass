#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief Tests for basic uint128 functionality
*/

#include "../common/cutlass_unit_test.h"

#include "cutlass/array.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/numeric_types.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/host_tensor.h"


/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Host
//
/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(uint128_t, host_arithmetic) {
  using T = cutlass::uint128_t;

  // only low 64bit
  for (uint64_t i = 0; i < 1024; ++i) {
    for (uint64_t j = 0; j < 1024; ++j) {
      T x = i;
      T y = j;

      EXPECT_TRUE(static_cast<uint64_t>(x + y) == (i + j));
    }
  }

  // carry overflow for low uint64_t 
  {
    for (uint64_t i = 0; i < 1024; ++i) {
      T x = static_cast<uint64_t>(0xFFFFFFFFFFFFFFFF);
      T y = i + 1;

      T z = x + y;

      EXPECT_EQ(z.hilo_.hi, static_cast<uint64_t>(0x1));
      EXPECT_EQ(z.hilo_.lo, i);
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////
//
// Device
//
/////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void uint128_add_operator(cutlass::uint128_t *output, cutlass::uint128_t const *input, cutlass::uint128_t base, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) {
    output[tid] = input[tid] + base;
  }
}

TEST(uint128_t, device_arithmetic) {
  using T = cutlass::uint128_t;

  int const N = 1024;

  cutlass::HostTensor<T, cutlass::layout::RowMajor> input({N, 1});
  cutlass::HostTensor<T, cutlass::layout::RowMajor> sum({N, 1});

  for (int i = 0; i < N; ++i) {
    input.at({i, 0}) = static_cast<uint64_t>(i + 1);
  }

  T b = static_cast<uint64_t>(0xFFFFFFFFFFFFFFFF);

  input.sync_device();

  uint128_add_operator<<< dim3(1,1), dim3(N, 1) >>>(sum.device_data(), input.device_data(), b, N);

  ASSERT_EQ(hipGetLastError(), hipSuccess) << "Kernel launch error.";

  sum.sync_host();

  for (int i = 0; i < N; ++i) {
    T got = sum.at({i, 0});
    uint64_t expected_hi = static_cast<uint64_t>(0x1);
    uint64_t expected_lo = static_cast<uint64_t>(i);

    EXPECT_EQ(got.hilo_.hi, expected_hi);
    EXPECT_EQ(got.hilo_.lo, expected_lo);
  }
}
