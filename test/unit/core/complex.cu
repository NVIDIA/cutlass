#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    \brief CUTLASS host-device template for complex numbers supporting all CUTLASS numeric types.
*/

#include <complex>
#include <cuda/std/complex>

#include "../common/cutlass_unit_test.h"

#include "cutlass/complex.h"
#include "cutlass/constants.h"
#include "cutlass/numeric_conversion.h"
#include "cutlass/tfloat32.h"
#include <type_traits>

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, f64_to_f32_conversion) {

  cutlass::complex<double> source = {1.5, -1.25};

  cutlass::complex<float> dest = cutlass::complex<float>(source); // explicit conversion

  EXPECT_TRUE(source.real() == 1.5 && source.imag() == -1.25 &&
    dest.real() == 1.5f && dest.imag() == -1.25f);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, f32_to_f64_conversion) {

  cutlass::complex<float> source = {-1.5f, 1.25f};

  cutlass::complex<double> dest = source;  // implicit conversion

  EXPECT_TRUE(source.real() == -1.5f && source.imag() == 1.25f &&
    dest.real() == -1.5 && dest.imag() == 1.25);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, s32_to_f64_conversion) {

  cutlass::complex<int> source = {-2, 1};

  cutlass::complex<double> dest = source;  // implicit conversion

  EXPECT_TRUE(source.real() == -2 && source.imag() == 1 &&
    dest.real() == -2 && dest.imag() == 1);
}


/////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, f16_to_f32_conversion) {

  cutlass::complex<cutlass::half_t> source = {1.5_hf, -1.25_hf};

  cutlass::complex<float> dest = cutlass::complex<float>(source); // explicit conversion

  EXPECT_TRUE(source.real() == 1.5_hf && source.imag() == -1.25_hf &&
    dest.real() == 1.5f && dest.imag() == -1.25f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, exp_f32) {
  cutlass::complex<float> Z[] = {
    {1, 1},
    {2   ,  cutlass::constants::pi<float>()/2.0f   },
    {0.5f,  cutlass::constants::pi<float>()        },
    {0.25f,  cutlass::constants::pi<float>()*3/4.0f },
    {0, 0},
  };

  cutlass::complex<double> Expected[] = {
    {1.4686939399158851, 2.2873552871788423},
    {4.524491950137825e-16, 7.38905609893065},
    {-1.6487212707001282, 2.019101226849069e-16},
    {-0.9079430793557842, 0.9079430793557843},
    {1, 0}
  };

  double tolerance = 0.00001;

  for (int i = 0; cutlass::real(Z[i]) != 0.0f; ++i) {
    double e_r = cutlass::real(Expected[i]);
    double e_i = cutlass::real(Expected[i]);

    cutlass::complex<float> got = cutlass::exp(Z[i]);
    float g_r = cutlass::real(got);
    float g_i = cutlass::real(got);

    EXPECT_TRUE(
      std::abs(g_r - e_r) < tolerance && std::abs(g_i - e_i) < tolerance
    ) << "Expected(" << Expected[i] << "), Got(" << got << ")";
  }
}

TEST(complex, absolute_value_real_and_imag) {
  {
    cutlass::complex z_d{3.0, 4.0};

    auto abs_d = cutlass::abs(z_d);
    static_assert(std::is_same_v<decltype(abs_d), double>);
    EXPECT_EQ(abs_d, 5.0);

    auto real_d = cutlass::real(z_d);
    static_assert(std::is_same_v<decltype(real_d), double>);
    EXPECT_EQ(real_d, 3.0);

    auto imag_d = cutlass::imag(z_d);
    static_assert(std::is_same_v<decltype(imag_d), double>);
    EXPECT_EQ(imag_d, 4.0);
  }

  {
    cutlass::complex z_f{3.0f, 4.0f};

    auto abs_f = cutlass::abs(z_f);
    static_assert(std::is_same_v<decltype(abs_f), float>);
    EXPECT_EQ(abs_f, 5.0f);

    auto real_f = cutlass::real(z_f);
    static_assert(std::is_same_v<decltype(real_f), float>);
    EXPECT_EQ(real_f, 3.0f);

    auto imag_f = cutlass::imag(z_f);
    static_assert(std::is_same_v<decltype(imag_f), float>);
    EXPECT_EQ(imag_f, 4.0f);
  }

  {
    cutlass::complex z_tf32{cutlass::tfloat32_t{3.0f}, cutlass::tfloat32_t{4.0f}};
    auto abs_tf32 = cutlass::abs(z_tf32);
    static_assert(std::is_same_v<decltype(abs_tf32), cutlass::tfloat32_t>);
    EXPECT_EQ(abs_tf32, cutlass::tfloat32_t{5.0f});

    auto real_tf32 = cutlass::real(z_tf32);
    static_assert(std::is_same_v<decltype(real_tf32), cutlass::tfloat32_t>);
    EXPECT_EQ(real_tf32, cutlass::tfloat32_t{3.0f});

    auto imag_tf32 = cutlass::imag(z_tf32);
    static_assert(std::is_same_v<decltype(imag_tf32), cutlass::tfloat32_t>);
    EXPECT_EQ(imag_tf32, cutlass::tfloat32_t{4.0f});
  }

  {
    cutlass::complex z_i{3, 4};

    // sqrt(int) isn't a valid overload, so cutlass::abs isn't tested.
    auto real_i = cutlass::real(z_i);
    static_assert(std::is_same_v<decltype(real_i), int>);
    EXPECT_EQ(real_i, 3);

    auto imag_i = cutlass::imag(z_i);
    static_assert(std::is_same_v<decltype(imag_i), int>);
    EXPECT_EQ(imag_i, 4);
  }

  {
    double x_d{3.0};

    auto real_d = cutlass::real(x_d);
    static_assert(std::is_same_v<decltype(real_d), double>);
    EXPECT_EQ(real_d, 3.0);

    auto imag_d = cutlass::imag(x_d);
    static_assert(std::is_same_v<decltype(imag_d), double>);
    EXPECT_EQ(imag_d, 0.0);
  }

  {
    float x_f{3.0f};

    auto real_f = cutlass::real(x_f);
    static_assert(std::is_same_v<decltype(real_f), float>);
    EXPECT_EQ(real_f, 3.0f);

    auto imag_f = cutlass::imag(x_f);
    static_assert(std::is_same_v<decltype(imag_f), float>);
    EXPECT_EQ(imag_f, 0.0f);
  }

  {
    cutlass::tfloat32_t x_tf32{3.0f};

    auto real_tf32 = cutlass::real(x_tf32);
    static_assert(std::is_same_v<decltype(real_tf32), cutlass::tfloat32_t>);
    EXPECT_EQ(real_tf32, cutlass::tfloat32_t{3.0f});

    auto imag_tf32 = cutlass::imag(x_tf32);
    static_assert(std::is_same_v<decltype(imag_tf32), cutlass::tfloat32_t>);
    EXPECT_EQ(imag_tf32, cutlass::tfloat32_t{0.0f});
  }

  {
    int x_i{3};

    auto real_i = cutlass::real(x_i);
    static_assert(std::is_same_v<decltype(real_i), int>);
    EXPECT_EQ(real_i, 3);

    auto imag_i = cutlass::imag(x_i);
    static_assert(std::is_same_v<decltype(imag_i), int>);
    EXPECT_EQ(imag_i, 0);
  }
}

// FakeReal and FakeComplex test whether cutlass::real and
// cutlass::imag correctly handle user-defined non-complex
// and complex number types.
namespace test {

// These classes have no conversions to or from arithmetic types, so
// that the test can ensure that the implementation does not silently
// convert to, say, float or int.
class FakeReal {
public:
  // cutlass::imag must be able to value-construct its noncomplex input.
  FakeReal() = default;

  static CUTLASS_HOST_DEVICE FakeReal make_FakeReal(int val) {
    return FakeReal{val};
  }

  friend CUTLASS_HOST_DEVICE bool operator==(FakeReal lhs, FakeReal rhs) {
    return lhs.value_ == rhs.value_;
  }

  friend CUTLASS_HOST_DEVICE FakeReal operator-(FakeReal const& x) {
    return make_FakeReal(-x.value_);
  }

private:
  CUTLASS_HOST_DEVICE FakeReal(int val) : value_(val) {}
  int value_ = 0;
};

class FakeComplex {
public:
  static CUTLASS_HOST_DEVICE FakeComplex
  make_FakeComplex(FakeReal re, FakeReal im) {
    return FakeComplex{re, im};
  }

  // Existence of member functions real and imag tell
  // CUTLASS that FakeComplex is a complex number type.
  CUTLASS_HOST_DEVICE FakeReal real() const { return real_; }
  CUTLASS_HOST_DEVICE FakeReal imag() const { return imag_; }

  friend CUTLASS_HOST_DEVICE bool operator==(FakeComplex lhs, FakeComplex rhs) {
    return lhs.real_ == rhs.real_ && lhs.imag_ == rhs.imag_;
  }

private:
  CUTLASS_HOST_DEVICE FakeComplex(FakeReal re, FakeReal im)
    : real_(re), imag_(im)
  {}

  FakeReal real_{};
  FakeReal imag_{};
};

CUTLASS_HOST_DEVICE FakeComplex conj(FakeComplex const& z) {
  return FakeComplex::make_FakeComplex(z.real(), -z.imag());
}

// Variant of FakeComplex that has a hidden friend conj instead of a
// nonmember conj defined outside the class.
class FakeComplexWithHiddenFriendConj {
public:
  static CUTLASS_HOST_DEVICE FakeComplexWithHiddenFriendConj
  make_FakeComplexWithHiddenFriendConj(FakeReal re, FakeReal im) {
    return FakeComplexWithHiddenFriendConj{re, im};
  }

  CUTLASS_HOST_DEVICE FakeReal real() const { return real_; }
  CUTLASS_HOST_DEVICE FakeReal imag() const { return imag_; }

  friend CUTLASS_HOST_DEVICE bool
  operator==(FakeComplexWithHiddenFriendConj lhs,
    FakeComplexWithHiddenFriendConj rhs)
  {
    return lhs.real_ == rhs.real_ && lhs.imag_ == rhs.imag_;
  }

  friend CUTLASS_HOST_DEVICE FakeComplexWithHiddenFriendConj
  conj(FakeComplexWithHiddenFriendConj const& z) {
    return FakeComplexWithHiddenFriendConj::make_FakeComplexWithHiddenFriendConj(z.real(), -z.imag());
  }

private:
  CUTLASS_HOST_DEVICE
  FakeComplexWithHiddenFriendConj(FakeReal re, FakeReal im)
    : real_(re), imag_(im)
  {}

  FakeReal real_{};
  FakeReal imag_{};
};

} // namespace test

TEST(complex, real_and_imag_with_custom_types) {
  using test::FakeReal;
  using test::FakeComplex;

  {
    FakeReal x = FakeReal::make_FakeReal(42);
    auto x_r = cutlass::real(x);
    static_assert(std::is_same_v<decltype(x_r), FakeReal>);
    EXPECT_EQ(x_r, FakeReal::make_FakeReal(42));
    auto x_i = cutlass::imag(x);
    static_assert(std::is_same_v<decltype(x_i), FakeReal>);
    EXPECT_EQ(x_i, FakeReal::make_FakeReal(0));
  }
  {
    FakeComplex z = FakeComplex::make_FakeComplex(
      FakeReal::make_FakeReal(3), FakeReal::make_FakeReal(4));
    auto z_r = cutlass::real(z);
    static_assert(std::is_same_v<decltype(z_r), FakeReal>);
    EXPECT_EQ(z_r, FakeReal::make_FakeReal(3));
    auto z_i = cutlass::imag(z);
    static_assert(std::is_same_v<decltype(z_i), FakeReal>);
    EXPECT_EQ(z_i, FakeReal::make_FakeReal(4));
  }
}

namespace test {

template<class T>
void conj_tester(T z, T z_c_expected, const char type_name[]) {
  // Use cutlass::conj just like std::swap (the "std::swap two-step").
  using cutlass::conj;
  auto z_c = conj(z);
  static_assert(std::is_same_v<decltype(z_c), T>);
  constexpr bool is_cuComplex = std::is_same_v<T, hipDoubleComplex> ||
    std::is_same_v<T, hipFloatComplex>;
  if constexpr (is_cuComplex) {
    EXPECT_EQ(z_c.x, z_c_expected.x);
    EXPECT_EQ(z_c.y, z_c_expected.y) << "conj failed for type " << type_name;
  }
  else {
    EXPECT_EQ(z_c, z_c_expected) << "conj failed for type " << type_name;
  }

  auto z_c2 = cutlass::conjugate<T>{}(z);
  static_assert(std::is_same_v<decltype(z_c2), T>);
  if constexpr (is_cuComplex) {
    // hipFloatComplex and hipDoubleComplex don't report conj(z) as
    // being well-formed, probably because they are type aliases of
    // some kind.  cutlass::conj works fine, though!
    static_assert(! cutlass::platform::is_arithmetic_v<T> &&
                  (cutlass::detail::has_unqualified_conj_v<T> ||
                   cutlass::detail::has_cutlass_conj_v<T>));
    
    EXPECT_EQ(z_c2.x, z_c_expected.x);
    EXPECT_EQ(z_c2.y, z_c_expected.y)
      << "conjugate failed for type " << type_name;
  }
  else {
    EXPECT_EQ(z_c2, z_c_expected) << "conjugate failed for type " << type_name;
  }
}

} // namespace test

TEST(complex, conj_with_standard_arithmetic_types) {
  {
    double x = 42.0;
    double x_c_expected = 42.0;
    test::conj_tester(x, x_c_expected, "double");
  }
  {
    float x = 42.0f;
    float x_c_expected = 42.0f;
    test::conj_tester(x, x_c_expected, "float");
  }
  {
    int x = 42;
    int x_c_expected = 42;
    test::conj_tester(x, x_c_expected, "int");
  }
}

TEST(complex, conj_with_cutlass_complex_types) {
  {
    cutlass::complex<double> z{3.0, 4.0};
    cutlass::complex<double> z_c_expected{3.0, -4.0};
    test::conj_tester(z, z_c_expected, "cutlass::complex<double>");
  }
  {
    cutlass::complex<float> z{3.0f, 4.0f};
    cutlass::complex<float> z_c_expected{3.0f, -4.0f};
    test::conj_tester(z, z_c_expected, "cutlass::complex<float>");
  }
  {
    cutlass::complex<cutlass::tfloat32_t> z{
      cutlass::tfloat32_t{3.0f}, cutlass::tfloat32_t{4.0f}};
    cutlass::complex<cutlass::tfloat32_t> z_c_expected{
      cutlass::tfloat32_t{3.0f}, cutlass::tfloat32_t{-4.0f}};
    test::conj_tester(z, z_c_expected, "cutlass::complex<cutlass::tfloat32_t>");
  }
}

TEST(complex, conj_with_noncomplex_type_not_in_cutlass_namespace) {
  test::FakeReal x = test::FakeReal::make_FakeReal(42);
  test::FakeReal x_c_expected = test::FakeReal::make_FakeReal(42);
  test::conj_tester(x, x_c_expected, "test::FakeReal");
}

TEST(complex, conj_with_noncomplex_type_in_cutlass_namespace) {
  cutlass::tfloat32_t x{42.0f};
  cutlass::tfloat32_t x_c_expected{42.0f};
  test::conj_tester(x, x_c_expected, "cutlass::tfloat32_t");
}

TEST(complex, conj_with_complex_types_not_in_cutlass_namespace) {
  using test::FakeReal;

  // conj defined as nonmember outside the class
  {
    test::FakeComplex z = test::FakeComplex::make_FakeComplex(
      FakeReal::make_FakeReal(3), FakeReal::make_FakeReal(4));
    test::FakeComplex z_c_expected = test::FakeComplex::make_FakeComplex(
      FakeReal::make_FakeReal(3), FakeReal::make_FakeReal(-4));
    test::conj_tester(z, z_c_expected, "test::FakeComplex");
  }
  // conj defined as hidden friend
  {
    test::FakeComplexWithHiddenFriendConj z =
      test::FakeComplexWithHiddenFriendConj::make_FakeComplexWithHiddenFriendConj(
        FakeReal::make_FakeReal(3),
        FakeReal::make_FakeReal(4));
    test::FakeComplexWithHiddenFriendConj z_c_expected =
      test::FakeComplexWithHiddenFriendConj::make_FakeComplexWithHiddenFriendConj(
        FakeReal::make_FakeReal(3),
        FakeReal::make_FakeReal(-4));
    test::conj_tester(z, z_c_expected, "test::FakeComplexWithHiddenFriendConj");
  }
}

TEST(complex, conj_with_cuda_std_complex_types) {
  {
    cuda::std::complex<double> z{3.0, 4.0};
    cuda::std::complex<double> z_c_expected{3.0, -4.0};
    test::conj_tester(z, z_c_expected, "cuda::std::complex<double>");
  }
  {
    cuda::std::complex<float> z{3.0f, 4.0f};
    cuda::std::complex<float> z_c_expected{3.0f, -4.0f};
    test::conj_tester(z, z_c_expected, "cuda::std::complex<float>");
  }
}

TEST(complex, conj_with_cuComplex_types) {
  {
    hipDoubleComplex z = make_hipDoubleComplex(3.0, 4.0);
    hipDoubleComplex z_c_expected = make_hipDoubleComplex(3.0, -4.0);
    test::conj_tester(z, z_c_expected, "hipDoubleComplex");
  }
  {
    hipFloatComplex z = make_hipFloatComplex(3.0f, 4.0f);
    hipFloatComplex z_c_expected = make_hipFloatComplex(3.0f, -4.0f);
    test::conj_tester(z, z_c_expected, "hipFloatComplex");
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {

  /// Thorough testing for basic complex math operators. Uses std::complex as a reference.
  template <typename T, int N, int M>
  struct ComplexOperators {
    ComplexOperators() {
      for (int ar = -N; ar <= N; ++ar) {
        for (int ai = -N; ai <= N; ++ai) {
          for (int br = -N; br <= N; ++br) {
            for (int bi = -N; bi <= N; ++bi) {

              cutlass::complex<T> Ae(T(ar) / T(M), T(ai) / T(M));
              cutlass::complex<T> Be(T(br) / T(M), T(bi) / T(M));

              std::complex<T> Ar(T(ar) / T(M), T(ai) / T(M));
              std::complex<T> Br(T(br) / T(M), T(bi) / T(M));

              cutlass::complex<T> add_e = Ae + Be;
              cutlass::complex<T> sub_e = Ae - Be;
              cutlass::complex<T> mul_e = Ae * Be;

              std::complex<T> add_r = (Ar + Br);
              std::complex<T> sub_r = (Ar - Br);
              std::complex<T> mul_r = (Ar * Br);

              EXPECT_EQ(real(add_e), real(add_r));
              EXPECT_EQ(imag(add_e), imag(add_r));

              EXPECT_EQ(real(sub_e), real(sub_r));
              EXPECT_EQ(imag(sub_e), imag(sub_r));

              EXPECT_EQ(real(mul_e), real(mul_r));
              EXPECT_EQ(imag(mul_e), imag(mul_r));

              if (!(br == 0 && bi == 0)) {

                cutlass::complex<T> div_e = Ae / Be;
                std::complex<T> div_r = Ar / Br;

                T const kRange = T(0.001);

                EXPECT_NEAR(real(div_e), real(div_r), kRange);
                EXPECT_NEAR(imag(div_e), imag(div_r), kRange);
              }
            }
          }
        }
      }
    }
  };
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, host_float) {
  test::ComplexOperators<float, 32, 8> test;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(complex, host_double) {
  test::ComplexOperators<double, 32, 8> test;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
