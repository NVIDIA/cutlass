/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <cute/tensor.hpp>

#include "../cooperative_gemm_common.hpp"

using namespace cute;

TEST(SM75_CuTe_Turing, CooperativeGemm1_MixedPrecisionFP16FP32_MMA) {
  using TA = cutlass::half_t;
  using TB = cutlass::half_t;
  using TC = float;

  constexpr uint32_t m = 64;
  constexpr uint32_t n = 64;
  constexpr uint32_t k = 64;

  constexpr uint32_t thread_block_size = 128;

  using tiled_mma_t =
      TiledMMA<
        MMA_Atom<SM75_16x8x8_F32F16F16F32_TN>,
        Layout<Shape<_2, _2, _1>>
      >;

  test_cooperative_gemm_col_major_layout<m, n, k, thread_block_size, tiled_mma_t, 128, TA, TB, TC>();
}
