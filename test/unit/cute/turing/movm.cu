#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include <cute/atom/copy_traits_sm75.hpp>

using namespace cute;

__global__ void
movm_test_device(uint16_t* g_in, uint16_t* g_out)
{
  int tid = threadIdx.x;

  // load input gmem -> register
  uint32_t reg = reinterpret_cast<uint32_t*>(g_in)[tid];

  // do two movmatrix calls (transpose twice => identity)
  uint32_t tmp = 0;
  uint32_t dst = 0;
  SM75_U32x1_MOVM_T::copy(reg, tmp);
  SM75_U32x1_MOVM_T::copy(tmp, dst);

  // store result
  reinterpret_cast<uint32_t*>(g_out)[tid] = dst;
}

template <class TiledCopy, class GmemLayout>
__global__ void
movm_test_device_cute(uint16_t* g_in, uint16_t* g_out,
                      TiledCopy tiled_copy, GmemLayout gmem_layout)
{
  using namespace cute;

  auto t_g_in  = make_tensor(make_gmem_ptr(reinterpret_cast<uint32_t*>(g_in)),  gmem_layout);
  auto t_g_out = make_tensor(make_gmem_ptr(reinterpret_cast<uint32_t*>(g_out)), gmem_layout);

  int tid = threadIdx.x;

  auto thr_copy = tiled_copy.get_thread_slice(tid);

  auto tXgS = thr_copy.partition_S(t_g_in);
  auto tXgD = thr_copy.partition_D(t_g_out);

  // Register tensors for intermediate and output data
  auto tXrS = make_tensor<uint32_t>(shape(tXgS)); // src
  auto tXrT = make_tensor<uint32_t>(shape(tXgS)); // tmp
  auto tXrD = make_tensor<uint32_t>(shape(tXgD)); // dst
  clear(tXrS);
  clear(tXrT);
  clear(tXrD);

  // Load gmem -> registers
  for (int i = 0; i < size(tXrS); ++i) {
    tXrS(i) = tXgS(i);
  }

  // do two movmatrix calls for identity
  copy(tiled_copy, tXrS, tXrT);
  copy(tiled_copy, tXrT, tXrD);

  // Store registers -> gmem
  for (int i = 0; i < size(tXrD); ++i) {
    tXgD(i) = tXrD(i);
  }
}

TEST(SM75_CuTe_Turing, Movm)
{
  constexpr int count = 1024;

  thrust::host_vector<uint16_t> h_in(count);
  for (int i = 0; i < count; ++i) {
    h_in[i] = uint16_t(i);
  } 
  thrust::device_vector<uint16_t> d_in = h_in;

  //
  // Direct MOVM
  //

  {
  thrust::device_vector<uint16_t> d_out(count);
  movm_test_device<<<1, 32>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()));
  thrust::host_vector<uint16_t> h_out = d_out;
  // applied movmatrix twice so result should equal input
  for (int i = 0; i < 64; ++i) {
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("MOVM movm_test_device SUCCESS\n");
  }

  //
  // CuTe MOVM
  //

  {
  thrust::device_vector<uint16_t> d_out(count);

  auto gmem_layout = Layout<Shape <_32, _1>, 
                            Stride< _1,_32>>{};
  auto tiled_copy = make_tiled_copy(Copy_Atom<SM75_U32x1_MOVM_T, uint32_t>{},
                                    Layout<Shape<_32, _1>>{}, 
                                    Layout<Shape< _1, _1>>{});

  movm_test_device_cute<<<1, int(size(tiled_copy))>>>(                              
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tiled_copy,
    gmem_layout);
  thrust::host_vector<uint16_t> h_out = d_out;
  for (int i = 0; i < (size(gmem_layout)*2); ++i) {
    EXPECT_EQ(h_out[i], h_in[i]);
  }
  CUTLASS_TRACE_HOST("CuTe MOVM SUCCESS\n");
  }

  CUTLASS_TRACE_HOST("PASS");
}
