#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

using namespace cute;

template <class ElementType, class SmemLayout>
struct SharedStorage
{
  cute::array_aligned<ElementType, cute::cosize_v<SmemLayout>> smem;
  cute::uint64_t tma_load_mbar[1];
};

#if CUDA_12_0_SM90_FEATURES_SUPPORTED
template <class T, class TiledCopy, class CTA_Tiler, class GmemLayout, class SmemLayout>
__global__ void
tma_test_device_cute(T const* g_in, T* g_out,
                     CUTE_GRID_CONSTANT TiledCopy const tma, CTA_Tiler cta_tiler,
                     GmemLayout gmem_layout, SmemLayout smem_layout)
{
  CUTE_STATIC_ASSERT_V(product_each(shape(cta_tiler)) == product_each(shape(smem_layout)));

  // Use Shared Storage structure to allocate and distribute aligned SMEM addresses
  extern __shared__ char shared_memory[];
  using SharedStorage = SharedStorage<T, SmemLayout>;
  SharedStorage& shared_storage = *reinterpret_cast<SharedStorage*>(shared_memory);
  // Construct SMEM tensor
  Tensor sA = make_tensor(make_smem_ptr(shared_storage.smem.data()), smem_layout);  // (CTA_TILE_M,CTA_TILE_N,...)
  // Shared memory barriers use 64bits in SMEM for synchronization
  uint64_t* tma_load_mbar = shared_storage.tma_load_mbar;

  // TMA requires special handling of strides to deal with coord codomain mapping
  // Represent the full tensors -- get these from TMA
  Tensor mA = tma.get_tma_tensor(shape(gmem_layout));
  Tensor mB = make_tensor(make_gmem_ptr(g_out), gmem_layout);

  constexpr int R = rank_v<CTA_Tiler>;
  Tensor gA = local_tile(mA, cta_tiler, repeat<R>(_));               // (CTA_TILE_M,CTA_TILE_N,...REST_M,REST_N,...)
  Tensor gB = local_tile(mB, cta_tiler, repeat<R>(_));               // (CTA_TILE_M,CTA_TILE_N,...REST_M,REST_N,...)

  //
  // Prepare the TMA_LOAD
  //

  auto cta_tma = tma.get_slice(Int<0>{});                            // CTA slice

  Tensor tAgA_x = cta_tma.partition_S(gA);                           // (TMA,TMA_M,TMA_N,REST_M,REST_N)
  Tensor tAsA_x = cta_tma.partition_D(sA);                           // (TMA,TMA_M,TMA_N)

#if 0
  if (thread0()) {
    print(tma);
    print("TILE  :  "); print(cta_tiler); print("\n");
    print("  mA  :  "); print(  mA.data());   print(" o "); print(  mA.layout());   print("\n");
    print("  gA  :  "); print(  gA.data());   print(" o "); print(  gA.layout());   print("\n");
    print("tAgA_x:  "); print(tAgA_x.data()); print(" o "); print(tAgA_x.layout()); print("\n");
    print("  sA  :  "); print(  sA.data());   print(" o "); print(  sA.layout());   print("\n");
    print("tAsA_x:  "); print(tAsA_x.data()); print(" o "); print(tAsA_x.layout()); print("\n");
  }
#endif

  //
  // Perform the TMA_LOAD
  //

  // INPUT: Group the REST_X modes and the TMA_X modes to easily iterate through the tiles
  Tensor tAgA = group_modes<1,rank(tAgA_x)>(tAgA_x);                 // (TMA,REST)
  Tensor tAsA = group_modes<1,rank(tAsA_x)>(tAsA_x);                 // (TMA,REST)
  static_assert(size<1>(tAsA) == 1);

  // OUTPUT: Group the CTA_TILE_X modes and REST_X modes for output
  Tensor tBgB = group_modes<0,R>(group_modes<R,rank(gB)>(gB));       // (CTA_TILE, REST)

#if 0
  if (thread0()) {
    print("tAgA  :  "); print(tAgA.data()); print(" o "); print(tAgA.layout()); print("\n");
    print("tAsA  :  "); print(tAsA.data()); print(" o "); print(tAsA.layout()); print("\n");
    print("tBgB  :  "); print(tBgB.data()); print(" o "); print(tBgB.layout()); print("\n");
  }
#endif

  // Loop over the TMA stages, using smem as our buffer
  for (int stage = 0; stage < size<1>(tAgA); ++stage)
  {
    // Set the bytes transferred in this TMA transaction (may involve multiple issues)
    constexpr int kTmaTransactionBytes = size(sA) * sizeof(T);

    if (threadIdx.x == 0)
    {
      /// Initialize shared memory barrier
      tma_load_mbar[0] = 0;
      cute::initialize_barrier(tma_load_mbar[0], 1 /*numThreads*/);
      cute::set_barrier_transaction_bytes(tma_load_mbar[0], kTmaTransactionBytes);

      copy(tma.with(tma_load_mbar[0]), tAgA(_,stage), tAsA(_,0));
    }
    __syncthreads();

    /// Wait on the shared memory barrier until the phase bit flips from kPhaseBit value
    constexpr int kPhaseBit = 0;
    cute::wait_barrier(tma_load_mbar[0], kPhaseBit);

    //
    // Write out trivially smem -> gmem
    //

    for (int i = threadIdx.x; i < size(sA); i += blockDim.x) {
      tBgB(i,stage) = sA(i);
    }
    __syncthreads();
  }
}

template <class T, class GMEM_Layout, class SMEM_Layout, class CTA_Tile>
void
test_tma_load(GMEM_Layout const& gmem_layout,
              SMEM_Layout const& smem_layout,
              CTA_Tile    const& cta_tile)
{
  thrust::host_vector<T> h_in(cosize(gmem_layout));
  for (int i = 0; i < h_in.size(); ++i) { h_in[i] = T(i); }
  thrust::device_vector<T> d_in = h_in;
  thrust::device_vector<T> d_out(h_in.size(), T(-1));

  Tensor gA = make_tensor(d_in.data().get(), gmem_layout);
  auto tma = make_tma_copy(SM90_TMA_LOAD{}, gA, smem_layout, cta_tile, Int<1>{});
  //print("TMA Box   size:  "); print(typename decltype(tma)::Tiler_MN{}); print("\n");
  //print("TMA Instr size:  "); print(decltype(tma)::NumValSrc); print("\n");

  int smem_size = int(sizeof(SharedStorage<T, decltype(smem_layout)>));
  tma_test_device_cute<<<1, 128, smem_size>>>(
    thrust::raw_pointer_cast(d_in.data()),
    thrust::raw_pointer_cast(d_out.data()),
    tma, cta_tile,
    gmem_layout,
    smem_layout);

  thrust::host_vector<T> h_out = d_out;
  Tensor hA_in  = make_tensor(h_in.data(),  gmem_layout);
  Tensor hA_out = make_tensor(h_out.data(), gmem_layout);
  for (int i = 0; i < size(gmem_layout); ++i) {
    EXPECT_EQ(hA_in(i), hA_out(i));
  }
}

template <class T, class GMEM_Layout, class SMEM_Layout>
void
test_tma_load(GMEM_Layout const& gmem_layout,
              SMEM_Layout const& smem_layout)
{
  return test_tma_load<T>(gmem_layout, smem_layout, product_each(shape(smem_layout)));
}

TEST(SM90_CuTe_Hopper, Tma_Load_32x32_Col)
{
  Layout smem_layout = Layout<Shape<_32,_32>, Stride<_1,_32>>{};
  {
  Layout gmem_layout = smem_layout;
  test_tma_load<int8_t>(gmem_layout, smem_layout);
  test_tma_load<half_t>(gmem_layout, smem_layout);
  test_tma_load< float>(gmem_layout, smem_layout);
  test_tma_load<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), GenColMajor{});
  test_tma_load<int8_t>(gmem_layout, smem_layout);
  test_tma_load<half_t>(gmem_layout, smem_layout);
  test_tma_load< float>(gmem_layout, smem_layout);
  test_tma_load<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), make_stride(Int<1>{}, 1024));
  test_tma_load<int8_t>(gmem_layout, smem_layout);
  test_tma_load<half_t>(gmem_layout, smem_layout);
  test_tma_load< float>(gmem_layout, smem_layout);
  test_tma_load<double>(gmem_layout, smem_layout);
  }
}

TEST(SM90_CuTe_Hopper, Tma_Load_32x32_Row)
{
  Layout smem_layout = Layout<Shape<_32,_32>, Stride<_32,_1>>{};
  {
  Layout gmem_layout = smem_layout;
  test_tma_load<int8_t>(gmem_layout, smem_layout);
  test_tma_load<half_t>(gmem_layout, smem_layout);
  test_tma_load< float>(gmem_layout, smem_layout);
  test_tma_load<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), GenRowMajor{});
  test_tma_load<int8_t>(gmem_layout, smem_layout);
  test_tma_load<half_t>(gmem_layout, smem_layout);
  test_tma_load< float>(gmem_layout, smem_layout);
  test_tma_load<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), make_stride(1024, Int<1>{}));
  test_tma_load<int8_t>(gmem_layout, smem_layout);
  test_tma_load<half_t>(gmem_layout, smem_layout);
  test_tma_load< float>(gmem_layout, smem_layout);
  test_tma_load<double>(gmem_layout, smem_layout);
  }
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_load_swizzle_atom_mn()
{
  auto   smem_layout = SWIZZLE_ATOM<T>{};
  Layout gmem_layout = make_layout(shape(smem_layout), GenColMajor{});
  return test_tma_load<T>(gmem_layout, smem_layout, product_each(shape(smem_layout)));
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_load_swizzle_atom_k()
{
  auto   smem_layout = SWIZZLE_ATOM<T>{};
  Layout gmem_layout = make_layout(shape(smem_layout), GenRowMajor{});
  return test_tma_load<T>(gmem_layout, smem_layout, product_each(shape(smem_layout)));
}

TEST(SM90_CuTe_Hopper, Tma_Load_Swizzle_Atoms)
{
  test_tma_load_swizzle_atom_mn<int8_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_load_swizzle_atom_mn<half_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_load_swizzle_atom_mn< float, GMMA::Layout_MN_SW128_Atom>();
  test_tma_load_swizzle_atom_mn<double, GMMA::Layout_MN_SW128_Atom>();

  test_tma_load_swizzle_atom_mn<int8_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_load_swizzle_atom_mn<half_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_load_swizzle_atom_mn< float, GMMA::Layout_MN_SW64_Atom>();
  test_tma_load_swizzle_atom_mn<double, GMMA::Layout_MN_SW64_Atom>();

  test_tma_load_swizzle_atom_mn<int8_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_load_swizzle_atom_mn<half_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_load_swizzle_atom_mn< float, GMMA::Layout_MN_SW32_Atom>();
  test_tma_load_swizzle_atom_mn<double, GMMA::Layout_MN_SW32_Atom>();

  test_tma_load_swizzle_atom_mn<int8_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_load_swizzle_atom_mn<half_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_load_swizzle_atom_mn< float, GMMA::Layout_MN_INTER_Atom>();
  test_tma_load_swizzle_atom_mn<double, GMMA::Layout_MN_INTER_Atom>();

  test_tma_load_swizzle_atom_k<int8_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_load_swizzle_atom_k<half_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_load_swizzle_atom_k< float, GMMA::Layout_K_SW128_Atom>();
  test_tma_load_swizzle_atom_k<double, GMMA::Layout_K_SW128_Atom>();

  test_tma_load_swizzle_atom_k<int8_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_load_swizzle_atom_k<half_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_load_swizzle_atom_k< float, GMMA::Layout_K_SW64_Atom>();
  test_tma_load_swizzle_atom_k<double, GMMA::Layout_K_SW64_Atom>();

  test_tma_load_swizzle_atom_k<int8_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_load_swizzle_atom_k<half_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_load_swizzle_atom_k< float, GMMA::Layout_K_SW32_Atom>();
  test_tma_load_swizzle_atom_k<double, GMMA::Layout_K_SW32_Atom>();

  test_tma_load_swizzle_atom_k<int8_t, GMMA::Layout_K_INTER_Atom>();
  test_tma_load_swizzle_atom_k<half_t, GMMA::Layout_K_INTER_Atom>();
  test_tma_load_swizzle_atom_k< float, GMMA::Layout_K_INTER_Atom>();
  test_tma_load_swizzle_atom_k<double, GMMA::Layout_K_INTER_Atom>();
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_load_swizzle_tile_mn()
{
  auto   smem_layout = tile_to_shape(SWIZZLE_ATOM<T>{}, Shape<_128,_128>{});
  Layout gmem_layout = make_layout(make_shape(int(size<0>(smem_layout)), int(size<1>(smem_layout))), GenColMajor{});
  return test_tma_load<T>(gmem_layout, smem_layout, product_each(shape(smem_layout)));
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_load_swizzle_tile_k()
{
  auto   smem_layout = tile_to_shape(SWIZZLE_ATOM<T>{}, Shape<_128,_128>{});
  Layout gmem_layout = make_layout(make_shape(int(size<0>(smem_layout)), int(size<1>(smem_layout))), GenRowMajor{});
  return test_tma_load<T>(gmem_layout, smem_layout, product_each(shape(smem_layout)));
}

TEST(SM90_CuTe_Hopper, Tma_Load_Swizzle_Tiles)
{
  // Other T-types use too much smem
  test_tma_load_swizzle_tile_mn<int8_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_load_swizzle_tile_mn<half_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_load_swizzle_tile_mn<int8_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_load_swizzle_tile_mn<half_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_load_swizzle_tile_mn<int8_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_load_swizzle_tile_mn<half_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_load_swizzle_tile_mn<int8_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_load_swizzle_tile_mn<half_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_load_swizzle_tile_k<int8_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_load_swizzle_tile_k<half_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_load_swizzle_tile_k<int8_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_load_swizzle_tile_k<half_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_load_swizzle_tile_k<int8_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_load_swizzle_tile_k<half_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_load_swizzle_tile_k<int8_t, GMMA::Layout_K_INTER_Atom>();
  test_tma_load_swizzle_tile_k<half_t, GMMA::Layout_K_INTER_Atom>();
}


TEST(SM90_CuTe_Hopper, Tma_Load_Metamode)
{
  {
  auto smem_layout = Layout<Shape<_32,_32>, Stride<_1,_32>>{};
    {
    Layout gmem_layout = make_layout(make_shape(make_shape(8,4), 32), GenColMajor{});
    test_tma_load<half_t>(gmem_layout, smem_layout);
    }
    {
    Layout gmem_layout = make_layout(make_shape(make_shape(8,32), 32), GenColMajor{});
    test_tma_load<half_t>(gmem_layout, smem_layout);
    }
    {
    Layout gmem_layout = make_layout(make_shape(make_shape(64,32), 32), GenColMajor{});
    test_tma_load<half_t>(gmem_layout, smem_layout);
    }
  }

  {
  auto smem_layout = Layout<Shape<_32,_32>, Stride<_32,_1>>{};
    {
    Layout gmem_layout = make_layout(make_shape(make_shape(8,4), 32), GenRowMajor{});
    test_tma_load<half_t>(gmem_layout, smem_layout);
    }
    {
    Layout gmem_layout = make_layout(make_shape(make_shape(8,32), 32), GenRowMajor{});
    test_tma_load<half_t>(gmem_layout, smem_layout);
    }
    {
    Layout gmem_layout = make_layout(make_shape(make_shape(64,32), 32), GenRowMajor{});
    test_tma_load<half_t>(gmem_layout, smem_layout);
    }
  }
}

TEST(SM90_CuTe_Hopper, Tma_Load_Tensor)
{
  // Tensor by-mode
  {
  Layout gmem_layout = make_layout(make_shape(make_shape(80,40),make_shape(32,12)));
  auto cta_tile      = Shape<Shape<_16,_8>,Shape<_32,_2>>{}; // GMEM Tiling:
                                                             //   Take 16-elem from m0, 8-elem from m1,
                                                             //   Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_128,_64>{});
  test_tma_load<half_t>(gmem_layout, smem_layout, cta_tile);
  }

  // Tensor Metamode -- Tiler selects flat elements from a multimode
  {
  Layout gmem_layout = make_layout(make_shape(make_shape(32,40),make_shape(make_shape(8,8),12)));
  auto cta_tile      = Shape<_128, Shape<_32,_2>>{};         // GMEM Tiling:
                                                             //   Take 128-elem from m: m0 must divide 128,
                                                             //                         m-last may be predicated
                                                             //   Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_128,_64>{});
  test_tma_load<half_t>(gmem_layout, smem_layout, cta_tile);
  }

  // Tensor Multimode -- TMA with more than 5 modes in GMEM (packs residual modes into last TMA mode)
  {
  Layout gmem_layout = make_layout(make_shape(make_shape(32,3,2,2),make_shape(32,4,2)));
  auto cta_tile      = Shape<Shape<_32>, Shape<_32,_2>>{};    // GMEM Tiling:
                                                              //  Take 32-elem from m0
                                                              //  Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_32,_64>{});
  test_tma_load<half_t>(gmem_layout, smem_layout, cta_tile);
  }

}

#endif
