/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include "../hopper/tma_store_testbed.hpp"

using namespace cute;
using namespace cutlass::test;

#if CUDA_12_0_SM90_FEATURES_SUPPORTED

template <class T, class TmaType = T, class GMEM_Layout, class SMEM_Layout, class CTA_Tile>
void
test_tma_store(GMEM_Layout const& gmem_layout,
               SMEM_Layout const& smem_layout,
               CTA_Tile    const& cta_tile)
{
  using namespace cute;
  return test_tma_store<T, TmaType>(SM90_TMA_STORE{}, gmem_layout, smem_layout, cta_tile);
}

template <class T, class TmaType = T, class GMEM_Layout, class SMEM_Layout>
void
test_tma_store(GMEM_Layout const& gmem_layout,
               SMEM_Layout const& smem_layout)
{
  using namespace cute;
  return test_tma_store<T, TmaType>(gmem_layout, smem_layout, product_each(shape(smem_layout)));
}

TEST(SM90_CuTe_Hopper, Tma_Load_1D)
{
  Layout smem_layout = Layout<_256, _1>{};
  {
  Layout gmem_layout = smem_layout;
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(128, GenColMajor{});
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }
}

TEST(SM90_CuTe_Hopper, Tma_Store_32x32_Col)
{
  Layout smem_layout = Layout<Shape<_32,_32>, Stride<_1,_32>>{};
  {
  Layout gmem_layout = smem_layout;
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), GenColMajor{});
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), make_stride(Int<1>{}, 1024));
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }
}

TEST(SM90_CuTe_Hopper, Tma_Store_32x32_Row)
{
  Layout smem_layout = Layout<Shape<_32,_32>, Stride<_32,_1>>{};
  {
  Layout gmem_layout = smem_layout;
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), GenRowMajor{});
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }

  {
  Layout gmem_layout = make_layout(make_shape(32,32), make_stride(1024, Int<1>{}));
  test_tma_store<int8_t>(gmem_layout, smem_layout);
  test_tma_store<half_t>(gmem_layout, smem_layout);
  test_tma_store< float>(gmem_layout, smem_layout);
  test_tma_store<double>(gmem_layout, smem_layout);
  }
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_store_swizzle_atom_mn()
{
  auto   smem_layout = SWIZZLE_ATOM<T>{};
  Layout gmem_layout = make_layout(make_shape(2*size<0>(smem_layout), 2*size<1>(smem_layout)), GenColMajor{});
  return test_tma_store<T>(gmem_layout, smem_layout);
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_store_swizzle_atom_k()
{
  auto   smem_layout = SWIZZLE_ATOM<T>{};
  Layout gmem_layout = make_layout(make_shape(2*size<0>(smem_layout), 2*size<1>(smem_layout)), GenRowMajor{});
  return test_tma_store<T>(gmem_layout, smem_layout);
}

TEST(SM90_CuTe_Hopper, Tma_Store_Swizzle_Atoms)
{
  test_tma_store_swizzle_atom_mn<int8_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_store_swizzle_atom_mn<half_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_store_swizzle_atom_mn< float, GMMA::Layout_MN_SW128_Atom>();
  test_tma_store_swizzle_atom_mn<double, GMMA::Layout_MN_SW128_Atom>();

  test_tma_store_swizzle_atom_mn<int8_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_store_swizzle_atom_mn<half_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_store_swizzle_atom_mn< float, GMMA::Layout_MN_SW64_Atom>();
  test_tma_store_swizzle_atom_mn<double, GMMA::Layout_MN_SW64_Atom>();

  test_tma_store_swizzle_atom_mn<int8_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_store_swizzle_atom_mn<half_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_store_swizzle_atom_mn< float, GMMA::Layout_MN_SW32_Atom>();
  test_tma_store_swizzle_atom_mn<double, GMMA::Layout_MN_SW32_Atom>();

  test_tma_store_swizzle_atom_mn<int8_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_store_swizzle_atom_mn<half_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_store_swizzle_atom_mn< float, GMMA::Layout_MN_INTER_Atom>();
  test_tma_store_swizzle_atom_mn<double, GMMA::Layout_MN_INTER_Atom>();

  test_tma_store_swizzle_atom_k<int8_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_store_swizzle_atom_k<half_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_store_swizzle_atom_k< float, GMMA::Layout_K_SW128_Atom>();
  test_tma_store_swizzle_atom_k<double, GMMA::Layout_K_SW128_Atom>();

  test_tma_store_swizzle_atom_k<int8_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_store_swizzle_atom_k<half_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_store_swizzle_atom_k< float, GMMA::Layout_K_SW64_Atom>();
  test_tma_store_swizzle_atom_k<double, GMMA::Layout_K_SW64_Atom>();

  test_tma_store_swizzle_atom_k<int8_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_store_swizzle_atom_k<half_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_store_swizzle_atom_k< float, GMMA::Layout_K_SW32_Atom>();
  test_tma_store_swizzle_atom_k<double, GMMA::Layout_K_SW32_Atom>();

  test_tma_store_swizzle_atom_k<int8_t, GMMA::Layout_K_INTER_Atom>();
  test_tma_store_swizzle_atom_k<half_t, GMMA::Layout_K_INTER_Atom>();
  test_tma_store_swizzle_atom_k< float, GMMA::Layout_K_INTER_Atom>();
  test_tma_store_swizzle_atom_k<double, GMMA::Layout_K_INTER_Atom>();
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_store_swizzle_tile_mn()
{
  auto   smem_layout = tile_to_shape(SWIZZLE_ATOM<T>{}, Shape<_128,_128>{});
  Layout gmem_layout = make_layout(make_shape(2*size<0>(smem_layout), 2*size<1>(smem_layout)), GenColMajor{});
  return test_tma_store<T>(gmem_layout, smem_layout);
}

template <class T, template <typename> typename SWIZZLE_ATOM>
void
test_tma_store_swizzle_tile_k()
{
  auto   smem_layout = tile_to_shape(SWIZZLE_ATOM<T>{}, Shape<_128,_128>{});
  Layout gmem_layout = make_layout(make_shape(2*size<0>(smem_layout), 2*size<1>(smem_layout)), GenRowMajor{});
  return test_tma_store<T>(gmem_layout, smem_layout);
}

TEST(SM90_CuTe_Hopper, Tma_Store_Swizzle_Tiles)
{
  // Other T-types use too much smem
  test_tma_store_swizzle_tile_mn<int8_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_store_swizzle_tile_mn<half_t, GMMA::Layout_MN_SW128_Atom>();
  test_tma_store_swizzle_tile_mn<int8_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_store_swizzle_tile_mn<half_t, GMMA::Layout_MN_SW64_Atom>();
  test_tma_store_swizzle_tile_mn<int8_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_store_swizzle_tile_mn<half_t, GMMA::Layout_MN_SW32_Atom>();
  test_tma_store_swizzle_tile_mn<int8_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_store_swizzle_tile_mn<half_t, GMMA::Layout_MN_INTER_Atom>();
  test_tma_store_swizzle_tile_k<int8_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_store_swizzle_tile_k<half_t, GMMA::Layout_K_SW128_Atom>();
  test_tma_store_swizzle_tile_k<int8_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_store_swizzle_tile_k<half_t, GMMA::Layout_K_SW64_Atom>();
  test_tma_store_swizzle_tile_k<int8_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_store_swizzle_tile_k<half_t, GMMA::Layout_K_SW32_Atom>();
  test_tma_store_swizzle_tile_k<int8_t, GMMA::Layout_K_INTER_Atom>();
  test_tma_store_swizzle_tile_k<half_t, GMMA::Layout_K_INTER_Atom>();
}

// Tensor by-mode
TEST(SM90_CuTe_Hopper, Tma_Store_Tensor)
{
  // 3-mode TMA
  {
  Layout gmem_layout = make_layout(make_shape(128, 64, 5));
  auto cta_tile      = Shape<_64, _32>{};                    // GMEM Tiling:
                                                             //   Take 64-elem from m
                                                             //   Take 32-elem from k
  auto smem_layout = make_layout(Shape<_64,_32>{});
  test_tma_store<half_t>(gmem_layout, smem_layout, cta_tile);
  }

  // 4-mode TMA
  {
  Layout gmem_layout = make_layout(make_shape(make_shape(80,40),make_shape(32,12)));
  auto cta_tile      = Shape<Shape<_16,_8>,Shape<_32,_2>>{}; // GMEM Tiling:
                                                             //   Take 16-elem from m0, 8-elem from m1,
                                                             //   Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_128,_64>{});
  test_tma_store<half_t>(gmem_layout, smem_layout, cta_tile);
  }

  // 5-mode TMA
  {
  Layout gmem_layout = make_layout(make_shape(make_shape(32,32,32),make_shape(32,12)));
  auto cta_tile      = Shape<Shape<_16,_4,_2>,Shape<_16,_2>>{}; // GMEM Tiling:
                                                             //   Take 4-elem from m0, 4-elem from m1, 5-elem from m2
                                                             //   Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_128,_32>{});
  test_tma_store<half_t>(gmem_layout, smem_layout, cta_tile);
  }
}

// Tensor Multimode -- TMA with more than 5 modes in GMEM (packs residual modes into last TMA mode)
TEST(SM90_CuTe_Hopper, Tma_Store_Tensor_Multimode)
{
  {
  Layout gmem_layout = make_layout(make_shape(make_shape(32,3,2,2),make_shape(32,4,2)));
  auto cta_tile      = Shape<Shape<_32>, Shape<_32,_2>>{};    // GMEM Tiling:
                                                              //  Take 32-elem from m0
                                                              //  Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_32,_64>{});
  test_tma_store<half_t>(gmem_layout, smem_layout, cta_tile);
  }

  {
  Layout gmem_layout = make_layout(make_shape(make_shape(64,3,2,2),make_shape(32,4,2)));
  auto cta_tile      = Shape<Shape<_32,_3>, Shape<_32,_2>>{}; // GMEM Tiling:
                                                              //  Take 32-elem from m0, 3-elem from m1
                                                              //  Take 32-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_96,_64>{});
  test_tma_store<half_t>(gmem_layout, smem_layout, cta_tile);
  }

  {
  Layout gmem_layout = make_layout(make_shape(make_shape(64,3,2,3,2),make_shape(32,4,2,2)));
  auto cta_tile      = Shape<Shape<_32>, Shape<_16,_2>>{};    // GMEM Tiling:
                                                              //  Take 32-elem from m0
                                                              //  Take 16-elem from k0, 2-elem from k1
  auto smem_layout = make_layout(Shape<_32,_32>{});
  test_tma_store<half_t>(gmem_layout, smem_layout, cta_tile);
  }
}

#endif
