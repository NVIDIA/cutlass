#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>
#include <tuple>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/swizzle.hpp> // cute::Swizzle
#include <cute/swizzle_layout.hpp> // cute::compose(cute::Swizzle)
#include <cute/numeric/numeric_types.hpp>

using namespace cute;

namespace cooperative_copy_mode {
  struct global_shared {};
  struct global_global {};
  struct shared_shared {};
}

// gs --> global to/from shared
template <int MaxVecBits, uint32_t ThreadBlockSize, class T, class GMemLayout, class SMemLayout>
__device__ void
cooperative_copy_default_gs(T const* g_in, T* g_out, GMemLayout const& gmem_layout, SMemLayout const& smem_layout)
{
  using namespace cute;
  extern __shared__ uint128_t smem_buf[];
  // Cast smem_buf to smem_uint8_ptr and move it by MaxVecBits bits
  // This is to make sure tests pass on pointer aligned to MaxVecBits bits
  uint8_t* smem_uint8_ptr = reinterpret_cast<uint8_t*>(smem_buf) + (MaxVecBits/8);
  T* smem = reinterpret_cast<T*>(smem_uint8_ptr);

  Tensor g_in_tensor  = make_tensor(make_gmem_ptr(g_in),  gmem_layout);
  Tensor g_out_tensor = make_tensor(make_gmem_ptr(g_out), gmem_layout);
  Tensor s_tensor     = make_tensor(make_smem_ptr(smem),  smem_layout);

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, g_in_tensor, s_tensor);

  cp_async_fence();
  cp_async_wait<0>();
  __syncthreads();

  if(thread0()) {
    for(int i = 0; i < size(s_tensor); ++i) {
      s_tensor(i) += T(i);
    }
  }
  __syncthreads();

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, s_tensor, g_out_tensor);
}

// ss --> shared to shared
template <int MaxVecBits, uint32_t ThreadBlockSize, class T, class Layout1, class Layout2>
__device__ void
cooperative_copy_default_ss(T const* g_in, T* g_out, Layout1 const& layout1, Layout2 const& layout2)
{
  using namespace cute;
  extern __shared__ uint128_t smem_buf[];
  // Cast smem_buf to smem_uint8_ptr and move it by MaxVecBits bits
  // This is to make sure tests pass on pointer aligned to MaxVecBits bits
  T* smem1 = reinterpret_cast<T*>(smem_buf);
  uint8_t* smem2_uint8_ptr = reinterpret_cast<uint8_t*>(smem_buf) + (MaxVecBits/8);
  T* smem2 = reinterpret_cast<T*>(smem2_uint8_ptr) + cute::cosize(layout2);

  Tensor g_in_tensor  = make_tensor(make_gmem_ptr(g_in),  layout1);
  Tensor g_out_tensor = make_tensor(make_gmem_ptr(g_out), layout2);

  Tensor s1_tensor    = make_tensor(make_smem_ptr(smem1), layout2);
  Tensor s2_tensor    = make_tensor(make_smem_ptr(smem2), layout1);

  cooperative_copy<ThreadBlockSize,  cute::sizeof_bits_v<T>>(threadIdx.x, g_in_tensor, s1_tensor);

  cp_async_fence();
  cp_async_wait<0>();
  __syncthreads();

  if(thread0()) {
    for(int i = 0; i < size(s1_tensor); ++i) {
      s1_tensor(i) += T(i);
    }
  }
  __syncthreads();

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, s1_tensor, s2_tensor);
  __syncthreads();

  cooperative_copy<ThreadBlockSize,  cute::sizeof_bits_v<T>>(threadIdx.x, s2_tensor, g_out_tensor);
}

// gg --> global to global
template <int MaxVecBits, uint32_t ThreadBlockSize, class T, class Layout1, class Layout2>
__device__ void
cooperative_copy_default_gg(T const* g_in, T* g_out, Layout1 const& layout1, Layout2 const& layout2)
{
  using namespace cute;

  Tensor g_in_tensor  = make_tensor(make_gmem_ptr(g_in),  layout1);
  Tensor g_out_tensor = make_tensor(make_gmem_ptr(g_out), layout2);

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, g_in_tensor, g_out_tensor);
}

template <class Mode, int MaxVecBits, uint32_t ThreadBlockSize, class T, class Layout1, class Layout2>
__global__ void
cooperative_copy_default_kernel(T const* g_in, T* g_out, Layout1 const layout1, Layout2 const layout2)
{
  if constexpr(std::is_same_v<Mode, cooperative_copy_mode::global_shared>) {
    cooperative_copy_default_gs<MaxVecBits, ThreadBlockSize>(g_in, g_out, layout1, layout2);
  } else if constexpr (std::is_same_v<Mode, cooperative_copy_mode::global_global>) {
    cooperative_copy_default_gg<MaxVecBits, ThreadBlockSize>(g_in, g_out, layout1, layout2);
  } else if constexpr (std::is_same_v<Mode, cooperative_copy_mode::shared_shared>) {
    cooperative_copy_default_ss<MaxVecBits, ThreadBlockSize>(g_in, g_out, layout1, layout2);
  }
}

// Mode - defines memory types of src and dst in cooperative_copy operation
// MaxVecBits - defines max vectorization in cooperative_copy operation, and enforces that
//              alignment on used pointers to ensure correct testing
template <class Mode, int MaxVecBits, uint32_t ThreadBlockSize, class T, class Layout1, class Layout2>
void test_cooperative_copy_default(Layout1 const& layout1, Layout2 const& layout2)
{
  using value_type = T;
  CUTE_STATIC_ASSERT_V(cute::size(layout1) == cute::size(layout2));

  auto gmem_layout_in  = layout1;
  auto gmem_layout_out = cute::conditional_return<std::is_same_v<Mode, cooperative_copy_mode::global_shared>>(layout1, layout2);

#if 0
  print("   "); print("layout1:     "); print(layout1); print("\n");
  print("   "); print("layout2:     "); print(layout2); print("\n");
  print("   "); print("threads:     "); print(ThreadBlockSize); print("\n");
  print("   "); print("maxvecbits:  "); print(MaxVecBits); print("\n");
#endif

  if constexpr (MaxVecBits < cute::sizeof_bits_v<value_type>) {
    GTEST_SKIP() << "Skipping test since MaxVecBits (=" << MaxVecBits
                 << ") < cute::sizeof_bits_v<value_type> (=" << cute::sizeof_bits_v<value_type> << ")";
  } else {
    constexpr auto max_vec_bytes = MaxVecBits / 8;
    static_assert((max_vec_bytes % sizeof(T)) == 0);

    uint32_t count = cute::cosize(gmem_layout_in);
    // Extra elements to force MaxVecBits alignment in global memory
    uint32_t extra_elements = max_vec_bytes / sizeof(value_type);

    // Allocate
    thrust::host_vector<value_type> h_in (count + extra_elements);
    thrust::host_vector<value_type> h_out(count + extra_elements);

    // Initialize
    Tensor h_in_tensor  = make_tensor(h_in.data()  + extra_elements, gmem_layout_in);
    Tensor h_out_tensor = make_tensor(h_out.data() + extra_elements, gmem_layout_out);
    for (int i = 0; i < cute::size(h_in_tensor); ++i) {
      h_in_tensor(i)  = value_type(float(i));
      // For global-to-global copy need to compare against the same value
      h_out_tensor(i) = std::is_same_v<Mode, cooperative_copy_mode::global_global> ? value_type(float(i)) : value_type(float(2 * i));
    }

    // To GPU
    thrust::device_vector<value_type> d_in = h_in;
    thrust::device_vector<value_type> d_out(d_in.size(), value_type(float(-2)));

    // Adds (MaxVecBits/8) bytes to shared memory as we'll move pointer by that many bytes inside the kernel to enforce
    // alignment to (MaxVecBits/8) bytes
    size_t shared_memory_bytes = (sizeof(value_type) * count) + max_vec_bytes;
    shared_memory_bytes += std::is_same_v<Mode, cooperative_copy_mode::shared_shared> * (sizeof(value_type) * count);

    // Launch
    auto coop_copy = cooperative_copy_default_kernel<Mode, MaxVecBits, ThreadBlockSize, value_type, Layout1, Layout2>;
    ASSERT_EQ(hipFuncSetAttribute(reinterpret_cast<const void*>(coop_copy), hipFuncAttributeMaxDynamicSharedMemorySize, static_cast<int>(shared_memory_bytes)), hipSuccess);

    auto d_in_ptr  = thrust::raw_pointer_cast(d_in.data()  + extra_elements);
    auto d_out_ptr = thrust::raw_pointer_cast(d_out.data() + extra_elements);
    coop_copy<<<1, ThreadBlockSize, shared_memory_bytes>>>(d_in_ptr, d_out_ptr, layout1, layout2);

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      hipError_t error = hipGetLastError();
      FAIL() << "Error at kernel sync: " << hipGetErrorString(error) << "\n";
    }

    // Validate
    thrust::host_vector<value_type> h_result        = d_out;
    Tensor                          h_result_tensor = make_tensor(h_result.data() + extra_elements, gmem_layout_out);
    for (int i = 0; i < cute::size(h_in_tensor); ++i) {
      ASSERT_EQ(h_result_tensor(i), h_out_tensor(i))
          << i << " - result:" << h_result_tensor(i) << " expected:" << h_out_tensor(i);
    }
  }
}

template<class T>
class SM80_CuTe_Ampere;

template<class Mode, class MaxVecBits>
class SM80_CuTe_Ampere<std::tuple<Mode, MaxVecBits>>: public testing::Test
{
public:
  using mode = Mode;
  static constexpr int max_vec_bits = MaxVecBits::value;
};

typedef testing::Types<
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<128>>,
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<64>>,
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<32>>,
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<16>>,

  std::tuple<cooperative_copy_mode::global_global, cute::Int<128>>,
  std::tuple<cooperative_copy_mode::global_global, cute::Int<64>>,
  std::tuple<cooperative_copy_mode::global_global, cute::Int<32>>,
  std::tuple<cooperative_copy_mode::global_global, cute::Int<16>>,

  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<128>>,
  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<64>>,
  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<32>>,
  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<16>>,
> CooperativeCopyModeMaxVecBitsList;

TYPED_TEST_SUITE(SM80_CuTe_Ampere, CooperativeCopyModeMaxVecBitsList);

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault1D)
{
  using value_type = float;
  constexpr uint32_t count = 512;
  auto gmem_layout = make_layout(make_shape(Int<count>{}));
  auto smem_layout = make_layout(make_shape(Int<count>{}));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault1DFallback)
{
  using value_type = float;
  constexpr uint32_t count = 99;
  auto gmem_layout = make_layout(make_shape(Int<count>{}));
  auto smem_layout = make_layout(make_shape(Int<count>{}));
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault2D)
{
  using value_type = float;
  constexpr uint32_t x = 32;
  constexpr uint32_t y = 32;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}));
  auto smem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

#if 0

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault2DDynamicStrides)
{
  using value_type = float;
  constexpr uint32_t x = 32;
  constexpr uint32_t y = 32;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(1, x));
  auto smem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(1, x));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}



// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault2DMixedStrides)
{
  using value_type = float;
  constexpr uint32_t x = 32;
  constexpr uint32_t y = 32;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}));
  auto smem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(1, x));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

#endif

TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault2DFallback)
{
  using value_type = float;
  constexpr uint32_t x = 37;
  constexpr uint32_t y = 37;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}));
  auto smem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast Path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault2DCustomStride)
{
  using value_type = float;
  constexpr uint32_t x = 16;
  constexpr uint32_t y = 16;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(Int<y>{}, Int<1>{}));
  auto smem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(Int<1>{}, Int<x>{}));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault3D)
{
  using value_type = cute::half_t;
  constexpr uint32_t x = 8;
  constexpr uint32_t y = 8;
  constexpr uint32_t z = 16;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}, Int<z>{}));
  auto smem_layout = make_layout(make_shape(Int<x>{}, Int<y>{}, Int<z>{}));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefault2Dto3D)
{
  using value_type = double;
  constexpr uint32_t x = 16;
  constexpr uint32_t y = 16;
  constexpr uint32_t z = 4;
  auto gmem_layout = make_layout(make_shape(Int<x>{}, Int<y*z>{}));
  auto smem_layout = make_layout(make_shape(Int<z>{}, Int<y>{}, Int<x>{}));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultCustom1)
{
  using value_type = double;
  auto gmem_layout = make_layout(
    make_shape(Int<8>{}, make_shape(Int<2>{}, Int<2>{})),
    make_stride(Int<2>{}, make_shape(Int<1>{}, Int<16>{}))
  );
  auto smem_layout = make_layout(
    make_shape(Int<8>{}, Int<4>{}),
    make_stride(Int<4>{}, Int<1>{})
  );
  constexpr uint32_t thread_block_size = 8;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast Path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultCustom2)
{
  using value_type = float;
  auto gmem_layout = make_layout(
    make_shape(make_shape(Int<4>{}, Int<2>{}), make_shape(Int<2>{}, Int<2>{})),
    make_stride(make_shape(Int<4>{}, Int<1>{}), make_shape(Int<16>{}, Int<2>{}))
  );
  auto smem_layout = make_layout(
    make_shape(make_shape(Int<2>{}, Int<2>{}, Int<2>{}), make_shape(Int<2>{}, Int<2>{})),
    make_stride(make_shape(Int<16>{}, Int<4>{}, Int<1>{}), make_shape(Int<8>{}, Int<2>{}))
  );
  constexpr uint32_t thread_block_size = 16;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast Path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzle1)
{
  using value_type = float;
  auto gmem_layout = Layout<Shape<_8, _64>, Stride<_64, _1>>{};
  auto smem_layout = composition(Swizzle<3, 3, 3>{}, Layout<Shape<_8, _64>, Stride<_64, _1>>{});
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast Path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzle2)
{
  using value_type = cute::half_t;
  auto gmem_layout = make_layout(make_shape(Int<64>{}, Int<64>{}));
  auto smem_atom_layout = composition(Swizzle<3, 2, 3>{}, Layout<Shape<_8, _32>, Stride<_32, _1>>{});
  auto smem_layout = tile_to_shape(
      smem_atom_layout,
      make_shape(shape<0>(gmem_layout), shape<1>(gmem_layout))
  );
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast Path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzle3)
{
  using value_type = cute::half_t;
  auto gmem_layout = make_layout(make_shape(Int<64>{}, Int<64>{}));
  auto smem_atom_layout = composition(Swizzle<2, 4, 3>{}, Layout<Shape<_16, _64>, Stride<_64, _1>>{});
  auto smem_layout = tile_to_shape(
      smem_atom_layout,
      make_shape(shape<0>(gmem_layout), shape<1>(gmem_layout))
  );
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzle4)
{
  using value_type = cute::half_t;
  auto gmem_atom_layout = composition(Swizzle<3, 2, 3>{}, Layout<Shape<_8, _32>, Stride<_32, _1>>{});
  auto smem_layout = make_layout(make_shape(Int<64>{}, Int<64>{}));
  auto gmem_layout = tile_to_shape(
      gmem_atom_layout,
      make_shape(shape<0>(smem_layout), shape<1>(smem_layout))
  );
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

// Needs coalescing to work on fast path
// OK if we enforce slow path
// Problem: Wrong condition when we select between slow and fast path
TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultCoalesceToCompose)
{
  constexpr int m = 96;
  using value_type = cute::half_t;
  auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<m>{}), GenColMajor{});
  auto smem_layout = make_layout(make_shape(Int<m>{}, Int<m>{}), GenColMajor{});
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                thread_block_size,
                                value_type>(gmem_layout, smem_layout);
}

 // Fast path (default): OK
 // Slow path (enforced): OK
 TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzle5)
 {
   constexpr int m = 64;
   constexpr int n = 128;
   using value_type = cute::half_t;
   auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenColMajor{});
   // auto smem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenColMajor{}));
   auto smem_atom_layout =
     composition(Swizzle<3,3,3>{},
                 Layout<Shape < _8,_64>,
                        Stride<_64, _1>>{});
   auto smem_layout = tile_to_shape(
     smem_atom_layout,
     make_shape(shape<0>(gmem_layout), shape<1>(gmem_layout))
   );

   constexpr uint32_t thread_block_size = 128;
   test_cooperative_copy_default<typename TestFixture::mode,
                                 TestFixture::max_vec_bits,
                                 thread_block_size,
                                 value_type>(gmem_layout, smem_layout);
 }

 // If condition not strict enought will go to fast path
 // This test needs checking if CuTe can compose layouts
 // Fast path (default): fail
 // Slow path (enforced): Should go to vectorized naive path
 TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzleNaiveVectorizable)
 {
   constexpr int m = 192;
   constexpr int n = 64;
   using value_type = cute::half_t;
   auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenColMajor{});
   // auto smem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenColMajor{});
   auto smem_atom_layout =
       composition(Swizzle<3,3,3>{},
                   Layout<Shape <_64, _8>,
                          Stride< _1,_64>>{});
   auto smem_layout = tile_to_shape(
     smem_atom_layout,
     shape(gmem_layout)
   );

   constexpr uint32_t thread_block_size = 128;
   test_cooperative_copy_default<typename TestFixture::mode,
                                 TestFixture::max_vec_bits,
                                 thread_block_size,
                                 value_type>(gmem_layout, smem_layout);
 }

 // fast path: ok (chosen)
 // slow path: ok
 TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultRowMajorSmall)
 {
   constexpr int m = 24;
   constexpr int n = 8;
   using value_type = cute::half_t;
   auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});
   auto smem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});

   constexpr uint32_t thread_block_size = 64;
   test_cooperative_copy_default<typename TestFixture::mode,
                                 TestFixture::max_vec_bits,
                                 thread_block_size,
                                 value_type>(gmem_layout, smem_layout);
 }

 // fast path: doesn't apply
 // slow path: ok
 TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSlowPath)
 {
   constexpr int m = 67;
   constexpr int n = 67;
   using value_type = cute::half_t;
   auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});
   auto smem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});

   constexpr uint32_t thread_block_size = 64;
   test_cooperative_copy_default<typename TestFixture::mode,
                                 TestFixture::max_vec_bits,
                                 thread_block_size,
                                 value_type>(gmem_layout, smem_layout);
 }

 // fast path: doesn't apply
 // slow path: should vectorize
 TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopyDefaultSwizzleSlowPathVectorize)
 {
   constexpr int m = 68;
   constexpr int n = 68;
   using value_type = cute::half_t;
   auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});
   auto smem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});

   constexpr uint32_t thread_block_size = 32;
   test_cooperative_copy_default<typename TestFixture::mode,
                                 TestFixture::max_vec_bits,
                                 thread_block_size,
                                 value_type>(gmem_layout, smem_layout);
 }

 TYPED_TEST(SM80_CuTe_Ampere, CooperativeCopy48x48Swizzle)
 {
   constexpr int m = 48;
   constexpr int n = 48;
   using value_type = cute::half_t;
   auto gmem_layout = make_layout(make_shape(Int<m>{}, Int<n>{}), GenRowMajor{});
   auto smem_layout = composition(Swizzle<2,2,3>{},
                                              Layout<Shape <Shape <_16,       _3, Int<48>>>,
                                                     Stride<Stride< _1, Int<768>,     _16>>>{});

   constexpr uint32_t thread_block_size = 8 * 32;
   test_cooperative_copy_default<cooperative_copy_mode::shared_shared,
                                 TestFixture::max_vec_bits,
                                 thread_block_size,
                                 value_type>(gmem_layout, smem_layout);
 }
