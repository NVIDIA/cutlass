/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "tiled_cp_async_testbed.hpp"

using namespace cute;

TEST(SM80_CuTe_tiled_cp_async, no_swizzle_mn_single_tile)
{
  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, double>{});
  using thr_layout = decltype(Layout<Shape <_16, _8>, Stride< _1,_16>>{});
  using val_layout = decltype(Layout<Shape<_2,_1>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using smem_layout_atom = decltype(Layout<Shape <_16, _4>, Stride< _1,_16>>{});
  using gmem_stride_type = decltype(LayoutLeft{});
  test_cp_async_no_swizzle<double, cute::Int<64>, cute::Int<16>, gmem_stride_type, smem_layout_atom, tiled_copy>();
  }

  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, double>{});
  using thr_layout = decltype(Layout<Shape <_16, _8>, Stride< _1,_16>>{});
  using val_layout = decltype(Layout<Shape<_2,_1>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using smem_layout_atom = decltype(Layout<Shape <_16, _4>, Stride< _1,_16>>{});
  using gmem_stride_type = decltype(LayoutLeft{});
  test_cp_async_no_swizzle<double, cute::Int<128>, cute::Int<16>, gmem_stride_type, smem_layout_atom, tiled_copy>();
  }
}

TEST(SM80_CuTe_tiled_cp_async, no_swizzle_k_single_tile)
{
  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, double>{});
  using thr_layout = decltype(Layout<Shape <_16, _8>, Stride< _8,_1>>{});
  using val_layout = decltype(Layout<Shape<_1,_2>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using smem_layout_atom = decltype(make_ordered_layout(Shape<_128,_16>{}, Step <_2, _1>{}));
  using gmem_stride_type = decltype(LayoutRight{});
  test_cp_async_no_swizzle<double, cute::Int<128>, cute::Int<16>, gmem_stride_type, smem_layout_atom, tiled_copy>();
  }
}

TEST(SM80_CuTe_tiled_cp_async, swizzle_mn_single_tile)
{
  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, double>{});
  using thr_layout = decltype(Layout<Shape <_16, _8>, Stride< _1,_16>>{});
  using val_layout = decltype(Layout<Shape<_2,_1>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using swizzle_atom = decltype(Swizzle<2,2,2>{});
  using smem_layout_atom = decltype(Layout<Shape <_16, _4>, Stride< _1,_16>>{});
  using gmem_stride_type = decltype(LayoutLeft{});
  test_cp_async_with_swizzle<double, cute::Int<64>, cute::Int<16>, gmem_stride_type, swizzle_atom, smem_layout_atom, tiled_copy>();
  }

  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, double>{});
  using thr_layout = decltype(Layout<Shape <_16, _8>, Stride< _1,_16>>{});
  using val_layout = decltype(Layout<Shape<_2,_1>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using swizzle_atom = decltype(Swizzle<2,2,2>{});
  using smem_layout_atom = decltype(Layout<Shape <_16, _4>, Stride< _1,_16>>{});
  using gmem_stride_type = decltype(LayoutLeft{});
  test_cp_async_with_swizzle<double, cute::Int<128>, cute::Int<16>, gmem_stride_type, swizzle_atom, smem_layout_atom, tiled_copy>();
  }
}

TEST(SM80_CuTe_tiled_cp_async, swizzle_k_single_tile)
{
  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<double>, double>{});
  using thr_layout = decltype(Layout<Shape < _8,_16>, Stride<_16, _1>>{});
  using val_layout = decltype(Layout<Shape<_1,_1>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using swizzle_atom = decltype(Swizzle<2,0,4>{});
  using smem_layout_atom = decltype(Layout<Shape <_4,_16>, Stride<_1, _4>>{});
  using gmem_stride_type = decltype(LayoutRight{});
  test_cp_async_with_swizzle<double, cute::Int<128>, cute::Int<16>, gmem_stride_type, swizzle_atom, smem_layout_atom, tiled_copy>();
  }

  {
  using copy_atom = decltype(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<cute::uint128_t>, tfloat32_t>{});
  using thr_layout = decltype(Layout<Shape <_16,_8>, Stride< _8,_1>>{});
  using val_layout = decltype(Layout<Shape < _1,_4>>{});
  using tiled_copy = decltype(make_tiled_copy(copy_atom{}, thr_layout{}, val_layout{}));
  using swizzle_atom = decltype(Swizzle<3,2,3>{});
  using smem_layout_atom = decltype(Layout<Shape < _8,_32>, Stride<_32, _1>>{});
  using gmem_stride_type = decltype(LayoutRight{});
  test_cp_async_with_swizzle<tfloat32_t, cute::Int<128>, cute::Int<32>, gmem_stride_type, swizzle_atom, smem_layout_atom, tiled_copy>();
  }
}
