/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <cute/tensor.hpp>
#include <cute/swizzle.hpp> // cute::Swizzle
#include <cute/swizzle_layout.hpp> // cute::compose(cute::Swizzle)

#include "../cooperative_gemm_common.hpp"

using namespace cute;

TEST(SM80_CuTe_Ampere, CooperativeGemm1_Half_MMA) {
  constexpr uint32_t thread_block_size = 128;
  using value_type = cutlass::half_t;

  auto shape_mnk = Shape<_64, _64, _64>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F16F16F16F16_TN>,
        Layout<Shape<_2, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, value_type>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm2_Double_MMA) {
  constexpr uint32_t thread_block_size = 128;
  using value_type = double;

  auto shape_mnk = Shape<_64, _64, _64>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_8x8x4_F64F64F64F64_TN>,
         Layout<Shape<_2,_2,_1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, value_type>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm3_Half_MMA_CustomSmemLayouts) {
  constexpr uint32_t thread_block_size = 128;
  constexpr uint32_t max_vec_bits = 128;
  using value_type = cutlass::half_t;

  auto shape_mnk = Shape<_128, _128, _128>{};
  auto tiled_mma =
    TiledMMA<
      MMA_Atom<SM80_16x8x16_F16F16F16F16_TN>,
      Layout<Shape<_2, _2, _1>>, // 2x2x1 thread group
      Tile<_32, _32, _16> // 32x32x16 MMA for LDSM, 1x2x1 value group`
    >{};

  auto smem_a_atom_layout = Layout<Shape<_64, _8>, Stride< _1,_64>>{};
  auto smem_b_atom_layout = Layout<Shape< _8,_32>, Stride<_32, _1>>{};
  auto smem_c_atom_layout = make_layout(select<0,1>(shape_mnk));

  test_cooperative_gemm_col_major_layout<thread_block_size,
                                         max_vec_bits,
                                         value_type,
                                         value_type,
                                         value_type>
    (smem_a_atom_layout,
    smem_b_atom_layout,
    smem_c_atom_layout,
    shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm4_Half_MMA_SwizzledSmemLayouts) {
  constexpr uint32_t thread_block_size = 128;
  constexpr uint32_t max_vec_bits = 128;
  using value_type = cutlass::half_t;

  auto shape_mnk = Shape<_128, _128, _128>{};
  auto tiled_mma =
    TiledMMA<
      MMA_Atom<SM80_16x8x16_F16F16F16F16_TN>,
      Layout<Shape<_2, _2, _1>>, // 2x2x1 thread group
      Tile<_32, _32, _16> // 32x32x16 MMA for LDSM, 1x2x1 value group`
    >{};

  // RowMajor
  auto smem_a_atom_layout =
    composition(Swizzle<3,3,3>{},
                Layout<Shape < _8,_64>,
                       Stride<_64, _1>>{});
  // ColMajor
  auto smem_b_atom_layout =
    composition(Swizzle<3,3,3>{},
                Layout<Shape <_64, _8>,
                       Stride< _1,_64>>{});

  auto smem_c_atom_layout = make_layout(select<0, 1>(shape_mnk), GenRowMajor{});

  auto gmem_a_layout = make_layout(select<0, 2>(shape_mnk), GenRowMajor{});
  auto gmem_b_layout = make_layout(select<1, 2>(shape_mnk), GenColMajor{});
  auto gmem_c_layout = make_layout(select<0, 1>(shape_mnk), GenRowMajor{});

  auto smem_a_layout = tile_to_shape(
      smem_a_atom_layout,
      make_shape(shape<0>(gmem_a_layout), shape<1>(gmem_a_layout)));

  auto smem_b_layout = tile_to_shape(
      smem_b_atom_layout,
      make_shape(shape<0>(gmem_b_layout), shape<1>(gmem_b_layout)));

  auto smem_c_layout = tile_to_shape(
      smem_c_atom_layout,
      make_shape(shape<0>(gmem_c_layout), shape<1>(gmem_c_layout)));

  test_cooperative_gemm<thread_block_size,
                        max_vec_bits,
                        value_type,
                        value_type,
                        value_type>
    (gmem_a_layout,
     gmem_b_layout,
     gmem_c_layout,
     smem_a_layout,
     smem_b_layout,
     smem_c_layout,
     tiled_mma,
     cute::identity{}, // TransformLoadA
     cute::identity{}, // TransformLoadB
     cute::identity{}, // TransformLoadC
     cute::identity{}, // TransformStoreC
     SM75_U32x4_LDSM_N{}, // A
     SM75_U16x8_LDSM_T{}, // B
     AutoVectorizingCopyWithAssumedAlignment<128>{}); // C
}

TEST(SM80_CuTe_Ampere, CooperativeGemm5_Double_MMA_SwizzledSmemLayouts) {
  constexpr uint32_t thread_block_size = 128;
  constexpr uint32_t max_vec_bits = 128;
  using value_type = double;

  auto shape_mnk = Shape<_128, _64, _16>{};
  auto tiled_mma =
      TiledMMA<MMA_Atom<SM80_8x8x4_F64F64F64F64_TN>,        // Atom
               Layout<Shape<_2, _2, _1>>,                   // Atom layout
               Tile<Layout<Shape<_16, _2>, Stride<_2, _1>>, // 32x32x4 MMA with perm for load vectorization
                    Layout<Shape<_16, _2>, Stride<_2, _1>>,
                    Underscore>>{};

  auto smem_a_atom_layout =
      composition(Swizzle<2,2,2>{},
                  Layout<Shape <_16, _4>,
                         Stride< _1,_16>>{}); // M, K
  auto smem_b_atom_layout =
      composition(Swizzle<2,2,2>{},
                  Layout<Shape <_16, _4>,
                         Stride< _1,_16>>{}); // N, K

  auto smem_c_atom_layout = make_layout(select<0, 1>(shape_mnk), GenRowMajor{});

  auto gmem_a_layout = make_layout(select<0, 2>(shape_mnk), GenRowMajor{});
  auto gmem_b_layout = make_layout(select<1, 2>(shape_mnk), GenColMajor{});
  auto gmem_c_layout = make_layout(select<0, 1>(shape_mnk), GenRowMajor{});

  auto smem_a_layout = tile_to_shape(
      smem_a_atom_layout,
      make_shape(shape<0>(gmem_a_layout), shape<1>(gmem_a_layout)));
  auto smem_b_layout = tile_to_shape(
      smem_b_atom_layout,
      make_shape(shape<0>(gmem_b_layout), shape<1>(gmem_b_layout)));
  auto smem_c_layout = tile_to_shape(
      smem_c_atom_layout,
      make_shape(shape<0>(gmem_c_layout), shape<1>(gmem_c_layout)));

  test_cooperative_gemm<thread_block_size,
                        max_vec_bits,
                        value_type,
                        value_type,
                        value_type>
    (gmem_a_layout,
     gmem_b_layout,
     gmem_c_layout,
     smem_a_layout,
     smem_b_layout,
     smem_c_layout,
     tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm6_MixedPrecisionFP16FP32_MMA) {
  constexpr uint32_t thread_block_size = 128;
  constexpr uint32_t max_vec_bits = 128;
  using TA = cutlass::half_t;
  using TB = cutlass::half_t;
  using TC = float;

  auto shape_mnk = Shape<_64, _64, _64>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F32F16F16F32_TN>,
        Layout<Shape<_2, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, max_vec_bits, TA, TB, TC>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm7_MixedPrecisionBF16FP32_MMA) {
  constexpr uint32_t thread_block_size = 128;
  constexpr uint32_t max_vec_bits = 128;
  using TA = cutlass::bfloat16_t;
  using TB = cutlass::bfloat16_t;
  using TC = float;

  auto shape_mnk = Shape<_64, _64, _64>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F32BF16BF16F32_TN>,
        Layout<Shape<_2, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, max_vec_bits, TA, TB, TC>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm8_MixedPrecisionTF32FP32_MMA) {
  constexpr uint32_t thread_block_size = 128;
  constexpr uint32_t max_vec_bits = 128;
  using TA = cutlass::tfloat32_t;
  using TB = cutlass::tfloat32_t;
  using TC = float;

  auto shape_mnk = Shape<_64, _64, _64>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F32TF32TF32F32_TN>,
        Layout<Shape<_2, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, max_vec_bits, TA, TB, TC>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm9_C64C64C64_MMA_Dynamic) {
  constexpr uint32_t thread_block_size = 256;
  constexpr int MaxVecBits = 128;
  using TA = cutlass::complex<double>;
  using TB = cutlass::complex<double>;
  using TC = cutlass::complex<double>;

  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_8x8x4_C64C64C64C64_TN>,
        Layout<Shape<_4, _4, _1>, Stride<_1, _4, _0>>,
        Tile<Underscore, Underscore, Underscore>
      >{};

  auto a_layout = make_layout(Shape<Int<13>,Int<35>>{}, make_stride(44, 1));
  auto b_layout = make_layout(Shape< Int<7>, Int<35>>{}, make_stride(44, 1));
  auto c_layout = make_layout(Shape<Int<13>,  Int<7>>{}, make_stride(1, 30));

  test_cooperative_gemm<thread_block_size,
                        MaxVecBits,
                        TA, TB, TC>
    (a_layout,
     b_layout,
     c_layout,
     a_layout,
     b_layout,
     c_layout,
     tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm9_C64C64C64_MMA) {
  constexpr uint32_t thread_block_size = 256;
  constexpr int MaxVecBits = 128;
  using TA = cutlass::complex<double>;
  using TB = cutlass::complex<double>;
  using TC = cutlass::complex<double>;

  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_8x8x4_C64C64C64C64_TN>,
        Layout<Shape<_4, _4, _1>, Stride<_1, _4, _0>>,
        Tile<Underscore, Underscore, Underscore>
      >{};

  auto a_layout = Layout<Shape<Int<13>,Int<35>>, Stride<Int<44>, Int<1> >>{};
  auto b_layout = Layout<Shape< Int<7>, Int<35>>, Stride<Int<44>, Int<1> >>{};
  auto c_layout = Layout<Shape<Int<13>,  Int<7>>, Stride< Int<1>, Int<30>>>{};

  test_cooperative_gemm<thread_block_size,
                        MaxVecBits,
                        TA, TB, TC>
    (a_layout,
     b_layout,
     c_layout,
     a_layout,
     b_layout,
     c_layout,
     tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm10_F16F64F16_FMA) {

  constexpr uint32_t thread_block_size = 256;
  constexpr int MaxVecBits = 128;
  using TA = cutlass::half_t;
  using TB = double;
  using TC = cutlass::half_t;

  auto tiled_mma =
      TiledMMA<
        MMA_Atom<UniversalFMA<half_t, half_t, double, half_t>>,
        Layout<Shape<_16, _16, _1>, Stride<_1, _16, _0>>,
        Tile<Underscore, Underscore, Underscore>
      >{};

  auto a_layout = Layout<Shape<Int<64>,Int<64>>, Stride<Int<64>, Int< 1>>>{};
  auto b_layout = Layout<Shape<Int<64>,Int<64>>, Stride<Int< 1>, Int<64>>>{};
  auto c_layout = Layout<Shape<Int<64>,Int<64>>, Stride<Int< 1>, Int<64>>>{};

  test_cooperative_gemm<thread_block_size,
                        MaxVecBits,
                        TA,
                        TB,
                        TC>
    (a_layout,
     b_layout,
     c_layout,
     a_layout,
     b_layout,
     c_layout,
     tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemmComposedStride) {

  constexpr uint32_t thread_block_size = 128;
  constexpr int MaxVecBits = 16;
  using T = cute::half_t;

  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x16_F16F16F16F16_TN>,
        Layout<Shape<_2, _2, _1>, Stride<_1, _2, _0>>,
        Tile<Underscore, Underscore, Underscore>
      >{};

  auto swizzle = cute::Swizzle<3, 3, 3>{};
  auto offset = cute::_0{};
  auto atom_tile_right = cute::make_layout(cute::Shape<cute::_8, cute::_64>{}, cute::LayoutRight{});
  auto FP16AtomLayoutRight = cute::composition(swizzle, offset, atom_tile_right);

  auto shape = cute::Shape<cute::Int<128>, cute::Int<128>>{};
  auto global_a_layout = cute::make_layout(shape, cute::LayoutRight{});
  auto global_b_layout = cute::make_layout(shape, cute::LayoutLeft{});
  auto global_c_layout = cute::make_layout(shape, cute::LayoutRight{});

  // This is for A row major, B col major according to CUTLASS default configs
  auto a_layout = cute::tile_to_shape(FP16AtomLayoutRight, global_a_layout);
  auto b_layout = cute::tile_to_shape(FP16AtomLayoutRight, global_b_layout);
  auto c_layout = global_c_layout;

  test_cooperative_gemm<thread_block_size,
                        MaxVecBits,
                        T, T, T>
    (a_layout,
     b_layout,
     c_layout,
     a_layout,
     b_layout,
     c_layout,
     tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm8_MixedPrecisionTF32FP32_Transform) {
  constexpr uint32_t thread_block_size = 64;
  constexpr uint32_t max_vec_bits = 16;
  using TA = cutlass::tfloat32_t;
  using TB = cutlass::tfloat32_t;
  using TC = float;

  auto shape_mnk = Shape<C<9>, C<9>, C<9>>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F32TF32TF32F32_TN>,
        Layout<Shape<_1, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, max_vec_bits, TA, TB, TC>
    (shape_mnk, tiled_mma, cute::negate{}, cute::negate{}, cute::negate{}, cute::negate{});
}

TEST(SM80_CuTe_Ampere, CooperativeGemm8_MixedPrecisionTF32FP32_TransformPrecision) {
  constexpr uint32_t thread_block_size = 64;
  constexpr uint32_t max_vec_bits = 16;
  using InputTA = cutlass::half_t;
  using InputTB = cutlass::half_t;
  using InputTC = cutlass::half_t;

  using ComputeTA = cutlass::tfloat32_t;
  using ComputeTB = cutlass::tfloat32_t;
  using ComputeTC = float;

  auto shape_mnk = Shape<C<9>, C<9>, C<9>>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F32TF32TF32F32_TN>,
        Layout<Shape<_1, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout<thread_block_size, max_vec_bits, InputTA, InputTB, InputTC>
    (shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm8_MixedPrecisionTF32FP32_TransformPrecisionReg) {
  constexpr uint32_t thread_block_size = 64;
  constexpr uint32_t max_vec_bits = 16;
  using InputTA = cutlass::half_t;
  using InputTB = cutlass::half_t;
  using InputTC = cutlass::half_t;

  using ComputeTA = cutlass::tfloat32_t;
  using ComputeTB = cutlass::tfloat32_t;
  using ComputeTC = float;

  auto shape_mnk = Shape<C<9>, C<9>, C<9>>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F32TF32TF32F32_TN>,
        Layout<Shape<_1, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout_rmem_c<thread_block_size, max_vec_bits, InputTA, InputTB, InputTC>
    (shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm1_Half_MMA_Reg) {
  using value_type = cutlass::half_t;

  auto shape_mnk = Shape<_64, _64, _64>{};

  constexpr uint32_t thread_block_size = 128;

  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_16x8x8_F16F16F16F16_TN>,
        Layout<Shape<_2, _2, _1>>
      >{};

  test_cooperative_gemm_col_major_layout_rmem_c<thread_block_size, value_type>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm2_Double_MMA_Reg) {
  constexpr uint32_t thread_block_size = 128;
  using value_type = double;

  auto shape_mnk = Shape<_64, _64, _64>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_8x8x4_F64F64F64F64_TN>,
         Layout<Shape<_2,_2,_1>>
      >{};

  test_cooperative_gemm_col_major_layout_rmem_c<thread_block_size, value_type>(shape_mnk, tiled_mma);
}

TEST(SM80_CuTe_Ampere, CooperativeGemm2_Double_MMA_Predicated_Reg) {
  constexpr uint32_t thread_block_size = 128;
  using value_type = double;

  auto shape_mnk = Shape<C<62>, C<62>, C<62>>{};
  auto tiled_mma =
      TiledMMA<
        MMA_Atom<SM80_8x8x4_F64F64F64F64_TN>,
         Layout<Shape<_2,_2,_1>>
      >{};

  test_cooperative_gemm_col_major_layout_rmem_c<thread_block_size, value_type>(shape_mnk, tiled_mma);
}
