#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>
#include <tuple>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/numeric/numeric_types.hpp>

using namespace cute;

namespace cooperative_copy_mode {
  struct global_shared {};
  struct global_global {};
  struct shared_shared {};
}

// gs --> global to/from shared
template <int MaxVecBits, class GMemLayout, class SMemLayout, uint32_t ThreadBlockSize, class T>
__device__ void
cooperative_copy_default_gs(T const* g_in, T* g_out)
{
  using namespace cute;
  extern __shared__ float4 smem_buf[];
  // Cast smem_buf to smem_uint8_ptr and move it by MaxVecBits bits
  // This is to make sure tests pass on pointer aligned to MaxVecBits bits
  uint8_t* smem_uint8_ptr = reinterpret_cast<uint8_t*>(smem_buf) + (MaxVecBits/8);
  T* smem = reinterpret_cast<T*>(smem_uint8_ptr);

  Tensor g_in_tensor  = make_tensor(make_gmem_ptr(g_in),  GMemLayout{});
  Tensor g_out_tensor = make_tensor(make_gmem_ptr(g_out), GMemLayout{});
  Tensor s_tensor     = make_tensor(make_smem_ptr(smem), SMemLayout{});

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, g_in_tensor, s_tensor);
  __syncthreads();

  if(thread0()) {
    for(int i = 0; i < size(s_tensor); ++i) {
      s_tensor(i) += T(i);
    }
  }
  __syncthreads();

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, s_tensor, g_out_tensor);
}

// ss --> shared to shared
template <int MaxVecBits, class Layout1, class Layout2, uint32_t ThreadBlockSize, class T>
__device__ void
cooperative_copy_default_ss(T const* g_in, T* g_out)
{
  using namespace cute;
  extern __shared__ float4 smem_buf[];
  // Cast smem_buf to smem_uint8_ptr and move it by MaxVecBits bits
  // This is to make sure tests pass on pointer aligned to MaxVecBits bits
  T* smem1 = reinterpret_cast<T*>(smem_buf);
  uint8_t* smem2_uint8_ptr = reinterpret_cast<uint8_t*>(smem_buf) + (MaxVecBits/8);
  T* smem2 = reinterpret_cast<T*>(smem2_uint8_ptr) + cute::cosize(Layout2{});

  Tensor g_in_tensor  = make_tensor(make_gmem_ptr(g_in), Layout1 {});
  Tensor g_out_tensor = make_tensor(make_gmem_ptr(g_out), Layout2 {});

  Tensor s1_tensor    = make_tensor(make_smem_ptr(smem1), Layout2 {});
  Tensor s2_tensor    = make_tensor(make_smem_ptr(smem2), Layout1 {});

  cooperative_copy<ThreadBlockSize,  cute::sizeof_bits_v<T>>(threadIdx.x, g_in_tensor, s1_tensor);
  __syncthreads();

  if(thread0()) {
    for(int i = 0; i < size(s1_tensor); ++i) {
      s1_tensor(i) += T(i);
    }
  }
  __syncthreads();

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, s1_tensor, s2_tensor);
  __syncthreads();

  cooperative_copy<ThreadBlockSize,  cute::sizeof_bits_v<T>>(threadIdx.x, s2_tensor, g_out_tensor);
}

// gg --> global to global
template <int MaxVecBits, class Layout1, class Layout2, uint32_t ThreadBlockSize, class T>
__device__ void
cooperative_copy_default_gg(T const* g_in, T* g_out)
{
  using namespace cute;

  Tensor g_in_tensor  = make_tensor(make_gmem_ptr(g_in), Layout1{});
  Tensor g_out_tensor = make_tensor(make_gmem_ptr(g_out), Layout2{});

  cooperative_copy<ThreadBlockSize, MaxVecBits>(threadIdx.x, g_in_tensor, g_out_tensor);
}

template <class Mode, int MaxVecBits, class Layout1, class Layout2, uint32_t ThreadBlockSize, class T>
__global__ void
cooperative_copy_default_kernel(T const* g_in, T* g_out)
{
  if constexpr(std::is_same_v<Mode, cooperative_copy_mode::global_shared>) {
    cooperative_copy_default_gs<MaxVecBits, Layout1, Layout2, ThreadBlockSize>(g_in, g_out);
  } else if constexpr (std::is_same_v<Mode, cooperative_copy_mode::global_global>) {
    cooperative_copy_default_gg<MaxVecBits, Layout1, Layout2, ThreadBlockSize>(g_in, g_out);
  } else if constexpr (std::is_same_v<Mode, cooperative_copy_mode::shared_shared>) {
    cooperative_copy_default_ss<MaxVecBits, Layout1, Layout2, ThreadBlockSize>(g_in, g_out);
  }
}

// Mode - defines memory types of src and dst in cooperative_copy operation
// MaxVecBits - defines max vectorization in cooperative_copy operation, and enforces that
//              alignment on used pointers to ensure correct testing
template <class Mode, int MaxVecBits, class Layout1, class Layout2, uint32_t ThreadBlockSize, class T>
void test_cooperative_copy_default()
{
  using value_type = T;
  static_assert(cute::size(Layout1{}) == cute::size(Layout2{}));

  using gmem_layout_in  = Layout1;
  using gmem_layout_out = std::conditional_t<std::is_same_v<Mode, cooperative_copy_mode::global_shared>, Layout1, Layout2>;

#if 0
  print("   "); print("layout1:  "); print(Layout1{}); print("\n");
  print("   "); print("layout2: "); print(Layout2{}); print("\n");
  print("   "); print("threads:  "); print(ThreadBlockSize); print("\n");
#endif

  if constexpr (MaxVecBits < cute::sizeof_bits_v<value_type>) {
    GTEST_SKIP() << "Skipping test since MaxVecBits (=" << MaxVecBits
                 << ") < cute::sizeof_bits_v<value_type> (=" << cute::sizeof_bits_v<value_type> << ")";
  } else {
    constexpr auto max_vec_bytes = MaxVecBits / 8;
    static_assert((max_vec_bytes % sizeof(T)) == 0);

    constexpr uint32_t count = cute::cosize(gmem_layout_in {});
    // Extra elements to force MaxVecBits alignment in global memory
    constexpr uint32_t extra_elements = max_vec_bytes / sizeof(value_type);

    // Allocate
    thrust::host_vector<value_type> h_in(count + extra_elements);
    thrust::host_vector<value_type> h_out(count + extra_elements);

    // Initialize
    Tensor h_in_tensor  = make_tensor((h_in.data() + extra_elements), gmem_layout_in {});
    Tensor h_out_tensor = make_tensor((h_out.data() + extra_elements), gmem_layout_out {});
    for (int i = 0; i < cute::size(h_in_tensor); ++i) {
      h_in_tensor(i)  = value_type(float(i));
      // For global-to-global copy need to compare against the same value
      h_out_tensor(i) = std::is_same_v<Mode, cooperative_copy_mode::global_global> ? value_type(float(i)) : value_type(float(2 * i));
    }

    // To GPU
    thrust::device_vector<value_type> d_in = h_in;
    thrust::device_vector<value_type> d_out(d_in.size(), value_type(float(-2)));

    // Adds (MaxVecBits/8) bytes to shared memory as we'll move pointer by that many bytes inside the kernel to enforce
    // alignment to (MaxVecBits/8) bytes
    size_t shared_memory_bytes = (sizeof(value_type) * count) + max_vec_bytes;
    shared_memory_bytes += std::is_same_v<Mode, cooperative_copy_mode::shared_shared> * (sizeof(value_type) * count);

    // Launch
    auto coop_copy = cooperative_copy_default_kernel<Mode, MaxVecBits, Layout1, Layout2, ThreadBlockSize, value_type>;
    ASSERT_EQ(hipFuncSetAttribute(reinterpret_cast<const void*>(coop_copy), hipFuncAttributeMaxDynamicSharedMemorySize, static_cast<int>(shared_memory_bytes)), hipSuccess);

    auto d_in_ptr = thrust::raw_pointer_cast(d_in.data() + extra_elements);
    auto d_out_ptr = thrust::raw_pointer_cast(d_out.data() + extra_elements);
    coop_copy<<<1, ThreadBlockSize, shared_memory_bytes>>>(d_in_ptr, d_out_ptr);

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      hipError_t error = hipGetLastError();
      FAIL() << "Error at kernel sync: " << hipGetErrorString(error) << "\n";
    }

    // Validate
    thrust::host_vector<value_type> h_result        = d_out;
    Tensor                          h_result_tensor = make_tensor((h_result.data() + extra_elements), gmem_layout_out {});
    for (int i = 0; i < cute::size(h_in_tensor); ++i) {
      ASSERT_EQ(h_result_tensor(i), h_out_tensor(i))
          << i << " - result:" << h_result_tensor(i) << " expected:" << h_out_tensor(i);
    }
  }
}

template<class T>
class SM70_CuTe_Volta;

template<class Mode, class MaxVecBits>
class SM70_CuTe_Volta<std::tuple<Mode, MaxVecBits>>: public testing::Test
{
public:
  using mode = Mode;
  static constexpr int max_vec_bits = MaxVecBits::value;
};

typedef testing::Types<
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<128>>,
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<64>>,
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<32>>,
  std::tuple<cooperative_copy_mode::global_shared, cute::Int<16>>,

  std::tuple<cooperative_copy_mode::global_global, cute::Int<128>>,
  std::tuple<cooperative_copy_mode::global_global, cute::Int<64>>,
  std::tuple<cooperative_copy_mode::global_global, cute::Int<32>>,
  std::tuple<cooperative_copy_mode::global_global, cute::Int<16>>,

  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<128>>,
  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<64>>,
  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<32>>,
  std::tuple<cooperative_copy_mode::shared_shared, cute::Int<16>>,
> CooperativeCopyModeMaxVecBitsList;

TYPED_TEST_SUITE(SM70_CuTe_Volta, CooperativeCopyModeMaxVecBitsList);

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefault1D)
{
  using value_type = float;
  constexpr uint32_t count = 512;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<count>{})));
  using smem_layout_t = decltype(make_layout(make_shape(Int<count>{})));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSG2D)
{
  using value_type = float;
  constexpr uint32_t x = 32;
  constexpr uint32_t y = 32;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y>{})));
  using smem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y>{})));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSG2DCustomStride)
{
  using value_type = float;
  constexpr uint32_t x = 16;
  constexpr uint32_t y = 16;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(Int<y>{}, Int<1>{})));
  using smem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y>{}), make_stride(Int<1>{}, Int<x>{})));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSG3D)
{
  using value_type = cute::half_t;
  constexpr uint32_t x = 8;
  constexpr uint32_t y = 8;
  constexpr uint32_t z = 16;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y>{}, Int<z>{})));
  using smem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y>{}, Int<z>{})));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSG2Dto3D)
{
  using value_type = double;
  constexpr uint32_t x = 16;
  constexpr uint32_t y = 16;
  constexpr uint32_t z = 4;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<x>{}, Int<y*z>{})));
  using smem_layout_t = decltype(make_layout(make_shape(Int<z>{}, Int<y>{}, Int<x>{})));
  constexpr uint32_t thread_block_size = 64;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSGCustom1)
{
  using value_type = double;
  using gmem_layout_t = decltype(make_layout(
    make_shape(Int<8>{}, make_shape(Int<2>{}, Int<2>{})),
    make_stride(Int<2>{}, make_shape(Int<1>{}, Int<16>{}))
  ));
  using smem_layout_t = decltype(make_layout(
    make_shape(Int<8>{}, Int<4>{}),
    make_stride(Int<4>{}, Int<1>{})
  ));
  constexpr uint32_t thread_block_size = 8;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSGCustom2)
{
  using value_type = float;
  using gmem_layout_t = decltype(make_layout(
    make_shape(make_shape(Int<4>{}, Int<2>{}), make_shape(Int<2>{}, Int<2>{})),
    make_stride(make_shape(Int<4>{}, Int<1>{}), make_shape(Int<16>{}, Int<2>{}))
  ));
  using smem_layout_t = decltype(make_layout(
    make_shape(make_shape(Int<2>{}, Int<2>{}, Int<2>{}), make_shape(Int<2>{}, Int<2>{})),
    make_stride(make_shape(Int<16>{}, Int<4>{}, Int<1>{}), make_shape(Int<8>{}, Int<2>{}))
  ));
  constexpr uint32_t thread_block_size = 16;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSGSwizzle1)
{
  using value_type = float;
  using gmem_layout_t = Layout<Shape<_8, _64>, Stride<_64, _1>>;
  using smem_layout_t = decltype(composition(Swizzle<3, 3, 3>{}, Layout<Shape<_8, _64>, Stride<_64, _1>>{}));
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSGSwizzle2)
{
  using value_type = cute::half_t;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<64>{}, Int<64>{})));
  using smem_atom_layout_t = decltype(composition(Swizzle<3, 2, 3> {}, Layout<Shape<_8, _32>, Stride<_32, _1>>{}));
  using smem_layout_t = decltype(tile_to_shape(
      smem_atom_layout_t{},
      make_shape(shape<0>(gmem_layout_t{}), shape<1>(gmem_layout_t{})))
  );
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSGSwizzle3)
{
  using value_type = cute::half_t;
  using gmem_layout_t = decltype(make_layout(make_shape(Int<64>{}, Int<64>{})));
  using smem_atom_layout_t = decltype(composition(Swizzle<2, 4, 3> {}, Layout<Shape<_16, _64>, Stride<_64, _1>>{}));
  using smem_layout_t = decltype(tile_to_shape(
      smem_atom_layout_t{},
      make_shape(shape<0>(gmem_layout_t{}), shape<1>(gmem_layout_t{})))
  );
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}

TYPED_TEST(SM70_CuTe_Volta, CooperativeCopyDefaultGSSGSwizzle4)
{
  using value_type = cute::half_t;
  using gmem_atom_layout_t = decltype(composition(Swizzle<3, 2, 3> {}, Layout<Shape<_8, _32>, Stride<_32, _1>>{}));
  using smem_layout_t = decltype(make_layout(make_shape(Int<64>{}, Int<64>{})));
  using gmem_layout_t = decltype(tile_to_shape(
      gmem_atom_layout_t{},
      make_shape(shape<0>(smem_layout_t{}), shape<1>(smem_layout_t{})))
  );
  constexpr uint32_t thread_block_size = 128;
  test_cooperative_copy_default<typename TestFixture::mode,
                                TestFixture::max_vec_bits,
                                gmem_layout_t,
                                smem_layout_t,
                                thread_block_size,
                                value_type>();
}
