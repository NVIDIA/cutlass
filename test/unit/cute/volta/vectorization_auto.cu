#include "hip/hip_runtime.h"

/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"

#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

using namespace cute;

template <class GmemTensor, class RmemTiler, class CopyPolicy>
__global__
void
kernel(GmemTensor gC, RmemTiler tiler, CopyPolicy policy)
{
  Tensor tCgC = local_tile(gC, tiler, 0);

  Tensor rC = make_tensor_like(tCgC);
  using T = typename GmemTensor::value_type;
  for (int i = 0; i < size(rC); ++i) {
    rC(i) = T(i % 13);
  }

#if 0
  print("  gC : "); print(  gC); print("\n");
  print("tCgC : "); print(tCgC); print("\n");
  print("  rC : "); print(  rC); print("\n");
#endif

  // NOTE: only 1 thread, this thread produce a block of 8x8 output. The fringe will not be touched.
  //copy(rC, tCgC);           // Enable auto-vectorization if static
  //copy_vec<T>(rC, tCgC);    // Disable auto-vectorization always
  copy(policy, rC, tCgC);     // Use a policy to establish vectorization assumptions
}

template <class T, class CopyPolicy, class GmemLayout, class RmemTiler>
void
test_copy_vectorization(CopyPolicy policy, GmemLayout gmem_layout, RmemTiler rmem_tiler)
{
  thrust::host_vector<T> h_in(cosize(gmem_layout), T(0));

  thrust::device_vector<T> d_in = h_in;
  Tensor m_in = make_tensor(make_gmem_ptr(raw_pointer_cast(d_in.data())), gmem_layout);

  kernel<<<1,1>>>(m_in, rmem_tiler, policy);

  thrust::host_vector<T> h_out = d_in;
  Tensor result = make_tensor(h_out.data(), gmem_layout);

  thrust::host_vector<T> h_true = h_in;
  Tensor ref = make_tensor(h_true.data(), gmem_layout);

  // Set the values directly in the reference tensor, no copy
  Tensor ref_tile = local_tile(ref, rmem_tiler, 0);
  for (int i = 0; i < size(ref_tile); ++i) {
    ref_tile(i) = T(i % 13);
  }

  // Compare the reference and the result. Print only the first 3 errors.
  // print_tensor(result);
  int count = 3;
  for (int i = 0; i < size(ref) && count > 0; ++i) {
    EXPECT_EQ(result(i), ref(i));
    if (result(i) != ref(i)) {
      --count;
    }
  }
}

template <class T, class GmemLayout, class RmemTiler>
void
test_copy_vectorization(GmemLayout gmem_layout, RmemTiler rmem_tiler)
{
  test_copy_vectorization<T>(DefaultCopy{}, gmem_layout, rmem_tiler);
}

TEST(SM70_CuTe_Volta, SimpleVec)
{
  // Fully static layouts are assumed to be aligned -- these will be vectorized
  test_copy_vectorization<float>(make_layout(make_shape(Int<8>{}, Int<8>{})), Shape<_8,_8>{});
  test_copy_vectorization<float>(make_layout(make_shape(Int<12>{}, Int<12>{})), Shape<_8,_8>{});
  // Fails in vectorization recast due to misalignment and static assertions
  //test_copy_vectorization<float>(make_layout(make_shape(Int<9>{}, Int<9>{})), Shape<_8,_8>{});

  // Dynamic layouts are not assumed to be aligned -- these will not be vectorized
  test_copy_vectorization<float>(make_layout(make_shape(12,12)), Shape<_8,_8>{});
  test_copy_vectorization<float>(make_layout(make_shape( 9, 9)), Shape<_8,_8>{});

  // Dynamic layouts that are assumed to be aligned -- these will be vectorized
  test_copy_vectorization<float>(AutoVectorizingCopyWithAssumedAlignment<128>{}, make_layout(make_shape( 8, 8)), Shape<_8,_8>{});
  test_copy_vectorization<float>(AutoVectorizingCopyWithAssumedAlignment<128>{}, make_layout(make_shape(12,12)), Shape<_8,_8>{});
  // Fails -- bad alignment assumption
  //test_copy_vectorization<float>(AutoVectorizingCopyWithAssumedAlignment<128>{}, make_layout(make_shape( 9, 9)), Shape<_8,_8>{});
}
