#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2025 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <bitset>
#include <cstdint>
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>

#include <hip/hip_runtime_api.h>

#include "cutlass/cutlass.h"
#include "cutlass/profiler/grouped_gemm_operation_profiler.h"
#include "cutlass/library/handle.h"
#include "cutlass/library/library.h"
/////////////////////////////////////////////////////////////////////////////////////////////////

namespace {
std::vector<std::vector<int>> parseProblemSizes(std::string const& input) {
  // input must be of the form:
  //  `[m0xn0xk0][m1xn1xk1]` where 0, 1 are the group indexes
  std::stringstream ss(input);
  std::string token;
  std::vector<std::vector<int>> result;
  while (std::getline(ss, token, ']')) {
    std::stringstream ss(token);
    std::string token;
    ss.get(); // discard '['
    std::getline(ss, token, 'x');
    auto m = std::stoi(token);
    std::getline(ss, token, 'x');
    auto n = std::stoi(token);
    std::getline(ss, token);
    auto k = std::stoi(token);
    result.push_back({m, n, k});
  }
  return result;
}
} // namespace

namespace cutlass {
namespace profiler {

GroupedGemmOperationProfiler::GroupedGemmOperationProfiler(Options const& options)
    : OperationProfiler(
        options,
        library::OperationKind::kGroupedGemm,
        {{ArgumentTypeID::kEnumerated,
          {"gemm_kind"},
          "Variant of GEMM (universal, gemm, planar_complex, planar_complex_array)"},
         {ArgumentTypeID::kInteger,
          {"m", "problem-size::m"},
          "M dimension of the GEMM problem space (for all groups)"},
         {ArgumentTypeID::kInteger,
          {"n", "problem-size::n"},
          "N dimension of the GEMM problem space (for all groups)"},
         {ArgumentTypeID::kInteger,
          {"k", "problem-size::k"},
          "K dimension of the GEMM problem space (for all groups)"},
         {ArgumentTypeID::kInteger,
          {"num_groups"},
          "If m,n,k are specified, run a grouped GEMM with this number of groups, where each GEMM "
          "uses the same m,n,k values."},
         {ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
         {ArgumentTypeID::kTensor, {"B"}, "Tensor storing the B operand"},
         {ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
         {ArgumentTypeID::kTensor, {"D"}, "Tensor storing the D output"},
         {ArgumentTypeID::kScalar,
          {"alpha", "epilogue::alpha"},
          "Epilogue scalar alpha (applied to all GEMMs in group)."},
         {ArgumentTypeID::kScalar,
          {"beta", "epilogue::beta"},
          "Epilogue scalar beta (applied to all GEMMs in group)."},
         {ArgumentTypeID::kScalar,
          {"problem-sizes"},
          "MxNxK Problem sizes for the grouped GEMM, where a group is enclosed by `[]`. E.g. "
          "--problem-sizes='[m1xn1xk1][m2xn2xk2]'"},
         {ArgumentTypeID::kScalar,
          {"problem-sizes-file"},
          "File containing grouped GEMM problem sizes, where each line represents a group whose "
          "GEMM dimensions are 'mxnxk'."}},
        {library::Provider::kReferenceDevice}) {

  description_ = "      Grouped matrix-matrix product. D[g] = alpha[g] * A[g] * B[g] + beta[g] * "
                 "C[g] for g in [0, num_groups)";
}

GroupedGemmOperationProfiler::~GroupedGemmOperationProfiler() {}

void GroupedGemmOperationProfiler::print_usage(std::ostream& out) const {
  OperationProfiler::print_usage(out);
}

void GroupedGemmOperationProfiler::print_examples(std::ostream& out) const {

  out
    << "\nExamples:\n\n"
    << "Profile a particular problem size (explicit shapes):\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --problem-sizes='[1024x1024x128][16x8x8]'\n\n"

    << "Profile a particular problem size (same M, N, K for all groups):\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --m=16 --n=32 --k=64 --num_groups=8'\n\n"

    << "Profile a particular problem size from a file:\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --problem-sizes-file=shapes.txt\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --problem-sizes='[8x8x8],[16x8x16][32x32x32]' "
       "--beta=0,1,2.5\n\n"

    << "Schmoo over accumulator types:\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --accumulator-type=f16,f32\n\n"

    << "Run when A is f16 with column-major and B is any datatype with row-major (For column "
       "major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --A=f16:column --B=*:row\n\n"

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=GroupedGemm --dist=sequential,start:0,delta:1\n\n"

    << "Test your changes to gemm kernels with a quick functional test and save results in "
       "functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=Gemm \\ \n"
    << "   --problem-sizes='[8x8x8][5x10x5],[16x8x16][32x32x32]' \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

Status GroupedGemmOperationProfiler::GroupedGemmProblem::parse(
  library::GemmDescription const& operation_desc,
  ProblemSpace const& problem_space,
  ProblemSpace::Problem const& problem) {

  this->mode = library::GemmUniversalMode::kGrouped;

  std::bitset<3> args_exist;
  std::string problem_sizes_str;
  args_exist[0] = arg_as_string(problem_sizes_str, "problem-sizes", problem_space, problem);
  int m, n, k;
  args_exist[1] = arg_as_int(m, "m", problem_space, problem) &&
                  arg_as_int(n, "n", problem_space, problem) &&
                  arg_as_int(k, "k", problem_space, problem);
  std::string problem_file;
  args_exist[2] = arg_as_string(problem_file, "problem-sizes-file", problem_space, problem);

  if (args_exist.count() == 0) {
    int num_groups = 8;
    problem_sizes.resize(num_groups);
    problem_sizes_3x.resize(num_groups);
    int m0 = 16;
    int n0 = 32;
    int k0 = 64;
    for (int i = 0; i < num_groups; i++) {
      auto m = m0 * (i + 1);
      auto n = n0 * (i + 1);
      auto k = k0 * (i + 1);
      problem_sizes[i] = {m, n, k};
      problem_sizes_3x[i] = {m, n, k};
    }
  }
  else if (args_exist.count() > 1) {
    std::cerr
      << "Exactly one of --problem-sizes, --problem-sizes-file, or --m --n --k may be specified.\n";
    return Status::kErrorInvalidProblem;
  }
  // --problem-sizes path
  else if (args_exist[0]) {
    auto problems = parseProblemSizes(problem_sizes_str);
    auto num_groups = problems.size();
    problem_sizes.resize(num_groups);
    problem_sizes_3x.resize(num_groups);
    for (size_t i = 0; i < num_groups; i++) {
      auto m = problems[i][0];
      auto n = problems[i][1];
      auto k = problems[i][2];
      problem_sizes[i] = {m, n, k};
      problem_sizes_3x[i] = {m, n, k};
    }
  }
  // m, n, k path
  else if (args_exist[1]) {
    int num_groups;
    if (!arg_as_int(num_groups, "num_groups", problem_space, problem)) {
      std::cerr << "num_groups must be specified if --m --n and --k are set.\n";
      return Status::kErrorInvalidProblem;
    }
    problem_sizes.resize(num_groups);
    problem_sizes_3x.resize(num_groups);
    for (int i = 0; i < num_groups; i++) {
      problem_sizes[i] = {m, n, k};
      problem_sizes_3x[i] = {m, n, k};
    }
  }
  // --problem-sizes-file path
  else if (args_exist[2]) {
    std::ifstream file(problem_file);
    if (!file.good()) {
      throw std::runtime_error("Failed to open file: " + problem_file);
    }

    for (std::string line; std::getline(file, line);) {
      std::istringstream iss(line);

      int m, n, k;
      char sep1, sep2;
      std::string remaining;

      if (iss >> m >> sep1 >> n >> sep2 >> k && sep1 == 'x' && sep2 == 'x' && !(iss >> remaining)) {
        problem_sizes.emplace_back(m, n, k);
        problem_sizes_3x.emplace_back(m, n, k);
      } else {
        throw std::runtime_error(
          "Invalid format in line: " + line + ". Each line in file expected to be 'mxnxk'.");
      }
    }
  }

  if (!arg_as_int(this->cluster_m, "cluster_m", problem_space, problem)) {
    // default value
    this->cluster_m = 1;
  }

  if (!arg_as_int(this->cluster_n, "cluster_n", problem_space, problem)) {
    // default value
    this->cluster_n = 1;
  }

  if (!arg_as_int(this->cluster_k, "cluster_k", problem_space, problem)) {
    // default value
    this->cluster_k = 1;
  }

  if (!arg_as_int(this->cluster_m_fallback, "cluster_m_fallback", problem_space, problem)) {
    // default value
    this->cluster_m_fallback = 0;
  }

  if (!arg_as_int(this->cluster_n_fallback, "cluster_n_fallback", problem_space, problem)) {
    // default value
    this->cluster_n_fallback = 0;
  }

  if (!arg_as_int(this->cluster_k_fallback, "cluster_k_fallback", problem_space, problem)) {
    // default value
    this->cluster_k_fallback = 0;
  }

  this->mode = library::GemmUniversalMode::kGrouped;

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.B, "B", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.D, "D", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
        this->alpha,
        operation_desc.element_epilogue,
        "alpha",
        problem_space,
        problem)) {

    if (!cast_from_double(this->alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }

  if (!arg_as_scalar(this->beta, operation_desc.element_epilogue, "beta", problem_space, problem)) {

    if (!cast_from_double(this->beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }

  auto num_groups = problem_sizes.size();
  this->lda.resize(num_groups);
  this->ldb.resize(num_groups);
  this->ldc.resize(num_groups);
  for (size_t group_idx = 0; group_idx < num_groups; group_idx++) {
    this->lda[group_idx] = DeviceAllocation::get_packed_layout(
                             operation_desc.A.layout,
                             {int(this->m(group_idx)), int(this->k(group_idx))})
                             .front();

    this->ldb[group_idx] = DeviceAllocation::get_packed_layout(
                             operation_desc.B.layout,
                             {int(this->k(group_idx)), int(this->n(group_idx))})
                             .front();

    this->ldc[group_idx] = DeviceAllocation::get_packed_layout(
                             operation_desc.C.layout,
                             {int(this->m(group_idx)), int(this->n(group_idx))})
                             .front();
  }

  return Status::kSuccess;
}

/// Total number of bytes loaded
int64_t GroupedGemmOperationProfiler::GroupedGemmProblem::bytes(
  library::GemmDescription const& operation_desc) const {
  // Input bytes read and Output bytes written for the gemm problem
  int64_t bytes = 0;
  for (size_t group_idx = 0, num_groups = problem_sizes.size(); group_idx < num_groups;
       group_idx++) {

    bytes +=
      int64_t(library::sizeof_bits(operation_desc.A.element) * m(group_idx) / 8) * k(group_idx) +
      int64_t(library::sizeof_bits(operation_desc.B.element) * n(group_idx) / 8) * k(group_idx) +
      int64_t(library::sizeof_bits(operation_desc.C.element) * m(group_idx) / 8) * n(group_idx);

    // Set is_beta_zero true if beta is zero
    bool is_beta_zero = std::all_of(beta.begin(), beta.end(), [](uint8_t i) { return i == 0; });
    // Output bytes read for the gemm problem for non-zero beta values
    if (!is_beta_zero) {
      bytes +=
        int64_t(library::sizeof_bits(operation_desc.C.element) * m(group_idx) / 8) * n(group_idx);
    }
  }

  return bytes;
}

/// Total number of flops computed
int64_t GroupedGemmOperationProfiler::GroupedGemmProblem::flops(
  library::GemmDescription const& operation_desc) const {
  int64_t flops_ = 0;
  for (size_t group_idx = 0, num_groups = problem_sizes.size(); group_idx < num_groups;
       group_idx++) {
    flops_ +=
      (int64_t(m(group_idx)) * n(group_idx) * k(group_idx) + m(group_idx) * n(group_idx)) * 2;
  }

  // complex-valued support
  switch (operation_desc.tile_description.math_instruction.math_operation) {
  case library::MathOperationID::kMultiplyAddComplex:
  case library::MathOperationID::kMultiplyAddComplexFastF32:
    flops_ *= 4;
    break;
  case library::MathOperationID::kMultiplyAddGaussianComplex:
    flops_ *= 3;
    break;

  default:
    break;
  }

  return flops_;
}

/// Initializes a performance result
void GroupedGemmOperationProfiler::GroupedGemmProblem::initialize_result(
  PerformanceResult& result,
  library::GemmDescription const& operation_desc,
  ProblemSpace const& problem_space) {

  result.arguments.resize(problem_space.rank());

  set_argument(result, "gemm_kind", problem_space, library::to_string(operation_desc.gemm_kind));

  set_argument(
    result,
    "A",
    problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" +
      library::to_string(operation_desc.A.layout));

  set_argument(
    result,
    "B",
    problem_space,
    std::string(library::to_string(operation_desc.B.element)) + ":" +
      library::to_string(operation_desc.B.layout));

  set_argument(
    result,
    "C",
    problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" +
      library::to_string(operation_desc.C.layout));

  set_argument(
    result,
    "D",
    problem_space,
    std::string(library::to_string(operation_desc.D.element)) + ":" +
      library::to_string(operation_desc.D.layout));

  {
    std::stringstream ss;
    ss << "'";
    for (auto const& problem_size : problem_sizes) {
      ss << "[";
      auto m = problem_size[0];
      auto n = problem_size[1];
      auto k = problem_size[2];
      ss << m << "x" << n << "x" << k;
      ss << "]";
    }
    ss << "'";
    set_argument(result, "problem-sizes", problem_space, ss.str());
  }

  set_argument(result, "cluster_m", problem_space, cluster_m);
  set_argument(result, "cluster_n", problem_space, cluster_n);
  set_argument(result, "cluster_k", problem_space, cluster_k);
  set_argument(result, "cluster_m_fallback", problem_space, cluster_m_fallback);
  set_argument(result, "cluster_n_fallback", problem_space, cluster_n_fallback);
  set_argument(result, "cluster_k_fallback", problem_space, cluster_k_fallback);

  set_argument(
    result,
    "alpha",
    problem_space,
    library::lexical_cast(alpha, operation_desc.element_epilogue));

  set_argument(
    result,
    "beta",
    problem_space,
    library::lexical_cast(beta, operation_desc.element_epilogue));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Extracts the problem dimensions
Status GroupedGemmOperationProfiler::initialize_configuration(
  Options const& options,
  PerformanceReport& report,
  DeviceContext& device_context,
  library::Operation const* operation,
  ProblemSpace const& problem_space,
  ProblemSpace::Problem const& problem) {

  library::GemmDescription const& operation_desc =
    static_cast<library::GemmDescription const&>(operation->description());

  if (operation_desc.gemm_kind != library::GemmKind::kGrouped) {
    return Status::kErrorInvalidProblem;
  }

  Status status = problem_.parse(operation_desc, problem_space, problem);
  if (status != Status::kSuccess) {
    return status;
  }

  auto num_groups = problem_.problem_sizes.size();
  gemm_workspace_.configuration.problem_count = num_groups;
  gemm_workspace_.configuration.lda = problem_.lda.data();
  gemm_workspace_.configuration.ldb = problem_.ldb.data();
  gemm_workspace_.configuration.ldc = problem_.ldc.data();

  initialize_result_(this->model_result_, options, operation_desc, problem_space);

  return status;
}

/// Initializes the performance result
void GroupedGemmOperationProfiler::initialize_result_(
  PerformanceResult& result,
  Options const& options,
  library::GemmDescription const& operation_desc,
  ProblemSpace const& problem_space) {

  result.provider = library::Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;

  problem_.initialize_result(result, operation_desc, problem_space);

  OperationProfiler::initialize_result_(result, operation_desc, problem_space);

  result.bytes = problem_.bytes(operation_desc);
  result.flops = problem_.flops(operation_desc);
  result.runtime = 0;

}

/// Initializes workspace
Status GroupedGemmOperationProfiler::initialize_workspace(
  Options const& options,
  PerformanceReport& report,
  DeviceContext& device_context,
  library::Operation const* operation,
  ProblemSpace const& problem_space,
  ProblemSpace::Problem const& problem) {

  if (options.device.devices.size() != 1) {
    throw std::runtime_error("This operation profiler only supports a single "
                             "device.");
  }

  hipError_t result;
  result = hipSetDevice(options.device.device_id(0));
  if (result != hipSuccess) {
    throw std::runtime_error("hipSetDevice() failed.");
  }

  library::Operation const* underlying_operation = operation;
  library::GemmDescription const& operation_desc =
    static_cast<library::GemmDescription const&>(operation->description());

  // Compute the number of copies of the problem to avoid L2 camping.
  if (!options.profiling.workspace_count) {
    int64_t bytes = problem_.bytes(operation_desc);
    if (bytes < 3 * int64_t(options.device.properties[0].l2CacheSize)) {
      gemm_workspace_.problem_count =
        1 + int((3 * int64_t(options.device.properties[0].l2CacheSize)) / bytes);
    }
    else {
      gemm_workspace_.problem_count = 1;
    }
  }
  else {
    gemm_workspace_.problem_count = options.profiling.workspace_count;
  }

  bool allocate_device_tensors = options.execution_mode != ExecutionMode::kDryRun;
  if (allocate_device_tensors) {
    size_t num_groups = problem_.problem_sizes.size();
    // input data
    gemm_workspace_.A_ptr_array_host.resize(num_groups);
    gemm_workspace_.B_ptr_array_host.resize(num_groups);
    gemm_workspace_.C_ptr_array_host.resize(num_groups);
    gemm_workspace_.D_ptr_array_host.resize(num_groups);
    static_assert(sizeof(void*) == 8); // allocating blocks for pointers, so verify pointer size
    // ldx
    gemm_workspace_.lda_array_device =
      device_context
        .allocate_block(options, "lda_array", library::NumericTypeID::kS64, num_groups, 0);
    gemm_workspace_.ldb_array_device =
      device_context
        .allocate_block(options, "ldb_array", library::NumericTypeID::kS64, num_groups, 0);
    gemm_workspace_.ldc_array_device =
      device_context
        .allocate_block(options, "ldc_array", library::NumericTypeID::kS64, num_groups, 0);
    gemm_workspace_.lda_array_device->copy_from_host(problem_.lda.data());
    gemm_workspace_.ldb_array_device->copy_from_host(problem_.ldb.data());
    gemm_workspace_.ldc_array_device->copy_from_host(problem_.ldc.data());
    // problem sizes
    gemm_workspace_.problem_sizes_array_device = device_context.allocate_block(
      options,
      "problem_sizes_array",
      library::NumericTypeID::kU8,
      num_groups * sizeof(gemm::GemmCoord),
      0);
    gemm_workspace_.problem_sizes_array_device->copy_from_host(problem_.problem_sizes.data());

    gemm_workspace_.problem_sizes_3x_array_device = device_context.allocate_block(
      options,
      "problem_sizes_array_3x",
      library::NumericTypeID::kU8,
      num_groups * sizeof(cute::Shape<int, int, int>),
      0);
    gemm_workspace_.problem_sizes_3x_array_device->copy_from_host(problem_.problem_sizes_3x.data());

    // reference
    gemm_workspace_.reference_ptr_array_host.resize(num_groups);

    int seed_shift = 0;
    for (size_t group_idx = 0; group_idx < num_groups; group_idx++) {
      auto group_str = std::to_string(group_idx);
      gemm_workspace_.A_ptr_array_host[group_idx] = device_context.allocate_and_initialize_tensor(
        options,
        "A_" + group_str,
        operation_desc.A.element,
        operation_desc.A.layout,
        {int(problem_.m(group_idx)), int(problem_.k(group_idx))},
        {int(problem_.lda[group_idx])},
        gemm_workspace_.problem_count,
        seed_shift++,
        0);
      gemm_workspace_.B_ptr_array_host[group_idx] = device_context.allocate_and_initialize_tensor(
        options,
        "B_" + group_str,
        operation_desc.B.element,
        operation_desc.B.layout,
        {int(problem_.k(group_idx)), int(problem_.n(group_idx))},
        {int(problem_.ldb[group_idx])},
        gemm_workspace_.problem_count,
        seed_shift++,
        0);
      gemm_workspace_.C_ptr_array_host[group_idx] = device_context.allocate_and_initialize_tensor(
        options,
        "C_" + group_str,
        operation_desc.C.element,
        operation_desc.C.layout,
        {int(problem_.m(group_idx)), int(problem_.n(group_idx))},
        {int(problem_.ldc[group_idx])},
        gemm_workspace_.problem_count,
        seed_shift++,
        0);
      gemm_workspace_.D_ptr_array_host[group_idx] = device_context.allocate_tensor(
        options,
        "D_" + group_str,
        operation_desc.D.element,
        operation_desc.D.layout,
        {int(problem_.m(group_idx)), int(problem_.n(group_idx))},
        {int(problem_.ldc[group_idx])},
        gemm_workspace_.problem_count,
        0);

      gemm_workspace_.reference_ptr_array_host[group_idx] = device_context.allocate_tensor(
        options,
        "Reference_" + group_str,
        operation_desc.D.element,
        operation_desc.D.layout,
        {int(problem_.m(group_idx)), int(problem_.n(group_idx))},
        {int(problem_.ldc[group_idx])},
        gemm_workspace_.problem_count,
        0);
    }

    // takes the allocated tensors and initializes an array of pointers per problem in the workspace
    auto create_dev_ptr_array_all_workspace = [&](
                                                std::vector<DeviceAllocation*>& dev_ptr_arrays,
                                                std::vector<DeviceAllocation*> const& input,
                                                std::string const& id) {
      auto num_workspaces = gemm_workspace_.problem_count;
      dev_ptr_arrays.resize(num_workspaces);
      // note "problem_count" here refers to input/output count for L2 cycling
      for (int i = 0; i < gemm_workspace_.problem_count; i++) {
        std::string name = id + "_ptr_array_workspace" + std::to_string(i);
        dev_ptr_arrays[i] =
          device_context.allocate_block(options, name, library::NumericTypeID::kU64, num_groups, 0);
        std::vector<void*> group_ptrs(num_groups);
        for (size_t group_idx = 0; group_idx < num_groups; group_idx++) {
          group_ptrs[group_idx] = input[group_idx]->batch_data(i);
        }
        dev_ptr_arrays[i]->copy_from_host(group_ptrs.data());
      }
    };
    create_dev_ptr_array_all_workspace(
      gemm_workspace_.A_ptr_array_device,
      gemm_workspace_.A_ptr_array_host,
      "A");
    create_dev_ptr_array_all_workspace(
      gemm_workspace_.B_ptr_array_device,
      gemm_workspace_.B_ptr_array_host,
      "B");
    create_dev_ptr_array_all_workspace(
      gemm_workspace_.C_ptr_array_device,
      gemm_workspace_.C_ptr_array_host,
      "C");
    create_dev_ptr_array_all_workspace(
      gemm_workspace_.D_ptr_array_device,
      gemm_workspace_.D_ptr_array_host,
      "D");
  }
  init_arguments(options);

  //
  // Initialize the CUTLASS operation
  //
  Status status = Status::kSuccess;
  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    if (options.execution_mode != ExecutionMode::kDryRun) {
      uint64_t workspace_size =
        underlying_operation->get_host_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.host_workspace.resize(workspace_size, 0);

      workspace_size = underlying_operation->get_device_workspace_size(
        &gemm_workspace_.configuration,
        &gemm_workspace_.arguments);
      gemm_workspace_.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

      status = underlying_operation->initialize(
        &gemm_workspace_.configuration,
        gemm_workspace_.host_workspace.data(),
        gemm_workspace_.device_workspace.data());
      if (status != Status::kSuccess) {
        return status;
      }

      status = underlying_operation->can_implement(
        &gemm_workspace_.configuration,
        &gemm_workspace_.arguments);
      if (status != Status::kSuccess) {
        return status;
      }
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //
    results_.push_back(model_result_);
    results_.back().provider = library::Provider::kCUTLASS;
    results_.back().op_kind = library::OperationKind::kGroupedGemm;
    results_.back().disposition = Disposition::kNotRun;

    for (auto provider : verification_providers_) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
    }
  }
  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GroupedGemmOperationProfiler::verify_cutlass(
  Options const& options,
  PerformanceReport& report,
  DeviceContext& device_context,
  library::Operation const* operation,
  ProblemSpace const& problem_space,
  ProblemSpace::Problem const& problem) {

  if (!options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  init_arguments(options);

  library::Operation const* underlying_operation = operation;
  results_.back().status = underlying_operation->run(
    &gemm_workspace_.arguments,
    gemm_workspace_.host_workspace.data(),
    gemm_workspace_.device_workspace.data());

  if (results_.back().status != Status::kSuccess) {
    results_.back().disposition = Disposition::kFailed;
    throw "failed";
    return false;
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // CUTLASS op ran the but not yet verified against any verification provider
  results_.back().disposition = Disposition::kNotVerified;

  //
  // Run verification providers
  //

  if (options.verification.enabled) {

#if CUTLASS_ENABLE_CUBLAS
    if (options.verification.provider_enabled(library::Provider::kCUBLAS)) {
      // set verification map for cublas to not supported
      results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotSupported;
    }
#endif // #if CUTLASS_ENABLE_CUBLAS

    library::GemmDescription const& gemm_desc =
      static_cast<library::GemmDescription const&>(operation->description());

    bool verification_status = verify_with_reference_(
      options,
      report,
      device_context,
      operation,
      problem_space,
      problem,
      gemm_desc.A.element,
      gemm_desc.B.element);

    // Update disposition to worst case verification outcome among all
    // verification providers which are supported
    bool is_any_verification_run_passed = false;
    for (auto& m : results_.back().verification_map) {
      if (m.second == Disposition::kFailed || m.second == Disposition::kIncorrect) {
        results_.back().disposition = m.second;
        return true;
      }
      if (!is_any_verification_run_passed && m.second == Disposition::kPassed) {
        is_any_verification_run_passed = true;
      }
    }

    if (is_any_verification_run_passed) {
      results_.back().disposition = Disposition::kPassed;
    }
  }

  // if verification.required is set, then return success iff at least one ref-check was run
  if (options.verification.required) {
    bool did_any_verification_run = false;
    for (auto provider : options.verification.providers) {
      did_any_verification_run |=
        (Disposition::kNotRun != results_.back().verification_map[provider]);
    }

    if (not did_any_verification_run) {
      results_.back().status = Status::kErrorNotSupported;
      return false;
    }
  }

  // Return true means continue profiling
  return true;
}

/// Verifies CUTLASS against host and device references
bool GroupedGemmOperationProfiler::verify_with_reference_(
  Options const& options,
  PerformanceReport& report,
  DeviceContext& device_context,
  library::Operation const* operation,
  ProblemSpace const& problem_space,
  ProblemSpace::Problem const& problem,
  cutlass::library::NumericTypeID element_A,
  cutlass::library::NumericTypeID element_B) {
  library::GemmDescription const& gemm_desc =
    static_cast<library::GemmDescription const&>(operation->description());

  for (auto provider : options.verification.providers) {

    // Skip providers that are not enabled
    if (!options.verification.provider_enabled(provider)) {
      continue;
    }

    auto status = Status::kSuccess;
    auto disposition = Disposition::kFailed;
    for (size_t group_idx = 0, num_groups = problem_.problem_sizes.size(); group_idx < num_groups;
         group_idx++) {
      void* ptr_A = gemm_workspace_.A_ptr_array_host[group_idx]->data();
      void* ptr_B = gemm_workspace_.B_ptr_array_host[group_idx]->data();
      void* ptr_C = gemm_workspace_.C_ptr_array_host[group_idx]->data();
      void* ptr_D = gemm_workspace_.reference_ptr_array_host[group_idx]->data();

      // To support the host-side reference, conditionally allocate and
      // copy tensors to host memory.
      std::vector<uint8_t> host_data_A;
      std::vector<uint8_t> host_data_B;
      std::vector<uint8_t> host_data_C;
      std::vector<uint8_t> host_data_D;

      if (provider == library::Provider::kReferenceHost) {
        host_data_A.resize(gemm_workspace_.A_ptr_array_host[group_idx]->bytes());
        ptr_A = host_data_A.data();
        gemm_workspace_.A_ptr_array_host[group_idx]->copy_to_host(
          ptr_A); // this is copying all the data for L2 busting as well

        host_data_B.resize(gemm_workspace_.B_ptr_array_host[group_idx]->bytes());
        ptr_B = host_data_B.data();
        gemm_workspace_.B_ptr_array_host[group_idx]->copy_to_host(ptr_B);

        host_data_C.resize(gemm_workspace_.C_ptr_array_host[group_idx]->bytes());
        ptr_C = host_data_C.data();
        gemm_workspace_.C_ptr_array_host[group_idx]->copy_to_host(ptr_C);

        host_data_D.resize(gemm_workspace_.reference_ptr_array_host[group_idx]->bytes());
        ptr_D = host_data_D.data();
      }

      library::Handle handle;
      handle.set_provider(provider);

      status = handle.gemm_universal(
        library::GemmUniversalMode::kGemm,
        problem_.m(group_idx),
        problem_.n(group_idx),
        problem_.k(group_idx),
        problem_.cluster_m,
        problem_.cluster_n,
        problem_.cluster_k,
        problem_.cluster_m_fallback,
        problem_.cluster_n_fallback,
        problem_.cluster_k_fallback,
        gemm_desc.tile_description.math_instruction.element_accumulator,
        gemm_desc.element_epilogue,
        problem_.alpha.data(),
        element_A,
        gemm_desc.A.layout,
        gemm_desc.transform_A,
        ptr_A,
        int(problem_.lda[group_idx]),
        element_B,
        gemm_desc.B.layout,
        gemm_desc.transform_B,
        ptr_B,
        int(problem_.ldb[group_idx]),
        problem_.beta.data(),
        gemm_desc.C.element,
        gemm_desc.C.layout,
        ptr_C,
        int(problem_.ldc[group_idx]),
        gemm_desc.D.element,
        gemm_desc.D.layout,
        ptr_D,
        int(problem_.ldc[group_idx]),
        1,
        gemm_workspace_.A_ptr_array_host[group_idx]->batch_stride(),
        gemm_workspace_.B_ptr_array_host[group_idx]->batch_stride(),
        gemm_workspace_.C_ptr_array_host[group_idx]->batch_stride(),
        gemm_workspace_.reference_ptr_array_host[group_idx]->batch_stride());

      if (status != Status::kSuccess)
        break;

      if (provider == library::Provider::kReferenceHost) {
        gemm_workspace_.reference_ptr_array_host[group_idx]->copy_from_host(ptr_D);
      }

      disposition = compare_tensors(
        options,
        *gemm_workspace_.D_ptr_array_host[group_idx],
        *gemm_workspace_.reference_ptr_array_host[group_idx],
        gemm_workspace_.D_ptr_array_host[group_idx]->batch_stride());
      if (disposition != Disposition::kPassed)
        break;
    }
    if (status != Status::kSuccess) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
      continue;
    }
    results_.back().status = status;
    results_.back().verification_map[provider] = disposition;

    // Save workspace if incorrect
    if (
      options.verification.save_workspace == SaveWorkspace::kIncorrect &&
      results_.back().verification_map[provider] == Disposition::kIncorrect) {

      save_workspace(device_context, options, gemm_desc, library::Provider::kCUTLASS, provider);
    }
  }

  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool GroupedGemmOperationProfiler::profile(
  Options const& options,
  PerformanceReport& report,
  DeviceContext& device_context,
  library::Operation const* operation,
  ProblemSpace const& problem_space,
  ProblemSpace::Problem const& problem) {

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    results_.back().status = profile_cutlass_(
      results_.back(),
      options,
      operation,
      &gemm_workspace_.arguments,
      gemm_workspace_.host_workspace.data(),
      gemm_workspace_.device_workspace.data());
  }
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Method to profile a CUTLASS Operation
Status GroupedGemmOperationProfiler::profile_cutlass_(
  PerformanceResult& result,
  Options const& options,
  library::Operation const* operation,
  void* arguments,
  void* host_workspace,
  void* device_workspace) {

  // initialize gemm underlying operation to handle parallel reduction
  library::Operation const* underlying_operation = operation;

  auto func = [&](hipStream_t stream, int iteration) {
    // Iterate over copies of the problem in memory
    int workspace_idx = options.profiling.warmup_iterations + iteration;
    int problem_idx = (workspace_idx % gemm_workspace_.problem_count);

    gemm_workspace_.arguments.ptr_A = gemm_workspace_.A_ptr_array_device[problem_idx]->data();
    gemm_workspace_.arguments.ptr_B = gemm_workspace_.B_ptr_array_device[problem_idx]->data();
    gemm_workspace_.arguments.ptr_C = gemm_workspace_.C_ptr_array_device[problem_idx]->data();
    gemm_workspace_.arguments.ptr_D = gemm_workspace_.D_ptr_array_device[problem_idx]->data();

    return underlying_operation->run(arguments, host_workspace, device_workspace);
  };
  return profile_kernel_(result, options, func);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
