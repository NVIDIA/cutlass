/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief 
*/

#include "device_context.h"

namespace cutlass {
namespace profiler {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Allocates memory of a given type, capacity (elements), and name
DeviceAllocation *DeviceContext::allocate_block(
  std::string const &name,
  library::NumericTypeID type, 
  size_t capacity) {

  device_memory_.emplace_back(type, capacity);
  DeviceAllocation *allocation = &device_memory_.back();
  
  allocations_[name] = allocation;
  return allocation;
}

/// Allocates memory of a given type, capacity (elements), and name
DeviceAllocation *DeviceContext::allocate_tensor(
  std::string const &name,
  library::NumericTypeID type, 
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent, 
  std::vector<int> const &stride) {

  device_memory_.emplace_back(type, layout_id, extent, stride);
  DeviceAllocation *allocation = &device_memory_.back();
  
  allocations_[name] = allocation;
  return allocation;
}

/// Allocates memory of a given type, capacity (elements), and name
DeviceAllocation *DeviceContext::allocate_tensor(
  Options const &options,
  std::string const &name,
  library::NumericTypeID type, 
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent, 
  std::vector<int> const &stride) {

  DeviceAllocation *allocation = 
    allocate_tensor(name, type, layout_id, extent, stride);

  if (options.initialization.enabled) {

    if (options.initialization.provider == Provider::kReferenceDevice) {
      allocation->initialize_random_device(
        options.initialization.seed, 
        options.initialization.data_distribution); 
    }
    else if (options.initialization.provider == Provider::kReferenceHost) {
      allocation->initialize_random_host(
        options.initialization.seed, 
        options.initialization.data_distribution);  
    }
  }

  return allocation;
}

/// Clears named allocations (but does not necessarily free memory)
void DeviceContext::clear() {
  allocations_.clear();
}

/// Frees all device memory allocations
void DeviceContext::free() {
  allocations_.clear();
  device_memory_.clear();
}

/// Gets the allocation by name
DeviceAllocation &DeviceContext::at(std::string const &name) {
  return *allocations_.at(name);
}

size_t DeviceContext::size() const {
  return allocations_.size();
}

DeviceContext::AllocationMap::iterator DeviceContext::begin() {
  return allocations_.begin();
}

DeviceContext::AllocationMap::iterator DeviceContext::end() {
  return allocations_.end();
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass
