/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <cstring>

#include "cutlass/numeric_types.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/layout/tensor.h"

#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "cutlass/util/reference/host/tensor_fill.h"

#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"

#include "device_allocation.h"

namespace cutlass {
namespace profiler {

/////////////////////////////////////////////////////////////////////////////////////////////////

size_t DeviceAllocation::bytes(library::NumericTypeID type, size_t capacity) {
  return size_t(cutlass::library::sizeof_bits(type)) * capacity / 8;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Layout>
static std::vector<int> get_packed_layout_stride(std::vector<int> const &extent) {

  typename Layout::TensorCoord extent_coord;
  typename Layout::Stride stride_coord;

  if (extent.size() != size_t(Layout::kRank)) {
    throw std::runtime_error("Layout does not have same rank as extent vector.");
  }

  for (int i = 0; i < Layout::kRank; ++i) {
    extent_coord[i] = extent.at(i);
  }

  std::vector<int> stride;
  stride.resize(Layout::kStrideRank, 0);

  Layout layout = Layout::packed(extent_coord);
  stride_coord = layout.stride();

  for (int i = 0; i < Layout::kStrideRank; ++i) {
    stride.at(i) = stride_coord[i];
  }

  return stride;
}

/// Returns the stride of a packed layout
std::vector<int> DeviceAllocation::get_packed_layout(
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent) {

  std::vector<int> stride;

  switch (layout_id) {
    case library::LayoutTypeID::kColumnMajor: 
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajor>(extent);
      break;
    case library::LayoutTypeID::kRowMajor: 
      stride = get_packed_layout_stride<cutlass::layout::RowMajor>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK4:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<4>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK4:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<4>>(extent);
      break;
    case library::LayoutTypeID::kColumnMajorInterleavedK16:
      stride = get_packed_layout_stride<cutlass::layout::ColumnMajorInterleaved<16>>(extent);
      break;
    case library::LayoutTypeID::kRowMajorInterleavedK16:
      stride = get_packed_layout_stride<cutlass::layout::RowMajorInterleaved<16>>(extent);
      break;
    case library::LayoutTypeID::kTensorNCHW:
      stride = get_packed_layout_stride<cutlass::layout::TensorNCHW>(extent);
      break;
    case library::LayoutTypeID::kTensorNHWC:
      stride = get_packed_layout_stride<cutlass::layout::TensorNHWC>(extent);
      break;


    default: break;
  }

  return stride;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Template to use CUTLASS Layout functions to 
template <typename Layout>
static size_t construct_layout_(
  void *bytes,
  library::LayoutTypeID layout_id,
  std::vector<int> const &extent,
  std::vector<int> &stride) {

  if (extent.size() != Layout::kRank) {
    throw std::runtime_error(
      "Layout must have same rank as extent vector.");
  }

  if (Layout::kStrideRank && stride.empty()) {

    stride = get_packed_layout_stride<Layout>(extent);

    return construct_layout_<Layout>(
      bytes, 
      layout_id, 
      extent,
      stride);
  }
  else if (Layout::kStrideRank && stride.size() != Layout::kStrideRank) {
    throw std::runtime_error(
      "Layout requires either empty stride or stride vector matching Layout::kStrideRank");
  }

  typename Layout::Stride stride_coord;
  for (int i = 0; i < Layout::kStrideRank; ++i) {
    stride_coord[i] = stride.at(i);
  }

  typename Layout::TensorCoord extent_coord;
  for (int i = 0; i < Layout::kRank; ++i) {
    extent_coord[i] = extent.at(i);
  }

  // Construct the CUTLASS layout object from the stride object
  Layout layout(stride_coord);

  // Pack it into bytes
  if (bytes) {
    *reinterpret_cast<Layout *>(bytes) = layout; 
  }

  // Return capacity
  size_t capacity_ = layout.capacity(extent_coord);

  return capacity_;
}

/// returns the capacity needed
size_t DeviceAllocation::construct_layout(
  void *bytes,
  library::LayoutTypeID layout_id,
  std::vector<int> const &extent,
  std::vector<int> &stride) {

  switch (layout_id) {
    case library::LayoutTypeID::kColumnMajor: 
      return construct_layout_<cutlass::layout::ColumnMajor>(bytes, layout_id, extent, stride);
      
    case library::LayoutTypeID::kRowMajor: 
      return construct_layout_<cutlass::layout::RowMajor>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK4:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<4>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK4:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<4>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kColumnMajorInterleavedK16:
      return construct_layout_<cutlass::layout::ColumnMajorInterleaved<16>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kRowMajorInterleavedK16:
      return construct_layout_<cutlass::layout::RowMajorInterleaved<16>>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNCHW:
      return construct_layout_<cutlass::layout::TensorNHWC>(bytes, layout_id, extent, stride);

    case library::LayoutTypeID::kTensorNHWC:
      return construct_layout_<cutlass::layout::TensorNHWC>(bytes, layout_id, extent, stride);

    default: break;
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

DeviceAllocation::DeviceAllocation(): 
  type_(library::NumericTypeID::kInvalid), 
  capacity_(0), 
  pointer_(nullptr),
  layout_(library::LayoutTypeID::kUnknown) {

}

DeviceAllocation::DeviceAllocation(
  library::NumericTypeID type, 
  size_t capacity
):
  type_(type), capacity_(capacity), pointer_(nullptr), layout_(library::LayoutTypeID::kUnknown) {

  hipError_t result = hipMalloc((void **)&pointer_, bytes(type, capacity));

  if (result != hipSuccess) {
    type_ = library::NumericTypeID::kInvalid;
    capacity_ = 0;
    pointer_ = nullptr;
    throw std::bad_alloc();
  }
}

DeviceAllocation::DeviceAllocation(
  library::NumericTypeID type, 
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent, 
  std::vector<int> const &stride
):
  type_(type), capacity_(size_t(0)), pointer_(nullptr) {

  reset(type, layout_id, extent, stride);
}

DeviceAllocation::~DeviceAllocation() {
  if (pointer_) {
    hipFree(pointer_);
  }
}

DeviceAllocation &DeviceAllocation::reset() {
  if (pointer_) {
    hipFree(pointer_);
  }

  type_ = library::NumericTypeID::kInvalid;
  capacity_ = 0;
  pointer_ = nullptr;
  layout_ = library::LayoutTypeID::kUnknown;
  stride_.clear();
  extent_.clear();
  tensor_ref_buffer_.clear();

  return *this;
}

DeviceAllocation &DeviceAllocation::reset(library::NumericTypeID type, size_t capacity) {

  reset();

  hipError_t result = hipMalloc((void **)&pointer_, bytes(type, capacity));
  if (result != hipSuccess) {
    throw std::bad_alloc();
  }

  type_ = type;
  capacity_ = capacity;
  layout_ = library::LayoutTypeID::kUnknown;
  stride_.clear();
  extent_.clear();

  tensor_ref_buffer_.resize(sizeof(pointer_), 0);
  std::memcpy(tensor_ref_buffer_.data(), &pointer_, sizeof(pointer_));

  return *this;
}

/// Allocates memory for a given layout and tensor
DeviceAllocation &DeviceAllocation::reset(
  library::NumericTypeID type, 
  library::LayoutTypeID layout_id, 
  std::vector<int> const &extent, 
  std::vector<int> const &stride) {

  reset();

  tensor_ref_buffer_.resize(sizeof(pointer_) + (sizeof(int) * library::get_layout_stride_rank(layout_id)), 0);

  type_ = type;

  layout_ = layout_id;
  stride_ = stride;
  extent_ = extent;

  capacity_ = construct_layout(
    tensor_ref_buffer_.data() + sizeof(pointer_), 
    layout_id, 
    extent, 
    stride_);

  hipError_t result = hipMalloc((void **)&pointer_, bytes(type, capacity_));
  if (result != hipSuccess) {
    throw std::bad_alloc();
  }

  std::memcpy(tensor_ref_buffer_.data(), &pointer_, sizeof(pointer_));

  return *this;
}

bool DeviceAllocation::good() const {
  return (capacity_ && pointer_);
}

library::NumericTypeID DeviceAllocation::type() const {
  return type_;
}

void *DeviceAllocation::data() const {
  return pointer_;
}

library::LayoutTypeID DeviceAllocation::layout() const {
  return layout_;
}

std::vector<int> const & DeviceAllocation::stride() const {
  return stride_;
}

/// Gets the extent vector
std::vector<int> const & DeviceAllocation::extent() const {
  return extent_;
}

size_t DeviceAllocation::capacity() const {
  return capacity_;
}

size_t DeviceAllocation::bytes() const {
  return bytes(type_, capacity_);
}

/// Copies from an equivalent-sized tensor in device memory
void DeviceAllocation::copy_from_device(void const *ptr) {
  hipError_t result = hipMemcpy(data(), ptr, bytes(), hipMemcpyDeviceToDevice);
  if (result != hipSuccess) {
    throw std::runtime_error("Failed device-to-device copy");
  }
}

/// Copies from an equivalent-sized tensor in device memory
void DeviceAllocation::copy_from_host(void const *ptr) {
  hipError_t result = hipMemcpy(data(), ptr, bytes(), hipMemcpyHostToDevice);
  if (result != hipSuccess) {
    throw std::runtime_error("Failed device-to-device copy");
  }
}

/// Copies from an equivalent-sized tensor in device memory
void DeviceAllocation::copy_to_host(void *ptr) {
  hipError_t result = hipMemcpy(ptr, data(), bytes(), hipMemcpyDeviceToHost);
  if (result != hipSuccess) {
    throw std::runtime_error("Failed device-to-device copy");
  }
}

void DeviceAllocation::initialize_random_device(int seed, Distribution dist) {
  if (!good()) {
    throw std::runtime_error("Attempting to initialize invalid allocation.");
  }

  // Instantiate calls to CURAND here. This file takes a long time to compile for
  // this reason.

  switch (type_) {
  case library::NumericTypeID::kF16:
    cutlass::reference::device::BlockFillRandom<cutlass::half_t>(
      reinterpret_cast<cutlass::half_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF32:
    cutlass::reference::device::BlockFillRandom<float>(
      reinterpret_cast<float *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF64:
    cutlass::reference::device::BlockFillRandom<double>(
      reinterpret_cast<double *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS8:
    cutlass::reference::device::BlockFillRandom<int8_t>(
      reinterpret_cast<int8_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS16:
    cutlass::reference::device::BlockFillRandom<int16_t>(
      reinterpret_cast<int16_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS32:
    cutlass::reference::device::BlockFillRandom<int32_t>(
      reinterpret_cast<int32_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS64:
    cutlass::reference::device::BlockFillRandom<int64_t>(
      reinterpret_cast<int64_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU8:
    cutlass::reference::device::BlockFillRandom<uint8_t>(
      reinterpret_cast<uint8_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU16:
    cutlass::reference::device::BlockFillRandom<uint16_t>(
      reinterpret_cast<uint16_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU32:
    cutlass::reference::device::BlockFillRandom<uint32_t>(
      reinterpret_cast<uint32_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU64:
    cutlass::reference::device::BlockFillRandom<uint64_t>(
      reinterpret_cast<uint64_t *>(pointer_),
      capacity_,
      seed,
      dist
    );
    break;
  default: break;
  }
}


void DeviceAllocation::initialize_random_host(int seed, Distribution dist) {
  if (!good()) {
    throw std::runtime_error("Attempting to initialize invalid allocation.");
  }

  std::vector<uint8_t> host_data(bytes());

  switch (type_) {
  case library::NumericTypeID::kF16:
    cutlass::reference::host::BlockFillRandom<cutlass::half_t>(
      reinterpret_cast<cutlass::half_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF32:
    cutlass::reference::host::BlockFillRandom<float>(
      reinterpret_cast<float *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kF64:
    cutlass::reference::host::BlockFillRandom<double>(
      reinterpret_cast<double *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS8:
    cutlass::reference::host::BlockFillRandom<int8_t>(
      reinterpret_cast<int8_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS16:
    cutlass::reference::host::BlockFillRandom<int16_t>(
      reinterpret_cast<int16_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS32:
    cutlass::reference::host::BlockFillRandom<int32_t>(
      reinterpret_cast<int32_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kS64:
    cutlass::reference::host::BlockFillRandom<int64_t>(
      reinterpret_cast<int64_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU8:
    cutlass::reference::host::BlockFillRandom<uint8_t>(
      reinterpret_cast<uint8_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU16:
    cutlass::reference::host::BlockFillRandom<uint16_t>(
      reinterpret_cast<uint16_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU32:
    cutlass::reference::host::BlockFillRandom<uint32_t>(
      reinterpret_cast<uint32_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  case library::NumericTypeID::kU64:
    cutlass::reference::host::BlockFillRandom<uint64_t>(
      reinterpret_cast<uint64_t *>(host_data.data()),
      capacity_,
      seed,
      dist
    );
    break;
  default: break;
  }

  copy_from_host(host_data.data());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns true if two blocks have exactly the same value
bool DeviceAllocation::block_compare_equal(
  library::NumericTypeID numeric_type, 
  void const *ptr_A, 
  void const *ptr_B, 
  size_t capacity) {

  switch (numeric_type) {
  case library::NumericTypeID::kF16:
    return reference::device::BlockCompareEqual<half_t>(
      reinterpret_cast<half_t const *>(ptr_A), 
      reinterpret_cast<half_t const *>(ptr_B), 
      capacity);
    
  case library::NumericTypeID::kF32:
    return reference::device::BlockCompareEqual<float>(
      reinterpret_cast<float const *>(ptr_A), 
      reinterpret_cast<float const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kF64:
    return reference::device::BlockCompareEqual<double>(
      reinterpret_cast<double const *>(ptr_A), 
      reinterpret_cast<double const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS8:
    return reference::device::BlockCompareEqual<int8_t>(
      reinterpret_cast<int8_t const *>(ptr_A), 
      reinterpret_cast<int8_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS16:
    return reference::device::BlockCompareEqual<int16_t>(
      reinterpret_cast<int16_t const *>(ptr_A), 
      reinterpret_cast<int16_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS32:
    return reference::device::BlockCompareEqual<int32_t>(
      reinterpret_cast<int32_t const *>(ptr_A), 
      reinterpret_cast<int32_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kS64:
    return reference::device::BlockCompareEqual<int64_t>(
      reinterpret_cast<int64_t const *>(ptr_A), 
      reinterpret_cast<int64_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU8:
    return reference::device::BlockCompareEqual<uint8_t>(
      reinterpret_cast<uint8_t const *>(ptr_A), 
      reinterpret_cast<uint8_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU16:
    return reference::device::BlockCompareEqual<uint16_t>(
      reinterpret_cast<uint16_t const *>(ptr_A), 
      reinterpret_cast<uint16_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU32:
    return reference::device::BlockCompareEqual<uint32_t>(
      reinterpret_cast<uint32_t const *>(ptr_A), 
      reinterpret_cast<uint32_t const *>(ptr_B), 
      capacity);

  case library::NumericTypeID::kU64:
    return reference::device::BlockCompareEqual<uint64_t>(
      reinterpret_cast<uint64_t const *>(ptr_A), 
      reinterpret_cast<uint64_t const *>(ptr_B), 
      capacity);

  default:
    throw std::runtime_error("Unsupported numeric type");
  }
}

/// Returns true if two blocks have approximately the same value
bool DeviceAllocation::block_compare_relatively_equal(
  library::NumericTypeID numeric_type, 
  void const *ptr_A, 
  void const *ptr_B, 
  size_t capacity,
  double epsilon,
  double nonzero_floor) {

  switch (numeric_type) {
  case library::NumericTypeID::kF16:
    return reference::device::BlockCompareRelativelyEqual<half_t>(
      reinterpret_cast<half_t const *>(ptr_A), 
      reinterpret_cast<half_t const *>(ptr_B),
      capacity, 
      static_cast<half_t>(epsilon), 
      static_cast<half_t>(nonzero_floor));
    
  case library::NumericTypeID::kF32:
    return reference::device::BlockCompareRelativelyEqual<float>(
      reinterpret_cast<float const *>(ptr_A), 
      reinterpret_cast<float const *>(ptr_B),
      capacity, 
      static_cast<float>(epsilon), 
      static_cast<float>(nonzero_floor));

  case library::NumericTypeID::kF64:
    return reference::device::BlockCompareRelativelyEqual<double>(
      reinterpret_cast<double const *>(ptr_A), 
      reinterpret_cast<double const *>(ptr_B),
      capacity, 
      static_cast<double>(epsilon), 
      static_cast<double>(nonzero_floor));

  case library::NumericTypeID::kS8:
    return reference::device::BlockCompareRelativelyEqual<int8_t>(
      reinterpret_cast<int8_t const *>(ptr_A), 
      reinterpret_cast<int8_t const *>(ptr_B),
      capacity, 
      static_cast<int8_t>(epsilon), 
      static_cast<int8_t>(nonzero_floor));

  case library::NumericTypeID::kS16:
    return reference::device::BlockCompareRelativelyEqual<int16_t>(
      reinterpret_cast<int16_t const *>(ptr_A), 
      reinterpret_cast<int16_t const *>(ptr_B),
      capacity, 
      static_cast<int16_t>(epsilon), 
      static_cast<int16_t>(nonzero_floor));

  case library::NumericTypeID::kS32:
    return reference::device::BlockCompareRelativelyEqual<int32_t>(
      reinterpret_cast<int32_t const *>(ptr_A), 
      reinterpret_cast<int32_t const *>(ptr_B),
      capacity, 
      static_cast<int32_t>(epsilon), 
      static_cast<int32_t>(nonzero_floor));

  case library::NumericTypeID::kS64:
    return reference::device::BlockCompareRelativelyEqual<int64_t>(
      reinterpret_cast<int64_t const *>(ptr_A), 
      reinterpret_cast<int64_t const *>(ptr_B),
      capacity, 
      static_cast<int64_t>(epsilon), 
      static_cast<int64_t>(nonzero_floor));

  case library::NumericTypeID::kU8:
    return reference::device::BlockCompareRelativelyEqual<uint8_t>(
      reinterpret_cast<uint8_t const *>(ptr_A), 
      reinterpret_cast<uint8_t const *>(ptr_B),
      capacity, 
      static_cast<uint8_t>(epsilon), 
      static_cast<uint8_t>(nonzero_floor));

  case library::NumericTypeID::kU16:
    return reference::device::BlockCompareRelativelyEqual<uint16_t>(
      reinterpret_cast<uint16_t const *>(ptr_A), 
      reinterpret_cast<uint16_t const *>(ptr_B),
      capacity, 
      static_cast<uint16_t>(epsilon), 
      static_cast<uint16_t>(nonzero_floor));

  case library::NumericTypeID::kU32:
    return reference::device::BlockCompareRelativelyEqual<uint32_t>(
      reinterpret_cast<uint32_t const *>(ptr_A), 
      reinterpret_cast<uint32_t const *>(ptr_B),
      capacity, 
      static_cast<uint32_t>(epsilon), 
      static_cast<uint32_t>(nonzero_floor));

  case library::NumericTypeID::kU64:
    return reference::device::BlockCompareRelativelyEqual<uint64_t>(
      reinterpret_cast<uint64_t const *>(ptr_A), 
      reinterpret_cast<uint64_t const *>(ptr_B),
      capacity, 
      static_cast<uint64_t>(epsilon), 
      static_cast<uint64_t>(nonzero_floor));

  default:
    throw std::runtime_error("Unsupported numeric type");
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Permits copying dynamic vectors into static-length vectors 
template <typename TensorCoord, int Rank>
struct vector_to_coord {
  
  vector_to_coord(TensorCoord &coord, std::vector<int> const &vec) {

    coord[Rank - 1] = vec.at(Rank - 1);
    
    if (Rank > 1) {
      vector_to_coord<TensorCoord, Rank - 1>(coord, vec);
    }
  }
};

/// Permits copying dynamic vectors into static-length vectors 
template <typename TensorCoord>
struct vector_to_coord<TensorCoord, 1> {
  
  vector_to_coord(TensorCoord &coord, std::vector<int> const &vec) {

    coord[0] = vec.at(0);
  }
};

/// Permits copying dynamic vectors into static-length vectors 
template <typename TensorCoord>
struct vector_to_coord<TensorCoord, 0> {
  
  vector_to_coord(TensorCoord &coord, std::vector<int> const &vec) {

  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Element, typename Layout>
static void write_tensor_csv_static_tensor_view(
  std::ostream &out, 
  DeviceAllocation &allocation) {

  Coord<Layout::kRank> extent;
  Coord<Layout::kStrideRank> stride;

  if (allocation.extent().size() != Layout::kRank) {
    throw std::runtime_error("Allocation extent has invalid rank");
  }

  if (allocation.stride().size() != Layout::kStrideRank) {
    throw std::runtime_error("Allocation stride has invalid rank");
  }

  vector_to_coord<Coord<Layout::kRank>, Layout::kRank>(extent, allocation.extent());
  vector_to_coord<Coord<Layout::kStrideRank>, Layout::kStrideRank>(stride, allocation.stride());

  Layout layout(stride);
  HostTensor<Element, Layout> host_tensor(extent, layout, false);

  if (host_tensor.capacity() != allocation.capacity()) {
    throw std::runtime_error("Unexpected capacity to equal.");
  }

  host_tensor.copy_in_device_to_host(static_cast<Element const *>(allocation.data()), host_tensor.capacity());

  TensorViewWrite(out, host_tensor.host_view());
}

/////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
static void write_tensor_csv_static_type(
  std::ostream &out, 
  DeviceAllocation &allocation) {

  switch (allocation.layout()) {
    case library::LayoutTypeID::kRowMajor:
      write_tensor_csv_static_tensor_view<T, layout::RowMajor>(out, allocation);
      break;
    case library::LayoutTypeID::kColumnMajor:
      write_tensor_csv_static_tensor_view<T, layout::ColumnMajor>(out, allocation);
      break;
    case library::LayoutTypeID::kTensorNHWC:
      write_tensor_csv_static_tensor_view<T, layout::TensorNHWC>(out, allocation);
      break;
    default:
      throw std::runtime_error("Unhandled layout");
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Writes a tensor to csv 
void DeviceAllocation::write_tensor_csv(
  std::ostream &out) {

  switch (this->type()) {
  case library::NumericTypeID::kF16:
    write_tensor_csv_static_type<half_t>(out, *this);
    break;
    
  case library::NumericTypeID::kF32:
    write_tensor_csv_static_type<float>(out, *this);
    break;

  case library::NumericTypeID::kF64:
    write_tensor_csv_static_type<double>(out, *this);
    break;

  case library::NumericTypeID::kS8:
    write_tensor_csv_static_type<int8_t>(out, *this);
    break;

  case library::NumericTypeID::kS16:
    write_tensor_csv_static_type<int16_t>(out, *this);
    break;

  case library::NumericTypeID::kS32:
    write_tensor_csv_static_type<int32_t>(out, *this);
    break;

  case library::NumericTypeID::kS64:
    write_tensor_csv_static_type<int64_t>(out, *this);
    break;

  case library::NumericTypeID::kU8:
    write_tensor_csv_static_type<uint8_t>(out, *this);
    break;

  case library::NumericTypeID::kU16:
    write_tensor_csv_static_type<uint16_t>(out, *this);
    break;

  case library::NumericTypeID::kU32:
    write_tensor_csv_static_type<uint32_t>(out, *this);
    break;

  case library::NumericTypeID::kU64:
    write_tensor_csv_static_type<uint64_t>(out, *this);
    break;

  default:
    throw std::runtime_error("Unsupported numeric type");
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass
