#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/



#include <iostream>
#include <stdexcept>
#include <iomanip>
#include <ios>
#include <vector>

#include "cutlass/core_io.h"

#include "cutlass/profiler/cublas_helpers.h"
#include "cutlass/profiler/block_scaled_gemm_operation_profiler.h"
#include "cutlass/profiler/gpu_timer.h"
#include "cutlass/library/singleton.h"
#include "cutlass/library/library.h"
#include "cutlass/library/handle.h"

#include "cutlass/util/reference/host/gett.hpp"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {


/////////////////////////////////////////////////////////////////////////////////////////////////

/// Ctor
BlockScaledGemmOperationProfiler::BlockScaledGemmOperationProfiler(Options const &options):
  OperationProfiler(
    options,
    library::OperationKind::kBlockScaledGemm,
    {
      {ArgumentTypeID::kEnumerated, {"gemm_kind"}, "Variant of GEMM (universal, gemm, planar_complex, planar_complex_array)"},
      {ArgumentTypeID::kInteger, {"m", "problem-size::m"}, "M dimension of the GEMM problem space"},
      {ArgumentTypeID::kInteger, {"n", "problem-size::n"}, "N dimension of the GEMM problem space"},
      {ArgumentTypeID::kInteger, {"k", "problem-size::k"}, "K dimension of the GEMM problem space"},
      {ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
      {ArgumentTypeID::kTensor, {"B"}, "Tensor storing the B operand"},
      {ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
      {ArgumentTypeID::kTensor, {"D"}, "Tensor storing the D output"},
      {ArgumentTypeID::kScalar, {"alpha", "epilogue::alpha"}, "Epilogue scalar alpha"},
      {ArgumentTypeID::kScalar, {"beta", "epilogue::beta"}, "Epilogue scalar beta"},
      // TODO: Bring these back once SM100 future audits are complete
      {ArgumentTypeID::kEnumerated, {"split_k_mode", "split-k-mode"}, "Variant of split K mode(serial, parallel)"},
      {ArgumentTypeID::kInteger, {"split_k_slices", "split-k-slices"}, "Number of partitions of K dimension"},
      {ArgumentTypeID::kInteger, {"batch_count", "batch-count"}, "Number of GEMMs computed in one batch"},
      {ArgumentTypeID::kEnumerated, {"runtime_input_datatype_a", "runtime-input-datatype::a"}, "Runtime datatype (e4m3, e5m2, e3m2, e2m3, e2m1)"}, 
      {ArgumentTypeID::kEnumerated, {"runtime_input_datatype_b", "runtime-input-datatype::b"}, "Runtime datatype (e4m3, e5m2, e3m2, e2m3, e2m1)"}, 
      {ArgumentTypeID::kEnumerated, {"raster_order", "raster-order"}, "Raster order (heuristic, along_n, along_m)"},
      {ArgumentTypeID::kInteger, {"swizzle_size", "swizzle-size"}, "Size to swizzle"},
      {ArgumentTypeID::kEnumerated, {"use_pdl", "use_pdl"}, "Use PDL (true, false)"},
    },
    { library::Provider::kCUBLAS}
  ) {

  description_ = "      General matrix-matrix product. D = alpha * A*B + beta * C";
}

/// Destructor
BlockScaledGemmOperationProfiler::~BlockScaledGemmOperationProfiler() {

}

/// Prints usage statement for the math function
void BlockScaledGemmOperationProfiler::print_usage(std::ostream &out) const {
  out << "Block Scaled GEMM" << "\n\n";

  OperationProfiler::print_usage(out);
}

/// Prints examples
void BlockScaledGemmOperationProfiler::print_examples(std::ostream &out) const {

  out << "\nExamples:\n\n"
    << "Profile a particular problem size:\n"
    << "  $ cutlass_profiler --operation=block_scaled_gemm --m=1024 --n=1024 --k=128\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=block_scaled_gemm --m=1024:4096:256 --n=1024:4096:256 --k=128:8192:128 --beta=0,1,2.5\n\n"

    // TODO: Bring these back once SM100 future audits are complete
#if 0
    << "Run when A is f16 with column-major and B is any datatype with row-major (For column major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=Gemm --A=f16:column --B=*:row\n\n"

    << "Profile a particular problem size with split K and parallel reduction:\n"
    << "  $ cutlass_profiler --operation=Gemm --split_k_mode=parallel --split_k_slices=2 --m=1024 --n=1024 --k=128\n\n"
#endif

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=sequential,start:0,delta:1\n\n"

    << "Run a kernel with cta tile size of 256x128x32 and save workspace if results are incorrect (note that --cta-tile::k=32 is default cta-tile size):\n"
    << " $ cutlass_profiler --operation=Gemm --cta_m=256 --cta_n=128  --cta_k=32 --save-workspace=incorrect\n\n"

    << "Test your changes to gemm kernels with a quick functional test and save results in functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=Gemm \\ \n"
    << "   --m=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --n=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --k=8,16,32,64,128,256,288,384,504,512,520 \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#if 0
// used this for debugging
static std::string byte_string(std::vector<uint8_t> const &bytes) {
  std::stringstream ss;

  ss << "0x";

  for (size_t idx = bytes.size(); idx > 0; --idx) {
    ss << std::hex << std::setw(2) << std::setfill('0') << uint32_t(bytes.at(idx - 1));
  }

  return ss.str();
}
#endif

Status BlockScaledGemmOperationProfiler::GemmProblem::parse(
  library::BlockScaledGemmDescription const &operation_desc,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  this->mode = library::GemmUniversalMode::kGemm;

  if (!arg_as_int(this->m, "m", problem_space, problem)) {
    // default value
    this->m = 1024;
  }

  if (!arg_as_int(this->n, "n", problem_space, problem)) {
    // default value
    this->n = 1024;
  }

  if (!arg_as_int(this->k, "k", problem_space, problem)) {
    // default value
    this->k = 1024;
  }

  
  if (!arg_as_int(this->cluster_m, "cluster_m", problem_space, problem)) {
    // default value
    this->cluster_m = 1;
  }

  if (!arg_as_int(this->cluster_n, "cluster_n", problem_space, problem)) {
    // default value
    this->cluster_n = 1;
  }

  if (!arg_as_int(this->cluster_k, "cluster_k", problem_space, problem)) {
    // default value
    this->cluster_k = 1;
  }

  if (!arg_as_int(this->cluster_m_fallback, "cluster_m_fallback", problem_space, problem)) {
    // default value
    this->cluster_m_fallback = 0;
  }

  if (!arg_as_int(this->cluster_n_fallback, "cluster_n_fallback", problem_space, problem)) {
    // default value
    this->cluster_n_fallback = 0;
  }

  if (!arg_as_int(this->cluster_k_fallback, "cluster_k_fallback", problem_space, problem)) {
    // default value
    this->cluster_k_fallback = 0;
  }
  

  if (!arg_as_SplitKModeID(this->split_k_mode, "split_k_mode", problem_space, problem)) {
    // default value
    this->split_k_mode = library::SplitKMode::kSerial;
  }

  this->mode = library::GemmUniversalMode::kGemm;
  if (this->split_k_mode == library::SplitKMode::kParallel) {
    this->mode = library::GemmUniversalMode::kGemmSplitKParallel;
  }

  if (!arg_as_int(this->split_k_slices, "split_k_slices", problem_space, problem)) {
    // default value
    this->split_k_slices = 1;
  }

  // TODO: Bring these back once SM100 future audits are complete
  if (this->split_k_mode != library::SplitKMode::kSerial) {
    std::cout<<"SplitK/StreamK feature is not supported yet!";
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_bool(this->use_pdl, "use_pdl", problem_space, problem)) {
    // default value
    this->use_pdl = false;
  }

  
  if (!arg_as_RuntimeDatatype(this->runtime_input_datatype_a, "runtime_input_datatype_a", problem_space, problem)) {
    // default value
    this->runtime_input_datatype_a = cutlass::library::RuntimeDatatype::kStatic;
  }

  if (!arg_as_RuntimeDatatype(this->runtime_input_datatype_b, "runtime_input_datatype_b", problem_space, problem)) {
    // default value
    this->runtime_input_datatype_b = cutlass::library::RuntimeDatatype::kStatic;
  }
  

  if (!arg_as_int(this->batch_count, "batch_count", problem_space, problem)) {
    // default value
    this->batch_count = 1;
  } else if (this->batch_count > 1) {
    this->mode = library::GemmUniversalMode::kBatched;
  }

  if (!arg_as_int(this->swizzle_size, "swizzle_size", problem_space, problem)) {
    // default value
    this->swizzle_size = 1;
  }

  if (!arg_as_RasterOrder(this->raster_order, "raster_order", problem_space, problem)) {
    // default value
    this->raster_order = library::RasterOrder::kHeuristic;
  }

  if (this->split_k_slices > 1 && this->batch_count > 1) {
    // At least one of these must be one
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.B, "B", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.D, "D", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
    this->alpha,
    operation_desc.element_epilogue,
    "alpha",
    problem_space,
    problem)) {

    if (!cast_from_double(this->alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }

  if (!arg_as_scalar(
    this->beta,
    operation_desc.element_epilogue,
    "beta",
    problem_space,
    problem)) {

    if (!cast_from_double(this->beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }

  this->lda = DeviceAllocation::get_packed_layout(
    operation_desc.A.layout, {int(this->m), int(this->k)}).front();

  this->ldb = DeviceAllocation::get_packed_layout(
    operation_desc.B.layout, {int(this->k), int(this->n)}).front();

  this->ldc = DeviceAllocation::get_packed_layout(
    operation_desc.C.layout, {int(this->m), int(this->n)}).front();

  return Status::kSuccess;
}

/// Total number of bytes loaded
int64_t BlockScaledGemmOperationProfiler::GemmProblem::bytes(library::BlockScaledGemmDescription const &operation_desc) const {
  // Input bytes read and Output bytes written for the gemm problem
  int64_t bytes =
    int64_t(library::sizeof_bits(operation_desc.A.element) * m / 8) * k +
    int64_t(library::sizeof_bits(operation_desc.B.element) * n / 8) * k +
    int64_t(library::sizeof_bits(operation_desc.C.element) * m / 8) * n;

  // Set is_beta_zero true if beta is zero
  bool is_beta_zero = std::all_of(beta.begin(), beta.end(), [](uint8_t i) { return i==0; });

  // Output bytes read for the gemm problem for non-zero beta values
  if (!is_beta_zero) {
    bytes += int64_t(library::sizeof_bits(operation_desc.C.element) * m / 8) * n;
  }

  bytes *= batch_count;

  return bytes;
}

/// Total number of flops computed
int64_t BlockScaledGemmOperationProfiler::GemmProblem::flops(library::BlockScaledGemmDescription const &operation_desc) const {
  int64_t flops_ = (int64_t(m) * n * k + m * n) * 2 * batch_count;

  // complex-valued support
  switch (operation_desc.tile_description.math_instruction.math_operation) {
  case library::MathOperationID::kMultiplyAddComplex:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddComplexFastF32:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddGaussianComplex:
    flops_ *= 3;
    break;

  default: break;
  }

  return flops_;
}


/// Initializes a performance result
void BlockScaledGemmOperationProfiler::GemmProblem::initialize_result(
  PerformanceResult &result,
  library::BlockScaledGemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.arguments.resize(problem_space.rank());

  set_argument(result, "gemm_kind", problem_space, library::to_string(operation_desc.gemm_kind));

  set_argument(result, "A", problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" + library::to_string(operation_desc.A.layout));

  set_argument(result, "B", problem_space,
    std::string(library::to_string(operation_desc.B.element)) + ":" + library::to_string(operation_desc.B.layout));

  set_argument(result, "C", problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" + library::to_string(operation_desc.C.layout));

  set_argument(result, "D", problem_space,
    std::string(library::to_string(operation_desc.D.element)) + ":" + library::to_string(operation_desc.D.layout));

  set_argument(result, "m", problem_space, m);
  set_argument(result, "n", problem_space, n);
  set_argument(result, "k", problem_space, k);

  
  set_argument(result, "cluster_m", problem_space, cluster_m);
  set_argument(result, "cluster_n", problem_space, cluster_n);
  set_argument(result, "cluster_k", problem_space, cluster_k);
  set_argument(result, "cluster_m_fallback", problem_space, cluster_m_fallback);
  set_argument(result, "cluster_n_fallback", problem_space, cluster_n_fallback);
  set_argument(result, "cluster_k_fallback", problem_space, cluster_k_fallback);
  

  // TODO: Bring these back once SM100 future audits are complete
  set_argument(result, "split_k_mode", problem_space, library::to_string(split_k_mode));
  set_argument(result, "split_k_slices", problem_space, split_k_slices);
  set_argument(result, "batch_count", problem_space, batch_count);
  set_argument(result, "raster_order", problem_space, library::to_string(raster_order));
  set_argument(result, "swizzle_size", problem_space, swizzle_size);
  set_argument(result, "use_pdl", problem_space, library::to_string(use_pdl));

  
  set_argument(result, "runtime_input_datatype_a", problem_space, library::to_string(runtime_input_datatype_a));
  set_argument(result, "runtime_input_datatype_b", problem_space, library::to_string(runtime_input_datatype_b));
  

  set_argument(result, "alpha", problem_space,
    library::lexical_cast(alpha, operation_desc.element_epilogue));

  set_argument(result, "beta", problem_space,
    library::lexical_cast(beta, operation_desc.element_epilogue));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Extracts the problem dimensions
Status BlockScaledGemmOperationProfiler::initialize_configuration(
    Options const &options,
    PerformanceReport &report,
    DeviceContext &device_context,
    library::Operation const *operation,
    ProblemSpace const &problem_space,
    ProblemSpace::Problem const &problem) {

  library::BlockScaledGemmDescription const &operation_desc =
    static_cast<library::BlockScaledGemmDescription const &>(operation->description());

  if (operation_desc.gemm_kind != library::GemmKind::kUniversal) {
    return Status::kErrorInvalidProblem;
  }

  Status status = problem_.parse(operation_desc, problem_space, problem);

  if (status != Status::kSuccess) {
    return status;
  }

  gemm_workspace_.configuration.mode = problem_.mode;
  gemm_workspace_.configuration.problem_size.m() = int(problem_.m);
  gemm_workspace_.configuration.problem_size.n() = int(problem_.n);
  gemm_workspace_.configuration.problem_size.k() = int(problem_.k);
  
  gemm_workspace_.configuration.cluster_shape.m() = int(problem_.cluster_m);
  gemm_workspace_.configuration.cluster_shape.n() = int(problem_.cluster_n);
  gemm_workspace_.configuration.cluster_shape.k() = int(problem_.cluster_k);
  gemm_workspace_.configuration.cluster_shape_fallback.m() = int(problem_.cluster_m_fallback);
  gemm_workspace_.configuration.cluster_shape_fallback.n() = int(problem_.cluster_n_fallback);
  gemm_workspace_.configuration.cluster_shape_fallback.k() = int(problem_.cluster_k_fallback);
  
  gemm_workspace_.configuration.lda = problem_.lda;
  gemm_workspace_.configuration.ldb = problem_.ldb;
  gemm_workspace_.configuration.ldc = problem_.ldc;
  gemm_workspace_.configuration.ldd = problem_.ldc;

  if (problem_.mode == library::GemmUniversalMode::kBatched) {
    gemm_workspace_.configuration.batch_count = problem_.batch_count;
  }
  else {
    gemm_workspace_.configuration.batch_count = problem_.split_k_slices;
  }

  gemm_workspace_.arguments.problem_size.m() = int(problem_.m);
  gemm_workspace_.arguments.problem_size.n() = int(problem_.n);
  gemm_workspace_.arguments.problem_size.k() = int(problem_.k);
  gemm_workspace_.arguments.batch_count = problem_.batch_count;

  gemm_workspace_.arguments.A = nullptr;
  gemm_workspace_.arguments.B = nullptr;
  gemm_workspace_.arguments.C = nullptr;
  gemm_workspace_.arguments.D = nullptr;
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;
  gemm_workspace_.arguments.swizzle_size = problem_.swizzle_size;
  gemm_workspace_.arguments.raster_order = problem_.raster_order;
  gemm_workspace_.arguments.norm_constant = 0;                    
  gemm_workspace_.arguments.cluster_shape = {int(problem_.cluster_m), int(problem_.cluster_n), int(problem_.cluster_k)}; 
  gemm_workspace_.arguments.cluster_shape_fallback = {int(problem_.cluster_m_fallback), int(problem_.cluster_n_fallback), int(problem_.cluster_k_fallback)}; 
  gemm_workspace_.arguments.split_k_slices = problem_.split_k_slices;

  
  gemm_workspace_.arguments.runtime_input_datatype_a = problem_.runtime_input_datatype_a;
  gemm_workspace_.arguments.runtime_input_datatype_b = problem_.runtime_input_datatype_b;
  

  gemm_workspace_.arguments.use_pdl = problem_.use_pdl;

  // initialize reduction operation for parallel splitKMode
  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!initialize_reduction_configuration_(operation, problem)) {
      return Status::kErrorInternal;
    }
  }

  initialize_result_(this->model_result_, options, operation_desc, problem_space);

  return operation->can_implement(&gemm_workspace_.configuration, &gemm_workspace_.arguments);
}

/// Initializes the performance result
void BlockScaledGemmOperationProfiler::initialize_result_(
    PerformanceResult &result,
    Options const &options,
    library::BlockScaledGemmDescription const &operation_desc,
    ProblemSpace const &problem_space) {

  result.provider = library::Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;

  problem_.initialize_result(result, operation_desc, problem_space);

  OperationProfiler::initialize_result_(result, operation_desc, problem_space);

  result.bytes = problem_.bytes(operation_desc);
  result.flops = problem_.flops(operation_desc);
  result.runtime = 0;

}

/// Initialize reduction problem dimensions and library::Operation
bool BlockScaledGemmOperationProfiler::initialize_reduction_configuration_(
  library::Operation const *operation,
  ProblemSpace::Problem const &problem) {

  // TODO: Bring these back once SM100 future audits are complete
#if 1
  library::BlockScaledGemmDescription const &gemm_desc =
    static_cast<library::BlockScaledGemmDescription const&>(operation->description());

  if (!cast_from_double(problem_.alpha_one, gemm_desc.element_epilogue, 1)) {
    return false;
  }

  if (!cast_from_double(problem_.beta_zero, gemm_desc.element_epilogue, 0)) {
    return false;
  }

  /// initialize library::ReductionConfiguration
  gemm_workspace_.reduction_configuration.problem_size      = gemm::GemmCoord(int(problem_.n), int(problem_.m), int(problem_.k)).mn();
  gemm_workspace_.reduction_configuration.partitions        = int(problem_.split_k_slices);
  gemm_workspace_.reduction_configuration.partition_stride  = gemm::GemmCoord(int(problem_.n), int(problem_.m), int(problem_.k)).mn().product();
  gemm_workspace_.reduction_configuration.ldw               = problem_.ldc;
  gemm_workspace_.reduction_configuration.lds               = problem_.ldc;
  gemm_workspace_.reduction_configuration.ldd               = problem_.ldc;

  // find reduction operation
  library::ReductionFunctionalKey reduction_key(
    library::Provider::kCUTLASS,
    gemm_desc.tile_description.math_instruction.element_accumulator,    // element workspace
    gemm_desc.tile_description.math_instruction.element_accumulator,    // element accumulator
    gemm_desc.D.element,                                                // element output
    gemm_desc.element_epilogue                                          // element compute
  );

  auto reduction_it = library::Singleton::get().operation_table.reduction_operations.find(reduction_key);

  if (reduction_it == library::Singleton::get().operation_table.reduction_operations.end()) {
    return false;
  }

  // initialize reduction operation required for parallel split-k operator
  reduction_op_ = reduction_it->second;

  // reduction operation found and initialized
  return true;
#endif
  return false;
}

/// Initializes workspace
Status BlockScaledGemmOperationProfiler::initialize_workspace(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.device.devices.size() != 1) {
    throw std::runtime_error("This operation profiler only supports a single "
                             "device.");
  }

  hipError_t result;
  result = hipSetDevice(options.device.device_id(0));
  if (result != hipSuccess) {
    throw std::runtime_error("hipSetDevice() failed.");
  }

  library::Operation const* underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      return Status::kErrorNotSupported;
    }
  }

  library::BlockScaledGemmDescription const &operation_desc =
    static_cast<library::BlockScaledGemmDescription const &>(operation->description());

  // Compute the number of copies of the problem to avoid L2 camping.
  if (!options.profiling.workspace_count) {
    int64_t bytes = problem_.bytes(operation_desc);
    if (bytes < 3 * int64_t(options.device.properties[0].l2CacheSize)) {
      gemm_workspace_.problem_count =
        1 + int((3 * int64_t(options.device.properties[0].l2CacheSize)) / bytes);
    }
    else {
      gemm_workspace_.problem_count = 1;
    }
  }
  else {
    gemm_workspace_.problem_count = options.profiling.workspace_count;
  }

  bool allocate_device_tensors = options.execution_mode != ExecutionMode::kDryRun;
  if (allocate_device_tensors) {
    int seed_shift = 0;
    gemm_workspace_.A = device_context.allocate_and_initialize_tensor(
      options,
      "A",
      operation_desc.A.element,
      operation_desc.A.layout,
      {int(problem_.m), int(problem_.k)},
      {int(problem_.lda)},
      problem_.batch_count * gemm_workspace_.problem_count,
      seed_shift++,
      0 // device_index
    );

    int sfa_m     = round_up(int(problem_.m), 128);
    int sfb_n     = round_up(int(problem_.n), 128);
    int sfa_sfb_k = round_up(ceil_div(int(problem_.k), operation_desc.SFVecSize), 4);
    
    int sfd_m     = operation_desc.SFD.layout == cutlass::library::LayoutTypeID::kRowMajor ?
                      sfa_m : round_up(ceil_div(int(problem_.m), operation_desc.EpilogueSFVecSize), 4);
    int sfd_n     = operation_desc.SFD.layout == cutlass::library::LayoutTypeID::kRowMajor ?
                      round_up(ceil_div(int(problem_.n), operation_desc.EpilogueSFVecSize), 4) : sfb_n;
    

    gemm_workspace_.SFA = device_context.allocate_and_initialize_tensor(
      options,
      "SFA",
      operation_desc.SFA.element,
      operation_desc.SFA.layout,
      {sfa_m, sfa_sfb_k},
      {sfa_sfb_k},
      problem_.batch_count * gemm_workspace_.problem_count,
      seed_shift++,
      0 // device_index
    );

    gemm_workspace_.SFB = device_context.allocate_and_initialize_tensor(
      options,
      "SFB",
      operation_desc.SFB.element,
      operation_desc.SFB.layout,
      {sfb_n, sfa_sfb_k},
      {sfa_sfb_k},
      problem_.batch_count * gemm_workspace_.problem_count,
      seed_shift++,
      0 // device_index
    );

    gemm_workspace_.B = device_context.allocate_and_initialize_tensor(
      options,
      "B",
      operation_desc.B.element,
      operation_desc.B.layout,
      {int(problem_.k), int(problem_.n)},
      {int(problem_.ldb)},
      problem_.batch_count * gemm_workspace_.problem_count,
      seed_shift++,
      0 // device_index
    );

    gemm_workspace_.C = device_context.allocate_and_initialize_tensor(
      options,
      "C",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)},
      problem_.batch_count * gemm_workspace_.problem_count,
      seed_shift++,
      0 // device_index
    );

    gemm_workspace_.Computed = device_context.allocate_tensor(
      options,
      "D",
      operation_desc.D.element,
      operation_desc.D.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)},
      problem_.batch_count * gemm_workspace_.problem_count,
      0 // device_index
    );

    gemm_workspace_.Reference = device_context.allocate_tensor(
      options,
      "Reference",
      operation_desc.D.element,
      operation_desc.D.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)},
      problem_.batch_count * gemm_workspace_.problem_count,
      0 // device_index
    );
    
    gemm_workspace_.Computed_SFD = device_context.allocate_tensor(
      options,
      "SFD",
      operation_desc.SFD.element,
      operation_desc.SFD.layout,
      {sfd_m, sfd_n},
      {sfd_n},
      problem_.batch_count * gemm_workspace_.problem_count,
      0 // device_index
    );

    gemm_workspace_.Reference_SFD = device_context.allocate_tensor(
      options,
      "Reference_SFD",
      operation_desc.SFD.element,
      operation_desc.SFD.layout,
      {sfd_m, sfd_n},
      {sfd_n},
      problem_.batch_count * gemm_workspace_.problem_count,
      0 // device_index
    );

    gemm_workspace_.Norm_constant = device_context.allocate_and_initialize_tensor(
      options,
      "Norm_constant",
      operation_desc.element_epilogue,
      operation_desc.A.layout,
      {1, 1},
      {1},
      1,
      seed_shift++,
      0 // device_index
    );
    
  }

  if (options.execution_mode != ExecutionMode::kDryRun) {
    
    // ScaleFactor tensor results may have some holes and will not be touched by the kernel.
    // If we randomly fill the two tensors, these holes may encounter refcheck errors.
    if (gemm_workspace_.Computed_SFD->type() != library::NumericTypeID::kVoid) {
      if (options.initialization.provider == library::Provider::kReferenceHost) {
        gemm_workspace_.Reference_SFD->fill_host(0);
        gemm_workspace_.Computed_SFD->fill_host(0);
      }
      else {
        gemm_workspace_.Reference_SFD->fill_device(0);
        gemm_workspace_.Computed_SFD->fill_device(0);
      }
    }
    

    // NOTE: the leading non-batch strides are duplicated here for 3.0 API kernels
    gemm_workspace_.arguments.problem_size = {int(problem_.m), int(problem_.n), int(problem_.k)};
    gemm_workspace_.arguments.cluster_shape = {int(problem_.cluster_m), int(problem_.cluster_n), int(problem_.cluster_k)}; 
    gemm_workspace_.arguments.cluster_shape_fallback = {int(problem_.cluster_m_fallback), int(problem_.cluster_n_fallback), int(problem_.cluster_k_fallback)}; 
    gemm_workspace_.arguments.split_k_slices = problem_.split_k_slices;
    gemm_workspace_.arguments.batch_count = problem_.batch_count;
    gemm_workspace_.arguments.lda = problem_.lda;
    gemm_workspace_.arguments.ldb = problem_.ldb;
    gemm_workspace_.arguments.ldc = problem_.ldc;
    gemm_workspace_.arguments.ldd = problem_.ldc;
    gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
    gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
    gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.C->batch_stride();
    gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Computed->batch_stride();
    gemm_workspace_.arguments.use_pdl = problem_.use_pdl;

    /* Query device SM count to pass onto the kernel as an argument, where needed */
    gemm_workspace_.arguments.sm_count = options.device.properties[0].multiProcessorCount;
  }

  //
  // Initialize the CUTLASS operation
  //
  Status status = Status::kSuccess;

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    if (options.execution_mode != ExecutionMode::kDryRun) {
      uint64_t workspace_size = underlying_operation->get_host_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.host_workspace.resize(workspace_size, 0);

      workspace_size = underlying_operation->get_device_workspace_size(&gemm_workspace_.configuration,
                                                            &gemm_workspace_.arguments);
      gemm_workspace_.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

      status = underlying_operation->initialize(
        &gemm_workspace_.configuration,
        gemm_workspace_.host_workspace.data(),
        gemm_workspace_.device_workspace.data());
      if (status != Status::kSuccess) {
        return status;
      }

      if (problem_.split_k_mode == library::SplitKMode::kParallel) {
        workspace_size = reduction_op_->get_host_workspace_size(&gemm_workspace_.reduction_configuration);
        gemm_workspace_.reduction_host_workspace.resize(workspace_size, 0);

        status = reduction_op_->initialize(
          &gemm_workspace_.reduction_configuration,
          gemm_workspace_.reduction_host_workspace.data(),
          nullptr);

        if (status != Status::kSuccess) {
          return status;
        }
      }
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //
    results_.push_back(model_result_);
    results_.back().provider = library::Provider::kCUTLASS;
    results_.back().op_kind = library::OperationKind::kGemm;
    results_.back().disposition = Disposition::kNotRun;

    for (auto provider : verification_providers_) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
    }
  }
  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool BlockScaledGemmOperationProfiler::verify_cutlass(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (!options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  // Initialize structure containing GEMM arguments
  gemm_workspace_.arguments.A = gemm_workspace_.A->data();
  gemm_workspace_.arguments.B = gemm_workspace_.B->data();
  gemm_workspace_.arguments.SFA = gemm_workspace_.SFA->data();
  gemm_workspace_.arguments.SFB = gemm_workspace_.SFB->data();
  gemm_workspace_.arguments.C = gemm_workspace_.C->data();
  gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
  gemm_workspace_.arguments.SFD = gemm_workspace_.Computed_SFD->data();
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.norm_constant = gemm_workspace_.Norm_constant->data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;
  gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
  gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
  gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.C->batch_stride();
  gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Computed->batch_stride();

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    gemm_workspace_.arguments.D                       = gemm_workspace_.device_workspace.data();
    gemm_workspace_.arguments.alpha                   = problem_.alpha_one.data();
    gemm_workspace_.arguments.beta                    = problem_.beta_zero.data();

    gemm_workspace_.reduction_arguments.workspace     = gemm_workspace_.device_workspace.data();
    gemm_workspace_.reduction_arguments.source        = gemm_workspace_.C->data();
    gemm_workspace_.reduction_arguments.destination   = gemm_workspace_.Computed->data();
    gemm_workspace_.reduction_arguments.alpha         = problem_.alpha.data();
    gemm_workspace_.reduction_arguments.beta          = problem_.beta.data();
    gemm_workspace_.reduction_arguments.pointer_mode  = library::ScalarPointerMode::kHost;
  }

  //
  // Run the CUTLASS operation
  //

  // initialize gemm underlying operation to handle parallel reduction
  library::Operation const * underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      results_.back().disposition = Disposition::kFailed;
      return false;
    }
  }

  results_.back().status = underlying_operation->run(
    &gemm_workspace_.arguments,
    gemm_workspace_.host_workspace.data(),
    gemm_workspace_.device_workspace.data(),
    nullptr);

  if (results_.back().status != Status::kSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // Run parallel reduction kernel for parallel split_k_mode
  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    results_.back().status = reduction_op_->run(
      &gemm_workspace_.reduction_arguments,
      gemm_workspace_.reduction_host_workspace.data(),
      nullptr,
      nullptr);

    if (results_.back().status != Status::kSuccess) {
      results_.back().disposition = Disposition::kFailed;
      return false;
    }
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // CUTLASS op ran the but not yet verified against any verification provider
  results_.back().disposition = Disposition::kNotVerified;

  //
  // Run verification providers
  //

  if (options.verification.enabled) {

#if CUTLASS_ENABLE_CUBLAS
    if (options.verification.provider_enabled(library::Provider::kCUBLAS)) {
      // set verification map for cublas to not supported
      results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotSupported;
    }
#endif // #if CUTLASS_ENABLE_CUBLAS

    
    cutlass::library::RuntimeDatatype runtime_datatype_a = gemm_workspace_.arguments.runtime_input_datatype_a;
    cutlass::library::RuntimeDatatype runtime_datatype_b = gemm_workspace_.arguments.runtime_input_datatype_b;

    bool is_runtime_datatype_a = runtime_datatype_a != cutlass::library::RuntimeDatatype::kStatic;
    bool is_runtime_datatype_b = runtime_datatype_b != cutlass::library::RuntimeDatatype::kStatic;

    assert(is_runtime_datatype_a == is_runtime_datatype_b && "runtime datatype should be both dynamic or static.");
    
    library::OperationDescription const &desc = operation->description();
    auto &gemm_desc = static_cast<library::BlockScaledGemmDescription const &>(desc);

    cutlass::library::NumericTypeID element_A = gemm_desc.A.element;
    cutlass::library::NumericTypeID element_B = gemm_desc.B.element;
    
    if (is_runtime_datatype_a) {
      element_A = cutlass::library::dynamic_datatype_to_id(runtime_datatype_a);
    }

    if (is_runtime_datatype_b) {
      element_B = cutlass::library::dynamic_datatype_to_id(runtime_datatype_b);
    }
    

    bool verification_status = verify_with_reference_(options, report, device_context, operation, problem_space, problem, element_A, element_B);

    // Update disposition to worst case verification outcome among all
    // verification providers which are supported
    bool is_any_verification_run_passed = false;
    for (auto &m : results_.back().verification_map) {
      if (m.second == Disposition::kFailed || m.second == Disposition::kIncorrect) {
        results_.back().disposition = m.second;
        return true;
      }
      if (!is_any_verification_run_passed && m.second == Disposition::kPassed) {
        is_any_verification_run_passed = true;
      }
    }

    if (is_any_verification_run_passed) {
      results_.back().disposition = Disposition::kPassed;
    }
  }

  // if verification.required is set, then return success iff at least one ref-check was run
  if (options.verification.required) {
    bool did_any_verification_run = false;
    for (auto provider : options.verification.providers) {
      did_any_verification_run |= (Disposition::kNotRun != results_.back().verification_map[provider]);
    }

    if (not did_any_verification_run) {
      results_.back().status = Status::kErrorNotSupported;
      return false;
    }
  }

  // Return true means continue profiling
  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool BlockScaledGemmOperationProfiler::verify_with_cublas_(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

#if CUTLASS_ENABLE_CUBLAS
  std::cerr << "cuBLAS is not supported" << std::endl;
#endif

  // Return true means continue profiling
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against host and device references
bool BlockScaledGemmOperationProfiler::verify_with_reference_(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem,
  cutlass::library::NumericTypeID element_A,
  cutlass::library::NumericTypeID element_B) {

  /// Verifies CUTLASS against host reference

  //
  // Find host reference operation using conv2d functional description key
  //
  library::OperationDescription const &desc = operation->description();

  auto &gemm_desc = static_cast<library::BlockScaledGemmDescription const &>(desc);

  library::BlockScaledGemmFunctionalKey blockScaledGemm_key(
    library::Provider::kReferenceHost,
    gemm_desc.gemm_kind,
    gemm_desc.kind,
    gemm_desc.tile_description.math_instruction.element_accumulator,
    gemm_desc.element_epilogue,
    element_A,
    gemm_desc.A.layout,
    gemm_desc.SFA.element,
    element_B,
    gemm_desc.B.layout,
    gemm_desc.SFB.element,
    gemm_desc.C.element,
    gemm_desc.C.layout,
    gemm_desc.D.element,
    gemm_desc.D.layout,
    gemm_desc.SFD.element,
    gemm_desc.SFD.layout,
    gemm_desc.SFVecSize
    , gemm_desc.EpilogueSFVecSize
  );

  auto operators_it = library::Singleton::get().operation_table.block_scaled_gemm_operations.find(blockScaledGemm_key);

  if (operators_it == library::Singleton::get().operation_table.block_scaled_gemm_operations.end()) {
    return true;
  }

  if (operators_it->second.empty()) {
    return true;
  }

  // Not use preference to filter the reference kernel.
  auto cc_it = operators_it->second.begin();

  if(cc_it == operators_it->second.end()) {
    std::cout<< "not find any reference kernel" << std::endl;
    results_.back().verification_map[library::Provider::kReferenceHost] = Disposition::kNotRun;
    return true;
  }

  // host reference has only one instances in BlockScaledOperationVectorMap
  library::Operation const *reference_op = cc_it->second[0];

  // To support the host-side reference, conditionally allocate and
  // copy tensors to host memory.
  std::vector<uint8_t> host_data_A;
  std::vector<uint8_t> host_data_SFA;
  std::vector<uint8_t> host_data_B;
  std::vector<uint8_t> host_data_SFB;
  std::vector<uint8_t> host_data_C;
  std::vector<uint8_t> host_data_D;
  std::vector<uint8_t> host_data_SFD;
  std::vector<uint8_t> host_data_Norm_constant;

  //
  // Copy input tensors A, B, and C from device to host buffers
  //

  host_data_A.resize(gemm_workspace_.A->bytes());
  void * ptr_A = host_data_A.data();
  gemm_workspace_.A->copy_to_host(ptr_A);

  host_data_SFA.resize(gemm_workspace_.SFA->bytes());
  void * ptr_SFA = host_data_SFA.data();
  gemm_workspace_.SFA->copy_to_host(ptr_SFA);

  host_data_B.resize(gemm_workspace_.B->bytes());
  void * ptr_B = host_data_B.data();
  gemm_workspace_.B->copy_to_host(ptr_B);

  host_data_SFB.resize(gemm_workspace_.SFB->bytes());
  void * ptr_SFB = host_data_SFB.data();
  gemm_workspace_.SFB->copy_to_host(ptr_SFB);

  host_data_C.resize(gemm_workspace_.C->bytes());
  void * ptr_C = host_data_C.data();
  gemm_workspace_.C->copy_to_host(ptr_C);
  
  host_data_Norm_constant.resize(gemm_workspace_.Norm_constant->bytes());
  void * ptr_Norm_constant = host_data_Norm_constant.data();
  gemm_workspace_.Norm_constant->copy_to_host(ptr_Norm_constant);
  
  host_data_D.resize(gemm_workspace_.Reference->bytes());
  void * ptr_D = host_data_D.data();

  host_data_SFD.resize(gemm_workspace_.Reference_SFD->bytes());
  void * ptr_SFD = host_data_SFD.data();

  /// Set reference kernel Arguments

  library::BlockScaledGemmArguments arguments {
    {int(problem_.m), int(problem_.n), int(problem_.k)},
    {int(problem_.cluster_m), int(problem_.cluster_n), int(problem_.cluster_k)},
    {int(problem_.cluster_m_fallback), int(problem_.cluster_n_fallback), int(problem_.cluster_k_fallback)},
    gemm_workspace_.configuration.batch_count,
    ptr_A,
    ptr_B,
    ptr_SFA,
    ptr_SFB,
    ptr_C,
    ptr_D,
    ptr_SFD,
    problem_.alpha.data(),
    problem_.beta.data(),
    library::ScalarPointerMode::kHost,
    int(gemm_workspace_.configuration.lda),
    int(gemm_workspace_.configuration.ldb),
    int(gemm_workspace_.configuration.ldc),
    int(gemm_workspace_.configuration.ldd),
    gemm_workspace_.A->batch_stride(),
    gemm_workspace_.B->batch_stride(),
    gemm_workspace_.C->batch_stride(),
    gemm_workspace_.Reference->batch_stride()
    , ptr_Norm_constant
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = reference_op->get_host_workspace_size(&gemm_workspace_.configuration);

  std::vector<char> host_workspace(host_workspace_size_needed);

  // Query device workspace size
  uint64_t device_workspace_size_needed = reference_op->get_device_workspace_size(&gemm_workspace_.configuration);
  // Initialize host and device workspaces
  Status status = reference_op->initialize(
    &gemm_workspace_.configuration,
    host_workspace.data()
  );

  if (status != cutlass::Status::kSuccess) {
    results_.back().verification_map[library::Provider::kReferenceHost] = Disposition::kNotRun;
    return true;
  }

  // Run the operator
  status = reference_op->run(&arguments, host_workspace.data());

  results_.back().status = status;

  gemm_workspace_.Reference->copy_from_host(ptr_D);
  gemm_workspace_.Reference_SFD->copy_from_host(ptr_SFD);

  //
  // Verify results
  //
  auto resultD = compare_tensors(
    options,
    *gemm_workspace_.Computed,
    *gemm_workspace_.Reference,
    gemm_workspace_.Computed->batch_stride()
  );
  
  auto resultSFD = Disposition::kPassed;
  if (gemm_desc.SFD.element != library::NumericTypeID::kVoid) {
    resultSFD = compare_tensors(
      options,
      *gemm_workspace_.Computed_SFD,
      *gemm_workspace_.Reference_SFD,
      gemm_workspace_.Computed_SFD->batch_stride()
    );
  }
  
  results_.back().verification_map[library::Provider::kReferenceHost] = resultD;
  
  if (resultSFD != Disposition::kPassed) {
      results_.back().verification_map[library::Provider::kReferenceHost] = resultSFD;
  }
  

  // Save workspace if incorrect
  if (options.verification.save_workspace == SaveWorkspace::kIncorrect &&
    results_.back().verification_map[library::Provider::kReferenceHost] == Disposition::kIncorrect) {
    save_workspace(
      device_context,
      options,
      gemm_desc,
      library::Provider::kCUTLASS,
      library::Provider::kReferenceHost);
  }

  // Return true means continue profiling
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool BlockScaledGemmOperationProfiler::profile(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.SFA = gemm_workspace_.SFA->data();
    gemm_workspace_.arguments.SFB = gemm_workspace_.SFB->data();
    gemm_workspace_.arguments.C = gemm_workspace_.C->data();
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;
    gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
    gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
    gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.C->batch_stride();
    gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Computed->batch_stride();

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_.arguments.D                       = gemm_workspace_.device_workspace.data();
      gemm_workspace_.arguments.alpha                   = problem_.alpha_one.data();
      gemm_workspace_.arguments.beta                    = problem_.beta_zero.data();

      gemm_workspace_.reduction_arguments.workspace     = gemm_workspace_.device_workspace.data();
      gemm_workspace_.reduction_arguments.source        = gemm_workspace_.C->data();
      gemm_workspace_.reduction_arguments.destination   = gemm_workspace_.Computed->data();
      gemm_workspace_.reduction_arguments.alpha         = problem_.alpha.data();
      gemm_workspace_.reduction_arguments.beta          = problem_.beta.data();
      gemm_workspace_.reduction_arguments.pointer_mode  = library::ScalarPointerMode::kHost;
    }

    results_.back().status = profile_cutlass_(
      results_.back(),
      options,
      operation,
      &gemm_workspace_.arguments,
      gemm_workspace_.host_workspace.data(),
      gemm_workspace_.device_workspace.data()
    );
  }
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Method to profile a CUTLASS Operation
Status BlockScaledGemmOperationProfiler::profile_cutlass_(
  PerformanceResult &result,
  Options const &options,
  library::Operation const *operation,
  void *arguments,
  void *host_workspace,
  void *device_workspace) {

  // initialize gemm underlying operation to handle parallel reduction
  library::Operation const * underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      return Status::kErrorNotSupported;
    }
  }

  auto func = [&](hipStream_t, int iteration) {
    // Iterate over copies of the problem in memory
    int problem_idx = (iteration % gemm_workspace_.problem_count) * problem_.batch_count;

    gemm_workspace_.arguments.A = gemm_workspace_.A->batch_data(problem_idx);
    gemm_workspace_.arguments.B = gemm_workspace_.B->batch_data(problem_idx);
    gemm_workspace_.arguments.C = gemm_workspace_.C->batch_data(problem_idx);
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->batch_data(problem_idx);

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_.arguments.D                     = gemm_workspace_.device_workspace.data();

      gemm_workspace_.reduction_arguments.workspace   = gemm_workspace_.device_workspace.data();
      gemm_workspace_.reduction_arguments.source      = gemm_workspace_.C->batch_data(problem_idx);
      gemm_workspace_.reduction_arguments.destination = gemm_workspace_.Computed->batch_data(problem_idx);
    }

    Status status = underlying_operation->run(
      arguments,
      host_workspace,
      device_workspace,
      nullptr);

    if (status != Status::kSuccess) {
      return status;
    }

    // Run parallel reduction kernel for parallel split_k_mode
    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      status = reduction_op_->run(
        &gemm_workspace_.reduction_arguments,
        gemm_workspace_.reduction_host_workspace.data(),
        nullptr,
        nullptr);

      if (status != Status::kSuccess) {
        return status;
      }
    }

    return status;
  };

  return profile_kernel_(result, options, func);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
