#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <iostream>
#include <stdexcept>
#include <iomanip>
#include <ios>
#include <vector>

#include "cutlass/core_io.h"
#include <hip/hip_runtime_api.h>
#include <cuda/atomic>

#include "cutlass/profiler/cublas_helpers.h"
#include "cutlass/profiler/gemm_operation_profiler.h"
#include "cutlass/profiler/gpu_timer.h"
#include "cutlass/library/singleton.h"
#include "cutlass/library/library.h"
#include "cutlass/library/handle.h"
/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Ctor
GemmOperationProfiler::GemmOperationProfiler(Options const &options):
  OperationProfiler(
    options,
    library::OperationKind::kGemm,
    {
      {ArgumentTypeID::kEnumerated, {"gemm_kind"}, "Variant of GEMM (universal, gemm, planar_complex, planar_complex_array)"},
      {ArgumentTypeID::kInteger, {"m", "problem-size::m"}, "M dimension of the GEMM problem space"},
      {ArgumentTypeID::kInteger, {"n", "problem-size::n"}, "N dimension of the GEMM problem space"},
      {ArgumentTypeID::kInteger, {"k", "problem-size::k"}, "K dimension of the GEMM problem space"},
      {ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
      {ArgumentTypeID::kTensor, {"B"}, "Tensor storing the B operand"},
      {ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
      {ArgumentTypeID::kTensor, {"D"}, "Tensor storing the D output"},
      {ArgumentTypeID::kScalar, {"alpha", "epilogue::alpha"}, "Epilogue scalar alpha"},
      {ArgumentTypeID::kScalar, {"beta", "epilogue::beta"}, "Epilogue scalar beta"},
      {ArgumentTypeID::kEnumerated, {"split_k_mode", "split-k-mode"}, "Variant of split K mode(serial, parallel)"},
      {ArgumentTypeID::kInteger, {"split_k_slices", "split-k-slices"}, "Number of partitions of K dimension"},
      {ArgumentTypeID::kInteger, {"batch_count", "batch-count"}, "Number of GEMMs computed in one batch"},
      {ArgumentTypeID::kEnumerated, {"raster_order", "raster-order"}, "Raster order (heuristic, along_n, along_m)"},
      {ArgumentTypeID::kEnumerated, {"runtime_input_datatype_a", "runtime-input-datatype::a"}, "Runtime datatype (e4m3, e5m2, e3m2, e2m3, e2m1)"}, 
      {ArgumentTypeID::kEnumerated, {"runtime_input_datatype_b", "runtime-input-datatype::b"}, "Runtime datatype (e4m3, e5m2, e3m2, e2m3, e2m1)"}, 
      {ArgumentTypeID::kInteger, {"use_pdl", "use-pdl"}, "Use PDL (true, false)"}, 
      {ArgumentTypeID::kEnumerated, {"enable_sm90_mixed_dtype_shuffle_test", "enable-sm90-mixed-dtype-shuffle-test"}, "Enable SM90 mixed input data type kernel shuffle layout test (true, false)"},
      {ArgumentTypeID::kInteger, {"swizzle_size", "swizzle-size"}, "Size to swizzle"},
    },
    { library::Provider::kCUBLAS}
  ) {

  description_ = "      General matrix-matrix product. D = alpha * A*B + beta * C";
}

/// Destructor
GemmOperationProfiler::~GemmOperationProfiler() {

}

/// Prints usage statement for the math function
void GemmOperationProfiler::print_usage(std::ostream &out) const {
  out << "GEMM" << "\n\n";

  OperationProfiler::print_usage(out);
}

/// Prints examples
void GemmOperationProfiler::print_examples(std::ostream &out) const {

  out << "\nExamples:\n\n"
    << "Profile a particular problem size:\n"
    << "  $ cutlass_profiler --operation=Gemm --m=1024 --n=1024 --k=128\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=Gemm --m=1024:4096:256 --n=1024:4096:256 --k=128:8192:128 --beta=0,1,2.5\n\n"

    << "Schmoo over accumulator types:\n"
    << "  $ cutlass_profiler --operation=Gemm --accumulator-type=f16,f32\n\n"

    << "Run when A is f16 with column-major and B is any datatype with row-major (For column major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=Gemm --A=f16:column --B=*:row\n\n"

    << "Profile a particular problem size with split K and parallel reduction:\n"
    << "  $ cutlass_profiler --operation=Gemm --split_k_mode=parallel --split_k_slices=2 --m=1024 --n=1024 --k=128\n\n"

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=sequential,start:0,delta:1\n\n"

    << "Run a kernel with cta tile size of 256x128x32 and save workspace if results are incorrect (note that --cta-tile::k=32 is default cta-tile size):\n"
    << " $ cutlass_profiler --operation=Gemm --cta_m=256 --cta_n=128  --cta_k=32 --save-workspace=incorrect\n\n"

    << "Test your changes to gemm kernels with a quick functional test and save results in functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=Gemm \\ \n"
    << "   --m=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --n=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --k=8,16,32,64,128,256,288,384,504,512,520 \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#if 0
// used this for debugging
static std::string byte_string(std::vector<uint8_t> const &bytes) {
  std::stringstream ss;

  ss << "0x";

  for (size_t idx = bytes.size(); idx > 0; --idx) {
    ss << std::hex << std::setw(2) << std::setfill('0') << uint32_t(bytes.at(idx - 1));
  }

  return ss.str();
}
#endif

Status GemmOperationProfiler::GemmProblem::parse(
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  this->mode = library::GemmUniversalMode::kGemm;

  if (!arg_as_int(this->m, "m", problem_space, problem)) {
    // default value
    this->m = 1024;
  }

  if (!arg_as_int(this->n, "n", problem_space, problem)) {
    // default value
    this->n = 1024;
  }

  if (!arg_as_int(this->k, "k", problem_space, problem)) {
    // default value
    this->k = 1024;
  }

  if (!arg_as_int(this->cluster_m, "cluster_m", problem_space, problem)) {
    // default value
    this->cluster_m = 1;
  }

  if (!arg_as_int(this->cluster_n, "cluster_n", problem_space, problem)) {
    // default value
    this->cluster_n = 1;
  }

  if (!arg_as_int(this->cluster_k, "cluster_k", problem_space, problem)) {
    // default value
    this->cluster_k = 1;
  }

  if (!arg_as_int(this->cluster_m_fallback, "cluster_m_fallback", problem_space, problem)) {
    // default value
    this->cluster_m_fallback = 0;
  }

  if (!arg_as_int(this->cluster_n_fallback, "cluster_n_fallback", problem_space, problem)) {
    // default value
    this->cluster_n_fallback = 0;
  }

  if (!arg_as_int(this->cluster_k_fallback, "cluster_k_fallback", problem_space, problem)) {
    // default value
    this->cluster_k_fallback = 0;
  }

  if (!arg_as_bool(this->use_pdl, "use_pdl", problem_space, problem)) {
    // default value
    this->use_pdl = false;
  }

  if (!arg_as_bool(this->enable_sm90_mixed_dtype_shuffle_test, "enable_sm90_mixed_dtype_shuffle_test", problem_space, problem)) {
    // default value
    this->enable_sm90_mixed_dtype_shuffle_test = false;
  }

  if (!arg_as_SplitKModeID(this->split_k_mode, "split_k_mode", problem_space, problem)) {
    // default value
    this->split_k_mode = library::SplitKMode::kSerial;
  }

  this->mode = library::GemmUniversalMode::kGemm;
  if (this->split_k_mode == library::SplitKMode::kParallel) {
    this->mode = library::GemmUniversalMode::kGemmSplitKParallel;
  }

  if (!arg_as_int(this->split_k_slices, "split_k_slices", problem_space, problem)) {
    // default value
    this->split_k_slices = 1;
  }

  
  if (!arg_as_RuntimeDatatype(this->runtime_input_datatype_a, "runtime_input_datatype_a", problem_space, problem)) {
    // default value
    this->runtime_input_datatype_a = cutlass::library::RuntimeDatatype::kStatic;
  }

  if (!arg_as_RuntimeDatatype(this->runtime_input_datatype_b, "runtime_input_datatype_b", problem_space, problem)) {
    // default value
    this->runtime_input_datatype_b = cutlass::library::RuntimeDatatype::kStatic;
  }
  

  if (!arg_as_int(this->batch_count, "batch_count", problem_space, problem)) {
    // default value
    this->batch_count = 1;
  } else if (this->batch_count > 1) {
    this->mode = library::GemmUniversalMode::kBatched;
  }

  if (!arg_as_int(this->swizzle_size, "swizzle_size", problem_space, problem)) {
    // default value
    this->swizzle_size = 1;
  }

  if (!arg_as_RasterOrder(this->raster_order, "raster_order", problem_space, problem)) {
    // default value
    this->raster_order = library::RasterOrder::kHeuristic;
  }

  if (this->split_k_slices > 1 && this->batch_count > 1) {
    // At least one of these must be one
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.B, "B", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.D, "D", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
    this->alpha,
    operation_desc.element_epilogue,
    "alpha",
    problem_space,
    problem)) {

    if (!cast_from_double(this->alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }

  if (!arg_as_scalar(
    this->beta,
    operation_desc.element_epilogue,
    "beta",
    problem_space,
    problem)) {

    if (!cast_from_double(this->beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }

  this->lda = DeviceAllocation::get_packed_layout(
    operation_desc.A.layout, {int(this->m), int(this->k)}).front();

  this->ldb = DeviceAllocation::get_packed_layout(
    operation_desc.B.layout, {int(this->k), int(this->n)}).front();

  this->ldc = DeviceAllocation::get_packed_layout(
    operation_desc.C.layout, {int(this->m), int(this->n)}).front();

  // instantiation
  int num_sizes = 8;
  this->problem_sizes.resize(num_sizes);
  this->leading_dims.resize(num_sizes, {0, 0, 0});
    
  int m0 = 1024;
  int n0 = 1024;
  int k0 = 1024;
  for (int i = 0; i < num_sizes; i++) {
    auto m = m0 * (i + 1);
    auto n = n0 * (i + 1);
    auto k = k0 * (i + 1);
    this->problem_sizes[i] = {m, n, k};
    this->leading_dims[i] = {
      DeviceAllocation::get_packed_layout(operation_desc.A.layout, {int(m), int(k)}).front(),
      DeviceAllocation::get_packed_layout(operation_desc.B.layout, {int(k), int(n)}).front(),
      DeviceAllocation::get_packed_layout(operation_desc.C.layout, {int(m), int(n)}).front()
    };

  }

  this->raster_orders = {
    cutlass::library::RasterOrder::kAlongN,
    cutlass::library::RasterOrder::kAlongM
  };

  this->swizzle_sizes = {1, 2, 4, 8};

  this->preferred_clusters = {
    {1, 1, 1}, {2, 1, 1}, {2, 2, 1}, {4, 1, 1}, {4, 2, 1}, {4, 4, 1}, {8, 2, 1}
  };

  this->fallback_clusters = {
    {1, 1, 1}, {2, 1, 1}, {2, 2, 1}
  };

  return Status::kSuccess;
}

int64_t GemmOperationProfiler::GemmProblem::bytes_with_problem_shape(
  library::GemmDescription const &operation_desc,
  gemm::GemmCoord const &problem_shape) const {

  // Input bytes read and Output bytes written for the gemm problem
  int64_t bytes =
    int64_t(library::sizeof_bits(operation_desc.A.element) * problem_shape.m() / 8) * problem_shape.k() +
    int64_t(library::sizeof_bits(operation_desc.B.element) * problem_shape.n() / 8) * problem_shape.k() +
    int64_t(library::sizeof_bits(operation_desc.C.element) * problem_shape.m() / 8) * problem_shape.n();

  // Set is_beta_zero true if beta is zero
  bool is_beta_zero = std::all_of(beta.begin(), beta.end(), [](uint8_t i) { return i==0; });

  // Output bytes read for the gemm problem for non-zero beta values
  if (!is_beta_zero) {
    bytes += int64_t(library::sizeof_bits(operation_desc.C.element) * problem_shape.m() / 8) * problem_shape.n();
  }

  bytes *= batch_count;

  return bytes;

}

/// Total number of bytes loaded
int64_t GemmOperationProfiler::GemmProblem::bytes(library::GemmDescription const &operation_desc) const {
  gemm::GemmCoord problem_shape({int(m), int(n), int(k)});
  return bytes_with_problem_shape(operation_desc, problem_shape);
}

/// Total number of flops computed
int64_t GemmOperationProfiler::GemmProblem::flops_with_problem_shape(
  library::GemmDescription const &operation_desc,
  gemm::GemmCoord const &problem_shape) const {
  int64_t flops_ = (int64_t(problem_shape.m()) * problem_shape.n() * problem_shape.k() + problem_shape.m() * problem_shape.n()) * 2 * batch_count;

  // complex-valued support
  switch (operation_desc.tile_description.math_instruction.math_operation) {
  case library::MathOperationID::kMultiplyAddComplex:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddComplexFastF32:
    flops_ *= 4;
    break;

  case library::MathOperationID::kMultiplyAddGaussianComplex:
    flops_ *= 3;
    break;

  default: break;
  }

  return flops_;
}

/// Total number of flops computed
int64_t GemmOperationProfiler::GemmProblem::flops(library::GemmDescription const &operation_desc) const {
  gemm::GemmCoord problem_shape({int(m), int(n), int(k)});
  return flops_with_problem_shape(operation_desc, problem_shape);
}


/// Initializes a performance result
void GemmOperationProfiler::GemmProblem::initialize_result(
  PerformanceResult &result,
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.arguments.resize(problem_space.rank());

  set_argument(result, "gemm_kind", problem_space, library::to_string(operation_desc.gemm_kind));

  set_argument(result, "A", problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" + library::to_string(operation_desc.A.layout));

  set_argument(result, "B", problem_space,
    std::string(library::to_string(operation_desc.B.element)) + ":" + library::to_string(operation_desc.B.layout));

  set_argument(result, "C", problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" + library::to_string(operation_desc.C.layout));

  set_argument(result, "D", problem_space,
    std::string(library::to_string(operation_desc.D.element)) + ":" + library::to_string(operation_desc.D.layout));

  set_argument(result, "m", problem_space, m);
  set_argument(result, "n", problem_space, n);
  set_argument(result, "k", problem_space, k);

  
  set_argument(result, "cluster_m", problem_space, cluster_m);
  set_argument(result, "cluster_n", problem_space, cluster_n);
  set_argument(result, "cluster_k", problem_space, cluster_k);
  set_argument(result, "cluster_m_fallback", problem_space, cluster_m_fallback);
  set_argument(result, "cluster_n_fallback", problem_space, cluster_n_fallback);
  set_argument(result, "cluster_k_fallback", problem_space, cluster_k_fallback);
  

  set_argument(result, "split_k_mode", problem_space, library::to_string(split_k_mode));
  set_argument(result, "split_k_slices", problem_space, split_k_slices);
  set_argument(result, "batch_count", problem_space, batch_count);
  set_argument(result, "raster_order", problem_space, library::to_string(raster_order));
  set_argument(result, "swizzle_size", problem_space, swizzle_size);
  set_argument(result, "use_pdl", problem_space, library::to_string(use_pdl));
  set_argument(result, "enable_sm90_mixed_dtype_shuffle_test", problem_space, library::to_string(enable_sm90_mixed_dtype_shuffle_test));

  
  set_argument(result, "runtime_input_datatype_a", problem_space, library::to_string(runtime_input_datatype_a));
  set_argument(result, "runtime_input_datatype_b", problem_space, library::to_string(runtime_input_datatype_b));
  

  set_argument(result, "alpha", problem_space,
    library::lexical_cast(alpha, operation_desc.element_epilogue));

  set_argument(result, "beta", problem_space,
    library::lexical_cast(beta, operation_desc.element_epilogue));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Extracts the problem dimensions
Status GemmOperationProfiler::initialize_configuration(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::GemmDescription const &operation_desc =
    static_cast<library::GemmDescription const &>(operation->description());

  if (operation_desc.gemm_kind != library::GemmKind::kUniversal) {
    return Status::kErrorInvalidProblem;
  }

  Status status = problem_.parse(operation_desc, problem_space, problem);

  // Note: this is a temporary workaround
  bool is_sm90_operation = (strstr(operation_desc.name, "_sm90") != NULL);
  bool is_sm90_mixed_dtype_shuffle_operation = (strstr(operation_desc.name, "_shfl") != NULL);
  if (is_sm90_mixed_dtype_shuffle_operation && (problem_.enable_sm90_mixed_dtype_shuffle_test == false)) {
    return Status::kErrorInvalidProblem;
  }

  if (status != Status::kSuccess) {
    return status;
  }

  auto const device_count = options.device.devices.size();

  gemm_workspace_.clear();

  library::NumericTypeID a_elem = library::get_real_type(operation_desc.A.element);
  library::NumericTypeID b_elem = library::get_real_type(operation_desc.B.element);
  int a_elem_bits = library::sizeof_bits(a_elem);
  int b_elem_bits = library::sizeof_bits(b_elem);
  bool is_sm90_mixed_dtype_operation = is_sm90_operation && (a_elem_bits != b_elem_bits);

  for (size_t i = 0; i < device_count; ++i) {
    hipSetDevice(options.device.device_id(i));
    gemm_workspace_.emplace_back();
    hipStreamCreateWithFlags(&gemm_workspace_[i].stream, hipStreamNonBlocking);
    gemm_workspace_[i].configuration.mode = problem_.mode;
    gemm_workspace_[i].configuration.problem_size.m() = int(problem_.m);
    gemm_workspace_[i].configuration.problem_size.n() = int(problem_.n);
    gemm_workspace_[i].configuration.problem_size.k() = int(problem_.k);
    
    gemm_workspace_[i].configuration.cluster_shape.m() = int(problem_.cluster_m);
    gemm_workspace_[i].configuration.cluster_shape.n() = int(problem_.cluster_n);
    gemm_workspace_[i].configuration.cluster_shape.k() = int(problem_.cluster_k);
    gemm_workspace_[i].configuration.cluster_shape_fallback.m() = int(problem_.cluster_m_fallback);
    gemm_workspace_[i].configuration.cluster_shape_fallback.n() = int(problem_.cluster_n_fallback);
    gemm_workspace_[i].configuration.cluster_shape_fallback.k() = int(problem_.cluster_k_fallback);
    gemm_workspace_[i].configuration.lda = problem_.lda;
    gemm_workspace_[i].configuration.ldb = problem_.ldb;
    gemm_workspace_[i].configuration.ldc = problem_.ldc;
    gemm_workspace_[i].configuration.ldd = problem_.ldc;

    gemm_workspace_[i].configuration.device_count = static_cast<int>(device_count);
    gemm_workspace_[i].arguments.device_index = static_cast<int>(i);
    gemm_workspace_[i].arguments.use_pdl = problem_.use_pdl;

    if (problem_.mode == library::GemmUniversalMode::kBatched) {
      gemm_workspace_[i].configuration.batch_count = problem_.batch_count;
    }
    else {
      gemm_workspace_[i].configuration.batch_count = problem_.split_k_slices;
    }

    gemm_workspace_[i].arguments.problem_size.m() = int(problem_.m);
    gemm_workspace_[i].arguments.problem_size.n() = int(problem_.n);
    gemm_workspace_[i].arguments.problem_size.k() = int(problem_.k);
    if (problem_.mode == library::GemmUniversalMode::kBatched) {
      gemm_workspace_[i].arguments.batch_count = problem_.batch_count;
    }
    else {
      gemm_workspace_[i].arguments.batch_count = problem_.split_k_slices;
    }

    gemm_workspace_[i].arguments.A = nullptr;
    gemm_workspace_[i].arguments.B = nullptr;
    gemm_workspace_[i].arguments.C = nullptr;
    gemm_workspace_[i].arguments.D = nullptr;
    gemm_workspace_[i].arguments.alpha = problem_.alpha.data();
    gemm_workspace_[i].arguments.beta = problem_.beta.data();
    gemm_workspace_[i].arguments.pointer_mode = library::ScalarPointerMode::kHost;
    gemm_workspace_[i].arguments.swizzle_size = problem_.swizzle_size;
    gemm_workspace_[i].arguments.raster_order = problem_.raster_order;
    gemm_workspace_[i].arguments.cluster_shape = {int(problem_.cluster_m), int(problem_.cluster_n), int(problem_.cluster_k)}; 
    gemm_workspace_[i].arguments.cluster_shape_fallback = {int(problem_.cluster_m_fallback), int(problem_.cluster_n_fallback), int(problem_.cluster_k_fallback)}; 
    gemm_workspace_[i].arguments.split_k_slices = problem_.split_k_slices;

    
    gemm_workspace_[i].arguments.runtime_input_datatype_a = problem_.runtime_input_datatype_a;
    gemm_workspace_[i].arguments.runtime_input_datatype_b = problem_.runtime_input_datatype_b;
    

    initialize_result_(this->model_result_, options, operation_desc, problem_space);
    if (is_sm90_mixed_dtype_operation)
    {
      const int options_g = problem_.k;
      const int options_l = problem_.batch_count;
      const int scale_k = (problem_.k + options_g - 1) / options_g;
      // We cannot get the mainloop's ElementScale and ElementZero here,
      // use the wide type to allocate a large enough workspace for S and Z.
      library::NumericTypeID wide_dtype;
      size_t SZ_mat_size = 0;
      if (a_elem_bits > b_elem_bits) {
        wide_dtype = a_elem;
        SZ_mat_size = static_cast<size_t>(problem_.n * scale_k);
      }
      else {
        wide_dtype = b_elem;
        SZ_mat_size = static_cast<size_t>(problem_.m * scale_k);
      }

      gemm_workspace_[i].Scale = device_context.allocate_tensor(
        options,
        "Scale",
        wide_dtype,
        library::LayoutTypeID::kRowMajor,
        {int(SZ_mat_size), int(options_l)},
        {int(options_l)},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        i // device_index
      );
      gemm_workspace_[i].Zero = device_context.allocate_tensor(
        options,
        "Zero",
        wide_dtype,
        library::LayoutTypeID::kRowMajor,
        {int(SZ_mat_size), int(options_l)},
        {int(options_l)},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        i // device_index
      );

      // Packed scale is for int4 * fp8, where the original scale is fp8, and
      // each scale element will be packed into an Array<fp8, 8> which is 64-bit
      gemm_workspace_[i].packed_Scale = device_context.allocate_tensor(
        options,
        "packed-Scale",
        library::NumericTypeID::kU64,
        library::LayoutTypeID::kRowMajor,
        {int(SZ_mat_size), int(options_l)},
        {int(options_l)},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        i // device_index
      );

      gemm_workspace_[i].arguments.problem_size = {int(problem_.m), int(problem_.n), int(problem_.k)};
      gemm_workspace_[i].arguments.batch_count = problem_.batch_count;

      // Here is the first touch of the arguments, mark the mixed dtype,
      // populate the scale and zero tensors in the following can_implement() call later.
      // A and B are not populated at this moment, so do not update the dequantized A or B
      gemm_workspace_[i].arguments.is_sm90_mixed_dtype = true;
      gemm_workspace_[i].arguments.wider_operand = (a_elem_bits > b_elem_bits) ? cutlass::library::Sm90MixedInputWiderOperand::A : cutlass::library::Sm90MixedInputWiderOperand::B;
      gemm_workspace_[i].arguments.generate_scale_and_zero = true;
      gemm_workspace_[i].arguments.generate_dequantized_AB = false;
      gemm_workspace_[i].arguments.Scale = gemm_workspace_[i].Scale->data();
      gemm_workspace_[i].arguments.Zero = gemm_workspace_[i].Zero->data();
      gemm_workspace_[i].arguments.packed_Scale = gemm_workspace_[i].packed_Scale->data();
    }  // End of "if (is_sm90_mixed_dtype_operation)"

    const auto can_implement = operation->can_implement(&gemm_workspace_[i].configuration, &gemm_workspace_[i].arguments);
    if (can_implement != Status::kSuccess) {
      return can_implement;
    }
  }

  // initialize reduction operation for parallel splitKMode
  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!initialize_reduction_configuration_(operation, problem)) {
      return Status::kErrorInternal;
    }
  }

  return status;
}

void GemmOperationProfiler::update_workspace_(
  GemmWorkspace &gemm_workspace,
  gemm::GemmCoord const &problem_shape,
  std::array<int64_t, 3> const &leading_dim,
  std::array<int64_t, 3> const &preferred_cluster,
  std::array<int64_t, 3> const &fallback_cluster,
  cutlass::library::RasterOrder const &raster_order,
  int swizzle_size
) {

  gemm_workspace.arguments.problem_size.m() = problem_shape.m();
  gemm_workspace.arguments.problem_size.n() = problem_shape.n();
  gemm_workspace.arguments.problem_size.k() = problem_shape.k();

  gemm_workspace.arguments.lda = leading_dim[0];
  gemm_workspace.arguments.ldb = leading_dim[1];
  gemm_workspace.arguments.ldc = leading_dim[2];

  gemm_workspace.arguments.swizzle_size = swizzle_size;
  gemm_workspace.arguments.raster_order = raster_order;

  gemm_workspace.arguments.cluster_shape = {int(preferred_cluster[0]), int(preferred_cluster[1]), int(preferred_cluster[2])};
  gemm_workspace.arguments.cluster_shape_fallback = {int(fallback_cluster[0]), int(fallback_cluster[1]), int(fallback_cluster[2])};

  gemm_workspace.configuration.problem_size.m() = problem_shape.m();
  gemm_workspace.configuration.problem_size.n() = problem_shape.n();
  gemm_workspace.configuration.problem_size.k() = problem_shape.k();

  gemm_workspace.configuration.cluster_shape = {int(preferred_cluster[0]), int(preferred_cluster[1]), int(preferred_cluster[2])};
  gemm_workspace.configuration.cluster_shape_fallback = {int(fallback_cluster[0]), int(fallback_cluster[1]), int(fallback_cluster[2])};

  gemm_workspace.configuration.lda = leading_dim[0];
  gemm_workspace.configuration.ldb = leading_dim[1];
  gemm_workspace.configuration.ldc = leading_dim[2];

}

void GemmOperationProfiler::update_result_(
  PerformanceResult &result,
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space,
  gemm::GemmCoord const &problem_shape,
  cutlass::library::RasterOrder const &raster_order,
  std::array<int64_t, 3> const &preferred_cluster,
  std::array<int64_t, 3> const &fallback_cluster,
  int swizzle_size
) {
  result.bytes = problem_.bytes_with_problem_shape(operation_desc, problem_shape);
  result.flops = problem_.flops_with_problem_shape(operation_desc, problem_shape);

  set_argument(result, "m", problem_space, problem_shape.m());
  set_argument(result, "n", problem_space, problem_shape.n());
  set_argument(result, "k", problem_space, problem_shape.k());

  set_argument(result, "raster_order", problem_space, library::to_string(raster_order));
  set_argument(result, "swizzle_size", problem_space, swizzle_size);

  set_argument(result, "cluster_m", problem_space, preferred_cluster[0]);
  set_argument(result, "cluster_n", problem_space, preferred_cluster[1]);
  set_argument(result, "cluster_k", problem_space, preferred_cluster[2]);
  set_argument(result, "cluster_m_fallback", problem_space, fallback_cluster[0]);
  set_argument(result, "cluster_n_fallback", problem_space, fallback_cluster[1]);
  set_argument(result, "cluster_k_fallback", problem_space, fallback_cluster[2]);

}

/// Initializes the performance result
void GemmOperationProfiler::initialize_result_(
  PerformanceResult &result,
  Options const &options,
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.provider = library::Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;

  problem_.initialize_result(result, operation_desc, problem_space);

  OperationProfiler::initialize_result_(result, operation_desc, problem_space);

  result.bytes = problem_.bytes(operation_desc);
  result.flops = problem_.flops(operation_desc);
  result.runtime = 0;
  result.runtime_vector.resize(options.device.devices.size(), 0);

}

/// Initialize reduction problem dimensions and library::Operation
bool GemmOperationProfiler::initialize_reduction_configuration_(
  library::Operation const *operation,
  ProblemSpace::Problem const &problem) {

  library::GemmDescription const &gemm_desc =
    static_cast<library::GemmDescription const&>(operation->description());

  if (!cast_from_double(problem_.alpha_one, gemm_desc.element_epilogue, 1)) {
    return false;
  }

  if (!cast_from_double(problem_.beta_zero, gemm_desc.element_epilogue, 0)) {
    return false;
  }

  /// initialize library::ReductionConfiguration
  for (auto &gemm_workspace : gemm_workspace_) {
    gemm_workspace.reduction_configuration.problem_size      = gemm::GemmCoord(int(problem_.n), int(problem_.m), int(problem_.k)).mn();
    gemm_workspace.reduction_configuration.partitions        = int(problem_.split_k_slices);
    gemm_workspace.reduction_configuration.partition_stride  = gemm::GemmCoord(int(problem_.n), int(problem_.m), int(problem_.k)).mn().product();
    gemm_workspace.reduction_configuration.ldw               = problem_.ldc;
    gemm_workspace.reduction_configuration.lds               = problem_.ldc;
    gemm_workspace.reduction_configuration.ldd               = problem_.ldc;
  }

  // find reduction operation
  library::ReductionFunctionalKey reduction_key(
    library::Provider::kCUTLASS,
    gemm_desc.tile_description.math_instruction.element_accumulator,    // element workspace
    gemm_desc.tile_description.math_instruction.element_accumulator,    // element accumulator
    gemm_desc.D.element,                                                // element output
    gemm_desc.element_epilogue                                          // element compute
  );

  auto reduction_it = library::Singleton::get().operation_table.reduction_operations.find(reduction_key);

  if (reduction_it == library::Singleton::get().operation_table.reduction_operations.end()) {
    return false;
  }

  // initialize reduction operation required for parallel split-k operator
  reduction_op_ = reduction_it->second;

  // reduction operation found and initialized
  return true;
}

/// Initializes workspace
Status GemmOperationProfiler::initialize_workspace(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  hipError_t result;
  result = hipSetDevice(options.device.device_id(0));
  if (result != hipSuccess) {
    throw std::runtime_error("hipSetDevice() failed.");
  }

  library::Operation const* underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      return Status::kErrorNotSupported;
    }
  }

  library::GemmDescription const &operation_desc =
    static_cast<library::GemmDescription const &>(operation->description());

  bool is_sparse = operation_desc.tile_description.math_instruction.opcode_class == cutlass::library::OpcodeClassID::kSparseTensorOp;

  for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
    hipSetDevice(options.device.device_id(i));

    // Compute the number of copies of the problem to avoid L2 camping.
    if (!options.profiling.workspace_count) {
      int64_t bytes = problem_.bytes(operation_desc);
      if (bytes < 3 * int64_t(options.device.properties[0].l2CacheSize)) {
        gemm_workspace_[i].problem_count =
          1 + int((3 * int64_t(options.device.properties[0].l2CacheSize)) / bytes);
      }
      else {
        gemm_workspace_[i].problem_count = 1;
      }
    }
    else {
      gemm_workspace_[i].problem_count = options.profiling.workspace_count;
    }

    bool allocate_device_tensors = options.execution_mode != ExecutionMode::kDryRun;
    if (allocate_device_tensors) {
      bool enable_deep_profiling = options.profiling.enable_kernel_performance_search;
      int seed_shift = 0;

      // When exhaustive performance search (deep profiling) option is enabled, device buffers are initialized to the largest problem shape
      // so that later performance search can re-use those buffers.
      int init_m = enable_deep_profiling ? std::max(int(problem_.m),  problem_.problem_sizes.back().m()) : int(problem_.m);
      int init_n = enable_deep_profiling ? std::max(int(problem_.n),  problem_.problem_sizes.back().n()) : int(problem_.n);
      int init_k = enable_deep_profiling ? std::max(int(problem_.k),  problem_.problem_sizes.back().k()) : int(problem_.k);
      int init_lda = enable_deep_profiling ? int(std::max(problem_.lda,  problem_.leading_dims.back()[0])) : int(problem_.lda);
      int init_ldb = enable_deep_profiling ? int(std::max(problem_.ldb,  problem_.leading_dims.back()[1])) : int(problem_.ldb);
      int init_ldc = enable_deep_profiling ? int(std::max(problem_.ldc,  problem_.leading_dims.back()[2])) : int(problem_.ldc);

      gemm_workspace_[i].A = device_context.allocate_and_initialize_tensor(
        options,
        "A",
        operation_desc.A.element,
        operation_desc.A.layout,
        {init_m, init_k},
        {init_lda},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        seed_shift++,
        i // device_index
      );

      gemm_workspace_[i].B = device_context.allocate_and_initialize_tensor(
        options,
        "B",
        operation_desc.B.element,
        operation_desc.B.layout,
        {init_k, init_n},
        {init_ldb},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        seed_shift++,
        i // device_index
      );

      gemm_workspace_[i].C = device_context.allocate_and_initialize_tensor(
        options,
        "C",
        operation_desc.C.element,
        operation_desc.C.layout,
        {init_m, init_n},
        {init_ldc},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        seed_shift++,
        i // device_index
      );

      gemm_workspace_[i].Computed = device_context.allocate_tensor(
        options,
        "D",
        operation_desc.D.element,
        operation_desc.D.layout,
        {init_m, init_n},
        {init_ldc},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        i // device_index
      );

      gemm_workspace_[i].Reference = device_context.allocate_tensor(
        options,
        "Reference",
        operation_desc.D.element,
        operation_desc.D.layout,
        {init_m, init_n},
        {init_ldc},
        problem_.batch_count * gemm_workspace_[i].problem_count,
        i // device_index
      );

      if (gemm_workspace_[i].arguments.is_sm90_mixed_dtype) {
        // Dequantized tensor has the same shape of the narrow data type tensor,
        // and the same data type as the wide data type tensor
        // Encoded tensor has the same shape and data type of the narrow data type tensor
        if (gemm_workspace_[i].arguments.wider_operand == cutlass::library::Sm90MixedInputWiderOperand::A) {
          gemm_workspace_[i].dequantized_AB = device_context.allocate_tensor(
            options,
            "dequantized-B",
            operation_desc.A.element,
            operation_desc.B.layout,
            {int(problem_.k), int(problem_.n)},
            {int(problem_.ldb)},
            problem_.batch_count * gemm_workspace_[i].problem_count,
            i // device_index
          );
          gemm_workspace_[i].encoded_AB = device_context.allocate_tensor(
            options,
            "encoded-B",
            operation_desc.B.element,
            operation_desc.B.layout,
            {int(problem_.k), int(problem_.n)},
            {int(problem_.ldb)},
            problem_.batch_count * gemm_workspace_[i].problem_count,
            i // device_index
          );
        }
        else {
          gemm_workspace_[i].dequantized_AB = device_context.allocate_tensor(
            options,
            "dequantized-A",
            operation_desc.B.element,
            operation_desc.A.layout,
            {int(problem_.m), int(problem_.k)},
            {int(problem_.lda)},
            problem_.batch_count * gemm_workspace_[i].problem_count,
            i // device_index
          );
          gemm_workspace_[i].encoded_AB = device_context.allocate_tensor(
            options,
            "encoded-A",
            operation_desc.A.element,
            operation_desc.A.layout,
            {int(problem_.m), int(problem_.k)},
            {int(problem_.lda)},
            problem_.batch_count * gemm_workspace_[i].problem_count,
            i // device_index
          );
        }
      }  // End of "if (gemm_workspace_[i].arguments.is_sm90_mixed_dtype)"
    }

    if (options.execution_mode != ExecutionMode::kDryRun) {
      // NOTE: the leading non-batch strides are duplicated here for 3.0 API kernels
      gemm_workspace_[i].arguments.problem_size = {int(problem_.m), int(problem_.n), int(problem_.k)};
      gemm_workspace_[i].arguments.cluster_shape = {int(problem_.cluster_m), int(problem_.cluster_n), int(problem_.cluster_k)}; 
      gemm_workspace_[i].arguments.cluster_shape_fallback = {int(problem_.cluster_m_fallback), int(problem_.cluster_n_fallback), int(problem_.cluster_k_fallback)};
      gemm_workspace_[i].arguments.split_k_slices = problem_.split_k_slices;
      gemm_workspace_[i].arguments.batch_count = problem_.batch_count;
      gemm_workspace_[i].arguments.lda = problem_.lda;
      gemm_workspace_[i].arguments.ldb = problem_.ldb;
      gemm_workspace_[i].arguments.ldc = problem_.ldc;
      gemm_workspace_[i].arguments.ldd = problem_.ldc;
      gemm_workspace_[i].arguments.batch_stride_A = gemm_workspace_[i].A->batch_stride();
      gemm_workspace_[i].arguments.batch_stride_B = gemm_workspace_[i].B->batch_stride();
      gemm_workspace_[i].arguments.batch_stride_C = gemm_workspace_[i].C->batch_stride();
      gemm_workspace_[i].arguments.batch_stride_D = gemm_workspace_[i].Computed->batch_stride();

      /* Query device SM count to pass onto the kernel as an argument, where needed */
      gemm_workspace_[i].arguments.sm_count = options.device.properties[i].multiProcessorCount;
      gemm_workspace_[i].arguments.device_index = static_cast<int>(i);
    }
  }

  //
  // Initialize the CUTLASS operation
  //
  Status status = Status::kSuccess;

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    if (options.execution_mode != ExecutionMode::kDryRun) {
      for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
        hipSetDevice(options.device.device_id(i));
        uint64_t workspace_size = underlying_operation->get_host_workspace_size(&gemm_workspace_[i].configuration);
        gemm_workspace_[i].host_workspace.resize(workspace_size, 0);

        workspace_size = underlying_operation->get_device_workspace_size(&gemm_workspace_[i].configuration,
                                                              &gemm_workspace_[i].arguments);
        if (is_sparse) {
          // sparse gemm get_device_workspace_size() only return device workspace size per iteration
          // Needs to multiply it w/ number of iteration
          workspace_size *= gemm_workspace_[i].problem_count;
        }
        gemm_workspace_[i].device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

        // Convert to structure sparse contents here.
        if (is_sparse) {
          uint8_t* profiler_workspaces[1];
          profiler_workspaces[0] = reinterpret_cast<uint8_t*>(gemm_workspace_[i].A->data());
          // Sparse operations have a different initialize interface.
          // initialize_with_profiler_workspace converts mxk tensorA to compressed mxk/sp tensorA and the tensorE
          auto modifiable_underlying_op = const_cast<library::Operation*>(underlying_operation);
          status = modifiable_underlying_op->initialize_with_profiler_workspace(
            &gemm_workspace_[i].configuration,
            gemm_workspace_[i].host_workspace.data(),
            gemm_workspace_[i].device_workspace.data(),
            profiler_workspaces,
            gemm_workspace_[i].problem_count,
            gemm_workspace_[i].stream);
        }
        else {
          status = underlying_operation->initialize(
            &gemm_workspace_[i].configuration,
            gemm_workspace_[i].host_workspace.data(),
            gemm_workspace_[i].device_workspace.data(),
            gemm_workspace_[i].stream);
        }

        if (status != Status::kSuccess) {
          return status;
        }

        if (problem_.split_k_mode == library::SplitKMode::kParallel) {
          workspace_size = reduction_op_->get_host_workspace_size(&gemm_workspace_[i].reduction_configuration);
          gemm_workspace_[i].reduction_host_workspace.resize(workspace_size, 0);

          status = reduction_op_->initialize(
            &gemm_workspace_[i].reduction_configuration,
            gemm_workspace_[i].reduction_host_workspace.data(),
            nullptr,
            gemm_workspace_[i].stream);

          if (status != Status::kSuccess) {
            return status;
          }
        }
      }
    }

    for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
      hipSetDevice(options.device.device_id(i));
      hipDeviceSynchronize();
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //
    results_.push_back(model_result_);
    results_.back().provider = library::Provider::kCUTLASS;
    results_.back().op_kind = library::OperationKind::kGemm;
    results_.back().disposition = Disposition::kNotRun;

    for (auto provider : verification_providers_) {
      results_.back().verification_map[provider] = Disposition::kNotRun;
    }
  }
  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GemmOperationProfiler::verify_cutlass(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (!options.profiling.provider_enabled(library::Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  // Initialize structure containing GEMM arguments
  for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
    gemm_workspace_[i].arguments.A = gemm_workspace_[i].A->data();
    gemm_workspace_[i].arguments.B = gemm_workspace_[i].B->data();
    gemm_workspace_[i].arguments.C = gemm_workspace_[i].C->data();
    gemm_workspace_[i].arguments.D = gemm_workspace_[i].Computed->data();
    gemm_workspace_[i].arguments.alpha = problem_.alpha.data();
    gemm_workspace_[i].arguments.beta = problem_.beta.data();
    gemm_workspace_[i].arguments.pointer_mode = library::ScalarPointerMode::kHost;
    gemm_workspace_[i].arguments.batch_stride_A = gemm_workspace_[i].A->batch_stride();
    gemm_workspace_[i].arguments.batch_stride_B = gemm_workspace_[i].B->batch_stride();
    gemm_workspace_[i].arguments.batch_stride_C = gemm_workspace_[i].C->batch_stride();
    gemm_workspace_[i].arguments.batch_stride_D = gemm_workspace_[i].Computed->batch_stride();

    if (gemm_workspace_[i].arguments.is_sm90_mixed_dtype) {
      // Scale and zero already generated in initialize_configuration(),
      // A and B already generated in initialize_workspace(), signal
      // GemmUniversal3xOperation::update_arguments_() (trigger by underlying_operation->run())
      // to generate the dequantized matrix for verification
      gemm_workspace_[i].arguments.generate_scale_and_zero = false;
      gemm_workspace_[i].arguments.generate_dequantized_AB = true;
      gemm_workspace_[i].arguments.dequantized_AB = gemm_workspace_[i].dequantized_AB->data();
      gemm_workspace_[i].arguments.encoded_AB = gemm_workspace_[i].encoded_AB->data();
    }

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_[i].arguments.D                       = gemm_workspace_[i].device_workspace.data();
      gemm_workspace_[i].arguments.alpha                   = problem_.alpha_one.data();
      gemm_workspace_[i].arguments.beta                    = problem_.beta_zero.data();

      gemm_workspace_[i].reduction_arguments.workspace     = gemm_workspace_[i].device_workspace.data();
      gemm_workspace_[i].reduction_arguments.source        = gemm_workspace_[i].C->data();
      gemm_workspace_[i].reduction_arguments.destination   = gemm_workspace_[i].Computed->data();
      gemm_workspace_[i].reduction_arguments.alpha         = problem_.alpha.data();
      gemm_workspace_[i].reduction_arguments.beta          = problem_.beta.data();
      gemm_workspace_[i].reduction_arguments.pointer_mode  = library::ScalarPointerMode::kHost;
    }
  }

  //
  // Run the CUTLASS operation
  //

 // initialize gemm underlying operation to handle parallel reduction
  library::Operation const * underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      results_.back().disposition = Disposition::kFailed;
      return false;
    }
  }

  for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
    hipSetDevice(options.device.device_id(i));

    results_.back().status = underlying_operation->run(
     &gemm_workspace_[i].arguments,
     gemm_workspace_[i].host_workspace.data(),
     gemm_workspace_[i].device_workspace.data(),
     gemm_workspace_[i].stream);

    if (results_.back().status != Status::kSuccess) {
      results_.back().disposition = Disposition::kFailed;
      return false;
    }

    // Run parallel reduction kernel for parallel split_k_mode
    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      results_.back().status = reduction_op_->run(
        &gemm_workspace_[i].reduction_arguments,
        gemm_workspace_[i].reduction_host_workspace.data(),
        nullptr,
        gemm_workspace_[i].stream);

      if (results_.back().status != Status::kSuccess) {
        results_.back().disposition = Disposition::kFailed;
        return false;
      }
    }
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  // CUTLASS op ran the but not yet verified against any verification provider
  results_.back().disposition = Disposition::kNotVerified;

  //
  // Run verification providers
  //

  if (options.verification.enabled) {

#if CUTLASS_ENABLE_CUBLAS
    if (options.verification.provider_enabled(library::Provider::kCUBLAS)) {

      // Guard against unsupported cases
      auto const & gemm_desc = static_cast<library::GemmDescription const &>(operation->description());

      if (cublas_satisfies(gemm_desc) == Status::kSuccess) {

        // call cublas verification if supported
        for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
          hipSetDevice(options.device.device_id(i));
          verify_with_cublas_(
           options,
           report,
           device_context,
           operation,
           problem_space,
           problem,
           gemm_workspace_[i]);
        }
        }

      else {
        // set verification map for cublas to not supported
        results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotSupported;
      }
    }
#endif // #if CUTLASS_ENABLE_CUBLAS

    
    cutlass::library::RuntimeDatatype runtime_datatype_a = gemm_workspace_.front().arguments.runtime_input_datatype_a;
    cutlass::library::RuntimeDatatype runtime_datatype_b = gemm_workspace_.front().arguments.runtime_input_datatype_b;

    bool is_runtime_datatype_a = runtime_datatype_a != cutlass::library::RuntimeDatatype::kStatic;
    bool is_runtime_datatype_b = runtime_datatype_b != cutlass::library::RuntimeDatatype::kStatic;

    assert(is_runtime_datatype_a == is_runtime_datatype_b && "runtime datatype should be both dynamic or static.");
    

    library::GemmDescription const &gemm_desc =
      static_cast<library::GemmDescription const &>(operation->description());


    cutlass::library::NumericTypeID element_A = gemm_desc.A.element;
    cutlass::library::NumericTypeID element_B = gemm_desc.B.element;
    
    if (is_runtime_datatype_a) {
      element_A = cutlass::library::dynamic_datatype_to_id(runtime_datatype_a);
    }

    if (is_runtime_datatype_b) {
      element_B = cutlass::library::dynamic_datatype_to_id(runtime_datatype_b);
    }
    

    bool verification_status = verify_with_reference_(options, report, device_context, operation, problem_space, problem, element_A, element_B);

    // Update disposition to worst case verification outcome among all
    // verification providers which are supported
    bool is_any_verification_run_passed = false;
    for (auto &m : results_.back().verification_map) {
      if (m.second == Disposition::kFailed || m.second == Disposition::kIncorrect) {
        results_.back().disposition = m.second;
        return true;
      }
      if (!is_any_verification_run_passed && m.second == Disposition::kPassed) {
        is_any_verification_run_passed = true;
      }
    }

    if (is_any_verification_run_passed) {
      results_.back().disposition = Disposition::kPassed;
    }
  }

  // if verification.required is set, then return success iff at least one ref-check was run
  if (options.verification.required) {
    bool did_any_verification_run = false;
    for (auto provider : options.verification.providers) {
      did_any_verification_run |= (Disposition::kNotRun != results_.back().verification_map[provider]);
    }

    if (not did_any_verification_run) {
      results_.back().status = Status::kErrorNotSupported;
      return false;
    }
  }

  // Return true means continue profiling
  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GemmOperationProfiler::verify_with_cublas_(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem,
  GemmWorkspace &gemm_workspace_) {

#if CUTLASS_ENABLE_CUBLAS

  library::GemmDescription const &gemm_desc =
    static_cast<library::GemmDescription const &>(operation->description());

  //
  // Construct cuBLAS operators
  //

  CublasLtCreate handle;
  hipblasStatus_t status = handle.get_cublaslt_create_status();

  if (status != HIPBLAS_STATUS_SUCCESS) {
    results_.back().verification_map[library::Provider::kCUBLAS] = get_cutlass_disposition(status);
    return true;
  }


  //
  // Initialize state
  //

  try {

    //
    // Construct dispatcher to hipblasGemmEx()
    //

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.batch_stride_A = gemm_workspace_.A->batch_stride();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.batch_stride_B = gemm_workspace_.B->batch_stride();
    gemm_workspace_.arguments.C = gemm_workspace_.Reference->data();
    gemm_workspace_.arguments.batch_stride_C = gemm_workspace_.Reference->batch_stride();
    gemm_workspace_.arguments.D = gemm_workspace_.Reference->data();
    gemm_workspace_.arguments.batch_stride_D = gemm_workspace_.Reference->batch_stride();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    detail::cublasLtGemmExDispatcher gemm_op(
      gemm_desc,
      gemm_workspace_.configuration,
      gemm_workspace_.arguments
    );

    gemm_op.initialize_cublaslt();

    if(!gemm_op.get_cublaslt_algo(handle, AlgorithmMode::kDefault)){
      return true;
    }

    if (gemm_op.status != Status::kSuccess) {
      results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kNotRun;
      return true;
    }

    status = gemm_op(handle);

    // Handle errors
    if (status != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "cublasLt Verification run failed with status : " << cublasLtGetStatusName(status) << "\n";
      results_.back().verification_map[library::Provider::kCUBLAS] = get_cutlass_disposition(status);
      return true;
    }

    results_.back().status = Status::kSuccess;

    //
    // Verify results
    //

    results_.back().verification_map[library::Provider::kCUBLAS] = compare_tensors(
      options,
      *gemm_workspace_.Computed,
      *gemm_workspace_.Reference,
      gemm_workspace_.Computed->batch_stride()
    );

    // Save workspace if incorrect
    if (options.verification.save_workspace == SaveWorkspace::kIncorrect &&
      results_.back().verification_map[library::Provider::kCUBLAS] == Disposition::kIncorrect) {

      save_workspace(
        device_context,
        options,
        gemm_desc,
        library::Provider::kCUTLASS,
        library::Provider::kCUBLAS);
    }
  }
  catch (...) {
    results_.back().verification_map[library::Provider::kCUBLAS] = Disposition::kFailed;
  }

#endif

  // Return true means continue profiling
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against host and device references
bool GemmOperationProfiler::verify_with_reference_(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem,
  cutlass::library::NumericTypeID element_A,
  cutlass::library::NumericTypeID element_B)
{
  library::GemmDescription const &gemm_desc =
    static_cast<library::GemmDescription const &>(operation->description());

  //
  // Initialize state
  //
  for (auto provider : options.verification.providers) {

    // Skip providers that are not enabled
    if (!options.verification.provider_enabled(provider)) {
      continue;
    }

    for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
      hipSetDevice(options.device.device_id(i));

      void *ptr_A = gemm_workspace_[i].A->data();
      void *ptr_B = gemm_workspace_[i].B->data();
      void *ptr_C = gemm_workspace_[i].C->data();
      void *ptr_D = gemm_workspace_[i].Reference->data();

      cutlass::library::NumericTypeID element_A_for_reference = element_A;
      cutlass::library::NumericTypeID element_B_for_reference = element_B;
      if (gemm_workspace_[i].arguments.is_sm90_mixed_dtype) {
        // Dequantized tensor has the same shape of the narrow data type tensor,
        // and the same data type as the wide data type tensor
        if (gemm_workspace_[i].arguments.wider_operand == cutlass::library::Sm90MixedInputWiderOperand::A) {
          ptr_B = gemm_workspace_[i].dequantized_AB->data();
          element_B_for_reference = element_A;
        }
        else {
          ptr_A = gemm_workspace_[i].dequantized_AB->data();
          element_A_for_reference = element_B;
        }
      }

      // To support the host-side reference, conditionally allocate and
      // copy tensors to host memory.
      std::vector<uint8_t> host_data_A;
      std::vector<uint8_t> host_data_B;
      std::vector<uint8_t> host_data_C;
      std::vector<uint8_t> host_data_D;

      if (provider == library::Provider::kReferenceHost) {

        host_data_A.resize(gemm_workspace_[i].A->bytes());
        ptr_A = host_data_A.data();
        gemm_workspace_[i].A->copy_to_host(ptr_A);

        host_data_B.resize(gemm_workspace_[i].B->bytes());
        ptr_B = host_data_B.data();
        gemm_workspace_[i].B->copy_to_host(ptr_B);

        host_data_C.resize(gemm_workspace_[i].C->bytes());
        ptr_C = host_data_C.data();
        gemm_workspace_[i].C->copy_to_host(ptr_C);

        host_data_D.resize(gemm_workspace_[i].Reference->bytes());
        ptr_D = host_data_D.data();
      }

      //
      // Launch
      //

      library::Handle handle;

      handle.set_provider(provider);

      Status status = handle.gemm_universal(
        problem_.mode,
        gemm_workspace_[i].configuration.problem_size.m(),
        gemm_workspace_[i].configuration.problem_size.n(),
        gemm_workspace_[i].configuration.problem_size.k(),
        
        gemm_workspace_[i].configuration.cluster_shape.m(),
        gemm_workspace_[i].configuration.cluster_shape.n(),
        gemm_workspace_[i].configuration.cluster_shape.k(),
        gemm_workspace_[i].configuration.cluster_shape_fallback.m(),
        gemm_workspace_[i].configuration.cluster_shape_fallback.n(),
        gemm_workspace_[i].configuration.cluster_shape_fallback.k(),
        
        gemm_desc.tile_description.math_instruction.element_accumulator,
        gemm_desc.element_epilogue,

        problem_.alpha.data(),

        element_A_for_reference,
        gemm_desc.A.layout,
        gemm_desc.transform_A,
        ptr_A,
        int(gemm_workspace_[i].configuration.lda),

        element_B_for_reference,
        gemm_desc.B.layout,
        gemm_desc.transform_B,
        ptr_B,
        int(gemm_workspace_[i].configuration.ldb),

        problem_.beta.data(),

        gemm_desc.C.element,
        gemm_desc.C.layout,
        ptr_C,
        int(gemm_workspace_[i].configuration.ldc),

        gemm_desc.D.element,
        gemm_desc.D.layout,
        ptr_D,
        int(gemm_workspace_[i].configuration.ldd),

        gemm_workspace_[i].configuration.batch_count,
        gemm_workspace_[i].A->batch_stride(),
        gemm_workspace_[i].B->batch_stride(),
        gemm_workspace_[i].C->batch_stride(),
        gemm_workspace_[i].Reference->batch_stride());

      if (status != Status::kSuccess) {
        results_.back().verification_map[provider] = Disposition::kNotRun;
        continue;
      }
      results_.back().status = status;

      if (provider == library::Provider::kReferenceHost) {
        gemm_workspace_[i].Reference->copy_from_host(ptr_D);
      }

      //
      // Verify results
      //

      results_.back().verification_map[provider] = compare_tensors(
        options,
        *gemm_workspace_[i].Computed,
        *gemm_workspace_[i].Reference,
        gemm_workspace_[i].Computed->batch_stride()
      );

      // Save workspace if incorrect
      if (options.verification.save_workspace == SaveWorkspace::kIncorrect &&
        results_.back().verification_map[provider] == Disposition::kIncorrect) {

        save_workspace(
          device_context,
          options,
          gemm_desc,
          library::Provider::kCUTLASS,
          provider);
        }
    }
  }

  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool GemmOperationProfiler::profile(
  Options const &options,
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.profiling.provider_enabled(library::Provider::kCUTLASS)) {

    // Case when we either screen the best performance number of kernels with or without a fixed problem shape fed in.
    if (options.profiling.enable_kernel_performance_search || options.profiling.enable_best_kernel_for_fixed_shape) {
      library::GemmDescription const &operation_desc =
        static_cast<library::GemmDescription const &>(operation->description());

      auto min_cc = operation_desc.tile_description.minimum_compute_capability;

      bool is_dynamic_cluster_enabled = (min_cc >= 100);

      // Helper function wrapping up performance test with flexible parameters.
      auto initialize_and_profile = [&](
        PerformanceResult const &result,
        gemm::GemmCoord const &problem_shape,
        std::array<int64_t, 3> const &leading_dim,
        std::array<int64_t, 3> const &preferred_cluster,
        std::array<int64_t, 3> const &fallback_cluster,
        cutlass::library::RasterOrder const &raster_order,
        int swizzle_size) -> std::optional<PerformanceResult> {

        for (size_t i = 0; i < gemm_workspace_.size(); ++i) {
          // Initialize structure containing GEMM arguments
          auto& workspace = gemm_workspace_[i];
          workspace.arguments.A = workspace.A->data();
          workspace.arguments.B = workspace.B->data();
          workspace.arguments.C = workspace.C->data();
          workspace.arguments.D = workspace.Computed->data();
          workspace.arguments.alpha = problem_.alpha.data();
          workspace.arguments.beta = problem_.beta.data();
          workspace.arguments.pointer_mode = library::ScalarPointerMode::kHost;
          workspace.arguments.batch_stride_A = workspace.A->batch_stride();
          workspace.arguments.batch_stride_B = workspace.B->batch_stride();
          workspace.arguments.batch_stride_C = workspace.C->batch_stride();
          workspace.arguments.batch_stride_D = workspace.Computed->batch_stride();

          if (problem_.split_k_mode == library::SplitKMode::kParallel) {
            workspace.arguments.D = workspace.device_workspace.data();
            workspace.arguments.alpha = problem_.alpha_one.data();
            workspace.arguments.beta = problem_.beta_zero.data();

            workspace.reduction_arguments.workspace = workspace.device_workspace.data();
            workspace.reduction_arguments.source = workspace.C->data();
            workspace.reduction_arguments.destination = workspace.Computed->data();
            workspace.reduction_arguments.alpha = problem_.alpha.data();
            workspace.reduction_arguments.beta = problem_.beta.data();
            workspace.reduction_arguments.pointer_mode = library::ScalarPointerMode::kHost;
          }

          update_workspace_(workspace, problem_shape, leading_dim, preferred_cluster, fallback_cluster, raster_order, swizzle_size);

          const auto can_implement = operation->can_implement(&workspace.configuration, &workspace.arguments);
          if (can_implement != Status::kSuccess) {
            return std::nullopt;  // Return nullopt to indicate failure
          }
          library::Operation const* underlying_operation = operation;
          hipSetDevice(options.device.device_id(i));
          uint64_t workspace_size = underlying_operation->get_host_workspace_size(&workspace.configuration);
          workspace.host_workspace.resize(workspace_size, 0);

          workspace_size = underlying_operation->get_device_workspace_size(&workspace.configuration,
                                                                &workspace.arguments);
          
          bool is_sparse = operation_desc.tile_description.math_instruction.opcode_class == cutlass::library::OpcodeClassID::kSparseTensorOp;
          if (is_sparse) {
            // sparse gemm get_device_workspace_size() only return device workspace size per iteration
            // Needs to multiply it w/ number of iteration
            workspace_size *= workspace.problem_count;
          }

          workspace.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

          Status status = Status::kSuccess;

          if (is_sparse) {
            uint8_t* profiler_workspaces[1];
            profiler_workspaces[0] = reinterpret_cast<uint8_t*>(workspace.A->data());
            // Sparse operations have a different initialize interface.
            // initialize_with_profiler_workspace converts mxk tensorA to compressed mxk/sp tensorA and the tensorE
            auto modifiable_underlying_op = const_cast<library::Operation*>(underlying_operation);
            status = modifiable_underlying_op->initialize_with_profiler_workspace(
              &workspace.configuration,
              workspace.host_workspace.data(),
              workspace.device_workspace.data(),
              profiler_workspaces,
              workspace.problem_count,
              workspace.stream);
          }
          else {
            status = underlying_operation->initialize(
              &workspace.configuration,
              workspace.host_workspace.data(),
              workspace.device_workspace.data(),
              workspace.stream);
          }

          if (status != Status::kSuccess) {
            return std::nullopt;  // Return nullopt to indicate failure
          }

        }

        PerformanceResult curr_result(result);
        update_result_(curr_result, operation_desc, problem_space, problem_shape, raster_order, preferred_cluster, fallback_cluster, swizzle_size);

        curr_result.status = profile_cutlass_(
          curr_result,
          options,
          operation,
          nullptr,
          nullptr,
          nullptr
        );

        return curr_result;
      };

      // Helper function to test validity of fallback cluster shapes and preferred cluster shapes.
      auto is_valid_dynamic_cluster_shape = [](const std::array<int64_t, 3>& preferred_cluster, const std::array<int64_t, 3>& fallback_cluster) {
        for (size_t i = 0; i < 3; ++i) {
          if (preferred_cluster[i] % fallback_cluster[i] != 0) {
            return false;
          }
        }
        return true;
      };

      // Helper function to select the best performance number among a list.
      auto select_best_candidate = [&](std::vector<PerformanceResult> &candidates) {
        assert(!candidates.empty() && "Candidates vector should not be empty");
        auto best_iter = std::max_element(
          candidates.begin(), candidates.end(),
          [](PerformanceResult const &a, PerformanceResult const &b) {
            return a.gflops_per_sec() < b.gflops_per_sec();
          }
        );
        assert(best_iter != candidates.end() && "No candidate found despite non-empty candidates vector");
        results_.push_back(std::move(*best_iter));
      };

      std::vector<PerformanceResult> candidates;
      PerformanceResult result_base = results_.back();
      results_.pop_back();
      
      bool dynamic_cluster = int64_t(operation_desc.tile_description.cluster_shape.m()) == 0 ||
                             int64_t(operation_desc.tile_description.cluster_shape.n()) == 0 ||
                             int64_t(operation_desc.tile_description.cluster_shape.k()) == 0;

      std::vector<std::array<int64_t, 3>> preferred_clusters;
      std::vector<std::array<int64_t, 3>> fallback_clusters;

      // Only loop over built-in cluster shape lists for dynamic cluster kernels
      // and for kernels that can leverage the dynamic cluster feature.
      if (dynamic_cluster && is_dynamic_cluster_enabled) {
        preferred_clusters = this->problem_.preferred_clusters;
        fallback_clusters = this->problem_.fallback_clusters;
      } 
      else {
        preferred_clusters = {{int(problem_.cluster_m), int(problem_.cluster_n), int(problem_.cluster_k)}};
        fallback_clusters = {{int(problem_.cluster_m_fallback), int(problem_.cluster_n_fallback), int(problem_.cluster_k_fallback)}};
      }

      for (auto preferred_cluster : preferred_clusters) {
        for (auto fallback_cluster : fallback_clusters) {
          if (dynamic_cluster && !is_valid_dynamic_cluster_shape(preferred_cluster, fallback_cluster)) {
            continue;
          }
          for (auto swizzle_size : this->problem_.swizzle_sizes) {
            for (auto raster_order : this->problem_.raster_orders) {
              // With the fixed shape option turned on, only a specific problem shape is tested.
              if (options.profiling.enable_best_kernel_for_fixed_shape) {
                this->problem_.problem_sizes = {{int(this->problem_.m), int(this->problem_.n), int(this->problem_.k)}};
                this->problem_.leading_dims = {{this->problem_.lda, this->problem_.ldb, this->problem_.ldc}};
              }

              for (int i = 0; i < int(this->problem_.problem_sizes.size()); i++) {
                gemm::GemmCoord problem_shape = problem_.problem_sizes[i];
                std::array<int64_t, 3> leading_dim = problem_.leading_dims[i];
                auto result_opt = initialize_and_profile(result_base, problem_shape, leading_dim, preferred_cluster, fallback_cluster, raster_order, swizzle_size);
                  
                if (result_opt) {  // Only add valid results
                  candidates.push_back(*result_opt);
                }

              }

            }// for raster_order
          }// for swizzle_size
        }// for fallback_cluster
      }// for swizzle_size

      if (candidates.empty()) {
        return false;
      }

      select_best_candidate(candidates);
    }
    // Basic case where we benchmark input parameters only.
    else {
      results_.back().status = profile_cutlass_(
        results_.back(),
        options,
        operation,
        nullptr,
        nullptr,
        nullptr
      );
    }

  }
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Method to profile a CUTLASS Operation
Status GemmOperationProfiler::profile_cutlass_(
  PerformanceResult &result,
  Options const &options,
  library::Operation const *operation,
  void *,
  void *,
  void *) {

  // initialize gemm underlying operation to handle parallel reduction
  library::Operation const * underlying_operation = operation;

  if (problem_.split_k_mode == library::SplitKMode::kParallel) {
    if (!(underlying_operation = library::find_gemm_operation_for_parallel_reduction(operation))) {
      return Status::kErrorNotSupported;
    }
  }

  auto launch_gemm = [&](int dev_id, hipStream_t stream, int iteration) {
    int problem_idx = (iteration % gemm_workspace_[dev_id].problem_count) * problem_.batch_count;

    gemm_workspace_[dev_id].arguments.A = gemm_workspace_[dev_id].A->batch_data(problem_idx);
    gemm_workspace_[dev_id].arguments.B = gemm_workspace_[dev_id].B->batch_data(problem_idx);
    gemm_workspace_[dev_id].arguments.C = gemm_workspace_[dev_id].C->batch_data(problem_idx);
    gemm_workspace_[dev_id].arguments.D = gemm_workspace_[dev_id].Computed->batch_data(problem_idx);

      if (gemm_workspace_[dev_id].arguments.is_sm90_mixed_dtype) {
        // Scale, zero, and dequantized tensors are already generated in
        // verify_cutlass(), no need to re-generate them in profiling
        gemm_workspace_[dev_id].arguments.generate_scale_and_zero = false;
        gemm_workspace_[dev_id].arguments.generate_dequantized_AB = false;
      }

    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      gemm_workspace_[dev_id].arguments.D                     = gemm_workspace_[dev_id].device_workspace.data();

      gemm_workspace_[dev_id].reduction_arguments.workspace   = gemm_workspace_[dev_id].device_workspace.data();
      gemm_workspace_[dev_id].reduction_arguments.source      = gemm_workspace_[dev_id].C->batch_data(problem_idx);
      gemm_workspace_[dev_id].reduction_arguments.destination = gemm_workspace_[dev_id].Computed->batch_data(problem_idx);
    }

    // Execute the CUTLASS operation
    Status status = underlying_operation->run(
      &gemm_workspace_[dev_id].arguments,
      gemm_workspace_[dev_id].host_workspace.data(),
      gemm_workspace_[dev_id].device_workspace.data(),
      stream);

    if (status != Status::kSuccess) {
      return status;
    }

    // Run parallel reduction kernel for parallel split_k_mode
    if (problem_.split_k_mode == library::SplitKMode::kParallel) {
      status = reduction_op_->run(
        &gemm_workspace_[dev_id].reduction_arguments,
        gemm_workspace_[dev_id].reduction_host_workspace.data(),
        nullptr,
        gemm_workspace_[dev_id].stream);

      if (status != Status::kSuccess) {
        return status;
      }
    }
    return Status::kSuccess;
  };

  std::vector<hipStream_t> streams(gemm_workspace_.size());
  for (size_t i = 0; i < streams.size(); i++) {
    streams[i] = gemm_workspace_[i].stream;
  }
  return profile_kernel_(result, options, launch_gemm, streams);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
