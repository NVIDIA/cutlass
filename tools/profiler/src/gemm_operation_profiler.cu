#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Execution environment
*/

#include <iostream>
#include <stdexcept>
#include <iomanip>
#include <ios>

#include "cublas_helpers.h"
#include "gemm_operation_profiler.h"
#include "gpu_timer.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace profiler {


/////////////////////////////////////////////////////////////////////////////////////////////////

/// Ctor
GemmOperationProfiler::GemmOperationProfiler(): 
  OperationProfiler(library::OperationKind::kGemm,{
  	{ArgumentTypeID::kEnumerated, {"Gemm_kind"}, "Variant of GEMM (e.g. gemm, planar complex, batched, ...)"},
	  {ArgumentTypeID::kInteger, {"m", "problem-size::m"}, "M dimension of the GEMM problem space"},
  	{ArgumentTypeID::kInteger, {"n", "problem-size::n"}, "N dimension of the GEMM problem space"},
	  {ArgumentTypeID::kInteger, {"k", "problem-size::k"}, "K dimension of the GEMM problem space"},
  	{ArgumentTypeID::kTensor, {"A"}, "Tensor storing the A operand"},
	  {ArgumentTypeID::kTensor, {"B"}, "Tensor storing the B operand"},
  	{ArgumentTypeID::kTensor, {"C"}, "Tensor storing the C operand"},
	  {ArgumentTypeID::kScalar, {"alpha", "epilogue::alpha"}, "Epilogue scalar alpha"},
  	{ArgumentTypeID::kScalar, {"beta", "epilogue::beta"}, "Epilogue scalar beta"},
	  {ArgumentTypeID::kInteger, {"split_k_slices"}, "Number of partitions of K dimension"},
  	{ArgumentTypeID::kInteger, {"batch_count"}, "Number of GEMMs computed in one batch"},
  }) {

  description_ = "General matrix-matrix product. D = alpha * A*B + beta * C";
}

/// Destructor
GemmOperationProfiler::~GemmOperationProfiler() {

}

/// Prints usage statement for the math function
void GemmOperationProfiler::print_usage(std::ostream &out) const {
  out << "GEMM" << "\n\n";

  OperationProfiler::print_usage(out);
}

/// Prints examples
void GemmOperationProfiler::print_examples(std::ostream &out) const {

  out << "\nExamples:\n\n"
    << "Profile a particular problem size:\n"
    << "  $ cutlass_profiler --operation=Gemm --m=1024 --n=1024 --k=128\n\n"

    << "Schmoo over problem size and beta:\n"
    << "  $ cutlass_profiler --operation=Gemm --m=1024:4096:256 --n=1024:4096:256 --k=128:8192:128 --beta=0,1,2.5\n\n"

    << "Schmoo over accumulator types:\n"
    << "  $ cutlass_profiler --operation=Gemm --accumulator-type=f16,f32\n\n"

    << "Run when A is f16 with column-major and B is any datatype with row-major (For column major, use column, col, or n. For row major use, row or t):\n"
    << "  $ cutlass_profiler --operation=Gemm --A=f16:column --B=*:row\n\n"

    << "Using various input value distribution:\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=uniform,min:0,max:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=gaussian,mean:0,stddev:3\n"
    << "  $ cutlass_profiler --operation=Gemm --dist=sequential,start:0,delta:1\n\n"

    << "Run a kernel with cta tile size of 256x128x32 and save workspace if results are incorrect (note that --cta-tile::k=32 is default cta-tile size):\n"
    << " $ cutlass_profiler --operation=Gemm --cta_m=256 --cta_n=128  --cta_k=32 --save-workspace=incorrect\n\n"
    
    << "Test your changes to gemm kernels with a quick functional test and save results in functional-test.csv:\n"
    << " $ cutlass_profiler  --operation=Gemm \\ \n"
    << "   --m=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --n=8,56,120,136,256,264,512,520,1024,1032,4096,8192,16384 \\ \n"
    << "   --k=8,16,32,64,128,256,288,384,504,512,520 \\ \n"
    << "   --beta=0,1,2 --profiling-iterations=1 \\ \n"
    << "   --providers=cutlass --output=functional-test.csv\n\n";
}

#if 0
// used this for debugging
static std::string byte_string(std::vector<uint8_t> const &bytes) {
  std::stringstream ss;

  ss << "0x";

  for (size_t idx = bytes.size(); idx > 0; --idx) {
    ss << std::hex << std::setw(2) << std::setfill('0') << uint32_t(bytes.at(idx - 1));
  }

  return ss.str();
}
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Extracts the problem dimensions
Status GemmOperationProfiler::initialize_configuration(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::GemmDescription const &operation_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  if (operation_desc.gemm_kind != library::GemmKind::kGemm) {
    return Status::kErrorInvalidProblem;
  }


  if (!arg_as_int(problem_.m, "m", problem_space, problem)) {
    // default value
    problem_.m = 1024;
  }

  if (!arg_as_int(problem_.n, "n", problem_space, problem)) {
    // default value
    problem_.n = 1024;
  }
  
  if (!arg_as_int(problem_.k, "k", problem_space, problem)) {
    // default value
    problem_.k = 1024;
  }
  
  if (!arg_as_int(problem_.split_k_slices, "split_k_slices", problem_space, problem)) {
    // default value
    problem_.split_k_slices = 1;
  }
  
  if (!arg_as_int(problem_.batch_count, "batch_count", problem_space, problem)) {
    // default value
    problem_.batch_count = 1;
  }

  if (!tensor_description_satisfies(operation_desc.A, "A", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.B, "B", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!tensor_description_satisfies(operation_desc.C, "C", problem_space, problem)) {
    return Status::kErrorInvalidProblem;
  }

  if (!arg_as_scalar(
    problem_.alpha, 
    operation_desc.element_epilogue, 
    "alpha", 
    problem_space, 
    problem)) {

    if (!cast_from_double(problem_.alpha, operation_desc.element_epilogue, 1)) {
      return Status::kErrorInternal;
    }
  }
  
  if (!arg_as_scalar(
    problem_.beta, 
    operation_desc.element_epilogue, 
    "beta", 
    problem_space, 
    problem)) {
    
    if (!cast_from_double(problem_.beta, operation_desc.element_epilogue, 0)) {
      return Status::kErrorInternal;
    }
  }

  problem_.lda = DeviceAllocation::get_packed_layout(
    operation_desc.A.layout, {int(problem_.m), int(problem_.k)}).front();

  problem_.ldb = DeviceAllocation::get_packed_layout(
    operation_desc.B.layout, {int(problem_.k), int(problem_.n)}).front();

  problem_.ldc = DeviceAllocation::get_packed_layout(
    operation_desc.C.layout, {int(problem_.m), int(problem_.n)}).front();

  gemm_workspace_.configuration.problem_size.m() = int(problem_.m);
  gemm_workspace_.configuration.problem_size.n() = int(problem_.n);
  gemm_workspace_.configuration.problem_size.k() = int(problem_.k);
  gemm_workspace_.configuration.lda = problem_.lda;
  gemm_workspace_.configuration.ldb = problem_.ldb;
  gemm_workspace_.configuration.ldc = problem_.ldc;
  gemm_workspace_.configuration.ldd = problem_.ldc;
  gemm_workspace_.configuration.split_k_slices = int(problem_.split_k_slices);

  gemm_workspace_.arguments.A = nullptr;
  gemm_workspace_.arguments.B = nullptr;
  gemm_workspace_.arguments.C = nullptr;
  gemm_workspace_.arguments.D = nullptr;
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  initialize_result_(this->model_result_, options, operation_desc, problem_space);
  
  return operation->can_implement(&gemm_workspace_.configuration, &gemm_workspace_.arguments);
}

/// Initializes the performance result
void GemmOperationProfiler::initialize_result_(
  PerformanceResult &result,
  Options const &options,  
  library::GemmDescription const &operation_desc,
  ProblemSpace const &problem_space) {

  result.provider = Provider::kCUTLASS;
  result.disposition = Disposition::kNotRun;
  result.status = Status::kSuccess;
  result.operation_name = operation_desc.name;

  result.arguments.resize(problem_space.rank());

  set_argument_(result, "A", problem_space,
    std::string(library::to_string(operation_desc.A.element)) + ":" + library::to_string(operation_desc.A.layout));

  set_argument_(result, "B", problem_space,
    std::string(library::to_string(operation_desc.B.element)) + ":" + library::to_string(operation_desc.B.layout));

  set_argument_(result, "C", problem_space,
    std::string(library::to_string(operation_desc.C.element)) + ":" + library::to_string(operation_desc.C.layout));

  set_argument_(result, "m", problem_space, problem_.m);
  set_argument_(result, "n", problem_space, problem_.n);
  set_argument_(result, "k", problem_space, problem_.k);

  set_argument_(result, "split_k_slices", problem_space, problem_.split_k_slices);
  set_argument_(result, "batch_count", problem_space, problem_.batch_count);

  set_argument_(result, "alpha", problem_space,
    library::lexical_cast(problem_.alpha, operation_desc.element_epilogue));

  set_argument_(result, "beta", problem_space,
    library::lexical_cast(problem_.beta, operation_desc.element_epilogue));

  OperationProfiler::initialize_result_(result, operation_desc, problem_space);

  result.bytes = 
    int64_t(library::sizeof_bits(operation_desc.A.element) * problem_.m / 8) * problem_.k + 
    int64_t(library::sizeof_bits(operation_desc.B.element) * problem_.n / 8) * problem_.k +
    int64_t(library::sizeof_bits(operation_desc.C.element) * problem_.m / 8) * problem_.n * 2;

  result.flops = 2 * (problem_.m * problem_.n * problem_.k + problem_.m * problem_.n);

  result.runtime = 0;

}

/// Initializes workspace
Status GemmOperationProfiler::initialize_workspace(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  library::GemmDescription const &operation_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  if (options.execution_mode != ExecutionMode::kDryRun) {

    gemm_workspace_.A = device_context.allocate_tensor(
      options,
      "A",
      operation_desc.A.element,
      operation_desc.A.layout,
      {int(problem_.m), int(problem_.k)},
      {int(problem_.lda)}
    );

    gemm_workspace_.B = device_context.allocate_tensor(
      options,
      "B",
      operation_desc.B.element,
      operation_desc.B.layout,
      {int(problem_.k), int(problem_.n)},
      {int(problem_.ldb)}
    );

    gemm_workspace_.C = device_context.allocate_tensor(
      options,
      "C",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)}
    );

    gemm_workspace_.Computed = device_context.allocate_tensor(
      "D",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)}
    );

    gemm_workspace_.Reference = device_context.allocate_tensor(
      "Reference",
      operation_desc.C.element,
      operation_desc.C.layout,
      {int(problem_.m), int(problem_.n)},
      {int(problem_.ldc)}
    );

    gemm_workspace_.Reference->copy_from_device(gemm_workspace_.C->data());
  }


  //
  // Initialize the CUTLASS operation
  //
  Status status = Status::kSuccess;

  if (options.profiling.provider_enabled(Provider::kCUTLASS)) {

    if (options.execution_mode != ExecutionMode::kDryRun) {

      uint64_t workspace_size = operation->get_host_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.host_workspace.resize(workspace_size, 0);

      workspace_size = operation->get_device_workspace_size(&gemm_workspace_.configuration);
      gemm_workspace_.device_workspace.reset(library::NumericTypeID::kU8, workspace_size);

      status = operation->initialize(
        &gemm_workspace_.configuration,
        gemm_workspace_.host_workspace.data(),
        gemm_workspace_.device_workspace.data());
    }

    //
    // If CUTLASS is enabled, generate a result for it
    //
    results_.push_back(model_result_);
    results_.back().provider = Provider::kCUTLASS;
    results_.back().disposition = Disposition::kNotRun;
  }

  return status;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GemmOperationProfiler::verify_cutlass(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (!options.profiling.provider_enabled(Provider::kCUTLASS)) {
    return true;
  }

  if (options.execution_mode == ExecutionMode::kDryRun) {
    return true;
  }

  // Initialize structure containing GEMM arguments
  gemm_workspace_.arguments.A = gemm_workspace_.A->data();
  gemm_workspace_.arguments.B = gemm_workspace_.B->data();
  gemm_workspace_.arguments.C = gemm_workspace_.C->data();
  gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
  gemm_workspace_.arguments.alpha = problem_.alpha.data();
  gemm_workspace_.arguments.beta = problem_.beta.data();
  gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

  //
  // Run the CUTLASS operation
  //

  results_.back().status = operation->run(
    &gemm_workspace_.arguments, 
    gemm_workspace_.host_workspace.data(),
    gemm_workspace_.device_workspace.data());

  if (results_.back().status != Status::kSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    results_.back().disposition = Disposition::kFailed;
    return false;
  }

  results_.back().disposition = Disposition::kNotVerified;

  if (options.verification.enabled) {

#if CUTLASS_ENABLE_CUBLAS
    if (options.verification.provider_enabled(Provider::kCUBLAS)) {

      // Guard against unsupported cases
      auto const & gemm_desc = static_cast<library::GemmDescription const &>(operation->description());

      if (cublas_satisfies(gemm_desc) != Status::kSuccess) {
        return true;
      }

      return verify_with_cublas_(
        options,
        report,
        device_context,
        operation,
        problem_space,
        problem);
    }
#endif // #if CUTLASS_ENABLE_CUBLAS
    
  }

  return true;
}


/////////////////////////////////////////////////////////////////////////////////////////////////

#if CUTLASS_ENABLE_CUBLAS

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace detail {

/// Selects one or more cuBLAS algorithms.
static void select_cublas_algorithms(
  std::vector<hipblasGemmAlgo_t> &algorithms,
  Options const &options, 
  library::GemmDescription const &op_desc) {

  library::OpcodeClassID const & opcode_class = 
    op_desc.tile_description.math_instruction.opcode_class;

  switch (options.library.algorithm_mode) {
    case AlgorithmMode::kMatching:
    {
      algorithms.push_back(get_cublas_gemm_algo(
        op_desc.tile_description.threadblock_shape.m(), 
        op_desc.tile_description.threadblock_shape.n(), 
        op_desc.tile_description.threadblock_shape.k(), 
        opcode_class));
      break;
    }

    case AlgorithmMode::kBest:
    {
      // Choose first enumerated mode. If none are enumerated, choose based on opcode class
      // and evaluate all of them.

      if (options.library.algorithms.empty()) {
        // Enumerate all algorithms
        if (opcode_class == library::OpcodeClassID::kSimt) {
          
          for (int algo = HIPBLAS_GEMM_DEFAULT; 
            algo <= CUBLAS_GEMM_ALGO23; 
            ++algo) {

            algorithms.push_back(hipblasGemmAlgo_t(algo));
          }
        }
        else {
          
          for (int algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP; 
            algo <= CUBLAS_GEMM_ALGO15_TENSOR_OP; 
            ++algo) {

            algorithms.push_back(hipblasGemmAlgo_t(algo));
          }
        }
      }
      else {
        // Use the listed algorithms
        algorithms.reserve(options.library.algorithms.size());

        for (int algo : options.library.algorithms) {
          algorithms.push_back(reinterpret_cast<hipblasGemmAlgo_t const &>(algo));
        }
      }

      break;
    }

    case AlgorithmMode::kDefault:
    {

      // Use the library's default algorithm
      algorithms.push_back((opcode_class == library::OpcodeClassID::kSimt ? 
        HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)); 

      break;
    }
    default:
    {
      break;
    }
  }
}

/// Dispatcher to hipblasGemmEx() 
struct cublasGemmExDispatcher {

  //
  // Data members
  //
  library::GemmConfiguration configuration;
  library::GemmArguments arguments;

  hipblasOperation_t trans_A;
  hipblasOperation_t trans_B;
  hipDataType data_type_A;
  hipDataType data_type_B;
  hipDataType data_type_C;
  hipDataType compute_type;
  hipblasGemmAlgo_t algo;
  Status status;
  
  //
  // Methods
  //

  cublasGemmExDispatcher( 
    library::GemmDescription const &op_desc,
    library::GemmConfiguration configuration_,
    library::GemmArguments arguments_,
    hipblasGemmAlgo_t algorithm = HIPBLAS_GEMM_DEFAULT
  ):
    configuration(configuration_), arguments(arguments_), algo(algorithm), status(Status::kSuccess) {

    trans_A = get_cublas_transpose_operation(op_desc.A.layout);
    trans_B = get_cublas_transpose_operation(op_desc.B.layout);

    bool good = true;
    good = (good && get_cublas_datatype(data_type_A, op_desc.A.element));
    good = (good && get_cublas_datatype(data_type_B, op_desc.B.element));
    good = (good && get_cublas_datatype(data_type_C, op_desc.C.element));

    good = (good && get_cublas_datatype(
      compute_type, 
      op_desc.tile_description.math_instruction.element_accumulator));

    if (!good) {
      status = Status::kErrorNotSupported;
    }
  }

  /// Executes GEMM using these arguments
  hipblasStatus_t operator()(hipblasHandle_t handle) {

    return hipblasGemmEx(
      handle,
      trans_A,
      trans_B,
      configuration.problem_size.m(),
      configuration.problem_size.n(),
      configuration.problem_size.k(),
      arguments.alpha,
      arguments.A,
      data_type_A,
      int(configuration.lda),
      arguments.B,
      data_type_B,
      int(configuration.ldb),
      arguments.beta,
      arguments.D,
      data_type_C,
      int(configuration.ldc),
      compute_type,
      algo
    );
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace detail

#endif // CUTLASS_ENABLE_CUBLAS

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Verifies CUTLASS against references
bool GemmOperationProfiler::verify_with_cublas_(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {


#if CUTLASS_ENABLE_CUBLAS

  library::GemmDescription const &gemm_desc = 
    static_cast<library::GemmDescription const &>(operation->description());

  CublasCreate handle;
  hipblasStatus_t status = handle.get_cublas_create_status();

  if (status != HIPBLAS_STATUS_SUCCESS) {

    results_.back().status = get_cutlass_status(status);
    results_.back().disposition = Disposition::kFailed;
    
    return true;
  }

  std::vector<hipblasGemmAlgo_t> algorithms;

  detail::select_cublas_algorithms(
    algorithms, 
    options, 
    gemm_desc);

  if (algorithms.empty()) {
    // no algorithm selected
    return true;
  }

  //
  // Initialize state
  //

  try {

    //
    // Construct dispatcher to hipblasGemmEx()
    //

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.C = gemm_workspace_.Reference->data();
    gemm_workspace_.arguments.D = gemm_workspace_.Reference->data();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    detail::cublasGemmExDispatcher gemm_op( 
      gemm_desc, 
      gemm_workspace_.configuration,
      gemm_workspace_.arguments,
      algorithms.front()
    );

    if (gemm_op.status != Status::kSuccess) {
      results_.back().disposition = Disposition::kNotVerified;
      return true;
    }

    results_.back().status = Status::kSuccess;

    status = gemm_op(handle);

    // Handle errors
    if (status != HIPBLAS_STATUS_SUCCESS) {
      results_.back().status = get_cutlass_status(status);
      results_.back().disposition = Disposition::kNotVerified;
      return true;
    }

    //
    // Verify results
    //

    results_.back().disposition = compare_tensors(
      options,
      *gemm_workspace_.Computed,
      *gemm_workspace_.Reference
    );

    // Save workspace if incorrect
    if (options.verification.save_workspace == SaveWorkspace::kIncorrect && 
      results_.back().disposition == Disposition::kIncorrect) {

      save_workspace(
        device_context,
        options,
        gemm_desc,
        Provider::kCUTLASS,
        Provider::kCUBLAS);
    }
  }
  catch (...) {
    results_.back().disposition = Disposition::kFailed;
    results_.back().status = Status::kErrorNotSupported;
  }

#endif

  // Return true means continue profiling
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Measures performance results
bool GemmOperationProfiler::profile(
  Options const &options,  
  PerformanceReport &report,
  DeviceContext &device_context,
  library::Operation const *operation,
  ProblemSpace const &problem_space,
  ProblemSpace::Problem const &problem) {

  if (options.profiling.provider_enabled(Provider::kCUTLASS)) {

    // Initialize structure containing GEMM arguments
    gemm_workspace_.arguments.A = gemm_workspace_.A->data();
    gemm_workspace_.arguments.B = gemm_workspace_.B->data();
    gemm_workspace_.arguments.C = gemm_workspace_.C->data();
    gemm_workspace_.arguments.D = gemm_workspace_.Computed->data();
    gemm_workspace_.arguments.alpha = problem_.alpha.data();
    gemm_workspace_.arguments.beta = problem_.beta.data();
    gemm_workspace_.arguments.pointer_mode = library::ScalarPointerMode::kHost;

    results_.back().status = profile_cutlass_(
      results_.back().runtime,
      options,
      operation,
      &gemm_workspace_.arguments,
      gemm_workspace_.host_workspace.data(),
      gemm_workspace_.device_workspace.data()
    );
  }
  return true;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace profiler
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////
