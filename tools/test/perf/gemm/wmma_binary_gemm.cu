/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "tools/test/perf/cutlass_perf_test.h"
#include "tools/test/perf/gemm/gemm_profiler.h"
#include "tools/test/perf/gemm/gemm_perf_testbed.h"

#include "cutlass/wmma_matrix.h"
#ifdef CUTLASS_USE_WMMA_API
#ifdef CUTLASS_USE_SUBBYTE_WMMA
#pragma warning( disable : 4503)
////////////////////////////////////////////////////////////////////////////////////////////////////

#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/wmma_gemm_traits.h"
#include "tools/test/perf/gemm/cutlass_dispatch.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Traits>
struct WmmaBinaryGemmDispatch {

  typedef cutlass::gemm::Gemm<Traits> Gemm;

  typedef typename Gemm::Params Params;

  /// Indicate warp-level GEMM
  static bool const kThreadMultiplyAdd = false;

  static bool const kRunCuBLAS = false;

  static cutlass::MatrixLayout::Kind const kLayoutA = Traits::kLayoutA;
  static cutlass::MatrixLayout::Kind const kLayoutB = Traits::kLayoutB;

  //
  // Data members
  //

  /// Params argument
  Params params;

  //
  // Methods
  //

  WmmaBinaryGemmDispatch() {}

  /// Initializes params object
  WmmaBinaryGemmDispatch(int m, int n, int k, int alpha,
                       cutlass::Vector<cutlass::bin1_t, 32> const* d_a, int lda,
                       cutlass::Vector<cutlass::bin1_t, 32> const* d_b, int ldb, int beta,
                       int const* d_c, int ldc, int* d_d, int ldd) {

    params.initialize(m, n, k * 32, alpha, d_a, lda, d_b, ldb, beta, d_c, ldc, d_d, ldd);
  }

  /// batched strided bmma
  WmmaBinaryGemmDispatch(int m, int n, int k, int alpha,
    cutlass::Vector<cutlass::bin1_t, 32> const* d_a, int lda, long long int batch_stride_a, 
    cutlass::Vector<cutlass::bin1_t, 32> const* d_b, int ldb, long long int batch_stride_b, int beta,
    int const* d_c, int ldc, long long int batch_stride_c, int* d_d, int ldd, long long int batch_stride_d,
    int batch_count) {
    assert(0);
  }

  /// Initializes params object
  WmmaBinaryGemmDispatch(Params const& _params) : params(_params) {}

  /// Launches kernel
  hipError_t operator()() { return Gemm::launch(params); }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace perf {

////////////////////////////////////////////////////////////////////////////////////////////////////

int profile_wmma_binary_gemm(TestbenchOutput<GemmProblem> &output, TestbenchOptions const &options, Config const &config) {
  typedef perf::GemmProfiler<cutlass::Vector<cutlass::bin1_t, 32>, cutlass::Vector<cutlass::bin1_t, 32>, int, int, int> GemmProfiler;

  int results = 0;

  // compute capability check
  if (!options.compute_capability_exact(7, 5)) {
    return 0;
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<1024, 128, 128>,
                                          cutlass::Vector<cutlass::bin1_t, 32>,
                                          cutlass::Vector<cutlass::bin1_t, 32>,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<1024, 32, 64>,
                                          cutlass::Shape<128, 8, 8>,
                                          128,
                                          128>
      WmmaGemmTraits;

    typedef WmmaBinaryGemmDispatch<WmmaGemmTraits> Dispatch;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_binary_gemm_tn", options, config);
  }

  return results;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace perf

////////////////////////////////////////////////////////////////////////////////////////////////////

#else // ! CUTLASS_USE_SUBBYTE_WMMA

namespace perf {

int profile_wmma_binary_gemm(TestbenchOutput<GemmProblem> &output, TestbenchOptions const &options, Config const &config) {
  return 0;
}

} // namespace perf

#endif

struct WmmaBinaryGemmRegistrar {
  WmmaBinaryGemmRegistrar() { perf::RegisterGemmProfileFunc(perf::profile_wmma_binary_gemm); }
};

volatile WmmaBinaryGemmRegistrar _WmmaBinaryGemmRegistrar;

#endif // CUTLASS_USE_WMMA_API
