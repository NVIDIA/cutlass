/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "tools/test/perf/cutlass_perf_test.h"
#include "tools/test/perf/gemm/gemm_perf_testbed.h"
#include "tools/test/perf/gemm/gemm_profiler.h"

#include "cutlass/wmma_matrix.h"
#ifdef CUTLASS_USE_WMMA_API
#ifdef CUTLASS_USE_INT_WMMA
#pragma warning( disable : 4503)
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/wmma_gemm_traits.h"
#include "tools/test/perf/gemm/cutlass_dispatch.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Traits, typename ScalarA, typename ScalarB>
struct WmmaIntegerGemmDispatch {

  typedef cutlass::gemm::Gemm<Traits> Gemm;

  typedef typename Gemm::Params Params;

  /// Indicate warp-level GEMM
  static bool const kThreadMultiplyAdd = false;

  static bool const kRunCuBLAS = false;

  static cutlass::MatrixLayout::Kind const kLayoutA = Traits::kLayoutA;
  static cutlass::MatrixLayout::Kind const kLayoutB = Traits::kLayoutB;

  //
  // Data members
  //

  /// Params argument
  Params params;

  //
  // Methods
  //

  WmmaIntegerGemmDispatch() {}

  /// Initializes params object
  WmmaIntegerGemmDispatch(int m, int n, int k, int alpha,
                       ScalarA const* d_a, int lda,
                       ScalarB const* d_b, int ldb, int beta,
                       int const* d_c, int ldc, int* d_d, int ldd) {

    params.initialize(m, n, k, alpha, d_a, lda, d_b, ldb, beta, d_c, ldc, d_d, ldd);
  }

  ///
  WmmaIntegerGemmDispatch(int m, int n, int k, int alpha, 
    ScalarA const* d_a, int lda, long long int batch_stride_a,
    ScalarB const* d_b, int ldb, long long int batch_stride_b, int beta,
    int const* d_c, int ldc, long long int batch_stride_c, int* d_d, int ldd, long long int batch_stride_d,
    int batch_count) {
    assert(0);
  }

  /// Initializes params object
  WmmaIntegerGemmDispatch(Params const& _params) : params(_params) {}

  /// Launches kernel
  hipError_t operator()() { return Gemm::launch(params); }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

#ifdef CUTLASS_USE_SUBBYTE_WMMA
template<typename Traits>
struct WmmaIntegerGemmDispatch<Traits,
                               cutlass::Vector<cutlass::int4_t, 8>,
                               cutlass::Vector<cutlass::int4_t, 8> > {

  typedef typename cutlass::Vector<cutlass::int4_t, 8> ScalarA;
  typedef typename cutlass::Vector<cutlass::int4_t, 8> ScalarB;

  typedef cutlass::gemm::Gemm<Traits> Gemm;

  typedef typename Gemm::Params Params;

  /// Indicate warp-level GEMM
  static bool const kThreadMultiplyAdd = false;

  static bool const kRunCuBLAS = false;

  static cutlass::MatrixLayout::Kind const kLayoutA = Traits::kLayoutA;
  static cutlass::MatrixLayout::Kind const kLayoutB = Traits::kLayoutB;

  //
  // Data members
  //

  /// Params argument
  Params params;

  //
  // Methods
  //

  WmmaIntegerGemmDispatch() {}

  /// Initializes params object
  WmmaIntegerGemmDispatch(int m, int n, int k, int alpha,
                       ScalarA const* d_a, int lda,
                       ScalarB const* d_b, int ldb, int beta,
                       int const* d_c, int ldc, int* d_d, int ldd) {

    params.initialize(m, n, k * 8, alpha, d_a, lda, d_b, ldb, beta, d_c, ldc, d_d, ldd);
  }

  ///
  WmmaIntegerGemmDispatch(int m, int n, int k, int alpha,
    ScalarA const* d_a, int lda, long long int batch_stride_a,
    ScalarB const* d_b, int ldb, long long int batch_stride_b, int beta,
    int const* d_c, int ldc, long long int batch_stride_c, int* d_d, int ldd, long long int batch_stride_d,
    int batch_count) {
    assert(0);
  }

  /// Initializes params object
  WmmaIntegerGemmDispatch(Params const& _params) : params(_params) {}

  /// Launches kernel
  hipError_t operator()() { return Gemm::launch(params); }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template<typename Traits>
struct WmmaIntegerGemmDispatch<Traits,
                               cutlass::Vector<cutlass::uint4_t, 8>,
                               cutlass::Vector<cutlass::uint4_t, 8> > {

  typedef typename cutlass::Vector<cutlass::uint4_t, 8> ScalarA;
  typedef typename cutlass::Vector<cutlass::uint4_t, 8> ScalarB;

  typedef cutlass::gemm::Gemm<Traits> Gemm;

  typedef typename Gemm::Params Params;

  /// Indicate warp-level GEMM
  static bool const kThreadMultiplyAdd = false;

  static bool const kRunCuBLAS = false;

  static cutlass::MatrixLayout::Kind const kLayoutA = Traits::kLayoutA;
  static cutlass::MatrixLayout::Kind const kLayoutB = Traits::kLayoutB;

  //
  // Data members
  //

  /// Params argument
  Params params;

  //
  // Methods
  //

  WmmaIntegerGemmDispatch() {}

  /// Initializes params object
  WmmaIntegerGemmDispatch(int m, int n, int k, int alpha,
                       ScalarA const* d_a, int lda,
                       ScalarB const* d_b, int ldb, int beta,
                       int const* d_c, int ldc, int* d_d, int ldd) {

    params.initialize(m, n, k * 8, alpha, d_a, lda, d_b, ldb, beta, d_c, ldc, d_d, ldd);
  }

  ///
  WmmaIntegerGemmDispatch(int m, int n, int k, int alpha,
    ScalarA const* d_a, int lda, long long int batch_stride_a,
    ScalarB const* d_b, int ldb, long long int batch_stride_b, int beta,
    int const* d_c, int ldc, long long int batch_stride_c, int* d_d, int ldd, long long int batch_stride_d,
    int batch_count) {
    assert(0);
  }

  /// Initializes params object
  WmmaIntegerGemmDispatch(Params const& _params) : params(_params) {}

  /// Launches kernel
  hipError_t operator()() { return Gemm::launch(params); }
};
#endif //ifdef CUTLASS_USE_SUBBYTE_WMMA

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace perf {

////////////////////////////////////////////////////////////////////////////////////////////////////

int profile_wmma_integer_gemm(TestbenchOutput<GemmProblem> &output, TestbenchOptions const &options, Config const &config) {

  int results = 0;

  // compute capability check
  if (!options.compute_capability(7, 2)) {
    return 0;
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          signed char,
                                          signed char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, signed char, signed char> Dispatch;

    typedef perf::GemmProfiler<signed char, signed char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_s8_16x16x16_nn", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::MatrixLayout::kRowMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          signed char,
                                          signed char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, signed char, signed char> Dispatch;

    typedef perf::GemmProfiler<signed char, signed char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_s8_16x16x16_nt", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          signed char,
                                          signed char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, signed char, signed char> Dispatch;

    typedef perf::GemmProfiler<signed char, signed char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_s8_16x16x16_tn", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kRowMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          signed char,
                                          signed char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, signed char, signed char> Dispatch;

    typedef perf::GemmProfiler<signed char, signed char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_s8_16x16x16_tt", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          unsigned char,
                                          unsigned char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, unsigned char, unsigned char> Dispatch;

    typedef perf::GemmProfiler<unsigned char, unsigned char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_u8_16x16x16_nn", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::MatrixLayout::kRowMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          unsigned char,
                                          unsigned char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, unsigned char, unsigned char> Dispatch;

    typedef perf::GemmProfiler<unsigned char, unsigned char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_u8_16x16x16_nt", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          unsigned char,
                                          unsigned char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, unsigned char, unsigned char> Dispatch;

    typedef perf::GemmProfiler<unsigned char, unsigned char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_u8_16x16x16_tn", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kRowMajor,
                                          cutlass::Shape<128, 128, 128>,
                                          unsigned char,
                                          unsigned char,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<128, 32, 32>,
                                          cutlass::Shape<16, 16, 16>,
                                          16,
                                          16> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits, unsigned char, unsigned char> Dispatch;

    typedef perf::GemmProfiler<unsigned char, unsigned char, int, int, int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_u8_16x16x16_tt", options, config);
  }

  // compute capability check
  if (!options.compute_capability_exact(7, 5)) {
    return 0;
  }

#ifdef CUTLASS_USE_SUBBYTE_WMMA
  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<256, 128, 128>,
                                          cutlass::Vector<cutlass::int4_t, 8>,
                                          cutlass::Vector<cutlass::int4_t, 8>,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<256, 32, 32>,
                                          cutlass::Shape<32, 8, 8>,
                                          32,
                                          32> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits,
                                 cutlass::Vector<cutlass::int4_t, 8>,
                                 cutlass::Vector<cutlass::int4_t, 8> > Dispatch;

    typedef perf::GemmProfiler<cutlass::Vector<cutlass::int4_t, 8>,
                               cutlass::Vector<cutlass::int4_t, 8>,
                               int,
                               int,
                               int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_s4_tn", options, config);
  }

  {
    typedef cutlass::gemm::WmmaGemmTraits<cutlass::MatrixLayout::kRowMajor,
                                          cutlass::MatrixLayout::kColumnMajor,
                                          cutlass::Shape<256, 128, 128>,
                                          cutlass::Vector<cutlass::uint4_t, 8>,
                                          cutlass::Vector<cutlass::uint4_t, 8>,
                                          int,
                                          cutlass::gemm::LinearScaling<int>,
                                          int,
                                          cutlass::Shape<256, 32, 32>,
                                          cutlass::Shape<32, 8, 8>,
                                          32,
                                          32> WmmaGemmTraits;

    typedef WmmaIntegerGemmDispatch<WmmaGemmTraits,
                                 cutlass::Vector<cutlass::uint4_t, 8>,
                                 cutlass::Vector<cutlass::uint4_t, 8> > Dispatch;

    typedef perf::GemmProfiler<cutlass::Vector<cutlass::uint4_t, 8>,
                               cutlass::Vector<cutlass::uint4_t, 8>,
                               int,
                               int,
                               int> GemmProfiler;

    results |= profile_gemm<Dispatch, GemmProfiler>(output, "wmma_integer_gemm_u4_tn", options, config);
  }
#endif //ifdef CUTLASS_USE_SUBBYTE_WMMA

  return results;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace perf

////////////////////////////////////////////////////////////////////////////////////////////////////

#else // ! CUTLASS_USE_INT_WMMA

namespace perf {

int profile_wmma_integer_gemm(TestbenchOutput<GemmProblem> &output, TestbenchOptions const &options, Config const &config) {
  return 0;
}

}

#endif

struct WmmaIntegerGemmRegistrar {
  WmmaIntegerGemmRegistrar() { perf::RegisterGemmProfileFunc(perf::profile_wmma_integer_gemm); }
};

volatile WmmaIntegerGemmRegistrar _WmmaIntegerGemmRegistrar;

#endif // ifdef CUTLASS_USE_WMMA_API
