#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file

  \brief Defines unit tests for HostTensor and HostMatrix.

  HostTensor is a utility class for allocating memory on the host and on the selected CUDA device
  and presenting a TensorView of this memory.

  HostMatrix is new in CUTLASS 1.1 that offers a matrix-like interface to a HostTensor with rank 2.
  Several examples are shown in this source file.
*/

#include "cutlass_unit_test.h"

#include "cutlass/matrix_traits.h"

#include "tools/util/tensor_view_io.h"
#include "tools/util/host_tensor.h"
#include "tools/util/host_matrix.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace test {

/// Kernel to compute a thread's unique coordinate within a CUDA kernel grid and write a value
/// using a CUTLASS TensorView.
template <typename TensorView>
__global__ void fill_sequential(TensorView view) {

  // Compute the thread's coordinate in the 2D CUDA kernel grid
  cutlass::Coord<2> coord = cutlass::make_Coord(
    blockIdx.x * blockDim.x + threadIdx.x,
    blockIdx.y * blockDim.y + threadIdx.y
  );

  // Write a value into the view
  if (view.contains(coord)) {
    view.at(coord) = coord[0] + view.size(0) * coord[1];
  }
}

} // namespace test

////////////////////////////////////////////////////////////////////////////////////////////////////

// This test constructs a CUTLASS HostTensor  with column-major layout.
TEST(HostTensor, fill_sequential_column_major) {

  int const M = 16;
  int const N = 32;

  cutlass::Coord<2> bounds = cutlass::make_Coord(M, N);

  // Construct a rank=2 host tensor of size M-by-N with leading dimension M
  cutlass::HostTensor<
    int,
    2,
    cutlass::MatrixLayout::ColumnMajor> host_tensor(cutlass::make_Coord(M, 1), bounds);

  // Fill it with zeros and synchronize device
  host_tensor.fill(0);
  host_tensor.sync_device();

  // Launch a CUDA kernel by obtaining a TensorView of the device memory
  dim3 block(16, 16);
  dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

  test::fill_sequential<<< grid, block >>>(host_tensor.device_view());

  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

  // Synchronize the host data
  host_tensor.sync_host();

  // Verify host_tensor contains sequential elements
  int errors = 0;
  for (int n = 0; n < N; ++n) {
    for (int m = 0; m < M; ++m) {
      int expected = m + n * M;
      int got = host_tensor.at(cutlass::make_Coord(m, n));
      if (expected != got) {
        ++errors;
      }
    }
  }

  EXPECT_EQ(errors, 0) << std::setw(4) << host_tensor << std::endl;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// This test constructs a CUTLASS HostTensor with column-major interleaved layout
TEST(HostTensor, fill_sequential_column_major_interleaved) {

  int const M = 16;
  int const N = 16;
  int const kInterleave = 4;

  cutlass::Coord<2> bounds = cutlass::make_Coord(M, N);

  // Define a mapping function for column-major interleaved layout
  typedef cutlass::MatrixLayout::ColumnMajorInterleaved<kInterleave> TensorRefMapFunc;

  // Construct a rank=2 host tensor of size M-by-N
  cutlass::HostTensor<
    int,
    2,
    TensorRefMapFunc > host_tensor(TensorRefMapFunc::stride(M), bounds);

  // Fill it with zeros and synchronize device
  host_tensor.fill(0);
  host_tensor.sync_device();

  // Launch a CUDA kernel by obtaining a TensorView of the device memory
  dim3 block(16, 16);
  dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

  test::fill_sequential<<< grid, block >>>(host_tensor.device_view());

  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

  // Synchronize the host data
  host_tensor.sync_host();

  // Verify host_tensor contains sequential elements
  int errors = 0;
  for (int n = 0; n < N; ++n) {
    for (int m = 0; m < M; ++m) {
      int expected = m + n * M;
      int got = host_tensor.at(cutlass::make_Coord(m, n));
      if (got != expected) {
        ++errors;
      }
    }
  }

  EXPECT_EQ(errors, 0) << std::setw(4) << host_tensor << std::endl;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
// cutlass::HostMatrix extends cutlass::HostTensor of rank=2 to facilitate allocate and operating
// on matrices in device memory.
//
// cutlass::HostMatrix<T> accommodates both row-major and column-major matrices with a single
// leading dimension.
//
// The first test demonstrates use of HostMatrix<> in the same circumstances as HostTensor but with
// simplifcations to the calling interface.
//
////////////////////////////////////////////////////////////////////////////////////////////////////

// This test constructs a CUTLASS cutlass::HostMatrix  with column-major layout.
TEST(HostMatrix, fill_sequential_column_major) {

  int const M = 16;
  int const N = 32;
  int const ldm = M + 2; // define leading dimension with padding

  cutlass::Coord<2> bounds = cutlass::make_Coord(M, N);

  // Construct a HostMatrix of size M-by-N with leading dimension ldm
  cutlass::HostMatrix<int> host_matrix(bounds, cutlass::MatrixLayout::kColumnMajor, ldm);

  // Fill it with zeros and synchronize device
  host_matrix.fill(0);
  host_matrix.sync_device();

  // Launch a CUDA kernel by obtaining a TensorView of the device memory
  dim3 block(16, 16);
  dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

  test::fill_sequential<<< grid, block >>>(host_matrix.device_view());

  ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);

  // Synchronize the host data
  host_matrix.sync_host();

  // Verify host_matrix contains sequential elements
  int errors = 0;
  for (int n = 0; n < N; ++n) {
    for (int m = 0; m < M; ++m) {
      int expected = m + n * M;
      int got = host_matrix.at(cutlass::make_Coord(m, n));
      if (expected != got) {
        ++errors;
      }
    }
  }

  EXPECT_EQ(errors, 0) << std::setw(4) << host_matrix << std::endl;
}

////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Previously, cutlass::HostTensorView<> offered a gemm() method defined for the H and W dimensions.
// The other dimensions were ignored.
//
// To improve the interface, we We have moved this into the HostMatrixView<> and HostMatrix<>
// classes which require rank=2. To accommodate matrix operands of differing layout, we have extracted
// the host-side GEMM implementation into cutlass::reference::host::Gemm() which can compute the
// general matrix product of matrices with arbitrary layout.
//
////////////////////////////////////////////////////////////////////////////////////////////////////

// This test constructs a CUTLASS cutlass::HostMatrix  with column-major layout.
TEST(HostMatrix, gemm) {

  // Problem size intentionally small, as reference check has complexity O(MNK).
  int const M = 32;
  int const N = 16;
  int const K = 4;

  int const lda = M;
  int const ldb = N;
  int const ldc = M;

  // Construct matrix operands
  cutlass::HostMatrix<int> A(cutlass::make_Coord(M, K), cutlass::MatrixLayout::kColumnMajor, lda);
  cutlass::HostMatrix<int> B(cutlass::make_Coord(K, N), cutlass::MatrixLayout::kRowMajor, ldb);
  cutlass::HostMatrix<int> C(cutlass::make_Coord(M, N), cutlass::MatrixLayout::kColumnMajor, ldc);

  A.fill_sequential();
  B.fill_sequential();
  C.fill(0);

  int alpha = 1;

  // Compute host-side GEMM reference
  cutlass::reference::host::Gemm(
    cutlass::gemm::GemmCoord(K, N, M),
    alpha,
    A.host_ref(),
    B.host_ref(),
    int(0), // beta
    C.host_ref());

  // Verify result
  int errors = 0;

  // Primitive reference implementation for matrix product
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      int result = 0;
      for (int k = 0; k < K; ++k) {
        result += A.at(cutlass::make_Coord(i, k)) * B.at(cutlass::make_Coord(k, j));
      }
      if (C.at(cutlass::make_Coord(i, j)) != alpha * result) {
        ++errors;
      }
    }
  }

  EXPECT_EQ(errors, 0) << "GEMM error\n"
    << "A =\n" << A << "\nB = \n" << B << "\nC =\n" << C << "\n";
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// When layout is known at compile time, we may be use the corresponding helper classes to smplify
// matrix instantiation. The matrix layout becomes part of the type which reduces the StorageRank
// of the internal stride vector.
//
// Apart from specifying the matrix layout at compile time, this test is functionally identical to
// HostMatrix.gemm.
//
TEST(HostMatrix, gemm_compile_time_layout) {

  // Problem size intentionally small, as reference check has complexity O(MNK).
  int const M = 32;
  int const N = 16;
  int const K = 4;

  int const lda = M;
  int const ldb = N;
  int const ldc = M;

  // Construct matrix operands
  cutlass::HostMatrixColumnMajor<int> A(cutlass::make_Coord(M, K), lda);
  cutlass::HostMatrixRowMajor<int>    B(cutlass::make_Coord(K, N), ldb);
  cutlass::HostMatrixColumnMajor<int> C(cutlass::make_Coord(M, N), ldc);

  A.fill_sequential();
  B.fill_sequential();
  C.fill(0);

  int alpha = 1;

  // Compute host-side GEMM reference
  cutlass::reference::host::Gemm(
    cutlass::gemm::GemmCoord(K, N, M),
    alpha,
    A.host_ref(),
    B.host_ref(),
    int(0), // beta
    C.host_ref());

  // Verify result
  int errors = 0;

  // Primitive reference implementation for matrix product
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      int result = 0;
      for (int k = 0; k < K; ++k) {
        result += A.at(cutlass::make_Coord(i, k)) * B.at(cutlass::make_Coord(k, j));
      }
      if (C.at(cutlass::make_Coord(i, j)) != alpha * result) {
        ++errors;
      }
    }
  }

  EXPECT_EQ(errors, 0) << "GEMM error\n"
    << "A =\n" << A << "\nB = \n" << B << "\nC =\n" << C << "\n";
}

////////////////////////////////////////////////////////////////////////////////////////////////////
