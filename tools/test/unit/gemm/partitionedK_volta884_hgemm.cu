/***************************************************************************************************
* Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Redistribution and use in source and binary forms, with or without modification, are permitted
* provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright notice, this list of
*       conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright notice, this list of
*       conditions and the following disclaimer in the documentation and/or other materials
*       provided with the distribution.
*     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
*       to endorse or promote products derived from this software without specific prior written
*       permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
* IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
* FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
* BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
* OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
* STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*
**************************************************************************************************/
#include <hipblas.h>
#include <cstring>
#include "cutlass_unit_test.h"

#include "tools/util/half.h"
#include "tools/util/host_tensor.h"
#include "tools/util/tensor_view_io.h"

#include "cutlass/gemm/volta884_gemm_traits.h"
#include "cutlass/gemm/gemm.h"

#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

#if CUTLASS_ENABLE_TENSOR_CORE_MMA

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x88x10_nn) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 88, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 88;
  int partitionK_count = 10;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x88x10_nt) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 88, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 88;
  int partitionK_count = 10;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x88x10_tn) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 88, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 88;
  int partitionK_count = 10;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x88x10_tt) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 88, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 88;
  int partitionK_count = 10;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x128x10_nn) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 128, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 12. 
  But if we require the partition mulitple to be 8, the first 9 partition 
  k = k - (k % partition_mulitiple) = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 56
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 128;
  int partitionK_count = 10;
  int partitionK_multiple = 8;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count, partitionK_multiple);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x128x10_nt) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 128, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 12.
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 56
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 128;
  int partitionK_count = 10;
  int partitionK_multiple = 8;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count, partitionK_multiple);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x128x10_tn) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 128, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 12.
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 56
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 128;
  int partitionK_count = 10;
  int partitionK_multiple = 8;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count, partitionK_multiple);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_partitionedK_64x64x32, volta884_h884gemm_128x256x128x10_tt) {
  /*
  for example
  partitionedK gemm, m = 128, n = 256, overall_K = 128, partitionK_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 12.
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 56
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  int m = 128;
  int n = 256;
  int overall_k = 128;
  int partitionK_count = 10;
  int partitionK_multiple = 8;

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_partitioned_k_gemm<GemmTraits>(m, n, overall_k, partitionK_count, partitionK_multiple);
}

#endif // if defined(CUTLASS_ENABLE_TENSOR_CORE_MMA)
