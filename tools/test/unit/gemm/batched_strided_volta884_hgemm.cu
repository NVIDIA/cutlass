/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <hipblas.h>
#include <cstring>
#include "cutlass_unit_test.h"

#include "tools/util/half.h"
#include "tools/util/host_tensor.h"
#include "tools/util/tensor_view_io.h"

#include "cutlass/gemm/volta884_gemm_traits.h"
#include "cutlass/gemm/gemm.h"

#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

#if CUTLASS_ENABLE_TENSOR_CORE_MMA

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_batched_strided_64x64x32_nt, 64x128x64x3) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_batched_strided_gemm<GemmTraits>(64, 128, 64, 3);
}


////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_batched_strided_64x64x32_nn, 64x128x64x3) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_batched_strided_gemm<GemmTraits>(64, 128, 64, 3);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_batched_strided_64x64x32_tn, 64x128x64x3) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_batched_strided_gemm<GemmTraits>(64, 128, 64, 3);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_batched_strided_64x64x32_tt, 64x128x64x3) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_batched_strided_gemm<GemmTraits>(64, 128, 64, 3);
}

#endif // if defined(CUTLASS_ENABLE_TENSOR_CORE_MMA)
