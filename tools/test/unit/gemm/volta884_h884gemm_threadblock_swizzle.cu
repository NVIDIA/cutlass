/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <hipblas.h>
#include <cstring>
#include "cutlass_unit_test.h"

#include "tools/util/half.h"
#include "tools/util/host_tensor.h"
#include "tools/util/tensor_view_io.h"

#include "cutlass/gemm/volta884_gemm_traits.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/threadblock_swizzle.h"
#include "cutlass/gemm/linear_scaling.h"

#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

#if CUTLASS_ENABLE_TENSOR_CORE_MMA

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 530

////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Contiguous - h884gemm
//
////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_RowMajorSwizzle) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_RowMajorSwizzle_groupCol2) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_RowMajorSwizzle_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_RowMajorSwizzle_groupCol2_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_ColumnMajorSwizzle) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_ColumnMajorSwizzle_groupCol2) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_ColumnMajorSwizzle_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_nt_swizzle, 520x264x136_ColumnMajorSwizzle_groupCol2_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_RowMajorSwizzle) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_RowMajorSwizzle_groupCol2) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_RowMajorSwizzle_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_RowMajorSwizzle_groupCol2_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::RowMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_ColumnMajorSwizzle) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_ColumnMajorSwizzle_groupCol2) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::OneDirection>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_ColumnMajorSwizzle_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<1, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn_swizzle, 520x264x136_ColumnMajorSwizzle_groupCol2_Boustrophedon) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2,
    cutlass::gemm::LinearScaling<half>,
    typename cutlass::gemm::ColumnMajorBlockSwizzle<2, cutlass::gemm::swizzleDirection::Boustrophedon>
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

#endif // #if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 530

#endif // defined(CUTLASS_ENABLE_TENSOR_CORE_MMA)

