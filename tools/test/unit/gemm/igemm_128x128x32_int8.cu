/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#if (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 610))

#include "cutlass_unit_test.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/igemm_traits.h"
#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x4_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 4);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x32_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 32);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x36_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 36);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x64_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x256_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 256);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x128x64_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x256x64_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x256x64_nt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x4_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 4);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x32_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 32);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x36_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 36);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x64_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x256_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 256);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x128x64_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x256x64_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x256x64_nn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// NB: I have removed tests in which k=1. These result in the test environment definining matrices
//     in which ld{a,b} = 1 which cannot be launched by cuBLAS.
//
// This problem size remains untested. --akerr
//

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x4_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 4);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x32_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 32);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x36_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 36);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x64_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x256_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 256);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x128x64_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x256x64_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x256x64_tn) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x4_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kColumnMajor,
                                     cutlass::Shape<32, 128, 128> , int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 4);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x32_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 32);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x36_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 36);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x64_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x128x256_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 128, 256);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x128x64_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 128, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_128x256x64_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(128, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Igemm_128x128x32_int8, igemm_256x256x64_tt) {
  typedef cutlass::gemm::IgemmTraits<cutlass::MatrixLayout::kRowMajor,
                                     cutlass::MatrixLayout::kRowMajor,
                                     cutlass::Shape<32, 128, 128>, int8_t>
      IgemmTraits;
  run_gemm<IgemmTraits>(256, 256, 64);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#endif // if (!defined(__CUDA_ARCH__) || (__CUDA_ARCH__ >= 610))
