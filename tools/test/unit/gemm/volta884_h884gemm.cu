/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <hipblas.h>
#include <cstring>
#include "cutlass_unit_test.h"

#include "tools/util/half.h"
#include "tools/util/host_tensor.h"
#include "tools/util/tensor_view_io.h"

#include "cutlass/gemm/volta884_gemm_traits.h"
#include "cutlass/gemm/gemm.h"

#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

#if CUTLASS_ENABLE_TENSOR_CORE_MMA

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 530

////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Contiguous - h884gemm
//
////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_h884gemm_64x64x32_nt, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_128x64x32_nt, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 128>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x128x32_nt, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 128, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_128x128x32_nt, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 128, 128>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_256x128x32_nt, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 128, 256>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_128x256x32_nt, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 256, 128>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x64x32_tn, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_128x64x32_tn, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 128>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_64x128x32_tn, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 128, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_128x128x32_tn, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 128, 128>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_256x128x32_tn, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 128, 256>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

TEST(Volta884_h884gemm_128x256x32_tn, 520x264x136) {

  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 256, 128>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;

  run_gemm<GemmTraits>(520, 264, 136);
}

#endif // #if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 530

#endif // defined(CUTLASS_ENABLE_TENSOR_CORE_MMA)

