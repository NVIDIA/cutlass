/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

#include "cutlass_unit_test.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/linear_scaling_device_ptr.h"
#include "cutlass/gemm/sgemm_traits.h"

#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

////////////////////////////////////////////////////////////////////////////////////////////////////

// This example defines an SGEMM with a linear scaling functor that supports optionally passing
// alpha and beta via device-side pointers as in cuBLAS.
TEST(Sgemm_epilogue_functor, device_ptr_mode_sgemm_1024x512x128_nt) {

  typedef cutlass::gemm::SgemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<8, 128, 128>,
    cutlass::gemm::LinearScalingDevicePtr<float>
  >
    SgemmTraits;

  // Define a GEMM problem size
  int const m = 1025;
  int const n = 512;
  int const k = 128;

  // Define scalars
  float alpha_host = 3;
  float beta_host = 2;

  // Define a device-backed tensor to contain the scalars
  cutlass::HostTensor<float, 1> device_scalars(2);

  // Copy scalar values to device memory for device-ptr mode
  device_scalars.at(0) = alpha_host;
  device_scalars.at(1) = beta_host;
  device_scalars.sync_device();

  // Construct a GemmTestbed instance
  test::GemmTestbed<
    float,  // AType
    float,  // BType
    float,  // CType
    float,  // Accumulator
    float   // Scalar
    >
    testbed(m,
            n,
            k,
            test::convert(SgemmTraits::kLayoutA),
            test::convert(SgemmTraits::kLayoutB),
            alpha_host,
            beta_host);

  testbed.initialize();

  //
  // Construct a CUTLASS GEMM and initialize parameters
  //
  typedef cutlass::gemm::Gemm<SgemmTraits> Gemm;
  typename Gemm::Params params;

  params.initialize(testbed.M(),
                    testbed.N(),
                    testbed.K(),
                    0,                // alpha ignored
                    testbed.ptr_A(),
                    testbed.lda(),
                    testbed.ptr_B(),
                    testbed.ldb(),
                    0,                // beta ignored
                    testbed.ptr_C_initial(),
                    testbed.ldc(),
                    testbed.ptr_computed(),
                    testbed.ldc());

  // Explicitly call the epilogue functor's initialize method to pass additional arguments
  params.epilogue.functor.initialize(
    device_scalars.device_data() + 0,   // pointer to alpha in device memory
    device_scalars.device_data() + 1);  // pointer to beta in device memory

  // Launch the CUTLASS SGEMM kernel
  Gemm::launch(params);

  // Report any errors
  hipError_t result = hipDeviceSynchronize();
  ASSERT_EQ(result, hipSuccess)
    << "\nCUDA kernel launch error: " << hipGetErrorString(result)
    << "\n";

  // Verify result
  ASSERT_TRUE(testbed.verify_with_cublas());
}

////////////////////////////////////////////////////////////////////////////////////////////////////
