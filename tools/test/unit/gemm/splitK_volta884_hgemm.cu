/***************************************************************************************************
* Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
*
* Redistribution and use in source and binary forms, with or without modification, are permitted
* provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright notice, this list of
*       conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright notice, this list of
*       conditions and the following disclaimer in the documentation and/or other materials
*       provided with the distribution.
*     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
*       to endorse or promote products derived from this software without specific prior written
*       permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
* IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
* FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
* FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
* BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
* OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
* STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*
**************************************************************************************************/
#include <hipblas.h>
#include <cstring>
#include "cutlass_unit_test.h"

#include "tools/util/half.h"
#include "tools/util/host_tensor.h"
#include "tools/util/tensor_view_io.h"

#include "cutlass/gemm/volta884_gemm_traits.h"
#include "cutlass/gemm/gemm.h"

#include "cutlass/reduction/batched_reduction_traits.h"

#include "tools/test/unit/gemm/gemm_testbed.h"
#include "tools/test/unit/gemm/run_gemm.h"

#if CUTLASS_ENABLE_TENSOR_CORE_MMA


////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits16, volta884_h884gemm_128x256x512_nn) {
  const int splits_count = 16;
  const int m = 128;
  const int n = 256;
  const int k = 512;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits16, volta884_h884gemm_128x256x512_nt) {
  const int splits_count = 16;
  const int m = 128;
  const int n = 256;
  const int k = 512;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits16, volta884_h884gemm_128x256x512_tn) {
  const int splits_count = 16;
  const int m = 128;
  const int n = 256;
  const int k = 512;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits16, volta884_h884gemm_128x256x512_tt) {
  const int splits_count = 16;
  const int m = 128;
  const int n = 256;
  const int k = 512;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x88_nn) {
  /*
  m = 128, n = 256, overall_K = 88, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */

  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 88;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x88_nt) {
  /*
  m = 128, n = 256, overall_K = 88, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 88;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x88_tn) {
  /*
  m = 128, n = 256, overall_K = 88, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 88;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x88_tt) {
  /*
  m = 128, n = 256, overall_K = 88, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 8
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 16
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 88;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}
////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x256_nn) {
  /*
  m = 128, n = 256, overall_K = 256, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 25
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 24
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 40
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 256;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x256_nt) {
  /*
  m = 128, n = 256, overall_K = 256, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 25
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 24
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 40
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 256;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x256_tn) {
  /*
  m = 128, n = 256, overall_K = 256, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 25
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 24
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 40
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 256;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kColumnMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}
////////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Volta884_splitK_h884gemm_64x64x32_splits10, volta884_h884gemm_128x256x256_tt) {
  /*
  m = 128, n = 256, overall_K = 256, splits_count = 10
  for the first 9 partition k = overall_k / partitionK_count = 25
  But if we require the partition mulitple to be 8, the first 9 partition
  k = k - (k % partition_mulitiple) = 24
  for the last partition last_k = overall_k - (partitionK_count - 1) * k = 40
  for volta884 it is safe to make sure leading dim are multiple of 8
  */
  const int splits_count = 10;
  const int m = 128;
  const int n = 256;
  const int k = 256;

  /*gemm traits*/
  typedef cutlass::gemm::Volta884GemmTraits<
    cutlass::MatrixLayout::kRowMajor,
    cutlass::MatrixLayout::kRowMajor,
    cutlass::Shape<32, 64, 64>,
    cutlass::Shape<32, 64, 64>,
    half,
    half,
    half,
    2
  > GemmTraits;
  /*batched reduction traits*/
  typedef cutlass::reduction::BatchedReductionTraits<half,
    half,
    half,
    half,
    half, /*accumulation type*/
    splits_count,
    cutlass::Shape<1, 1, 128>,
    cutlass::Shape<1, 1, 64>,
    cutlass::Shape<1, 1, 2> >
    BatchedReductionTraits;

  run_splitK_gemm<GemmTraits, BatchedReductionTraits>(m, n, k, 8/*partitionK_multiple*/, 1.0f, 0.0f);
}

#endif
