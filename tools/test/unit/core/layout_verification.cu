#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include "cutlass_unit_test.h"
#include <algorithm>
#include "tools/test/unit/core/layout_verification.h"


namespace test {

Layout::Layout() {

}

Layout::Layout(Layout::SpanVector const &_layout) {
    reset(_layout);
}

struct SpanCompareDim {
    bool operator()(Layout::Span const &a, Layout::Span const &b) const {
        return a.dim < b.dim;
    }
};

/// Updates the layout
void Layout::reset(Layout::SpanVector const &_layout) {
    layout_ = _layout;

    extent_.clear();
    extent_.resize(layout_.size(), 1);

    int _rank = std::max_element(layout_.begin(), layout_.end(), SpanCompareDim())->dim + 1;

    dim_extent_.clear();
    dim_extent_.resize(_rank, extent_);

    // initialize extent vector
    for (size_t i = layout_.size(); i > 0; --i) {
        extent_.at(i - 1) = layout_.at(i - 1).size * (i < layout_.size() ? extent_.at(i) : 1);
    }

    // initialize the dim_extent vector
    for (size_t rank_idx = 0; rank_idx < dim_extent_.size(); ++rank_idx) {
        ExtentVector &_extent = dim_extent_.at(rank_idx);
        for (size_t i = layout_.size(); i > 0; --i) {
            int _size = (rank_idx == layout_.at(i - 1).dim ? layout_.at(i - 1).size : 1);
            _extent.at(i - 1) = _size * (i < layout_.size() ? _extent.at(i) : 1);
        }
    }
}

/// Computes the rank of the layout
int Layout::rank() const {
    return int(dim_extent_.size());
}

/// Prints a layout
std::ostream & Layout::write(std::ostream &out) const {
    std::cout << "Layout: [";
    for (size_t i = 0; i < layout_.size(); ++i) {
        std::cout << "(" << layout_.at(i).dim << ": " << layout_.at(i).size << ") ";
    }
    std::cout << "] - rank: " << rank() << "\n";

    std::cout << "Extent: [";
    for (size_t i = 0; i < layout_.size(); ++i) {
        std::cout << (i ? ", " : "") << extent_.at(i);
    }
    std::cout << "]\n";
    for (size_t r = 0; r < dim_extent_.size(); ++r) {
        std::cout << " Dim " << r << ": [";
        for (int i = 0; i < dim_extent_.at(r).size(); ++i) {
            std::cout << (i ? ", " : "") << dim_extent_.at(r).at(i);
        }
        std::cout << "]\n";
    }
    return out;
}

/// Maps an index to a given coordinate
Layout::Coordinate Layout::operator()(int index) const {

    Coordinate coord(rank(), 0);

    for (size_t i = 0; i < layout_.size() - 1; ++i) {

        int quotient = (index / extent_.at(i + 1));
        index = (index % extent_.at(i + 1));

        coord.at(layout_.at(i).dim) += quotient * dim_extent_.at(layout_.at(i).dim).at(i + 1);
    }

    coord.at(layout_.back().dim) += index;

    return coord;
}

/// Maps a coordinate to an index
int Layout::operator()(Layout::Coordinate const &_coord) const {

    Coordinate coord(_coord);
    int index = 0;

    for (size_t i = layout_.size(); i > 0; --i) {
        size_t idx = i - 1;

        int dim = layout_.at(idx).dim;
        int size = layout_.at(idx).size;

        int items = coord.at(dim);

        int quotient = items / size;
        int remainder = items % size;

        index += remainder * (i < layout_.size() ? extent_.at(idx + 1) : 1);
        coord.at(dim) = quotient;
    }

    return index;
}

}

std::ostream & operator<<(std::ostream &out, test::Layout::Coordinate const &coord) {
    for (int i = 0; i < coord.size(); ++i) {
        out << (i ? ", " : "") << coord.at(i);
    }
    return out;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Layout, igemm) {

    test::Layout::SpanVector layout_def;
    typedef test::Layout::Span Span;

    layout_def.push_back(Span(0, 8));
    layout_def.push_back(Span(1, 4));
    layout_def.push_back(Span(0, 4));

    test::Layout layout(layout_def);

    for (int i = 0; i < 33; ++i) {
        test::Layout::Coordinate coord = layout(i);
        int index = layout(coord);
        EXPECT_EQ(i, index)
            << "[" << i << "] - (" << layout(i) << ") => " << layout(layout(i)) << std::endl;
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

TEST(Layout, sgemm_accum) {

    test::Layout::SpanVector layout_def;
    typedef test::Layout::Span Span;

    layout_def.push_back(Span(0, 2));
    layout_def.push_back(Span(1, 8));
    layout_def.push_back(Span(0, 2));

    test::Layout layout(layout_def);

    for (int i = 0; i < 32; ++i) {
        test::Layout::Coordinate coord = layout(i);
        int index = layout(coord);
        EXPECT_EQ(i, index)
            << "[" << i << "] - (" << layout(i) << ") => " << layout(layout(i)) << std::endl;
    }
}

///////////////////////////////////////////////////////////////////////////////////////////////////
