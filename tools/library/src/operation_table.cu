#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*
  \file
  \brief Defines a data structure in which a set of functionally equivalent library::Operation
        instances may be queried.
*/

#include "cutlass/library/operation_table.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

/////////////////////////////////////////////////////////////////////////////////////////////////

void OperationTable::append(Manifest const &manifest) {

  // Insert operations into appropriate data structure
  for (auto const & operation : manifest) {

    OperationDescription const &desc = operation->description();

    // insert all gemm operation into operation table
    if (desc.kind == OperationKind::kGemm) {
      GemmDescription const &gemm_desc = static_cast<GemmDescription const &>(desc);
    

      GemmFunctionalKey functional_key(
        gemm_desc.provider,
        gemm_desc.gemm_kind,
        gemm_desc.tile_description.math_instruction.element_accumulator,
        gemm_desc.element_epilogue,
        gemm_desc.A.element,
        gemm_desc.A.layout,
        gemm_desc.transform_A,
        gemm_desc.B.element,
        gemm_desc.B.layout,
        gemm_desc.transform_B,
        gemm_desc.C.element
      );

      Operation const *op = operation.get();

      int cc = gemm_desc.tile_description.minimum_compute_capability;
        
      int alignment = std::max(std::max(
        gemm_desc.A.alignment, gemm_desc.B.alignment), gemm_desc.C.alignment);

      GemmPreferenceKey preference_key(cc, alignment);

      gemm_operations[functional_key][preference_key].push_back(op);
    }


  }

}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace library
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////

