#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*
  \file
  \brief Defines a data structure in which a set of functionally equivalent library::Operation
        instances may be queried.
*/

#include <fstream>

#include "cutlass/library/library.h"
#include "cutlass/library/operation_table.h"
#include "cutlass/library/util.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

std::ostream & operator<<(std::ostream &out, cutlass::library::GemmFunctionalKey const &k) {

  out << "{\n"
    << "  element_compute: " << to_string(k.element_compute) << "\n"
    << "   element_scalar: " << to_string(k.element_scalar) << "\n"
    << "        element_A: " << to_string(k.element_A) << "\n"
    << "         layout_A: " << to_string(k.layout_A) << "\n"
    << "      transform_A: " << to_string(k.transform_A) << "\n"
    << "        element_B: " << to_string(k.element_B) << "\n"
    << "         layout_B: " << to_string(k.layout_B) << "\n"
    << "      transform_B: " << to_string(k.transform_B) << "\n"
    << "        element_C: " << to_string(k.element_C) << "\n"
    << "}";

  return out;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

/////////////////////////////////////////////////////////////////////////////////////////////////

void OperationTable::append(Manifest const &manifest) {

  // Insert operations into appropriate data structure
  for (auto const & operation : manifest) {

    OperationDescription const &desc = operation->description();

    if (desc.kind == OperationKind::kGemm) {
      GemmDescription const &gemm_desc = static_cast<GemmDescription const &>(desc);
    
      if (gemm_desc.gemm_kind == GemmKind::kGemm) {

        GemmFunctionalKey functional_key(
          gemm_desc.tile_description.math_instruction.element_accumulator,
          gemm_desc.element_epilogue,
          gemm_desc.A.element,
          gemm_desc.A.layout,
          gemm_desc.transform_A,
          gemm_desc.B.element,
          gemm_desc.B.layout,
          gemm_desc.transform_B,
          gemm_desc.C.element
        );

        Operation const *op = operation.get();

        int cc = gemm_desc.tile_description.minimum_compute_capability;
        
        int alignment = std::max(std::max(
          gemm_desc.A.alignment, gemm_desc.B.alignment), gemm_desc.C.alignment);

        GemmPreferenceKey preference_key(cc, alignment);

        gemm_operations[functional_key][preference_key].push_back(op);
      }
      else if (gemm_desc.gemm_kind == GemmKind::kPlanarComplex) {

        GemmFunctionalKey functional_key(
          gemm_desc.tile_description.math_instruction.element_accumulator,
          gemm_desc.element_epilogue,
          gemm_desc.A.element,
          gemm_desc.A.layout,
          gemm_desc.transform_A,
          gemm_desc.B.element,
          gemm_desc.B.layout,
          gemm_desc.transform_B,
          gemm_desc.C.element
        );

        Operation const *op = operation.get();

        int cc = gemm_desc.tile_description.minimum_compute_capability;
        
        int alignment = std::max(std::max(
          gemm_desc.A.alignment, gemm_desc.B.alignment), gemm_desc.C.alignment);

        GemmPreferenceKey preference_key(cc, alignment);

        gemm_planar_complex_operations[functional_key][preference_key].push_back(op);
      }
      else if (gemm_desc.gemm_kind == GemmKind::kPlanarComplexArray) {

        GemmFunctionalKey functional_key(
          gemm_desc.tile_description.math_instruction.element_accumulator,
          gemm_desc.element_epilogue,
          gemm_desc.A.element,
          gemm_desc.A.layout,
          gemm_desc.transform_A,
          gemm_desc.B.element,
          gemm_desc.B.layout,
          gemm_desc.transform_B,
          gemm_desc.C.element
        );

        Operation const *op = operation.get();

        int cc = gemm_desc.tile_description.minimum_compute_capability;
        
        int alignment = std::max(std::max(
          gemm_desc.A.alignment, gemm_desc.B.alignment), gemm_desc.C.alignment);

        GemmPreferenceKey preference_key(cc, alignment);

        gemm_planar_complex_array_operations[functional_key][preference_key].push_back(op);
      }
    }
  }

}

/////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace library
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////

