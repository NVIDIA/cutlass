/***************************************************************************************************
 * Copyright (c) 2017 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief Instantiates GEMM reference implementations.
*/

#include "cutlass/cutlass.h"
#include "cutlass/library/library.h"
#include "cutlass/library/manifest.h"

#include "gemm_reference_operation.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

///////////////////////////////////////////////////////////////////////////////////////////////////

// A/B: u8
// Acc : s32
// C/D: some variance

// 1. u8_u8_s32_s32_s32 (s32 epi scalar)
// 2. u8_u8_s32_s32_s32 (f32 epi scalar)
// 3. u8_8_s32_s8_s8 (f32 epi scalar)
// 3. u8_8_s32_s8_s8 (s epi scalar)

void initialize_gemm_reference_operations_u8_u8_s32(Manifest &manifest) {
  // 1.
  make_gemm_real_canonical_layouts<
    uint8_t,                          // ElementA
    uint8_t,                          // ElementB
    int32_t,                          // ElementC
    int32_t,                          // ElementScalar / ElementCompute
    int32_t,                          // ElementAccumulator
    int32_t                           // ElementD
  >(manifest);

  // 2.
  make_gemm_real_canonical_layouts<
    uint8_t,                          // ElementA
    uint8_t,                          // ElementB
    int32_t,                          // ElementC
    float,                            // ElementScalar / ElementCompute
    int32_t,                          // ElementAccumulator
    int32_t,                          // ElementD
    NumericConverterClamp<int32_t, float> // From Scalar to D
  >(manifest);

  // 3.
  make_gemm_real_canonical_layouts<
    uint8_t,                          // ElementA
    uint8_t,                          // ElementB
    int8_t,                           // ElementC
    float,                            // ElementScalar / ElementCompute
    int32_t,                          // ElementAccumulator
    int8_t,                           // ElementD
    NumericConverterClamp<int8_t, float> // From Scalar to D
  >(manifest);

}

///////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace library
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////////////////////////

