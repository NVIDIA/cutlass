/***************************************************************************************************
 * Copyright (c) 2017 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/* \file
   \brief

*/

#include "cutlass/cutlass.h"
#include "cutlass/library/library.h"
#include "cutlass/library/manifest.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace library {

// note: init methods for the same op-class may be split into multiple to parallelize compilation
void initialize_gemm_reference_operations_int4(Manifest &manifest);
void initialize_gemm_reference_operations_int8_interleaved_32(Manifest &manifest);
void initialize_gemm_reference_operations_int8_interleaved_64(Manifest &manifest);
void initialize_gemm_reference_operations_int8_canonical(Manifest &manifest);
void initialize_gemm_reference_operations_e4m3a_e4m3out(Manifest &manifest);
void initialize_gemm_reference_operations_e5m2a_e4m3out(Manifest &manifest);
void initialize_gemm_reference_operations_e4m3a_e5m2out(Manifest &manifest);
void initialize_gemm_reference_operations_e5m2a_e5m2out(Manifest &manifest);
void initialize_gemm_reference_operations_fp8in_fp16out(Manifest &manifest);
void initialize_gemm_reference_operations_fp8in_bf16out(Manifest &manifest);
void initialize_gemm_reference_operations_fp8in_fp32out(Manifest &manifest);
void initialize_gemm_reference_operations_fp32out(Manifest &manifest);
void initialize_gemm_reference_operations_fp_other(Manifest &manifest);
void initialize_gemm_reference_operations_fp_mixed_input(Manifest &manifest);

void initialize_conv2d_reference_operations(Manifest &manifest);
void initialize_conv3d_reference_operations(Manifest &manifest);

///////////////////////////////////////////////////////////////////////////////////////////////////

void initialize_reference_operations(Manifest &manifest) {
  initialize_conv2d_reference_operations(manifest);
  initialize_conv3d_reference_operations(manifest);

  initialize_gemm_reference_operations_int4(manifest);

  initialize_gemm_reference_operations_int8_interleaved_32(manifest);
  initialize_gemm_reference_operations_int8_interleaved_64(manifest);
  initialize_gemm_reference_operations_int8_canonical(manifest);

  initialize_gemm_reference_operations_e4m3a_e4m3out(manifest);
  initialize_gemm_reference_operations_e5m2a_e4m3out(manifest);
  initialize_gemm_reference_operations_e4m3a_e5m2out(manifest);
  initialize_gemm_reference_operations_e5m2a_e5m2out(manifest);
  initialize_gemm_reference_operations_fp8in_fp16out(manifest);
  initialize_gemm_reference_operations_fp8in_bf16out(manifest);
  initialize_gemm_reference_operations_fp8in_fp32out(manifest);

  initialize_gemm_reference_operations_fp32out(manifest);
  initialize_gemm_reference_operations_fp_other(manifest);
  initialize_gemm_reference_operations_fp_mixed_input(manifest);

}

///////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace library
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////////////////////////

