#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/*! \file
    \brief CUTLASS Library handle.
*/
#include <iostream> 
#include <stdexcept>
#include <cstdint>

#include "cutlass/library/handle.h"
#include "cutlass/library/singleton.h"
#include "cutlass/library/util.h"

namespace cutlass {
namespace library {

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Constructor
Handle::Handle(
  hipStream_t stream, 
  size_t workspace_size
):
  provider_(Provider::kCUTLASS), 
  stream_(stream), 
  workspace_(nullptr), 
  workspace_size_(0), 
  scalar_pointer_mode_(ScalarPointerMode::kHost), 
  last_operation_(nullptr) {

  int device_idx = -1;

  hipError_t error = hipGetDevice(&device_idx);
  if (error != hipSuccess) {
    throw std::runtime_error("hipGetDevice() failed");
  }

  error = hipGetDeviceProperties(&device_, device_idx);
  if (error != hipSuccess) {
    throw std::runtime_error("hipGetDeviceProperties() failed");
  }

  set_workspace_size(workspace_size);

  Singleton::get();
}

/// Destructor
Handle::~Handle() {
  if (workspace_) {

    if (workspace_) {
      hipFree(workspace_);
    }

    workspace_ = nullptr;
    workspace_size_ = 0;
  }
}

/// Move constructor
Handle::Handle(Handle && handle) {
  device_ = handle.device_;
  workspace_size_ = handle.workspace_size_;
  workspace_ = handle.workspace_;
  stream_ = handle.stream_;
  scalar_pointer_mode_ = handle.scalar_pointer_mode_;
  
  handle.workspace_ = nullptr;
  handle.workspace_size_ = 0;
}

/// Move assignment operator
Handle & Handle::operator=(Handle && handle) {

  provider_ = handle.provider_;
  device_ = handle.device_;
  workspace_size_ = handle.workspace_size_;
  workspace_ = handle.workspace_;
  stream_ = handle.stream_;
  scalar_pointer_mode_ = handle.scalar_pointer_mode_;

  handle.workspace_ = nullptr;
  handle.workspace_size_ = 0;

  return *this;
}

int Handle::compute_capability() const {
  return device_.major * 10 + device_.minor;
}

/// Sets the current CUDA stream
void Handle::set_stream(hipStream_t stream) {
  stream_ = stream;
}

/// Gets the current CUDA stream
hipStream_t Handle::get_stream() const {
  return stream_;
}

/// Gets the current provider
Provider Handle::get_provider() const {
  return provider_;
}

/// Sets the provider of operations
void Handle::set_provider(Provider provider) {
  provider_ = provider;
}

/// Gets the device workspace size
size_t Handle::get_workspace_size() const {
  return workspace_size_;
}

/// Gets a pointer to the device workspace allocation in Global Memory
void *Handle::get_workspace() const {
  return workspace_;
}

/// Sets the size of device workspace, invalidating previous calls to get_device_workspace()
void Handle::set_workspace_size(size_t bytes) {
  if (bytes != workspace_size_) {

    if (workspace_) {
      hipFree(workspace_);
    }
      
    workspace_ = nullptr;
    workspace_size_ = bytes;

    if (workspace_size_) {
  
      hipError_t error = hipMalloc((void **)&workspace_, workspace_size_);
  
      if (error != hipSuccess) {
        throw std::runtime_error("Failed to allocate workspace");
      }
    }
  }

  if (workspace_) {
    hipError_t error = hipMemset(workspace_, 0, workspace_size_);

    if (error != hipSuccess) {
      throw std::runtime_error("Failed to clear workspace");
    }
  }
}

/// Gets the scalar pointer mode
ScalarPointerMode Handle::get_scalar_pointer_mode() const {
  return scalar_pointer_mode_;
}

/// Sets the scalar pointer mode
void Handle::set_scalar_pointer_mode(ScalarPointerMode mode) {
  scalar_pointer_mode_ = mode;
}

/// Gets the last operation
Operation const *Handle::get_last_operation() const {
  return last_operation_;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Returns the maximum required alignment for each operator
static int maximum_alignment_requirement(GemmDescription const &desc) {
  return std::max(
    std::max(desc.A.alignment, desc.B.alignment), desc.C.alignment);
}

/// Returns the largest alignment (in units of elements) the problem satisfies, starting from a
/// given upper limit.
static int gemm_problem_alignment(
  int M,
  int N,
  int K,
  NumericTypeID element_A,
  void const *ptr_A,
  int lda,
  int64_t batch_stride_A,
  NumericTypeID element_B,
  void const *ptr_B,
  int ldb,
  int64_t batch_stride_B,
  NumericTypeID element_C,
  void const * ptr_C,
  int ldc,
  int64_t batch_stride_C,
  void const * ptr_D,
  int ldd,
  int64_t batch_stride_D,
  int max_alignment_in_bytes = 16
) {

  void const *pointers[] = {
    ptr_A, ptr_B, ptr_C, ptr_D
  };

  int64_t extents[] = {
    M, N, K, lda, ldb, ldc, ldd, batch_stride_A, batch_stride_B, batch_stride_C, batch_stride_D
  };

  NumericTypeID elements[] = {
    element_A, element_B, element_C
  };

  for (; max_alignment_in_bytes > 0; max_alignment_in_bytes /= 2) {
    
    bool satisfied = true;

    // Can pointers satisfy this?
    for (void const *ptr : pointers) {
      std::uintptr_t int_ptr = reinterpret_cast<std::uintptr_t>(ptr);

      if (int_ptr % max_alignment_in_bytes) {
        satisfied = false;
        break;
      }
    }

    if (!satisfied) {
      continue;
    }

    // Compute the maximum alignment based on element data types
    int max_element_alignment = 0;

    for (NumericTypeID type_id : elements) {
      int element_alignment = max_alignment_in_bytes * 8 / library::sizeof_bits(type_id); 
      max_element_alignment = std::max(max_element_alignment, element_alignment);
    }

    // Can the problem size and leading dimensions satisfy this?
    for (int64_t extent : extents) {
      if (extent % max_element_alignment) {
        satisfied = false;
        break;
      }
    }

    if (!satisfied) {
      continue;
    }

    // Yes
    return max_element_alignment;
  }

  // No alignment satisfies this problem
  return 0;
}

/// Find the best kernel in descending order of preference.
static Operation const * find_gemm_operation(
  GemmOperationFunctionalMap::const_iterator operators_it, 
  GemmPreferenceKey const preference_key) {

  auto cc_it = operators_it->second.upper_bound(preference_key);

  if (cc_it == operators_it->second.begin()) {
    return nullptr;
  }

  Operation const *operation = nullptr;

  // Search in descending order of compute capability
  do {
    --cc_it;

    // Search tile sizes in order, for now.
    for (auto const * op : cc_it->second) {

      GemmDescription const &desc = static_cast<GemmDescription const &>(op->description());

      int min_cc = desc.tile_description.minimum_compute_capability;
      int max_cc = desc.tile_description.maximum_compute_capability;

      int op_alignment = maximum_alignment_requirement(desc);

      if ((min_cc <= preference_key.compute_capability) &&
        (preference_key.compute_capability <= max_cc) &&
        (op_alignment <= preference_key.alignment)) {

        operation = op;
        break;
      }
    }
  } while (!operation && cc_it != operators_it->second.begin());

  return operation;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Executes a GEMM computation: D <= alpha * A*B + beta * C
Status Handle::gemm(

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix - ignored for real-valued matrices

  void const * ptr_A,                       /// Pointer to A matrix in Global Memory
  int lda,                                  /// Leading dimension of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix - ignored for real-valued matrices

  void const * ptr_B,                       /// Pointer to B matrix in Global Memory
  int ldb,                                  /// Leading dimension of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrices

  void const * ptr_C,                       /// Pointer to C matrix
  int ldc,                                  /// Leading dimension of C matrix

  void * ptr_D,                             /// Pointer to D matrix
  int ldd                                   /// Leading dimension of D matrix
) {
  
  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kGemm,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = gemm_problem_alignment(
    M, N, K, 
    element_A, ptr_A, lda, 0,
    element_B, ptr_B, ldb, 0,
    element_C, ptr_C, ldc, 0,
    ptr_D, ldd, 0, kMaximumAlignmentSize
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmConfiguration configuration{
    {M, N, K},
    lda,
    ldb,
    ldc,
    ldd,
    1
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmArguments arguments{
    ptr_A,
    ptr_B,
    ptr_C,
    ptr_D,
    alpha,
    beta,
    scalar_pointer_mode_
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Executes a GEMM computation: D <= alpha * A*B + beta * C.
//
// Supports batched-strided, batched array or split-K serial or split-K parallel.
//
Status Handle::gemm_universal(

  GemmUniversalMode mode,                   /// indicates the mode in which the kUniversal GEMM is launched

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix - ignored for real-valued matrices

  void const * ptr_A,                       /// Pointer to A matrix in Global Memory
  int lda,                                  /// Leading dimension of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix - ignored for real-valued matrices

  void const * ptr_B,                       /// Pointer to B matrix in Global Memory
  int ldb,                                  /// Leading dimension of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrices

  void const * ptr_C,                       /// Pointer to C matrix
  int ldc,                                  /// Leading dimension of C matrix

  void * ptr_D,                             /// Pointer to D matrix
  int ldd,                                  /// Leading dimension of D matrix

  int batch_count,                          /// Batch count or number of split-K slices

  int64_t batch_stride_A,                   /// Batch stride of A operand
  int64_t batch_stride_B,                   /// Batch stride of B operand
  int64_t batch_stride_C,                   /// Batch stride of C operand
  int64_t batch_stride_D                    /// Batch stride of D operand
) {
  
  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kUniversal,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  void const *ptr_A_check = ptr_A;
  void const *ptr_B_check = ptr_B;
  void const *ptr_C_check = ptr_C;
  void *      ptr_D_check = ptr_D;

  // Ignore alignment of pointers to pointers. We can't check this from the host,
  // as each batch index has its own pointer in device memory.
  if (mode == GemmUniversalMode::kArray) {
    ptr_A_check = nullptr; 
    ptr_B_check = nullptr; 
    ptr_C_check = nullptr; 
    ptr_D_check = nullptr; 
  }

  int alignment = gemm_problem_alignment(
    M, N, K, 
    element_A, ptr_A_check, lda, 0,
    element_B, ptr_B_check, ldb, 0,
    element_C, ptr_C_check, ldc, 0,
    ptr_D_check, ldd, 0, kMaximumAlignmentSize
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmUniversalConfiguration configuration{
    mode,
    {M, N, K},
    batch_count,
    lda,
    ldb,
    ldc,
    ldd
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmUniversalArguments arguments{
    ptr_A,
    ptr_B,
    ptr_C,
    ptr_D,
    alpha,
    beta,
    scalar_pointer_mode_,
    batch_stride_A,
    batch_stride_B,
    batch_stride_C,
    batch_stride_D
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Planar complex GEMM
Status Handle::gemm_planar_complex(

  int M,                                    /// GEMM M dimension
  int N,                                    /// GEMM N dimension
  int K,                                    /// GEMM K dimension

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix

  void const * ptr_A_real,                  /// Pointer to real part of A matrix
  void const * ptr_A_imag,                  /// Pointer to imaginary part of A matrix
  int lda_real,                             /// Leading dimension of real part of A matrix
  int lda_imag,                             /// Leading dimension of imaginary part of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix

  void const * ptr_B_real,                  /// Pointer to real part of B matrix
  void const * ptr_B_imag,                  /// Pointer to imaginary part of B matrix
  int ldb_real,                             /// Leading dimension of real part of B matrix
  int ldb_imag,                             /// Leading dimension of imaginary part of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrix

  void const * ptr_C_real,                  /// Pointer to real part of C matrix
  void const * ptr_C_imag,                  /// Pointer to imaginary part of C matrix
  int ldc_real,                             /// Leading dimension of real part of C matrix
  int ldc_imag,                             /// Leading dimension of imaginary part of C matrix

  void * ptr_D_real,                        /// Pointer to real part of D matrix
  void * ptr_D_imag,                        /// Pointer to imaginary part of D matrix
  int ldd_real,                             /// Leading dimension of real part of D matrix
  int ldd_imag,                             /// Leading dimension of imaginary part of D matrix

  int batch_count,                          /// Number of batched GEMMs to execute

  int64_t batch_stride_A_real,
  int64_t batch_stride_A_imag,

  int64_t batch_stride_B_real,
  int64_t batch_stride_B_imag,

  int64_t batch_stride_C_real,
  int64_t batch_stride_C_imag,

  int64_t batch_stride_D_real,
  int64_t batch_stride_D_imag
) {

  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kPlanarComplex,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = std::max(
    gemm_problem_alignment(
      M, N, K, 
      element_A, ptr_A_real, lda_real, batch_stride_A_real,
      element_B, ptr_B_real, ldb_real, batch_stride_B_real,
      element_C, ptr_C_real, ldc_real, batch_stride_C_real,
      ptr_D_real, ldd_real, batch_stride_D_real, kMaximumAlignmentSize
    ),
    gemm_problem_alignment(
      M, N, K, 
      element_A, ptr_A_imag, lda_imag, batch_stride_A_imag,
      element_B, ptr_B_imag, ldb_imag, batch_stride_B_imag,
      element_C, ptr_C_imag, ldc_imag, batch_stride_C_imag,
      ptr_D_imag, ldd_imag, batch_stride_D_imag, kMaximumAlignmentSize
    )
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmPlanarComplexConfiguration configuration{
    GemmUniversalMode::kBatched,
    {M, N, K},
    batch_count,
    lda_real,
    lda_imag,
    ldb_real,
    ldb_imag,
    ldc_real,
    ldc_imag,
    ldd_real,
    ldd_imag
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmPlanarComplexArguments arguments{
    ptr_A_real,
    ptr_A_imag,
    ptr_B_real,
    ptr_B_imag,
    ptr_C_real,
    ptr_C_imag,
    ptr_D_real,
    ptr_D_imag,
    alpha,
    beta,
    scalar_pointer_mode_,
    batch_stride_A_real,
    batch_stride_A_imag,
    batch_stride_B_real,
    batch_stride_B_imag,
    batch_stride_C_real,
    batch_stride_C_imag,
    batch_stride_D_real,
    batch_stride_D_imag
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Planar complex batched GEMM loading pointers from arrays in global memory
Status Handle::gemm_planar_complex_array(

  int expected_M,                           /// Expected GEMM M dimension (used for sizing CUDA grid)
  int expected_N,                           /// Expected GEMM N dimension (used for sizing CUDA grid)
  int expected_K,                           /// Expected GEMM K dimension
  int batch_count,                          /// Number of independent GEMM computations to execute

  int const *M,                             /// Array containing the GEMM M dimension for each batch index
  int const *N,                             /// Array containing the GEMM N dimension for each batch index
  int const *K,                             /// Array containing the GEMM K dimension for each batch index

  NumericTypeID element_compute,            /// Data type of internal accumulation

  NumericTypeID element_scalar,             /// Data type of alpha/beta scalars

  void const *alpha,                        /// Pointer to alpha scalar

  NumericTypeID element_A,                  /// Data type of A matrix elements
  LayoutTypeID layout_A,                    /// Layout of A matrix
  ComplexTransform transform_A,             /// Complex transformation applied to A matrix

  void const * const * ptr_A_real,          /// Pointer to array containing pointers to real part of A matrices
  void const * const * ptr_A_imag,          /// Pointer to array containing pointers to imaginary part of A matrices

  int lda_real,                             /// Leading dimension of real part of A matrix
  int lda_imag,                             /// Leading dimension of imaginary part of A matrix

  NumericTypeID element_B,                  /// Data type of B matrix elements
  LayoutTypeID layout_B,                    /// Layout of B matrix
  ComplexTransform transform_B,             /// Complex transformation applied to B matrix

  void const * const * ptr_B_real,          /// Pointer to array containing pointers to real part of B matrices
  void const * const * ptr_B_imag,          /// Pointer to array containing pointers to imaginary part of B matrices

  int ldb_real,                             /// Leading dimension of real part of B matrix
  int ldb_imag,                             /// Leading dimension of imaginary part of B matrix

  void const * beta,                        /// Pointer to beta scalar

  NumericTypeID element_C,                  /// Data type of C and D matrix

  void const * const * ptr_C_real,          /// Pointer to array containing pointers to real part of C matrices
  void const * const * ptr_C_imag,          /// Pointer to array containing poitners to imaginary part of C matrices

  int ldc_real,                             /// Leading dimension of real part of C matrix
  int ldc_imag,                             /// Leading dimension of imaginary part of C matrix

  void * const * ptr_D_real,                /// Pointer to array containing pointers to real part of D matrices
  void * const * ptr_D_imag,                /// Pointer to array containing poitners to imaginary part of D matrices

  int ldd_real,                             /// Leading dimension of real part of D matrix
  int ldd_imag                              /// Leading dimension of imaginary part of D matrix
) {
  
  //
  // Find the operation
  //

  GemmFunctionalKey key(
    provider_,
    GemmKind::kPlanarComplexArray,
    element_compute,
    element_scalar,
    element_A,
    layout_A,
    transform_A,
    element_B,
    layout_B,
    transform_B,
    element_C
  );

  auto operators_it = Singleton::get().operation_table.gemm_operations.find(key);

  if (operators_it == Singleton::get().operation_table.gemm_operations.end()) {
    return cutlass::Status::kErrorNotSupported;
  }
  
  if (operators_it->second.empty()) {
    return cutlass::Status::kErrorNotSupported;
  }

  //
  // Compute the largest alignment restriction the kernel can satisfy.
  //

  // Maximum alignment expectation among all kernels (in units of bytes)
  int const kMaximumAlignmentSize = 16;

  int alignment = std::max(
    gemm_problem_alignment(
      expected_M, expected_N, expected_K, 
      element_A, nullptr, lda_real, 0,
      element_B, nullptr, ldb_real, 0,
      element_C, nullptr, ldc_real, 0,
      nullptr, ldd_real, 0, kMaximumAlignmentSize
    ),
    gemm_problem_alignment(
      expected_M, expected_N, expected_K, 
      element_A, nullptr, lda_imag, 0,
      element_B, nullptr, ldb_imag, 0,
      element_C, nullptr, ldc_imag, 0,
      nullptr, ldd_imag, 0, kMaximumAlignmentSize
    )
  );

  //
  // Find the best kernel in descending order of preference.
  //

  GemmPreferenceKey preference_key(compute_capability(), alignment);

  Operation const *operation = find_gemm_operation(operators_it, preference_key);

  if (!operation) {
    return cutlass::Status::kErrorNotSupported;
  }

  last_operation_ = operation;

  //
  // Configure operation
  //

  GemmPlanarComplexArrayConfiguration configuration{
    {expected_M, expected_N, expected_K},
    batch_count,
    lda_real,
    lda_imag,
    ldb_real,
    ldb_imag,
    ldc_real,
    ldc_imag,
    ldd_real,
    ldd_imag
  };

  // Query host work space size
  uint64_t host_workspace_size_needed = operation->get_host_workspace_size(&configuration);

  if (uint64_t(kHostWorkspaceSize) < host_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  char host_workspace[kHostWorkspaceSize];

  // Query device workspace size
  uint64_t device_workspace_size_needed = operation->get_device_workspace_size(&configuration);

  if (uint64_t(workspace_size_) < device_workspace_size_needed) {
    return cutlass::Status::kErrorNotSupported;
  }

  // Initialize host and device workspaces
  Status status = operation->initialize(
    &configuration,
    host_workspace,
    workspace_,
    stream_);

  if (status != cutlass::Status::kSuccess) {
    return status;
  }

  // Run the operator
  GemmPlanarComplexArrayArguments arguments{
    M, N, K,
    ptr_A_real,
    ptr_A_imag,
    ptr_B_real,
    ptr_B_imag,
    ptr_C_real,
    ptr_C_imag,
    ptr_D_real,
    ptr_D_imag,
    alpha,
    beta,
    scalar_pointer_mode_
  };

  return operation->run(&arguments, host_workspace, workspace_, stream_);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
} // namespace library
} // namespace cutlass

/////////////////////////////////////////////////////////////////////////////////////////////////

